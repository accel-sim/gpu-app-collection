#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in DAC'10
  paper "An Effective GPU Implementation of Breadth-First Search"

  Copyright (c) 2010 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Author: Lijiuan Luo (lluo3@uiuc.edu)
  Revised for Parboil 2 Benchmark Suite by: Geng Daniel Liu (gengliu2@illinois.edu)
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <parboil.h>
#include <deque>
#include <iostream>

#include "config.h"
FILE *fp;

typedef int2 Node;
typedef int2 Edge;

#include "kernel.cu"
//Somehow "hipMemset" does not work. So I use hipMemcpy of constant variables for initialization
const int h_top = 1;
const int zero = 0;

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
  //the number of nodes in the graph
  int num_of_nodes = 0; 
  //the number of edges in the graph
  int num_of_edges = 0;
  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
  {
    fprintf(stderr, "Expecting one input filename\n");
    exit(-1);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  //printf("Reading File\n");
  //Read in Graph from a file
  fp = fopen(params->inpFiles[0],"r");
  if(!fp)
  {
    printf("Error Reading graph file\n");
    return 0;
  }
  int source;

  fscanf(fp,"%d",&num_of_nodes);
  // allocate host memory
  Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*num_of_nodes);
  int *color = (int*) malloc(sizeof(int)*num_of_nodes);
  int start, edgeno;   
  // initalize the memory
  for( unsigned int i = 0; i < num_of_nodes; i++) 
  {
    fscanf(fp,"%d %d",&start,&edgeno);
    h_graph_nodes[i].x = start;
    h_graph_nodes[i].y = edgeno;
    color[i]=WHITE;
  }
  //read the source node from the file
  fscanf(fp,"%d",&source);
  fscanf(fp,"%d",&num_of_edges);
  int id,cost;
  Edge* h_graph_edges = (Edge*) malloc(sizeof(Edge)*num_of_edges);
  for(int i=0; i < num_of_edges ; i++)
  {
    fscanf(fp,"%d",&id);
    fscanf(fp,"%d",&cost);
    h_graph_edges[i].x = id;
    h_graph_edges[i].y = cost;
  }
  if(fp)
    fclose(fp);    

  // allocate mem for the result on host side
  int* h_cost = (int*) malloc( sizeof(int)*num_of_nodes);
  for(int i = 0; i < num_of_nodes; i++){
    h_cost[i] = INF;
  }
  h_cost[source] = 0;

  pb_SwitchToTimer(&timers, pb_TimerID_COPY);

  //Copy the Node list to device memory
  Node* d_graph_nodes;
  hipMalloc((void**) &d_graph_nodes, sizeof(Node)*num_of_nodes);
  hipMemcpy(d_graph_nodes, h_graph_nodes, sizeof(Node)*num_of_nodes, hipMemcpyHostToDevice);
  //Copy the Edge List to device Memory
  Edge* d_graph_edges;
  hipMalloc((void**) &d_graph_edges, sizeof(Edge)*num_of_edges);
  hipMemcpy(d_graph_edges, h_graph_edges, sizeof(Edge)*num_of_edges, hipMemcpyHostToDevice);

  int* d_color;
  hipMalloc((void**) &d_color, sizeof(int)*num_of_nodes);
  int* d_cost;
  hipMalloc((void**) &d_cost, sizeof(int)*num_of_nodes);
  int * d_q1;
  int * d_q2;
  hipMalloc( (void**) &d_q1, sizeof(int)*num_of_nodes);
  hipMalloc( (void**) &d_q2, sizeof(int)*num_of_nodes);
  int * tail;
  hipMalloc( (void**) &tail, sizeof(int));
  int *front_cost_d;
  hipMalloc( (void**) &front_cost_d, sizeof(int));
  hipMemcpy( d_color, color, sizeof(int)*num_of_nodes, hipMemcpyHostToDevice);
  hipMemcpy( d_cost, h_cost, sizeof(int)*num_of_nodes, hipMemcpyHostToDevice);

  //bind the texture memory with global memory
  hipBindTexture(0,g_graph_node_ref,d_graph_nodes, sizeof(Node)*num_of_nodes);
  hipBindTexture(0,g_graph_edge_ref,d_graph_edges,sizeof(Edge)*num_of_edges);

  printf("Starting GPU kernel\n");
  (hipDeviceSynchronize());
  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
  
  int num_of_blocks; 
  int num_of_threads_per_block;

  hipMemcpy(tail,&h_top,sizeof(int),hipMemcpyHostToDevice);
  hipMemcpy(&d_cost[source],&zero,sizeof(int),hipMemcpyHostToDevice);

  hipMemcpy( &d_q1[0], &source, sizeof(int), hipMemcpyHostToDevice);
  int num_t;//number of threads
  int k=0;//BFS level index

  do
  {
    (hipMemcpy(&num_t, tail, sizeof(int), hipMemcpyDeviceToHost) );
    (hipMemcpy(tail,&zero,sizeof(int),hipMemcpyHostToDevice));

    if(num_t == 0){//frontier is empty
      break;
    }

    num_of_blocks = 1;
    num_of_threads_per_block = num_t;
    if(num_of_threads_per_block <NUM_BIN)
      num_of_threads_per_block = NUM_BIN;
    if(num_t>MAX_THREADS_PER_BLOCK)
    {
      num_of_blocks = (int)ceil(num_t/(double)MAX_THREADS_PER_BLOCK); 
      num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
    }
    if(num_of_blocks == 1)//will call "BFS_in_GPU_kernel" 
      num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
    if(num_of_blocks >1 && num_of_blocks <= NUM_SM)// will call "BFS_kernel_multi_blk_inGPU"
      num_of_blocks = NUM_SM;

    dim3  grid( num_of_blocks, 1, 1);
    dim3  threads( num_of_threads_per_block, 1, 1);

    if(k%2 == 0){
      BFS_kernel<<< grid, threads >>>(d_q1,d_q2, d_graph_nodes, 
          d_graph_edges, d_color, d_cost, num_t, tail,GRAY0,k);
    }
    else{
      BFS_kernel<<< grid, threads >>>(d_q2,d_q1, d_graph_nodes, 
          d_graph_edges, d_color, d_cost, num_t, tail, GRAY1,k);
    }
    k++;
  }
  while(1);
  hipDeviceSynchronize();
  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
  printf("GPU kernel done\n");

  // copy result from device to host
  hipMemcpy(h_cost, d_cost, sizeof(int)*num_of_nodes, hipMemcpyDeviceToHost);
  hipMemcpy(color, d_color, sizeof(int)*num_of_nodes, hipMemcpyDeviceToHost);
  hipUnbindTexture(g_graph_node_ref);
  hipUnbindTexture(g_graph_edge_ref);

  hipFree(d_graph_nodes);
  hipFree(d_graph_edges);
  hipFree(d_color);
  hipFree(d_cost);
  hipFree(tail);
  hipFree(front_cost_d);
  //Store the result into a file
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  FILE *fp = fopen(params->outFile,"w");
  fprintf(fp, "%d\n", num_of_nodes);
  for(int i=0;i<num_of_nodes;i++)
    fprintf(fp,"%d %d\n",i,h_cost[i]);
  fclose(fp);

  // cleanup memory
  free( h_graph_nodes);
  free( h_graph_edges);
  free( color);
  free( h_cost);
  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);
  return 0;
}
