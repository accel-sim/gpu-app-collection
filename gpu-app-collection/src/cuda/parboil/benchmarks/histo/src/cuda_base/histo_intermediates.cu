#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "util.h"

__device__ void calculateBin (
        const unsigned int bin,
        uchar4 *sm_mapping)
{
        unsigned char offset  =  bin        %   4;
        unsigned char indexlo = (bin >>  2) % 256;
        unsigned char indexhi = (bin >> 10) %  KB;
        unsigned char block   =  bin / BINS_PER_BLOCK;

        offset *= 8;

        uchar4 sm;
        sm.x = block;
        sm.y = indexhi;
        sm.z = indexlo;
        sm.w = offset;

        *sm_mapping = sm;
}

__global__ void histo_intermediates_kernel (
        uint2 *input,
        unsigned int height,
        unsigned int width,
        unsigned int input_pitch,
        uchar4 *sm_mappings)
{
        unsigned int line = UNROLL * blockIdx.x;// 16 is the unroll factor;

        uint2 *load_bin = input + line * input_pitch + threadIdx.x;

        unsigned int store = line * width + threadIdx.x;
        bool skip = (width % 2) && (threadIdx.x == (blockDim.x - 1));

        #pragma unroll
        for (int i = 0; i < UNROLL; i++)
        {
                uint2 bin_value = *load_bin;

                calculateBin (
                        bin_value.x,
                        &sm_mappings[store]
                );

                if (!skip) calculateBin (
                        bin_value.y,
                        &sm_mappings[store + blockDim.x]
                );

                load_bin += input_pitch;
                store += width;
        }
}
