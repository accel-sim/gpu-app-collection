
#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>

#include "file.h"
#include "gpu_info.h"
#include "spmv_jds.h"
#include "jds_kernels.cu"
#include "convert_dataset.h"

static int generate_vector(float *x_vector, int dim) 
{	
	srand(54321);	
	for(int i=0;i<dim;i++)
	{
		x_vector[i] = (rand() / (float) RAND_MAX);
	}
	return 0;
}

int main(int argc, char** argv) {
	struct pb_TimerSet timers;
	struct pb_Parameters *parameters;
	
	
	
	
	
	printf("CUDA accelerated sparse matrix vector multiplication****\n");
	printf("Original version by Li-Wen Chang <lchang20@illinois.edu> and Shengzhao Wu<wu14@illinois.edu>\n");
	printf("This version maintained by Chris Rodrigues  ***********\n");
	parameters = pb_ReadParameters(&argc, argv);
	if ((parameters->inpFiles[0] == NULL) || (parameters->inpFiles[1] == NULL))
    {
      fprintf(stderr, "Expecting one two filenames\n");
      exit(-1);
    }

	
	pb_InitializeTimerSet(&timers);
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	//parameters declaration
	int len;
	int depth;
	int dim;
	int pad=32;
	int nzcnt_len;
	
	//host memory allocation
	//matrix
	float *h_data;
	int *h_indices;
	int *h_ptr;
	int *h_perm;
	int *h_nzcnt;
	//vector
	float *h_Ax_vector;
  float *h_x_vector;
	
	//device memory allocation
	//matrix
	float *d_data;
	int *d_indices;
	int *d_ptr;
	int *d_perm;
	int *d_nzcnt;
	//vector
	float *d_Ax_vector;
    float *d_x_vector;
    

    //load matrix from files
	pb_SwitchToTimer(&timers, pb_TimerID_IO);

	//inputData(parameters->inpFiles[0], &len, &depth, &dim,&nzcnt_len,&pad,
	//    &h_data, &h_indices, &h_ptr,
	//    &h_perm, &h_nzcnt);

	// HACK: remove the .bin from the end of data, remove later
	int col_count;
	//parameters->inpFiles[0][strlen(parameters->inpFiles[0])-4] = 0x00;
	printf("Input file %s\n", parameters->inpFiles[0]);
	 coo_to_jds(
		parameters->inpFiles[0], // bcsstk32.mtx, fidapm05.mtx, jgl009.mtx
		1, // row padding
		pad, // warp size
		1, // pack size
		1, // is mirrored?
		0, // binary matrix
		1, // debug level [0:2]
		&h_data, &h_ptr, &h_nzcnt, &h_indices, &h_perm,
		&col_count, &dim, &len, &nzcnt_len, &depth
	);	

	int i;
	for (i=0; i<dim; i++) {
		//printf("%d = %d\n", h_perm[i], h_perm2[i]);
	}

  h_Ax_vector=(float*)malloc(sizeof(float)*dim);
  h_x_vector=(float*)malloc(sizeof(float)*dim);
  input_vec( parameters->inpFiles[1],h_x_vector,dim);
  
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  
	
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
	
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//memory allocation
	hipMalloc((void **)&d_data, len*sizeof(float));
	hipMalloc((void **)&d_indices, len*sizeof(int));
	hipMalloc((void **)&d_ptr, depth*sizeof(int));
	hipMalloc((void **)&d_perm, dim*sizeof(int));
	hipMalloc((void **)&d_nzcnt, nzcnt_len*sizeof(int));
	hipMalloc((void **)&d_x_vector, dim*sizeof(float));
	hipMalloc((void **)&d_Ax_vector,dim*sizeof(float));
	hipMemset( (void *) d_Ax_vector, 0, dim*sizeof(float));
	
	//memory copy
	hipMemcpy(d_data, h_data, len*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_indices, h_indices, len*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_perm, h_perm, dim*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_x_vector, h_x_vector, dim*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(jds_ptr_int), h_ptr, depth*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(sh_zcnt_int), h_nzcnt,nzcnt_len*sizeof(int));
	
	hipDeviceSynchronize();
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	unsigned int grid;
	unsigned int block;
    compute_active_thread(&block, &grid,nzcnt_len,pad, deviceProp.major,deviceProp.minor,
					deviceProp.warpSize,deviceProp.multiProcessorCount);
	
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(spmv_jds_naive), hipFuncCachePreferL1);

	//main execution
	pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

	for (int i=0; i<50; i++)
	spmv_jds_naive<<<grid, block>>>(d_Ax_vector,
  				d_data,d_indices,d_perm,
				d_x_vector,d_nzcnt,dim);
							
    CUERR // check and clear any existing errors
	
	hipDeviceSynchronize();
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//HtoD memory copy
	hipMemcpy(h_Ax_vector, d_Ax_vector,dim*sizeof(float), hipMemcpyDeviceToHost);	

	hipDeviceSynchronize();
	hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_ptr);
	hipFree(d_perm);
    hipFree(d_nzcnt);
    hipFree(d_x_vector);
	hipFree(d_Ax_vector);
 
	if (parameters->outFile) {
		pb_SwitchToTimer(&timers, pb_TimerID_IO);
		//int temp = ((dim + 31)/32)*32; // hack because of "gold" version including padding
		outputData(parameters->outFile,h_Ax_vector,dim);
		
	}
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	free (h_data);
	free (h_indices);
	free (h_ptr);
	free (h_perm);
	free (h_nzcnt);
	free (h_Ax_vector);
	free (h_x_vector);
	pb_SwitchToTimer(&timers, pb_TimerID_NONE);

	pb_PrintTimerSet(&timers);
	pb_FreeParameters(parameters);

	return 0;

}
