#include "hip/hip_runtime.h"
/**
 * 2mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size. */
# define NI 2048
# define NJ 2048
# define NK 2048
# define NL 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i*NI + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i*NK + j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}

	for (i = 0; i < NL; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NL + j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;	
		}
	}
}


void compareResults(DATA_TYPE *E, DATA_TYPE *E_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NL; i++)
	{
		for (j=0; j < NI; j++)
		{
			if (percentDiff(E[i*NI + j], E_outputFromGpu[i*NI + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void mm2_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{ 
		int k;
		for (k = 0; k < NK; k++)
		{
			C[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}


__global__ void mm2_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{ 
		int k;
		for (k = 0; k < NJ; k++)
		{
			E[i * NL + j] += C[i * NJ + k] * D[k * NL + j];
		}
	}
}


void mm2_cpu(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E)
{
	int i, j, k;
	
  	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			C[i*NJ + j] = 0.0;
			for (k = 0; k < NK; ++k)
			{
				C[i*NJ + j] += A[i*NK + k] * B[k*NJ + j];
			}
		}
	}
	
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			E[i*NL + j] = 0.0;
			for (k = 0; k < NJ; ++k)
			{
				E[i*NL + j] += C[i*NJ + k] * D[k*NL + j];
			}
		}
	}
}


void mm2Cuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, DATA_TYPE* E_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NL);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
		
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)ceil( ((float)NJ) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
	dim3 grid2((size_t)ceil( ((float)NL) / ((float)block.x) ), (size_t)ceil( ((float)NI) / ((float)block.y)) );
	t_start = rtclock();
	mm2_kernel1<<<grid1,block>>>(A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();
	mm2_kernel2<<<grid2,block>>>(C_gpu, D_gpu, E_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	hipMemcpy(E_outputFromGpu, E_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
}


int main(int argc, char** argv)
{
	double t_start, t_end;
	
	DATA_TYPE* C;
	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* D;
	DATA_TYPE* E;
	DATA_TYPE* E_outputFromGpu;

	C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));
	D = (DATA_TYPE*)malloc(NJ*NL*sizeof(DATA_TYPE));
	E = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));
	E_outputFromGpu = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));

  	init_array(A, B, C, D);
	GPU_argv_init();

	mm2Cuda(A, B, C, D, E, E_outputFromGpu);

	t_start = rtclock();
	mm2_cpu(A, B, C, D, E);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(E, E_outputFromGpu);

	free(C);
	free(A);
	free(B);
	free(D);
	free(E);
	free(E_outputFromGpu);

  	return 0;
}

