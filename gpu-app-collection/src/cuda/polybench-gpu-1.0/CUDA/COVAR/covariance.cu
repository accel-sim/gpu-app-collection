#include "hip/hip_runtime.h"
/**
 * covariance.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 1.05

#define GPU_DEVICE 0

/* Problem size */
#define M 2048
#define N 2048

/* Thread block dimensions for kernel 1*/
#define DIM_THREAD_BLOCK_KERNEL_1_X 256
#define DIM_THREAD_BLOCK_KERNEL_1_Y 1

/* Thread block dimensions for kernel 2*/
#define DIM_THREAD_BLOCK_KERNEL_2_X 32
#define DIM_THREAD_BLOCK_KERNEL_2_Y 8

/* Thread block dimensions for kernel 3*/
#define DIM_THREAD_BLOCK_KERNEL_3_X 256
#define DIM_THREAD_BLOCK_KERNEL_3_Y 1

#define sqrt_of_array_cell(x,j) sqrt(x[j])

#define FLOAT_N 3214212.01
#define EPS 0.005

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_arrays(DATA_TYPE* data)
{
	int i, j;

	for (i = 1; i < (M+1); i++)
	{
		for (j = 1; j < (N+1); j++)
		{
			data[i*(N+1) + j] = ((DATA_TYPE) i*j) / M;
		}
	}
}


void covariance(DATA_TYPE* data, DATA_TYPE* symmat, DATA_TYPE* mean)
{
	int i, j, j1,j2;

  	/* Determine mean of column vectors of input data matrix */
	for (j = 1; j < (M+1); j++)
	{
		mean[j] = 0.0;
		for (i = 1; i < (N+1); i++)
		{
        		mean[j] += data[i*(M+1) + j];
		}
		mean[j] /= FLOAT_N;
	}

  	/* Center the column vectors. */
	for (i = 1; i < (N+1); i++)
	{
		for (j = 1; j < (M+1); j++)
		{
			data[i*(M+1) + j] -= mean[j];
		}
	}

  	/* Calculate the m * m covariance matrix. */
	for (j1 = 1; j1 < (M+1); j1++)
	{
		for (j2 = j1; j2 < (M+1); j2++)
     		{
       		symmat[j1*(M+1) + j2] = 0.0;
			for (i = 1; i < N+1; i++)
			{
				symmat[j1*(M+1) + j2] += data[i*(M+1) + j1] * data[i*(M+1) + j2];
			}
        		symmat[j2*(M+1) + j1] = symmat[j1*(M+1) + j2];
      		}
	}
}


void compareResults(DATA_TYPE* symmat, DATA_TYPE* symmat_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=1; i < (M+1); i++)
	{
		for (j=1; j < (N+1); j++)
		{
			if (percentDiff(symmat[i*(N+1) + j], symmat_outputFromGpu[i*(N+1) + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}			
		}
	}
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
	
	return;
}


__global__ void mean_kernel(DATA_TYPE *mean, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

	if ((j >= 1) && (j < (M+1)))
	{
		mean[j] = 0.0;

		int i;
		for(i = 1; i < (N+1); i++)
		{
			mean[j] += data[i * (M+1) + j];
		}
		mean[j] /= (DATA_TYPE)FLOAT_N;
	}
}


__global__ void reduce_kernel(DATA_TYPE *mean, DATA_TYPE *data)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
		
	if ((i >= 1) && (i < (N+1)) && (j >= 1) && (j < (M+1)))
	{
		data[i * (M+1) + j] -= mean[j];	
	}
}


__global__ void covar_kernel(DATA_TYPE *symmat, DATA_TYPE *data)
{
	int j1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int i, j2;

	if ((j1 >= 1) && (j1 < (M+1)))
	{
		for (j2 = j1; j2 < (M+1); j2++)
		{		
			symmat[j1*(M+1) + j2] = 0.0;
			for(i = 1; i < (N+1); i++)
			{
				symmat[j1 * (M+1) + j2] += data[i *(M+1) + j1] * data[i *(M+1) + j2];
			}
			symmat[j2 * (M+1) + j1] = symmat[j1 * (M+1) + j2];
		}
	}
}


void covarianceCuda(DATA_TYPE* data, DATA_TYPE* symmat, DATA_TYPE* mean, DATA_TYPE* symmat_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *data_gpu;
	DATA_TYPE *mean_gpu;
	DATA_TYPE *symmat_gpu;

	hipMalloc((void **)&data_gpu, sizeof(DATA_TYPE) * (M+1) * (N+1));
	hipMalloc((void **)&symmat_gpu, sizeof(DATA_TYPE) * (M+1) * (M+1));
	hipMalloc((void **)&mean_gpu, sizeof(DATA_TYPE) * (M+1));
	hipMemcpy(data_gpu, data, sizeof(DATA_TYPE) * (M+1) * (N+1), hipMemcpyHostToDevice);
	hipMemcpy(symmat_gpu, symmat, sizeof(DATA_TYPE) * (M+1) * (M+1), hipMemcpyHostToDevice);
	hipMemcpy(mean_gpu, mean, sizeof(DATA_TYPE) * (M+1), hipMemcpyHostToDevice);
	
	dim3 block1(DIM_THREAD_BLOCK_KERNEL_1_X, DIM_THREAD_BLOCK_KERNEL_1_Y);
	dim3 grid1((size_t)(ceil((float)M) / ((float)DIM_THREAD_BLOCK_KERNEL_1_X)), 1);
	
	dim3 block2(DIM_THREAD_BLOCK_KERNEL_2_X, DIM_THREAD_BLOCK_KERNEL_2_Y);
	dim3 grid2((size_t)(ceil((float)M) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)), (size_t)(ceil((float)N) / ((float)DIM_THREAD_BLOCK_KERNEL_2_X)));
	
	dim3 block3(DIM_THREAD_BLOCK_KERNEL_3_X, DIM_THREAD_BLOCK_KERNEL_3_Y);
	dim3 grid3((size_t)(ceil((float)M) / ((float)DIM_THREAD_BLOCK_KERNEL_3_X)), 1);
	
	t_start = rtclock();

	mean_kernel<<<grid1, block1>>>(mean_gpu,data_gpu);
	hipDeviceSynchronize();
	reduce_kernel<<<grid2, block2>>>(mean_gpu,data_gpu);
	hipDeviceSynchronize();
	covar_kernel<<<grid3, block3>>>(symmat_gpu,data_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	hipMemcpy(symmat_outputFromGpu, symmat_gpu, sizeof(DATA_TYPE) * (M+1) * (N+1), hipMemcpyDeviceToHost);
	
	hipFree(data_gpu);
	hipFree(symmat_gpu);
	hipFree(mean_gpu);
}


int main()
{
	double t_start, t_end;

	DATA_TYPE* data;
	DATA_TYPE* symmat;
	DATA_TYPE* mean;
	DATA_TYPE* symmat_outputFromGpu;	

	data = (DATA_TYPE*)malloc((M+1)*(N+1)*sizeof(DATA_TYPE));
	symmat = (DATA_TYPE*)malloc((M+1)*(M+1)*sizeof(DATA_TYPE));
	mean = (DATA_TYPE*)malloc((M+1)*sizeof(DATA_TYPE));
	symmat_outputFromGpu = (DATA_TYPE*)malloc((M+1)*(M+1)*sizeof(DATA_TYPE));	

	init_arrays(data);
    
	GPU_argv_init();

	covarianceCuda(data, symmat, mean, symmat_outputFromGpu);
	
	t_start = rtclock();
	covariance(data, symmat, mean);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(symmat, symmat_outputFromGpu);

	free(data);
	free(symmat);
	free(mean);
	free(symmat_outputFromGpu);	

  	return 0;
}

