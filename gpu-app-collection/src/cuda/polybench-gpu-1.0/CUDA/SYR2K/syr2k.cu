#include "hip/hip_runtime.h"
/**
 * syr2k.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define N 2048
#define M 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 12435
#define BETA 4546

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_arrays(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j;
  
	for (i = 0; i < N; i++)
    	{
    		for (j = 0; j < N; j++)
		{
			C[i*N + j] = ((DATA_TYPE) i*j + 2) / N;
		}
      	
		for (j = 0; j < M; j++)
		{
	  		A[i*N + j] = ((DATA_TYPE) i*j) / N;
	  		B[i*N + j] = ((DATA_TYPE) i*j + 1) / N;
		}
    	}
}


void syr2k(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j, k;
		
  	for (i = 0; i < N; i++)
	{
   		for (j = 0; j < N; j++)
		{
     			C[i*N + j] *= BETA;
		}
	}

  	for (i = 0; i < N; i++)
	{
   		for (j = 0; j < N; j++)
		{
      			for (k = 0; k < M; k++)
			{
	  			C[i*N + j] += ALPHA * A[i*M + k] * B[j*M + k];
	 		 	C[i*N + j] += ALPHA * B[i*M + k] * A[j*M + k];
			}
		}
	}
}


void compareResults(DATA_TYPE *C, DATA_TYPE *C_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	// Compare C with D
	for (i=0; i<N; i++)
	{
		for (j=0; j<N; j++)
		{
			if (percentDiff(C[i*N + j], C_outputFromGpu[i*N + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{ 
				fail++;
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void syr2k_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < N) && (j < N))
	{
		c[i * N + j] *= BETA;
		
		int k;
		for(k = 0; k < M; k++)
		{
			c[i * N + j] += ALPHA * a[i * M + k] * b[j * M + k] + ALPHA * b[i * M + k] * a[j * M + k];
		}
	}
}


void syr2kCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu) 
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * M);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * N * M);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * N * N);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)ceil( ((float)N) / ((float)DIM_THREAD_BLOCK_X) ), (size_t)(ceil( ((float)N) / ((float)DIM_THREAD_BLOCK_Y) )));
	
	t_start = rtclock();
	syr2k_kernel<<<grid,block>>>(A_gpu,B_gpu,C_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}


int main()
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* C;
	DATA_TYPE* C_outputFromGpu;

	A = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));
	C_outputFromGpu = (DATA_TYPE*)malloc(N*M*sizeof(DATA_TYPE));

	init_arrays(A, B, C);
    
	GPU_argv_init();
	syr2kCuda(A, B, C, C_outputFromGpu);
	
	t_start = rtclock();
	syr2k(A, B, C);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(C, C_outputFromGpu);

	free(A);
	free(B);
	free(C);
	free(C_outputFromGpu);

  	return 0;
}

