#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <tuple>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsparse.h>

#include "tensor.h"
#include "cuda_helper.h"
#include "sparse_gemm_problems.h"

hiprandGenerator_t curand_gen;

std::string getCusparseErrorString(hipsparseStatus_t status) {
    if (status == HIPSPARSE_STATUS_SUCCESS)
        return "HIPSPARSE_STATUS_SUCCESS";
    else if (status == HIPSPARSE_STATUS_NOT_INITIALIZED)
        return "HIPSPARSE_STATUS_NOT_INITIALIZED";
    else if (status == HIPSPARSE_STATUS_ALLOC_FAILED)
        return "HIPSPARSE_STATUS_ALLOC_FAILED";
    else if (status == HIPSPARSE_STATUS_ARCH_MISMATCH)
        return "HIPSPARSE_STATUS_ARCH_MISMATCH";
    else if (status == HIPSPARSE_STATUS_MAPPING_ERROR)
        return "HIPSPARSE_STATUS_MAPPING_ERROR";
    else if (status == HIPSPARSE_STATUS_EXECUTION_FAILED)
        return "HIPSPARSE_STATUS_EXECUTION_FAILED";
    else if (status == HIPSPARSE_STATUS_INTERNAL_ERROR)
        return "HIPSPARSE_STATUS_INTERNAL_ERROR";
    else if (status == HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED)
        return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    else if (status == HIPSPARSE_STATUS_ZERO_PIVOT)
        return "HIPSPARSE_STATUS_ZERO_PIVOT";
    else
        return "Unknown CUSPARSE error type";
}

void throw_cusparse_err(hipsparseStatus_t status, int line, const char* filename) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        std::stringstream ss;
        ss << "CUSPARSE failure: " << getCusparseErrorString(status) <<
              " in " << filename << " at line: " << line << std::endl;
        throw std::runtime_error(ss.str());
    }
}

#define CHECK_CUSPARSE_ERROR(status) throw_cusparse_err(status, __LINE__, __FILE__)

class CusparseHandle {
    std::shared_ptr<hipsparseHandle_t> ptr_;

    struct CusparseHandleDeleter {
        void operator()(hipsparseHandle_t * handle) {
            hipsparseDestroy(*handle);
        }
    };

public:

    CusparseHandle() : ptr_(new hipsparseHandle_t, CusparseHandleDeleter()) {
        if (hipsparseCreate(ptr_.get()) != HIPSPARSE_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cusparseHandle");
        }

    }

    hipsparseHandle_t  get() { return *ptr_; };
};

class CusparseMatDesc {
    std::shared_ptr<hipsparseMatDescr_t> ptr_;

    struct CusparseMatDescDeleter {
        void operator()(hipsparseMatDescr_t * handle) {
            hipsparseDestroyMatDescr(*handle);
        }
    };

    public:

    CusparseMatDesc() : ptr_(new hipsparseMatDescr_t, CusparseMatDescDeleter()) {
        if (hipsparseCreateMatDescr(ptr_.get()) != HIPSPARSE_STATUS_SUCCESS) {
            throw std::runtime_error("Failed to create cusparse matrix descriptor");
        }

    }

    hipsparseMatDescr_t get() { return *ptr_; };

};

template <typename T>
class SparseOp {

    Tensor<T> csrVal_;
    Tensor<int> csrRowPtr_;
    Tensor<int> csrColInd_;

    Tensor<int> nnzPerRow_;

    int rows_, cols_;
    int nnzTotal_;

    // Cusparse handle
    CusparseHandle cusparse_handle_;

    // Cusparse matrix descriptor.
    CusparseMatDesc cusparse_descr_;


    float alpha_ = 1.0;
    float beta_ = 0.0;

public:

    SparseOp(int rows, int cols, Tensor<T> dense_array) :
        rows_(rows),
        cols_(cols),
        cusparse_handle_(),
        cusparse_descr_() {


        //Assign descr attributes
        hipsparseSetMatType(cusparse_descr_.get(),HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(cusparse_descr_.get(),HIPSPARSE_INDEX_BASE_ZERO);

        nnzPerRow_ = Tensor<int>({rows_});

        // compute total non zeros and non zeros per row.
        CHECK_CUSPARSE_ERROR(hipsparseSnnz(cusparse_handle_.get(),
                                          HIPSPARSE_DIRECTION_ROW,
                                          rows_,
                                          cols_,
                                          cusparse_descr_.get(),
                                          dense_array.begin(),
                                          rows_,
                                          nnzPerRow_.begin(),
                                          &nnzTotal_));

        //allocate memory for csr matrix arrays.
        csrVal_ = Tensor<T>({nnzTotal_});
        csrRowPtr_ = Tensor<int>({rows_+1});
        csrColInd_ = Tensor<int>({nnzTotal_});

        // Convert Dense matrix to CSR format
        CHECK_CUSPARSE_ERROR(hipsparseSdense2csr(cusparse_handle_.get(),
                                                rows_,
                                                cols_,
                                                cusparse_descr_.get(),
                                                dense_array.begin(),
                                                rows_,
                                                nnzPerRow_.begin(),
                                                csrVal_.begin(),
                                                csrRowPtr_.begin(),
                                                csrColInd_.begin()));

    }

    void sparse_gemm(Tensor<float> b, Tensor<float> c, int mini_batch) {

        if (mini_batch == 1) {
            // Sparse gemv
            CHECK_CUSPARSE_ERROR(hipsparseScsrmv(cusparse_handle_.get(),
                                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                rows_,
                                                cols_,
                                                nnzTotal_,
                                                &alpha_,
                                                cusparse_descr_.get(),
                                                csrVal_.begin(),
                                                csrRowPtr_.begin(),
                                                csrColInd_.begin(),
                                                b.begin(),
                                                &beta_,
                                                c.begin()));
        } else {
            // Sparse gemm
            CHECK_CUSPARSE_ERROR(hipsparseScsrmm(cusparse_handle_.get(),
                                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                rows_,
                                                mini_batch,
                                                cols_,
                                                nnzTotal_,
                                                &alpha_,
                                                cusparse_descr_.get(),
                                                csrVal_.begin(),
                                                csrRowPtr_.begin(),
                                                csrColInd_.begin(),
                                                b.begin(),
                                                cols_,
                                                &beta_,
                                                c.begin(),
                                                rows_));
        }

    }

};

Tensor<float> generate_sparse_matrix(int rows, int cols, float sparsity) {

    auto dense_mat = rand<float>({rows, cols}, curand_gen);

    float * cpu_sp_mat;

    cpu_sp_mat = new float [rows*cols];

    CHECK_CUDA_ERROR(hipMemcpy(cpu_sp_mat, dense_mat.begin(), rows*cols*sizeof(float), hipMemcpyDeviceToHost));

    int count = 0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (cpu_sp_mat[i*cols+j] < sparsity) {
                cpu_sp_mat[i*cols+j] = 0.f;
                count++;
            }
        }
    }

    CHECK_CUDA_ERROR(hipMemcpy(dense_mat.begin(), cpu_sp_mat, rows*cols*sizeof(float), hipMemcpyHostToDevice));

    delete[] cpu_sp_mat;

    return dense_mat;
}

std::tuple<int, int> time_sparse_gemm(Tensor<float> A, Tensor<float> B, Tensor<float> C,
                                      bool a_t, bool b_t, hipblasHandle_t cublas_handle) {

    auto a_dims = A.dims();
    auto b_dims = B.dims();

    int m = C.dims()[0];
    int k = a_t ? A.dims()[0] : A.dims()[1];
    int n = C.dims()[1];

    const float alpha = 1.f / static_cast<float>(A.dims()[1]);
    const float beta  = 1.f;

    int numRepeats = std::max(std::ceil(1e11 / (m * k * n)), 10.);

    // Warm up
    hipblasStatus_t stat = hipblasSgemm(cublas_handle,
                a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                m,
                n,
                k,
                &alpha,
                A.begin(), A.dims()[0],
                B.begin(), B.dims()[0],
                &beta,
                C.begin(), C.dims()[0]);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("sgemm failed");
    }

    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i) {
        hipblasStatus_t stat = hipblasSgemm(cublas_handle,
                    a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    m,
                    n,
                    k,
                    &alpha,
                    A.begin(), A.dims()[0],
                    B.begin(), B.dims()[0],
                    &beta,
                    C.begin(), C.dims()[0]);
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("sgemm failed");
        }
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();

    int d_time = static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / numRepeats);

    // Only support 2D arrays in this benchmark.
    assert (a_dims.size() == 2);
    assert (b_dims.size() == 2);

    auto csr_A = SparseOp<float>(a_dims[0], a_dims[1], A);

    csr_A.sparse_gemm(B, C, b_dims[1]);
    hipDeviceSynchronize();

    start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i) {
        csr_A.sparse_gemm(B, C, b_dims[1]);
    }

    hipDeviceSynchronize();
    end = std::chrono::steady_clock::now();
    int sp_time = static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / numRepeats);


    return std::tuple<int, int>(sp_time, d_time);
}

int main() {

    hipFree(0);

    hipblasHandle_t cublas_handle;
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS init failed" << std::endl;
    }


    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    std::cout << std::setw(30) << "Times" << std::endl;
    std::cout << std::setfill('-') << std::setw(110) << "-" << std::endl;
    std::cout << std::setfill(' ');
    std::cout << "    m       n      k      a_t     b_t    sparsity   sparse time (usec)  dense time (usec)  speedup" << std::endl;
    for (const auto &problem : inference_server_set) {
        int m,n,k;
        bool a_t, b_t;
        float sparsity;

        std::tie(m, n, k, a_t, b_t, sparsity) = problem;

        auto a = generate_sparse_matrix(m, k, sparsity);
        auto b = rand<float>({k, n}, curand_gen);
        auto c = zeros<float>({m, n});

        std::cout << std::setw(7) << m;
        std::cout << std::setw(7) << n;
        std::cout << std::setw(7) << k;
        std::cout << std::setw(7) << a_t ? "t" : "n";
        std::cout << std::setw(7) << b_t ? "t" : "n";
        std::cout << std::setw(11) << sparsity;
        std::cout << std::setw(13) << std::setprecision(6);

        int sp_time, d_time;
        std::tie(sp_time, d_time) = time_sparse_gemm(a, b, c, a_t, b_t, cublas_handle);

        std::cout << std::setw(16) << sp_time;
        std::cout << std::setw(16) << d_time;
        std::cout << std::setw(20) << float(d_time)/sp_time;
        std::cout << std::endl;
    }

    return 0;
}
