#include "hip/hip_runtime.h"
//========================================================================================================================================================================================================200
//	findRangeK function
//========================================================================================================================================================================================================200

__global__ void 
findRangeK(	long height,

			knode *knodesD,
			long knodes_elem,

			long *currKnodeD,
			long *offsetD,
			long *lastKnodeD,
			long *offset_2D,
			int *startD,
			int *endD,
			int *RecstartD, 
			int *ReclenD)
{

	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;

	// ???
	int i;
	for(i = 0; i < height; i++){

		if((knodesD[currKnodeD[bid]].keys[thid] <= startD[bid]) && (knodesD[currKnodeD[bid]].keys[thid+1] > startD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[currKnodeD[bid]].indices[thid] < knodes_elem){
				offsetD[bid] = knodesD[currKnodeD[bid]].indices[thid];
			}
		}
		if((knodesD[lastKnodeD[bid]].keys[thid] <= endD[bid]) && (knodesD[lastKnodeD[bid]].keys[thid+1] > endD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset_2[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[lastKnodeD[bid]].indices[thid] < knodes_elem){
				offset_2D[bid] = knodesD[lastKnodeD[bid]].indices[thid];
			}
		}
		__syncthreads();

		// set for next tree level
		if(thid==0){
			currKnodeD[bid] = offsetD[bid];
			lastKnodeD[bid] = offset_2D[bid];
		}
		__syncthreads();
	}

	// Find the index of the starting record
	if(knodesD[currKnodeD[bid]].keys[thid] == startD[bid]){
		RecstartD[bid] = knodesD[currKnodeD[bid]].indices[thid];
	}
	__syncthreads();

	// Find the index of the ending record
	if(knodesD[lastKnodeD[bid]].keys[thid] == endD[bid]){
		ReclenD[bid] = knodesD[lastKnodeD[bid]].indices[thid] - RecstartD[bid]+1;
	}

}

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200
