#include "hip/hip_runtime.h"
/// 
/// @file    rdwt53.cu
/// @brief   CUDA implementation of reverse 5/3 2D DWT.
/// @author  Martin Jirman (207962@mail.muni.cz)
/// @date    2011-02-04 14:19
///
///
/// Copyright (c) 2011 Martin Jirman
/// All rights reserved.
/// 
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions are met:
/// 
///     * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
///     * Redistributions in binary form must reproduce the above copyright
///       notice, this list of conditions and the following disclaimer in the
///       documentation and/or other materials provided with the distribution.
/// 
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
/// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
/// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
/// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
/// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
/// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
/// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
/// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
/// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
/// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.
///


#include "common.h"
#include "transform_buffer.h"
#include "io.h"


namespace dwt_cuda {

  

  /// Wraps shared momory buffer and algorithms needed for computing 5/3 RDWT
  /// using sliding window and lifting schema.
  /// @tparam WIN_SIZE_X  width of sliding window
  /// @tparam WIN_SIZE_Y  height of sliding window
  template <int WIN_SIZE_X, int WIN_SIZE_Y>
  class RDWT53 {
  private: 
    
    /// Shared memory buffer used for 5/3 DWT transforms.
    typedef TransformBuffer<int, WIN_SIZE_X, WIN_SIZE_Y + 3, 2> RDWT53Buffer;

    /// Shared buffer used for reverse 5/3 DWT.
    RDWT53Buffer buffer;

    /// Difference between indices of two vertically neighboring items in buffer.
    enum { STRIDE = RDWT53Buffer::VERTICAL_STRIDE };


    /// Info needed for loading of one input column from input image.
    /// @tparam CHECKED  true if loader should check boundaries
    template <bool CHECKED>
    struct RDWT53Column {
      /// loader of pixels from column in input image
      VerticalDWTBandLoader<int, CHECKED> loader;
      
      /// Offset of corresponding column in shared buffer.
      int offset;
      
      /// Sets all fields to some values to avoid 'uninitialized' warnings.
      __device__ void clear() {
        offset = 0;
        loader.clear();
      }
    };


    /// 5/3 DWT reverse update operation.
    struct Reverse53Update {
      __device__ void operator() (const int p, int & c, const int n) const {
        c -= (p + n + 2) / 4;  // F.3, page 118, ITU-T Rec. T.800 final draft
      }
    };


    /// 5/3 DWT reverse predict operation.
    struct Reverse53Predict {
      __device__ void operator() (const int p, int & c, const int n) const {
        c += (p + n) / 2;      // F.4, page 118, ITU-T Rec. T.800 final draft
      }
    };


    /// Horizontal 5/3 RDWT on specified lines of transform buffer.
    /// @param lines      number of lines to be transformed
    /// @param firstLine  index of the first line to be transformed
    __device__ void horizontalTransform(const int lines, const int firstLine) {
      __syncthreads();
      buffer.forEachHorizontalEven(firstLine, lines, Reverse53Update());
      __syncthreads();
      buffer.forEachHorizontalOdd(firstLine, lines, Reverse53Predict());
      __syncthreads();
    }


    /// Using given loader, it loads another WIN_SIZE_Y coefficients
    /// into specified column.
    /// @tparam CHECKED  true if loader should check image boundaries
    /// @param input     input coefficients to load from
    /// @param col       info about loaded column
    template <bool CHECKED>
    inline __device__ void loadWindowIntoColumn(const int * const input,
                                                RDWT53Column<CHECKED> & col) {
      for(int i = 3; i < (3 + WIN_SIZE_Y); i += 2) {
        buffer[col.offset + i * STRIDE] = col.loader.loadLowFrom(input);
        buffer[col.offset + (i + 1) * STRIDE] = col.loader.loadHighFrom(input);
      }
    }


    /// Initializes one column of shared transform buffer with 7 input pixels.
    /// Those 7 pixels will not be transformed. Also initializes given loader.
    /// @tparam CHECKED  true if loader should check image boundaries
    /// @param columnX   x coordinate of column in shared transform buffer
    /// @param input     input image
    /// @param sizeX     width of the input image
    /// @param sizeY     height of the input image
    /// @param loader    (uninitialized) info about loaded column
    template <bool CHECKED>
    __device__ void initColumn(const int columnX, const int * const input, 
                               const int sizeX, const int sizeY,
                               RDWT53Column<CHECKED> & column,
                               const int firstY) {
      // coordinates of the first coefficient to be loaded
      const int firstX = blockIdx.x * WIN_SIZE_X + columnX;

      // offset of the column with index 'colIndex' in the transform buffer
      column.offset = buffer.getColumnOffset(columnX);

      if(blockIdx.y == 0) {
        // topmost block - apply mirroring rules when loading first 3 rows
        column.loader.init(sizeX, sizeY, firstX, firstY);

        // load pixels in mirrored way
        buffer[column.offset + 1 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 0 * STRIDE] =
        buffer[column.offset + 2 * STRIDE] = column.loader.loadHighFrom(input);
      } else {
        // non-topmost row - regular loading:
        column.loader.init(sizeX, sizeY, firstX, firstY - 1);
        buffer[column.offset + 0 * STRIDE] = column.loader.loadHighFrom(input);
        buffer[column.offset + 1 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 2 * STRIDE] = column.loader.loadHighFrom(input);
      }
      // Now, the next coefficient, which will be loaded by loader, is #2.
    }


    /// Actual GPU 5/3 RDWT implementation.
    /// @tparam CHECKED_LOADS   true if boundaries must be checked when reading
    /// @tparam CHECKED_WRITES  true if boundaries must be checked when writing
    /// @param in        input image (5/3 transformed coefficients)
    /// @param out       output buffer (for reverse transformed image)
    /// @param sizeX     width of the output image 
    /// @param sizeY     height of the output image
    /// @param winSteps  number of sliding window steps
    template<bool CHECKED_LOADS, bool CHECKED_WRITES>
    __device__ void transform(const int * const in, int * const out,
                              const int sizeX, const int sizeY,
                              const int winSteps) {
      // info about one main and one boundary column
      RDWT53Column<CHECKED_LOADS> column, boundaryColumn;

      // index of first row to be transformed
      const int firstY = blockIdx.y * WIN_SIZE_Y * winSteps;

      // some threads initialize boundary columns
      boundaryColumn.clear();
      if(threadIdx.x < 3) {
        // First 3 threads also handle boundary columns. Thread #0 gets right
        // column #0, thread #1 get right column #1 and thread #2 left column.
        const int colId = threadIdx.x + ((threadIdx.x != 2) ? WIN_SIZE_X : -3);

        // Thread initializes offset of the boundary column (in shared 
        // buffer), first 3 pixels of the column and a loader for this column.
        initColumn(colId, in, sizeX, sizeY, boundaryColumn, firstY);
      }

      // All threads initialize central columns.
      initColumn(parityIdx<WIN_SIZE_X>(), in, sizeX, sizeY, column, firstY);

      // horizontally transform first 3 rows
      horizontalTransform(3, 0);

      // writer of output pixels - initialize it
      const int outX = blockIdx.x * WIN_SIZE_X + threadIdx.x;
      VerticalDWTPixelWriter<int, CHECKED_WRITES> writer;
      writer.init(sizeX, sizeY, outX, firstY);

      // offset of column (in transform buffer) saved by this thread
      const int outputColumnOffset = buffer.getColumnOffset(threadIdx.x);

      // (Each iteration assumes that first 3 rows of transform buffer are
      // already loaded with horizontally transformed pixels.)
      for(int w = 0; w < winSteps; w++) {
        // Load another WIN_SIZE_Y lines of this thread's column
        // into the transform buffer.
        loadWindowIntoColumn(in, column);

        // possibly load boundary columns
        if(threadIdx.x < 3) {
          loadWindowIntoColumn(in, boundaryColumn);
        }

        // horizontally transform all newly loaded lines
        horizontalTransform(WIN_SIZE_Y, 3);

        // Using 3 registers, remember current values of last 3 rows 
        // of transform buffer. These rows are transformed horizontally 
        // only and will be used in next iteration.
        int last3Lines[3];
        last3Lines[0] = buffer[outputColumnOffset + (WIN_SIZE_Y + 0) * STRIDE];
        last3Lines[1] = buffer[outputColumnOffset + (WIN_SIZE_Y + 1) * STRIDE];
        last3Lines[2] = buffer[outputColumnOffset + (WIN_SIZE_Y + 2) * STRIDE];

        // vertically transform all central columns
        buffer.forEachVerticalOdd(outputColumnOffset, Reverse53Update());
        buffer.forEachVerticalEven(outputColumnOffset, Reverse53Predict());

        // Save all results of current window. Results are in transform buffer
        // at rows from #1 to #(1 + WIN_SIZE_Y). Other rows are invalid now.
        // (They only served as a boundary for vertical RDWT.)
        for(int i = 1; i < (1 + WIN_SIZE_Y); i++) {
          writer.writeInto(out, buffer[outputColumnOffset + i * STRIDE]);
        }

        // Use last 3 remembered lines as first 3 lines for next iteration.
        // As expected, these lines are already horizontally transformed.
        buffer[outputColumnOffset + 0 * STRIDE] = last3Lines[0];
        buffer[outputColumnOffset + 1 * STRIDE] = last3Lines[1];
        buffer[outputColumnOffset + 2 * STRIDE] = last3Lines[2];

        // Wait for all writing threads before proceeding to loading new
        // coeficients in next iteration. (Not to overwrite those which
        // are not written yet.)
        __syncthreads();
      }
    }


  public:
    /// Main GPU 5/3 RDWT entry point.
    /// @param in     input image (5/3 transformed coefficients)
    /// @param out    output buffer (for reverse transformed image)
    /// @param sizeX  width of the output image 
    /// @param sizeY  height of the output image
    /// @param winSteps  number of sliding window steps
    __device__ static void run(const int * const input, int * const output,
                               const int sx, const int sy, const int steps) {
      // prepare instance with buffer in shared memory
      __shared__ RDWT53<WIN_SIZE_X, WIN_SIZE_Y> rdwt53;

      // Compute limits of this threadblock's block of pixels and use them to
      // determine, whether this threadblock will have to deal with boundary.
      // (1 in next expressions is for radius of impulse response of 5/3 RDWT.)
      const int maxX = (blockIdx.x + 1) * WIN_SIZE_X + 1;
      const int maxY = (blockIdx.y + 1) * WIN_SIZE_Y * steps + 1;
      const bool atRightBoudary = maxX >= sx;
      const bool atBottomBoudary = maxY >= sy;

      // Select specialized version of code according to distance of this
      // threadblock's pixels from image boundary.
      if(atBottomBoudary) {
        // near bottom boundary => check both writing and reading
        rdwt53.transform<true, true>(input, output, sx, sy, steps);
      } else if(atRightBoudary) {
        // near right boundary only => check writing only
        rdwt53.transform<false, true>(input, output, sx, sy, steps);
      } else {
        // no nearby boundary => check nothing
        rdwt53.transform<false, false>(input, output, sx, sy, steps);
      }
    }

  }; // end of class RDWT53
  
  
  
  /// Main GPU 5/3 RDWT entry point.
  /// @param in     input image (5/3 transformed coefficients)
  /// @param out    output buffer (for reverse transformed image)
  /// @param sizeX  width of the output image 
  /// @param sizeY  height of the output image
  /// @param winSteps  number of sliding window steps
  template <int WIN_SX, int WIN_SY>
  __launch_bounds__(WIN_SX, CTMIN(SHM_SIZE/sizeof(RDWT53<WIN_SX, WIN_SY>), 8))
  __global__ void rdwt53Kernel(const int * const in, int * const out,
                               const int sx, const int sy, const int steps) {
    RDWT53<WIN_SX, WIN_SY>::run(in, out, sx, sy, steps);
  }
  
  
  
  /// Only computes optimal number of sliding window steps, 
  /// number of threadblocks and then lanches the 5/3 RDWT kernel.
  /// @tparam WIN_SX  width of sliding window
  /// @tparam WIN_SY  height of sliding window
  /// @param in       input image
  /// @param out      output buffer
  /// @param sx       width of the input image 
  /// @param sy       height of the input image
  template <int WIN_SX, int WIN_SY>
  void launchRDWT53Kernel (int * in, int * out, const int sx, const int sy) {
    // compute optimal number of steps of each sliding window
    const int steps = divRndUp(sy, 15 * WIN_SY);
    
    // prepare grid size
    dim3 gSize(divRndUp(sx, WIN_SX), divRndUp(sy, WIN_SY * steps));
    
    // finally transform this level
    PERF_BEGIN
    rdwt53Kernel<WIN_SX, WIN_SY><<<gSize, WIN_SX>>>(in, out, sx, sy, steps);
    PERF_END("        RDWT53", sx, sy)
    CudaDWTTester::checkLastKernelCall("RDWT 5/3 kernel");
  }
    
  
  
  /// Reverse 5/3 2D DWT. See common rules (above) for more details.
  /// @param in      Input DWT coefficients. Format described in common rules.
  ///                Will not be preserved (will be overwritten).
  /// @param out     output buffer on GPU - will contain original image
  ///                in normalized range [-128, 127].
  /// @param sizeX   width of input image (in pixels)
  /// @param sizeY   height of input image (in pixels)
  /// @param levels  number of recursive DWT levels
  void rdwt53(int * in, int * out, int sizeX, int sizeY, int levels) {
    if(levels > 1) {
      // let this function recursively reverse transform deeper levels first
      const int llSizeX = divRndUp(sizeX, 2);
      const int llSizeY = divRndUp(sizeY, 2);
      rdwt53(in, out, llSizeX, llSizeY, levels - 1);
      
      // copy reverse transformed LL band from output back into the input
      memCopy(in, out, llSizeX, llSizeY);
    }
    
    // select right width of kernel for the size of the image
    if(sizeX >= 960) {
      launchRDWT53Kernel<192, 8>(in, out, sizeX, sizeY);
    } else if (sizeX >= 480) {
      launchRDWT53Kernel<128, 8>(in, out, sizeX, sizeY);
    } else {
      launchRDWT53Kernel<64, 8>(in, out, sizeX, sizeY);
    }
  }
  

} // end of namespace dwt_cuda
