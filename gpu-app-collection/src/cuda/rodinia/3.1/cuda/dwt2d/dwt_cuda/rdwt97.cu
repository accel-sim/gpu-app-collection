#include "hip/hip_runtime.h"
/// 
/// @file    rdwt97.cu
/// @brief   CUDA implementation of reverse 9/7 2D DWT.
/// @author  Martin Jirman (207962@mail.muni.cz)
/// @date    2011-02-03 21:59
///
///
/// Copyright (c) 2011 Martin Jirman
/// All rights reserved.
/// 
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions are met:
/// 
///     * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
///     * Redistributions in binary form must reproduce the above copyright
///       notice, this list of conditions and the following disclaimer in the
///       documentation and/or other materials provided with the distribution.
/// 
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
/// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
/// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
/// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
/// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
/// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
/// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
/// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
/// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
/// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.
///


#include "common.h"
#include "transform_buffer.h"
#include "io.h"


namespace dwt_cuda {

  
  /// Wraps shared memory buffer and methods for computing 9/7 RDWT using
  /// lifting schema and sliding window.
  /// @tparam WIN_SIZE_X  width of the sliding window
  /// @tparam WIN_SIZE_Y  height of the sliding window
  template <int WIN_SIZE_X, int WIN_SIZE_Y>
  class RDWT97 {
  private:
    
    /// Info related to loading of one input column.
    /// @tparam CHECKED true if boundary chould be checked,
    ///                 false if there is no near boudnary
    template <bool CHECKED>
    struct RDWT97Column  {
      /// laoder of input pxels for given column.
      VerticalDWTBandLoader<float, CHECKED> loader;
      
      /// Offset of loaded column in shared memory buffer.
      int offset;
      
      /// Sets all fields to some values to avoid 'uninitialized' warnings.
      __device__ void clear() {
        loader.clear();
        offset = 0;
      }
    };


    /// Shared memory buffer used for 9/7 DWT transforms.
    typedef TransformBuffer<float, WIN_SIZE_X, WIN_SIZE_Y + 7, 4> RDWT97Buffer;

    /// Shared buffer used for reverse 9/7 DWT.
    RDWT97Buffer buffer;

    /// Difference between indices of two vertical neighbors in buffer.
    enum { STRIDE = RDWT97Buffer::VERTICAL_STRIDE };


    /// Horizontal 9/7 RDWT on specified lines of transform buffer.
    /// @param lines      number of lines to be transformed
    /// @param firstLine  index of the first line to be transformed
    __device__ void horizontalRDWT97(int lines, int firstLine) {
      __syncthreads();
      buffer.scaleHorizontal(scale97Mul, scale97Div, firstLine, lines);
      __syncthreads();
      buffer.forEachHorizontalEven(firstLine, lines, AddScaledSum(r97update2));
      __syncthreads();
      buffer.forEachHorizontalOdd(firstLine, lines, AddScaledSum(r97predict2));
      __syncthreads();
      buffer.forEachHorizontalEven(firstLine, lines, AddScaledSum(r97update1));
      __syncthreads();
      buffer.forEachHorizontalOdd(firstLine, lines, AddScaledSum(r97Predict1));
      __syncthreads();
    }


    /// Initializes one column of shared transform buffer with 7 input pixels.
    /// Those 7 pixels will not be transformed. Also initializes given loader.
    /// @tparam CHECKED  true if there are near image boundaries
    /// @param colIndex  index of column in shared transform buffer
    /// @param input     input image
    /// @param sizeX     width of the input image
    /// @param sizeY     height of the input image
    /// @param column    (uninitialized) info about loading one column
    /// @param firstY    index of first image row to be transformed
    template <bool CHECKED>
    __device__ void initColumn(const int colIndex, const float * const input, 
                               const int sizeX, const int sizeY,
                               RDWT97Column<CHECKED> & column,
                               const int firstY) {
      // coordinates of the first coefficient to be loaded
      const int firstX = blockIdx.x * WIN_SIZE_X + colIndex;

      // offset of the column with index 'colIndex' in the transform buffer
      column.offset = buffer.getColumnOffset(colIndex);

      if(blockIdx.y == 0) {
        // topmost block - apply mirroring rules when loading first 7 rows
        column.loader.init(sizeX, sizeY, firstX, firstY);

        // load pixels in mirrored way
        buffer[column.offset + 3 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 4 * STRIDE] =
        buffer[column.offset + 2 * STRIDE] = column.loader.loadHighFrom(input);
        buffer[column.offset + 5 * STRIDE] =
        buffer[column.offset + 1 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 6 * STRIDE] = 
        buffer[column.offset + 0 * STRIDE] = column.loader.loadHighFrom(input);
      } else {
        // non-topmost row - regular loading:
        column.loader.init(sizeX, sizeY, firstX, firstY - 3);
        buffer[column.offset + 0 * STRIDE] = column.loader.loadHighFrom(input);
        buffer[column.offset + 1 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 2 * STRIDE] = column.loader.loadHighFrom(input);
        buffer[column.offset + 3 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 4 * STRIDE] = column.loader.loadHighFrom(input);
        buffer[column.offset + 5 * STRIDE] = column.loader.loadLowFrom(input);
        buffer[column.offset + 6 * STRIDE] = column.loader.loadHighFrom(input);
      }
      // Now, the next coefficient, which will be loaded by loader, is #4.
    }


    /// Using given loader, it loads another WIN_SIZE_Y coefficients
    /// into specified column.
    /// @tparam CHECKED  true if there are near image boundaries
    /// @param col       info about loaded column
    /// @param input     buffer with input coefficients
    template <bool CHECKED>
    inline __device__ void loadWindowIntoColumn(RDWT97Column<CHECKED> & col,
                                                const float * const input) {
      for(int i = 7; i < (7 + WIN_SIZE_Y); i += 2) {
        buffer[col.offset + i * STRIDE] = col.loader.loadLowFrom(input);
        buffer[col.offset + (i + 1) * STRIDE] = col.loader.loadHighFrom(input);
      }
    }


    /// Actual GPU 9/7 RDWT sliding window lifting schema implementation.
    /// @tparam CHECKED_LOADS   true if loader should check boundaries
    /// @tparam CHECKED_WRITES  true if boundaries should be taken into account
    ///                         when writing into output buffer
    /// @param in        input image (9/7 transformed coefficients)
    /// @param out       output buffer (for reverse transformed image)
    /// @param sizeX     width of the output image 
    /// @param sizeY     height of the output image
    /// @param winSteps  number of steps of sliding window
    template <bool CHECKED_LOADS, bool CHECKED_WRITES>
    __device__ void transform(const float * const in, float * const out,
                              const int sizeX, const int sizeY,
                              const int winSteps) {
      // info about one main column and one boundary column
      RDWT97Column<CHECKED_LOADS> column;
      RDWT97Column<CHECKED_LOADS> boundaryColumn;

      // index of first image row to be transformed
      const int firstY = blockIdx.y * WIN_SIZE_Y * winSteps;

      // initialize boundary columns
      boundaryColumn.clear();
      if(threadIdx.x < 7) {
        // each thread among first 7 ones gets index of one of boundary columns
        const int colId = threadIdx.x + ((threadIdx.x < 4) ? WIN_SIZE_X : -7);

        // Thread initializes offset of the boundary column (in shared  
        // buffer), first 7 pixels of the column and a loader for this column.
        initColumn(colId, in, sizeX, sizeY, boundaryColumn, firstY);
      }

      // All threads initialize central columns.
      initColumn(parityIdx<WIN_SIZE_X>(), in, sizeX, sizeY, column, firstY);

      // horizontally transform first 7 rows
      horizontalRDWT97(7, 0);

      // writer of output pixels - initialize it
      const int outputX = blockIdx.x * WIN_SIZE_X + threadIdx.x;
      VerticalDWTPixelWriter<float, CHECKED_WRITES> writer;
      writer.init(sizeX, sizeY, outputX, firstY);

      // offset of column (in transform buffer) saved by this thread
      const int outColumnOffset = buffer.getColumnOffset(threadIdx.x);

      // (Each iteration assumes that first 7 rows of transform buffer are 
      // already loaded with horizontally transformed pixels.)
      for(int w = 0; w < winSteps; w++) {
        // Load another WIN_SIZE_Y lines of this thread's column
        // into the transform buffer.
        loadWindowIntoColumn(column, in);

        // possibly load boundary columns
        if(threadIdx.x < 7) {
          loadWindowIntoColumn(boundaryColumn, in);
        }

        // horizontally transform all newly loaded lines
        horizontalRDWT97(WIN_SIZE_Y, 7);

        // Using 7 registers, remember current values of last 7 rows 
        // of transform buffer. These rows are transformed horizontally 
        // only and will be used in next iteration.
        float last7Lines[7];
        for(int i = 0; i < 7; i++) {
          last7Lines[i] = buffer[outColumnOffset + (WIN_SIZE_Y + i) * STRIDE];
        }

        // vertically transform all central columns
        buffer.scaleVertical(scale97Div, scale97Mul, outColumnOffset,
                             WIN_SIZE_Y + 7, 0);
        buffer.forEachVerticalOdd(outColumnOffset, AddScaledSum(r97update2));
        buffer.forEachVerticalEven(outColumnOffset, AddScaledSum(r97predict2));
        buffer.forEachVerticalOdd(outColumnOffset, AddScaledSum(r97update1));
        buffer.forEachVerticalEven(outColumnOffset, AddScaledSum(r97Predict1));

        // Save all results of current window. Results are in transform buffer
        // at rows from #3 to #(3 + WIN_SIZE_Y). Other rows are invalid now.
        // (They only served as a boundary for vertical RDWT.)
        for(int i = 3; i < (3 + WIN_SIZE_Y); i++) {
          writer.writeInto(out, buffer[outColumnOffset + i * STRIDE]);
        }

        // Use last 7 remembered lines as first 7 lines for next iteration.
        // As expected, these lines are already horizontally transformed.
        for(int i = 0; i < 7; i++) {
          buffer[outColumnOffset + i * STRIDE] = last7Lines[i];
        }

        // Wait for all writing threads before proceeding to loading new
        // coeficients in next iteration. (Not to overwrite those which
        // are not written yet.)
        __syncthreads();
      }
    }


  public:
    /// Main GPU 9/7 RDWT entry point.
    /// @param in     input image (9/7 transformed coefficients)
    /// @param out    output buffer (for reverse transformed image)
    /// @param sizeX  width of the output image 
    /// @param sizeY  height of the output image
    __device__ static void run(const float * const input, float * const output,
                               const int sx, const int sy, const int steps) {
      // prepare instance with buffer in shared memory
      __shared__ RDWT97<WIN_SIZE_X, WIN_SIZE_Y> rdwt97;
      
      // Compute limits of this threadblock's block of pixels and use them to
      // determine, whether this threadblock will have to deal with boundary.
      // (3 in next expressions is for radius of impulse response of 9/7 RDWT.)
      const int maxX = (blockIdx.x + 1) * WIN_SIZE_X + 3;
      const int maxY = (blockIdx.y + 1) * WIN_SIZE_Y * steps + 3;
      const bool atRightBoudary = maxX >= sx;
      const bool atBottomBoudary = maxY >= sy;

      // Select specialized version of code according to distance of this
      // threadblock's pixels from image boundary.
      if(atBottomBoudary) {
        // near bottom boundary => check both writing and reading
        rdwt97.transform<true, true>(input, output, sx, sy, steps);
      } else if(atRightBoudary) {
        // near right boundary only => check writing only
        rdwt97.transform<false, true>(input, output, sx, sy, steps);
      } else {
        // no nearby boundary => check nothing
        rdwt97.transform<false, false>(input, output, sx, sy, steps);
      }
    }
    
  }; // end of class RDWT97
  
    
  
  /// Main GPU 9/7 RDWT entry point.
  /// @param in     input image (9/7 transformed coefficients)
  /// @param out    output buffer (for reverse transformed image)
  /// @param sizeX  width of the output image 
  /// @param sizeY  height of the output image
  template <int WIN_SX, int WIN_SY>
  __launch_bounds__(WIN_SX, CTMIN(SHM_SIZE/sizeof(RDWT97<WIN_SX, WIN_SY>), 8))
  __global__ void rdwt97Kernel(const float * const in, float * const out,
                               const int sx, const int sy, const int steps) {
    RDWT97<WIN_SX, WIN_SY>::run(in, out, sx, sy, steps);
  }
  
  
  
  /// Only computes optimal number of sliding window steps, 
  /// number of threadblocks and then lanches the 9/7 RDWT kernel.
  /// @tparam WIN_SX  width of sliding window
  /// @tparam WIN_SY  height of sliding window
  /// @param in       input image
  /// @param out      output buffer
  /// @param sx       width of the input image 
  /// @param sy       height of the input image
  template <int WIN_SX, int WIN_SY>
  void launchRDWT97Kernel (float * in, float * out, int sx, int sy) {
    // compute optimal number of steps of each sliding window
    const int steps = divRndUp(sy, 15 * WIN_SY);
    
    // prepare grid size
    dim3 gSize(divRndUp(sx, WIN_SX), divRndUp(sy, WIN_SY * steps));
    
    // finally launch kernel
    PERF_BEGIN
    rdwt97Kernel<WIN_SX, WIN_SY><<<gSize, WIN_SX>>>(in, out, sx, sy, steps);
    PERF_END("        RDWT97", sx, sy)
    CudaDWTTester::checkLastKernelCall("RDWT 9/7 kernel");
  }
  
  
  
  /// Reverse 9/7 2D DWT. See common rules (dwt.h) for more details.
  /// @param in      Input DWT coefficients. Format described in common rules.
  ///                Will not be preserved (will be overwritten).
  /// @param out     output buffer on GPU - will contain original image
  ///                in normalized range [-0.5, 0.5].
  /// @param sizeX   width of input image (in pixels)
  /// @param sizeY   height of input image (in pixels)
  /// @param levels  number of recursive DWT levels
  void rdwt97(float * in, float * out, int sizeX, int sizeY, int levels) {
    if(levels > 1) {
      // let this function recursively reverse transform deeper levels first
      const int llSizeX = divRndUp(sizeX, 2);
      const int llSizeY = divRndUp(sizeY, 2);
      rdwt97(in, out, llSizeX, llSizeY, levels - 1);
      
      // copy reverse transformed LL band from output back into the input
      memCopy(in, out, llSizeX, llSizeY);
    }
    
    // select right width of kernel for the size of the image
    if(sizeX >= 960) {
      launchRDWT97Kernel<192, 8>(in, out, sizeX, sizeY);
    } else if (sizeX >= 480) {
      launchRDWT97Kernel<128, 6>(in, out, sizeX, sizeY);
    } else {
      launchRDWT97Kernel<64, 6>(in, out, sizeX, sizeY);
    }
  }
  

  
} // end of namespace dwt_cuda
