#include "hip/hip_runtime.h"
/// 
/// @file    fdwt97.cu
/// @brief   CUDA implementation of forward 9/7 2D DWT.
/// @author  Martin Jirman (207962@mail.muni.cz)
/// @date    2011-01-20 13:18
///
///
/// Copyright (c) 2011 Martin Jirman
/// All rights reserved.
/// 
/// Redistribution and use in source and binary forms, with or without
/// modification, are permitted provided that the following conditions are met:
/// 
///     * Redistributions of source code must retain the above copyright
///       notice, this list of conditions and the following disclaimer.
///     * Redistributions in binary form must reproduce the above copyright
///       notice, this list of conditions and the following disclaimer in the
///       documentation and/or other materials provided with the distribution.
/// 
/// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
/// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
/// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
/// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
/// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
/// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
/// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
/// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
/// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
/// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
/// POSSIBILITY OF SUCH DAMAGE.
///


#include "common.h"
#include "transform_buffer.h"
#include "io.h"


namespace dwt_cuda {

 
  
  /// Wraps a buffer and methods for computing 9/7 FDWT with sliding window
  /// of specified size. Template arguments specify this size.
  /// @tparam WIN_SIZE_X  width of sliding window
  /// @tparam WIN_SIZE_Y  height of sliding window
  template <int WIN_SIZE_X, int WIN_SIZE_Y>
  class FDWT97 {
  private:
    /// Type of shared memory buffer used for 9/7 DWT.
    typedef TransformBuffer<float, WIN_SIZE_X, WIN_SIZE_Y + 7, 4> FDWT97Buffer;

    /// Actual shared buffer used for forward 9/7 DWT.
    FDWT97Buffer buffer;

    /// Difference of indices of two vertically neighboring items in buffer.
    enum { STRIDE = FDWT97Buffer::VERTICAL_STRIDE };


    /// One thread's info about loading input image
    /// @tparam CHECKED  true if loader should check for image boundaries
    template <bool CHECKED>
    struct FDWT97ColumnLoadingInfo {
      /// Loader of pixels from some input image.
      VerticalDWTPixelLoader<float, CHECKED> loader;  
      
      /// Offset of column loaded by loader. (Offset in shared buffer.)
      int offset;
    };


    /// Horizontal 9/7 FDWT on specified lines of transform buffer.
    /// @param lines      number of lines to be transformed
    /// @param firstLine  index of the first line to be transformed
    __device__ void horizontalFDWT97(const int lines, const int firstLine) {
      __syncthreads();
      buffer.forEachHorizontalOdd(firstLine, lines, AddScaledSum(f97Predict1));
      __syncthreads();
      buffer.forEachHorizontalEven(firstLine, lines, AddScaledSum(f97Update1));
      __syncthreads();
      buffer.forEachHorizontalOdd(firstLine, lines, AddScaledSum(f97Predict2));
      __syncthreads();
      buffer.forEachHorizontalEven(firstLine, lines, AddScaledSum(f97Update2));
      __syncthreads();
      buffer.scaleHorizontal(scale97Div, scale97Mul, firstLine, lines);
      __syncthreads();
    }


    /// Initializes one column of shared transform buffer with 7 input pixels.
    /// Those 7 pixels will not be transformed. Also initializes given loader.
    /// @tparam CHECKED     true if loader should check for image boundaries
    /// @param column       (uninitialized) object for loading input pixels
    /// @param columnIndex  index (not offset!) of the column to be loaded
    ///                     (relative to threadblock's first column)
    /// @param input        pointer to input image in GPU memory
    /// @param sizeX        width of the input image
    /// @param sizeY        height of the input image
    /// @param firstY       index of first row to be loaded from image
    template <bool CHECKED>
    __device__ void initColumn(FDWT97ColumnLoadingInfo<CHECKED> & column,
                              const int columnIndex, const float * const input, 
                              const int sizeX, const int sizeY,
                              const int firstY) {
      // get offset of the column with index 'columnIndex'
      column.offset = buffer.getColumnOffset(columnIndex);

      // x-coordinate of the first pixel to be loaded by given loader
      const int firstX = blockIdx.x * WIN_SIZE_X + columnIndex;
      
      if(blockIdx.y == 0) {
        // topmost block - apply mirroring rules when loading first 7 rows
        column.loader.init(sizeX, sizeY, firstX, firstY);

        // load pixels in mirrored way
        buffer[column.offset + 4 * STRIDE] = column.loader.loadFrom(input);
        buffer[column.offset + 3 * STRIDE] =
        buffer[column.offset + 5 * STRIDE] = column.loader.loadFrom(input);
        buffer[column.offset + 2 * STRIDE] =
        buffer[column.offset + 6 * STRIDE] = column.loader.loadFrom(input);
        buffer[column.offset + 1 * STRIDE] = column.loader.loadFrom(input);
        buffer[column.offset + 0 * STRIDE] = column.loader.loadFrom(input);

        // reinitialize loader to start with pixel #3 again
        column.loader.init(sizeX, sizeY, firstX, firstY + 3);
      } else {
        // non-topmost row - regular loading:
        column.loader.init(sizeX, sizeY, firstX, firstY - 4);

        // load 7 rows into the transform buffer
        for(int i = 0; i < 7; i++) {
          buffer[column.offset + i * STRIDE] = column.loader.loadFrom(input);
        }
      }
      // Now, the next pixel, which will be loaded by loader, is pixel #3.
    }


    /// Loads another WIN_SIZE_Y pixels into given column using given loader.
    /// @tparam CHECKED  true if loader should check for image boundaries
    /// @param input     input image to load from
    /// @param column    loader and offset of loaded column in shared buffer
    template <bool CHECKED>
    inline __device__ void loadWindowIntoColumn(const float * const input,
                                  FDWT97ColumnLoadingInfo<CHECKED> & column) {
      for(int i = 7; i < (7 + WIN_SIZE_Y); i++) {
        buffer[column.offset + i * STRIDE] = column.loader.loadFrom(input);
      }
    }


    /// Main GPU 9/7 FDWT entry point.
    /// @tparam CHECK_LOADS   true if boundaries should be checked when loading
    /// @tparam CHECK_WRITES  true if boundaries should be checked when writing
    /// @param in        input image
    /// @param out       output buffer
    /// @param sizeX     width of the input image 
    /// @param sizeY     height of the input image
    /// @param winSteps  number of steps of sliding window
    template <bool CHECK_LOADS, bool CHECK_WRITES>
    __device__ void transform(const float * const in, float * const out,
                              const int sizeX, const int sizeY,
                              const int winSteps) {
      // info about columns loaded by this thread: one main column and possibly
      // one boundary column. (Only some threads load some boundary column.)
      FDWT97ColumnLoadingInfo<CHECK_LOADS> loadedColumn;
      FDWT97ColumnLoadingInfo<CHECK_LOADS> boundaryColumn;

      // Initialize first 7 lines of transform buffer.
      const int firstY = blockIdx.y * WIN_SIZE_Y * winSteps;
      initColumn(loadedColumn, threadIdx.x, in, sizeX, sizeY, firstY);

      // Some threads initialize boundary columns.
      boundaryColumn.offset = 0;
      boundaryColumn.loader.clear();
      if(threadIdx.x < 7) {
        // each thread among first 7 ones gets index of one of boundary columns
        const int colId = threadIdx.x + ((threadIdx.x < 3) ? WIN_SIZE_X : -7);

        // Thread initializes offset of the boundary column (in shared buffer),
        // first 7 pixels of the column and a loader for this column.
        initColumn(boundaryColumn, colId, in, sizeX, sizeY, firstY);
      }

      // horizontally transform first 7 rows in all columns
      horizontalFDWT97(7, 0);

      // Index of column handled by this thread. (First half of threads handle
      // even columns and others handle odd columns.)
      const int outColumnIndex = parityIdx<WIN_SIZE_X>();

      // writer of output linear bands - initialize it
      const int firstX = blockIdx.x * WIN_SIZE_X + outColumnIndex;
      VerticalDWTBandWriter<float, CHECK_WRITES> writer;
      writer.init(sizeX, sizeY, firstX, firstY);

      // transform buffer offset of column transformed and saved by this thread
      const int outColumnOffset = buffer.getColumnOffset(outColumnIndex);

      // (Each iteration of this loop assumes that first 7 rows of transform 
      // buffer are already loaded with horizontally transformed coefficients.)
      for(int w = 0; w < winSteps; w++) {
        // Load another WIN_SIZE_Y lines of thread's column into the buffer.
        loadWindowIntoColumn(in, loadedColumn);

        // some threads also load boundary columns
        if(threadIdx.x < 7) {
          loadWindowIntoColumn(in, boundaryColumn);
        }

        // horizontally transform all newly loaded lines
        horizontalFDWT97(WIN_SIZE_Y, 7);

        // Using 7 registers, remember current values of last 7 rows of
        // transform buffer. These rows are transformed horizontally only 
        // and will be used in next iteration.
        float last7Lines[7];
        for(int i = 0; i < 7; i++) {
          last7Lines[i] = buffer[outColumnOffset + (WIN_SIZE_Y + i) * STRIDE];
        }

        // vertically transform all central columns (do not scale yet)
        buffer.forEachVerticalOdd(outColumnOffset, AddScaledSum(f97Predict1));
        buffer.forEachVerticalEven(outColumnOffset, AddScaledSum(f97Update1));
        buffer.forEachVerticalOdd(outColumnOffset, AddScaledSum(f97Predict2));
        buffer.forEachVerticalEven(outColumnOffset, AddScaledSum(f97Update2));

        // Save all results of current window. Results are in transform buffer
        // at rows from #4 to #(4 + WIN_SIZE_Y). Other rows are invalid now.
        // (They only served as a boundary for vertical FDWT.)
        for(int i = 4; i < (4 + WIN_SIZE_Y); i += 2) {
          const int index = outColumnOffset + i * STRIDE;
          // Write low coefficients from column into low band ...
          writer.writeLowInto(out, buffer[index] * scale97Div);
          // ... and high coeficients into the high band.
          writer.writeHighInto(out, buffer[index + STRIDE] * scale97Mul);
        }

        // Use last 7 remembered lines as first 7 lines for next iteration.
        // As expected, these lines are already horizontally transformed.
        for(int i = 0; i < 7; i++) {
          buffer[outColumnOffset + i * STRIDE] = last7Lines[i];
        }

        // Wait for all writing threads before proceeding to loading new
        // pixels in next iteration. (Not to overwrite those which
        // are not written yet.)
        __syncthreads();
      }
    }
    
    
  public:
    /// Runs one of specialized variants of 9/7 FDWT according to distance of
    /// processed pixels to image boudnary. Some variants do not check for 
    /// boudnary and thus are slightly faster.
    /// @param in     input image
    /// @param out    output buffer
    /// @param sx     width of the input image 
    /// @param sy     height of the input image
    /// @param steps  number of steps of sliding window
    __device__ static void run(const float * const input, float * const output,
                               const int sx, const int sy, const int steps) {
      // object with transform buffer in shared memory
      __shared__ FDWT97<WIN_SIZE_X, WIN_SIZE_Y> fdwt97;

      // Compute limits of this threadblock's block of pixels and use them to
      // determine, whether this threadblock will have to deal with boundary.
      // (3 in next expressions is for radius of impulse response of 9/7 FDWT.)
      const int maxX = (blockIdx.x + 1) * WIN_SIZE_X + 3;
      const int maxY = (blockIdx.y + 1) * WIN_SIZE_Y * steps + 3;
      const bool atRightBoudary = maxX >= sx;
      const bool atBottomBoudary = maxY >= sy;

      // Select specialized version of code according to distance of this
      // threadblock's pixels from image boundary.
      if(atBottomBoudary) {
        // near bottom boundary => check both writing and reading
        fdwt97.transform<true, true>(input, output, sx, sy, steps);
      } else if(atRightBoudary) {
        // near right boundary only => check writing only
        fdwt97.transform<false, true>(input, output, sx, sy, steps);
      } else {
        // no nearby boundary => check nothing
        fdwt97.transform<false, false>(input, output, sx, sy, steps);
      }
    }
    
  }; // end of class FDWT97
  
  
    
  /// Main GPU 9/7 FDWT entry point.
  /// @param input   input image
  /// @parma output  output buffer
  /// @param sx      width of the input image 
  /// @param sy      height of the input image
  /// @param steps   number of steps of sliding window
  template <int WIN_SX, int WIN_SY>
  __launch_bounds__(WIN_SX, CTMIN(SHM_SIZE/sizeof(FDWT97<WIN_SX, WIN_SY>), 8))
  __global__ void fdwt97Kernel(const float * const input, float * const output,
                               const int sx, const int sy, const int steps) {
    // Excuse me, dear reader of this code - this call have to be here. If you
    // try to simply put contents of following method right here, CUDA compiler
    // (version 3.2) will spit tons of nonsense messy errors ...
    // Hope they will not break it even more in future releases.
    FDWT97<WIN_SX, WIN_SY>::run(input, output, sx, sy, steps);
  }

  
  
  /// Only computes optimal number of sliding window steps, 
  /// number of threadblocks and then lanches the 9/7 FDWT kernel.
  /// @tparam WIN_SX  width of sliding window
  /// @tparam WIN_SY  height of sliding window
  /// @param in       input image
  /// @param out      output buffer
  /// @param sx       width of the input image 
  /// @param sy       height of the input image
  template <int WIN_SX, int WIN_SY>
  void launchFDWT97Kernel (float * in, float * out, int sx, int sy) {
    // compute optimal number of steps of each sliding window
    const int steps = divRndUp(sy, 15 * WIN_SY);
    
    // prepare grid size
    dim3 gSize(divRndUp(sx, WIN_SX), divRndUp(sy, WIN_SY * steps));
    
    // run kernel, possibly measure time and finally check the call
    PERF_BEGIN
    fdwt97Kernel<WIN_SX, WIN_SY><<<gSize, WIN_SX>>>(in, out, sx, sy, steps);
    PERF_END("        FDWT97", sx, sy)
    CudaDWTTester::checkLastKernelCall("FDWT 9/7 kernel");
  }
  
  
  
  /// Forward 9/7 2D DWT. See common rules (dwt.h) for more details.
  /// @param in      Input DWT coefficients. Should be normalized (in range 
  ///                [-0.5, 0.5]). Will not be preserved (will be overwritten).
  /// @param out     output buffer on GPU - format specified in common rules
  /// @param sizeX   width of input image (in pixels)
  /// @param sizeY   height of input image (in pixels)
  /// @param levels  number of recursive DWT levels
  void fdwt97(float * in, float * out, int sizeX, int sizeY, int levels) {
    // select right width of kernel for the size of the image
    if(sizeX >= 960) {
      launchFDWT97Kernel<192, 8>(in, out, sizeX, sizeY);
    } else if (sizeX >= 480) {
      launchFDWT97Kernel<128, 6>(in, out, sizeX, sizeY);
    } else {
      launchFDWT97Kernel<64, 6>(in, out, sizeX, sizeY);
    }
    
    // if this was not the last level, continue recursively with other levels
    if(levels > 1) {
      // copy output's LL band back into input buffer
      const int llSizeX = divRndUp(sizeX, 2);
      const int llSizeY = divRndUp(sizeY, 2);
      memCopy(in, out, llSizeX, llSizeY);
      
      // run remaining levels of FDWT
      fdwt97(in, out, llSizeX, llSizeY, levels - 1);
    }
  }
  
  

} // end of namespace dwt_cuda
