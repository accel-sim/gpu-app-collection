#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2009, Jiri Matela
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */
 
#include <unistd.h>
#include <error.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <assert.h>

#include "components.h"
#include "common.h"

#define THREADS 256

/* Store 3 RGB float components */
__device__ void storeComponents(float *d_r, float *d_g, float *d_b, float r, float g, float b, int pos)
{
    d_r[pos] = (r/255.0f) - 0.5f;
    d_g[pos] = (g/255.0f) - 0.5f;
    d_b[pos] = (b/255.0f) - 0.5f;
}

/* Store 3 RGB intege components */
__device__ void storeComponents(int *d_r, int *d_g, int *d_b, int r, int g, int b, int pos)
{
    d_r[pos] = r - 128;
    d_g[pos] = g - 128;
    d_b[pos] = b - 128;
} 

/* Store float component */
__device__ void storeComponent(float *d_c, float c, int pos)
{
    d_c[pos] = (c/255.0f) - 0.5f;
}

/* Store integer component */
__device__ void storeComponent(int *d_c, int c, int pos)
{
    d_c[pos] = c - 128;
}

/* Copy img src data into three separated component buffers */
template<typename T>
__global__ void c_CopySrcToComponents(T *d_r, T *d_g, T *d_b, 
                                  unsigned char * d_src, 
                                  int pixels)
{
    int x  = threadIdx.x;
    int gX = blockDim.x*blockIdx.x;

    __shared__ unsigned char sData[THREADS*3];

    /* Copy data to shared mem by 4bytes 
       other checks are not necessary, since 
       d_src buffer is aligned to sharedDataSize */
    if ( (x*4) < THREADS*3 ) {
        float *s = (float *)d_src;
        float *d = (float *)sData;
        d[x] = s[((gX*3)>>2) + x];
    }
    __syncthreads();

    T r, g, b;

    int offset = x*3;
    r = (T)(sData[offset]);
    g = (T)(sData[offset+1]);
    b = (T)(sData[offset+2]);

    int globalOutputPosition = gX + x;
    if (globalOutputPosition < pixels) {
        storeComponents(d_r, d_g, d_b, r, g, b, globalOutputPosition);
    }
}

/* Copy img src data into three separated component buffers */
template<typename T>
__global__ void c_CopySrcToComponent(T *d_c, unsigned char * d_src, int pixels)
{
    int x  = threadIdx.x;
    int gX = blockDim.x*blockIdx.x;

    __shared__ unsigned char sData[THREADS];

    /* Copy data to shared mem by 4bytes 
       other checks are not necessary, since 
       d_src buffer is aligned to sharedDataSize */
    if ( (x*4) < THREADS) {
        float *s = (float *)d_src;
        float *d = (float *)sData;
        d[x] = s[(gX>>2) + x];
    }
    __syncthreads();

    T c;

    c = (T)(sData[x]);

    int globalOutputPosition = gX + x;
    if (globalOutputPosition < pixels) {
        storeComponent(d_c, c, globalOutputPosition);
    }
}


/* Separate compoents of 8bit RGB source image */
template<typename T>
void rgbToComponents(T *d_r, T *d_g, T *d_b, unsigned char * src, int width, int height)
{
    unsigned char * d_src;
    int pixels      = width*height;
    int alignedSize =  DIVANDRND(width*height, THREADS) * THREADS * 3; //aligned to thread block size -- THREADS

    /* Alloc d_src buffer */
    hipMalloc((void **)&d_src, alignedSize);
    cudaCheckAsyncError("Cuda malloc")
    hipMemset(d_src, 0, alignedSize);

    /* Copy data to device */
    hipMemcpy(d_src, src, pixels*3, hipMemcpyHostToDevice);
    cudaCheckError("Copy data to device")

    /* Kernel */
    dim3 threads(THREADS);
    dim3 grid(alignedSize/(THREADS*3));
    assert(alignedSize%(THREADS*3) == 0);
    c_CopySrcToComponents<<<grid, threads>>>(d_r, d_g, d_b, d_src, pixels);
    cudaCheckAsyncError("CopySrcToComponents kernel")

    /* Free Memory */
    hipFree(d_src);
    cudaCheckAsyncError("Free memory")
}
template void rgbToComponents<float>(float *d_r, float *d_g, float *d_b, unsigned char * src, int width, int height);
template void rgbToComponents<int>(int *d_r, int *d_g, int *d_b, unsigned char * src, int width, int height);


/* Copy a 8bit source image data into a color compoment of type T */
template<typename T>
void bwToComponent(T *d_c, unsigned char * src, int width, int height)
{
    unsigned char * d_src;
    int pixels      = width*height;
    int alignedSize =  DIVANDRND(pixels, THREADS) * THREADS; //aligned to thread block size -- THREADS

    /* Alloc d_src buffer */
    hipMalloc((void **)&d_src, alignedSize);
    cudaCheckAsyncError("Cuda malloc")
    hipMemset(d_src, 0, alignedSize);

    /* Copy data to device */
    hipMemcpy(d_src, src, pixels, hipMemcpyHostToDevice);
    cudaCheckError("Copy data to device")

    /* Kernel */
    dim3 threads(THREADS);
    dim3 grid(alignedSize/(THREADS));
    assert(alignedSize%(THREADS) == 0);
    c_CopySrcToComponent<<<grid, threads>>>(d_c, d_src, pixels);
    cudaCheckAsyncError("CopySrcToComponent kernel")

    /* Free Memory */
    hipFree(d_src);
    cudaCheckAsyncError("Free memory")
}

template void bwToComponent<float>(float *d_c, unsigned char *src, int width, int height);
template void bwToComponent<int>(int *d_c, unsigned char *src, int width, int height);
