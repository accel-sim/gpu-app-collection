#include "hip/hip_runtime.h"
#include "track_ellipse_kernel.h"
#include "misc_math.h"
#include <cutil.h>

// Constants used in the MGVF computation
#define ONE_OVER_PI (1.0 / PI)
#define MU 0.5
#define LAMBDA (8.0 * MU + 1.0)


// Host and device arrays to hold device pointers to input matrices
float **host_I_array, **host_IMGVF_array;
float **device_I_array, **device_IMGVF_array;
// Host and device arrays to hold sizes of input matrices
int *host_m_array, *host_n_array;
int *device_m_array, *device_n_array;

// Host array to hold matrices for all cells
// (so we can copy to and from the device in a single transfer)
float *host_I_all;
int total_mem_size;

// The number of threads per thread block
const int threads_per_block = 320;
// next_lowest_power_of_two = 2^(floor(log2(threads_per_block)))
const int next_lowest_power_of_two = 256;


// Regularized version of the Heaviside step function:
// He(x) = (atan(x) / pi) + 0.5
__device__ float heaviside(float x) {
	return (atan(x) * ONE_OVER_PI) + 0.5;

	// A simpler, faster approximation of the Heaviside function
	/* float out = 0.0;
	if (x > -0.0001) out = 0.5;
	if (x >  0.0001) out = 1.0;
	return out; */
}


// Kernel to compute the Motion Gradient Vector Field (MGVF) matrix for multiple cells
__global__ void IMGVF_kernel(float **IMGVF_array, float **I_array, int *m_array, int *n_array,
							 float vx, float vy, float e, int max_iterations, float cutoff) {
	
	// Shared copy of the matrix being computed
	__shared__ float IMGVF[41 * 81];
	
	// Shared buffer used for two purposes:
	// 1) To temporarily store newly computed matrix values so that only
	//    values from the previous iteration are used in the computation.
	// 2) To store partial sums during the tree reduction which is performed
	//    at the end of each iteration to determine if the computation has converged.
	__shared__ float buffer[threads_per_block];
	
	// Figure out which cell this thread block is working on
	int cell_num = blockIdx.x;
	
	// Get pointers to current cell's input image and inital matrix
	float *IMGVF_global = IMGVF_array[cell_num];
	float *I = I_array[cell_num];
	
	// Get current cell's matrix dimensions
	int m = m_array[cell_num];
	int n = n_array[cell_num];
	
	// Compute the number of virtual thread blocks
	int max = (m * n + threads_per_block - 1) / threads_per_block;
	
	// Load the initial IMGVF matrix into shared memory
	int thread_id = threadIdx.x, thread_block, i, j;
	for (thread_block = 0; thread_block < max; thread_block++) {
		int offset = thread_block * threads_per_block;
		i = (thread_id + offset) / n;
		j = (thread_id + offset) % n;
		if (i < m) IMGVF[(i * n) + j] = IMGVF_global[(i * n) + j];
	}
	__syncthreads();
	
	// Set the converged flag to false
	__shared__ int cell_converged;
	if (threadIdx.x == 0) cell_converged = 0;
	__syncthreads();
	
	// Constants used to iterate through virtual thread blocks
	const float one_nth = 1.f / (float) n;
	const int tid_mod = thread_id % n;
	const int tbsize_mod = threads_per_block % n;
	
	// Constant used in the computation of Heaviside values
	float one_over_e = 1.0 / e;
	
	// Iteratively compute the IMGVF matrix until the computation has
	//  converged or we have reached the maximum number of iterations
	int iterations = 0;
	while ((! cell_converged) && (iterations < max_iterations)) {
	
		// The total change to this thread's matrix elements in the current iteration
		float total_diff = 0.0f;
		
		int old_i = 0, old_j = 0;
		j = tid_mod - tbsize_mod;
		
		// Iterate over virtual thread blocks
		for (thread_block = 0; thread_block < max; thread_block++) {
			// Store the index of this thread's previous matrix element
			//  (used in the buffering scheme below)
			old_i = i;
			old_j = j;
			
			// Determine the index of this thread's current matrix element 
			int offset = thread_block * threads_per_block;
			i = (thread_id + offset) * one_nth;
			j += tbsize_mod;
			if (j >= n) j -= n;
			
			float new_val = 0.0, old_val = 0.0;
			
			// Make sure the thread has not gone off the end of the matrix
			if (i < m) {
				// Compute neighboring matrix element indices
				int rowU = (i == 0) ? 0 : i - 1;
				int rowD = (i == m - 1) ? m - 1 : i + 1;
				int colL = (j == 0) ? 0 : j - 1;
				int colR = (j == n - 1) ? n - 1 : j + 1;
				
				// Compute the difference between the matrix element and its eight neighbors
				old_val = IMGVF[(i * n) + j];
				float U  = IMGVF[(rowU * n) + j   ] - old_val;
				float D  = IMGVF[(rowD * n) + j   ] - old_val;
				float L  = IMGVF[(i    * n) + colL] - old_val;
				float R  = IMGVF[(i    * n) + colR] - old_val;
				float UR = IMGVF[(rowU * n) + colR] - old_val;
				float DR = IMGVF[(rowD * n) + colR] - old_val;
				float UL = IMGVF[(rowU * n) + colL] - old_val;
				float DL = IMGVF[(rowD * n) + colL] - old_val;
				
				// Compute the regularized heaviside value for these differences
				float UHe  = heaviside((U  *       -vy)  * one_over_e);
				float DHe  = heaviside((D  *        vy)  * one_over_e);
				float LHe  = heaviside((L  *  -vx     )  * one_over_e);
				float RHe  = heaviside((R  *   vx     )  * one_over_e);
				float URHe = heaviside((UR * ( vx - vy)) * one_over_e);
				float DRHe = heaviside((DR * ( vx + vy)) * one_over_e);
				float ULHe = heaviside((UL * (-vx - vy)) * one_over_e);
				float DLHe = heaviside((DL * (-vx + vy)) * one_over_e);
				
				// Update the IMGVF value in two steps:
				// 1) Compute IMGVF += (mu / lambda)(UHe .*U  + DHe .*D  + LHe .*L  + RHe .*R +
				//                                   URHe.*UR + DRHe.*DR + ULHe.*UL + DLHe.*DL);
				new_val = old_val + (MU / LAMBDA) * (UHe  * U  + DHe  * D  + LHe  * L  + RHe  * R +
													 URHe * UR + DRHe * DR + ULHe * UL + DLHe * DL);
				// 2) Compute IMGVF -= (1 / lambda)(I .* (IMGVF - I))
				float vI = I[(i * n) + j];
				new_val -= ((1.0 / LAMBDA) * vI * (new_val - vI));
			}
			
			// Save the previous virtual thread block's value (if it exists)
			if (thread_block > 0) {
				offset = (thread_block - 1) * threads_per_block;
				if (old_i < m) IMGVF[(old_i * n) + old_j] = buffer[thread_id];
			}
			if (thread_block < max - 1) {
				// Write the new value to the buffer
				buffer[thread_id] = new_val;
			} else {
				// We've reached the final virtual thread block,
				//  so write directly to the matrix
				if (i < m) IMGVF[(i * n) + j] = new_val;
			}
			
			// Keep track of the total change of this thread's matrix elements
			total_diff += fabs(new_val - old_val);
			
			// We need to synchronize between virtual thread blocks to prevent
			//  threads from writing the values from the buffer to the actual
			//  IMGVF matrix too early
			__syncthreads();
		}
		
		// We need to compute the overall sum of the change at each matrix element
		//  by performing a tree reduction across the whole threadblock
		buffer[thread_id] = total_diff;
		__syncthreads();
		
		// Account for thread block sizes that are not a power of 2
		if (thread_id >= next_lowest_power_of_two) {
			buffer[thread_id - next_lowest_power_of_two] += buffer[thread_id];
		}
		__syncthreads();
		
		// Perform the tree reduction
		int th;
		for (th = next_lowest_power_of_two / 2; th > 0; th /= 2) {
			if (thread_id < th) {
				buffer[thread_id] += buffer[thread_id + th];
			}
			__syncthreads();
		}
		
		// Figure out if we have converged
		if(thread_id == 0) {
			float mean = buffer[thread_id] / (float) (m * n);
			if (mean < cutoff) {
				// We have converged, so set the appropriate flag
				cell_converged = 1;
			}
		}
		
		// We need to synchronize to ensure that all threads
		//  read the correct value of the convergence flag
		__syncthreads();
		
		// Keep track of the number of iterations we have performed
		iterations++;
	}
	
	// Save the final IMGVF matrix to global memory
	for (thread_block = 0; thread_block < max; thread_block++) {
		int offset = thread_block * threads_per_block;
		i = (thread_id + offset) / n;
		j = (thread_id + offset) % n;
		if (i < m) IMGVF_global[(i * n) + j] = IMGVF[(i * n) + j];
	}
}


// Host function that launches a CUDA kernel to compute the MGVF matrices for the specified cells
void IMGVF_cuda(MAT **I, MAT **IMGVF, double vx, double vy, double e, int max_iterations, double cutoff, int num_cells) {
	
	// Initialize the data on the GPU
	IMGVF_cuda_init(I, num_cells);
	
	// Compute the MGVF on the GPU
	IMGVF_kernel <<< num_cells, threads_per_block >>>
				( device_IMGVF_array, device_I_array, device_m_array, device_n_array,
				  (float) vx, (float) vy, (float) e, max_iterations, (float) cutoff );
	
	// Check for kernel errors
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("MGVF kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
	// Copy back the final results from the GPU
	IMGVF_cuda_cleanup(IMGVF, num_cells);
}


// Initializes data on the GPU for the MGVF kernel
void IMGVF_cuda_init(MAT **IE, int num_cells) {
	// Allocate arrays of pointers to device memory
	host_I_array = (float **) malloc(sizeof(float *) * num_cells);
	host_IMGVF_array = (float **) malloc(sizeof(float *) * num_cells);
	hipMalloc( (void**) &device_I_array, num_cells * sizeof(float *));
	hipMalloc( (void**) &device_IMGVF_array, num_cells * sizeof(float *));
	
	// Allocate arrays of memory dimensions
	host_m_array = (int *) malloc(sizeof(int) * num_cells);
	host_n_array = (int *) malloc(sizeof(int) * num_cells);
	hipMalloc( (void**) &device_m_array, num_cells * sizeof(int));
	hipMalloc( (void**) &device_n_array, num_cells * sizeof(int));
	
	// Figure out the size of all of the matrices combined
	int i, j, cell_num;
	int total_size = 0;
	for (cell_num = 0; cell_num < num_cells; cell_num++) {
		MAT *I = IE[cell_num];
		int size = I->m * I->n;
		total_size += size;
	}
	total_mem_size = total_size * sizeof(float);
	
	// Allocate host memory just once for all cells
	host_I_all = (float *) malloc(total_mem_size);
	
	// Allocate device memory just once for all cells
	float *device_I_all, *device_IMGVF_all;
	hipMalloc( (void**) &device_I_all, total_mem_size);
	hipMalloc( (void**) &device_IMGVF_all, total_mem_size);
	
	// Copy each initial matrix into the allocated host memory
	int offset = 0;
	for (cell_num = 0; cell_num < num_cells; cell_num++) {
		MAT *I = IE[cell_num];
		
		// Determine the size of the matrix
		int m = I->m, n = I->n;
		int size = m * n;
		
		// Store memory dimensions
		host_m_array[cell_num] = m;
		host_n_array[cell_num] = n;
		
		// Store pointers to allocated memory
		float *device_I = &(device_I_all[offset]);
		float *device_IMGVF = &(device_IMGVF_all[offset]);
		host_I_array[cell_num] = device_I;
		host_IMGVF_array[cell_num] = device_IMGVF;
		
		// Copy matrix I (which is also the initial IMGVF matrix) into the overall array
		for (i = 0; i < m; i++)
			for (j = 0; j < n; j++)
				host_I_all[offset + (i * n) + j] = (float) m_get_val(I, i, j);
		
		offset += size;
	}
	
	// Copy I matrices (which are also the initial IMGVF matrices) to device
	hipMemcpy(device_I_all, host_I_all, total_mem_size, hipMemcpyHostToDevice);
	hipMemcpy(device_IMGVF_all, host_I_all, total_mem_size, hipMemcpyHostToDevice);
	
	// Copy pointer arrays to device
	hipMemcpy(device_I_array, host_I_array, num_cells * sizeof(float *), hipMemcpyHostToDevice);
	hipMemcpy(device_IMGVF_array, host_IMGVF_array, num_cells * sizeof(float *), hipMemcpyHostToDevice);
	
	// Copy memory dimension arrays to device
	hipMemcpy(device_m_array, host_m_array, num_cells * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_n_array, host_n_array, num_cells * sizeof(int), hipMemcpyHostToDevice);
}


// Copies the results of the MGVF kernel back to the host
void IMGVF_cuda_cleanup(MAT **IMGVF_out_array, int num_cells) {
	// Copy the result matrices from the device to the host
	hipMemcpy(host_I_all, host_IMGVF_array[0], total_mem_size, hipMemcpyDeviceToHost);
	
	// Copy each result matrix into its appropriate host matrix
	int cell_num, offset = 0;	
	for (cell_num = 0; cell_num < num_cells; cell_num++) {
		MAT *IMGVF_out = IMGVF_out_array[cell_num];
		
		// Determine the size of the matrix
		int m = IMGVF_out->m, n = IMGVF_out->n, i, j;
		// Pack the result into the matrix
		for (i = 0; i < m; i++)
			for (j = 0; j < n; j++)
				m_set_val(IMGVF_out, i, j, (double) host_I_all[offset + (i * n) + j]);
		
		offset += (m * n);
	}
	
	// Free device memory
	hipFree(device_m_array);
	hipFree(device_n_array);
	hipFree(device_IMGVF_array);
	hipFree(device_I_array);
	hipFree(host_IMGVF_array[0]);
	hipFree(host_I_array[0]);
	
	// Free host memory
	free(host_m_array);
	free(host_n_array);
	free(host_IMGVF_array);
	free(host_I_array);
	free(host_I_all);
}
