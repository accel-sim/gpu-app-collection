#include "hip/hip_runtime.h"
#include "find_ellipse_kernel.h"
#include <cutil.h>
#include <stdio.h>


// The number of sample points in each ellipse (stencil)
#define NPOINTS 150
// The maximum radius of a sample ellipse
#define MAX_RAD 20
// The total number of sample ellipses
#define NCIRCLES 7
// The size of the structuring element used in dilation
#define STREL_SIZE (12 * 2 + 1)


// Matrix used to store the maximal GICOV score at each pixels
// Produced by the GICOV kernel and consumed by the dilation kernel
float *device_gicov;


// Constant device arrays holding the stencil parameters used by the GICOV kernel
__constant__ float c_sin_angle[NPOINTS];
__constant__ float c_cos_angle[NPOINTS];
__constant__ int c_tX[NCIRCLES * NPOINTS];
__constant__ int c_tY[NCIRCLES * NPOINTS];

// Texture references to the gradient matrices used by the GICOV kernel
texture<float, 1, hipReadModeElementType> t_grad_x;
texture<float, 1, hipReadModeElementType> t_grad_y;

// Kernel to find the maximal GICOV value at each pixel of a
//  video frame, based on the input x- and y-gradient matrices
__global__ void GICOV_kernel(int grad_m, float *gicov) {
	int i, j, k, n, x, y;
	
	// Determine this thread's pixel
	i = blockIdx.x + MAX_RAD + 2;
	j = threadIdx.x + MAX_RAD + 2;

	// Initialize the maximal GICOV score to 0
	float max_GICOV = 0.f;

	// Iterate across each stencil
	for (k = 0; k < NCIRCLES; k++) {
		// Variables used to compute the mean and variance
		//  of the gradients along the current stencil
		float sum = 0.f, M2 = 0.f, mean = 0.f;		
		
		// Iterate across each sample point in the current stencil
		for (n = 0; n < NPOINTS; n++) {
			// Determine the x- and y-coordinates of the current sample point
			y = j + c_tY[(k * NPOINTS) + n];
			x = i + c_tX[(k * NPOINTS) + n];
			
			// Compute the combined gradient value at the current sample point
			int addr = x * grad_m + y;
			float p = tex1Dfetch(t_grad_x,addr) * c_cos_angle[n] + 
					  tex1Dfetch(t_grad_y,addr) * c_sin_angle[n];
			
			// Update the running total
			sum += p;
			
			// Partially compute the variance
			float delta = p - mean;
			mean = mean + (delta / (float) (n + 1));
			M2 = M2 + (delta * (p - mean));
		}
		
		// Finish computing the mean
		mean = sum / ((float) NPOINTS);
		
		// Finish computing the variance
		float var = M2 / ((float) (NPOINTS - 1));
		
		// Keep track of the maximal GICOV value seen so far
		if (((mean * mean) / var) > max_GICOV) max_GICOV = (mean * mean) / var;
	}
	
	// Store the maximal GICOV value
	gicov[(i * grad_m) + j] = max_GICOV;
}


// Sets up and invokes the GICOV kernel and returns its output
float *GICOV_CUDA(int grad_m, int grad_n, float *host_grad_x, float *host_grad_y) {

	int MaxR = MAX_RAD + 2;

	// Allocate device memory
	unsigned int grad_mem_size = sizeof(float) * grad_m * grad_n;
	float *device_grad_x, *device_grad_y;
	hipMalloc((void**) &device_grad_x, grad_mem_size);
	hipMalloc((void**) &device_grad_y, grad_mem_size);

	// Copy the input gradients to the device
	hipMemcpy(device_grad_x, host_grad_x, grad_mem_size, hipMemcpyHostToDevice);
	hipMemcpy(device_grad_y, host_grad_y, grad_mem_size, hipMemcpyHostToDevice);
    
	// Bind the device arrays to texture references
    hipBindTexture(0, t_grad_x, device_grad_x, grad_mem_size);
    hipBindTexture(0, t_grad_y, device_grad_y, grad_mem_size);

	// Allocate & initialize device memory for result
	// (some elements are not assigned values in the kernel)
	hipMalloc((void**) &device_gicov, grad_mem_size);
	hipMemset(device_gicov, 0, grad_mem_size);

	// Setup execution parameters
	int num_blocks = grad_n - (2 * MaxR);
	int threads_per_block = grad_m - (2 * MaxR);
    
	// Execute the GICOV kernel
	GICOV_kernel <<< num_blocks, threads_per_block >>> (grad_m, device_gicov);
	
	// Check for kernel errors
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("GICOV kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy the result to the host
	float *host_gicov = (float *) malloc(grad_mem_size);
	hipMemcpy(host_gicov, device_gicov, grad_mem_size, hipMemcpyDeviceToHost);

	// Cleanup memory
	hipUnbindTexture(t_grad_x);
	hipUnbindTexture(t_grad_y);
	hipFree(device_grad_x);
	hipFree(device_grad_y);

	return host_gicov;
}


// Constant device array holding the structuring element used by the dilation kernel
__constant__ float c_strel[STREL_SIZE * STREL_SIZE];

// Texture reference to the GICOV matrix used by the dilation kernel
texture<float, 1, hipReadModeElementType> t_img;

// Kernel to compute the dilation of the GICOV matrix produced by the GICOV kernel
// Each element (i, j) of the output matrix is set equal to the maximal value in
//  the neighborhood surrounding element (i, j) in the input matrix
// Here the neighborhood is defined by the structuring element (c_strel)
__global__ void dilate_kernel(int img_m, int img_n, int strel_m, int strel_n, float *dilated) {	
	// Find the center of the structuring element
	int el_center_i = strel_m / 2;
	int el_center_j = strel_n / 2;

	// Determine this thread's location in the matrix
	int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i = thread_id % img_m;
	int j = thread_id / img_m;

	// Initialize the maximum GICOV score seen so far to zero
	float max = 0.0;

	// Iterate across the structuring element in one dimension
	int el_i, el_j, x, y;
	for(el_i = 0; el_i < strel_m; el_i++) {
		y = i - el_center_i + el_i;
		// Make sure we have not gone off the edge of the matrix
		if( (y >= 0) && (y < img_m) ) {
			// Iterate across the structuring element in the other dimension
			for(el_j = 0; el_j < strel_n; el_j++) {
				x = j - el_center_j + el_j;
				// Make sure we have not gone off the edge of the matrix
				//  and that the current structuring element value is not zero
				if( (x >= 0) &&
					(x < img_n) &&
					(c_strel[(el_i * strel_n) + el_j] != 0) ) {
						// Determine if this is maximal value seen so far
						int addr = (x * img_m) + y;
						float temp = tex1Dfetch(t_img, addr);
						if (temp > max) max = temp;
				}
			}
		}
	}
	
	// Store the maximum value found
	dilated[(i * img_n) + j] = max;
}


// Sets up and invokes the dilation kernel and returns its output
float *dilate_CUDA(int max_gicov_m, int max_gicov_n, int strel_m, int strel_n) {
	// Allocate device memory for result
	unsigned int max_gicov_mem_size = sizeof(float) * max_gicov_m * max_gicov_n;
	float* device_img_dilated;
	hipMalloc( (void**) &device_img_dilated, max_gicov_mem_size);
	
	// Bind the input matrix of GICOV values to a texture reference
	hipBindTexture(0, t_img, device_gicov, max_gicov_mem_size);
    
	// Setup execution parameters
	int num_threads = max_gicov_m * max_gicov_n;
	int threads_per_block = 176;
	int num_blocks = (int) (((float) num_threads / (float) threads_per_block) + 0.5);

	// Execute the dilation kernel
	dilate_kernel <<< num_blocks, threads_per_block >>> (max_gicov_m, max_gicov_n, strel_m, strel_n, device_img_dilated);
	
	// Check for kernel errors
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("Dilation kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Copy the result to the host
	float *host_img_dilated = (float*) malloc(max_gicov_mem_size);
	hipMemcpy(host_img_dilated, device_img_dilated, max_gicov_mem_size, hipMemcpyDeviceToHost);

	// Cleanup memory
	hipUnbindTexture(t_img);
	hipFree(device_gicov);
	hipFree(device_img_dilated);

	return host_img_dilated;
}


// Chooses the most appropriate GPU on which to execute
void select_device() {
	// Figure out how many devices exist
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	
	// Choose the device with the largest number of multiprocessors
	if (num_devices > 0) {
		int max_multiprocessors = 0, max_device = -1;
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}
	
	// The following is to remove the API initialization overhead from the runtime measurements
	hipFree(0);
}


// Transfers pre-computed constants used by the two kernels to the GPU
void transfer_constants(float *host_sin_angle, float *host_cos_angle, int *host_tX, int *host_tY, int strel_m, int strel_n, float *host_strel) {

	// Compute the sizes of the matrices
	unsigned int angle_mem_size = sizeof(float) * NPOINTS;
	unsigned int t_mem_size = sizeof(int) * NCIRCLES * NPOINTS;
	unsigned int strel_mem_size = sizeof(float) * strel_m * strel_n;

	// Copy the matrices from host memory to device constant memory
	hipMemcpyToSymbol(HIP_SYMBOL("c_sin_angle"), host_sin_angle, angle_mem_size, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("c_cos_angle"), host_cos_angle, angle_mem_size, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("c_tX"), host_tX, t_mem_size, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("c_tY"), host_tY, t_mem_size, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("c_strel"), host_strel, strel_mem_size, 0, hipMemcpyHostToDevice);
}
