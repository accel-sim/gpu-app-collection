/*

                 Copyright (c) 2010.
      Lawrence Livermore National Security, LLC.
Produced at the Lawrence Livermore National Laboratory.
                  LLNL-CODE-461231
                All rights reserved.

This file is part of LULESH, Version 1.0.
Please also read this link -- http://www.opensource.org/licenses/index.php

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the disclaimer below.

   * Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the disclaimer (as noted below)
     in the documentation and/or other materials provided with the
     distribution.

   * Neither the name of the LLNS/LLNL nor the names of its contributors
     may be used to endorse or promote products derived from this software
     without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL LAWRENCE LIVERMORE NATIONAL SECURITY, LLC,
THE U.S. DEPARTMENT OF ENERGY OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


Additional BSD Notice

1. This notice is required to be provided under our contract with the U.S.
   Department of Energy (DOE). This work was produced at Lawrence Livermore
   National Laboratory under Contract No. DE-AC52-07NA27344 with the DOE.

2. Neither the United States Government nor Lawrence Livermore National
   Security, LLC nor any of their employees, makes any warranty, express
   or implied, or assumes any liability or responsibility for the accuracy,
   completeness, or usefulness of any information, apparatus, product, or
   process disclosed, or represents that its use would not infringe
   privately-owned rights.

3. Also, reference herein to any specific commercial products, process, or
   services by trade name, trademark, manufacturer or otherwise does not
   necessarily constitute or imply its endorsement, recommendation, or
   favoring by the United States Government or Lawrence Livermore National
   Security, LLC. The views and opinions of authors expressed herein do not
   necessarily state or reflect those of the United States Government or
   Lawrence Livermore National Security, LLC, and shall not be used for
   advertising or product endorsement purposes.

*/

#include <vector>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define LULESH_SHOW_PROGRESS 1

enum { VolumeError = -1, QStopError = -2 } ;

/****************************************************/
/* Allow flexibility for arithmetic representations */
/****************************************************/

/* Could also support fixed point and interval arithmetic types */
typedef float        real4 ;
typedef double       real8 ;
typedef long double  real10 ;  /* 10 bytes on x86 */

typedef int    Index_t ; /* array subscript and loop index */
typedef real8  Real_t ;  /* floating point representation */
typedef int    Int_t ;   /* integer representation */

__host__ __device__ inline real4  SQRT(real4  arg) { return sqrtf(arg) ; }
__host__ __device__ inline real8  SQRT(real8  arg) { return sqrt(arg) ; }
__host__            inline real10 SQRT(real10 arg) { return sqrtl(arg) ; }

__host__ __device__ inline real4  CBRT(real4  arg) { return cbrtf(arg) ; }
__host__ __device__ inline real8  CBRT(real8  arg) { return cbrt(arg) ; }
__host__            inline real10 CBRT(real10 arg) { return cbrtl(arg) ; }

__host__ __device__ inline real4  FABS(real4  arg) { return fabsf(arg) ; }
__host__ __device__ inline real8  FABS(real8  arg) { return fabs(arg) ; }
__host__            inline real10 FABS(real10 arg) { return fabsl(arg) ; }

__host__ __device__ inline real4  FMAX(real4  arg1,real4  arg2) { return fmaxf(arg1,arg2) ; }
__host__ __device__ inline real8  FMAX(real8  arg1,real8  arg2) { return fmax(arg1,arg2) ; }
__host__            inline real10 FMAX(real10 arg1,real10 arg2) { return fmaxl(arg1,arg2) ; }

#define CUDA_SAFE_CALL( call) do {                                           \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
    exit(EXIT_FAILURE);                                                      \
    } } while (0)
    
#define CUDA(call) CUDA_SAFE_CALL(call)

#ifdef CUDA_SYNC_ALL
#define CUDA_DEBUGSYNC CUDA(cudaThreadSynchronize())
#else
#define CUDA_DEBUGSYNC
#endif

#define BLOCKSIZE 256

/* Given a number of bytes, nbytes, and a byte alignment, align, (e.g., 2,
 * 4, 8, or 16), return the smallest integer that is larger than nbytes and
 * a multiple of align.
 */
#define PAD_DIV(nbytes, align)  (((nbytes) + (align) - 1) / (align))
#define PAD(nbytes, align)  (PAD_DIV((nbytes),(align)) * (align))

   /* More general version of reduceInPlacePOT (this works for arbitrary
    * numThreadsPerBlock <= 1024). Again, conditionals on
    * numThreadsPerBlock are evaluated at compile time.
    */
template <class T, int numThreadsPerBlock>
__device__ void
reduceSum(T *sresult, const int threadID)
{
    /* If number of threads is not a power of two, first add the ones
       after the last power of two into the beginning. At most one of
       these conditionals will be true for a given NPOT block size. */
    if (numThreadsPerBlock > 512 && numThreadsPerBlock <= 1024)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-512)
            sresult[threadID] += sresult[threadID + 512];
    }
    
    if (numThreadsPerBlock > 256 && numThreadsPerBlock < 512)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-256)
            sresult[threadID] += sresult[threadID + 256];
    }
    
    if (numThreadsPerBlock > 128 && numThreadsPerBlock < 256)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-128)
            sresult[threadID] += sresult[threadID + 128];
    }
    
    if (numThreadsPerBlock > 64 && numThreadsPerBlock < 128)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-64)
            sresult[threadID] += sresult[threadID + 64];
    }
    
    if (numThreadsPerBlock > 32 && numThreadsPerBlock < 64)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-32)
            sresult[threadID] += sresult[threadID + 32];
    }
    
    if (numThreadsPerBlock > 16 && numThreadsPerBlock < 32)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-16)
            sresult[threadID] += sresult[threadID + 16];
    }
    
    if (numThreadsPerBlock > 8 && numThreadsPerBlock < 16)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-8)
            sresult[threadID] += sresult[threadID + 8];
    }
    
    if (numThreadsPerBlock > 4 && numThreadsPerBlock < 8)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-4)
            sresult[threadID] += sresult[threadID + 4];
    }
    
    if (numThreadsPerBlock > 2 && numThreadsPerBlock < 4)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-2)
            sresult[threadID] += sresult[threadID + 2];
    }
    
    if (numThreadsPerBlock >= 512) {
        __syncthreads();
        if (threadID < 256)
            sresult[threadID] += sresult[threadID + 256];
    }
    
    if (numThreadsPerBlock >= 256) {
        __syncthreads();
        if (threadID < 128)
            sresult[threadID] += sresult[threadID + 128];
    }
    if (numThreadsPerBlock >= 128) {
        __syncthreads();
        if (threadID < 64)
            sresult[threadID] += sresult[threadID + 64];
    }
    __syncthreads();
#ifdef _DEVICEEMU
    if (numThreadsPerBlock >= 64) {
        __syncthreads();
        if (threadID < 32)
            sresult[threadID] += sresult[threadID + 32];
    }
    if (numThreadsPerBlock >= 32) {
        __syncthreads();
        if (threadID < 16)
            sresult[threadID] += sresult[threadID + 16];
    }
    if (numThreadsPerBlock >= 16) {
        __syncthreads();
        if (threadID < 8)
            sresult[threadID] += sresult[threadID + 8];
    }
    if (numThreadsPerBlock >= 8) {
        __syncthreads();
        if (threadID < 4)
            sresult[threadID] += sresult[threadID + 4];
    }
    if (numThreadsPerBlock >= 4) {
        __syncthreads();
        if (threadID < 2)
            sresult[threadID] += sresult[threadID + 2];
    }
    if (numThreadsPerBlock >= 2) {
        __syncthreads();
        if (threadID < 1)
            sresult[threadID] += sresult[threadID + 1];
    }
#else
    if (threadID < 32) {
        volatile T *vol = sresult;
        if (numThreadsPerBlock >= 64) vol[threadID] += vol[threadID + 32];
        if (numThreadsPerBlock >= 32) vol[threadID] += vol[threadID + 16];
        if (numThreadsPerBlock >= 16) vol[threadID] += vol[threadID + 8];
        if (numThreadsPerBlock >= 8) vol[threadID] += vol[threadID + 4];
        if (numThreadsPerBlock >= 4) vol[threadID] += vol[threadID + 2];
        if (numThreadsPerBlock >= 2) vol[threadID] += vol[threadID + 1];
    }
#endif
    __syncthreads();
}

#define MINEQ(a,b) (a)=(((a)<(b))?(a):(b))

template <class T, int numThreadsPerBlock>
__device__ void
reduceMin(T *sresult, const int threadID)
{
    /* If number of threads is not a power of two, first add the ones
       after the last power of two into the beginning. At most one of
       these conditionals will be true for a given NPOT block size. */
    if (numThreadsPerBlock > 512 && numThreadsPerBlock <= 1024)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-512)
            MINEQ(sresult[threadID],sresult[threadID + 512]);
    }
    
    if (numThreadsPerBlock > 256 && numThreadsPerBlock < 512)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-256)
            MINEQ(sresult[threadID],sresult[threadID + 256]);
    }
    
    if (numThreadsPerBlock > 128 && numThreadsPerBlock < 256)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-128)
            MINEQ(sresult[threadID],sresult[threadID + 128]);
    }
    
    if (numThreadsPerBlock > 64 && numThreadsPerBlock < 128)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-64)
            MINEQ(sresult[threadID],sresult[threadID + 64]);
    }
    
    if (numThreadsPerBlock > 32 && numThreadsPerBlock < 64)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-32)
            MINEQ(sresult[threadID],sresult[threadID + 32]);
    }
    
    if (numThreadsPerBlock > 16 && numThreadsPerBlock < 32)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-16)
            MINEQ(sresult[threadID],sresult[threadID + 16]);
    }
    
    if (numThreadsPerBlock > 8 && numThreadsPerBlock < 16)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-8)
            MINEQ(sresult[threadID],sresult[threadID + 8]);
    }
    
    if (numThreadsPerBlock > 4 && numThreadsPerBlock < 8)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-4)
            MINEQ(sresult[threadID],sresult[threadID + 4]);
    }
    
    if (numThreadsPerBlock > 2 && numThreadsPerBlock < 4)
    {
        __syncthreads();
        if (threadID < numThreadsPerBlock-2)
            MINEQ(sresult[threadID],sresult[threadID + 2]);
    }
    
    if (numThreadsPerBlock >= 512) {
        __syncthreads();
        if (threadID < 256)
            MINEQ(sresult[threadID],sresult[threadID + 256]);
    }
    
    if (numThreadsPerBlock >= 256) {
        __syncthreads();
        if (threadID < 128)
            MINEQ(sresult[threadID],sresult[threadID + 128]);
    }
    if (numThreadsPerBlock >= 128) {
        __syncthreads();
        if (threadID < 64)
            MINEQ(sresult[threadID],sresult[threadID + 64]);
    }
    __syncthreads();
#ifdef _DEVICEEMU
    if (numThreadsPerBlock >= 64) {
        __syncthreads();
        if (threadID < 32)
            MINEQ(sresult[threadID],sresult[threadID + 32]);
    }
    if (numThreadsPerBlock >= 32) {
        __syncthreads();
        if (threadID < 16)
            MINEQ(sresult[threadID],sresult[threadID + 16]);
    }
    if (numThreadsPerBlock >= 16) {
        __syncthreads();
        if (threadID < 8)
            MINEQ(sresult[threadID],sresult[threadID + 8]);
    }
    if (numThreadsPerBlock >= 8) {
        __syncthreads();
        if (threadID < 4)
            MINEQ(sresult[threadID],sresult[threadID + 4]);
    }
    if (numThreadsPerBlock >= 4) {
        __syncthreads();
        if (threadID < 2)
            MINEQ(sresult[threadID],sresult[threadID + 2]);
    }
    if (numThreadsPerBlock >= 2) {
        __syncthreads();
        if (threadID < 1)
            MINEQ(sresult[threadID],sresult[threadID + 1]);
    }
#else
    if (threadID < 32) {
        volatile T *vol = sresult;
        if (numThreadsPerBlock >= 64) MINEQ(vol[threadID],vol[threadID + 32]);
        if (numThreadsPerBlock >= 32) MINEQ(vol[threadID],vol[threadID + 16]);
        if (numThreadsPerBlock >= 16) MINEQ(vol[threadID],vol[threadID + 8]);
        if (numThreadsPerBlock >= 8)  MINEQ(vol[threadID],vol[threadID + 4]);
        if (numThreadsPerBlock >= 4)  MINEQ(vol[threadID],vol[threadID + 2]);
        if (numThreadsPerBlock >= 2)  MINEQ(vol[threadID],vol[threadID + 1]);
    }
#endif
    __syncthreads();
}

void cuda_init()
{
    int deviceCount, dev;
    hipDeviceProp_t cuda_deviceProp;
    char *s;
    
    CUDA( hipGetDeviceCount(&deviceCount) );
    if (deviceCount == 0) {
        fprintf(stderr, "cuda_init(): no devices supporting CUDA.\n");
        exit(1);
    }
    if (s=getenv("CUDA_DEVICE")) dev=atoi(s);
    else dev=0;
    if ((dev < 0) || (dev > deviceCount-1)) {
        fprintf(stderr, "cuda_init(): requested device (%d) out of range [%d,%d]\n",
                dev, 0, deviceCount-1);
        exit(1);
    }
    CUDA( hipGetDeviceProperties(&cuda_deviceProp, dev) );
    if (cuda_deviceProp.major < 1) {
        fprintf(stderr, "cuda_init(): device %d does not support CUDA.\n", dev);
        exit(1);
    }
    fprintf(stderr, "setting CUDA device %d\n",dev);
    CUDA( hipSetDevice(dev) );
}

/************************************************************/
/* Allow for flexible data layout experiments by separating */
/* array interface from underlying implementation.          */
/************************************************************/

struct Mesh {

/* This first implementation allows for runnable code */
/* and is not meant to be optimal. Final implementation */
/* should separate declaration and allocation phases */
/* so that allocation can be scheduled in a cache conscious */
/* manner. */
    
    friend struct MeshGPU;
    
public:

   /**************/
   /* Allocation */
   /**************/

   void AllocateNodalPersistent(size_t size)
   {
      m_x.resize(size) ;
      m_y.resize(size) ;
      m_z.resize(size) ;

      m_xd.resize(size, Real_t(0.)) ;
      m_yd.resize(size, Real_t(0.)) ;
      m_zd.resize(size, Real_t(0.)) ;

      m_xdd.resize(size, Real_t(0.)) ;
      m_ydd.resize(size, Real_t(0.)) ;
      m_zdd.resize(size, Real_t(0.)) ;

      m_fx.resize(size) ;
      m_fy.resize(size) ;
      m_fz.resize(size) ;

      m_nodalMass.resize(size, Real_t(0.)) ;
   }

   void AllocateElemPersistent(size_t size)
   {
      m_matElemlist.resize(size) ;
      m_nodelist.resize(8*size) ;

      m_lxim.resize(size) ;
      m_lxip.resize(size) ;
      m_letam.resize(size) ;
      m_letap.resize(size) ;
      m_lzetam.resize(size) ;
      m_lzetap.resize(size) ;

      m_elemBC.resize(size) ;

      m_e.resize(size, Real_t(0.)) ;

      m_p.resize(size, Real_t(0.)) ;
      m_q.resize(size) ;
      m_ql.resize(size) ;
      m_qq.resize(size) ;

      m_v.resize(size, 1.0) ;
      m_volo.resize(size) ;
      m_delv.resize(size) ;
      m_vdov.resize(size) ;

      m_arealg.resize(size) ;
   
      m_ss.resize(size) ;

      m_elemMass.resize(size) ;
   }

   /* Temporaries should not be initialized in bulk but */
   /* this is a runnable placeholder for now */
   void AllocateElemTemporary(size_t size)
   {
      m_dxx.resize(size) ;
      m_dyy.resize(size) ;
      m_dzz.resize(size) ;

      m_delv_xi.resize(size) ;
      m_delv_eta.resize(size) ;
      m_delv_zeta.resize(size) ;

      m_delx_xi.resize(size) ;
      m_delx_eta.resize(size) ;
      m_delx_zeta.resize(size) ;

      m_vnew.resize(size) ;
   }

   void AllocateNodesets(size_t size)
   {
      m_symmX.resize(size) ;
      m_symmY.resize(size) ;
      m_symmZ.resize(size) ;
   }

   void AllocateNodeElemIndexes()
   {
     Index_t i,j,nidx;
       
       /* set up node-centered indexing of elements */
       m_nodeElemCount.resize(m_numNode);
       for (i=0;i<m_numNode;i++) m_nodeElemCount[i]=0;
       m_nodeElemCornerList.resize(m_numNode*8);
       for (i=0;i<m_numElem;i++) {
           for (j=0;j<8;j++) {
               nidx=nodelist(i,j);
               m_nodeElemCornerList[nidx+m_numNode*m_nodeElemCount[nidx]++] = i+m_numElem*j;
	       if (m_nodeElemCount[nidx]>8) {
		 fprintf(stderr, "Node degree is higher than 8!\n"); 
		 exit(1);
	       }
           }
       }
   }
    
   /**********/
   /* Access */
   /**********/

   /* Node-centered */

   Real_t& x(Index_t idx)    { return m_x[idx] ; }
   Real_t& y(Index_t idx)    { return m_y[idx] ; }
   Real_t& z(Index_t idx)    { return m_z[idx] ; }

   Real_t& xd(Index_t idx)   { return m_xd[idx] ; }
   Real_t& yd(Index_t idx)   { return m_yd[idx] ; }
   Real_t& zd(Index_t idx)   { return m_zd[idx] ; }

   Real_t& xdd(Index_t idx)  { return m_xdd[idx] ; }
   Real_t& ydd(Index_t idx)  { return m_ydd[idx] ; }
   Real_t& zdd(Index_t idx)  { return m_zdd[idx] ; }

   Real_t& fx(Index_t idx)   { return m_fx[idx] ; }
   Real_t& fy(Index_t idx)   { return m_fy[idx] ; }
   Real_t& fz(Index_t idx)   { return m_fz[idx] ; }

   Real_t& nodalMass(Index_t idx) { return m_nodalMass[idx] ; }

   Index_t&  symmX(Index_t idx) { return m_symmX[idx] ; }
   Index_t&  symmY(Index_t idx) { return m_symmY[idx] ; }
   Index_t&  symmZ(Index_t idx) { return m_symmZ[idx] ; }
    
   /* Element-centered */

   Index_t&  matElemlist(Index_t idx) { return m_matElemlist[idx] ; }
   Index_t&  nodelist(Index_t idx,Index_t nidx)    { return m_nodelist[idx+nidx*m_numElem] ; }

   Index_t&  lxim(Index_t idx) { return m_lxim[idx] ; }
   Index_t&  lxip(Index_t idx) { return m_lxip[idx] ; }
   Index_t&  letam(Index_t idx) { return m_letam[idx] ; }
   Index_t&  letap(Index_t idx) { return m_letap[idx] ; }
   Index_t&  lzetam(Index_t idx) { return m_lzetam[idx] ; }
   Index_t&  lzetap(Index_t idx) { return m_lzetap[idx] ; }

   Int_t&  elemBC(Index_t idx) { return m_elemBC[idx] ; }

   Real_t& dxx(Index_t idx)  { return m_dxx[idx] ; }
   Real_t& dyy(Index_t idx)  { return m_dyy[idx] ; }
   Real_t& dzz(Index_t idx)  { return m_dzz[idx] ; }

   Real_t& delv_xi(Index_t idx)    { return m_delv_xi[idx] ; }
   Real_t& delv_eta(Index_t idx)   { return m_delv_eta[idx] ; }
   Real_t& delv_zeta(Index_t idx)  { return m_delv_zeta[idx] ; }

   Real_t& delx_xi(Index_t idx)    { return m_delx_xi[idx] ; }
   Real_t& delx_eta(Index_t idx)   { return m_delx_eta[idx] ; }
   Real_t& delx_zeta(Index_t idx)  { return m_delx_zeta[idx] ; }

   Real_t& e(Index_t idx)          { return m_e[idx] ; }

   Real_t& p(Index_t idx)          { return m_p[idx] ; }
   Real_t& q(Index_t idx)          { return m_q[idx] ; }
   Real_t& ql(Index_t idx)         { return m_ql[idx] ; }
   Real_t& qq(Index_t idx)         { return m_qq[idx] ; }

   Real_t& v(Index_t idx)          { return m_v[idx] ; }
   Real_t& volo(Index_t idx)       { return m_volo[idx] ; }
   Real_t& vnew(Index_t idx)       { return m_vnew[idx] ; }
   Real_t& delv(Index_t idx)       { return m_delv[idx] ; }
   Real_t& vdov(Index_t idx)       { return m_vdov[idx] ; }

   Real_t& arealg(Index_t idx)     { return m_arealg[idx] ; }
   
   Real_t& ss(Index_t idx)         { return m_ss[idx] ; }

   Real_t& elemMass(Index_t idx)  { return m_elemMass[idx] ; }

   /* Params */

   Real_t& dtfixed()              { return m_dtfixed ; }
   Real_t& time()                 { return m_time ; }
   Real_t& deltatime()            { return m_deltatime ; }
   Real_t& deltatimemultlb()      { return m_deltatimemultlb ; }
   Real_t& deltatimemultub()      { return m_deltatimemultub ; }
   Real_t& stoptime()             { return m_stoptime ; }

   Real_t& u_cut()                { return m_u_cut ; }
   Real_t& hgcoef()               { return m_hgcoef ; }
   Real_t& qstop()                { return m_qstop ; }
   Real_t& monoq_max_slope()      { return m_monoq_max_slope ; }
   Real_t& monoq_limiter_mult()   { return m_monoq_limiter_mult ; }
   Real_t& e_cut()                { return m_e_cut ; }
   Real_t& p_cut()                { return m_p_cut ; }
   Real_t& ss4o3()                { return m_ss4o3 ; }
   Real_t& q_cut()                { return m_q_cut ; }
   Real_t& v_cut()                { return m_v_cut ; }
   Real_t& qlc_monoq()            { return m_qlc_monoq ; }
   Real_t& qqc_monoq()            { return m_qqc_monoq ; }
   Real_t& qqc()                  { return m_qqc ; }
   Real_t& eosvmax()              { return m_eosvmax ; }
   Real_t& eosvmin()              { return m_eosvmin ; }
   Real_t& pmin()                 { return m_pmin ; }
   Real_t& emin()                 { return m_emin ; }
   Real_t& dvovmax()              { return m_dvovmax ; }
   Real_t& refdens()              { return m_refdens ; }

   Real_t& dtcourant()            { return m_dtcourant ; }
   Real_t& dthydro()              { return m_dthydro ; }
   Real_t& dtmax()                { return m_dtmax ; }

   Int_t&  cycle()                { return m_cycle ; }

   Index_t&  sizeX()              { return m_sizeX ; }
   Index_t&  sizeY()              { return m_sizeY ; }
   Index_t&  sizeZ()              { return m_sizeZ ; }
   Index_t&  numElem()            { return m_numElem ; }
   Index_t&  numNode()            { return m_numNode ; }
    
    
//private:

   /******************/
   /* Implementation */
   /******************/

   /* Node-centered */

   std::vector<Real_t> m_x ;  /* coordinates */
   std::vector<Real_t> m_y ;
   std::vector<Real_t> m_z ;

   std::vector<Real_t> m_xd ; /* velocities */
   std::vector<Real_t> m_yd ;
   std::vector<Real_t> m_zd ;

   std::vector<Real_t> m_xdd ; /* accelerations */
   std::vector<Real_t> m_ydd ;
   std::vector<Real_t> m_zdd ;

   std::vector<Real_t> m_fx ;  /* forces */
   std::vector<Real_t> m_fy ;
   std::vector<Real_t> m_fz ;

   std::vector<Real_t> m_nodalMass ;  /* mass */

   std::vector<Index_t> m_symmX ;  /* symmetry plane nodesets */
   std::vector<Index_t> m_symmY ;
   std::vector<Index_t> m_symmZ ;
    
   std::vector<Int_t> m_nodeElemCount ;
   std::vector<Index_t> m_nodeElemCornerList ;
    
   /* Element-centered */

   std::vector<Index_t>  m_matElemlist ;  /* material indexset */
   std::vector<Index_t>  m_nodelist ;     /* elemToNode connectivity */

   std::vector<Index_t>  m_lxim ;  /* element connectivity across each face */
   std::vector<Index_t>  m_lxip ;
   std::vector<Index_t>  m_letam ;
   std::vector<Index_t>  m_letap ;
   std::vector<Index_t>  m_lzetam ;
   std::vector<Index_t>  m_lzetap ;

   std::vector<Int_t>    m_elemBC ;  /* symmetry/free-surface flags for each elem face */

   std::vector<Real_t> m_dxx ;  /* principal strains -- temporary */
   std::vector<Real_t> m_dyy ;
   std::vector<Real_t> m_dzz ;

   std::vector<Real_t> m_delv_xi ;    /* velocity gradient -- temporary */
   std::vector<Real_t> m_delv_eta ;
   std::vector<Real_t> m_delv_zeta ;

   std::vector<Real_t> m_delx_xi ;    /* coordinate gradient -- temporary */
   std::vector<Real_t> m_delx_eta ;
   std::vector<Real_t> m_delx_zeta ;
   
   std::vector<Real_t> m_e ;   /* energy */

   std::vector<Real_t> m_p ;   /* pressure */
   std::vector<Real_t> m_q ;   /* q */
   std::vector<Real_t> m_ql ;  /* linear term for q */
   std::vector<Real_t> m_qq ;  /* quadratic term for q */

   std::vector<Real_t> m_v ;     /* relative volume */
   std::vector<Real_t> m_volo ;  /* reference volume */
   std::vector<Real_t> m_vnew ;  /* new relative volume -- temporary */
   std::vector<Real_t> m_delv ;  /* m_vnew - m_v */
   std::vector<Real_t> m_vdov ;  /* volume derivative over volume */

   std::vector<Real_t> m_arealg ;  /* characteristic length of an element */
   
   std::vector<Real_t> m_ss ;      /* "sound speed" */

   std::vector<Real_t> m_elemMass ;  /* mass */

   /* Parameters */

   Real_t  m_dtfixed ;           /* fixed time increment */
   Real_t  m_time ;              /* current time */
   Real_t  m_deltatime ;         /* variable time increment */
   Real_t  m_deltatimemultlb ;
   Real_t  m_deltatimemultub ;
   Real_t  m_stoptime ;          /* end time for simulation */

   Real_t  m_u_cut ;             /* velocity tolerance */
   Real_t  m_hgcoef ;            /* hourglass control */
   Real_t  m_qstop ;             /* excessive q indicator */
   Real_t  m_monoq_max_slope ;
   Real_t  m_monoq_limiter_mult ;
   Real_t  m_e_cut ;             /* energy tolerance */
   Real_t  m_p_cut ;             /* pressure tolerance */
   Real_t  m_ss4o3 ;
   Real_t  m_q_cut ;             /* q tolerance */
   Real_t  m_v_cut ;             /* relative volume tolerance */
   Real_t  m_qlc_monoq ;         /* linear term coef for q */
   Real_t  m_qqc_monoq ;         /* quadratic term coef for q */
   Real_t  m_qqc ;
   Real_t  m_eosvmax ;
   Real_t  m_eosvmin ;
   Real_t  m_pmin ;              /* pressure floor */
   Real_t  m_emin ;              /* energy floor */
   Real_t  m_dvovmax ;           /* maximum allowable volume change */
   Real_t  m_refdens ;           /* reference density */

   Real_t  m_dtcourant ;         /* courant constraint */
   Real_t  m_dthydro ;           /* volume change constraint */
   Real_t  m_dtmax ;             /* maximum allowable time increment */

   Int_t   m_cycle ;             /* iteration count for simulation */

   Index_t   m_sizeX ;           /* X,Y,Z extent of this block */
   Index_t   m_sizeY ;
   Index_t   m_sizeZ ;

   Index_t   m_numElem ;         /* Elements/Nodes in this domain */
   Index_t   m_numNode ;
} mesh ;

template <typename T>
T *Allocate(size_t size)
{
   return static_cast<T *>(malloc(sizeof(T)*size)) ;
}

template <typename T>
void Release(T **ptr)
{
   if (*ptr != NULL) {
      free(*ptr) ;
      *ptr = NULL ;
   }
}


#define GPU_STALE 0
#define CPU_STALE 1
#define ALL_FRESH 2

template<typename T>
void freshenGPU(std::vector<T>&cpu,T **gpu,int& stale) {
    if (stale!=GPU_STALE) return;
    if (!(*gpu)) {CUDA( hipMalloc(gpu,sizeof(T)*cpu.size()) );}
    CUDA( hipMemcpy(*gpu,&cpu[0],sizeof(T)*cpu.size(),hipMemcpyHostToDevice) );
    stale=ALL_FRESH;
}

template<typename T>
void freshenCPU(std::vector<T>&cpu,T *gpu,int& stale) {
    if (stale!=CPU_STALE) return;
    if (!gpu) {fprintf(stderr,"freshenCPU(): NULL GPU data!\n");exit(1);}
    CUDA( hipMemcpy(&cpu[0],gpu,sizeof(T)*cpu.size(),hipMemcpyDeviceToHost) );
    stale=ALL_FRESH;
}

// freshen helpers
#define FC(var) freshenCPU(mesh.m_ ## var , meshGPU.m_ ## var ,meshGPU.m_ ## var ## _stale ); // freshen CPU
#define FG(var) freshenGPU(mesh.m_ ## var , &meshGPU.m_ ## var ,meshGPU.m_ ## var ## _stale ); // freshen GPU
// stale helpers
#define SC(var) meshGPU.m_ ## var ## _stale = CPU_STALE; // stale CPU
#define SG(var) meshGPU.m_ ## var ## _stale = GPU_STALE; // stale GPU
    
struct MeshGPU {
    Mesh *m_mesh;
    
   /******************/
   /* Implementation */
   /******************/

   /* Node-centered */

   Real_t *m_x ;  /* coordinates */
   Real_t *m_y ;
   Real_t *m_z ;

   Real_t *m_xd ; /* velocities */
   Real_t *m_yd ;
   Real_t *m_zd ;

   Real_t *m_xdd ; /* accelerations */
   Real_t *m_ydd ;
   Real_t *m_zdd ;

   Real_t *m_fx ;  /* forces */
   Real_t *m_fy ;
   Real_t *m_fz ;

   Real_t *m_nodalMass ;  /* mass */

   Index_t *m_symmX ;  /* symmetry plane nodesets */
   Index_t *m_symmY ;
   Index_t *m_symmZ ;
    
   Int_t   *m_nodeElemCount ;
   Index_t *m_nodeElemCornerList ;
    
   /* Element-centered */

   Index_t * m_matElemlist ;  /* material indexset */
   Index_t * m_nodelist ;     /* elemToNode connectivity */

   Index_t * m_lxim ;  /* element connectivity across each face */
   Index_t * m_lxip ;
   Index_t * m_letam ;
   Index_t * m_letap ;
   Index_t * m_lzetam ;
   Index_t * m_lzetap ;

   Int_t *   m_elemBC ;  /* symmetry/free-surface flags for each elem face */

   Real_t *m_dxx ;  /* principal strains -- temporary */
   Real_t *m_dyy ;
   Real_t *m_dzz ;

   Real_t *m_delv_xi ;    /* velocity gradient -- temporary */
   Real_t *m_delv_eta ;
   Real_t *m_delv_zeta ;

   Real_t *m_delx_xi ;    /* coordinate gradient -- temporary */
   Real_t *m_delx_eta ;
   Real_t *m_delx_zeta ;
   
   Real_t *m_e ;   /* energy */

   Real_t *m_p ;   /* pressure */
   Real_t *m_q ;   /* q */
   Real_t *m_ql ;  /* linear term for q */
   Real_t *m_qq ;  /* quadratic term for q */

   Real_t *m_v ;     /* relative volume */
   Real_t *m_volo ;  /* reference volume */
   Real_t *m_vnew ;  /* new relative volume -- temporary */
   Real_t *m_delv ;  /* m_vnew - m_v */
   Real_t *m_vdov ;  /* volume derivative over volume */

   Real_t *m_arealg ;  /* characteristic length of an element */
   
   Real_t *m_ss ;      /* "sound speed" */

   Real_t *m_elemMass ;  /* mass */
    
   /* Stale flags */
    int m_x_stale,m_y_stale,m_z_stale;
    int m_xd_stale,m_yd_stale,m_zd_stale;
    int m_xdd_stale,m_ydd_stale,m_zdd_stale;
    int m_fx_stale,m_fy_stale,m_fz_stale;
    int m_nodalMass_stale;
    int m_symmX_stale,m_symmY_stale,m_symmZ_stale;
    int m_nodeElemCount_stale,m_nodeElemCornerList_stale;
    int m_matElemlist_stale,m_nodelist_stale;
    int m_lxim_stale,m_lxip_stale,m_letam_stale,m_letap_stale,m_lzetam_stale,m_lzetap_stale;
    int m_elemBC_stale;
    int m_dxx_stale,m_dyy_stale,m_dzz_stale;
    int m_delv_xi_stale,m_delv_eta_stale,m_delv_zeta_stale;
    int m_delx_xi_stale,m_delx_eta_stale,m_delx_zeta_stale;
    int m_e_stale;
    int m_p_stale,m_q_stale,m_ql_stale,m_qq_stale;
    int m_v_stale,m_volo_stale,m_vnew_stale,m_delv_stale,m_vdov_stale;
    int m_arealg_stale;
    int m_ss_stale;
    int m_elemMass_stale;
    
    void init(Mesh *mesh) {
        m_mesh=mesh;
        m_x=m_y=m_z=NULL;
        m_xd=m_yd=m_zd=NULL;
        m_xdd=m_ydd=m_zdd=NULL;
        m_fx=m_fy=m_fz=NULL;
        m_nodalMass=NULL;
        m_symmX=m_symmY=m_symmZ=NULL;
        m_nodeElemCount=m_nodeElemCornerList=NULL;
        m_matElemlist=m_nodelist=NULL;
        m_lxim=m_lxip=m_letam=m_letap=m_lzetam=m_lzetap=NULL;
        m_elemBC=NULL;
        m_dxx=m_dyy=m_dzz=NULL;
        m_delv_xi=m_delv_eta=m_delv_zeta=NULL;
        m_delx_xi=m_delx_eta=m_delx_zeta=NULL;
        m_e=NULL;
        m_p=m_q=m_ql=m_qq=NULL;
        m_v=m_volo=m_vnew=m_delv=m_vdov=NULL;
        m_arealg=NULL;
        m_ss=NULL;
        m_elemMass=NULL;
        m_x_stale=m_y_stale=m_z_stale=
            m_xd_stale=m_yd_stale=m_zd_stale=
            m_xdd_stale=m_ydd_stale=m_zdd_stale=
            m_fx_stale=m_fy_stale=m_fz_stale=
            m_nodalMass_stale=
            m_symmX_stale=m_symmY_stale=m_symmZ_stale=
            m_nodeElemCount_stale=m_nodeElemCornerList_stale=
            m_matElemlist_stale=m_nodelist_stale=
            m_lxim_stale=m_lxip_stale=m_letam_stale=m_letap_stale=m_lzetam_stale=m_lzetap_stale=
            m_elemBC_stale=
            m_dxx_stale=m_dyy_stale=m_dzz_stale=
            m_delv_xi_stale=m_delv_eta_stale=m_delv_zeta_stale=
            m_delx_xi_stale=m_delx_eta_stale=m_delx_zeta_stale=
            m_e_stale=
            m_p_stale=m_q_stale=m_ql_stale=m_qq_stale=
            m_v_stale=m_volo_stale=m_vnew_stale=m_delv_stale=m_vdov_stale=
            m_arealg_stale=
            m_ss_stale=
            m_elemMass_stale=
            GPU_STALE;
    }
    void freshenGPU() {
#define F(var) ::freshenGPU(m_mesh->m_ ## var , &m_ ## var ,m_ ## var ## _stale);
        F(x); F(y); F(z);
        F(xd); F(yd); F(zd);
        F(xdd); F(ydd); F(zdd);
        F(fx); F(fy); F(fz);
        F(nodalMass);
        F(symmX); F(symmY); F(symmZ);
        F(nodeElemCount); F(nodeElemCornerList);
        F(matElemlist); F(nodelist);
        F(lxim); F(lxip); F(letam); F(letap); F(lzetam); F(lzetap);
        F(elemBC);
        F(dxx); F(dyy); F(dzz);
        F(delv_xi); F(delv_eta); F(delv_zeta);
        F(delx_xi); F(delx_eta); F(delx_zeta);
        F(e);
        F(p); F(q); F(ql); F(qq);
        F(v); F(volo); F(vnew); F(delv); F(vdov);
        F(arealg);
        F(ss);
        F(elemMass);
#undef F
    }
    void freshenCPU() {
#define F(var) ::freshenCPU(m_mesh->m_ ## var , m_ ## var ,m_ ## var ## _stale);
        F(x); F(y); F(z);
        F(xd); F(yd); F(zd);
        F(xdd); F(ydd); F(zdd);
        F(fx); F(fy); F(fz);
        F(nodalMass);
        F(symmX); F(symmY); F(symmZ);
        F(nodeElemCount); F(nodeElemCornerList);
        F(matElemlist); F(nodelist);
        F(lxim); F(lxip); F(letam); F(letap); F(lzetam); F(lzetap);
        F(elemBC);
        F(dxx); F(dyy); F(dzz);
        F(delv_xi); F(delv_eta); F(delv_zeta);
        F(delx_xi); F(delx_eta); F(delx_zeta);
        F(e);
        F(p); F(q); F(ql); F(qq);
        F(v); F(volo); F(vnew); F(delv); F(vdov);
        F(arealg);
        F(ss);
        F(elemMass);
#undef F
    }
} meshGPU;



/* Stuff needed for boundary conditions */
/* 2 BCs on each of 6 hexahedral faces (12 bits) */
#define XI_M        0x003
#define XI_M_SYMM   0x001
#define XI_M_FREE   0x002

#define XI_P        0x00c
#define XI_P_SYMM   0x004
#define XI_P_FREE   0x008

#define ETA_M       0x030
#define ETA_M_SYMM  0x010
#define ETA_M_FREE  0x020

#define ETA_P       0x0c0
#define ETA_P_SYMM  0x040
#define ETA_P_FREE  0x080

#define ZETA_M      0x300
#define ZETA_M_SYMM 0x100
#define ZETA_M_FREE 0x200

#define ZETA_P      0xc00
#define ZETA_P_SYMM 0x400
#define ZETA_P_FREE 0x800


static inline
void TimeIncrement()
{
   Real_t targetdt = mesh.stoptime() - mesh.time() ;

   if ((mesh.dtfixed() <= Real_t(0.0)) && (mesh.cycle() != Int_t(0))) {
      Real_t ratio ;
      Real_t olddt = mesh.deltatime() ;

      /* This will require a reduction in parallel */
      Real_t newdt = Real_t(1.0e+20) ;
      if (mesh.dtcourant() < newdt) {
         newdt = mesh.dtcourant() / Real_t(2.0) ;
      }
      if (mesh.dthydro() < newdt) {
         newdt = mesh.dthydro() * Real_t(2.0) / Real_t(3.0) ;
      }

      ratio = newdt / olddt ;
      if (ratio >= Real_t(1.0)) {
         if (ratio < mesh.deltatimemultlb()) {
            newdt = olddt ;
         }
         else if (ratio > mesh.deltatimemultub()) {
            newdt = olddt*mesh.deltatimemultub() ;
         }
      }

      if (newdt > mesh.dtmax()) {
         newdt = mesh.dtmax() ;
      }
      mesh.deltatime() = newdt ;
   }

   /* TRY TO PREVENT VERY SMALL SCALING ON THE NEXT CYCLE */
   if ((targetdt > mesh.deltatime()) &&
       (targetdt < (Real_t(4.0) * mesh.deltatime() / Real_t(3.0))) ) {
      targetdt = Real_t(2.0) * mesh.deltatime() / Real_t(3.0) ;
   }

   if (targetdt < mesh.deltatime()) {
      mesh.deltatime() = targetdt ;
   }

   mesh.time() += mesh.deltatime() ;

   ++mesh.cycle() ;
}

__global__
void InitStressTermsForElems_kernel(
    int numElem,Real_t *sigxx, Real_t *sigyy, Real_t *sigzz, Real_t *p, Real_t *q)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numElem)
        sigxx[i] = sigyy[i] = sigzz[i] =  - p[i] - q[i] ;
}

static inline
void InitStressTermsForElems_gpu(Index_t numElem, 
                                 Real_t *sigxx, Real_t *sigyy, Real_t *sigzz)
{
    dim3 dimBlock(BLOCKSIZE,1,1);
    dim3 dimGrid(PAD_DIV(numElem,dimBlock.x),1,1);
    //cudaFuncSetCacheConfig(InitStressTermsForElems_kernel,cudaFuncCachePreferL1); // set as default for all kernels after this one
    InitStressTermsForElems_kernel<<<dimGrid, dimBlock>>>
        (numElem,sigxx,sigyy,sigzz,meshGPU.m_p,meshGPU.m_q);
    CUDA_DEBUGSYNC;
}

static inline
void InitStressTermsForElems_cpu(Index_t numElem, 
                                 Real_t *sigxx, Real_t *sigyy, Real_t *sigzz)
{
   //
   // pull in the stresses appropriate to the hydro integration
   //
   for (Index_t i = 0 ; i < numElem ; ++i){
      sigxx[i] =  sigyy[i] = sigzz[i] =  - mesh.p(i) - mesh.q(i) ;
   }
}

static inline
void InitStressTermsForElems(Index_t numElem, 
                             Real_t *sigxx, Real_t *sigyy, Real_t *sigzz,
                             int useCPU)
{
    if (useCPU) {
        FC(p); FC(q);
        InitStressTermsForElems_cpu(numElem,sigxx,sigyy,sigzz);
    }
    else {
        FG(p); FG(q);    
        InitStressTermsForElems_gpu(numElem,sigxx,sigyy,sigzz);
    }
}

__host__ __device__
static inline
void CalcElemShapeFunctionDerivatives( const Real_t* const x,
                                       const Real_t* const y,
                                       const Real_t* const z,
                                       Real_t b[][8],
                                       Real_t* const volume )
{
  const Real_t x0 = x[0] ;   const Real_t x1 = x[1] ;
  const Real_t x2 = x[2] ;   const Real_t x3 = x[3] ;
  const Real_t x4 = x[4] ;   const Real_t x5 = x[5] ;
  const Real_t x6 = x[6] ;   const Real_t x7 = x[7] ;

  const Real_t y0 = y[0] ;   const Real_t y1 = y[1] ;
  const Real_t y2 = y[2] ;   const Real_t y3 = y[3] ;
  const Real_t y4 = y[4] ;   const Real_t y5 = y[5] ;
  const Real_t y6 = y[6] ;   const Real_t y7 = y[7] ;

  const Real_t z0 = z[0] ;   const Real_t z1 = z[1] ;
  const Real_t z2 = z[2] ;   const Real_t z3 = z[3] ;
  const Real_t z4 = z[4] ;   const Real_t z5 = z[5] ;
  const Real_t z6 = z[6] ;   const Real_t z7 = z[7] ;

  Real_t fjxxi, fjxet, fjxze;
  Real_t fjyxi, fjyet, fjyze;
  Real_t fjzxi, fjzet, fjzze;
  Real_t cjxxi, cjxet, cjxze;
  Real_t cjyxi, cjyet, cjyze;
  Real_t cjzxi, cjzet, cjzze;

  fjxxi = Real_t(.125) * ( (x6-x0) + (x5-x3) - (x7-x1) - (x4-x2) );
  fjxet = Real_t(.125) * ( (x6-x0) - (x5-x3) + (x7-x1) - (x4-x2) );
  fjxze = Real_t(.125) * ( (x6-x0) + (x5-x3) + (x7-x1) + (x4-x2) );

  fjyxi = Real_t(.125) * ( (y6-y0) + (y5-y3) - (y7-y1) - (y4-y2) );
  fjyet = Real_t(.125) * ( (y6-y0) - (y5-y3) + (y7-y1) - (y4-y2) );
  fjyze = Real_t(.125) * ( (y6-y0) + (y5-y3) + (y7-y1) + (y4-y2) );

  fjzxi = Real_t(.125) * ( (z6-z0) + (z5-z3) - (z7-z1) - (z4-z2) );
  fjzet = Real_t(.125) * ( (z6-z0) - (z5-z3) + (z7-z1) - (z4-z2) );
  fjzze = Real_t(.125) * ( (z6-z0) + (z5-z3) + (z7-z1) + (z4-z2) );

  /* compute cofactors */
  cjxxi =    (fjyet * fjzze) - (fjzet * fjyze);
  cjxet =  - (fjyxi * fjzze) + (fjzxi * fjyze);
  cjxze =    (fjyxi * fjzet) - (fjzxi * fjyet);

  cjyxi =  - (fjxet * fjzze) + (fjzet * fjxze);
  cjyet =    (fjxxi * fjzze) - (fjzxi * fjxze);
  cjyze =  - (fjxxi * fjzet) + (fjzxi * fjxet);

  cjzxi =    (fjxet * fjyze) - (fjyet * fjxze);
  cjzet =  - (fjxxi * fjyze) + (fjyxi * fjxze);
  cjzze =    (fjxxi * fjyet) - (fjyxi * fjxet);

  /* calculate partials :
     this need only be done for l = 0,1,2,3   since , by symmetry ,
     (6,7,4,5) = - (0,1,2,3) .
  */
  b[0][0] =   -  cjxxi  -  cjxet  -  cjxze;
  b[0][1] =      cjxxi  -  cjxet  -  cjxze;
  b[0][2] =      cjxxi  +  cjxet  -  cjxze;
  b[0][3] =   -  cjxxi  +  cjxet  -  cjxze;
  b[0][4] = -b[0][2];
  b[0][5] = -b[0][3];
  b[0][6] = -b[0][0];
  b[0][7] = -b[0][1];

  b[1][0] =   -  cjyxi  -  cjyet  -  cjyze;
  b[1][1] =      cjyxi  -  cjyet  -  cjyze;
  b[1][2] =      cjyxi  +  cjyet  -  cjyze;
  b[1][3] =   -  cjyxi  +  cjyet  -  cjyze;
  b[1][4] = -b[1][2];
  b[1][5] = -b[1][3];
  b[1][6] = -b[1][0];
  b[1][7] = -b[1][1];

  b[2][0] =   -  cjzxi  -  cjzet  -  cjzze;
  b[2][1] =      cjzxi  -  cjzet  -  cjzze;
  b[2][2] =      cjzxi  +  cjzet  -  cjzze;
  b[2][3] =   -  cjzxi  +  cjzet  -  cjzze;
  b[2][4] = -b[2][2];
  b[2][5] = -b[2][3];
  b[2][6] = -b[2][0];
  b[2][7] = -b[2][1];

  /* calculate jacobian determinant (volume) */
  *volume = Real_t(8.) * ( fjxet * cjxet + fjyet * cjyet + fjzet * cjzet);
}

__host__ __device__
static inline
void SumElemFaceNormal(Real_t *normalX0, Real_t *normalY0, Real_t *normalZ0,
                       Real_t *normalX1, Real_t *normalY1, Real_t *normalZ1,
                       Real_t *normalX2, Real_t *normalY2, Real_t *normalZ2,
                       Real_t *normalX3, Real_t *normalY3, Real_t *normalZ3,
                       const Real_t x0, const Real_t y0, const Real_t z0,
                       const Real_t x1, const Real_t y1, const Real_t z1,
                       const Real_t x2, const Real_t y2, const Real_t z2,
                       const Real_t x3, const Real_t y3, const Real_t z3)
{
   Real_t bisectX0 = Real_t(0.5) * (x3 + x2 - x1 - x0);
   Real_t bisectY0 = Real_t(0.5) * (y3 + y2 - y1 - y0);
   Real_t bisectZ0 = Real_t(0.5) * (z3 + z2 - z1 - z0);
   Real_t bisectX1 = Real_t(0.5) * (x2 + x1 - x3 - x0);
   Real_t bisectY1 = Real_t(0.5) * (y2 + y1 - y3 - y0);
   Real_t bisectZ1 = Real_t(0.5) * (z2 + z1 - z3 - z0);
   Real_t areaX = Real_t(0.25) * (bisectY0 * bisectZ1 - bisectZ0 * bisectY1);
   Real_t areaY = Real_t(0.25) * (bisectZ0 * bisectX1 - bisectX0 * bisectZ1);
   Real_t areaZ = Real_t(0.25) * (bisectX0 * bisectY1 - bisectY0 * bisectX1);

   *normalX0 += areaX;
   *normalX1 += areaX;
   *normalX2 += areaX;
   *normalX3 += areaX;

   *normalY0 += areaY;
   *normalY1 += areaY;
   *normalY2 += areaY;
   *normalY3 += areaY;

   *normalZ0 += areaZ;
   *normalZ1 += areaZ;
   *normalZ2 += areaZ;
   *normalZ3 += areaZ;
}

__host__ __device__
static inline
void CalcElemNodeNormals(Real_t pfx[8],
                         Real_t pfy[8],
                         Real_t pfz[8],
                         const Real_t x[8],
                         const Real_t y[8],
                         const Real_t z[8])
{
   for (Index_t i = 0 ; i < 8 ; ++i) {
      pfx[i] = Real_t(0.0);
      pfy[i] = Real_t(0.0);
      pfz[i] = Real_t(0.0);
   }
   /* evaluate face one: nodes 0, 1, 2, 3 */
   SumElemFaceNormal(&pfx[0], &pfy[0], &pfz[0],
                  &pfx[1], &pfy[1], &pfz[1],
                  &pfx[2], &pfy[2], &pfz[2],
                  &pfx[3], &pfy[3], &pfz[3],
                  x[0], y[0], z[0], x[1], y[1], z[1],
                  x[2], y[2], z[2], x[3], y[3], z[3]);
   /* evaluate face two: nodes 0, 4, 5, 1 */
   SumElemFaceNormal(&pfx[0], &pfy[0], &pfz[0],
                  &pfx[4], &pfy[4], &pfz[4],
                  &pfx[5], &pfy[5], &pfz[5],
                  &pfx[1], &pfy[1], &pfz[1],
                  x[0], y[0], z[0], x[4], y[4], z[4],
                  x[5], y[5], z[5], x[1], y[1], z[1]);
   /* evaluate face three: nodes 1, 5, 6, 2 */
   SumElemFaceNormal(&pfx[1], &pfy[1], &pfz[1],
                  &pfx[5], &pfy[5], &pfz[5],
                  &pfx[6], &pfy[6], &pfz[6],
                  &pfx[2], &pfy[2], &pfz[2],
                  x[1], y[1], z[1], x[5], y[5], z[5],
                  x[6], y[6], z[6], x[2], y[2], z[2]);
   /* evaluate face four: nodes 2, 6, 7, 3 */
   SumElemFaceNormal(&pfx[2], &pfy[2], &pfz[2],
                  &pfx[6], &pfy[6], &pfz[6],
                  &pfx[7], &pfy[7], &pfz[7],
                  &pfx[3], &pfy[3], &pfz[3],
                  x[2], y[2], z[2], x[6], y[6], z[6],
                  x[7], y[7], z[7], x[3], y[3], z[3]);
   /* evaluate face five: nodes 3, 7, 4, 0 */
   SumElemFaceNormal(&pfx[3], &pfy[3], &pfz[3],
                  &pfx[7], &pfy[7], &pfz[7],
                  &pfx[4], &pfy[4], &pfz[4],
                  &pfx[0], &pfy[0], &pfz[0],
                  x[3], y[3], z[3], x[7], y[7], z[7],
                  x[4], y[4], z[4], x[0], y[0], z[0]);
   /* evaluate face six: nodes 4, 7, 6, 5 */
   SumElemFaceNormal(&pfx[4], &pfy[4], &pfz[4],
                  &pfx[7], &pfy[7], &pfz[7],
                  &pfx[6], &pfy[6], &pfz[6],
                  &pfx[5], &pfy[5], &pfz[5],
                  x[4], y[4], z[4], x[7], y[7], z[7],
                  x[6], y[6], z[6], x[5], y[5], z[5]);
}

__host__ __device__
static inline
void SumElemStressesToNodeForces( const Real_t B[][8],
                                  const Real_t stress_xx,
                                  const Real_t stress_yy,
                                  const Real_t stress_zz,
                                  Real_t* const fx,
                                  Real_t* const fy,
                                  Real_t* const fz,
				  int stride)
{
  Real_t pfx0 = B[0][0] ;   Real_t pfx1 = B[0][1] ;
  Real_t pfx2 = B[0][2] ;   Real_t pfx3 = B[0][3] ;
  Real_t pfx4 = B[0][4] ;   Real_t pfx5 = B[0][5] ;
  Real_t pfx6 = B[0][6] ;   Real_t pfx7 = B[0][7] ;

  Real_t pfy0 = B[1][0] ;   Real_t pfy1 = B[1][1] ;
  Real_t pfy2 = B[1][2] ;   Real_t pfy3 = B[1][3] ;
  Real_t pfy4 = B[1][4] ;   Real_t pfy5 = B[1][5] ;
  Real_t pfy6 = B[1][6] ;   Real_t pfy7 = B[1][7] ;

  Real_t pfz0 = B[2][0] ;   Real_t pfz1 = B[2][1] ;
  Real_t pfz2 = B[2][2] ;   Real_t pfz3 = B[2][3] ;
  Real_t pfz4 = B[2][4] ;   Real_t pfz5 = B[2][5] ;
  Real_t pfz6 = B[2][6] ;   Real_t pfz7 = B[2][7] ;

  fx[0*stride] = -( stress_xx * pfx0 );
  fx[1*stride] = -( stress_xx * pfx1 );
  fx[2*stride] = -( stress_xx * pfx2 );
  fx[3*stride] = -( stress_xx * pfx3 );
  fx[4*stride] = -( stress_xx * pfx4 );
  fx[5*stride] = -( stress_xx * pfx5 );
  fx[6*stride] = -( stress_xx * pfx6 );
  fx[7*stride] = -( stress_xx * pfx7 );

  fy[0*stride] = -( stress_yy * pfy0  );
  fy[1*stride] = -( stress_yy * pfy1  );
  fy[2*stride] = -( stress_yy * pfy2  );
  fy[3*stride] = -( stress_yy * pfy3  );
  fy[4*stride] = -( stress_yy * pfy4  );
  fy[5*stride] = -( stress_yy * pfy5  );
  fy[6*stride] = -( stress_yy * pfy6  );
  fy[7*stride] = -( stress_yy * pfy7  );

  fz[0*stride] = -( stress_zz * pfz0 );
  fz[1*stride] = -( stress_zz * pfz1 );
  fz[2*stride] = -( stress_zz * pfz2 );
  fz[3*stride] = -( stress_zz * pfz3 );
  fz[4*stride] = -( stress_zz * pfz4 );
  fz[5*stride] = -( stress_zz * pfz5 );
  fz[6*stride] = -( stress_zz * pfz6 );
  fz[7*stride] = -( stress_zz * pfz7 );
}

__global__
void IntegrateStressForElems_kernel( Index_t numElem, Index_t *nodelist,
                                     Real_t *x, Real_t *y, Real_t *z,
                                     Real_t *fx_elem, Real_t *fy_elem, Real_t *fz_elem,
                                     Real_t *sigxx, Real_t *sigyy, Real_t *sigzz,
                                     Real_t *determ)
{
  Real_t B[3][8] ;// shape function derivatives
  Real_t x_local[8] ;
  Real_t y_local[8] ;
  Real_t z_local[8] ;

  int k=blockDim.x*blockIdx.x + threadIdx.x;
  if (k<numElem) {
    // get nodal coordinates from global arrays and copy into local arrays.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = nodelist[k+lnode*numElem];
      x_local[lnode] = x[gnode];
      y_local[lnode] = y[gnode];
      z_local[lnode] = z[gnode]; 
   }

    /* Volume calculation involves extra work for numerical consistency. */
    CalcElemShapeFunctionDerivatives(x_local, y_local, z_local,
                                         B, &determ[k]);

    CalcElemNodeNormals( B[0] , B[1], B[2],
                         x_local, y_local, z_local );

    SumElemStressesToNodeForces( B, sigxx[k], sigyy[k], sigzz[k],
                                 &fx_elem[k], &fy_elem[k], &fz_elem[k], numElem ) ;
  }
}

__global__
void AddNodeForcesFromElems_kernel( Index_t numNode,
                                    Int_t *nodeElemCount, Index_t *nodeElemCornerList,
                                    Real_t *fx_elem, Real_t *fy_elem, Real_t *fz_elem,
                                    Real_t *fx_node, Real_t *fy_node, Real_t *fz_node)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numNode) {
        Int_t count=nodeElemCount[i];
        Real_t fx,fy,fz;
        fx=fy=fz=Real_t(0.0);
        for (int j=0;j<count;j++) {
            Index_t elem=nodeElemCornerList[i+numNode*j];
            fx+=fx_elem[elem]; fy+=fy_elem[elem]; fz+=fz_elem[elem];
        }
        fx_node[i]=fx; fy_node[i]=fy; fz_node[i]=fz;
    }
}

__global__
void AddNodeForcesFromElems2_kernel( Index_t numNode,
                                    Int_t *nodeElemCount, Index_t *nodeElemCornerList,
                                    Real_t *fx_elem, Real_t *fy_elem, Real_t *fz_elem,
                                    Real_t *fx_node, Real_t *fy_node, Real_t *fz_node)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numNode) {
        Int_t count=nodeElemCount[i];
        Real_t fx,fy,fz;
        fx=fy=fz=Real_t(0.0);
        for (int j=0;j<count;j++) {
            Index_t elem=nodeElemCornerList[i+numNode*j];
            fx+=fx_elem[elem]; fy+=fy_elem[elem]; fz+=fz_elem[elem];
        }
        fx_node[i]+=fx; fy_node[i]+=fy; fz_node[i]+=fz;
    }
}

static inline
void IntegrateStressForElems_gpu( Index_t numElem,
                                  Real_t *sigxx, Real_t *sigyy, Real_t *sigzz,
                                  Real_t *determ, int& badvol)
{
    Real_t *fx_elem,*fy_elem,*fz_elem;

    CUDA( hipMalloc(&fx_elem,numElem*8*sizeof(Real_t)) );
    CUDA( hipMalloc(&fy_elem,numElem*8*sizeof(Real_t)) );
    CUDA( hipMalloc(&fz_elem,numElem*8*sizeof(Real_t)) );
    
    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(numElem,dimBlock.x),1,1);
    IntegrateStressForElems_kernel<<<dimGrid,dimBlock>>>
        (numElem, meshGPU.m_nodelist, meshGPU.m_x, meshGPU.m_y, meshGPU.m_z,
         fx_elem, fy_elem, fz_elem, sigxx, sigyy, sigzz, determ);
    CUDA_DEBUGSYNC;

    dimGrid=dim3(PAD_DIV(mesh.numNode(),dimBlock.x),1,1);
    AddNodeForcesFromElems_kernel<<<dimGrid,dimBlock>>>
        (mesh.numNode(),meshGPU.m_nodeElemCount,meshGPU.m_nodeElemCornerList,
         fx_elem,fy_elem,fz_elem,meshGPU.m_fx,meshGPU.m_fy,meshGPU.m_fz);
    CUDA_DEBUGSYNC;

    CUDA( hipFree(fx_elem) );
    CUDA( hipFree(fy_elem) );
    CUDA( hipFree(fz_elem) );
    
    // JDC -- need a reduction step to check for non-positive element volumes
    badvol=0; 
}

static inline
void IntegrateStressForElems_cpu( Index_t numElem,
                                  Real_t *sigxx, Real_t *sigyy, Real_t *sigzz,
                                  Real_t *determ, int& badvol)
{
  Real_t B[3][8] ;// shape function derivatives
  Real_t x_local[8] ;
  Real_t y_local[8] ;
  Real_t z_local[8] ; 
  Real_t fx_local[8] ;
  Real_t fy_local[8] ;
  Real_t fz_local[8] ;

  // loop over all elements
  for( Index_t k=0 ; k<numElem ; ++k )
  {
    // get nodal coordinates from global arrays and copy into local arrays.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = mesh.nodelist(k,lnode);
      x_local[lnode] = mesh.x(gnode);
      y_local[lnode] = mesh.y(gnode);
      z_local[lnode] = mesh.z(gnode);
    }

    /* Volume calculation involves extra work for numerical consistency. */
    CalcElemShapeFunctionDerivatives(x_local, y_local, z_local,
                                         B, &determ[k]);

    CalcElemNodeNormals( B[0] , B[1], B[2],
                          x_local, y_local, z_local );

    SumElemStressesToNodeForces( B, sigxx[k], sigyy[k], sigzz[k],
				 fx_local, fy_local, fz_local, 1 ) ;

    // copy nodal force contributions to global force arrray.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = mesh.nodelist(k,lnode);
      mesh.fx(gnode) += fx_local[lnode];
      mesh.fy(gnode) += fy_local[lnode];
      mesh.fz(gnode) += fz_local[lnode];
    }
  }

  badvol=0;
  for ( Index_t k=0 ; k<numElem ; ++k ) {
      if (determ[k] <= Real_t(0.0)) {
          badvol=1;
      }
  }
}

static inline
void IntegrateStressForElems( Index_t numElem,
                              Real_t *sigxx, Real_t *sigyy, Real_t *sigzz,
                              Real_t *determ, int& badvol, int useCPU)
{
    if (useCPU) {
        FC(nodelist); FC(x); FC(y); FC(z);
        IntegrateStressForElems_cpu(numElem,sigxx,sigyy,sigzz,determ,badvol);
        SG(fx); SG(fy); SG(fz);
    }
    else {
        FG(nodelist); FG(nodeElemCount); FG(nodeElemCornerList);
        FG(x); FG(y); FG(z);
        IntegrateStressForElems_gpu(numElem,sigxx,sigyy,sigzz,determ,badvol);
        SC(fx); SC(fy); SC(fz);
    }
    
}


static inline
void CollectDomainNodesToElemNodes(const Index_t elemNum,
                                   Real_t elemX[8],
                                   Real_t elemY[8],
                                   Real_t elemZ[8])
{
   Index_t nd0i = mesh.nodelist(elemNum,0) ;
   Index_t nd1i = mesh.nodelist(elemNum,1) ;
   Index_t nd2i = mesh.nodelist(elemNum,2) ;
   Index_t nd3i = mesh.nodelist(elemNum,3) ;
   Index_t nd4i = mesh.nodelist(elemNum,4) ;
   Index_t nd5i = mesh.nodelist(elemNum,5) ;
   Index_t nd6i = mesh.nodelist(elemNum,6) ;
   Index_t nd7i = mesh.nodelist(elemNum,7) ;

   elemX[0] = mesh.x(nd0i);
   elemX[1] = mesh.x(nd1i);
   elemX[2] = mesh.x(nd2i);
   elemX[3] = mesh.x(nd3i);
   elemX[4] = mesh.x(nd4i);
   elemX[5] = mesh.x(nd5i);
   elemX[6] = mesh.x(nd6i);
   elemX[7] = mesh.x(nd7i);

   elemY[0] = mesh.y(nd0i);
   elemY[1] = mesh.y(nd1i);
   elemY[2] = mesh.y(nd2i);
   elemY[3] = mesh.y(nd3i);
   elemY[4] = mesh.y(nd4i);
   elemY[5] = mesh.y(nd5i);
   elemY[6] = mesh.y(nd6i);
   elemY[7] = mesh.y(nd7i);

   elemZ[0] = mesh.z(nd0i);
   elemZ[1] = mesh.z(nd1i);
   elemZ[2] = mesh.z(nd2i);
   elemZ[3] = mesh.z(nd3i);
   elemZ[4] = mesh.z(nd4i);
   elemZ[5] = mesh.z(nd5i);
   elemZ[6] = mesh.z(nd6i);
   elemZ[7] = mesh.z(nd7i);

}


__host__ 
static inline
void VoluDer(const Real_t x0, const Real_t x1, const Real_t x2,
             const Real_t x3, const Real_t x4, const Real_t x5,
             const Real_t y0, const Real_t y1, const Real_t y2,
             const Real_t y3, const Real_t y4, const Real_t y5,
             const Real_t z0, const Real_t z1, const Real_t z2,
             const Real_t z3, const Real_t z4, const Real_t z5,
             Real_t* dvdx, Real_t* dvdy, Real_t* dvdz)
{
   const Real_t twelfth = Real_t(1.0) / Real_t(12.0) ;

   *dvdx =
      (y1 + y2) * (z0 + z1) - (y0 + y1) * (z1 + z2) +
      (y0 + y4) * (z3 + z4) - (y3 + y4) * (z0 + z4) -
      (y2 + y5) * (z3 + z5) + (y3 + y5) * (z2 + z5);
   *dvdy =
      - (x1 + x2) * (z0 + z1) + (x0 + x1) * (z1 + z2) -
      (x0 + x4) * (z3 + z4) + (x3 + x4) * (z0 + z4) +
      (x2 + x5) * (z3 + z5) - (x3 + x5) * (z2 + z5);

   *dvdz =
      - (y1 + y2) * (x0 + x1) + (y0 + y1) * (x1 + x2) -
      (y0 + y4) * (x3 + x4) + (y3 + y4) * (x0 + x4) +
      (y2 + y5) * (x3 + x5) - (y3 + y5) * (x2 + x5);

   *dvdx *= twelfth;
   *dvdy *= twelfth;
   *dvdz *= twelfth;
}

#if 0
__device__ 
static inline
void VOLUDER(const Real_t a0, const Real_t a1, const Real_t a2,
             const Real_t a3, const Real_t a4, const Real_t a5,
             const Real_t b0, const Real_t b1, const Real_t b2,
             const Real_t b3, const Real_t b4, const Real_t b5,
             Real_t& dvdc)
{
   const Real_t twelfth = Real_t(1.0) / Real_t(12.0) ;

   dvdc=
      (a1 + a2) * (b0 + b1) - (a0 + a1) * (b1 + b2) +
      (a0 + a4) * (b3 + b4) - (a3 + a4) * (b0 + b4) -
      (a2 + a5) * (b3 + b5) + (a3 + a5) * (b2 + b5);
   dvdc *= twelfth;
}
#else
// Even though the above version is inlined, it seems to prohibit some kind of compiler optimization.
// This macro version uses many fewer registers and avoids spill-over into local memory.
#define VOLUDER(a0,a1,a2,a3,a4,a5,b0,b1,b2,b3,b4,b5,dvdc)		\
{									\
  const Real_t twelfth = Real_t(1.0) / Real_t(12.0) ;			\
									\
   dvdc= 								\
     ((a1) + (a2)) * ((b0) + (b1)) - ((a0) + (a1)) * ((b1) + (b2)) +	\
     ((a0) + (a4)) * ((b3) + (b4)) - ((a3) + (a4)) * ((b0) + (b4)) -	\
     ((a2) + (a5)) * ((b3) + (b5)) + ((a3) + (a5)) * ((b2) + (b5));	\
   dvdc *= twelfth;							\
}
#endif

__host__
static inline
void CalcElemVolumeDerivative(Real_t dvdx[8],
                              Real_t dvdy[8],
                              Real_t dvdz[8],
                              const Real_t x[8],
                              const Real_t y[8],
                              const Real_t z[8])
{
   VoluDer(x[1], x[2], x[3], x[4], x[5], x[7],
           y[1], y[2], y[3], y[4], y[5], y[7],
           z[1], z[2], z[3], z[4], z[5], z[7],
           &dvdx[0], &dvdy[0], &dvdz[0]);
   VoluDer(x[0], x[1], x[2], x[7], x[4], x[6],
           y[0], y[1], y[2], y[7], y[4], y[6],
           z[0], z[1], z[2], z[7], z[4], z[6],
           &dvdx[3], &dvdy[3], &dvdz[3]);
   VoluDer(x[3], x[0], x[1], x[6], x[7], x[5],
           y[3], y[0], y[1], y[6], y[7], y[5],
           z[3], z[0], z[1], z[6], z[7], z[5],
           &dvdx[2], &dvdy[2], &dvdz[2]);
   VoluDer(x[2], x[3], x[0], x[5], x[6], x[4],
           y[2], y[3], y[0], y[5], y[6], y[4],
           z[2], z[3], z[0], z[5], z[6], z[4],
           &dvdx[1], &dvdy[1], &dvdz[1]);
   VoluDer(x[7], x[6], x[5], x[0], x[3], x[1],
           y[7], y[6], y[5], y[0], y[3], y[1],
           z[7], z[6], z[5], z[0], z[3], z[1],
           &dvdx[4], &dvdy[4], &dvdz[4]);
   VoluDer(x[4], x[7], x[6], x[1], x[0], x[2],
           y[4], y[7], y[6], y[1], y[0], y[2],
           z[4], z[7], z[6], z[1], z[0], z[2],
           &dvdx[5], &dvdy[5], &dvdz[5]);
   VoluDer(x[5], x[4], x[7], x[2], x[1], x[3],
           y[5], y[4], y[7], y[2], y[1], y[3],
           z[5], z[4], z[7], z[2], z[1], z[3],
           &dvdx[6], &dvdy[6], &dvdz[6]);
   VoluDer(x[6], x[5], x[4], x[3], x[2], x[0],
           y[6], y[5], y[4], y[3], y[2], y[0],
           z[6], z[5], z[4], z[3], z[2], z[0],
           &dvdx[7], &dvdy[7], &dvdz[7]);
}

__device__
static inline
void CalcElemVolumeDerivative(Real_t& dvdx,
                              Real_t& dvdy,
                              Real_t& dvdz,
                              const Real_t x,
                              const Real_t y,
                              const Real_t z,
			      unsigned int node)
{
  __shared__ Real_t array1[256],array2[256];
  volatile Real_t *va1;
  volatile Real_t *va2;

  unsigned int idx,elem;
  unsigned int ind0,ind1,ind2,ind3,ind4,ind5;

  switch(node) {
  case 0:
    {ind0=1; ind1=2; ind2=3; ind3=4; ind4=5; ind5=7;
    break;}
  case 1:
    {ind0=2; ind1=3; ind2=0; ind3=5; ind4=6; ind5=4;
    break;}
  case 2:
    {ind0=3; ind1=0; ind2=1; ind3=6; ind4=7; ind5=5;
    break;}
  case 3:
    {ind0=0; ind1=1; ind2=2; ind3=7; ind4=4; ind5=6;
    break;}
  case 4:
    {ind0=7; ind1=6; ind2=5; ind3=0; ind4=3; ind5=1;
    break;}
  case 5:
    {ind0=4; ind1=7; ind2=6; ind3=1; ind4=0; ind5=2;
    break;}
  case 6:
    {ind0=5; ind1=4; ind2=7; ind3=2; ind4=1; ind5=3;
    break;}
  case 7:
    {ind0=6; ind1=5; ind2=4; ind3=3; ind4=2; ind5=0;
    break;}
  default:
    {ind0=ind1=ind2=ind3=ind4=ind5=0xFFFFFFFF;
    break;}
  }
  
  idx=threadIdx.x;
  elem=idx /*& 0x1F*/ - node*32;

  va1=&array1[0];
  va2=&array2[0];

  // load y and z
  __syncthreads();
  va1[idx]=y; va2[idx]=z;
  __syncthreads();
  VOLUDER(va1[ind0*32+elem],va1[ind1*32+elem],va1[ind2*32+elem],
	  va1[ind3*32+elem],va1[ind4*32+elem],va1[ind5*32+elem],
	  va2[ind0*32+elem],va2[ind1*32+elem],va2[ind2*32+elem],
	  va2[ind3*32+elem],va2[ind4*32+elem],va2[ind5*32+elem],
	  dvdx);

  // load x
  __syncthreads();
  va1[idx]=x;
  __syncthreads();
  VOLUDER(va2[ind0*32+elem],va2[ind1*32+elem],va2[ind2*32+elem],
	  va2[ind3*32+elem],va2[ind4*32+elem],va2[ind5*32+elem],
	  va1[ind0*32+elem],va1[ind1*32+elem],va1[ind2*32+elem],
	  va1[ind3*32+elem],va1[ind4*32+elem],va1[ind5*32+elem],
	  dvdy);
  __syncthreads();

  // load y
  __syncthreads();
  va2[idx]=y;
  __syncthreads();
  VOLUDER(va1[ind0*32+elem],va1[ind1*32+elem],va1[ind2*32+elem],
	  va1[ind3*32+elem],va1[ind4*32+elem],va1[ind5*32+elem],
	  va2[ind0*32+elem],va2[ind1*32+elem],va2[ind2*32+elem],
	  va2[ind3*32+elem],va2[ind4*32+elem],va2[ind5*32+elem],
	  dvdz);
  __syncthreads();
}

__host__
static inline
void CalcElemFBHourglassForce(Real_t *xd, Real_t *yd, Real_t *zd,  Real_t *hourgam0,
                              Real_t *hourgam1, Real_t *hourgam2, Real_t *hourgam3,
                              Real_t *hourgam4, Real_t *hourgam5, Real_t *hourgam6,
                              Real_t *hourgam7, Real_t coefficient,
                              Real_t *hgfx, Real_t *hgfy, Real_t *hgfz )
{
   Index_t i00=0;
   Index_t i01=1;
   Index_t i02=2;
   Index_t i03=3;

   Real_t h00 =
      hourgam0[i00] * xd[0] + hourgam1[i00] * xd[1] +
      hourgam2[i00] * xd[2] + hourgam3[i00] * xd[3] +
      hourgam4[i00] * xd[4] + hourgam5[i00] * xd[5] +
      hourgam6[i00] * xd[6] + hourgam7[i00] * xd[7];

   Real_t h01 =
      hourgam0[i01] * xd[0] + hourgam1[i01] * xd[1] +
      hourgam2[i01] * xd[2] + hourgam3[i01] * xd[3] +
      hourgam4[i01] * xd[4] + hourgam5[i01] * xd[5] +
      hourgam6[i01] * xd[6] + hourgam7[i01] * xd[7];

   Real_t h02 =
      hourgam0[i02] * xd[0] + hourgam1[i02] * xd[1]+
      hourgam2[i02] * xd[2] + hourgam3[i02] * xd[3]+
      hourgam4[i02] * xd[4] + hourgam5[i02] * xd[5]+
      hourgam6[i02] * xd[6] + hourgam7[i02] * xd[7];

   Real_t h03 =
      hourgam0[i03] * xd[0] + hourgam1[i03] * xd[1] +
      hourgam2[i03] * xd[2] + hourgam3[i03] * xd[3] +
      hourgam4[i03] * xd[4] + hourgam5[i03] * xd[5] +
      hourgam6[i03] * xd[6] + hourgam7[i03] * xd[7];

   hgfx[0] = coefficient *
      (hourgam0[i00] * h00 + hourgam0[i01] * h01 +
       hourgam0[i02] * h02 + hourgam0[i03] * h03);

   hgfx[1] = coefficient *
      (hourgam1[i00] * h00 + hourgam1[i01] * h01 +
       hourgam1[i02] * h02 + hourgam1[i03] * h03);

   hgfx[2] = coefficient *
      (hourgam2[i00] * h00 + hourgam2[i01] * h01 +
       hourgam2[i02] * h02 + hourgam2[i03] * h03);

   hgfx[3] = coefficient *
      (hourgam3[i00] * h00 + hourgam3[i01] * h01 +
       hourgam3[i02] * h02 + hourgam3[i03] * h03);

   hgfx[4] = coefficient *
      (hourgam4[i00] * h00 + hourgam4[i01] * h01 +
       hourgam4[i02] * h02 + hourgam4[i03] * h03);

   hgfx[5] = coefficient *
      (hourgam5[i00] * h00 + hourgam5[i01] * h01 +
       hourgam5[i02] * h02 + hourgam5[i03] * h03);

   hgfx[6] = coefficient *
      (hourgam6[i00] * h00 + hourgam6[i01] * h01 +
       hourgam6[i02] * h02 + hourgam6[i03] * h03);

   hgfx[7] = coefficient *
      (hourgam7[i00] * h00 + hourgam7[i01] * h01 +
       hourgam7[i02] * h02 + hourgam7[i03] * h03);

   h00 =
      hourgam0[i00] * yd[0] + hourgam1[i00] * yd[1] +
      hourgam2[i00] * yd[2] + hourgam3[i00] * yd[3] +
      hourgam4[i00] * yd[4] + hourgam5[i00] * yd[5] +
      hourgam6[i00] * yd[6] + hourgam7[i00] * yd[7];

   h01 =
      hourgam0[i01] * yd[0] + hourgam1[i01] * yd[1] +
      hourgam2[i01] * yd[2] + hourgam3[i01] * yd[3] +
      hourgam4[i01] * yd[4] + hourgam5[i01] * yd[5] +
      hourgam6[i01] * yd[6] + hourgam7[i01] * yd[7];

   h02 =
      hourgam0[i02] * yd[0] + hourgam1[i02] * yd[1]+
      hourgam2[i02] * yd[2] + hourgam3[i02] * yd[3]+
      hourgam4[i02] * yd[4] + hourgam5[i02] * yd[5]+
      hourgam6[i02] * yd[6] + hourgam7[i02] * yd[7];

   h03 =
      hourgam0[i03] * yd[0] + hourgam1[i03] * yd[1] +
      hourgam2[i03] * yd[2] + hourgam3[i03] * yd[3] +
      hourgam4[i03] * yd[4] + hourgam5[i03] * yd[5] +
      hourgam6[i03] * yd[6] + hourgam7[i03] * yd[7];


   hgfy[0] = coefficient *
      (hourgam0[i00] * h00 + hourgam0[i01] * h01 +
       hourgam0[i02] * h02 + hourgam0[i03] * h03);

   hgfy[1] = coefficient *
      (hourgam1[i00] * h00 + hourgam1[i01] * h01 +
       hourgam1[i02] * h02 + hourgam1[i03] * h03);

   hgfy[2] = coefficient *
      (hourgam2[i00] * h00 + hourgam2[i01] * h01 +
       hourgam2[i02] * h02 + hourgam2[i03] * h03);

   hgfy[3] = coefficient *
      (hourgam3[i00] * h00 + hourgam3[i01] * h01 +
       hourgam3[i02] * h02 + hourgam3[i03] * h03);

   hgfy[4] = coefficient *
      (hourgam4[i00] * h00 + hourgam4[i01] * h01 +
       hourgam4[i02] * h02 + hourgam4[i03] * h03);

   hgfy[5] = coefficient *
      (hourgam5[i00] * h00 + hourgam5[i01] * h01 +
       hourgam5[i02] * h02 + hourgam5[i03] * h03);

   hgfy[6] = coefficient *
      (hourgam6[i00] * h00 + hourgam6[i01] * h01 +
       hourgam6[i02] * h02 + hourgam6[i03] * h03);

   hgfy[7] = coefficient *
      (hourgam7[i00] * h00 + hourgam7[i01] * h01 +
       hourgam7[i02] * h02 + hourgam7[i03] * h03);

   h00 =
      hourgam0[i00] * zd[0] + hourgam1[i00] * zd[1] +
      hourgam2[i00] * zd[2] + hourgam3[i00] * zd[3] +
      hourgam4[i00] * zd[4] + hourgam5[i00] * zd[5] +
      hourgam6[i00] * zd[6] + hourgam7[i00] * zd[7];

   h01 =
      hourgam0[i01] * zd[0] + hourgam1[i01] * zd[1] +
      hourgam2[i01] * zd[2] + hourgam3[i01] * zd[3] +
      hourgam4[i01] * zd[4] + hourgam5[i01] * zd[5] +
      hourgam6[i01] * zd[6] + hourgam7[i01] * zd[7];

   h02 =
      hourgam0[i02] * zd[0] + hourgam1[i02] * zd[1]+
      hourgam2[i02] * zd[2] + hourgam3[i02] * zd[3]+
      hourgam4[i02] * zd[4] + hourgam5[i02] * zd[5]+
      hourgam6[i02] * zd[6] + hourgam7[i02] * zd[7];

   h03 =
      hourgam0[i03] * zd[0] + hourgam1[i03] * zd[1] +
      hourgam2[i03] * zd[2] + hourgam3[i03] * zd[3] +
      hourgam4[i03] * zd[4] + hourgam5[i03] * zd[5] +
      hourgam6[i03] * zd[6] + hourgam7[i03] * zd[7];


   hgfz[0] = coefficient *
      (hourgam0[i00] * h00 + hourgam0[i01] * h01 +
       hourgam0[i02] * h02 + hourgam0[i03] * h03);

   hgfz[1] = coefficient *
      (hourgam1[i00] * h00 + hourgam1[i01] * h01 +
       hourgam1[i02] * h02 + hourgam1[i03] * h03);

   hgfz[2] = coefficient *
      (hourgam2[i00] * h00 + hourgam2[i01] * h01 +
       hourgam2[i02] * h02 + hourgam2[i03] * h03);

   hgfz[3] = coefficient *
      (hourgam3[i00] * h00 + hourgam3[i01] * h01 +
       hourgam3[i02] * h02 + hourgam3[i03] * h03);

   hgfz[4] = coefficient *
      (hourgam4[i00] * h00 + hourgam4[i01] * h01 +
       hourgam4[i02] * h02 + hourgam4[i03] * h03);

   hgfz[5] = coefficient *
      (hourgam5[i00] * h00 + hourgam5[i01] * h01 +
       hourgam5[i02] * h02 + hourgam5[i03] * h03);

   hgfz[6] = coefficient *
      (hourgam6[i00] * h00 + hourgam6[i01] * h01 +
       hourgam6[i02] * h02 + hourgam6[i03] * h03);

   hgfz[7] = coefficient *
      (hourgam7[i00] * h00 + hourgam7[i01] * h01 +
       hourgam7[i02] * h02 + hourgam7[i03] * h03);
}


__shared__ Real_t shm_array[32*8];

__device__
static inline
Real_t SumOverNodes(Real_t val) {
    // Sum up 8 node values for each element
    // Assumes 256 threads: 32 elements, 8 nodes per element.
    // NOTE: we could probably avoid some of the __syncthreads() if we map 8 nodes 
    //       of an element to the same warp.
    unsigned int tid=threadIdx.x;

#if 1
#if 0
    unsigned int node=tid>>5;
    unsigned int elem=tid-(node<<5);
#elif 1
    unsigned int node=tid/32;
    unsigned int elem=tid-(node*32);
#else
    unsigned int elem=tid & 0x1F;
#endif
    __syncthreads();
    shm_array[tid]=val;
    __syncthreads();
    if (tid<128) shm_array[tid]+=shm_array[tid+128];
    __syncthreads();
    if (tid<64)  shm_array[tid]+=shm_array[tid+64];
    __syncthreads();
    if (tid<32)  shm_array[tid]+=shm_array[tid+32];
    __syncthreads();
    Real_t ret=shm_array[elem];
    __syncthreads();
    return ret;
#else
#if 0
    unsigned int node=tid>>5;
    unsigned int elem=tid-(node<<5);
#else
    unsigned int node=tid/32;
    unsigned int elem=tid-(node*32);
#endif
    unsigned int idx=elem*8+node;
    __syncthreads();
    shm_array[idx]=val;
    __syncthreads();
    if (node<4) shm_array[idx]+=shm_array[idx+4];
    if (node<2) shm_array[idx]+=shm_array[idx+2];
    if (node<1) shm_array[idx]+=shm_array[idx+1];
    __syncthreads();
    return shm_array[elem*8];
#endif
}

__device__
static inline
void CalcElemFBHourglassForce(Real_t xd,Real_t yd,Real_t zd,
                              Real_t *hourgam,Real_t coefficient,
                              Real_t &hgfx, Real_t &hgfy, Real_t &hgfz)
{
    hgfx=0;
    for (int i=0;i<4;i++) {
        Real_t h;
        h=hourgam[i]*xd;
        h=SumOverNodes(h);
        hgfx+=hourgam[i]*h;
    }
    hgfx *= coefficient;

    hgfy=0;
    for (int i=0;i<4;i++) {
        Real_t h;
        h=hourgam[i]*yd;
        h=SumOverNodes(h);
        hgfy+=hourgam[i]*h;
    }
    hgfy *= coefficient;

    hgfz=0;
    for (int i=0;i<4;i++) {
        Real_t h;
        h=hourgam[i]*zd;
        h=SumOverNodes(h);
        hgfz+=hourgam[i]*h;
    }
    hgfz *= coefficient;
}

__global__
void CalcFBHourglassForceForElems_kernel(
    Real_t *determ,
    Real_t *x8n,      Real_t *y8n,      Real_t *z8n,
    Real_t *dvdx,     Real_t *dvdy,     Real_t *dvdz,
    Real_t hourg,
    Index_t numElem, Index_t *nodelist,
    Real_t *ss, Real_t *elemMass,
    Real_t *xd, Real_t *yd, Real_t *zd,
    Real_t *fx_elem, Real_t *fy_elem, Real_t *fz_elem)
{
   /*************************************************
    *
    *     FUNCTION: Calculates the Flanagan-Belytschko anti-hourglass
    *               force.
    *
    *************************************************/

    Real_t hgfx, hgfy, hgfz;
    
    Real_t coefficient;
    
    Real_t hourgam[4];
    Real_t xd1, yd1, zd1;
    
/*************************************************/
/*    compute the hourglass modes */
    
    const Real_t posf = Real_t( 1.);
    const Real_t negf = Real_t(-1.);

    // Assume we will launch 256 threads, which we map to 32 elements, each
    // with 8 per-node threads. Organize so each warp of 32 consecutive
    // threads operates on the same node of different elements.
    

    // THESE ARE ALL GIVING ME DIFFERENT ANSWERS IN CUDA 4.0 !!?!!?!!
    unsigned int tid=threadIdx.x;
    unsigned int bid=blockIdx.x;
#if 0
    unsigned int node=tid>>5;
    unsigned int elem=bid<<5 + (tid - (node<<5));
#elif 1
    unsigned int node=tid/32;
    unsigned int elem=bid*32 + (tid-node*32);
#elif 0
    unsigned int node=tid/32;;
    unsigned int elem=bid*32 + (tid & 0x1F);
#elif 0
    unsigned int node=tid/32;
    unsigned int elem=bid<<5 + (tid & 0x1F);
#elif 0
    unsigned int node=tid>>5;
    unsigned int elem=bid*32 + (tid & 0x1F);
#else
    unsigned int node=tid>>5;
    unsigned int elem=bid<<5 + (tid & 0x1F);
#endif

    if (elem>=numElem) elem=numElem-1; // don't return -- need thread to participate in sync operations

    //if (elem<0) elem=0; // debugging test

    Real_t volinv=Real_t(1.0)/determ[elem];
    Real_t ss1, mass1, volume13 ;

    Real_t xn,yn,zn,dvdxn,dvdyn,dvdzn;
    Real_t hourmodx, hourmody, hourmodz;


#if 1
    xn=x8n[elem+numElem*node]; yn=y8n[elem+numElem*node]; zn=z8n[elem+numElem*node];
    dvdxn=dvdx[elem+numElem*node]; dvdyn=dvdy[elem+numElem*node]; dvdzn=dvdz[elem+numElem*node]; 
#else
    xn=yn=zn=posf; dvdxn=dvdyn=dvdzn=negf;
#endif

#if 1
    hourmodx=xn; hourmody=yn; hourmodz=zn;
    if (node==2 || node==3 || node==4 || node==5) {
        hourmodx *= negf; hourmody *= negf; hourmodz *= negf;
        hourgam[0] = negf;
    }
    else hourgam[0] = posf;
    hourmodx = SumOverNodes(hourmodx);
    hourmody = SumOverNodes(hourmody);
    hourmodz = SumOverNodes(hourmodz);
    hourgam[0] -= volinv*(dvdxn*hourmodx + dvdyn*hourmody + dvdzn*hourmodz);

    
    hourmodx=xn; hourmody=yn; hourmodz=zn;
    if (node==1 || node==2 || node==4 || node==7) {
        hourmodx *= negf; hourmody *= negf; hourmodz *= negf;
        hourgam[1] = negf;
    }
    else hourgam[1] = posf;
    hourmodx = SumOverNodes(hourmodx);
    hourmody = SumOverNodes(hourmody);
    hourmodz = SumOverNodes(hourmodz);
    hourgam[1] -= volinv*(dvdxn*hourmodx + dvdyn*hourmody + dvdzn*hourmodz);

    
    hourmodx=xn; hourmody=yn; hourmodz=zn;
    if (node==1 || node==3 || node==5 || node==7) {
        hourmodx *= negf; hourmody *= negf; hourmodz *= negf;
        hourgam[2] = negf;
    }
    else hourgam[2] = posf;
    hourmodx = SumOverNodes(hourmodx);
    hourmody = SumOverNodes(hourmody);
    hourmodz = SumOverNodes(hourmodz);
    hourgam[2] -= volinv*(dvdxn*hourmodx + dvdyn*hourmody + dvdzn*hourmodz);

    
    hourmodx=xn; hourmody=yn; hourmodz=zn;
    if (node==0 || node==2 || node==5 || node==7) {
        hourmodx *= negf; hourmody *= negf; hourmodz *= negf;
        hourgam[3] = negf;
    }
    else hourgam[3] = posf;
    hourmodx = SumOverNodes(hourmodx);
    hourmody = SumOverNodes(hourmody);
    hourmodz = SumOverNodes(hourmodz);
    hourgam[3] -= volinv*(dvdxn*hourmodx + dvdyn*hourmody + dvdzn*hourmodz);
    
    
    /* compute forces */
    /* store forces into h arrays (force arrays) */
    
    ss1=ss[elem];
    mass1=elemMass[elem];
    volume13=CBRT(determ[elem]);
    
    Index_t ni = nodelist[elem+numElem*node];
    xd1=xd[ni]; yd1=yd[ni]; zd1=zd[ni];
    
    coefficient = - hourg * Real_t(0.01) * ss1 * mass1 / volume13;
    
    CalcElemFBHourglassForce(xd1,yd1,zd1,hourgam,coefficient,hgfx,hgfy,hgfz);
#else
    hgfx=xn+dvdxn; hgfy=yn+dvdyn; hgfz=zn+dvdzn;
#endif
#if 1
    fx_elem[elem+numElem*node]=hgfx; fy_elem[elem+numElem*node]=hgfy; fz_elem[elem+numElem*node]=hgfz;
#else
    fx_elem[0]=hgfx; fy_elem[0]=hgfy; fz_elem[0]=hgfz;
#endif
}


static inline
void CalcFBHourglassForceForElems_cpu(Real_t *determ,
            Real_t *x8n,      Real_t *y8n,      Real_t *z8n,
            Real_t *dvdx,     Real_t *dvdy,     Real_t *dvdz,
            Real_t hourg)
{
   /*************************************************
    *
    *     FUNCTION: Calculates the Flanagan-Belytschko anti-hourglass
    *               force.
    *
    *************************************************/

   Index_t numElem = mesh.numElem() ;

   Real_t hgfx[8], hgfy[8], hgfz[8] ;

   Real_t coefficient;

   Real_t  gamma[4][8];
   Real_t hourgam0[4], hourgam1[4], hourgam2[4], hourgam3[4] ;
   Real_t hourgam4[4], hourgam5[4], hourgam6[4], hourgam7[4];
   Real_t xd1[8], yd1[8], zd1[8] ;

   gamma[0][0] = Real_t( 1.);
   gamma[0][1] = Real_t( 1.);
   gamma[0][2] = Real_t(-1.);
   gamma[0][3] = Real_t(-1.);
   gamma[0][4] = Real_t(-1.);
   gamma[0][5] = Real_t(-1.);
   gamma[0][6] = Real_t( 1.);
   gamma[0][7] = Real_t( 1.);
   gamma[1][0] = Real_t( 1.);
   gamma[1][1] = Real_t(-1.);
   gamma[1][2] = Real_t(-1.);
   gamma[1][3] = Real_t( 1.);
   gamma[1][4] = Real_t(-1.);
   gamma[1][5] = Real_t( 1.);
   gamma[1][6] = Real_t( 1.);
   gamma[1][7] = Real_t(-1.);
   gamma[2][0] = Real_t( 1.);
   gamma[2][1] = Real_t(-1.);
   gamma[2][2] = Real_t( 1.);
   gamma[2][3] = Real_t(-1.);
   gamma[2][4] = Real_t( 1.);
   gamma[2][5] = Real_t(-1.);
   gamma[2][6] = Real_t( 1.);
   gamma[2][7] = Real_t(-1.);
   gamma[3][0] = Real_t(-1.);
   gamma[3][1] = Real_t( 1.);
   gamma[3][2] = Real_t(-1.);
   gamma[3][3] = Real_t( 1.);
   gamma[3][4] = Real_t( 1.);
   gamma[3][5] = Real_t(-1.);
   gamma[3][6] = Real_t( 1.);
   gamma[3][7] = Real_t(-1.);

/*************************************************/
/*    compute the hourglass modes */


   for(Index_t i2=0;i2<numElem;++i2){
      Index_t i3=8*i2;
      Real_t volinv=Real_t(1.0)/determ[i2];
      Real_t ss1, mass1, volume13 ;
      for(Index_t i1=0;i1<4;++i1){

         Real_t hourmodx =
            x8n[i3] * gamma[i1][0] + x8n[i3+1] * gamma[i1][1] +
            x8n[i3+2] * gamma[i1][2] + x8n[i3+3] * gamma[i1][3] +
            x8n[i3+4] * gamma[i1][4] + x8n[i3+5] * gamma[i1][5] +
            x8n[i3+6] * gamma[i1][6] + x8n[i3+7] * gamma[i1][7];

         Real_t hourmody =
            y8n[i3] * gamma[i1][0] + y8n[i3+1] * gamma[i1][1] +
            y8n[i3+2] * gamma[i1][2] + y8n[i3+3] * gamma[i1][3] +
            y8n[i3+4] * gamma[i1][4] + y8n[i3+5] * gamma[i1][5] +
            y8n[i3+6] * gamma[i1][6] + y8n[i3+7] * gamma[i1][7];

         Real_t hourmodz =
            z8n[i3] * gamma[i1][0] + z8n[i3+1] * gamma[i1][1] +
            z8n[i3+2] * gamma[i1][2] + z8n[i3+3] * gamma[i1][3] +
            z8n[i3+4] * gamma[i1][4] + z8n[i3+5] * gamma[i1][5] +
            z8n[i3+6] * gamma[i1][6] + z8n[i3+7] * gamma[i1][7];

         hourgam0[i1] = gamma[i1][0] -  volinv*(dvdx[i3  ] * hourmodx +
                                                  dvdy[i3  ] * hourmody +
                                                  dvdz[i3  ] * hourmodz );

         hourgam1[i1] = gamma[i1][1] -  volinv*(dvdx[i3+1] * hourmodx +
                                                  dvdy[i3+1] * hourmody +
                                                  dvdz[i3+1] * hourmodz );

         hourgam2[i1] = gamma[i1][2] -  volinv*(dvdx[i3+2] * hourmodx +
                                                  dvdy[i3+2] * hourmody +
                                                  dvdz[i3+2] * hourmodz );

         hourgam3[i1] = gamma[i1][3] -  volinv*(dvdx[i3+3] * hourmodx +
                                                  dvdy[i3+3] * hourmody +
                                                  dvdz[i3+3] * hourmodz );

         hourgam4[i1] = gamma[i1][4] -  volinv*(dvdx[i3+4] * hourmodx +
                                                  dvdy[i3+4] * hourmody +
                                                  dvdz[i3+4] * hourmodz );

         hourgam5[i1] = gamma[i1][5] -  volinv*(dvdx[i3+5] * hourmodx +
                                                  dvdy[i3+5] * hourmody +
                                                  dvdz[i3+5] * hourmodz );

         hourgam6[i1] = gamma[i1][6] -  volinv*(dvdx[i3+6] * hourmodx +
                                                  dvdy[i3+6] * hourmody +
                                                  dvdz[i3+6] * hourmodz );

         hourgam7[i1] = gamma[i1][7] -  volinv*(dvdx[i3+7] * hourmodx +
                                                  dvdy[i3+7] * hourmody +
                                                  dvdz[i3+7] * hourmodz );

      }

      /* compute forces */
      /* store forces into h arrays (force arrays) */

      ss1=mesh.ss(i2);
      mass1=mesh.elemMass(i2);
      volume13=CBRT(determ[i2]);

      Index_t n0si2 = mesh.nodelist(i2,0);
      Index_t n1si2 = mesh.nodelist(i2,1);
      Index_t n2si2 = mesh.nodelist(i2,2);
      Index_t n3si2 = mesh.nodelist(i2,3);
      Index_t n4si2 = mesh.nodelist(i2,4);
      Index_t n5si2 = mesh.nodelist(i2,5);
      Index_t n6si2 = mesh.nodelist(i2,6);
      Index_t n7si2 = mesh.nodelist(i2,7);

      xd1[0] = mesh.xd(n0si2);
      xd1[1] = mesh.xd(n1si2);
      xd1[2] = mesh.xd(n2si2);
      xd1[3] = mesh.xd(n3si2);
      xd1[4] = mesh.xd(n4si2);
      xd1[5] = mesh.xd(n5si2);
      xd1[6] = mesh.xd(n6si2);
      xd1[7] = mesh.xd(n7si2);

      yd1[0] = mesh.yd(n0si2);
      yd1[1] = mesh.yd(n1si2);
      yd1[2] = mesh.yd(n2si2);
      yd1[3] = mesh.yd(n3si2);
      yd1[4] = mesh.yd(n4si2);
      yd1[5] = mesh.yd(n5si2);
      yd1[6] = mesh.yd(n6si2);
      yd1[7] = mesh.yd(n7si2);

      zd1[0] = mesh.zd(n0si2);
      zd1[1] = mesh.zd(n1si2);
      zd1[2] = mesh.zd(n2si2);
      zd1[3] = mesh.zd(n3si2);
      zd1[4] = mesh.zd(n4si2);
      zd1[5] = mesh.zd(n5si2);
      zd1[6] = mesh.zd(n6si2);
      zd1[7] = mesh.zd(n7si2);

      coefficient = - hourg * Real_t(0.01) * ss1 * mass1 / volume13;

      CalcElemFBHourglassForce(xd1,yd1,zd1,
                      hourgam0,hourgam1,hourgam2,hourgam3,
                      hourgam4,hourgam5,hourgam6,hourgam7,
                      coefficient, hgfx, hgfy, hgfz);

      mesh.fx(n0si2) += hgfx[0];
      mesh.fy(n0si2) += hgfy[0];
      mesh.fz(n0si2) += hgfz[0];

      mesh.fx(n1si2) += hgfx[1];
      mesh.fy(n1si2) += hgfy[1];
      mesh.fz(n1si2) += hgfz[1];

      mesh.fx(n2si2) += hgfx[2];
      mesh.fy(n2si2) += hgfy[2];
      mesh.fz(n2si2) += hgfz[2];

      mesh.fx(n3si2) += hgfx[3];
      mesh.fy(n3si2) += hgfy[3];
      mesh.fz(n3si2) += hgfz[3];

      mesh.fx(n4si2) += hgfx[4];
      mesh.fy(n4si2) += hgfy[4];
      mesh.fz(n4si2) += hgfz[4];

      mesh.fx(n5si2) += hgfx[5];
      mesh.fy(n5si2) += hgfy[5];
      mesh.fz(n5si2) += hgfz[5];

      mesh.fx(n6si2) += hgfx[6];
      mesh.fy(n6si2) += hgfy[6];
      mesh.fz(n6si2) += hgfz[6];

      mesh.fx(n7si2) += hgfx[7];
      mesh.fy(n7si2) += hgfy[7];
      mesh.fz(n7si2) += hgfz[7];
   }
}

static inline
void CalcFBHourglassForceForElems_gpu(Real_t *determ,
            Real_t *x8n,      Real_t *y8n,      Real_t *z8n,
            Real_t *dvdx,     Real_t *dvdy,     Real_t *dvdz,
            Real_t hourg)
{
    Index_t numElem = mesh.numElem();
    Real_t *fx_elem,*fy_elem,*fz_elem;
    
    CUDA( hipMalloc(&fx_elem,numElem*8*sizeof(Real_t)) );
    CUDA( hipMalloc(&fy_elem,numElem*8*sizeof(Real_t)) );
    CUDA( hipMalloc(&fz_elem,numElem*8*sizeof(Real_t)) );
    
    dim3 dimBlock=dim3(256,1,1);
    dim3 dimGrid=dim3(PAD_DIV(numElem*8,dimBlock.x),1,1);
    CalcFBHourglassForceForElems_kernel<<<dimGrid,dimBlock>>>(
        determ,x8n,y8n,z8n,dvdx,dvdy,dvdz,hourg,
        numElem,meshGPU.m_nodelist,
        meshGPU.m_ss,meshGPU.m_elemMass,
        meshGPU.m_xd,meshGPU.m_yd,meshGPU.m_zd,
        fx_elem,fy_elem,fz_elem);
    CUDA_DEBUGSYNC;
    
    dimGrid=dim3(PAD_DIV(mesh.numNode(),dimBlock.x),1,1);
    AddNodeForcesFromElems2_kernel<<<dimGrid,dimBlock>>>
        (mesh.numNode(),meshGPU.m_nodeElemCount,meshGPU.m_nodeElemCornerList,
         fx_elem,fy_elem,fz_elem,meshGPU.m_fx,meshGPU.m_fy,meshGPU.m_fz);
    CUDA_DEBUGSYNC;

    CUDA( hipFree(fx_elem) );
    CUDA( hipFree(fy_elem) );
    CUDA( hipFree(fz_elem) );
}


__global__
void CalcHourglassControlForElems_kernel(Int_t numElem,Index_t *nodelist,
                                         Real_t *x,Real_t *y,Real_t *z,
                                         Real_t *determ,Real_t *volo,Real_t *v,
                                         Real_t *dvdx,Real_t *dvdy,Real_t *dvdz,
                                         Real_t *x8n,Real_t *y8n,Real_t *z8n)
{
    Real_t  x1,y1,z1;
    Real_t pfx,pfy,pfz;
  
    // THESE ARE ALL GIVING ME DIFFERENT ANSWERS IN CUDA 4.0 !!?!!?!!
    unsigned int tid=threadIdx.x;
    unsigned int bid=blockIdx.x;
#if 0
    unsigned int node=tid>>5;
    unsigned int elem=bid<<5 + (tid - (node<<5));
#elif 1
    unsigned int node=tid/32;
    unsigned int elem=bid*32 + (tid-node*32);
#elif 0
    unsigned int node=tid/32;;
    unsigned int elem=bid*32 + (tid & 0x1F);
#elif 0
    unsigned int node=tid/32;
    unsigned int elem=bid<<5 + (tid & 0x1F);
#elif 0
    unsigned int node=tid>>5;
    unsigned int elem=bid*32 + (tid & 0x1F);
#else
    unsigned int node=tid>>5;
    unsigned int elem=bid<<5 + (tid & 0x1F);
#endif
    
    if (elem>=numElem) elem=numElem-1; // don't return -- need thread to participate in sync operations

    Index_t idx=elem+numElem*node;

    Index_t ni = nodelist[idx];
    x1=x[ni]; y1=y[ni]; z1=z[ni];
    
    CalcElemVolumeDerivative(pfx, pfy, pfz, x1, y1, z1, node);
    
    /* load into temporary storage for FB Hour Glass control */
      
    dvdx[idx] = pfx;
    dvdy[idx] = pfy;
    dvdz[idx] = pfz;
    x8n[idx]  = x1;
    y8n[idx]  = y1;
    z8n[idx]  = z1;
    
    //if (node==0)
      determ[elem] = volo[elem] * v[elem];
    
#if 0 // JDC
      /* Do a check for negative volumes */
    if ( mesh.v(i) <= Real_t(0.0) ) {
      exit(VolumeError) ;
    }
#endif
}


static inline
void CalcHourglassControlForElems_gpu(Real_t determ[], Real_t hgcoef)
{
   Index_t numElem = mesh.numElem() ;
   Index_t numElem8 = numElem * 8 ;
   Real_t *dvdx,*dvdy,*dvdz;
   Real_t *x8n,*y8n,*z8n;

   CUDA( hipMalloc(&dvdx,sizeof(Real_t)*numElem8) );
   CUDA( hipMalloc(&dvdy,sizeof(Real_t)*numElem8) );
   CUDA( hipMalloc(&dvdz,sizeof(Real_t)*numElem8) );
   CUDA( hipMalloc(&x8n,sizeof(Real_t)*numElem8) );
   CUDA( hipMalloc(&y8n,sizeof(Real_t)*numElem8) );
   CUDA( hipMalloc(&z8n,sizeof(Real_t)*numElem8) );

   dim3 dimBlock=dim3(256,1,1);
   dim3 dimGrid=dim3(PAD_DIV(numElem*8,dimBlock.x),1,1);
   CalcHourglassControlForElems_kernel<<<dimGrid,dimBlock>>>
       (numElem, meshGPU.m_nodelist,
        meshGPU.m_x,meshGPU.m_y,meshGPU.m_z,
        determ,meshGPU.m_volo,meshGPU.m_v,
        dvdx,dvdy,dvdz,x8n,y8n,z8n);
   CUDA_DEBUGSYNC;
   
   // JDC -- need a reduction to check for negative volumes

   if ( hgcoef > Real_t(0.) ) {
       CalcFBHourglassForceForElems_gpu(determ,x8n,y8n,z8n,dvdx,dvdy,dvdz,hgcoef) ;
   }
   
   CUDA( hipFree(dvdx) );
   CUDA( hipFree(dvdy) );
   CUDA( hipFree(dvdz) );
   CUDA( hipFree(x8n) );
   CUDA( hipFree(y8n) );
   CUDA( hipFree(z8n) );
   
   return ;
}


static inline
void CalcHourglassControlForElems_cpu(Real_t determ[], Real_t hgcoef)
{
   Index_t i, ii, jj ;
   Real_t  x1[8],  y1[8],  z1[8] ;
   Real_t pfx[8], pfy[8], pfz[8] ;
   Index_t numElem = mesh.numElem() ;
   Index_t numElem8 = numElem * 8 ;
   Real_t *dvdx = Allocate<Real_t>(numElem8) ;
   Real_t *dvdy = Allocate<Real_t>(numElem8) ;
   Real_t *dvdz = Allocate<Real_t>(numElem8) ;
   Real_t *x8n  = Allocate<Real_t>(numElem8) ;
   Real_t *y8n  = Allocate<Real_t>(numElem8) ;
   Real_t *z8n  = Allocate<Real_t>(numElem8) ;

   /* start loop over elements */
   for (i=0 ; i<numElem ; ++i){

      CollectDomainNodesToElemNodes(i, x1, y1, z1);

      CalcElemVolumeDerivative(pfx, pfy, pfz, x1, y1, z1);

      /* load into temporary storage for FB Hour Glass control */
      for(ii=0;ii<8;++ii){
         jj=8*i+ii;

         dvdx[jj] = pfx[ii];
         dvdy[jj] = pfy[ii];
         dvdz[jj] = pfz[ii];
         x8n[jj]  = x1[ii];
         y8n[jj]  = y1[ii];
         z8n[jj]  = z1[ii];
      }

      determ[i] = mesh.volo(i) * mesh.v(i);

      /* Do a check for negative volumes */
      if ( mesh.v(i) <= Real_t(0.0) ) {
         exit(VolumeError) ;
      }
   }

   if ( hgcoef > Real_t(0.) ) {
      CalcFBHourglassForceForElems_cpu(determ,x8n,y8n,z8n,dvdx,dvdy,dvdz,hgcoef) ;
   }

   Release(&z8n) ;
   Release(&y8n) ;
   Release(&x8n) ;
   Release(&dvdz) ;
   Release(&dvdy) ;
   Release(&dvdx) ;

   return ;
}


static inline
void CalcHourglassControlForElems(Real_t determ[], Real_t hgcoef, int useCPU)
{
    if (useCPU) {
        FC(x); FC(y); FC(z); FC(xd); FC(yd); FC(zd);
        FC(nodelist); FC(ss); FC(elemMass);
        FC(xd); FC(yd); FC(zd);
        FC(fx); FC(fy); FC(fz);
        CalcHourglassControlForElems_cpu(determ,hgcoef);
        SG(fx); SG(fy); SG(fz);
    }
    else {
        FG(x); FG(y); FG(z); FG(xd); FG(yd); FG(zd);
        FG(nodelist); FG(ss); FG(elemMass);
        FG(xd); FG(yd); FG(zd); 
        FG(fx); FG(fy); FG(fz);
        CalcHourglassControlForElems_gpu(determ,hgcoef);
        SC(fx); SC(fy); SC(fz);
    }
}


static inline
void CalcVolumeForceForElems_gpu()
{
   Index_t numElem = mesh.numElem() ;
   if (numElem != 0) {
      Real_t  hgcoef = mesh.hgcoef() ;
      Real_t *sigxx, *sigyy, *sigzz, *determ;
      int badvol;
      
      CUDA( hipMalloc(&sigxx,numElem*sizeof(Real_t)) );
      CUDA( hipMalloc(&sigyy,numElem*sizeof(Real_t)) );
      CUDA( hipMalloc(&sigzz,numElem*sizeof(Real_t)) );
      CUDA( hipMalloc(&determ,numElem*sizeof(Real_t)) );

      /* Sum contributions to total stress tensor */
      InitStressTermsForElems(numElem, sigxx, sigyy, sigzz, 0);

      // call elemlib stress integration loop to produce nodal forces from
      // material stresses.
      IntegrateStressForElems( numElem, sigxx, sigyy, sigzz, determ, badvol, 0) ;
      
      CUDA( hipFree(sigxx) );
      CUDA( hipFree(sigyy) );
      CUDA( hipFree(sigzz) );
      
      // check for negative element volume
      if (badvol) exit(VolumeError) ;

      CalcHourglassControlForElems(determ, hgcoef, 0) ;

      CUDA( hipFree(determ) );
   }
}


static inline
void CalcVolumeForceForElems_cpu()
{
   Index_t numElem = mesh.numElem() ;
   if (numElem != 0) {
      Real_t  hgcoef = mesh.hgcoef() ;
      Real_t *sigxx  = Allocate<Real_t>(numElem) ;
      Real_t *sigyy  = Allocate<Real_t>(numElem) ;
      Real_t *sigzz  = Allocate<Real_t>(numElem) ;
      Real_t *determ = Allocate<Real_t>(numElem) ;
      int badvol;
      
      /* Sum contributions to total stress tensor */
      InitStressTermsForElems(numElem, sigxx, sigyy, sigzz, 1);

      // call elemlib stress integration loop to produce nodal forces from
      // material stresses.
      IntegrateStressForElems( numElem, sigxx, sigyy, sigzz, determ, badvol, 1) ;
      
      Release(&sigzz) ;
      Release(&sigyy) ;
      Release(&sigxx) ;
      
      // check for negative element volume
      if (badvol) exit(VolumeError);
#if 0
      for ( Index_t k=0 ; k<numElem ; ++k ) {
         if (determ[k] <= Real_t(0.0)) {
            exit(VolumeError) ;
         }
      }
#endif
      
      CalcHourglassControlForElems(determ, hgcoef, 1) ;

      Release(&determ) ;
   }
}

static inline void CalcForceForNodes_gpu()
{
  /* Calcforce calls partial, force, hourq */
  CalcVolumeForceForElems_gpu() ;

  /* Calculate Nodal Forces at domain boundaries */
  /* problem->commSBN->Transfer(CommSBN::forces); */
  
  
}

static inline void CalcForceForNodes_cpu()
{
  Index_t numNode = mesh.numNode() ;
  for (Index_t i=0; i<numNode; ++i) {
     mesh.fx(i) = Real_t(0.0) ;
     mesh.fy(i) = Real_t(0.0) ;
     mesh.fz(i) = Real_t(0.0) ;
  }

  /* Calcforce calls partial, force, hourq */
  CalcVolumeForceForElems_cpu() ;

  /* Calculate Nodal Forces at domain boundaries */
  /* problem->commSBN->Transfer(CommSBN::forces); */

}

static inline void CalcForceForNodes(int useCPU)
{
    if (useCPU) {
        CalcForceForNodes_cpu();
    }
    else {
        CalcForceForNodes_gpu();
    }
}

__global__
void CalcAccelerationForNodes_kernel(int numNode,
                                     Real_t *xdd, Real_t *ydd, Real_t *zdd,
                                     Real_t *fx, Real_t *fy, Real_t *fz,
                                     Real_t *nodalMass)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numNode) {
        xdd[i]=fx[i]/nodalMass[i];
        ydd[i]=fy[i]/nodalMass[i];
        zdd[i]=fz[i]/nodalMass[i];
    }
}

static inline
void CalcAccelerationForNodes_gpu()
{
    dim3 dimBlock = dim3(BLOCKSIZE,1,1);
    dim3 dimGrid = dim3(PAD_DIV(mesh.numNode(),dimBlock.x),1,1);
    CalcAccelerationForNodes_kernel<<<dimGrid, dimBlock>>>
        (mesh.numNode(),
         meshGPU.m_xdd,meshGPU.m_ydd,meshGPU.m_zdd,
         meshGPU.m_fx,meshGPU.m_fy,meshGPU.m_fz,
         meshGPU.m_nodalMass);
    CUDA_DEBUGSYNC;
}


static inline
void CalcAccelerationForNodes_cpu()
{
   Index_t numNode = mesh.numNode() ;
   for (Index_t i = 0; i < numNode; ++i) {
      mesh.xdd(i) = mesh.fx(i) / mesh.nodalMass(i);
      mesh.ydd(i) = mesh.fy(i) / mesh.nodalMass(i);
      mesh.zdd(i) = mesh.fz(i) / mesh.nodalMass(i);
   }
}

static inline
void CalcAccelerationForNodes(int useCPU)
{
    if (useCPU) {
        FC(fx); FC(fy); FC(fz); FC(nodalMass);
        CalcAccelerationForNodes_cpu();
        SG(xdd); SG(ydd); SG(zdd);
    }
    else {
        FG(fx); FG(fy); FG(fz); FG(nodalMass);
        CalcAccelerationForNodes_gpu();
        SC(xdd); SC(ydd); SC(zdd);
    }
}

__global__
void ApplyAccelerationBoundaryConditionsForNodes_kernel(
    int numNodeBC, Real_t *xdd, Real_t *ydd, Real_t *zdd,
    Index_t *symmX, Index_t *symmY, Index_t *symmZ)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numNodeBC) {
        xdd[symmX[i]] = Real_t(0.0) ;
        ydd[symmY[i]] = Real_t(0.0) ;
        zdd[symmZ[i]] = Real_t(0.0) ;
    }
}

static inline
void ApplyAccelerationBoundaryConditionsForNodes_gpu()
{
    Index_t numNodeBC = (mesh.sizeX()+1)*(mesh.sizeX()+1) ;
    dim3 dimBlock(BLOCKSIZE,1,1);
    dim3 dimGrid(PAD_DIV(numNodeBC,dimBlock.x),1,1);
    ApplyAccelerationBoundaryConditionsForNodes_kernel<<<dimGrid, dimBlock>>>
        (numNodeBC,
         meshGPU.m_xdd,meshGPU.m_ydd,meshGPU.m_zdd,
         meshGPU.m_symmX,meshGPU.m_symmY,meshGPU.m_symmZ);
    CUDA_DEBUGSYNC;
}

static inline
void ApplyAccelerationBoundaryConditionsForNodes_cpu()
{
  Index_t numNodeBC = (mesh.sizeX()+1)*(mesh.sizeX()+1) ;
  for(Index_t i=0 ; i<numNodeBC ; ++i)
     mesh.xdd(mesh.symmX(i)) = Real_t(0.0) ;

  for(Index_t i=0 ; i<numNodeBC ; ++i)
     mesh.ydd(mesh.symmY(i)) = Real_t(0.0) ;

  for(Index_t i=0 ; i<numNodeBC ; ++i)
     mesh.zdd(mesh.symmZ(i)) = Real_t(0.0) ;
}

static inline
void ApplyAccelerationBoundaryConditionsForNodes(int useCPU)
{
    if (useCPU) {
        FC(xdd); FC(ydd); FC(zdd); FC(symmX); FC(symmY); FC(symmZ);
        ApplyAccelerationBoundaryConditionsForNodes_cpu();
        SG(xdd); SG(ydd); SG(zdd);
    }
    else {
        FG(xdd); FG(ydd); FG(zdd); FG(symmX); FG(symmY); FG(symmZ);
        ApplyAccelerationBoundaryConditionsForNodes_gpu();
        SC(xdd); SC(ydd); SC(zdd);
    }
}


__global__
void CalcVelocityForNodes_kernel(int numNode, const Real_t dt, const Real_t u_cut,
                                 Real_t *xd, Real_t *yd, Real_t *zd,
                                 Real_t *xdd, Real_t *ydd, Real_t *zdd)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numNode) {
        Real_t xdtmp, ydtmp, zdtmp ;
        
        xdtmp = xd[i] + xdd[i] * dt ;
        if( FABS(xdtmp) < u_cut ) xdtmp = 0.0;//Real_t(0.0);
        xd[i] = xdtmp ;
        
        ydtmp = yd[i] + ydd[i] * dt ;
        if( FABS(ydtmp) < u_cut ) ydtmp = Real_t(0.0);
        yd[i] = ydtmp ;
        
        zdtmp = zd[i] + zdd[i] * dt ;
        if( FABS(zdtmp) < u_cut ) zdtmp = Real_t(0.0);
        zd[i] = zdtmp ;
    }
}

static inline
void CalcVelocityForNodes_gpu(const Real_t dt, const Real_t u_cut)
{
    dim3 dimBlock(BLOCKSIZE,1,1);
    dim3 dimGrid(PAD_DIV(mesh.numNode(),dimBlock.x),1,1);
    CalcVelocityForNodes_kernel<<<dimGrid, dimBlock>>>
        (mesh.numNode(),dt,u_cut,
         meshGPU.m_xd,meshGPU.m_yd,meshGPU.m_zd,
         meshGPU.m_xdd,meshGPU.m_ydd,meshGPU.m_zdd);
    CUDA_DEBUGSYNC;
}

static inline
void CalcVelocityForNodes_cpu(const Real_t dt, const Real_t u_cut)
{
   Index_t numNode = mesh.numNode() ;

   for ( Index_t i = 0 ; i < numNode ; ++i )
   {
     Real_t xdtmp, ydtmp, zdtmp ;

     xdtmp = mesh.xd(i) + mesh.xdd(i) * dt ;
     if( FABS(xdtmp) < u_cut ) xdtmp = Real_t(0.0);
     mesh.xd(i) = xdtmp ;

     ydtmp = mesh.yd(i) + mesh.ydd(i) * dt ;
     if( FABS(ydtmp) < u_cut ) ydtmp = Real_t(0.0);
     mesh.yd(i) = ydtmp ;

     zdtmp = mesh.zd(i) + mesh.zdd(i) * dt ;
     if( FABS(zdtmp) < u_cut ) zdtmp = Real_t(0.0);
     mesh.zd(i) = zdtmp ;
   }
}

static inline
void CalcVelocityForNodes(const Real_t dt, const Real_t u_cut, int useCPU)
{
    if (useCPU) {
        FC(xd); FC(yd); FC(zd); FC(xdd); FC(ydd); FC(zdd);
        CalcVelocityForNodes_cpu(dt,u_cut);
        SG(xd); SG(yd); SG(zd);
    }
    else {
        FG(xd); FG(yd); FG(zd); FG(xdd); FG(ydd); FG(zdd);
        CalcVelocityForNodes_gpu(dt,u_cut);
        SC(xd); SC(yd); SC(zd);
    }
}

__global__
void CalcPositionForNodes_kernel(int numNode, Real_t dt,
                                 Real_t *x, Real_t *y, Real_t *z,
                                 Real_t *xd, Real_t *yd, Real_t *zd)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numNode) {
        x[i] += xd[i] * dt;
        y[i] += yd[i] * dt;
        z[i] += zd[i] * dt;
    }
}

static inline
void CalcPositionForNodes_gpu(const Real_t dt)
{
    dim3 dimBlock(BLOCKSIZE,1,1);
    dim3 dimGrid(PAD_DIV(mesh.numNode(),dimBlock.x),1,1);
    CalcPositionForNodes_kernel<<<dimGrid, dimBlock>>>
        (mesh.numNode(),dt,meshGPU.m_x,meshGPU.m_y,meshGPU.m_z,meshGPU.m_xd,meshGPU.m_yd,meshGPU.m_zd);
    CUDA_DEBUGSYNC;
}

static inline
void CalcPositionForNodes_cpu(const Real_t dt)
{
   Index_t numNode = mesh.numNode() ;

   for ( Index_t i = 0 ; i < numNode ; ++i )
   {
     mesh.x(i) += mesh.xd(i) * dt ;
     mesh.y(i) += mesh.yd(i) * dt ;
     mesh.z(i) += mesh.zd(i) * dt ;
   }
}

static inline
void CalcPositionForNodes(const Real_t dt,int useCPU)
{
    if (useCPU) {
        FC(x); FC(y); FC(z); FC(xd); FC(yd); FC(zd);
        CalcPositionForNodes_cpu(dt);
        SG(x); SG(y); SG(z);
    }
    else {
        FG(x); FG(y); FG(z); FG(xd); FG(yd); FG(zd);
        CalcPositionForNodes_gpu(dt);
        SC(x); SC(y); SC(z);
    }
}

static inline
void LagrangeNodal(int useCPU)
{
  const Real_t delt = mesh.deltatime() ;
  Real_t u_cut = mesh.u_cut() ;

  /* time of boundary condition evaluation is beginning of step for force and
   * acceleration boundary conditions. */
  CalcForceForNodes(/*0*/useCPU);

  CalcAccelerationForNodes(useCPU);

  ApplyAccelerationBoundaryConditionsForNodes(useCPU);

  CalcVelocityForNodes( delt, u_cut, useCPU ) ;

  CalcPositionForNodes( delt, useCPU );

  return;
}

__host__ __device__
static inline
Real_t CalcElemVolume( const Real_t x0, const Real_t x1,
               const Real_t x2, const Real_t x3,
               const Real_t x4, const Real_t x5,
               const Real_t x6, const Real_t x7,
               const Real_t y0, const Real_t y1,
               const Real_t y2, const Real_t y3,
               const Real_t y4, const Real_t y5,
               const Real_t y6, const Real_t y7,
               const Real_t z0, const Real_t z1,
               const Real_t z2, const Real_t z3,
               const Real_t z4, const Real_t z5,
               const Real_t z6, const Real_t z7 )
{
  Real_t twelveth = Real_t(1.0)/Real_t(12.0);

  Real_t dx61 = x6 - x1;
  Real_t dy61 = y6 - y1;
  Real_t dz61 = z6 - z1;

  Real_t dx70 = x7 - x0;
  Real_t dy70 = y7 - y0;
  Real_t dz70 = z7 - z0;

  Real_t dx63 = x6 - x3;
  Real_t dy63 = y6 - y3;
  Real_t dz63 = z6 - z3;

  Real_t dx20 = x2 - x0;
  Real_t dy20 = y2 - y0;
  Real_t dz20 = z2 - z0;

  Real_t dx50 = x5 - x0;
  Real_t dy50 = y5 - y0;
  Real_t dz50 = z5 - z0;

  Real_t dx64 = x6 - x4;
  Real_t dy64 = y6 - y4;
  Real_t dz64 = z6 - z4;

  Real_t dx31 = x3 - x1;
  Real_t dy31 = y3 - y1;
  Real_t dz31 = z3 - z1;

  Real_t dx72 = x7 - x2;
  Real_t dy72 = y7 - y2;
  Real_t dz72 = z7 - z2;

  Real_t dx43 = x4 - x3;
  Real_t dy43 = y4 - y3;
  Real_t dz43 = z4 - z3;

  Real_t dx57 = x5 - x7;
  Real_t dy57 = y5 - y7;
  Real_t dz57 = z5 - z7;

  Real_t dx14 = x1 - x4;
  Real_t dy14 = y1 - y4;
  Real_t dz14 = z1 - z4;

  Real_t dx25 = x2 - x5;
  Real_t dy25 = y2 - y5;
  Real_t dz25 = z2 - z5;

#define TRIPLE_PRODUCT(x1, y1, z1, x2, y2, z2, x3, y3, z3) \
   ((x1)*((y2)*(z3) - (z2)*(y3)) + (x2)*((z1)*(y3) - (y1)*(z3)) + (x3)*((y1)*(z2) - (z1)*(y2)))

  Real_t volume =
    TRIPLE_PRODUCT(dx31 + dx72, dx63, dx20,
       dy31 + dy72, dy63, dy20,
       dz31 + dz72, dz63, dz20) +
    TRIPLE_PRODUCT(dx43 + dx57, dx64, dx70,
       dy43 + dy57, dy64, dy70,
       dz43 + dz57, dz64, dz70) +
    TRIPLE_PRODUCT(dx14 + dx25, dx61, dx50,
       dy14 + dy25, dy61, dy50,
       dz14 + dz25, dz61, dz50);

#undef TRIPLE_PRODUCT

  volume *= twelveth;

  return volume ;
}

__host__ __device__
static inline
Real_t CalcElemVolume( const Real_t x[8], const Real_t y[8], const Real_t z[8] )
{
return CalcElemVolume( x[0], x[1], x[2], x[3], x[4], x[5], x[6], x[7],
                       y[0], y[1], y[2], y[3], y[4], y[5], y[6], y[7],
                       z[0], z[1], z[2], z[3], z[4], z[5], z[6], z[7]);
}

__host__ __device__
static inline
Real_t AreaFace( const Real_t x0, const Real_t x1,
                 const Real_t x2, const Real_t x3,
                 const Real_t y0, const Real_t y1,
                 const Real_t y2, const Real_t y3,
                 const Real_t z0, const Real_t z1,
                 const Real_t z2, const Real_t z3)
{
   Real_t fx = (x2 - x0) - (x3 - x1);
   Real_t fy = (y2 - y0) - (y3 - y1);
   Real_t fz = (z2 - z0) - (z3 - z1);
   Real_t gx = (x2 - x0) + (x3 - x1);
   Real_t gy = (y2 - y0) + (y3 - y1);
   Real_t gz = (z2 - z0) + (z3 - z1);
   Real_t area =
      (fx * fx + fy * fy + fz * fz) *
      (gx * gx + gy * gy + gz * gz) -
      (fx * gx + fy * gy + fz * gz) *
      (fx * gx + fy * gy + fz * gz);
   return area ;
}

__host__ __device__
static inline
Real_t CalcElemCharacteristicLength( const Real_t x[8],
                                     const Real_t y[8],
                                     const Real_t z[8],
                                     const Real_t volume)
{
   Real_t a, charLength = Real_t(0.0);

   a = AreaFace(x[0],x[1],x[2],x[3],
                y[0],y[1],y[2],y[3],
                z[0],z[1],z[2],z[3]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[4],x[5],x[6],x[7],
                y[4],y[5],y[6],y[7],
                z[4],z[5],z[6],z[7]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[0],x[1],x[5],x[4],
                y[0],y[1],y[5],y[4],
                z[0],z[1],z[5],z[4]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[1],x[2],x[6],x[5],
                y[1],y[2],y[6],y[5],
                z[1],z[2],z[6],z[5]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[2],x[3],x[7],x[6],
                y[2],y[3],y[7],y[6],
                z[2],z[3],z[7],z[6]) ;
   charLength = FMAX(a,charLength) ;

   a = AreaFace(x[3],x[0],x[4],x[7],
                y[3],y[0],y[4],y[7],
                z[3],z[0],z[4],z[7]) ;
   charLength = FMAX(a,charLength) ;

   charLength = Real_t(4.0) * volume / SQRT(charLength);

   return charLength;
}

__host__ __device__
static inline
void CalcElemVelocityGradient( const Real_t* const xvel,
                                const Real_t* const yvel,
                                const Real_t* const zvel,
                                const Real_t b[][8],
                                const Real_t detJ,
                                Real_t* const d )
{
  const Real_t inv_detJ = Real_t(1.0) / detJ ;
  Real_t dyddx, dxddy, dzddx, dxddz, dzddy, dyddz;
  const Real_t* const pfx = b[0];
  const Real_t* const pfy = b[1];
  const Real_t* const pfz = b[2];

  d[0] = inv_detJ * ( pfx[0] * (xvel[0]-xvel[6])
                     + pfx[1] * (xvel[1]-xvel[7])
                     + pfx[2] * (xvel[2]-xvel[4])
                     + pfx[3] * (xvel[3]-xvel[5]) );

  d[1] = inv_detJ * ( pfy[0] * (yvel[0]-yvel[6])
                     + pfy[1] * (yvel[1]-yvel[7])
                     + pfy[2] * (yvel[2]-yvel[4])
                     + pfy[3] * (yvel[3]-yvel[5]) );

  d[2] = inv_detJ * ( pfz[0] * (zvel[0]-zvel[6])
                     + pfz[1] * (zvel[1]-zvel[7])
                     + pfz[2] * (zvel[2]-zvel[4])
                     + pfz[3] * (zvel[3]-zvel[5]) );

  dyddx  = inv_detJ * ( pfx[0] * (yvel[0]-yvel[6])
                      + pfx[1] * (yvel[1]-yvel[7])
                      + pfx[2] * (yvel[2]-yvel[4])
                      + pfx[3] * (yvel[3]-yvel[5]) );

  dxddy  = inv_detJ * ( pfy[0] * (xvel[0]-xvel[6])
                      + pfy[1] * (xvel[1]-xvel[7])
                      + pfy[2] * (xvel[2]-xvel[4])
                      + pfy[3] * (xvel[3]-xvel[5]) );

  dzddx  = inv_detJ * ( pfx[0] * (zvel[0]-zvel[6])
                      + pfx[1] * (zvel[1]-zvel[7])
                      + pfx[2] * (zvel[2]-zvel[4])
                      + pfx[3] * (zvel[3]-zvel[5]) );

  dxddz  = inv_detJ * ( pfz[0] * (xvel[0]-xvel[6])
                      + pfz[1] * (xvel[1]-xvel[7])
                      + pfz[2] * (xvel[2]-xvel[4])
                      + pfz[3] * (xvel[3]-xvel[5]) );

  dzddy  = inv_detJ * ( pfy[0] * (zvel[0]-zvel[6])
                      + pfy[1] * (zvel[1]-zvel[7])
                      + pfy[2] * (zvel[2]-zvel[4])
                      + pfy[3] * (zvel[3]-zvel[5]) );

  dyddz  = inv_detJ * ( pfz[0] * (yvel[0]-yvel[6])
                      + pfz[1] * (yvel[1]-yvel[7])
                      + pfz[2] * (yvel[2]-yvel[4])
                      + pfz[3] * (yvel[3]-yvel[5]) );
  d[5]  = Real_t( .5) * ( dxddy + dyddx );
  d[4]  = Real_t( .5) * ( dxddz + dzddx );
  d[3]  = Real_t( .5) * ( dzddy + dyddz );
}

__global__
void CalcKinematicsForElems_kernel(
    Index_t numElem, Real_t dt,
    Index_t *nodelist,Real_t *volo,Real_t *v,
    Real_t *x,Real_t *y,Real_t *z,Real_t *xd,Real_t *yd,Real_t *zd,
    Real_t *vnew,Real_t *delv,Real_t *arealg,Real_t *dxx,Real_t *dyy,Real_t *dzz
    )
{
  Real_t B[3][8] ; /** shape function derivatives */
  Real_t D[6] ;
  Real_t x_local[8] ;
  Real_t y_local[8] ;
  Real_t z_local[8] ;
  Real_t xd_local[8] ;
  Real_t yd_local[8] ;
  Real_t zd_local[8] ;
  Real_t detJ = Real_t(0.0) ;

  int k=blockDim.x*blockIdx.x + threadIdx.x;
  if (k<numElem) {

    Real_t volume ;
    Real_t relativeVolume ;

    // get nodal coordinates from global arrays and copy into local arrays.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = nodelist[k+lnode*numElem];
      x_local[lnode] = x[gnode];
      y_local[lnode] = y[gnode];
      z_local[lnode] = z[gnode];
    }

    // volume calculations
    volume = CalcElemVolume(x_local, y_local, z_local );
    relativeVolume = volume / volo[k] ;
    vnew[k] = relativeVolume ;
    delv[k] = relativeVolume - v[k] ;

    // set characteristic length
    arealg[k] = CalcElemCharacteristicLength(x_local,y_local,z_local,volume);

    // get nodal velocities from global array and copy into local arrays.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = nodelist[k+lnode*numElem];
      xd_local[lnode] = xd[gnode];
      yd_local[lnode] = yd[gnode];
      zd_local[lnode] = zd[gnode];
    }

    Real_t dt2 = Real_t(0.5) * dt;
    for ( Index_t j=0 ; j<8 ; ++j )
    {
       x_local[j] -= dt2 * xd_local[j];
       y_local[j] -= dt2 * yd_local[j];
       z_local[j] -= dt2 * zd_local[j];
    }

    CalcElemShapeFunctionDerivatives(x_local,y_local,z_local,B,&detJ );

    CalcElemVelocityGradient(xd_local,yd_local,zd_local,B,detJ,D);

    // put velocity gradient quantities into their global arrays.
    dxx[k] = D[0];
    dyy[k] = D[1];
    dzz[k] = D[2];
  }
}


static inline
void CalcKinematicsForElems_gpu( Index_t numElem, Real_t dt )
{
    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(numElem,dimBlock.x),1,1);
    CalcKinematicsForElems_kernel<<<dimGrid,dimBlock>>>
        (numElem,dt,meshGPU.m_nodelist,meshGPU.m_volo,meshGPU.m_v,
         meshGPU.m_x,meshGPU.m_y,meshGPU.m_z,meshGPU.m_xd,meshGPU.m_yd,meshGPU.m_zd,
         meshGPU.m_vnew,meshGPU.m_delv,meshGPU.m_arealg,meshGPU.m_dxx,meshGPU.m_dyy,meshGPU.m_dzz);
    CUDA_DEBUGSYNC;
}


static inline
void CalcKinematicsForElems_cpu( Index_t numElem, Real_t dt )
{
  Real_t B[3][8] ; /** shape function derivatives */
  Real_t D[6] ;
  Real_t x_local[8] ;
  Real_t y_local[8] ;
  Real_t z_local[8] ;
  Real_t xd_local[8] ;
  Real_t yd_local[8] ;
  Real_t zd_local[8] ;
  Real_t detJ = Real_t(0.0) ;

  // loop over all elements
  for( Index_t k=0 ; k<numElem ; ++k )
  {
    Real_t volume ;
    Real_t relativeVolume ;

    // get nodal coordinates from global arrays and copy into local arrays.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = mesh.nodelist(k,lnode);
      x_local[lnode] = mesh.x(gnode);
      y_local[lnode] = mesh.y(gnode);
      z_local[lnode] = mesh.z(gnode);
    }

    // volume calculations
    volume = CalcElemVolume(x_local, y_local, z_local );
    relativeVolume = volume / mesh.volo(k) ;
    mesh.vnew(k) = relativeVolume ;
    mesh.delv(k) = relativeVolume - mesh.v(k) ;

    // set characteristic length
    mesh.arealg(k) = CalcElemCharacteristicLength(x_local,
                                                  y_local,
                                                  z_local,
                                                  volume);

    // get nodal velocities from global array and copy into local arrays.
    for( Index_t lnode=0 ; lnode<8 ; ++lnode )
    {
      Index_t gnode = mesh.nodelist(k,lnode);
      xd_local[lnode] = mesh.xd(gnode);
      yd_local[lnode] = mesh.yd(gnode);
      zd_local[lnode] = mesh.zd(gnode);
    }

    Real_t dt2 = Real_t(0.5) * dt;
    for ( Index_t j=0 ; j<8 ; ++j )
    {
       x_local[j] -= dt2 * xd_local[j];
       y_local[j] -= dt2 * yd_local[j];
       z_local[j] -= dt2 * zd_local[j];
    }

    CalcElemShapeFunctionDerivatives( x_local,
                                      y_local,
                                      z_local,
                                      B, &detJ );

    CalcElemVelocityGradient( xd_local,
                               yd_local,
                               zd_local,
                               B, detJ, D );

    // put velocity gradient quantities into their global arrays.
    mesh.dxx(k) = D[0];
    mesh.dyy(k) = D[1];
    mesh.dzz(k) = D[2];
  }
}


static inline
void CalcKinematicsForElems( Index_t numElem, Real_t dt, int useCPU )
{
    if (useCPU) {
        FC(nodelist); FC(volo); FC(v); FC(x); FC(y); FC(z); FC(xd); FC(yd); FC(zd);
        CalcKinematicsForElems_cpu(numElem,dt);
        SG(vnew); SG(delv); SG(arealg); SG(dxx); SG(dyy); SG(dzz);
    }
    else {
        FG(nodelist); FG(volo); FG(v); FG(x); FG(y); FG(z); FG(xd); FG(yd); FG(zd);
        CalcKinematicsForElems_gpu(numElem,dt);
        SC(vnew); SC(delv); SC(arealg); SC(dxx); SC(dyy); SC(dzz);
    }
}


__global__
void CalcLagrangeElementsPart2_kernel(
    Index_t numElem,
    Real_t *dxx,Real_t *dyy, Real_t *dzz,
    Real_t *vdov
    )
{
    int k=blockDim.x*blockIdx.x + threadIdx.x;
    if (k<numElem) {

        // calc strain rate and apply as constraint (only done in FB element)
        Real_t vdovNew = dxx[k] + dyy[k] + dzz[k] ;
        Real_t vdovthird = vdovNew/Real_t(3.0) ;
        
        // make the rate of deformation tensor deviatoric
        vdov[k] = vdovNew ;
        dxx[k] -= vdovthird ;
        dyy[k] -= vdovthird ;
        dzz[k] -= vdovthird ;
        
        // See if any volumes are negative, and take appropriate action.
        //if (mesh.vnew(k) <= Real_t(0.0))
        //{
        //    exit(VolumeError) ;
        //}
    }
}


static inline
void CalcLagrangeElementsPart2_gpu()
{
    Index_t numElem = mesh.numElem();
    
    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(numElem,dimBlock.x),1,1);
    CalcLagrangeElementsPart2_kernel<<<dimGrid,dimBlock>>>
        (numElem,
         meshGPU.m_dxx,meshGPU.m_dyy,meshGPU.m_dzz,
         meshGPU.m_vdov);
    CUDA_DEBUGSYNC;
}


static inline
void CalcLagrangeElementsPart2_cpu()
{
   Index_t numElem = mesh.numElem() ;

   // element loop to do some stuff not included in the elemlib function.
   for ( Index_t k=0 ; k<numElem ; ++k )
   {
       // calc strain rate and apply as constraint (only done in FB element)
       Real_t vdov = mesh.dxx(k) + mesh.dyy(k) + mesh.dzz(k) ;
       Real_t vdovthird = vdov/Real_t(3.0) ;
       
       // make the rate of deformation tensor deviatoric
       mesh.vdov(k) = vdov ;
       mesh.dxx(k) -= vdovthird ;
       mesh.dyy(k) -= vdovthird ;
       mesh.dzz(k) -= vdovthird ;
       
       // See if any volumes are negative, and take appropriate action.
       if (mesh.vnew(k) <= Real_t(0.0))
       {
           exit(VolumeError) ;
       }
   }
}


static inline
void CalcLagrangeElementsPart2(int useCPU)
{
    if (useCPU) {
        FC(dxx); FC(dyy); FC(dzz);
        CalcLagrangeElementsPart2_cpu();
        SG(vdov); SG(dxx); SG(dyy); SG(dzz);
    }
    else {
        FG(dxx); FG(dyy); FG(dzz);
        CalcLagrangeElementsPart2_gpu();
        SC(vdov); SC(dxx); SC(dyy); SC(dzz);
    }
}

static inline
void CalcLagrangeElements(Real_t deltatime, int useCPU)
{
   Index_t numElem = mesh.numElem() ;
   if (numElem > 0) {
       CalcKinematicsForElems(numElem, deltatime, useCPU);
       CalcLagrangeElementsPart2(useCPU);
   }
}


__global__
void CalcMonotonicQGradientsForElems_kernel(
    Index_t numElem,
    Index_t *nodelist,
    Real_t *x,Real_t *y,Real_t *z,Real_t *xd,Real_t *yd,Real_t *zd,
    Real_t *volo,Real_t *vnew,
    Real_t *delx_zeta,Real_t *delv_zeta,
    Real_t *delx_xi,Real_t *delv_xi,
    Real_t *delx_eta,Real_t *delv_eta
    )
{
#define SUM4(a,b,c,d) (a + b + c + d)
   const Real_t ptiny = Real_t(1.e-36) ;

   int i=blockDim.x*blockIdx.x + threadIdx.x;
   if (i<numElem) {
      Real_t ax,ay,az ;
      Real_t dxv,dyv,dzv ;

      Index_t n0 = nodelist[i+0*numElem] ;
      Index_t n1 = nodelist[i+1*numElem] ;
      Index_t n2 = nodelist[i+2*numElem] ;
      Index_t n3 = nodelist[i+3*numElem] ;
      Index_t n4 = nodelist[i+4*numElem] ;
      Index_t n5 = nodelist[i+5*numElem] ;
      Index_t n6 = nodelist[i+6*numElem] ;
      Index_t n7 = nodelist[i+7*numElem] ;

      Real_t x0 = x[n0] ;
      Real_t x1 = x[n1] ;
      Real_t x2 = x[n2] ;
      Real_t x3 = x[n3] ;
      Real_t x4 = x[n4] ;
      Real_t x5 = x[n5] ;
      Real_t x6 = x[n6] ;
      Real_t x7 = x[n7] ;

      Real_t y0 = y[n0] ;
      Real_t y1 = y[n1] ;
      Real_t y2 = y[n2] ;
      Real_t y3 = y[n3] ;
      Real_t y4 = y[n4] ;
      Real_t y5 = y[n5] ;
      Real_t y6 = y[n6] ;
      Real_t y7 = y[n7] ;

      Real_t z0 = z[n0] ;
      Real_t z1 = z[n1] ;
      Real_t z2 = z[n2] ;
      Real_t z3 = z[n3] ;
      Real_t z4 = z[n4] ;
      Real_t z5 = z[n5] ;
      Real_t z6 = z[n6] ;
      Real_t z7 = z[n7] ;

      Real_t xv0 = xd[n0] ;
      Real_t xv1 = xd[n1] ;
      Real_t xv2 = xd[n2] ;
      Real_t xv3 = xd[n3] ;
      Real_t xv4 = xd[n4] ;
      Real_t xv5 = xd[n5] ;
      Real_t xv6 = xd[n6] ;
      Real_t xv7 = xd[n7] ;

      Real_t yv0 = yd[n0] ;
      Real_t yv1 = yd[n1] ;
      Real_t yv2 = yd[n2] ;
      Real_t yv3 = yd[n3] ;
      Real_t yv4 = yd[n4] ;
      Real_t yv5 = yd[n5] ;
      Real_t yv6 = yd[n6] ;
      Real_t yv7 = yd[n7] ;

      Real_t zv0 = zd[n0] ;
      Real_t zv1 = zd[n1] ;
      Real_t zv2 = zd[n2] ;
      Real_t zv3 = zd[n3] ;
      Real_t zv4 = zd[n4] ;
      Real_t zv5 = zd[n5] ;
      Real_t zv6 = zd[n6] ;
      Real_t zv7 = zd[n7] ;

      Real_t vol = volo[i]*vnew[i] ;
      Real_t norm = Real_t(1.0) / ( vol + ptiny ) ;

      Real_t dxj = Real_t(-0.25)*(SUM4(x0,x1,x5,x4) - SUM4(x3,x2,x6,x7)) ;
      Real_t dyj = Real_t(-0.25)*(SUM4(y0,y1,y5,y4) - SUM4(y3,y2,y6,y7)) ;
      Real_t dzj = Real_t(-0.25)*(SUM4(z0,z1,z5,z4) - SUM4(z3,z2,z6,z7)) ;

      Real_t dxi = Real_t( 0.25)*(SUM4(x1,x2,x6,x5) - SUM4(x0,x3,x7,x4)) ;
      Real_t dyi = Real_t( 0.25)*(SUM4(y1,y2,y6,y5) - SUM4(y0,y3,y7,y4)) ;
      Real_t dzi = Real_t( 0.25)*(SUM4(z1,z2,z6,z5) - SUM4(z0,z3,z7,z4)) ;

      Real_t dxk = Real_t( 0.25)*(SUM4(x4,x5,x6,x7) - SUM4(x0,x1,x2,x3)) ;
      Real_t dyk = Real_t( 0.25)*(SUM4(y4,y5,y6,y7) - SUM4(y0,y1,y2,y3)) ;
      Real_t dzk = Real_t( 0.25)*(SUM4(z4,z5,z6,z7) - SUM4(z0,z1,z2,z3)) ;

      /* find delvk and delxk ( i cross j ) */

      ax = dyi*dzj - dzi*dyj ;
      ay = dzi*dxj - dxi*dzj ;
      az = dxi*dyj - dyi*dxj ;

      delx_zeta[i] = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

      ax *= norm ;
      ay *= norm ;
      az *= norm ;

      dxv = Real_t(0.25)*(SUM4(xv4,xv5,xv6,xv7) - SUM4(xv0,xv1,xv2,xv3)) ;
      dyv = Real_t(0.25)*(SUM4(yv4,yv5,yv6,yv7) - SUM4(yv0,yv1,yv2,yv3)) ;
      dzv = Real_t(0.25)*(SUM4(zv4,zv5,zv6,zv7) - SUM4(zv0,zv1,zv2,zv3)) ;

      delv_zeta[i] = ax*dxv + ay*dyv + az*dzv ;

      /* find delxi and delvi ( j cross k ) */

      ax = dyj*dzk - dzj*dyk ;
      ay = dzj*dxk - dxj*dzk ;
      az = dxj*dyk - dyj*dxk ;

      delx_xi[i] = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

      ax *= norm ;
      ay *= norm ;
      az *= norm ;

      dxv = Real_t(0.25)*(SUM4(xv1,xv2,xv6,xv5) - SUM4(xv0,xv3,xv7,xv4)) ;
      dyv = Real_t(0.25)*(SUM4(yv1,yv2,yv6,yv5) - SUM4(yv0,yv3,yv7,yv4)) ;
      dzv = Real_t(0.25)*(SUM4(zv1,zv2,zv6,zv5) - SUM4(zv0,zv3,zv7,zv4)) ;

      delv_xi[i] = ax*dxv + ay*dyv + az*dzv ;

      /* find delxj and delvj ( k cross i ) */

      ax = dyk*dzi - dzk*dyi ;
      ay = dzk*dxi - dxk*dzi ;
      az = dxk*dyi - dyk*dxi ;

      delx_eta[i] = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

      ax *= norm ;
      ay *= norm ;
      az *= norm ;

      dxv = Real_t(-0.25)*(SUM4(xv0,xv1,xv5,xv4) - SUM4(xv3,xv2,xv6,xv7)) ;
      dyv = Real_t(-0.25)*(SUM4(yv0,yv1,yv5,yv4) - SUM4(yv3,yv2,yv6,yv7)) ;
      dzv = Real_t(-0.25)*(SUM4(zv0,zv1,zv5,zv4) - SUM4(zv3,zv2,zv6,zv7)) ;

      delv_eta[i] = ax*dxv + ay*dyv + az*dzv ;
   }
#undef SUM4
}


static inline
void CalcMonotonicQGradientsForElems_gpu()
{
    Index_t numElem = mesh.numElem();
    
    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(numElem,dimBlock.x),1,1);
    CalcMonotonicQGradientsForElems_kernel<<<dimGrid,dimBlock>>>
        (numElem,
         meshGPU.m_nodelist,
         meshGPU.m_x,meshGPU.m_y,meshGPU.m_z,meshGPU.m_xd,meshGPU.m_yd,meshGPU.m_zd,
         meshGPU.m_volo,meshGPU.m_vnew,
         meshGPU.m_delx_zeta,meshGPU.m_delv_zeta,
         meshGPU.m_delx_xi,meshGPU.m_delv_xi,
         meshGPU.m_delx_eta,meshGPU.m_delv_eta);
    CUDA_DEBUGSYNC;
}


static inline
void CalcMonotonicQGradientsForElems_cpu()
{
#define SUM4(a,b,c,d) (a + b + c + d)
   Index_t numElem = mesh.numElem() ;
   const Real_t ptiny = Real_t(1.e-36) ;

   for (Index_t i = 0 ; i < numElem ; ++i ) {
      Real_t ax,ay,az ;
      Real_t dxv,dyv,dzv ;

      Index_t n0 = mesh.nodelist(i,0) ;
      Index_t n1 = mesh.nodelist(i,1) ;
      Index_t n2 = mesh.nodelist(i,2) ;
      Index_t n3 = mesh.nodelist(i,3) ;
      Index_t n4 = mesh.nodelist(i,4) ;
      Index_t n5 = mesh.nodelist(i,5) ;
      Index_t n6 = mesh.nodelist(i,6) ;
      Index_t n7 = mesh.nodelist(i,7) ;

      Real_t x0 = mesh.x(n0) ;
      Real_t x1 = mesh.x(n1) ;
      Real_t x2 = mesh.x(n2) ;
      Real_t x3 = mesh.x(n3) ;
      Real_t x4 = mesh.x(n4) ;
      Real_t x5 = mesh.x(n5) ;
      Real_t x6 = mesh.x(n6) ;
      Real_t x7 = mesh.x(n7) ;

      Real_t y0 = mesh.y(n0) ;
      Real_t y1 = mesh.y(n1) ;
      Real_t y2 = mesh.y(n2) ;
      Real_t y3 = mesh.y(n3) ;
      Real_t y4 = mesh.y(n4) ;
      Real_t y5 = mesh.y(n5) ;
      Real_t y6 = mesh.y(n6) ;
      Real_t y7 = mesh.y(n7) ;

      Real_t z0 = mesh.z(n0) ;
      Real_t z1 = mesh.z(n1) ;
      Real_t z2 = mesh.z(n2) ;
      Real_t z3 = mesh.z(n3) ;
      Real_t z4 = mesh.z(n4) ;
      Real_t z5 = mesh.z(n5) ;
      Real_t z6 = mesh.z(n6) ;
      Real_t z7 = mesh.z(n7) ;

      Real_t xv0 = mesh.xd(n0) ;
      Real_t xv1 = mesh.xd(n1) ;
      Real_t xv2 = mesh.xd(n2) ;
      Real_t xv3 = mesh.xd(n3) ;
      Real_t xv4 = mesh.xd(n4) ;
      Real_t xv5 = mesh.xd(n5) ;
      Real_t xv6 = mesh.xd(n6) ;
      Real_t xv7 = mesh.xd(n7) ;

      Real_t yv0 = mesh.yd(n0) ;
      Real_t yv1 = mesh.yd(n1) ;
      Real_t yv2 = mesh.yd(n2) ;
      Real_t yv3 = mesh.yd(n3) ;
      Real_t yv4 = mesh.yd(n4) ;
      Real_t yv5 = mesh.yd(n5) ;
      Real_t yv6 = mesh.yd(n6) ;
      Real_t yv7 = mesh.yd(n7) ;

      Real_t zv0 = mesh.zd(n0) ;
      Real_t zv1 = mesh.zd(n1) ;
      Real_t zv2 = mesh.zd(n2) ;
      Real_t zv3 = mesh.zd(n3) ;
      Real_t zv4 = mesh.zd(n4) ;
      Real_t zv5 = mesh.zd(n5) ;
      Real_t zv6 = mesh.zd(n6) ;
      Real_t zv7 = mesh.zd(n7) ;

      Real_t vol = mesh.volo(i)*mesh.vnew(i) ;
      Real_t norm = Real_t(1.0) / ( vol + ptiny ) ;

      Real_t dxj = Real_t(-0.25)*(SUM4(x0,x1,x5,x4) - SUM4(x3,x2,x6,x7)) ;
      Real_t dyj = Real_t(-0.25)*(SUM4(y0,y1,y5,y4) - SUM4(y3,y2,y6,y7)) ;
      Real_t dzj = Real_t(-0.25)*(SUM4(z0,z1,z5,z4) - SUM4(z3,z2,z6,z7)) ;

      Real_t dxi = Real_t( 0.25)*(SUM4(x1,x2,x6,x5) - SUM4(x0,x3,x7,x4)) ;
      Real_t dyi = Real_t( 0.25)*(SUM4(y1,y2,y6,y5) - SUM4(y0,y3,y7,y4)) ;
      Real_t dzi = Real_t( 0.25)*(SUM4(z1,z2,z6,z5) - SUM4(z0,z3,z7,z4)) ;

      Real_t dxk = Real_t( 0.25)*(SUM4(x4,x5,x6,x7) - SUM4(x0,x1,x2,x3)) ;
      Real_t dyk = Real_t( 0.25)*(SUM4(y4,y5,y6,y7) - SUM4(y0,y1,y2,y3)) ;
      Real_t dzk = Real_t( 0.25)*(SUM4(z4,z5,z6,z7) - SUM4(z0,z1,z2,z3)) ;

      /* find delvk and delxk ( i cross j ) */

      ax = dyi*dzj - dzi*dyj ;
      ay = dzi*dxj - dxi*dzj ;
      az = dxi*dyj - dyi*dxj ;

      mesh.delx_zeta(i) = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

      ax *= norm ;
      ay *= norm ;
      az *= norm ;

      dxv = Real_t(0.25)*(SUM4(xv4,xv5,xv6,xv7) - SUM4(xv0,xv1,xv2,xv3)) ;
      dyv = Real_t(0.25)*(SUM4(yv4,yv5,yv6,yv7) - SUM4(yv0,yv1,yv2,yv3)) ;
      dzv = Real_t(0.25)*(SUM4(zv4,zv5,zv6,zv7) - SUM4(zv0,zv1,zv2,zv3)) ;

      mesh.delv_zeta(i) = ax*dxv + ay*dyv + az*dzv ;

      /* find delxi and delvi ( j cross k ) */

      ax = dyj*dzk - dzj*dyk ;
      ay = dzj*dxk - dxj*dzk ;
      az = dxj*dyk - dyj*dxk ;

      mesh.delx_xi(i) = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

      ax *= norm ;
      ay *= norm ;
      az *= norm ;

      dxv = Real_t(0.25)*(SUM4(xv1,xv2,xv6,xv5) - SUM4(xv0,xv3,xv7,xv4)) ;
      dyv = Real_t(0.25)*(SUM4(yv1,yv2,yv6,yv5) - SUM4(yv0,yv3,yv7,yv4)) ;
      dzv = Real_t(0.25)*(SUM4(zv1,zv2,zv6,zv5) - SUM4(zv0,zv3,zv7,zv4)) ;

      mesh.delv_xi(i) = ax*dxv + ay*dyv + az*dzv ;

      /* find delxj and delvj ( k cross i ) */

      ax = dyk*dzi - dzk*dyi ;
      ay = dzk*dxi - dxk*dzi ;
      az = dxk*dyi - dyk*dxi ;

      mesh.delx_eta(i) = vol / SQRT(ax*ax + ay*ay + az*az + ptiny) ;

      ax *= norm ;
      ay *= norm ;
      az *= norm ;

      dxv = Real_t(-0.25)*(SUM4(xv0,xv1,xv5,xv4) - SUM4(xv3,xv2,xv6,xv7)) ;
      dyv = Real_t(-0.25)*(SUM4(yv0,yv1,yv5,yv4) - SUM4(yv3,yv2,yv6,yv7)) ;
      dzv = Real_t(-0.25)*(SUM4(zv0,zv1,zv5,zv4) - SUM4(zv3,zv2,zv6,zv7)) ;

      mesh.delv_eta(i) = ax*dxv + ay*dyv + az*dzv ;
   }
#undef SUM4
}


static inline
void CalcMonotonicQGradientsForElems(int useCPU)
{
    if (useCPU) {
        FC(nodelist); FC(x); FC(y); FC(z); FC(xd); FC(yd); FC(zd); FC(volo); FC(vnew);
        CalcMonotonicQGradientsForElems_cpu();
        SG(delx_zeta); SG(delv_zeta); SG(delx_xi); SG(delv_xi); SG(delx_eta); SG(delv_eta);
    }
    else {
        FG(nodelist); FG(x); FG(y); FG(z); FG(xd); FG(yd); FG(zd); FG(volo); FG(vnew);
        CalcMonotonicQGradientsForElems_gpu();
        SC(delx_zeta); SC(delv_zeta); SC(delx_xi); SC(delv_xi); SC(delx_eta); SC(delv_eta);
    }
}


__global__
void CalcMonotonicQRegionForElems_kernel(
    Real_t qlc_monoq,
    Real_t qqc_monoq,
    Real_t monoq_limiter_mult,
    Real_t monoq_max_slope,
    Real_t ptiny,
    
    // the elementset length
    Index_t elength,
    
    Index_t *matElemlist,Index_t *elemBC,
    Index_t *lxim,Index_t *lxip,
    Index_t *letam,Index_t *letap,
    Index_t *lzetam,Index_t *lzetap,
    Real_t *delv_xi,Real_t *delv_eta,Real_t *delv_zeta,
    Real_t *delx_xi,Real_t *delx_eta,Real_t *delx_zeta,
    Real_t *vdov,Real_t *elemMass,Real_t *volo,Real_t *vnew,
    Real_t *qq,Real_t *ql
    )
{
    int ielem=blockDim.x*blockIdx.x + threadIdx.x;
    if (ielem<elength) {
      Real_t qlin, qquad ;
      Real_t phixi, phieta, phizeta ;
      Index_t i = matElemlist[ielem];
      Int_t bcMask = elemBC[i] ;
      Real_t delvm, delvp ;

      /*  phixi     */
      Real_t norm = Real_t(1.) / ( delv_xi[i] + ptiny ) ;

      switch (bcMask & XI_M) {
         case 0:         delvm = delv_xi[lxim[i]] ; break ;
         case XI_M_SYMM: delvm = delv_xi[i] ;            break ;
         case XI_M_FREE: delvm = Real_t(0.0) ;                break ;
         default:        /* ERROR */ ;                        break ;
      }
      switch (bcMask & XI_P) {
         case 0:         delvp = delv_xi[lxip[i]] ; break ;
         case XI_P_SYMM: delvp = delv_xi[i] ;            break ;
         case XI_P_FREE: delvp = Real_t(0.0) ;                break ;
         default:        /* ERROR */ ;                        break ;
      }

      delvm = delvm * norm ;
      delvp = delvp * norm ;

      phixi = Real_t(.5) * ( delvm + delvp ) ;

      delvm *= monoq_limiter_mult ;
      delvp *= monoq_limiter_mult ;

      if ( delvm < phixi ) phixi = delvm ;
      if ( delvp < phixi ) phixi = delvp ;
      if ( phixi < Real_t(0.)) phixi = Real_t(0.) ;
      if ( phixi > monoq_max_slope) phixi = monoq_max_slope;


      /*  phieta     */
      norm = Real_t(1.) / ( delv_eta[i] + ptiny ) ;

      switch (bcMask & ETA_M) {
         case 0:          delvm = delv_eta[letam[i]] ; break ;
         case ETA_M_SYMM: delvm = delv_eta[i] ;             break ;
         case ETA_M_FREE: delvm = Real_t(0.0) ;                  break ;
         default:         /* ERROR */ ;                          break ;
      }
      switch (bcMask & ETA_P) {
         case 0:          delvp = delv_eta[letap[i]] ; break ;
         case ETA_P_SYMM: delvp = delv_eta[i] ;             break ;
         case ETA_P_FREE: delvp = Real_t(0.0) ;                  break ;
         default:         /* ERROR */ ;                          break ;
      }

      delvm = delvm * norm ;
      delvp = delvp * norm ;

      phieta = Real_t(.5) * ( delvm + delvp ) ;

      delvm *= monoq_limiter_mult ;
      delvp *= monoq_limiter_mult ;

      if ( delvm  < phieta ) phieta = delvm ;
      if ( delvp  < phieta ) phieta = delvp ;
      if ( phieta < Real_t(0.)) phieta = Real_t(0.) ;
      if ( phieta > monoq_max_slope)  phieta = monoq_max_slope;

      /*  phizeta     */
      norm = Real_t(1.) / ( delv_zeta[i] + ptiny ) ;

      switch (bcMask & ZETA_M) {
         case 0:           delvm = delv_zeta[lzetam[i]] ; break ;
         case ZETA_M_SYMM: delvm = delv_zeta[i] ;              break ;
         case ZETA_M_FREE: delvm = Real_t(0.0) ;                    break ;
         default:          /* ERROR */ ;                            break ;
      }
      switch (bcMask & ZETA_P) {
         case 0:           delvp = delv_zeta[lzetap[i]] ; break ;
         case ZETA_P_SYMM: delvp = delv_zeta[i] ;              break ;
         case ZETA_P_FREE: delvp = Real_t(0.0) ;                    break ;
         default:          /* ERROR */ ;                            break ;
      }

      delvm = delvm * norm ;
      delvp = delvp * norm ;

      phizeta = Real_t(.5) * ( delvm + delvp ) ;

      delvm *= monoq_limiter_mult ;
      delvp *= monoq_limiter_mult ;

      if ( delvm   < phizeta ) phizeta = delvm ;
      if ( delvp   < phizeta ) phizeta = delvp ;
      if ( phizeta < Real_t(0.)) phizeta = Real_t(0.);
      if ( phizeta > monoq_max_slope  ) phizeta = monoq_max_slope;

      /* Remove length scale */

      if ( vdov[i] > Real_t(0.) )  {
         qlin  = Real_t(0.) ;
         qquad = Real_t(0.) ;
      }
      else {
         Real_t delvxxi   = delv_xi[i]   * delx_xi[i]   ;
         Real_t delvxeta  = delv_eta[i]  * delx_eta[i]  ;
         Real_t delvxzeta = delv_zeta[i] * delx_zeta[i] ;

         if ( delvxxi   > Real_t(0.) ) delvxxi   = Real_t(0.) ;
         if ( delvxeta  > Real_t(0.) ) delvxeta  = Real_t(0.) ;
         if ( delvxzeta > Real_t(0.) ) delvxzeta = Real_t(0.) ;

         Real_t rho = elemMass[i] / (volo[i] * vnew[i]) ;

         qlin = -qlc_monoq * rho *
            (  delvxxi   * (Real_t(1.) - phixi) +
               delvxeta  * (Real_t(1.) - phieta) +
               delvxzeta * (Real_t(1.) - phizeta)  ) ;

         qquad = qqc_monoq * rho *
            (  delvxxi*delvxxi     * (Real_t(1.) - phixi*phixi) +
               delvxeta*delvxeta   * (Real_t(1.) - phieta*phieta) +
               delvxzeta*delvxzeta * (Real_t(1.) - phizeta*phizeta)  ) ;
      }

      qq[i] = qquad ;
      ql[i] = qlin  ;
   }
}


static inline
void CalcMonotonicQRegionForElems_gpu(// parameters
                          Real_t qlc_monoq,
                          Real_t qqc_monoq,
                          Real_t monoq_limiter_mult,
                          Real_t monoq_max_slope,
                          Real_t ptiny,

                          // the elementset length
                          Index_t elength )
{
    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(elength,dimBlock.x),1,1);
    CalcMonotonicQRegionForElems_kernel<<<dimGrid,dimBlock>>>
        (qlc_monoq,qqc_monoq,monoq_limiter_mult,monoq_max_slope,ptiny,elength,
         meshGPU.m_matElemlist,meshGPU.m_elemBC,
         meshGPU.m_lxim,meshGPU.m_lxip,
         meshGPU.m_letam,meshGPU.m_letap,
         meshGPU.m_lzetam,meshGPU.m_lzetap,
         meshGPU.m_delv_xi,meshGPU.m_delv_eta,meshGPU.m_delv_zeta,
         meshGPU.m_delx_xi,meshGPU.m_delx_eta,meshGPU.m_delx_zeta,
         meshGPU.m_vdov,meshGPU.m_elemMass,meshGPU.m_volo,meshGPU.m_vnew,
         meshGPU.m_qq,meshGPU.m_ql);
    CUDA_DEBUGSYNC;
}


static inline
void CalcMonotonicQRegionForElems_cpu(// parameters
                          Real_t qlc_monoq,
                          Real_t qqc_monoq,
                          Real_t monoq_limiter_mult,
                          Real_t monoq_max_slope,
                          Real_t ptiny,

                          // the elementset length
                          Index_t elength )
{
   for ( Index_t ielem = 0 ; ielem < elength; ++ielem ) {
      Real_t qlin, qquad ;
      Real_t phixi, phieta, phizeta ;
      Index_t i = mesh.matElemlist(ielem);
      Int_t bcMask = mesh.elemBC(i) ;
      Real_t delvm, delvp ;

      /*  phixi     */
      Real_t norm = Real_t(1.) / ( mesh.delv_xi(i) + ptiny ) ;

      switch (bcMask & XI_M) {
         case 0:         delvm = mesh.delv_xi(mesh.lxim(i)) ; break ;
         case XI_M_SYMM: delvm = mesh.delv_xi(i) ;            break ;
         case XI_M_FREE: delvm = Real_t(0.0) ;                break ;
         default:        /* ERROR */ ;                        break ;
      }
      switch (bcMask & XI_P) {
         case 0:         delvp = mesh.delv_xi(mesh.lxip(i)) ; break ;
         case XI_P_SYMM: delvp = mesh.delv_xi(i) ;            break ;
         case XI_P_FREE: delvp = Real_t(0.0) ;                break ;
         default:        /* ERROR */ ;                        break ;
      }

      delvm = delvm * norm ;
      delvp = delvp * norm ;

      phixi = Real_t(.5) * ( delvm + delvp ) ;

      delvm *= monoq_limiter_mult ;
      delvp *= monoq_limiter_mult ;

      if ( delvm < phixi ) phixi = delvm ;
      if ( delvp < phixi ) phixi = delvp ;
      if ( phixi < Real_t(0.)) phixi = Real_t(0.) ;
      if ( phixi > monoq_max_slope) phixi = monoq_max_slope;


      /*  phieta     */
      norm = Real_t(1.) / ( mesh.delv_eta(i) + ptiny ) ;

      switch (bcMask & ETA_M) {
         case 0:          delvm = mesh.delv_eta(mesh.letam(i)) ; break ;
         case ETA_M_SYMM: delvm = mesh.delv_eta(i) ;             break ;
         case ETA_M_FREE: delvm = Real_t(0.0) ;                  break ;
         default:         /* ERROR */ ;                          break ;
      }
      switch (bcMask & ETA_P) {
         case 0:          delvp = mesh.delv_eta(mesh.letap(i)) ; break ;
         case ETA_P_SYMM: delvp = mesh.delv_eta(i) ;             break ;
         case ETA_P_FREE: delvp = Real_t(0.0) ;                  break ;
         default:         /* ERROR */ ;                          break ;
      }

      delvm = delvm * norm ;
      delvp = delvp * norm ;

      phieta = Real_t(.5) * ( delvm + delvp ) ;

      delvm *= monoq_limiter_mult ;
      delvp *= monoq_limiter_mult ;

      if ( delvm  < phieta ) phieta = delvm ;
      if ( delvp  < phieta ) phieta = delvp ;
      if ( phieta < Real_t(0.)) phieta = Real_t(0.) ;
      if ( phieta > monoq_max_slope)  phieta = monoq_max_slope;

      /*  phizeta     */
      norm = Real_t(1.) / ( mesh.delv_zeta(i) + ptiny ) ;

      switch (bcMask & ZETA_M) {
         case 0:           delvm = mesh.delv_zeta(mesh.lzetam(i)) ; break ;
         case ZETA_M_SYMM: delvm = mesh.delv_zeta(i) ;              break ;
         case ZETA_M_FREE: delvm = Real_t(0.0) ;                    break ;
         default:          /* ERROR */ ;                            break ;
      }
      switch (bcMask & ZETA_P) {
         case 0:           delvp = mesh.delv_zeta(mesh.lzetap(i)) ; break ;
         case ZETA_P_SYMM: delvp = mesh.delv_zeta(i) ;              break ;
         case ZETA_P_FREE: delvp = Real_t(0.0) ;                    break ;
         default:          /* ERROR */ ;                            break ;
      }

      delvm = delvm * norm ;
      delvp = delvp * norm ;

      phizeta = Real_t(.5) * ( delvm + delvp ) ;

      delvm *= monoq_limiter_mult ;
      delvp *= monoq_limiter_mult ;

      if ( delvm   < phizeta ) phizeta = delvm ;
      if ( delvp   < phizeta ) phizeta = delvp ;
      if ( phizeta < Real_t(0.)) phizeta = Real_t(0.);
      if ( phizeta > monoq_max_slope  ) phizeta = monoq_max_slope;

      /* Remove length scale */

      if ( mesh.vdov(i) > Real_t(0.) )  {
         qlin  = Real_t(0.) ;
         qquad = Real_t(0.) ;
      }
      else {
         Real_t delvxxi   = mesh.delv_xi(i)   * mesh.delx_xi(i)   ;
         Real_t delvxeta  = mesh.delv_eta(i)  * mesh.delx_eta(i)  ;
         Real_t delvxzeta = mesh.delv_zeta(i) * mesh.delx_zeta(i) ;

         if ( delvxxi   > Real_t(0.) ) delvxxi   = Real_t(0.) ;
         if ( delvxeta  > Real_t(0.) ) delvxeta  = Real_t(0.) ;
         if ( delvxzeta > Real_t(0.) ) delvxzeta = Real_t(0.) ;

         Real_t rho = mesh.elemMass(i) / (mesh.volo(i) * mesh.vnew(i)) ;

         qlin = -qlc_monoq * rho *
            (  delvxxi   * (Real_t(1.) - phixi) +
               delvxeta  * (Real_t(1.) - phieta) +
               delvxzeta * (Real_t(1.) - phizeta)  ) ;

         qquad = qqc_monoq * rho *
            (  delvxxi*delvxxi     * (Real_t(1.) - phixi*phixi) +
               delvxeta*delvxeta   * (Real_t(1.) - phieta*phieta) +
               delvxzeta*delvxzeta * (Real_t(1.) - phizeta*phizeta)  ) ;
      }

      mesh.qq(i) = qquad ;
      mesh.ql(i) = qlin  ;
   }
}

static inline
void CalcMonotonicQRegionForElems(// parameters
                          Real_t qlc_monoq,
                          Real_t qqc_monoq,
                          Real_t monoq_limiter_mult,
                          Real_t monoq_max_slope,
                          Real_t ptiny,

                          // the elementset length
                          Index_t elength,
                          int useCPU)
{
    if (useCPU) {
        FC(matElemlist); FC(elemBC); FC(lxim); FC(lxip); FC(letam); FC(letap); FC(lzetam); FC(lzetap);
        FC(delv_xi); FC(delv_eta); FC(delv_zeta); FC(delx_xi); FC(delx_eta); FC(delx_zeta);
        FC(vdov); FC(elemMass); FC(volo); FC(vnew);
        CalcMonotonicQRegionForElems_cpu(qlc_monoq,qqc_monoq,
                                     monoq_limiter_mult,monoq_max_slope,ptiny,
                                     elength);
        SG(qq); SG(ql);
    }
    else {
        FG(matElemlist); FG(elemBC); FG(lxim); FG(lxip); FG(letam); FG(letap); FG(lzetam); FG(lzetap);
        FG(delv_xi); FG(delv_eta); FG(delv_zeta); FG(delx_xi); FG(delx_eta); FG(delx_zeta);
        FG(vdov); FG(elemMass); FG(volo); FG(vnew);
        CalcMonotonicQRegionForElems_gpu(qlc_monoq,qqc_monoq,
                                     monoq_limiter_mult,monoq_max_slope,ptiny,
                                     elength);
        SC(qq); SC(ql);
    }
}

static inline
void CalcMonotonicQForElems(int useCPU)
{  
   //
   // initialize parameters
   // 
   const Real_t ptiny        = Real_t(1.e-36) ;
   Real_t monoq_max_slope    = mesh.monoq_max_slope() ;
   Real_t monoq_limiter_mult = mesh.monoq_limiter_mult() ;

   //
   // calculate the monotonic q for pure regions
   //
   Index_t elength = mesh.numElem() ;
   if (elength > 0) {
      Real_t qlc_monoq = mesh.qlc_monoq();
      Real_t qqc_monoq = mesh.qqc_monoq();
      CalcMonotonicQRegionForElems(// parameters
                           qlc_monoq,
                           qqc_monoq,
                           monoq_limiter_mult,
                           monoq_max_slope,
                           ptiny,

                           // the elemset length
                           elength,
                           useCPU);
   }
}

static inline
void CalcQForElems(int useCPU)
{
   Real_t qstop = mesh.qstop() ;
   Index_t numElem = mesh.numElem() ;

   //
   // MONOTONIC Q option
   //

   /* Calculate velocity gradients */
   CalcMonotonicQGradientsForElems(useCPU) ;

   /* Transfer veloctiy gradients in the first order elements */
   /* problem->commElements->Transfer(CommElements::monoQ) ; */
   CalcMonotonicQForElems(useCPU) ;

   /* Don't allow excessive artificial viscosity */
   /*
   if (numElem != 0) {
      Index_t idx = -1; 
      for (Index_t i=0; i<numElem; ++i) {
         if ( mesh.q(i) > qstop ) {
            idx = i ;
            break ;
         }
      }

      if(idx >= 0) {
         exit(QStopError) ;
      }
    }
   */  
}


__global__
void CalcPressureForElems_kernel(Real_t* p_new, Real_t* bvc,
                                 Real_t* pbvc, Real_t* e_old,
                                 Real_t* compression, Real_t *vnewc,
                                 Real_t pmin,
                                 Real_t p_cut, Real_t eosvmax,
                                 Index_t length, Real_t c1s)
{
   int i=blockDim.x*blockIdx.x + threadIdx.x;
   if (i<length) {
       
      bvc[i] = c1s * (compression[i] + Real_t(1.));
      pbvc[i] = c1s;

      p_new[i] = bvc[i] * e_old[i] ;

      if    (FABS(p_new[i]) <  p_cut   )
         p_new[i] = Real_t(0.0) ;

      if    ( vnewc[i] >= eosvmax ) /* impossible condition here? */
         p_new[i] = Real_t(0.0) ;

      if    (p_new[i]       <  pmin)
         p_new[i]   = pmin ;
   }
}


static inline
void CalcPressureForElems_gpu(Real_t* p_new, Real_t* bvc,
                              Real_t* pbvc, Real_t* e_old,
                              Real_t* compression, Real_t *vnewc,
                              Real_t pmin,
                              Real_t p_cut, Real_t eosvmax,
                              Index_t length)
{
    Real_t c1s = Real_t(2.0)/Real_t(3.0) ;
    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(length,dimBlock.x),1,1);
    CalcPressureForElems_kernel<<<dimGrid,dimBlock>>>
        (p_new,bvc,pbvc,e_old,compression,vnewc,pmin,p_cut,eosvmax,length,c1s);
    CUDA_DEBUGSYNC;
}


static inline
void CalcPressureForElems_cpu(Real_t* p_new, Real_t* bvc,
                          Real_t* pbvc, Real_t* e_old,
                          Real_t* compression, Real_t *vnewc,
                          Real_t pmin,
                          Real_t p_cut, Real_t eosvmax,
                          Index_t length)
{
   Real_t c1s = Real_t(2.0)/Real_t(3.0) ;
   for (Index_t i = 0; i < length ; ++i) {
      bvc[i] = c1s * (compression[i] + Real_t(1.));
      pbvc[i] = c1s;
   }

   for (Index_t i = 0 ; i < length ; ++i){
      p_new[i] = bvc[i] * e_old[i] ;

      if    (FABS(p_new[i]) <  p_cut   )
         p_new[i] = Real_t(0.0) ;

      if    ( vnewc[i] >= eosvmax ) /* impossible condition here? */
         p_new[i] = Real_t(0.0) ;

      if    (p_new[i]       <  pmin)
         p_new[i]   = pmin ;
   }
}

__global__
void CalcEnergyForElemsPart1_kernel(
    Index_t length,Real_t emin,
    Real_t *e_old,Real_t *delvc,Real_t *p_old,Real_t *q_old,Real_t *work,
    Real_t *e_new)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {
        e_new[i] = e_old[i] - Real_t(0.5) * delvc[i] * (p_old[i] + q_old[i])
            + Real_t(0.5) * work[i];
        
        if (e_new[i]  < emin ) {
            e_new[i] = emin ;
        }
    }
}


__global__
void CalcEnergyForElemsPart2_kernel(
    Index_t length,Real_t rho0,Real_t e_cut,Real_t emin,
    Real_t *compHalfStep,Real_t *delvc,Real_t *pbvc,Real_t *bvc,
    Real_t *pHalfStep,Real_t *ql,Real_t *qq,Real_t *p_old,Real_t *q_old,Real_t *work,
    Real_t *e_new,
    Real_t *q_new
    )
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {

      Real_t vhalf = Real_t(1.) / (Real_t(1.) + compHalfStep[i]) ;

      if ( delvc[i] > Real_t(0.) ) {
         q_new[i] /* = qq[i] = ql[i] */ = Real_t(0.) ;
      }
      else {
         Real_t ssc = ( pbvc[i] * e_new[i]
                 + vhalf * vhalf * bvc[i] * pHalfStep[i] ) / rho0 ;

         if ( ssc <= Real_t(0.) ) {
            ssc =Real_t(.333333e-36) ;
         } else {
            ssc = SQRT(ssc) ;
         }

         q_new[i] = (ssc*ql[i] + qq[i]) ;
      }

      e_new[i] = e_new[i] + Real_t(0.5) * delvc[i]
         * (  Real_t(3.0)*(p_old[i]     + q_old[i])
              - Real_t(4.0)*(pHalfStep[i] + q_new[i])) ;

      e_new[i] += Real_t(0.5) * work[i];

      if (FABS(e_new[i]) < e_cut) {
         e_new[i] = Real_t(0.)  ;
      }
      if (     e_new[i]  < emin ) {
         e_new[i] = emin ;
      }
   }
}


__global__
void CalcEnergyForElemsPart3_kernel(
    Index_t length,Real_t rho0,Real_t sixth,Real_t e_cut,Real_t emin,
    Real_t *pbvc,Real_t *vnewc,Real_t *bvc,Real_t *p_new,Real_t *ql,Real_t *qq,
    Real_t *p_old,Real_t *q_old,Real_t *pHalfStep,Real_t *q_new,Real_t *delvc,
    Real_t *e_new)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {
      Real_t q_tilde ;

      if (delvc[i] > Real_t(0.)) {
         q_tilde = Real_t(0.) ;
      }
      else {
         Real_t ssc = ( pbvc[i] * e_new[i]
                 + vnewc[i] * vnewc[i] * bvc[i] * p_new[i] ) / rho0 ;

         if ( ssc <= Real_t(0.) ) {
            ssc = Real_t(.333333e-36) ;
         } else {
            ssc = SQRT(ssc) ;
         }

         q_tilde = (ssc*ql[i] + qq[i]) ;
      }

      e_new[i] = e_new[i] - (  Real_t(7.0)*(p_old[i]     + q_old[i])
                               - Real_t(8.0)*(pHalfStep[i] + q_new[i])
                               + (p_new[i] + q_tilde)) * delvc[i]*sixth ;

      if (FABS(e_new[i]) < e_cut) {
         e_new[i] = Real_t(0.)  ;
      }
      if (     e_new[i]  < emin ) {
         e_new[i] = emin ;
      }
   }
}


__global__
void CalcEnergyForElemsPart4_kernel(
    Index_t length,Real_t rho0,Real_t q_cut,
    Real_t *delvc,Real_t *pbvc,Real_t *e_new,Real_t *vnewc,Real_t *bvc,
    Real_t *p_new,Real_t *ql,Real_t *qq,
    Real_t *q_new)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {

      if ( delvc[i] <= Real_t(0.) ) {
         Real_t ssc = ( pbvc[i] * e_new[i]
                 + vnewc[i] * vnewc[i] * bvc[i] * p_new[i] ) / rho0 ;

         if ( ssc <= Real_t(0.) ) {
            ssc = Real_t(.333333e-36) ;
         } else {
            ssc = SQRT(ssc) ;
         }

         q_new[i] = (ssc*ql[i] + qq[i]) ;

         if (FABS(q_new[i]) < q_cut) q_new[i] = Real_t(0.) ;
      }
   }
}

static inline
void CalcEnergyForElems_gpu(Real_t* p_new, Real_t* e_new, Real_t* q_new,
                            Real_t* bvc, Real_t* pbvc,
                            Real_t* p_old, Real_t* e_old, Real_t* q_old,
                            Real_t* compression, Real_t* compHalfStep,
                            Real_t* vnewc, Real_t* work, Real_t* delvc, Real_t pmin,
                            Real_t p_cut, Real_t  e_cut, Real_t q_cut, Real_t emin,
                            Real_t* qq, Real_t* ql,
                            Real_t rho0,
                            Real_t eosvmax,
                            Index_t length)
{
   const Real_t sixth = Real_t(1.0) / Real_t(6.0) ;
   Real_t *pHalfStep;

   dim3 dimBlock=dim3(BLOCKSIZE,1,1);
   dim3 dimGrid=dim3(PAD_DIV(length,dimBlock.x),1,1);
   
   CUDA( hipMalloc(&pHalfStep,sizeof(Real_t)*length) );

   CalcEnergyForElemsPart1_kernel<<<dimGrid,dimBlock>>>
       (length,emin,e_old,delvc,p_old,q_old,work,e_new);
   CUDA_DEBUGSYNC;
   
   CalcPressureForElems_gpu(pHalfStep, bvc, pbvc, e_new, compHalfStep, vnewc,
                   pmin, p_cut, eosvmax, length);

   CalcEnergyForElemsPart2_kernel<<<dimGrid,dimBlock>>>
       (length,rho0,e_cut,emin,
        compHalfStep,delvc,pbvc,bvc,pHalfStep,ql,qq,p_old,q_old,work,
        e_new,
        q_new);
   CUDA_DEBUGSYNC;
   
   CalcPressureForElems_gpu(p_new, bvc, pbvc, e_new, compression, vnewc,
                   pmin, p_cut, eosvmax, length);

   CalcEnergyForElemsPart3_kernel<<<dimGrid,dimBlock>>>
       (length,rho0,sixth,e_cut,emin,
        pbvc,vnewc,bvc,p_new,ql,qq,
        p_old,q_old,pHalfStep,q_new,delvc,
        e_new);
   CUDA_DEBUGSYNC;
   
   CalcPressureForElems_gpu(p_new, bvc, pbvc, e_new, compression, vnewc,
                   pmin, p_cut, eosvmax, length);

   CalcEnergyForElemsPart4_kernel<<<dimGrid,dimBlock>>>
       (length,rho0,q_cut,
        delvc,pbvc,e_new,vnewc,bvc,
        p_new,ql,qq,
        q_new);
   CUDA_DEBUGSYNC;
   
   CUDA( hipFree(pHalfStep) );

   return ;
}

static inline
void CalcEnergyForElems_cpu(Real_t* p_new, Real_t* e_new, Real_t* q_new,
                            Real_t* bvc, Real_t* pbvc,
                            Real_t* p_old, Real_t* e_old, Real_t* q_old,
                            Real_t* compression, Real_t* compHalfStep,
                            Real_t* vnewc, Real_t* work, Real_t* delvc, Real_t pmin,
                            Real_t p_cut, Real_t  e_cut, Real_t q_cut, Real_t emin,
                            Real_t* qq, Real_t* ql,
                            Real_t rho0,
                            Real_t eosvmax,
                            Index_t length)
{
   const Real_t sixth = Real_t(1.0) / Real_t(6.0) ;
   Real_t *pHalfStep = Allocate<Real_t>(length) ;

   for (Index_t i = 0 ; i < length ; ++i) {
      e_new[i] = e_old[i] - Real_t(0.5) * delvc[i] * (p_old[i] + q_old[i])
         + Real_t(0.5) * work[i];

      if (e_new[i]  < emin ) {
         e_new[i] = emin ;
      }
   }

   CalcPressureForElems_cpu(pHalfStep, bvc, pbvc, e_new, compHalfStep, vnewc,
                   pmin, p_cut, eosvmax, length);

   for (Index_t i = 0 ; i < length ; ++i) {
      Real_t vhalf = Real_t(1.) / (Real_t(1.) + compHalfStep[i]) ;

      if ( delvc[i] > Real_t(0.) ) {
         q_new[i] /* = qq[i] = ql[i] */ = Real_t(0.) ;
      }
      else {
         Real_t ssc = ( pbvc[i] * e_new[i]
                 + vhalf * vhalf * bvc[i] * pHalfStep[i] ) / rho0 ;

         if ( ssc <= Real_t(0.) ) {
            ssc =Real_t(.333333e-36) ;
         } else {
            ssc = SQRT(ssc) ;
         }

         q_new[i] = (ssc*ql[i] + qq[i]) ;
      }

      e_new[i] = e_new[i] + Real_t(0.5) * delvc[i]
         * (  Real_t(3.0)*(p_old[i]     + q_old[i])
              - Real_t(4.0)*(pHalfStep[i] + q_new[i])) ;
   }

   for (Index_t i = 0 ; i < length ; ++i) {

      e_new[i] += Real_t(0.5) * work[i];

      if (FABS(e_new[i]) < e_cut) {
         e_new[i] = Real_t(0.)  ;
      }
      if (     e_new[i]  < emin ) {
         e_new[i] = emin ;
      }
   }

   CalcPressureForElems_cpu(p_new, bvc, pbvc, e_new, compression, vnewc,
                   pmin, p_cut, eosvmax, length);

   for (Index_t i = 0 ; i < length ; ++i){
      Real_t q_tilde ;

      if (delvc[i] > Real_t(0.)) {
         q_tilde = Real_t(0.) ;
      }
      else {
         Real_t ssc = ( pbvc[i] * e_new[i]
                 + vnewc[i] * vnewc[i] * bvc[i] * p_new[i] ) / rho0 ;

         if ( ssc <= Real_t(0.) ) {
            ssc = Real_t(.333333e-36) ;
         } else {
            ssc = SQRT(ssc) ;
         }

         q_tilde = (ssc*ql[i] + qq[i]) ;
      }

      e_new[i] = e_new[i] - (  Real_t(7.0)*(p_old[i]     + q_old[i])
                               - Real_t(8.0)*(pHalfStep[i] + q_new[i])
                               + (p_new[i] + q_tilde)) * delvc[i]*sixth ;

      if (FABS(e_new[i]) < e_cut) {
         e_new[i] = Real_t(0.)  ;
      }
      if (     e_new[i]  < emin ) {
         e_new[i] = emin ;
      }
   }

   CalcPressureForElems_cpu(p_new, bvc, pbvc, e_new, compression, vnewc,
                   pmin, p_cut, eosvmax, length);

   for (Index_t i = 0 ; i < length ; ++i){

      if ( delvc[i] <= Real_t(0.) ) {
         Real_t ssc = ( pbvc[i] * e_new[i]
                 + vnewc[i] * vnewc[i] * bvc[i] * p_new[i] ) / rho0 ;

         if ( ssc <= Real_t(0.) ) {
            ssc = Real_t(.333333e-36) ;
         } else {
            ssc = SQRT(ssc) ;
         }

         q_new[i] = (ssc*ql[i] + qq[i]) ;

         if (FABS(q_new[i]) < q_cut) q_new[i] = Real_t(0.) ;
      }
   }

   Release(&pHalfStep) ;

   return ;
}


__global__
void CalcSoundSpeedForElems_kernel(Real_t *vnewc, Real_t rho0, Real_t *enewc,
                            Real_t *pnewc, Real_t *pbvc,
                            Real_t *bvc, Real_t ss4o3, Index_t nz,Index_t *matElemlist,
                            Real_t *ss)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<nz) {
    
      Index_t iz = matElemlist[i];
      Real_t ssTmp = (pbvc[i] * enewc[i] + vnewc[i] * vnewc[i] *
                 bvc[i] * pnewc[i]) / rho0;
      if (ssTmp <= Real_t(1.111111e-36)) {
         ssTmp = Real_t(1.111111e-36);
      }
      ss[iz] = SQRT(ssTmp);
   }
}


static inline
void CalcSoundSpeedForElems_gpu(Real_t *vnewc, Real_t rho0, Real_t *enewc,
                            Real_t *pnewc, Real_t *pbvc,
                            Real_t *bvc, Real_t ss4o3, Index_t nz)
{
   dim3 dimBlock=dim3(BLOCKSIZE,1,1);
   dim3 dimGrid=dim3(PAD_DIV(nz,dimBlock.x),1,1);
   CalcSoundSpeedForElems_kernel<<<dimGrid,dimBlock>>>
       (vnewc,rho0,enewc,pnewc,pbvc,bvc,ss4o3,nz,meshGPU.m_matElemlist,meshGPU.m_ss);
   CUDA_DEBUGSYNC;
    
}

static inline
void CalcSoundSpeedForElems_cpu(Real_t *vnewc, Real_t rho0, Real_t *enewc,
                            Real_t *pnewc, Real_t *pbvc,
                            Real_t *bvc, Real_t ss4o3, Index_t nz)
{
   for (Index_t i = 0; i < nz ; ++i) {
      Index_t iz = mesh.matElemlist(i);
      Real_t ssTmp = (pbvc[i] * enewc[i] + vnewc[i] * vnewc[i] *
                 bvc[i] * pnewc[i]) / rho0;
      if (ssTmp <= Real_t(1.111111e-36)) {
         ssTmp = Real_t(1.111111e-36);
      }
      mesh.ss(iz) = SQRT(ssTmp);
   }
}


__global__
void EvalEOSForElemsPart1_kernel(
    Index_t length,Real_t eosvmin,Real_t eosvmax,
    Index_t *matElemlist,
    Real_t *e,Real_t *delv,Real_t *p,Real_t *q,Real_t *qq,Real_t *ql,
    Real_t *vnewc,
    Real_t *e_old,Real_t *delvc,Real_t *p_old,Real_t *q_old,
    Real_t *compression,Real_t *compHalfStep,
    Real_t *qq_old,Real_t *ql_old,Real_t *work)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {
        Index_t zidx = matElemlist[i];
        e_old[i] = e[zidx];
        delvc[i] = delv[zidx];
        p_old[i] = p[zidx];
        q_old[i] = q[zidx];

        Real_t vchalf ;
        compression[i] = Real_t(1.) / vnewc[i] - Real_t(1.);
        vchalf = vnewc[i] - delvc[i] * Real_t(.5);
        compHalfStep[i] = Real_t(1.) / vchalf - Real_t(1.);

        if ( eosvmin != Real_t(0.) ) {
            if (vnewc[i] <= eosvmin) { /* impossible due to calling func? */
                compHalfStep[i] = compression[i] ;
            }
        }
        if ( eosvmax != Real_t(0.) ) {
            if (vnewc[i] >= eosvmax) { /* impossible due to calling func? */
                p_old[i]        = Real_t(0.) ;
                compression[i]  = Real_t(0.) ;
                compHalfStep[i] = Real_t(0.) ;
            }
        }

        qq_old[i] = qq[zidx] ;
        ql_old[i] = ql[zidx] ;
        work[i] = Real_t(0.) ; 
    }
}


__global__
void EvalEOSForElemsPart2_kernel(
    Index_t length,
    Index_t *matElemlist,Real_t *p_new,Real_t *e_new,Real_t *q_new,
    Real_t *p,Real_t *e,Real_t *q)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {
        Index_t zidx = matElemlist[i] ;
        p[zidx] = p_new[i];
        e[zidx] = e_new[i];
        q[zidx] = q_new[i];
    }
}


static inline
void EvalEOSForElems_gpu(Real_t *vnewc, Index_t length)
{
   Real_t  e_cut = mesh.e_cut();
   Real_t  p_cut = mesh.p_cut();
   Real_t  ss4o3 = mesh.ss4o3();
   Real_t  q_cut = mesh.q_cut();

   Real_t eosvmax = mesh.eosvmax() ;
   Real_t eosvmin = mesh.eosvmin() ;
   Real_t pmin    = mesh.pmin() ;
   Real_t emin    = mesh.emin() ;
   Real_t rho0    = mesh.refdens() ;

   Real_t *e_old,*delvc,*p_old,*q_old;
   Real_t *compression,*compHalfStep;
   Real_t *qq,*ql,*work,*p_new,*e_new,*q_new,*bvc,*pbvc;

   CUDA( hipMalloc(&e_old,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&delvc,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&p_old,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&q_old,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&compression,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&compHalfStep,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&qq,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&ql,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&work,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&p_new,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&e_new,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&q_new,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&bvc,sizeof(Real_t)*length) );
   CUDA( hipMalloc(&pbvc,sizeof(Real_t)*length) );

   dim3 dimBlock=dim3(BLOCKSIZE,1,1);
   dim3 dimGrid=dim3(PAD_DIV(length,dimBlock.x),1,1);

   EvalEOSForElemsPart1_kernel<<<dimGrid,dimBlock>>>
       (length,eosvmin,eosvmax,
        meshGPU.m_matElemlist,
        meshGPU.m_e,meshGPU.m_delv,meshGPU.m_p,meshGPU.m_q,meshGPU.m_qq,meshGPU.m_ql,
        vnewc,
        e_old,delvc,p_old,q_old,
        compression,compHalfStep,qq,ql,work);
   CUDA_DEBUGSYNC;

   CalcEnergyForElems_gpu(p_new, e_new, q_new, bvc, pbvc,
                 p_old, e_old,  q_old, compression, compHalfStep,
                 vnewc, work,  delvc, pmin,
                 p_cut, e_cut, q_cut, emin,
                 qq, ql, rho0, eosvmax, length);


   EvalEOSForElemsPart2_kernel<<<dimGrid,dimBlock>>>
       (length,
        meshGPU.m_matElemlist,p_new,e_new,q_new,
        meshGPU.m_p,meshGPU.m_e,meshGPU.m_q);
   CUDA_DEBUGSYNC;

   CalcSoundSpeedForElems_gpu(vnewc, rho0, e_new, p_new,
             pbvc, bvc, ss4o3, length) ;

   CUDA( hipFree(pbvc) );
   CUDA( hipFree(bvc) );
   CUDA( hipFree(q_new) );
   CUDA( hipFree(e_new) );
   CUDA( hipFree(p_new) );
   CUDA( hipFree(work) );
   CUDA( hipFree(ql) );
   CUDA( hipFree(qq) );
   CUDA( hipFree(compHalfStep) );
   CUDA( hipFree(compression) );
   CUDA( hipFree(q_old) );
   CUDA( hipFree(p_old) );
   CUDA( hipFree(delvc) );
   CUDA( hipFree(e_old) );
}


static inline
void EvalEOSForElems_cpu(Real_t *vnewc, Index_t length)
{
   Real_t  e_cut = mesh.e_cut();
   Real_t  p_cut = mesh.p_cut();
   Real_t  ss4o3 = mesh.ss4o3();
   Real_t  q_cut = mesh.q_cut();

   Real_t eosvmax = mesh.eosvmax() ;
   Real_t eosvmin = mesh.eosvmin() ;
   Real_t pmin    = mesh.pmin() ;
   Real_t emin    = mesh.emin() ;
   Real_t rho0    = mesh.refdens() ;

   Real_t *e_old = Allocate<Real_t>(length) ;
   Real_t *delvc = Allocate<Real_t>(length) ;
   Real_t *p_old = Allocate<Real_t>(length) ;
   Real_t *q_old = Allocate<Real_t>(length) ;
   Real_t *compression = Allocate<Real_t>(length) ;
   Real_t *compHalfStep = Allocate<Real_t>(length) ;
   Real_t *qq = Allocate<Real_t>(length) ;
   Real_t *ql = Allocate<Real_t>(length) ;
   Real_t *work = Allocate<Real_t>(length) ;
   Real_t *p_new = Allocate<Real_t>(length) ;
   Real_t *e_new = Allocate<Real_t>(length) ;
   Real_t *q_new = Allocate<Real_t>(length) ;
   Real_t *bvc = Allocate<Real_t>(length) ;
   Real_t *pbvc = Allocate<Real_t>(length) ;

   /* compress data, minimal set */
   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      e_old[i] = mesh.e(zidx) ;
   }

   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      delvc[i] = mesh.delv(zidx) ;
   }

   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      p_old[i] = mesh.p(zidx) ;
   }

   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      q_old[i] = mesh.q(zidx) ;
   }

   for (Index_t i = 0; i < length ; ++i) {
      Real_t vchalf ;
      compression[i] = Real_t(1.) / vnewc[i] - Real_t(1.);
      vchalf = vnewc[i] - delvc[i] * Real_t(.5);
      compHalfStep[i] = Real_t(1.) / vchalf - Real_t(1.);
   }

   /* Check for v > eosvmax or v < eosvmin */
   if ( eosvmin != Real_t(0.) ) {
      for(Index_t i=0 ; i<length ; ++i) {
         if (vnewc[i] <= eosvmin) { /* impossible due to calling func? */
            compHalfStep[i] = compression[i] ;
         }
      }
   }
   if ( eosvmax != Real_t(0.) ) {
      for(Index_t i=0 ; i<length ; ++i) {
         if (vnewc[i] >= eosvmax) { /* impossible due to calling func? */
            p_old[i]        = Real_t(0.) ;
            compression[i]  = Real_t(0.) ;
            compHalfStep[i] = Real_t(0.) ;
         }
      }
   }

   for (Index_t i = 0 ; i < length ; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      qq[i] = mesh.qq(zidx) ;
      ql[i] = mesh.ql(zidx) ;
      work[i] = Real_t(0.) ; 
   }

   CalcEnergyForElems_cpu(p_new, e_new, q_new, bvc, pbvc,
                 p_old, e_old,  q_old, compression, compHalfStep,
                 vnewc, work,  delvc, pmin,
                 p_cut, e_cut, q_cut, emin,
                 qq, ql, rho0, eosvmax, length);


   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      mesh.p(zidx) = p_new[i] ;
   }

   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      mesh.e(zidx) = e_new[i] ;
   }

   for (Index_t i=0; i<length; ++i) {
      Index_t zidx = mesh.matElemlist(i) ;
      mesh.q(zidx) = q_new[i] ;
   }

   CalcSoundSpeedForElems_cpu(vnewc, rho0, e_new, p_new,
             pbvc, bvc, ss4o3, length) ;

   Release(&pbvc) ;
   Release(&bvc) ;
   Release(&q_new) ;
   Release(&e_new) ;
   Release(&p_new) ;
   Release(&work) ;
   Release(&ql) ;
   Release(&qq) ;
   Release(&compHalfStep) ;
   Release(&compression) ;
   Release(&q_old) ;
   Release(&p_old) ;
   Release(&delvc) ;
   Release(&e_old) ;
}


__global__
void ApplyMaterialPropertiesForElemsPart1_kernel(
    Index_t length,Real_t eosvmin,Real_t eosvmax,
    Index_t *matElemlist,Real_t *vnew,
    Real_t *vnewc)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<length) {
        Index_t zn = matElemlist[i] ;
        vnewc[i] = vnew[zn] ;

        if (eosvmin != Real_t(0.)) {
            if (vnewc[i] < eosvmin)
                vnewc[i] = eosvmin ;
        }

        if (eosvmax != Real_t(0.)) {
            if (vnewc[i] > eosvmax)
                vnewc[i] = eosvmax ;
        }
    }
}


static inline
void ApplyMaterialPropertiesForElems_gpu()
{
  Index_t length = mesh.numElem() ;

  if (length != 0) {
    /* Expose all of the variables needed for material evaluation */
    Real_t eosvmin = mesh.eosvmin() ;
    Real_t eosvmax = mesh.eosvmax() ;
    Real_t *vnewc;

    CUDA( hipMalloc(&vnewc,sizeof(Real_t)*length) );

    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(length,dimBlock.x),1,1);
    ApplyMaterialPropertiesForElemsPart1_kernel<<<dimGrid,dimBlock>>>
        (length,eosvmin,eosvmax,
         meshGPU.m_matElemlist,meshGPU.m_vnew,
         vnewc);
    CUDA_DEBUGSYNC;
    
    /*
    for (Index_t i=0; i<length; ++i) {
       Index_t zn = mesh.matElemlist(i) ;
       Real_t vc = mesh.v(zn) ;
       if (eosvmin != Real_t(0.)) {
          if (vc < eosvmin)
             vc = eosvmin ;
       }
       if (eosvmax != Real_t(0.)) {
          if (vc > eosvmax)
             vc = eosvmax ;
       }
       if (vc <= 0.) {
          exit(VolumeError) ;
       }
    }
    */
    
    EvalEOSForElems_gpu(vnewc, length);

    CUDA( hipFree(vnewc) );
  }
}

static inline
void ApplyMaterialPropertiesForElems_cpu()
{
  Index_t length = mesh.numElem() ;

  if (length != 0) {
    /* Expose all of the variables needed for material evaluation */
    Real_t eosvmin = mesh.eosvmin() ;
    Real_t eosvmax = mesh.eosvmax() ;
    Real_t *vnewc = Allocate<Real_t>(length) ;

    for (Index_t i=0 ; i<length ; ++i) {
       Index_t zn = mesh.matElemlist(i) ;
       vnewc[i] = mesh.vnew(zn) ;
    }

    if (eosvmin != Real_t(0.)) {
       for(Index_t i=0 ; i<length ; ++i) {
          if (vnewc[i] < eosvmin)
             vnewc[i] = eosvmin ;
       }
    }

    if (eosvmax != Real_t(0.)) {
       for(Index_t i=0 ; i<length ; ++i) {
          if (vnewc[i] > eosvmax)
             vnewc[i] = eosvmax ;
       }
    }

    for (Index_t i=0; i<length; ++i) {
       Index_t zn = mesh.matElemlist(i) ;
       Real_t vc = mesh.v(zn) ;
       if (eosvmin != Real_t(0.)) {
          if (vc < eosvmin)
             vc = eosvmin ;
       }
       if (eosvmax != Real_t(0.)) {
          if (vc > eosvmax)
             vc = eosvmax ;
       }
       if (vc <= 0.) {
          exit(VolumeError) ;
       }
    }

    EvalEOSForElems_cpu(vnewc, length);

    Release(&vnewc) ;

  }
}

static inline
void ApplyMaterialPropertiesForElems(int useCPU)
{
    if (useCPU) {
        FC(matElemlist); FC(vnew); FC(v); FC(e); FC(delv); FC(p); FC(q); FC(qq); FC(ql);
        ApplyMaterialPropertiesForElems_cpu();
        SG(p); SG(e); SG(q); SG(ss);
    }
    else {
        FG(matElemlist); FG(vnew); FG(v); FG(e); FG(delv); FG(p); FG(q); FG(qq); FG(ql);
        ApplyMaterialPropertiesForElems_gpu();
        SC(p); SC(e); SC(q); SC(ss);
    }
}

__global__
void UpdateVolumesForElems_kernel(Index_t numElem,Real_t v_cut,
                                  Real_t *vnew,
                                  Real_t *v)
{
    int i=blockDim.x*blockIdx.x + threadIdx.x;
    if (i<numElem) {
         Real_t tmpV ;
         tmpV = vnew[i] ;

         if ( FABS(tmpV - Real_t(1.0)) < v_cut )
            tmpV = Real_t(1.0) ;
         v[i] = tmpV ;
    }
}


static inline
void UpdateVolumesForElems_gpu()
{
   Index_t numElem = mesh.numElem();
   if (numElem != 0) {
      Real_t v_cut = mesh.v_cut();
      dim3 dimBlock=dim3(BLOCKSIZE,1,1);
      dim3 dimGrid=dim3(PAD_DIV(numElem,dimBlock.x),1,1);
      UpdateVolumesForElems_kernel<<<dimGrid,dimBlock>>>
          (numElem,v_cut,meshGPU.m_vnew,meshGPU.m_v);
   }
}


static inline
void UpdateVolumesForElems_cpu()
{
   Index_t numElem = mesh.numElem();
   if (numElem != 0) {
      Real_t v_cut = mesh.v_cut();

      for(Index_t i=0 ; i<numElem ; ++i) {
         Real_t tmpV ;
         tmpV = mesh.vnew(i) ;

         if ( FABS(tmpV - Real_t(1.0)) < v_cut )
            tmpV = Real_t(1.0) ;
         mesh.v(i) = tmpV ;
      }
   }

   return ;
}

static inline
void UpdateVolumesForElems(int useCPU)
{
    if (useCPU) {
        FC(vnew);
        UpdateVolumesForElems_cpu();
        SG(v);
    }
    else {
        FG(vnew);
        UpdateVolumesForElems_gpu();
        SC(v);
    }
}


static inline
void LagrangeElements(int useCPU)
{
  const Real_t deltatime = mesh.deltatime() ;

  CalcLagrangeElements(deltatime, useCPU) ;

  /* Calculate Q.  (Monotonic q option requires communication) */
  CalcQForElems(useCPU) ;

  ApplyMaterialPropertiesForElems(useCPU) ;

  UpdateVolumesForElems(useCPU) ;
}


__global__
void CalcCourantConstraintForElems_kernel(
    Index_t length,Real_t qqc2,
    Index_t *matElemlist,Real_t *ss,Real_t *vdov,Real_t *arealg,
    Real_t *mindtcourant)
{
    __shared__ Real_t minArray[BLOCKSIZE];

    int i=blockDim.x*blockIdx.x + threadIdx.x;
    
    Real_t dtcourant = Real_t(1.0e+20) ;
    if (i<length) {
        Index_t indx = matElemlist[i] ;
        Real_t dtf = ss[indx] * ss[indx] ;
        if ( vdov[indx] < Real_t(0.) ) {
            dtf = dtf
                + qqc2 * arealg[indx] * arealg[indx]
                * vdov[indx] * vdov[indx] ;
        }
        dtf = SQRT(dtf) ;
        dtf = arealg[indx] / dtf ;

        /* determine minimum timestep with its corresponding elem */
        if (vdov[indx] != Real_t(0.)) {
            if ( dtf < dtcourant ) {
                dtcourant = dtf ;
            }
        }
    }
    minArray[threadIdx.x]=dtcourant;
    reduceMin<Real_t,BLOCKSIZE>(minArray,threadIdx.x);
    if (threadIdx.x==0)
        mindtcourant[blockIdx.x]=minArray[0];
}


static inline
void CalcCourantConstraintForElems_gpu()
{
    Real_t qqc = mesh.qqc();
    Real_t qqc2 = Real_t(64.0) * qqc * qqc ;
    Index_t length = mesh.numElem() ;

    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(length,dimBlock.x),1,1);

    Real_t *dev_mindtcourant;
    CUDA( hipMalloc(&dev_mindtcourant,sizeof(Real_t)*dimGrid.x) );

    CalcCourantConstraintForElems_kernel<<<dimGrid,dimBlock>>>
        (length,qqc2,
         meshGPU.m_matElemlist,meshGPU.m_ss,meshGPU.m_vdov,meshGPU.m_arealg,
         dev_mindtcourant);
    CUDA_DEBUGSYNC;

    Real_t *mindtcourant = (Real_t*)malloc(sizeof(Real_t)*dimGrid.x);
    CUDA( hipMemcpy(mindtcourant,dev_mindtcourant,sizeof(Real_t)*dimGrid.x,hipMemcpyDeviceToHost) );
    CUDA( hipFree(dev_mindtcourant) );

    // finish the MIN computation over the thread blocks
    Real_t dtcourant;
    dtcourant=mindtcourant[0];
    for (int i=1; i<dimGrid.x; i++) {
        MINEQ(dtcourant,mindtcourant[i]);
    }
    free(mindtcourant);

    if (dtcourant < Real_t(1.0e+20))
        mesh.dtcourant() = dtcourant ;
}

static inline
void CalcCourantConstraintForElems_cpu()
{
   Real_t dtcourant = Real_t(1.0e+20) ;
   Index_t   courant_elem = -1 ;
   Real_t      qqc = mesh.qqc() ;
   Index_t length = mesh.numElem() ;

   Real_t  qqc2 = Real_t(64.0) * qqc * qqc ;

   for (Index_t i = 0 ; i < length ; ++i) {
      Index_t indx = mesh.matElemlist(i) ;

      Real_t dtf = mesh.ss(indx) * mesh.ss(indx) ;

      if ( mesh.vdov(indx) < Real_t(0.) ) {

         dtf = dtf
            + qqc2 * mesh.arealg(indx) * mesh.arealg(indx)
            * mesh.vdov(indx) * mesh.vdov(indx) ;
      }

      dtf = SQRT(dtf) ;

      dtf = mesh.arealg(indx) / dtf ;

   /* determine minimum timestep with its corresponding elem */
      if (mesh.vdov(indx) != Real_t(0.)) {
         if ( dtf < dtcourant ) {
            dtcourant = dtf ;
            courant_elem = indx ;
         }
      }
   }

   /* Don't try to register a time constraint if none of the elements
    * were active */
   if (courant_elem != -1) {
      mesh.dtcourant() = dtcourant ;
   }

   return ;
}


static inline
void CalcCourantConstraintForElems(int useCPU)
{
    if (useCPU) {
        FC(matElemlist); FC(ss); FC(vdov); FC(arealg);
        CalcCourantConstraintForElems_cpu();
    }
    else {
        FG(matElemlist); FG(ss); FG(vdov); FG(arealg);
        CalcCourantConstraintForElems_gpu();
    }
}


__global__
void CalcHydroConstraintForElems_kernel(
    Index_t length,Real_t dvovmax,
    Index_t *matElemlist,Real_t *vdov,
    Real_t *mindthydro)
{
    __shared__ Real_t minArray[BLOCKSIZE];

    int i=blockDim.x*blockIdx.x + threadIdx.x;

    Real_t dthydro = Real_t(1.0e+20) ;
    if (i<length) {
      Index_t indx = matElemlist[i] ;
      if (vdov[indx] != Real_t(0.)) {
         Real_t dtdvov = dvovmax / (FABS(vdov[indx])+Real_t(1.e-20)) ;
         if ( dthydro > dtdvov ) {
            dthydro = dtdvov ;
         }
      }
    }
    minArray[threadIdx.x]=dthydro;
    reduceMin<Real_t,BLOCKSIZE>(minArray,threadIdx.x);
    if (threadIdx.x==0)
        mindthydro[blockIdx.x]=minArray[0];
}


static inline
void CalcHydroConstraintForElems_gpu()
{
    Real_t dvovmax = mesh.dvovmax() ;
    Index_t length = mesh.numElem() ;

    dim3 dimBlock=dim3(BLOCKSIZE,1,1);
    dim3 dimGrid=dim3(PAD_DIV(length,dimBlock.x),1,1);

    Real_t *dev_mindthydro;
    CUDA( hipMalloc(&dev_mindthydro,sizeof(Real_t)*dimGrid.x) );

    CalcHydroConstraintForElems_kernel<<<dimGrid,dimBlock>>>
        (length,dvovmax,
         meshGPU.m_matElemlist,meshGPU.m_vdov,
         dev_mindthydro);
    CUDA_DEBUGSYNC;

    Real_t *mindthydro = (Real_t*)malloc(sizeof(Real_t)*dimGrid.x);
    CUDA( hipMemcpy(mindthydro,dev_mindthydro,sizeof(Real_t)*dimGrid.x,hipMemcpyDeviceToHost) );
    CUDA( hipFree(dev_mindthydro) );

    // finish the MIN computation over the thread blocks
    Real_t dthydro=mindthydro[0];
    for (int i=1; i<dimGrid.x; i++) {
        MINEQ(dthydro,mindthydro[i]);
    }
    free(mindthydro);
    
    if (dthydro < Real_t(1.0e+20))
        mesh.dthydro() = dthydro ;
}

static inline
void CalcHydroConstraintForElems_cpu()
{
   Real_t dthydro = Real_t(1.0e+20) ;
   Index_t hydro_elem = -1 ;
   Real_t dvovmax = mesh.dvovmax() ;
   Index_t length = mesh.numElem() ;

   for (Index_t i = 0 ; i < length ; ++i) {
      Index_t indx = mesh.matElemlist(i) ;

      if (mesh.vdov(indx) != Real_t(0.)) {
         Real_t dtdvov = dvovmax / (FABS(mesh.vdov(indx))+Real_t(1.e-20)) ;
         if ( dthydro > dtdvov ) {
            dthydro = dtdvov ;
            hydro_elem = indx ;
         }
      }
   }

   if (hydro_elem != -1) {
      mesh.dthydro() = dthydro ;
   }

   return ;
}


static inline
void CalcHydroConstraintForElems(int useCPU)
{
    if (useCPU) {
        FC(matElemlist); FC(vdov);
        CalcHydroConstraintForElems_cpu();
    }
    else {
        FG(matElemlist); FG(vdov);
        CalcHydroConstraintForElems_gpu();
    }
}



static inline
void CalcTimeConstraintsForElems(int useCPU) {
   /* evaluate time constraint */
   CalcCourantConstraintForElems(useCPU) ;

   /* check hydro constraint */
   CalcHydroConstraintForElems(useCPU) ;
}

static inline
void LagrangeLeapFrog(int useCPU)
{
   /* calculate nodal forces, accelerations, velocities, positions, with
    * applied boundary conditions and slide surface considerations */

   LagrangeNodal(useCPU);

   /* calculate element quantities (i.e. velocity gradient & q), and update
    * material states */
   LagrangeElements(useCPU);

   CalcTimeConstraintsForElems(useCPU);

   // LagrangeRelease() ;  Creation/destruction of temps may be important to capture 
}

int main(int argc, char *argv[])
{
   Index_t edgeElems = 45 ;
   Index_t edgeNodes = edgeElems+1 ;
   // Real_t ds = Real_t(1.125)/Real_t(edgeElems) ; /* may accumulate roundoff */
   Real_t tx, ty, tz ;
   Index_t nidx, zidx ;
   Index_t meshElems ;

   /* get run options to measure various metrics */

   /* ... */
   
   cuda_init();
   
   /****************************/
   /*   Initialize Sedov Mesh  */
   /****************************/

   /* construct a uniform box for this processor */

   mesh.sizeX()   = edgeElems ;
   mesh.sizeY()   = edgeElems ;
   mesh.sizeZ()   = edgeElems ;
   mesh.numElem() = edgeElems*edgeElems*edgeElems ;
   mesh.numNode() = edgeNodes*edgeNodes*edgeNodes ;

   meshElems = mesh.numElem() ;


   /* allocate field memory */

   mesh.AllocateElemPersistent(mesh.numElem()) ;
   mesh.AllocateElemTemporary (mesh.numElem()) ;

   mesh.AllocateNodalPersistent(mesh.numNode()) ;
   mesh.AllocateNodesets(edgeNodes*edgeNodes) ;


   /* initialize nodal coordinates */

   nidx = 0 ;
   tz  = Real_t(0.) ;
   for (Index_t plane=0; plane<edgeNodes; ++plane) {
      ty = Real_t(0.) ;
      for (Index_t row=0; row<edgeNodes; ++row) {
         tx = Real_t(0.) ;
         for (Index_t col=0; col<edgeNodes; ++col) {
            mesh.x(nidx) = tx ;
            mesh.y(nidx) = ty ;
            mesh.z(nidx) = tz ;
            ++nidx ;
            // tx += ds ; /* may accumulate roundoff... */
            tx = Real_t(1.125)*Real_t(col+1)/Real_t(edgeElems) ;
         }
         // ty += ds ;  /* may accumulate roundoff... */
         ty = Real_t(1.125)*Real_t(row+1)/Real_t(edgeElems) ;
      }
      // tz += ds ;  /* may accumulate roundoff... */
      tz = Real_t(1.125)*Real_t(plane+1)/Real_t(edgeElems) ;
   }


   /* embed hexehedral elements in nodal point lattice */

   nidx = 0 ;
   zidx = 0 ;
   for (Index_t plane=0; plane<edgeElems; ++plane) {
      for (Index_t row=0; row<edgeElems; ++row) {
         for (Index_t col=0; col<edgeElems; ++col) {
            mesh.nodelist(zidx,0) = nidx                                       ;
            mesh.nodelist(zidx,1) = nidx                                   + 1 ;
            mesh.nodelist(zidx,2) = nidx                       + edgeNodes + 1 ;
            mesh.nodelist(zidx,3) = nidx                       + edgeNodes     ;
            mesh.nodelist(zidx,4) = nidx + edgeNodes*edgeNodes                 ;
            mesh.nodelist(zidx,5) = nidx + edgeNodes*edgeNodes             + 1 ;
            mesh.nodelist(zidx,6) = nidx + edgeNodes*edgeNodes + edgeNodes + 1 ;
            mesh.nodelist(zidx,7) = nidx + edgeNodes*edgeNodes + edgeNodes     ;
            ++zidx ;
            ++nidx ;
         }
         ++nidx ;
      }
      nidx += edgeNodes ;
   }

   /* Create a material IndexSet (entire mesh same material for now) */
   for (Index_t i=0; i<meshElems; ++i) {
      mesh.matElemlist(i) = i ;
   }
   
   /* initialize material parameters */
   mesh.dtfixed() = Real_t(-1.0e-7) ;
   mesh.deltatime() = Real_t(1.0e-7) ;
   mesh.deltatimemultlb() = Real_t(1.1) ;
   mesh.deltatimemultub() = Real_t(1.2) ;
   if (argc == 2) {
       mesh.stoptime()  = Real_t(atof(argv[1])) ;
   } else {
       mesh.stoptime()  = Real_t(1.0e-2) ;
   }
   mesh.dtcourant() = Real_t(1.0e+20) ;
   mesh.dthydro()   = Real_t(1.0e+20) ;
   mesh.dtmax()     = Real_t(1.0e-2) ;
   mesh.time()    = Real_t(0.) ;
   mesh.cycle()   = 0 ;

   mesh.e_cut() = Real_t(1.0e-7) ;
   mesh.p_cut() = Real_t(1.0e-7) ;
   mesh.q_cut() = Real_t(1.0e-7) ;
   mesh.u_cut() = Real_t(1.0e-7) ;
   mesh.v_cut() = Real_t(1.0e-10) ;

   mesh.hgcoef()      = Real_t(3.0) ;
   mesh.ss4o3()       = Real_t(4.0)/Real_t(3.0) ;

   mesh.qstop()              =  Real_t(1.0e+12) ;
   mesh.monoq_max_slope()    =  Real_t(1.0) ;
   mesh.monoq_limiter_mult() =  Real_t(2.0) ;
   mesh.qlc_monoq()          = Real_t(0.5) ;
   mesh.qqc_monoq()          = Real_t(2.0)/Real_t(3.0) ;
   mesh.qqc()                = Real_t(2.0) ;

   mesh.pmin() =  Real_t(0.) ;
   mesh.emin() = Real_t(-1.0e+15) ;

   mesh.dvovmax() =  Real_t(0.1) ;

   mesh.eosvmax() =  Real_t(1.0e+9) ;
   mesh.eosvmin() =  Real_t(1.0e-9) ;

   mesh.refdens() =  Real_t(1.0) ;

   /* initialize field data */
   for (Index_t i=0; i<meshElems; ++i) {
      Real_t x_local[8], y_local[8], z_local[8] ;
      for( Index_t lnode=0 ; lnode<8 ; ++lnode )
      {
        Index_t gnode = mesh.nodelist(i,lnode);
        x_local[lnode] = mesh.x(gnode);
        y_local[lnode] = mesh.y(gnode);
        z_local[lnode] = mesh.z(gnode);
      }

      // volume calculations
      Real_t volume = CalcElemVolume(x_local, y_local, z_local );
      mesh.volo(i) = volume ;
      mesh.elemMass(i) = volume ;
      for (Index_t j=0; j<8; ++j) {
	 Index_t idx = mesh.nodelist(i,j);
         mesh.nodalMass(idx) += volume / Real_t(8.0) ;
      }
   }

   /* deposit energy */
   mesh.e(0) = Real_t(3.948746e+7) ;

   /* set up symmetry nodesets */
   nidx = 0 ;
   for (Index_t i=0; i<edgeNodes; ++i) {
      Index_t planeInc = i*edgeNodes*edgeNodes ;
      Index_t rowInc   = i*edgeNodes ;
      for (Index_t j=0; j<edgeNodes; ++j) {
         mesh.symmX(nidx) = planeInc + j*edgeNodes ;
         mesh.symmY(nidx) = planeInc + j ;
         mesh.symmZ(nidx) = rowInc   + j ;
         ++nidx ;
      }
   }

   /* set up elemement connectivity information */
   mesh.lxim(0) = 0 ;
   for (Index_t i=1; i<meshElems; ++i) {
      mesh.lxim(i)   = i-1 ;
      mesh.lxip(i-1) = i ;
   }
   mesh.lxip(meshElems-1) = meshElems-1 ;

   for (Index_t i=0; i<edgeElems; ++i) {
      mesh.letam(i) = i ; 
      mesh.letap(meshElems-edgeElems+i) = meshElems-edgeElems+i ;
   }
   for (Index_t i=edgeElems; i<meshElems; ++i) {
      mesh.letam(i) = i-edgeElems ;
      mesh.letap(i-edgeElems) = i ;
   }

   for (Index_t i=0; i<edgeElems*edgeElems; ++i) {
      mesh.lzetam(i) = i ;
      mesh.lzetap(meshElems-edgeElems*edgeElems+i) = meshElems-edgeElems*edgeElems+i ;
   }
   for (Index_t i=edgeElems*edgeElems; i<meshElems; ++i) {
      mesh.lzetam(i) = i - edgeElems*edgeElems ;
      mesh.lzetap(i-edgeElems*edgeElems) = i ;
   }

   /* set up boundary condition information */
   for (Index_t i=0; i<meshElems; ++i) {
      mesh.elemBC(i) = 0 ;  /* clear BCs by default */
   }

   /* faces on "external" boundaries will be */
   /* symmetry plane or free surface BCs */
   for (Index_t i=0; i<edgeElems; ++i) {
      Index_t planeInc = i*edgeElems*edgeElems ;
      Index_t rowInc   = i*edgeElems ;
      for (Index_t j=0; j<edgeElems; ++j) {
         mesh.elemBC(planeInc+j*edgeElems) |= XI_M_SYMM ;
         mesh.elemBC(planeInc+j*edgeElems+edgeElems-1) |= XI_P_FREE ;
         mesh.elemBC(planeInc+j) |= ETA_M_SYMM ;
         mesh.elemBC(planeInc+j+edgeElems*edgeElems-edgeElems) |= ETA_P_FREE ;
         mesh.elemBC(rowInc+j) |= ZETA_M_SYMM ;
         mesh.elemBC(rowInc+j+meshElems-edgeElems*edgeElems) |= ZETA_P_FREE ;
      }
   }

   mesh.AllocateNodeElemIndexes();
   

   
   /* initialize meshGPU */
   meshGPU.init(&mesh);
   meshGPU.freshenGPU();
   
   /* timestep to solution */
   int its=0;
#if 0
   while (its<50) {
#else
   while(mesh.time() < mesh.stoptime() ) {
#endif
      TimeIncrement() ;
      LagrangeLeapFrog(0) ;
      its++;
      /* problem->commNodes->Transfer(CommNodes::syncposvel) ; */
#if LULESH_SHOW_PROGRESS
      printf("time = %e, dt=%e\n",
             double(mesh.time()), double(mesh.deltatime()) ) ;
#endif
   }
   printf("iterations: %d\n",its);

//   FC(x);
//   FILE *fp = fopen("x.asc","wb");
//   for (Index_t i=0; i<mesh.numElem(); i++)
//       fprintf(fp,"%.6f\n",mesh.x(i));
//   fclose(fp);

   return 0 ;
}

