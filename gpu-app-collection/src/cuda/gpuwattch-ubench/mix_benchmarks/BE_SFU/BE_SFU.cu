#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <math.h>
// Includes
#include <stdio.h>

// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
#include "../include/ContAcq-IntClk.h"
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 240
#define ITERATIONS REPLACE_ITERATIONS

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal1(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=A[i];
    float Value2=0;
    float Value3=0;
    float Value=0;
    float I1=A[i];
    float I2=B[i];
    // exponential function
    for(unsigned k=0; k<ITERATIONS*(blockDim.x/blockDim.x+50);k++) {
    	Value2=exp(Value1);
    	Value3=exp(Value2);
    	Value1=exp(Value3);
    	Value2=exp(Value1);
    }

  

   Value=Value3-Value2;		

    C[i]=Value;
    __syncthreads();

}

__global__ void PowerKernal2(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=A[i];
    float Value2=0;
    float Value3=0;
    float Value=0;
    float I1=A[i];
    float I2=B[i];

    //sinusoidal functions
    for(unsigned k=0; k<ITERATIONS*(blockDim.x/blockDim.x+50);k++) {
        Value2=cos(Value1);
        Value3=sin(Value2);
        Value2=cos(Value1);
    	Value1=sin(Value2);
    }



   Value=Value3-Value2;		

    C[i]=Value;
    __syncthreads();

}


__global__ void PowerKernal3(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=0;
    float Value2=99999;
    float Value3=0;
    float Value=0;
    float I1=A[i];
    float I2=B[i];

    //square root
    for(unsigned long k=0; k<ITERATIONS*(blockDim.x/blockDim.x+100);k++) {
	Value1=Value2*Value2;
	Value1=sqrt(abs(Value1));
	Value2=sqrt(abs(I2))*sqrt(abs(I2));
	Value3=sqrt(abs(Value2));
	Value2=sqrt(abs(Value1));
    }



 


   Value=Value3-Value2;		

    C[i]=Value;
    __syncthreads();

}

__global__ void PowerKernalEmpty(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned Value1=0;
    unsigned Value2=0;
    unsigned Value3=0;
    unsigned Value=0;
    unsigned I1=A[i];
    unsigned I2=B[i];
    

    __syncthreads();
   // Excessive Mod/Div Operations
    for(unsigned long k=0; k<ITERATIONS*(blockDim.x+299);k++) {
    	Value1=(I1)+k;
        Value2=(I2)+k;
        Value3=(Value2)+k;
        Value2=(Value1)+k;
       	__asm volatile (
        			"B0: bra.uni B1;\n\t"
        			"B1: bra.uni B2;\n\t"
        			"B2: bra.uni B3;\n\t"
        			"B3: bra.uni B4;\n\t"
        			"B4: bra.uni B5;\n\t"
        			"B5: bra.uni B6;\n\t"
        			"B6: bra.uni B7;\n\t"
        			"B7: bra.uni B8;\n\t"
        			"B8: bra.uni B9;\n\t"
        			"B9: bra.uni B10;\n\t"
        			"B10: bra.uni B11;\n\t"
        			"B11: bra.uni B12;\n\t"
        			"B12: bra.uni B13;\n\t"
        			"B13: bra.uni B14;\n\t"
        			"B14: bra.uni B15;\n\t"
        			"B15: bra.uni B16;\n\t"
        			"B16: bra.uni B17;\n\t"
        			"B17: bra.uni B18;\n\t"
        			"B18: bra.uni B19;\n\t"
        			"B19: bra.uni B20;\n\t"
        			"B20: bra.uni B21;\n\t"
        			"B21: bra.uni B22;\n\t"
        			"B22: bra.uni B23;\n\t"
        			"B23: bra.uni B24;\n\t"
        			"B24: bra.uni B25;\n\t"
        			"B25: bra.uni B26;\n\t"
        			"B26: bra.uni B27;\n\t"
        			"B27: bra.uni B28;\n\t"
        			"B28: bra.uni B29;\n\t"
        			"B29: bra.uni B30;\n\t"
        			"B30: bra.uni B31;\n\t"
        			"B31: bra.uni LOOP;\n\t"
        			"LOOP:"
        			);

    }


    C[i]=I1;
    __syncthreads();

}

__global__ void PowerKernal4(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=0;
    float Value2=0;
    float Value3=0;
    float Value=0;
    float I1=A[i];
    float I2=B[i];


    // logarithmic
    for(unsigned k=0; k<ITERATIONS*(blockDim.x+50);k++) {
    Value1=log2((I1));
    Value2=log2((I2));
    Value3=log2((Value2));
    Value2=log2((Value1));
    }


 


   Value=Value3-Value2;		

    C[i]=Value;
    __syncthreads();

}


// Host code

int main()
{
 printf("Power Microbenchmarks\n");
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(float);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (float*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (float*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (float*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
 printf("before malloc in GPU0\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 printf("before malloc in GPU1\n");
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 printf("before malloc in GPU2\n");
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
 printf("after malloc in GPU\n");

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);
 
CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
TaskHandle taskhandle = LaunchDAQ();
CUT_SAFE_CALL(cutStartTimer(my_timer)); 
 PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A, d_B, d_C, N);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
printf("execution time = %f\n", cutGetTimerValue(my_timer));
//sleep(0.5);
dimGrid.y = NUM_OF_BLOCKS;
for (int i=0; i<3; i++) {
	dimGrid.y /= 3;
	PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
}
 
 
dimGrid.y = NUM_OF_BLOCKS;
for (int i=0; i<3; i++) {
	dimGrid.y /= 3;
	PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
	 PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
 
	 PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
}
 
dimGrid.y = NUM_OF_BLOCKS;
for (int i=0; i<3; i++) {
	dimGrid.y /= 3;
	 PowerKernal3<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	 
	 PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A, d_B, d_C, N);
	//sleep(0.5);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
}

dimGrid.y = NUM_OF_BLOCKS;
for (int i=0; i<3; i++) {
	dimGrid.y /= 3;
	 PowerKernal4<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	 PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A, d_B, d_C, N);

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
}
	 getLastCudaError("kernel launch failure");

CUDA_SAFE_CALL( hipDeviceSynchronize() );
CUT_SAFE_CALL(cutStopTimer(my_timer));
TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
printf("execution time = %f\n", cutGetTimerValue(my_timer));
CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 
#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
  for (int i = 0; i < n; ++i)
	data[i] = rand() / (float)RAND_MAX;
}






