#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <math.h>
// Includes
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>
#include "../include/ContAcq-IntClk.h"

// includes CUDA
#include <hip/hip_runtime.h>

#define MAX_THREADS_PER_BLOCK 256

#define LINE_SIZE 	128
#define SETS		4
#define ASSOC		24
#define SIMD_WIDTH	32

// Variables
int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(int*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line ){
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line ){
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
#define ITERATIONS REPLACE_ITERATIONS

texture<float,1,hipReadModeElementType> texmem1;
texture<float,1,hipReadModeElementType> texmem2;
texture<float,1,hipReadModeElementType> texmem3;
texture<float,1,hipReadModeElementType> texmem4;
texture<float,1,hipReadModeElementType> texmem5;
texture<float,1,hipReadModeElementType> texmem6;
texture<float,1,hipReadModeElementType> texmem7;
texture<float,1,hipReadModeElementType> texmem9;
texture<float,1,hipReadModeElementType> texmem8;



__global__ void tex_bm_kernel( float* out, unsigned size)
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	float Value=0;float Value1=0;float Value2=0;float Value3=0;float Value4=0;float Value5=0;
	if(tid < size){
		for(unsigned i=0; i<ITERATIONS; ++i){
			Value1 = tex1Dfetch(texmem1,tid) + Value5;
			Value2 = tex1Dfetch(texmem2,tid) + Value4;
			Value3 = tex1Dfetch(texmem3,tid) + Value3;
			Value4 = tex1Dfetch(texmem4,tid) + Value1;
			Value5 = tex1Dfetch(texmem5,tid) + Value2;
			Value+=i+Value5+Value3;
		}
	}
    __syncthreads();
	out[tid]=Value;
}


////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	int texmem_size = LINE_SIZE*SETS*ASSOC;

	float *host_texture1 = (float*) malloc(texmem_size*sizeof(float));
	for (int i=0; i< texmem_size; i++) {
		host_texture1[i] = i;
	}
	float *device_texture1;
	float *device_texture2;
	float *device_texture3;
	float *device_texture4;
	float *device_texture5;
	float *device_texture6;
	float *device_texture7;
	float *device_texture8;
	float *device_texture9;

	float *host_out = (float*) malloc(texmem_size*sizeof(float)*10);
	float *device_out;

	hipMalloc((void**) &device_texture1, texmem_size);
	hipMalloc((void**) &device_texture2, texmem_size);
	hipMalloc((void**) &device_texture3, texmem_size);
	hipMalloc((void**) &device_texture4, texmem_size);
	hipMalloc((void**) &device_texture5, texmem_size);
	hipMalloc((void**) &device_texture6, texmem_size);
	hipMalloc((void**) &device_texture7, texmem_size);
	hipMalloc((void**) &device_texture8, texmem_size);
	hipMalloc((void**) &device_texture9, texmem_size);

	hipMalloc((void**) &device_out, texmem_size*10);

	hipMemcpy(device_texture1, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture2, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture3, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture4, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture5, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture6, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture7, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture8, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture9, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);

	hipBindTexture(0, texmem1, device_texture1, texmem_size);
	hipBindTexture(0, texmem2, device_texture2, texmem_size);
	hipBindTexture(0, texmem3, device_texture3, texmem_size);
	hipBindTexture(0, texmem4, device_texture4, texmem_size);
	hipBindTexture(0, texmem5, device_texture5, texmem_size);
	hipBindTexture(0, texmem6, device_texture6, texmem_size);
	hipBindTexture(0, texmem7, device_texture7, texmem_size);
	hipBindTexture(0, texmem8, device_texture8, texmem_size);
	hipBindTexture(0, texmem9, device_texture9, texmem_size);


	unsigned num_blocks = (texmem_size / MAX_THREADS_PER_BLOCK) + 1;
	dim3  grid( num_blocks, 1, 1);
	dim3  threads( MAX_THREADS_PER_BLOCK, 1, 1);

	CUT_SAFE_CALL(cutCreateTimer(&my_timer));
	TaskHandle taskhandle = LaunchDAQ();
	CUT_SAFE_CALL(cutStartTimer(my_timer));

	tex_bm_kernel<<< grid, threads, 0 >>>(device_out, texmem_size);
	hipDeviceSynchronize();

	CUT_SAFE_CALL(cutStopTimer(my_timer));
	TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	CUT_SAFE_CALL(cutDeleteTimer(my_timer));


	printf("Kernel DONE, probably correctly\n");
	hipMemcpy(host_out, device_out, texmem_size*sizeof(float), hipMemcpyDeviceToHost);

	/*
	printf("Output: ");
	float error = false;
	for (int i=0; i< texmem_size; i++){
		printf("%.1f ", host_out[i]);
		if (host_out[i] - i > 0.0001) error = true;
	}
	printf("\n");
	if (error) printf("\nFAILED\n");
	else printf("\nPASSED\n");
	*/
}

void CleanupResources(void){
  // Free device memory


}

// Allocates an array with random float entries.
void RandomInit(int* data, int n){
  for (int i = 0; i < n; ++i)
	data[i] = (int)(rand() / RAND_MAX);
}






