#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

#define THREADS_PER_BLOCK 1024
#define THREADS_PER_SM 2048
#define BLOCKS_NUM 160
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 16

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	//register T s1 = data1[gid];
	//register T s2 = data2[gid];
	//register T result = 0;
    
	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		atomicAdd(&data1[gid], 10);
	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = data1[0];
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	int32_t *data1 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	//int32_t *data2 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *res = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	int32_t *data1_g;
	//int32_t *data2_g;
	int32_t *res_g;

	for (uint32_t i=0; i<TOTAL_THREADS; i++) {
		data1[i] = (int32_t)i;
		//data2[i] = (int32_t)i;
	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, TOTAL_THREADS*sizeof(int32_t)) );
	//gpuErrchk( cudaMalloc(&data2_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(int32_t)) );

	gpuErrchk( hipMemcpy(data1_g, data1, TOTAL_THREADS*sizeof(int32_t), hipMemcpyHostToDevice) );
	//gpuErrchk( cudaMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(int32_t), cudaMemcpyHostToDevice) );

	max_flops<int32_t><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(int32_t), hipMemcpyDeviceToHost) );

	float bw;
	uint32_t total_time = *std::max_element(&stopClk[0],&stopClk[TOTAL_THREADS-1])-*std::min_element(&startClk[0],&startClk[TOTAL_THREADS-1]);
	bw = ((float)(REPEAT_TIMES*TOTAL_THREADS*4)/(float)(total_time));
	printf("int32 bendwidth = %f (byte/clk)\n", bw);
	printf("Total Clk number = %u \n", total_time);

	return 0;
} 

