#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kmeans.h"

// FIXME: Make this a runtime selectable variable!
#define ASSUMED_NR_CLUSTERS 32

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

// t_features has the layout dim0[points 0-m-1]dim1[ points 0-m-1]...
texture<float, 1, hipReadModeElementType> t_features;
// t_features_flipped has the layout point0[dim 0-n-1]point1[dim 0-n-1]
texture<float, 1, hipReadModeElementType> t_features_flipped;
texture<float, 1, hipReadModeElementType> t_clusters;


__constant__ float c_clusters[ASSUMED_NR_CLUSTERS*34];		/* constant memory for cluster centers */

/* ----------------- invert_mapping() --------------------- */
/* inverts data array from row-major to column-major.

   [p0,dim0][p0,dim1][p0,dim2] ... 
   [p1,dim0][p1,dim1][p1,dim2] ... 
   [p2,dim0][p2,dim1][p2,dim2] ... 
										to
   [dim0,p0][dim0,p1][dim0,p2] ...
   [dim1,p0][dim1,p1][dim1,p2] ...
   [dim2,p0][dim2,p1][dim2,p2] ...
*/
__global__ void invert_mapping(float *input,			/* original */
							   float *output,			/* inverted */
							   int npoints,				/* npoints */
							   int nfeatures)			/* nfeatures */
{
	int point_id = threadIdx.x + blockDim.x*blockIdx.x;	/* id of thread */
	int i;

	if(point_id < npoints){
		for(i=0;i<nfeatures;i++)
			output[point_id + npoints*i] = input[point_id*nfeatures + i];
	}
	return;
}
/* ----------------- invert_mapping() end --------------------- */

/* to turn on the GPU delta and center reduction */
//#define GPU_DELTA_REDUCTION
//#define GPU_NEW_CENTER_REDUCTION


/* ----------------- kmeansPoint() --------------------- */
/* find the index of nearest cluster centers and change membership*/
__global__ void
kmeansPoint(float  *features,			/* in: [npoints*nfeatures] */
            int     nfeatures,
            int     npoints,
            int     nclusters,
            int    *membership,
			float  *clusters,
			float  *block_clusters,
			int    *block_deltas) 
{

	// block ID
	const unsigned int block_id = gridDim.x*blockIdx.y+blockIdx.x;
	// point/thread ID  
	const unsigned int point_id = block_id*blockDim.x*blockDim.y + threadIdx.x;
  
	int  index = -1;

	if (point_id < npoints)
	{
		int i, j;
		float min_dist = FLT_MAX;
		float dist;													/* distance square between a point to cluster center */
		
		/* find the cluster center id with min distance to pt */
		for (i=0; i<nclusters; i++) {
			int cluster_base_index = i*nfeatures;					/* base index of cluster centers for inverted array */			
			float ans=0.0;												/* Euclidean distance sqaure */

			for (j=0; j < nfeatures; j++)
			{					
				int addr = point_id + j*npoints;					/* appropriate index of data point */
				float diff = (tex1Dfetch(t_features,addr) -
							  c_clusters[cluster_base_index + j]);	/* distance between a data point to cluster centers */
				ans += diff*diff;									/* sum of squares */
			}
			dist = ans;		

			/* see if distance is smaller than previous ones:
			if so, change minimum distance and save index of cluster center */
			if (dist < min_dist) {
				min_dist = dist;
				index    = i;
			}
		}
	}
	

#ifdef GPU_DELTA_REDUCTION
    // count how many points are now closer to a different cluster center	
	__shared__ int deltas[THREADS_PER_BLOCK];
	if(threadIdx.x < THREADS_PER_BLOCK) {
		deltas[threadIdx.x] = 0;
	}
#endif
	if (point_id < npoints)
	{
#ifdef GPU_DELTA_REDUCTION
		/* if membership changes, increase delta by 1 */
		if (membership[point_id] != index) {
			deltas[threadIdx.x] = 1;
		}
#endif
		/* assign the membership to object point_id */
		membership[point_id] = index;
	}

#ifdef GPU_DELTA_REDUCTION
	// make sure all the deltas have finished writing to shared memory
	__syncthreads();

	// now let's count them
	// primitve reduction follows
	unsigned int threadids_participating = THREADS_PER_BLOCK / 2;
	for(;threadids_participating > 1; threadids_participating /= 2) {
   		if(threadIdx.x < threadids_participating) {
			deltas[threadIdx.x] += deltas[threadIdx.x + threadids_participating];
		}
   		__syncthreads();
	}
	if(threadIdx.x < 1)	{deltas[threadIdx.x] += deltas[threadIdx.x + 1];}
	__syncthreads();
		// propagate number of changes to global counter
	if(threadIdx.x == 0) {
		block_deltas[blockIdx.y * gridDim.x + blockIdx.x] = deltas[0];
		//printf("original id: %d, modified: %d\n", blockIdx.y*gridDim.x+blockIdx.x, blockIdx.x);
		
	}

#endif


#ifdef GPU_NEW_CENTER_REDUCTION
	int center_id = threadIdx.x / nfeatures;    
	int dim_id = threadIdx.x - nfeatures*center_id;

	__shared__ int new_center_ids[THREADS_PER_BLOCK];

	new_center_ids[threadIdx.x] = index;
	__syncthreads();

	/***
	determine which dimension calculte the sum for
	mapping of threads is
	center0[dim0,dim1,dim2,...]center1[dim0,dim1,dim2,...]...
	***/ 	

	int new_base_index = (point_id - threadIdx.x)*nfeatures + dim_id;
	float accumulator = 0.f;

	if(threadIdx.x < nfeatures * nclusters) {
		// accumulate over all the elements of this threadblock 
		for(int i = 0; i< (THREADS_PER_BLOCK); i++) {
			float val = tex1Dfetch(t_features_flipped,new_base_index+i*nfeatures);
			if(new_center_ids[i] == center_id) 
				accumulator += val;
		}
	
		// now store the sum for this threadblock
		/***
		mapping to global array is
		block0[center0[dim0,dim1,dim2,...]center1[dim0,dim1,dim2,...]...]block1[...]...
		***/
		block_clusters[(blockIdx.y*gridDim.x + blockIdx.x) * nclusters * nfeatures + threadIdx.x] = accumulator;
	}
#endif

}
#endif // #ifndef _KMEANS_CUDA_KERNEL_H_
