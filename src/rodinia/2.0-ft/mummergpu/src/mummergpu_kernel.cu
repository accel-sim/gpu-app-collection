#include "hip/hip_runtime.h"
#ifndef _MUMMERGPU_KERNEL_H_
#define _MUMMERGPU_KERNEL_H_

#include <stdio.h>
#include <common.cu>

#ifdef  n__DEVICE_EMULATION__

#define XPRINTF(...)  printf(__VA_ARGS__)
#define VERBOSE 0
#else
#define XPRINTF(...)  do{}while(0)
#define VERBOSE 0
#endif

#define WARP_SIZE 16

#if REORDER_TREE            
#define fNID "%d,%d"
#define NID(addr) (addr & 0x0000FFFF), ((addr & 0xFFFF0000)>>16)
#define GOROOT(addr) addr = 0x00010000
//#define GOROOT(addr) addr.x = 0; addr.y = 1
#else
#define fNID "%d"
#define NID(addr) addr
#define GOROOT(addr) addr = 1
#endif

#if COALESCED_QUERIES
#define GETQCHAR(qrypos) ((queries[((qrypos) >> 2) << 4]) & ((0xFF) << (((qrypos) & 0x00000003)) << 3)) >> ((((qrypos) & 0x00000003 )) << 3)
#elif QRYTEX
#define GETQCHAR(qrypos) tex1Dfetch(qrytex, qryAddr + qrypos)
#else
#define GETQCHAR(qrypos) queries[qrypos]
#endif

#if COALESCED_QUERIES
#define RESULT_SPAN WARP_SIZE
#define MATCH_BASE(match_coords, qryid) (_MatchCoord*)match_coords + coordAddrs[qryid]
#else
#define RESULT_SPAN 1
#define MATCH_BASE(match_coords, qryid) (_MatchCoord*)match_coords + qryAddr - __umul24(qryid, min_match_len + 1)
#endif

#if REFTEX
#define GETRCHAR(refpos) getRef(refpos)
#else
#define GETRCHAR(refpos) getRef(refpos, ref)
#endif



#if MERGETEX

#if TREE_ACCESS_HISTOGRAM

#if NODETEX
#define GETNODE(addr, two_level)         getMerged(addr, two_level, 0, NULL, NULL)
#define GETNODEHIST(addr, two_level)     getMerged(addr, two_level, 0, node_hist, child_hist)
#define GETCHILDREN(addr, two_level)     getMerged(addr, two_level, 1, NULL, NULL)
#define GETCHILDRENHIST(addr, two_level) getMerged(addr, two_level, 1, node_hist, child_hist)
#else
#define GETNODE(addr, two_level)         getMerged(nodes, childrenarr, addr, two_level, 0, NULL, NULL)
#define GETNODEHIST(addr, two_level)     getMerged(nodes, childrenarr, addr, two_level, 0, node_hist, child_hist)
#define GETCHILDREN(addr, two_level)     getMerged(nodes, childrenarr, addr, two_level, 1, NULL, NULL)
#define GETCHILDRENHIST(addr, two_level) getMerged(nodes, childrenarr, addr, two_level, 1, node_hist, child_hist)
#endif

#else
#if NODETEX
#define GETNODE(addr, two_level)         getMerged(addr, two_level, 0)
#define GETNODEHIST(addr, two_level)     getMerged(addr, two_level, 0)
#define GETCHILDREN(addr, two_level)     getMerged(addr, two_level, 1)
#define GETCHILDRENHIST(addr, two_level) getMerged(addr, two_level, 1)
#else
#define GETNODE(addr, two_level)         getMerged(nodes, childrenarr, addr, two_level, 0)
#define GETNODEHIST(addr, two_level)     getMerged(nodes, childrenarr, addr, two_level, 0)
#define GETCHILDREN(addr, two_level)     getMerged(nodes, childrenarr, addr, two_level, 1)
#define GETCHILDRENHIST(addr, two_level) getMerged(nodes, childrenarr, addr, two_level, 1)
#endif
#endif

#else

#if NODETEX
#if TREE_ACCESS_HISTOGRAM
#define GETNODEHIST(addr, two_level)    getNode(addr, two_level, node_hist)
#define GETNODE(addr, two_level)        getNode(addr, two_level, NULL)
#else
#define GETNODEHIST(addr, two_level)    getNode(addr, two_level)
#define GETNODE(addr, two_level)        getNode(addr, two_level)
#endif
#else
#if TREE_ACCESS_HISTOGRAM
#define GETNODEHIST(addr, two_level)    getNode(addr, two_level, nodes, node_hist)
#define GETNODE(addr, two_level)        getNode(addr, two_level, nodes, NULL)
#else
#define GETNODEHIST(addr, two_level)    getNode(addr, two_level, nodes)
#define GETNODE(addr, two_level)        getNode(addr, two_level, nodes)
#endif
#endif

#if CHILDTEX
#if TREE_ACCESS_HISTOGRAM
#define GETCHILDRENHIST(addr, two_level)    getChildren(addr, two_level, child_hist)
#define GETCHILDREN(addr, two_level)        getChildren(addr, two_level, NULL)
#else
#define GETCHILDRENHIST(addr, two_level)    getChildren(addr, two_level)
#define GETCHILDREN(addr, two_level)        getChildren(addr, two_level)
#endif
#else
#if TREE_ACCESS_HISTOGRAM
#define GETCHILDRENHIST(addr, two_level)    getChildren(addr, two_level, childrenarr, child_hist) 
#define GETCHILDREN(addr, two_level)        getChildren(addr, two_level, childrenarr, NULL)
#else
#define GETCHILDRENHIST(addr, two_level)    getChildren(addr, two_level, childrenarr)
#define GETCHILDREN(addr, two_level)        getChildren(addr, two_level, childrenarr)
#endif
#endif

#endif


#if QRYTEX
#define SHIFT_QUERIES(queries, qryAddr) 
#else
#define SHIFT_QUERIES(queries, qryAddr) queries += qryAddr
#endif

#if REORDER_TREE
texture<uint4, 2, hipReadModeElementType> nodetex;
texture<uint4, 2, hipReadModeElementType> childrentex;
#else
texture<uint4, 1, hipReadModeElementType> nodetex;
texture<uint4, 1, hipReadModeElementType> childrentex;
#endif


#if REORDER_REF
texture<char, 2, hipReadModeElementType> reftex;
#else
texture<char, 1, hipReadModeElementType> reftex;
#endif

texture<char, 1, hipReadModeElementType> qrytex;

 struct __align__(8) _MatchCoord
 {
 	union
	{
 		int2 data;
 		struct
		{
 			int node;               // match node
     		int edge_match_length;  // number of matching characters UP the parent edge
 		};
 	};
};

// If leafchar is 0, store the ACGT$ links, else store the leafid
struct _PixelOfChildren
{
  union
  {
    uint4 data;

    union
    {
      struct
      {
        uchar3 a;
        uchar3 c;
        uchar3 g;
        uchar3 t;
        uchar3 d;

        char leafchar;
      };
      
      struct
      {
        uchar3 leafid;
        unsigned char pad [12];
        char leafchar0;
      };
    };
  };
};

// Store the start, end coordinate of node, and the parent, suffix links
struct _PixelOfNode
{
  union
  {
    uint4 data;
    struct
    {
      uchar3 parent;
      uchar3 suffix;

      uchar3 start;
      uchar3 end;
      uchar3 depth;

      unsigned char pad;
    };
  };
};

#if TWO_LEVEL_CHILD_TREE
#define CHILD_THRESH 128
__constant__ _PixelOfChildren child_tree_top[CHILD_THRESH];
#endif

#if TWO_LEVEL_NODE_TREE
#define NODE_THRESH 128
__constant__ _PixelOfNode node_tree_top[NODE_THRESH];
#endif


////////////////////////////////////////////////////////////////////


//////////////////////////////////
/// addr2id
//////////////////////////////////

__device__ int addr2id(unsigned int addr)
{
#if MERGETEX & REORDER_TREE
	addr |= (((addr & 0x800) << 1) << 16);
	addr &= 0xFFFF07FF;
    
	int blocky = (addr >> 16) & 0x1F;
	
    int bigy = (addr >> 16) >> 5;
    int bigx = ((addr & 0x0000FFFF) << 5) + blocky;
    return bigx + (bigy << 16);
    
#elif REORDER_TREE
    int blocky = (addr >> 16) & 0x1F;
    int bigy   = (addr >> 16) >> 5;
    int bigx   = ((addr & 0x0000FFFF) << 5) + blocky;
    return bigx + (bigy << 17);
    
#elif MERGETEX
    return addr;
    
#else
    return addr;
#endif
}

 __device__ TextureAddress id2addr(int id)
 {
     TextureAddress retval;
 
 #if MERGETEX & REORDER_TREE
     // Half width is 2048 => 11 bits
     // TEXBLOCKSIZE is 32 => 5 bits
     int bigx = id & 0xFFFF; // 11 + 5 bits
    int bigy = id >> 16;
 
     retval.y = (bigy << 5) + (bigx & 0x1F);
     retval.x = bigx >> 5;
 
     // now stuff y's 13th bit into x's 12th bit
 
     retval.x |= (retval.y & 0x1000) >> 1;
     retval.y &= 0xFFF;
 
 #elif REORDER_TREE
     int bigx = id & 0x1FFFF;
     int bigy = id >> 17;
     retval.y = (bigy << 5) + (bigx & 0x1F);
     retval.x = bigx >> 5;
 
 #elif MERGETEX
     retval.x = id;
 #else
     retval.x = id;
 #endif
     return retval;
 }


#define MKI(uc3) (uc3.x | (uc3.y << 8) | (uc3.z << 16))

//////////////////////////////////
/// arrayToAddress
//////////////////////////////////

__device__ void arrayToAddress(uchar3 arr, unsigned int& addr)
{
#if REORDER_TREE
  addr = (arr.x | ((arr.z & 0xF) << 8)) | ((arr.y | ((arr.z & 0xF0) << 4)) << 16); 
#else
  addr = MKI(arr);
#endif
}

//////////////////////////////////
/// getRef
//////////////////////////////////

__device__ char getRef(int refpos
#if !REFTEX
					   ,char* ref
#endif
)
{
#if REORDER_REF
    int bigx = refpos & 0x3FFFF;
    int bigy = refpos >> 18;
    int y = (bigy << 2) + (bigx & 0x3);
    int x = bigx >> 2;
   
	#if REFTEX
		return tex2D(reftex, x, y);
	#else
		return *(ref + 65536 * y + x);
	#endif
#else
	#if REFTEX
		return tex1Dfetch(reftex, refpos);
	#else
		return ref[refpos];
	#endif
#endif	
}

//////////////////////////////////
/// RC
//////////////////////////////////

__device__ char rc(char c)
{
  switch(c)
  {
    case 'A': return 'T';
    case 'C': return 'G';
    case 'G': return 'C';
    case 'T': return 'A';
    case 'q': return '\0';
    default:  return c;
  };
}

//////////////////////////////////
/// getNode
//////////////////////////////////

__device__ uint4 getNode(unsigned int cur, 
                          bool use_two_level
#if !NODETEX
                          , _PixelOfNode* nodes
#endif
#if TREE_ACCESS_HISTOGRAM
                          , int* node_hist
#endif
                         ) 
{
#if TREE_ACCESS_HISTOGRAM
  int id = addr2id(cur);
  if (node_hist) { node_hist[id]++; }
#endif

#if TWO_LEVEL_NODE_TREE
  int id = addr2id(cur);
  if (use_two_level && id < NODE_THRESH) { return node_tree_top[id].data; }
#endif

#if NODETEX
#if REORDER_TREE
  return tex2D(nodetex, cur & 0x0000FFFF, (cur & 0xFFFF0000) >> 16);
#else
  return tex1Dfetch(nodetex, cur);
#endif

#else

#if REORDER_TREE
  return (nodes + (cur & 0x0000FFFF) + (((cur & 0xFFFF0000)>>16) * MAX_TEXTURE_DIMENSION))->data;
#else
  return (nodes + cur)->data;
#endif
#endif
}



//////////////////////////////////
/// getChildren
//////////////////////////////////

__device__ uint4 getChildren(unsigned int cur, 
                              bool use_two_level
#if !CHILDTEX
                          , _PixelOfChildren* childrenarr
#endif
#if TREE_ACCESS_HISTOGRAM
                          , int* child_hist
#endif
                          )
{
#if TREE_ACCESS_HISTOGRAM
  int id = addr2id(cur);
  if (child_hist) { child_hist[id]++; }
#endif

#if TWO_LEVEL_CHILD_TREE
  int id = addr2id(cur);
  if (id < CHILD_THRESH) { return child_tree_top[id].data; }
#endif

#if CHILDTEX
#if REORDER_TREE
  return tex2D(childrentex, cur & 0x0000FFFF, (cur & 0xFFFF0000) >> 16);
#else
  return tex1Dfetch(childrentex, cur);
#endif
#else
#if REORDER_TREE
  return (childrenarr +  (cur & 0x0000FFFF) + (((cur & 0xFFFF0000)>>16) * MAX_TEXTURE_DIMENSION))->data;
#else
  return (childrenarr + cur)->data;
#endif
#endif
}

#if MERGETEX

//////////////////////////////////
/// getMerged
//////////////////////////////////

__device__ uint4 getMerged(
#if !NODETEX
                            _PixelOfNode * nodes,
                            _PixelOfChildren * childrenarr,
#endif
                            unsigned int cur, 
                            int  use_two_level,
                            int   getChildrenData
#if TREE_ACCESS_HISTOGRAM
                          , int* node_hist
                          , int* child_hist
#endif
                            )
{
//	TextureAddress cur = _cur;
#if !REORDER_TREE
  //cur.x *= 2;
	unsigned int x = cur * 2;
  int useChildrenForData = 0;

  if (x >= MAX_TEXTURE_DIMENSION*MAX_TEXTURE_DIMENSION)
  {
    x -= MAX_TEXTURE_DIMENSION*MAX_TEXTURE_DIMENSION;
    useChildrenForData = 1;
  }

#else
	unsigned short x = cur & 0x0000FFFF;
	unsigned short y = (cur & 0xFFFF0000) >> 16;
  int useChildrenForData = 0; 

  // WARNING INSANE HACK TO WORK AROUND NVCC BUG

  goto TEST;
  MASK:

  x &= 0x7FF;
  x *= 2;

  goto INC;
  TEST:

  if (x >= 2048)
  {
    useChildrenForData = 1;
  }

  goto MASK;
  INC:

#endif

  x += getChildrenData;

#if !REORDER_TREE
	cur = x;
#else
	cur = (y << 16) | x;
#endif

	  if (useChildrenForData)
	  {
	    return getChildren(cur, use_two_level
	#if !CHILDTEX
	                       , childrenarr
	#endif

	#if TREE_ACCESS_HISTOGRAM
	                       , child_hist
	#endif
	                       );
	  }
	  else
	  {
	    return getNode(cur, use_two_level
	#if !NODETEX
	                   , nodes
	#endif

	#if TREE_ACCESS_HISTOGRAM
	                   , node_hist
	#endif
	                   );
	  }
}

#endif




//////////////////////////////////
/// printNode, Emulator only
//////////////////////////////////

#if VERBOSE

#if CHILDTEX && NODETEX
#define PRINTNODE(id)   printNode(id)
#define PRINTNODES(s,e) printNodes(s,e)
#elif CHILDTEX
#define PRINTNODE(id)   printNode(id, nodes)
#define PRINTNODES(s,e) printNodes(s, e, nodes)
#elif NODETEX
#define PRINTNODE(id)   printNode(id, childarr)
#define PRINTNODES(s,e) printNodes(s, e, childrenarr)
#else
#define PRINTNODE(id)   printNode(id, nodes, childrenarr)
#define PRINTNODES(s,e) printNodes(s, e, nodes, childrenarr)
#endif

__device__ void printNode(int nodeid
#if !NODETEX
                          , _PixelOfNode* nodes
#endif
#if !CHILDTEX
                          , _PixelOfChildren* childrenarr
#endif

                          )
{
  TextureAddress addr = id2addr(nodeid);

  _PixelOfNode nd;
  nd.data = GETNODE(addr.data, false);
  
  _PixelOfChildren cd;
  cd.data = GETCHILDREN(addr.data, false);

  unsigned int a; arrayToAddress(cd.a, a);
  unsigned int c; arrayToAddress(cd.c, c);
  unsigned int g; arrayToAddress(cd.g, g);
  unsigned int t; arrayToAddress(cd.t, t);
  unsigned int d; arrayToAddress(cd.d, d);

  unsigned int p; arrayToAddress(nd.parent, p);
  unsigned int s; arrayToAddress(nd.suffix, s);

  int start = MKI(nd.start);
  int end   = MKI(nd.end);
  int depth = MKI(nd.depth);

  char leafchar = cd.leafchar;


  XPRINTF("%d\t"fNID"\t%d\t%d\t%d\t%d\t"fNID"\t"fNID"\t"fNID"\t"fNID"\t"fNID"\t"fNID"\t"fNID"\n",
          nodeid, NID(addr), start, end, depth, leafchar, 
          NID(a), NID(c), NID(g), NID(t), NID(d), NID(p), NID(s));
}


__device__ void printNodes(int start, int end
#if !NODETEX
                , _PixelOfNode * nodes
#endif
#if !CHILDTEX
                ,_PixelOfChildren * childrenarr
#endif
                )
{
  XPRINTF("id\taddr\tstart\tend\tdepth\tleaf\ta\tc\tg\tt\t$\tp\ts\n");

  for (int i = start; i <= end; i++)
  {
    PRINTNODE(i);
  }
}
              

#else // !VERBOSE

#define PRINTNODE(id) 
#define PRINTNODES(s,e)

#endif

#if VERBOSE
#if NODETEX && CHILDTEX
#define SET_RESULT(c, r, e, q, m, rc) set_result(c, r, e, q, m, rc)
#elif NODETEX
#define SET_RESULT(c, r, e, q, m, rc) set_result(c, r, e, q, m, rc, childrenarr)
#elif CHILDTEX
#define SET_RESULT(c, r, e, q, m, rc) set_result(c, r, e, q, m, rc, nodes)
#else
#define SET_RESULT(c, r, e, q, m, rc) set_result(c, r, e, q, m, rc, nodes, childrenarr)
#endif

#else

#define SET_RESULT(c, r, e, q, m, rc) set_result(c, r, e, q, m, rc)

#endif



//////////////////////////////////
/// set_result
//////////////////////////////////

__device__ void set_result(unsigned int cur,
					   _MatchCoord* result, 
					   int edge_match_length,
                       int qry_match_len,
                       int min_match_len,
                       int rc
#if VERBOSE
#if !NODETEX
                       , _PixelOfNode * nodes
#endif
#if !CHILDTEX
                       , _PixelOfChildren * childrenarr
#endif
#endif
                       )
{
  if (qry_match_len > min_match_len)
  {
    edge_match_length |= rc;
	result->data = make_int2(cur, edge_match_length);

#if VERBOSE
    _PixelOfNode nd; nd.data = GETNODE(cur, false);

    XPRINTF("  saving match cur=%d "fNID" len=%d edge_match=%d depth=%d\n",
            result->data.x, NID(cur), qry_match_len, edge_match_length, MKI(nd.depth));

#endif
  }
  else
  {
    XPRINTF("  match too short (%d < %d)\n", qry_match_len, min_match_len);
  }
}



/////////////////////////////////////
// Compute forward substring matches
/////////////////////////////////////

__global__ void
mummergpuKernel(void* match_coords,
#if COALESCED_QUERIES
				int*		coordAddrs,
#endif

#if !QRYTEX
#if COALESCED_QUERIES
                int* queries,
#else 			
				char* queries,
#endif
#endif
                
#if !NODETEX
                _PixelOfNode* nodes,
#endif
                
#if !CHILDTEX
                _PixelOfChildren* childrenarr,
#endif
                
#if !REFTEX
                char* ref,
#endif
                const int* queryAddrs,
                const int* queryLengths,
                const int numQueries,
                const int min_match_len
#if TREE_ACCESS_HISTOGRAM
				,int* node_hist,
				int* child_hist
#endif
) 
{
    int qryid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if (qryid >= numQueries) { return; }
    XPRINTF("> qryid: %d\n", qryid);

    if (qryid == 0)
    {
      PRINTNODES(0,200);
    }

    int qlen = queryLengths[qryid];
    int qryAddr = queryAddrs[qryid];

    //TextureAddress cur;
	unsigned int cur = 0;
    //cur.data = 0;

    int mustmatch = 0;
    int qry_match_len = 0;
    
    _MatchCoord * result = MATCH_BASE(match_coords, qryid);

    SHIFT_QUERIES(queries, qryAddr);
    
    int last = qlen - min_match_len;
    for (int qrystart = 0;
             qrystart <= last;
             qrystart++, 
             result += RESULT_SPAN) 
    {
		//_PixelOfNode node;
		unsigned int node_start;
        unsigned int prev;

        if ((cur == 0) || (qry_match_len < 1)) 
        {
            // start at root of tree
            GOROOT(cur);
            qry_match_len = 1;
            mustmatch = 0;
        }

        char c = GETQCHAR(qrystart + qry_match_len);

        XPRINTF("In node ("fNID"): starting with %c [%d] =>  \n",
                NID(cur), c, qry_match_len);

        int refpos = 0;
        while ((c != '\0')) 
        {
            XPRINTF("Next edge to follow: %c (%d)\n", c, qry_match_len);
            
			_PixelOfChildren children;
            children.data = GETCHILDRENHIST(cur, false);
			prev = cur;
			uchar3 next;
            switch (c) 
            {
				case 'A': next = children.a; break;
                case 'C': next = children.c; break;
                case 'G': next = children.g; break;
                case 'T': next = children.t; break;
				default: next = make_uchar3(0,0,0); break;
            };
			
			arrayToAddress(next, cur);
				
            XPRINTF(" In node: ("fNID")\n", NID(cur));

            // No edge to follow out of the node
            if (cur == 0) {
                XPRINTF(" no edge\n");
                SET_RESULT(prev, result, 0, qry_match_len, min_match_len, FORWARD);
                           
                qry_match_len -= 1;
                mustmatch = 0;
                
                goto NEXT_SUBSTRING;
            }

			_PixelOfNode node;
			node.data = GETNODEHIST(cur, true);
			node_start = MKI(node.start);
			unsigned int node_end = MKI(node.end);
			
            XPRINTF(" Edge coordinates: %d - %d\n", node_start, node_end);
            {
                int edgelen = node_end - node_start + 1;
                int edge_matchlen = node_start + mustmatch;
                int past_node_end = node_end + 1;
                int dist_to_edge_end = mustmatch - edgelen;
                if (mustmatch) {
                    refpos = min(edge_matchlen, past_node_end);
                    qry_match_len += min(edgelen, mustmatch);
                    mustmatch = max(dist_to_edge_end, 0);
                }
                else {
                    // Try to walk the edge, the first char definitely matches
                    qry_match_len++;
                    refpos = node_start + 1;
                }
            }
            
            c = GETQCHAR(qrystart + qry_match_len);

            while (refpos <= node_end && c != '\0') 
            {
                char r = GETRCHAR(refpos);
                               
                XPRINTF(" Edge cmp ref: %d %c, qry: %d %c\n", refpos, r, qry_match_len, c);
                
                if (r != c) 
                {
                    // mismatch on edge
                    XPRINTF("mismatch on edge: %d, edge_pos: %d\n", qry_match_len, refpos - (node_start));
                    goto RECORD_RESULT;
                }
                
                qry_match_len++;
                refpos++;

                c = GETQCHAR(qrystart + qry_match_len);
            }
        }
        
        XPRINTF("end of string\n");
        
RECORD_RESULT:
		{
			//_PixelOfNode node;
			//node.data = getnodehist(cur, false);
	        SET_RESULT(cur, result, refpos - node_start, qry_match_len,
	                   min_match_len, FORWARD);
                   
	        mustmatch = refpos - node_start;
	        qry_match_len -= mustmatch + 1;
    	}
NEXT_SUBSTRING:
		{
			_PixelOfNode node;
			node.data = GETNODEHIST(prev, false);
	        arrayToAddress(node.suffix, cur);
		}
        //XPRINTF(" following suffix link. mustmatch:%d qry_match_len:%d sl:("fNID")\n",
        //       mustmatch, qry_match_len, NID(cur));
        do {} while (0);
    }

    return;
}

///////////////////////////////////////
//// Compute reverse substring matches
///////////////////////////////////////

__global__ void
mummergpuRCKernel(MatchCoord* match_coords,
               char* queries, 
               const int* queryAddrs,
			   const int* queryLengths,
               const int numQueries,
			   const int min_match_len) 
{
/*
   int qryid = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
   if (qryid >= numQueries) { return; }

   int qlen = queryLengths[qryid];

   XPRINTF("> rc qryid: %d\n", qryid);

   queries++; // skip the 'q' character


   // start at root for first query character
   TextureAddress cur;

   int mustmatch = 0;
   int qry_match_len = 0;

   int qryAddr=queryAddrs[qryid];
   MatchCoord * result = match_coords + qryAddr - __umul24(qryid, min_match_len + 1);
   queries += qryAddr;

   for (int qrystart = qlen;
       qrystart >= min_match_len ;
       qrystart--, result++)
   {
      #if VERBOSE
      queries[qrystart] = '\0';
	  XPRINTF("qry: ", queries);
      for (int j = qrystart-1; j >= 0; j--)
      { XPRINTF("%c", rc(queries[j])); }
      XPRINTF("\n");
      #endif

	  _PixelOfNode node;
      TextureAddress prev;

      if (((cur.data == 0)) || (qry_match_len < 1))
      {
	    // start at root of tree
	    cur.x = 0; cur.y = 1;
	    qry_match_len = 1; 
        mustmatch = 0;
      }

	  char c = rc(queries[qrystart-qry_match_len]);

	  XPRINTF("In node (%d,%d): starting with %c [%d] =>  \n", cur.x, cur.y, c, qry_match_len);

	  int refpos = 0;
	  while ((c != '\0'))
	  {
		 XPRINTF("Next edge to follow: %c (%d)\n", c, qry_match_len);

	     _PixelOfChildren children;
		 children.data = tex2D(childrentex,cur.x, cur.y);
		 prev = cur;

		 switch(c)
		 {
			case 'A': cur=children.children[0]; break;
			case 'C': cur=children.children[1]; break;
			case 'G': cur=children.children[2]; break;
			case 'T': cur=children.children[3]; break;
            default: cur.data = 0; break;
		 };		 

		 XPRINTF(" In node: (%d,%d)\n", cur.x, cur.y);

		 // No edge to follow out of the node
         if (cur.data == 0)
		 {
			XPRINTF(" no edge\n");

	        SET_RESULT(prev, (_MatchCoord*)result, 0, qry_match_len, min_match_len, 

                       REVERSE);

            qry_match_len -= 1;
            mustmatch = 0;

			goto NEXT_SUBSTRING;
		 }

         {
		   node.data = tex2D(nodetex, cur.data & 0xFFFF, cur.data >> 16);
         }

		 XPRINTF(" Edge coordinates: %d - %d\n", MKI(node.start), MKI(node.end));

         if (mustmatch)
         {
           int edgelen = MKI(node.end) - MKI(node.start)+1;
           if (mustmatch >= edgelen)
           {
             XPRINTF(" mustmatch(%d) >= edgelen(%d), skipping edge\n", mustmatch, edgelen);

             refpos = MKI(node.end)+1;
             qry_match_len += edgelen;
             mustmatch -= edgelen;
           }
           else
           {
             XPRINTF(" mustmatch(%d) < edgelen(%d), skipping to:%d\n", 
                     mustmatch, edgelen, MKI(node.start)+mustmatch);

             qry_match_len += mustmatch;
             refpos = MKI(node.start) + mustmatch;
             mustmatch = 0;
           }
         }
         else
         {
           // Try to walk the edge, the first char definitely matches
           qry_match_len++;
           refpos = MKI(node.start)+1;
         }

		 c = rc(queries[qrystart-qry_match_len]);

		 while (refpos <= MKI(node.end) && c != '\0')
		 { 
            char r = getRef(refpos
#if !REFTEX
							//FIXME: this needs to be a pointer to ref->d_ref_array
							,NULL
#endif
);

			XPRINTF(" Edge cmp ref: %d %c, qry: %d %c\n", refpos, r, qry_match_len, c);
						
			if (r != c)
			{
			   // mismatch on edge
			   XPRINTF("mismatch on edge: %d, edge_pos: %d\n", qry_match_len,refpos - (MKI(node.start)));
               goto RECORD_RESULT;
			}

	        qry_match_len++;
			refpos++;
			c = rc(queries[qrystart-qry_match_len]);
		 }
	  }

	  XPRINTF("end of string\n");

      RECORD_RESULT:
	
      SET_RESULT(cur, (_MatchCoord*)result, refpos - MKI(node.start), qry_match_len, 

                 min_match_len, REVERSE);

      mustmatch = refpos - MKI(node.start);
      qry_match_len -= mustmatch + 1;

      NEXT_SUBSTRING:

      node.data = tex2D(nodetex, prev.x, prev.y);
      cur = node.suffix;

      XPRINTF(" following suffix link. mustmatch:%d qry_match_len:%d sl:(%d,%d)\n", 
              mustmatch, qry_match_len, cur.x, cur.y);

      do {} while(0);
   }
*/	
   return;
}

__global__ void
printKernel(MatchInfo * matches,
            int totalMatches,
            Alignment * alignments,
#if !QRYTEX
#if COALESCED_QUERIES
            int  * queries,
#else
            char * queries,
#endif
#endif

#if !NODETEX
            _PixelOfNode* nodes,
#endif
                
#if !CHILDTEX
            _PixelOfChildren* childrenarr,
#endif
            const int * queryAddrs,
            const int * queryLengths,
            const int page_begin,
            const int page_end,
            const int page_shadow_left,
            const int page_shadow_right,
            const int min_match_length

#if TREE_ACCESS_HISTOGRAM
				,int* node_hist,
				int* child_hist
#endif
            ) 
{
  int matchid = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  if (matchid >= totalMatches) { return; }
    
  XPRINTF(">matchid: %d qry: %d\n", matchid, matches[matchid].queryid);

  int qryAddr = queryAddrs[matches[matchid].queryid];

  SHIFT_QUERIES(queries, qryAddr);
  
#if !QRYTEX
  XPRINTF("query: %s\n", queries);
#endif
  
  char queryflankingbase = GETQCHAR(matches[matchid].qrystartpos);
  
  // Find the top node to start printing from
  unsigned int matchaddr = matches[matchid].matchnode.data;
  unsigned int cur = matchaddr;
  unsigned int printParent = cur;
  
  _PixelOfNode node;
  node.data = GETNODE(cur, true);
  
  XPRINTF("starting node: %d "fNID" depth: %d\n", matches[matchid].matchnode, NID(cur), MKI(node.depth));

  while (MKI(node.depth) > min_match_length)
  {
    printParent = cur;
    arrayToAddress(node.parent, cur);
    node.data = GETNODE(cur, true);

    XPRINTF("par: "fNID" depth: %d\n", NID(cur), MKI(node.depth));
  }

  
  // traverse the tree starting at printParent
  unsigned int badParent = cur;
  cur = printParent;
  
  XPRINTF(" printParent: "fNID"\n", NID(printParent));
  
  char curchild = 'A';
  bool forceToParent = false;
  
  node.data = GETNODE(printParent, true);
  
  int matchlen = MKI(node.depth) - 1;
  int depthToGoldenPath = 0;
  
  int matchnum = matches[matchid].resultsoffset;

  // If the printparent is the matchnode, then we are already off the golden path
  if (printParent == matchaddr)
  {
    if (matches[matchid].edgematch > 0)
    {
      node.data = GETNODE(badParent, true);
      matchlen = MKI(node.depth)-1+matches[matchid].edgematch;
    }

    depthToGoldenPath = 1;
  }


  // keep going until I hit the printParent's parent
  while (cur != badParent)
  {
    _PixelOfChildren children;
    children.data = GETCHILDREN(cur, true);
    char isLeaf = children.leafchar;

    XPRINTF(" cur: "fNID" curchild: %c isLeaf:%d forceToParent:%d\n", 
            NID(cur), curchild, isLeaf, forceToParent);

    if (isLeaf || forceToParent)
    {
      // See if I am left maximal and print
      if (isLeaf && isLeaf != queryflankingbase)
      {
        int leafid = MKI(children.leafid);
        int left_in_ref = (leafid - 1) + page_begin; 
        int right_in_ref = left_in_ref + matchlen;

        if ((left_in_ref != page_begin || page_shadow_left == -1) && 
            (right_in_ref != page_end || page_shadow_right == -1))
        {
          if (!(left_in_ref > page_begin && right_in_ref < page_shadow_left))
          {
            //sprintf(buf, "%8d%10d%10d\n", left_in_ref, qrystartpos+1, matchlen);
            XPRINTF("%8d%10d%10d\n", 
                    left_in_ref, 
                    matches[matchid].qrystartpos+1, 
                    matchlen);

            alignments[matchnum].left_in_ref = left_in_ref;
            alignments[matchnum].matchlen = matchlen;
            matchnum++;
          }
        }
      }

      forceToParent = false;
      
      // now return to my parent and advance curchild

      node.data = GETNODE(cur, true);

      unsigned int myParent;
      arrayToAddress(node.parent, myParent);

      _PixelOfChildren pchildren;
      pchildren.data = GETCHILDREN(myParent, true);

      unsigned int pa, pc, pg, pt;
      arrayToAddress(pchildren.a, pa);
      arrayToAddress(pchildren.c, pc);
      arrayToAddress(pchildren.g, pg);
      arrayToAddress(pchildren.t, pt);

      if      (pa == cur) { curchild = 'C'; }
      else if (pc == cur) { curchild = 'G'; }
      else if (pg == cur) { curchild = 'T'; }
      else if (pt == cur) { curchild = '$'; }
      else  // I must be the $ child, go up a level
      {
        forceToParent = true;
      }

      cur = myParent;

      if (depthToGoldenPath) { depthToGoldenPath--; }

      if (depthToGoldenPath == 0)
      {
        node.data = GETNODE(cur, true);
        matchlen = MKI(node.depth)-1;
      }
    }
    else
    {
      // try to walk down the tree
      _PixelOfChildren children;
      children.data = GETCHILDREN(cur, true);

      char goldenChild = 0;

      if (depthToGoldenPath == 0)
      {
        // we are currently on the golden path
        // one of the children is also on the golden path
        goldenChild = GETQCHAR(matches[matchid].qrystartpos+matchlen+1);
      }

      do
      {
        if (curchild == 'A')
        {
          if (children.a.x || children.a.y || children.a.z)
          { 
            XPRINTF("  -> A\n"); 
            arrayToAddress(children.a, cur);
            break; 
          }

          curchild = 'C';
        }

        if (curchild == 'C')
        {
          if (children.c.x || children.c.y || children.c.z)
          { 
            XPRINTF("  -> C\n"); 
            arrayToAddress(children.c, cur); 
            break; 
          }
          curchild = 'G';
        }

        if (curchild == 'G')
        {
          if (children.g.x || children.g.y || children.g.z)
          { 
            XPRINTF("  -> G\n"); 
            arrayToAddress(children.g, cur); 
            break; 
          }
          curchild = 'T';
        }

        if (curchild == 'T')
        {
          if (children.t.x || children.t.y || children.t.z)
          { 
            XPRINTF("  -> T\n"); 
            arrayToAddress(children.t, cur); 
            break; 
          }
          curchild = '$';
        }

        if (curchild == '$')
        {
          if (children.d.x || children.d.y || children.d.z)
          { 
            XPRINTF("  -> $\n"); 
            arrayToAddress(children.d, cur); 
            break; 
          }
        }

        // checked all of the children, go back to parent
        forceToParent = true;
      } 
      while (0);

      if (!forceToParent)
      {
        if (depthToGoldenPath == 0)
        {
          if (curchild == goldenChild)
          {
            node.data = GETNODE(cur, true);
            matchlen = MKI(node.depth)-1;

            if (cur == matchaddr)
            {
              // we overextended the golden path
              depthToGoldenPath = 1;

              if (matches[matchid].edgematch > 0)
              {
				unsigned int par;
                arrayToAddress(node.parent, par);

                node.data = GETNODE(par, true);
                matchlen = MKI(node.depth) - 1 + matches[matchid].edgematch;
              }
            }
          }
          else
          {
            depthToGoldenPath = 1;
          }
        }
        else
        {
          depthToGoldenPath++;
        }
          
        curchild = 'A';
      }
    }
  }
}
  
#endif // #ifndef _MUMMERGPU_HH_
