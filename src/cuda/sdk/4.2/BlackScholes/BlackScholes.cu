#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

// Utilities and system includes
#include <shrUtils.h>
#include <shrQATest.h>
#include <cutil_inline.h>

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 100000;
const int  NUM_ITERATIONS = 1;


const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
  shrQAStart(argc, argv);

  // Start logs
  shrSetLogFileName ("BlackScholes.txt");
  shrLog("%s Starting...\n\n", argv[0]);

    //'h_' prefix - CPU (host) memory space
    float
        //Results calculated by CPU for reference
        *h_CallResultCPU,
        *h_PutResultCPU,
        //CPU copy of GPU results
        *h_CallResultGPU,
        *h_PutResultGPU,
        //CPU instance of input data
        *h_StockPrice,
        *h_OptionStrike,
        *h_OptionYears;

    //'d_' prefix - GPU (device) memory space
    float
        //Results calculated by GPU
        *d_CallResult,
        *d_PutResult,
        //GPU instance of input data
        *d_StockPrice,
        *d_OptionStrike,
        *d_OptionYears;

    double
        delta, ref, sum_delta, sum_ref, max_delta, L1norm, gpuTime;

    unsigned int hTimer;
    int i, devID;


    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        devID = cutilDeviceInit(argc, argv);
        if (devID < 0) {
           printf("No CUDA Capable devices found, exiting...\n");
           shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
	    }
    } else {
        hipSetDevice( devID = cutGetMaxGflopsDeviceId() );
    }
		
    cutilCheckError( cutCreateTimer(&hTimer) );

    shrLog("Initializing data...\n");
        shrLog("...allocating CPU memory for options.\n");
        h_CallResultCPU = (float *)malloc(OPT_SZ);
        h_PutResultCPU  = (float *)malloc(OPT_SZ);
        h_CallResultGPU = (float *)malloc(OPT_SZ);
        h_PutResultGPU  = (float *)malloc(OPT_SZ);
        h_StockPrice    = (float *)malloc(OPT_SZ);
        h_OptionStrike  = (float *)malloc(OPT_SZ);
        h_OptionYears   = (float *)malloc(OPT_SZ);

        shrLog("...allocating GPU memory for options.\n");
        cutilSafeCall( hipMalloc((void **)&d_CallResult,   OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_PutResult,    OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_StockPrice,   OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_OptionStrike, OPT_SZ) );
        cutilSafeCall( hipMalloc((void **)&d_OptionYears,  OPT_SZ) );

        shrLog("...generating input data in CPU mem.\n");
        srand(5347);
        //Generate options set
        for(i = 0; i < OPT_N; i++){
            h_CallResultCPU[i] = 0.0f;
            h_PutResultCPU[i]  = -1.0f;
            h_StockPrice[i]    = RandFloat(5.0f, 30.0f);
            h_OptionStrike[i]  = RandFloat(1.0f, 100.0f);
            h_OptionYears[i]   = RandFloat(0.25f, 10.0f);
        }

        shrLog("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing
        cutilSafeCall( hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice) );
    shrLog("Data init done.\n\n");


    shrLog("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
        cutilSafeCall( cutilDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        for(i = 0; i < NUM_ITERATIONS; i++){
            BlackScholesGPU<<<480, 128>>>(
                d_CallResult,
                d_PutResult,
                d_StockPrice,
                d_OptionStrike,
                d_OptionYears,
                RISKFREE,
                VOLATILITY,
                OPT_N
            );
            cutilCheckMsg("BlackScholesGPU() execution failed\n");
        }
        cutilSafeCall( cutilDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
        gpuTime = cutGetTimerValue(hTimer) / NUM_ITERATIONS;
        
    //Both call and put is calculated
    shrLog("Options count             : %i     \n", 2 * OPT_N);
    shrLog("BlackScholesGPU() time    : %f msec\n", gpuTime);
    shrLog("Effective memory bandwidth: %f GB/s\n", ((double)(5 * OPT_N * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
    shrLog("Gigaoptions per second    : %f     \n\n", ((double)(2 * OPT_N) * 1E-9) / (gpuTime * 1E-3));

    shrLogEx(LOGBOTH | MASTER, 0, "BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u options, NumDevsUsed = %u, Workgroup = %u\n", 
           (((double)(2.0 * OPT_N) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime*1e-3, (2 * OPT_N), 1, 128);

    shrLog("\nReading back GPU results...\n");
        //Read back GPU results to compare them to CPU results
        cutilSafeCall( hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost) );
        cutilSafeCall( hipMemcpy(h_PutResultGPU,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost) );


    shrLog("Checking the results...\n");
        shrLog("...running CPU calculations.\n\n");
        //Calculate options values on CPU
        BlackScholesCPU(
            h_CallResultCPU,
            h_PutResultCPU,
            h_StockPrice,
            h_OptionStrike,
            h_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );

        shrLog("Comparing the results...\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        max_delta = 0;
        for(i = 0; i < OPT_N; i++){
            ref   = h_CallResultCPU[i];
            delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);
            if(delta > max_delta) max_delta = delta;
            sum_delta += delta;
            sum_ref   += fabs(ref);
        }
        L1norm = sum_delta / sum_ref;
        shrLog("L1 norm: %E\n", L1norm);
        shrLog("Max absolute error: %E\n\n", max_delta);

    shrLog("Shutting down...\n");
        shrLog("...releasing GPU memory.\n");
        cutilSafeCall( hipFree(d_OptionYears)  );
        cutilSafeCall( hipFree(d_OptionStrike) );
        cutilSafeCall( hipFree(d_StockPrice)  );
        cutilSafeCall( hipFree(d_PutResult)    );
        cutilSafeCall( hipFree(d_CallResult)   );

        shrLog("...releasing CPU memory.\n");
        free(h_OptionYears);
        free(h_OptionStrike);
        free(h_StockPrice);
        free(h_PutResultGPU);
        free(h_CallResultGPU);
        free(h_PutResultCPU);
        free(h_CallResultCPU);
        cutilCheckError( cutDeleteTimer(hTimer) );
	shrLog("Shutdown done.\n");

	printf("\n[BlackScholes] - Test Summary\n");

    cutilDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, (L1norm < 1e-6) ? QA_PASSED : QA_FAILED);
}
