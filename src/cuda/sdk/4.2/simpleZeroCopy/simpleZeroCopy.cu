#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)
#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);                                                  
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv)
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    // If the command-line has a device number specified, use it
    if (checkCmdLineFlag(argc, argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, argv, "device=");
        if (devID < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(-1);
        }
        else
        {
            devID = gpuDeviceInit(devID);
            if (devID < 0)
            {
                printf("exiting...\n");
                shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                exit(-1);
            }
        }
    }
    else
    {
        // Otherwise pick the device with highest Gflops/s
        devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors( hipSetDevice( devID ) );
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
    return devID;
}
// end of CUDA Helper Functions

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N)
    c[idx] = a[idx] + b[idx];
}

// Allocate generic memory with malloc() and pin it laster instead of using hipHostAlloc()
bool bPinGenericMemory = false;

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

int main(int argc, char **argv)
{
  int n, nelem, deviceCount;
  int idev = 0; // use default device 0
  char *device = NULL;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;                      // Pinned memory allocated on the CPU
  float *a_UA, *b_UA, *c_UA;             // Non-4K Aligned Pinned memory on the CPU
  float *d_a, *d_b, *d_c;                // Device pointers for mapped memory  
  float errorNorm, refNorm, ref, diff;
  hipDeviceProp_t deviceProp;

  shrQAStart(argc, argv);

  if(checkCmdLineFlag(argc, (const char **)argv, "help"))
  {
    printf("Usage:  simpleZeroCopy [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    printf("  --use_generic_memory (optional) use generic page-aligned for system memory\n");
    shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
  }

  /* Get the device selected by the user or default to 0, and then set it. */
  if(getCmdLineArgumentString(argc, (const char**)argv, "device", &device))
  {
    hipGetDeviceCount(&deviceCount);
    idev = atoi(device);
    if(idev >= deviceCount || idev < 0)
    {
      fprintf(stderr, "Device number %d is invalid, will use default CUDA device 0.\n", idev);
      idev = 0;
    }
  }
  
  if( checkCmdLineFlag( argc, (const char **)argv, "use_generic_memory") ) 
  {
#if defined(__APPLE__) || defined(MACOSX)
    bPinGenericMemory = false;  // Generic Pinning of System Paged memory is not currently supported on Mac OSX 
#else
    bPinGenericMemory = true;
#endif
  }

  if (bPinGenericMemory) {
     printf("> Using Generic System Paged Memory (malloc)\n");
  } else {
     printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
  }

  checkCudaErrors(hipSetDevice(idev));

  /* Verify the selected device supports mapped memory and set the device
     flags for mapping host memory. */

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020
  if(!deviceProp.canMapHostMemory)
  {
    fprintf(stderr, "Device %d does not support mapping CPU host memory!\n", idev);
    hipDeviceReset();	
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
  }
  checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
    fprintf(stderr, "CUDART version %d.%d does not support <hipDeviceProp_t.canMapHostMemory> field\n", , CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    hipDeviceReset();	
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
#endif

#if CUDART_VERSION < 4000
  if (bPinGenericMemory)
  {
    fprintf(stderr, "CUDART version %d.%d does not support <hipHostRegister> function\n", CUDART_VERSION/1000, (CUDART_VERSION%100)/10);
    hipDeviceReset();	
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
  }
#endif

  /* Allocate mapped CPU memory. */

  nelem = 1048576;
  bytes = nelem*sizeof(float);
  if (bPinGenericMemory)
  {
#if CUDART_VERSION >= 4000
    a_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
    b_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );
    c_UA = (float *) malloc( bytes + MEMORY_ALIGNMENT );

    // We need to ensure memory is aligned to 4K (so we will need to padd memory accordingly)
    a = (float *) ALIGN_UP( a_UA, MEMORY_ALIGNMENT );
    b = (float *) ALIGN_UP( b_UA, MEMORY_ALIGNMENT );
    c = (float *) ALIGN_UP( c_UA, MEMORY_ALIGNMENT );

    checkCudaErrors(hipHostRegister(a, bytes, hipHostMallocMapped));
    checkCudaErrors(hipHostRegister(b, bytes, hipHostMallocMapped));
    checkCudaErrors(hipHostRegister(c, bytes, hipHostMallocMapped));
#endif
  }
  else
  {
#if CUDART_VERSION >= 2020
    flags = hipHostMallocMapped;
    checkCudaErrors(hipHostAlloc((void **)&a, bytes, flags));
    checkCudaErrors(hipHostAlloc((void **)&b, bytes, flags));
    checkCudaErrors(hipHostAlloc((void **)&c, bytes, flags));
#endif
  }

  /* Initialize the vectors. */

  for(n = 0; n < nelem; n++)
  {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  /* Get the device pointers for the pinned CPU memory mapped into the GPU
     memory space. */

#if CUDART_VERSION >= 2020
  checkCudaErrors(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
  checkCudaErrors(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
  checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

  /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory. */ 
  printf("> vectorAddGPU kernel will add vectors using mapped CPU memory...\n");
  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem/(float)block.x));
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);  
  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("vectorAddGPU() execution failed");

  /* Compare the results */

  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;
  for(n = 0; n < nelem; n++)
  {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff*diff;
    refNorm += ref*ref;
  }
  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);

  /* Memory clean up */

  printf("> Releasing CPU memory...\n");
  if (bPinGenericMemory)
  {
#if CUDART_VERSION >= 4000
    checkCudaErrors(hipHostUnregister(a));
    checkCudaErrors(hipHostUnregister(b));
    checkCudaErrors(hipHostUnregister(c));
    free(a_UA);
    free(b_UA);
    free(c_UA);
#endif
  }
  else
  {
#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostFree(a));
    checkCudaErrors(hipHostFree(b));
    checkCudaErrors(hipHostFree(c));
#endif
  }

  hipDeviceReset();	
  shrQAFinishExit(argc, (const char **)argv, (errorNorm/refNorm < 1.e-6f) ? QA_PASSED : QA_FAILED);
}
