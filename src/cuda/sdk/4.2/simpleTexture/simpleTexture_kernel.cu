#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _SIMPLETEXTURE_KERNEL_H_
#define _SIMPLETEXTURE_KERNEL_H_

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
transformKernel( float* g_odata, int width, int height, float theta) 
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = x / (float) width;
    float v = y / (float) height;

    // transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u*cosf(theta) - v*sinf(theta) + 0.5f;
    float tv = v*cosf(theta) + u*sinf(theta) + 0.5f;

    // read from texture and write to global memory
    g_odata[y*width + x] = tex2D(tex, tu, tv);
}

#endif // #ifndef _SIMPLETEXTURE_KERNEL_H_
