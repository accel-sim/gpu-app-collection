/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
* 1D DWT for Haar wavelet and signals with a length which is a power of 2.
* The code reduces bank conflicts and non-coalesced reads / writes as 
* appropriate but does not fully remove them because the computational 
* overhead to achieve this would outweighs the benefit (see inline comments
* for more details).
* Large signals are subdivided into sub-signals with 512 elements and the 
* wavelet transform for these is computed with one block over 10 decomposition 
* levels. The resulting signal consisting of the approximation coefficients at
* level X is then processed in a subsequent step on the device. This requires 
* interblock syncronization which is only possible on host side.
* Detail coefficients which have been computed are not further referenced 
* during the decomposition so that they can be stored directly in their final  
* position in global memory. The transform and its storing scheme preserve  
* locality in the coefficients so that these writes are coalesced. 
* Approximation coefficients are stored in shared memory because they are 
* needed to compute the subsequent decomposition step. The top most 
* approximation coefficient for a sub-signal processed by one block is stored 
* in a special global memory location to simplify the processing after the 
* interblock synchronization. 
* Most books on wavelets explain the Haar wavelet decompositon. A good freely
* available resource is the Wavelet primer by Stollnitz et al.
* http://grail.cs.washington.edu/projects/wavelets/article/wavelet1.pdf
* http://grail.cs.washington.edu/projects/wavelets/article/wavelet2.pdf
* The basic of all Wavelet transforms is to decompose a signal into 
* approximation (a) and detail (d) coefficients where the detail tends to be 
* small or zero which allows / simplifies compression. The following "graphs" 
* demonstrate the transform for a signal
* of length eight. The index always describes the decomposition level where
* a coefficient arises. The input signal is interpreted as approximation signal
* at level 0. The coefficients computed on the device are stored in the same 
* scheme as in the example. This data strucure is particularly well suited for 
* compression and also preserves the hierachical strucure of the decomposition.

-------------------------------------------------
| a_0 | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 |
-------------------------------------------------

-------------------------------------------------
| a_1 | a_1 | a_1 | a_1 | d_1 | d_1 | d_1 | d_1 |
-------------------------------------------------

-------------------------------------------------
| a_2 | a_2 | d_2 | d_2 | d_1 | d_1 | d_1 | d_1 |
-------------------------------------------------

-------------------------------------------------
| a_3 | d_3 | d_2 | d_2 | d_1 | d_1 | d_1 | d_1 |
-------------------------------------------------

* Host code.
*/

#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil_inline.h>
#include <shrQATest.h>

// constants which are used in host and device code
#define             INV_SQRT_2      0.70710678118654752440f;
const unsigned int  LOG_NUM_BANKS = 4;
const unsigned int  NUM_BANKS     = 16;

////////////////////////////////////////////////////////////////////////////////
// includes, kernels
#include <dwtHaar1D_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
CUTBoolean  getLevels( unsigned int len, unsigned int* levels);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    // run test
    runTest( argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Perform the wavelet decomposition
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    bool bResult = false; // flag for final validation of the results

    char* s_fname;
    char* r_fname;
    char* r_gold_fname;
    const char usage[] = 
    {
        "\nUsage:\n"
        "  dwtHaar1D --signal=<signal_file> --result=<result_file> --gold=<gold_file>\n\n"
        "  <signal_file> Input file containing the signal\n"
        "  <result_file> Output file storing the result of the wavelet decomposition\n"
        "  <gold_file>   Input file containing the reference result of the wavelet decomposition\n"
        "\nExample:\n"
        "  bin\\win32\\release\\dwtHaar1D\n"
        "       --signal=projects\\dwtHaar1D\\data\\signal.dat\n"
        "       --result=projects\\dwtHaar1D\\data\\regression.dat\n"
        "       --gold=projects\\dwtHaar1D\\data\\regression.gold.dat\n"
    };

    shrQAStart(argc, argv);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    // file names, either specified as cmd line args or use default
    if( argc == 4) 
    {
        if ((cutGetCmdLineArgumentstr(argc, (const char**)argv, "signal", &s_fname) != CUTTrue) ||
            (cutGetCmdLineArgumentstr(argc, (const char**)argv, "result", &r_fname) != CUTTrue) ||
            (cutGetCmdLineArgumentstr(argc, (const char**)argv, "gold", &r_gold_fname) != CUTTrue)) 
        {
            fprintf(stderr, "Invalid input syntax.\n%s", usage);
            shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
        }
    }
    else 
    {
        s_fname = cutFindFilePath("signal.dat", argv[0]);
        r_fname = cutFindFilePath("regression.dat", argv[0]);
        r_gold_fname = cutFindFilePath("regression.gold.dat", argv[0]);
    }

    // read in signal
    unsigned int slength = 0;
    float* signal = NULL;
    if (s_fname == NULL)
    {
        fprintf(stderr, "Cannot find the file containing the signal.\n%s", usage);
        cutilDeviceReset();
        exit(1);
    }
    if (cutReadFilef( s_fname, &signal, &slength) == CUTTrue) {
        printf("Reading signal from %s\n", s_fname);
    } else {
        cutilDeviceReset();
        shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }

    // get the number of decompositions necessary to perform a full decomposition
    unsigned int dlevels_complete = 0;
    if( CUTTrue != getLevels( slength, &dlevels_complete)) 
    {
        // error message
        fprintf( stderr, "Signal length not supported.\n");
        // cleanup and abort
        cutFree( signal);
        return;
    }

    // device in data
    float* d_idata = NULL;
    // device out data
    float* d_odata = NULL;
    // device approx_final data
    float* approx_final = NULL;
    // The very final approximation coefficient has to be written to the output
    // data, all others are reused as input data in the next global step and 
    // therefore have to be written to the input data again.
    // The following flag indicates where to copy approx_final data
    //   - 0 is input, 1 is output
    int approx_is_input;

    // allocate device mem
    const unsigned int smem_size = sizeof(float) * slength;
    cutilSafeCall( hipMalloc( (void**) &d_idata, smem_size));
    cutilSafeCall( hipMalloc( (void**) &d_odata, smem_size));
    cutilSafeCall( hipMalloc( (void**) &approx_final, smem_size));
    // copy input data to device
    cutilSafeCall( hipMemcpy( d_idata, signal, smem_size, 
        hipMemcpyHostToDevice) ); 

    // clear result memory
    float* tmp = (float*) malloc( smem_size);
    for( unsigned int i = 0; i < slength; ++i) 
    {
        tmp[i] = 0.0;
    }
    cutilSafeCall( hipMemcpy( d_odata, tmp, smem_size, 
                                hipMemcpyHostToDevice) ); 
    free( tmp);

    // total number of threads
    // in the first decomposition step always one thread computes the average and
    // detail signal for one pair of adjacent values
    unsigned int num_threads_total_left = slength / 2;
    // decomposition levels performed in the current / next step
    unsigned int  dlevels_step = dlevels_complete;

    // 1D signal so the arrangement of elements is also 1D
    dim3  block_size;
    dim3  grid_size;  

    // number of decomposition levels left after one iteration on the device
    unsigned int dlevels_left = dlevels_complete;

    // if less or equal 1k elements, then the data can be processed in one block,
    // this avoids the Wait-For-Idle (WFI) on host side which is necessary if the 
    // computation is split accross multiple SM's if enough input data
    if( dlevels_complete <= 10) 
    {
        // decomposition can be performed at once
        block_size.x = num_threads_total_left;   
        approx_is_input = 0;
}
    else 
    {
        // 512 threads per block
        grid_size.x = (num_threads_total_left / 512);
        block_size.x = 512;

        // 512 threads corresponds to 10 decomposition steps
        dlevels_step = 10;
        dlevels_left -= 10;

        approx_is_input = 1;
    }

    // do until full decomposition is accomplished
    while( 0 != num_threads_total_left) 
    {
        // double the number of threads as bytes
        unsigned int mem_shared = (2 * block_size.x) * sizeof( float);
        // extra memory requirements to avoid bank conflicts
        mem_shared += ((2 * block_size.x) / NUM_BANKS) * sizeof( float);

        // run kernel
        dwtHaar1D<<<grid_size, block_size, mem_shared >>>( d_idata, d_odata,
                                                           approx_final,
                                                           dlevels_step,
                                                           num_threads_total_left,
                                                           block_size.x );

	// Copy approx_final to appropriate location
        if (approx_is_input)
          {
            cutilSafeCall (hipMemcpy (d_idata, approx_final, grid_size.x * 4,
                                        hipMemcpyDeviceToDevice) );
          }
        else
          {
            cutilSafeCall (hipMemcpy (d_odata, approx_final, grid_size.x * 4,
                                        hipMemcpyDeviceToDevice) );
          }

        // update level variables
        if( dlevels_left < 10) 
          {
            // approx_final = d_odata;
            approx_is_input = 0;
          }

        // more global steps necessary
        dlevels_step = (dlevels_left > 10) ? dlevels_left - 10 : dlevels_left;
        dlevels_left -= 10;

        // after each step only half the threads are used any longer
        // therefore after 10 steps 2^10 less threads
        num_threads_total_left = num_threads_total_left >> 10;

        // update block and grid size
        grid_size.x = (num_threads_total_left / 512) 
                       + (0 != (num_threads_total_left % 512)) ? 1 : 0;
        if( grid_size.x <= 1) 
        {
            block_size.x = num_threads_total_left;
        }
    }

    // get the result back from the server
    // allocate mem for the result
    float* odata = (float*) malloc( smem_size);
    cutilSafeCall( hipMemcpy( odata, d_odata, smem_size, 
                                hipMemcpyDeviceToHost));

    // post processing
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test
        if (r_fname == NULL) {
            fprintf(stderr, "Cannot write the output file storing the result of the wavelet decomposition.\n%s", usage);
            cutilDeviceReset();
            exit(1);
        }
        if (cutWriteFilef( r_fname, odata, slength, 0.001f, false) == CUTTrue) 
            printf("Writing result to %s\n", r_fname);
        else {
            cutilDeviceReset();
            shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
        }
    }
    else 
    {
        // load the reference solution
        unsigned int len_reference = 0;
        float* reference = NULL;
        if (r_gold_fname == NULL)
        {
            fprintf(stderr, "Cannot read the file containing the reference result of the wavelet decomposition.\n%s", usage);
            cutilDeviceReset();
            exit(1);
        }
        if (cutReadFilef( r_gold_fname, &reference, &len_reference) == CUTTrue) 
            printf("Reading reference result from %s\n", r_gold_fname);
        else {
            cutilDeviceReset();
            shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
        }
        cutilCondition( slength == len_reference);

        // compare the computed solution and the reference
        bResult = (bool)cutComparefe( reference, odata, slength, 0.001f);
        cutFree( reference);
    }

    // free allocated host and device memory
    cutilSafeCall(hipFree(d_odata));
    cutilSafeCall(hipFree(d_idata));
    cutilSafeCall(hipFree(approx_final));

    cutFree( signal);
    free( odata);
    cutFree( s_fname); 
    cutFree( r_fname);  
    cutFree( r_gold_fname);   

    cutilDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bResult ? QA_PASSED : QA_FAILED) );
}

////////////////////////////////////////////////////////////////////////////////
//! Get number of decomposition levels to perform a full decomposition
//! Also check if the input signal size is suitable
//! @return  CUTTrue if the number of decomposition levels could be determined
//!          and the signal length is supported by the implementation,
//!          otherwise CUTFalse
//! @param   len  length of input signal
//! @param   levels  number of decomposition levels necessary to perform a full 
//!           decomposition
////////////////////////////////////////////////////////////////////////////////
CUTBoolean
getLevels( unsigned int len, unsigned int* levels) 
{
    CUTBoolean retval = CUTFalse;

    // currently signals up to a length of 2^20 supported
    for( unsigned int i = 0; i < 20; ++i) 
    {
        if( len == (1 << i)) 
        {
            *levels = i;
            retval = CUTTrue;
            break;
        }
    }

    return retval;
}
