#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 *
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to 
 * initialize an array to a specific value, after which the array is 
 * copied to the host (CPU) memory.  To increase performance, multiple 
 * kernel/memcopy pairs are launched asynchronously, each pair in its 
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5 
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

const char *sSDKsample = "simpleStreams";

const char *sEventSyncMethod[] = 
{ 
	"hipEventDefault", 
	"hipEventBlockingSync", 
	"hipEventDisableTiming", 
	NULL 
};

const char *sDeviceSyncMethod[] = 
{ 
	"hipDeviceScheduleAuto", 
	"hipDeviceScheduleSpin", 
	"hipDeviceScheduleYield", 
	"INVALID", 
	"hipDeviceScheduleBlockingSync", 
	NULL 
};

// Include headers
#include <stdio.h>

// CUDA utilities and system includes
#include <hip/hip_runtime.h>

// Shared Library Test Functions
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrUtils.h>
#include <shrQATest.h>

#ifndef WIN32
#include <sys/mman.h> // for mmap() / munmap()
#endif

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);                                                  
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv)
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    // If the command-line has a device number specified, use it
    if (checkCmdLineFlag(argc, argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, argv, "device=");
        if (devID < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(-1);
        }
        else
        {
            devID = gpuDeviceInit(devID);
            if (devID < 0)
            {
                printf("exiting...\n");
                shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                exit(-1);
            }
        }
    }
    else
    {
        // Otherwise pick the device with highest Gflops/s
        devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors( hipSetDevice( devID ) );
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
    return devID;
}
// end of CUDA Helper Functions


// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT  4096
#define ALIGN_UP(x,size) ( ((size_t)x+(size-1))&(~(size-1)) )

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=0;i<num_iterations;i++)
	    g_data[idx] += *factor;	// non-coalesced on purpose, to burn time
}

int correct_data(int *a, const int n, const int c)
{
    for(int i = 0; i < n; i++) {
        if(a[i] != c) {
           printf("%d: %d %d\n", i, a[i], c);
           return 0;
        }
    }
    return 1;
}

inline void 
AllocateHostMemory(bool bPinGenericMemory, int **pp_a, int **ppAligned_a, int nbytes)
{
#if CUDART_VERSION >= 4000
    if (bPinGenericMemory)
    {
        // allocate a generic page-aligned chunk of system memory
    #ifdef WIN32
        printf("> VirtualAlloc() allocating %4.2f Mbytes of (generic page-aligned system memory)\n", (float)nbytes/1048576.0f);
        *pp_a = (int *) VirtualAlloc( NULL, (nbytes + MEMORY_ALIGNMENT), MEM_RESERVE|MEM_COMMIT, PAGE_READWRITE );
    #else
        printf("> mmap() allocating %4.2f Mbytes (generic page-aligned system memory)\n", (float)nbytes/1048576.0f);
        *pp_a = (int *) mmap        ( NULL, (nbytes + MEMORY_ALIGNMENT), PROT_READ|PROT_WRITE, MAP_PRIVATE|MAP_ANON, -1, 0 );
    #endif

        *ppAligned_a = (int *)ALIGN_UP(*pp_a, MEMORY_ALIGNMENT);

        printf("> hipHostRegister() registering %4.2f Mbytes of generic allocated system memory\n", (float)nbytes/1048576.0f);
        // pin allocate memory
        checkCudaErrors( hipHostRegister(*ppAligned_a, nbytes, hipHostRegisterMapped) );
    }
    else
#endif
    {
        printf("> hipHostMalloc() allocating %4.2f Mbytes of system memory\n", (float)nbytes/1048576.0f);
        // allocate host memory (pinned is required for achieve asynchronicity)
        checkCudaErrors( hipHostMalloc((void**)pp_a, nbytes) ); 
        *ppAligned_a = *pp_a; 
    }
}

inline void
FreeHostMemory(bool bPinGenericMemory, int **pp_a, int **ppAligned_a, int nbytes)
{
#if CUDART_VERSION >= 4000
    // CUDA 4.0 support pinning of generic host memory
    if (bPinGenericMemory)
    {
        // unpin and delete host memory
        checkCudaErrors( hipHostUnregister(*ppAligned_a) );
  #ifdef WIN32
        VirtualFree(*pp_a, 0, MEM_RELEASE);
  #else
        munmap(*pp_a, nbytes);
  #endif
    }
    else
#endif
    {
        hipHostFree(*pp_a);
    }
}

static char *sSyncMethod[] = 
{
    "0 (Automatic Blocking)",
    "1 (Spin Blocking)",
    "2 (Yield Blocking)",
    "3 (Undefined Blocking Method)",
    "4 (Blocking Sync Event) = low CPU utilization",
    NULL
};

void printHelp()
{
    printf("Usage: %s [options below]\n", sSDKsample);
    printf("\t--sync_method=n for CPU/GPU synchronization\n");
    printf("\t             n=%s\n", sSyncMethod[0]);
    printf("\t             n=%s\n", sSyncMethod[1]);
    printf("\t             n=%s\n", sSyncMethod[2]);
    printf("\t   <Default> n=%s\n", sSyncMethod[4]);
    printf("\t--use_generic_memory (default) use generic page-aligned for system memory\n");
    printf("\t--use_cuda_malloc_host (optional) use hipHostMalloc to allocate system memory\n");
}

#if defined(__APPLE__) || defined(MACOSX)
#define DEFAULT_PINNED_GENERIC_MEMORY false
#else
#define DEFAULT_PINNED_GENERIC_MEMORY true
#endif

int main(int argc, char **argv)
{
    int cuda_device = 0;
    int nstreams = 4;               // number of streams for CUDA calls
    int nreps = 10;                 // number of times each experiment is repeated
    int n = 16 * 1024 * 1024;       // number of ints in the data set
    int nbytes = n * sizeof(int);   // number of data bytes
    dim3 threads, blocks;           // kernel launch configuration
    float elapsed_time, time_memcpy, time_kernel;   // timing variables
    float scale_factor = 1.0f;

    // allocate generic memory and pin it laster instead of using hipHostAlloc()

    bool bPinGenericMemory  = DEFAULT_PINNED_GENERIC_MEMORY; // we want this to be the default behavior
    int  device_sync_method = hipDeviceScheduleBlockingSync; // by default we use BlockingSync

    int niterations;	// number of iterations for the loop inside the kernel

    shrQAStart(argc, argv);

    printf("[ %s ]\n\n", sSDKsample);
    if( checkCmdLineFlag( argc, (const char **)argv, "help") ) {
        printHelp();
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
    }
    if( (device_sync_method = getCmdLineArgumentInt( argc, (const char **)argv, "sync_method" )) >= 0 ) {
        if (device_sync_method == 0 || device_sync_method == 1 || device_sync_method == 2 || device_sync_method == 4) {
            printf("Device synchronization method set to = %s\n", sSyncMethod[device_sync_method]);
            printf("Setting reps to 100 to demonstrate steady state\n");
            nreps = 100;
        } else {
            printf("Invalid command line option sync_method=\"%d\"\n", device_sync_method);
            shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
        }
	} else {
        printHelp();
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
	}

    if( checkCmdLineFlag( argc, (const char **)argv, "use_generic_memory") ) {
#if defined(__APPLE__) || defined(MACOSX)
        bPinGenericMemory = false;  // Generic Pinning of System Paged memory not currently supported on Mac OSX 
#else
        bPinGenericMemory = true;
#endif
    }
    if( checkCmdLineFlag( argc, (const char **)argv, "use_cuda_malloc_host") ) {
        bPinGenericMemory = false;
    }

    printf("\n> ");
    cuda_device = findCudaDevice(argc, (const char **)argv);

    // check the compute capability of the device
    int num_devices=0;
    checkCudaErrors( hipGetDeviceCount(&num_devices) );
    if(0==num_devices)
    {
        printf("your system does not have a CUDA capable device, waiving test...\n");
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
    }
	
    // check if the command-line chosen device ID is within range, exit if not
    if( cuda_device >= num_devices )
    {
        printf("cuda_device=%d is invalid, must choose device ID between 0 and %d\n", cuda_device, num_devices-1);
        shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }

    hipSetDevice( cuda_device );

    // Checking for compute capabilities
    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (1 == deviceProp.major) && (deviceProp.minor < 1)) {
        printf("%s does not have Compute Capability 1.1 or newer.  Reducing workload.\n", deviceProp.name);
    }

    if(deviceProp.major >= 2) {
        niterations = 100;
    } else {		    
        if(deviceProp.minor > 1) {
            niterations = 5;
        } else {
            niterations = 1; // reduced workload for compute capability 1.0 and 1.1
        }
    }

    // Check if GPU can map host memory (Generic Method), if not then we override bPinGenericMemory to be false
    if (bPinGenericMemory) {
       printf("Device: <%s> canMapHostMemory: %s\n", deviceProp.name, deviceProp.canMapHostMemory ? "Yes" : "No");
       if (deviceProp.canMapHostMemory == 0) {
          printf("Using hipHostMalloc, CUDA device does not support mapping of generic host memory\n");
          bPinGenericMemory = false;
       }
    }

    // Anything that is less than 32 Cores will have scaled down workload
    scale_factor = max((32.0f / (ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * (float)deviceProp.multiProcessorCount) ), 1.0f);
    n = (int)rint( (float)n / scale_factor );

    printf("> CUDA Capable: SM %d.%d hardware\n", deviceProp.major, deviceProp.minor);
    printf("> %d Multiprocessor(s) x %d (Cores/Multiprocessor) = %d (Cores)\n", 
            deviceProp.multiProcessorCount,
            ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
            ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

    printf("> scale_factor = %1.4f\n", 1.0f/scale_factor);
    printf("> array_size   = %d\n\n", n);

    // enable use of blocking sync, to reduce CPU usage
    printf("> Using CPU/GPU Device Synchronization method (%s)\n", sDeviceSyncMethod[device_sync_method]);
    hipSetDeviceFlags( device_sync_method | (bPinGenericMemory ? hipDeviceMapHost : 0 ) );

    // allocate host memory
    int c = 5;                      // value to which the array will be initialized
    int *h_a = 0;                   // pointer to the array data in host memory
    int *hAligned_a = 0;           // pointer to the array data in host memory (aligned to MEMORY_ALIGNMENT)

    // Allocate Host memory (could be using hipHostMalloc or VirtualAlloc/mmap if using the new CUDA 4.0 features
    AllocateHostMemory(bPinGenericMemory, &h_a, &hAligned_a, nbytes);

    // allocate device memory
    int *d_a = 0, *d_c = 0;             // pointers to data and init value in the device memory
    checkCudaErrors( hipMalloc((void**)&d_a, nbytes) );
    checkCudaErrors( hipMalloc((void**)&d_c, sizeof(int)) );
    checkCudaErrors( hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice) );

    printf("\nStarting Test\n");

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++) {
        checkCudaErrors( hipStreamCreate(&(streams[i])) );
    }

    // create CUDA event handles
    // use blocking sync
    hipEvent_t start_event, stop_event;
    int eventflags = ( (device_sync_method == hipDeviceScheduleBlockingSync) ? hipEventBlockingSync: hipEventDefault );

    checkCudaErrors( hipEventCreateWithFlags(&start_event, eventflags) );
    checkCudaErrors( hipEventCreateWithFlags(&stop_event, eventflags) );

    // time memcopy from device
    hipEventRecord(start_event, 0);     // record in stream-0, to ensure that all previous CUDA calls have completed
    hipMemcpyAsync(hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost, streams[0]);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);   // block until the event is actually recorded
    checkCudaErrors( hipEventElapsedTime(&time_memcpy, start_event, stop_event) );
    printf("memcopy:\t%.2f\n", time_memcpy);
    
    // time kernel
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    checkCudaErrors( hipEventElapsedTime(&time_kernel, start_event, stop_event) );
    printf("kernel:\t\t%.2f\n", time_kernel);

    //////////////////////////////////////////////////////////////////////
    // time non-streamed execution for reference
    threads=dim3(512, 1);
    blocks=dim3(n / threads.x, 1);
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        init_array<<<blocks, threads>>>(d_a, d_c, niterations);
        hipMemcpy(hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    checkCudaErrors( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("non-streamed:\t%.2f (%.2f expected)\n", elapsed_time / nreps, time_kernel + time_memcpy);

    //////////////////////////////////////////////////////////////////////
    // time execution with nstreams streams
    threads=dim3(512,1);
    blocks=dim3(n/(nstreams*threads.x),1);
    memset(hAligned_a, 255, nbytes);     // set host memory bits to all 1s, for testing correctness
    hipMemset(d_a, 0, nbytes); // set device memory to all 0s, for testing correctness
    hipEventRecord(start_event, 0);
    for(int k = 0; k < nreps; k++)
    {
        // asynchronously launch nstreams kernels, each operating on its own portion of data
        for(int i = 0; i < nstreams; i++)
            init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams, d_c, niterations);

        // asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
        //   commence executing when all previous CUDA calls in stream x have completed
        for(int i = 0; i < nstreams; i++)
            hipMemcpyAsync(hAligned_a + i * n / nstreams, d_a + i * n / nstreams, nbytes / nstreams, hipMemcpyDeviceToHost, streams[i]);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    checkCudaErrors( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    printf("%d streams:\t%.2f (%.2f expected with compute capability 1.1 or later)\n", nstreams, elapsed_time / nreps, time_kernel + time_memcpy / nstreams);

    // check whether the output is correct
    printf("-------------------------------\n");
    bool bResults = correct_data(hAligned_a, n, c*nreps*niterations);

    // release resources
    for(int i = 0; i < nstreams; i++) {
        hipStreamDestroy(streams[i]);
    }
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // Free hipHostMalloc or Generic Host allocated memory (from CUDA 4.0)
    FreeHostMemory( bPinGenericMemory, &h_a, &hAligned_a, nbytes );

    hipFree(d_a);
    hipFree(d_c);

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, bResults ? QA_PASSED : QA_FAILED);
}
