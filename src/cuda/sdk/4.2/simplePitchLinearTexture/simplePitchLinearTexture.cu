#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
 
/* pitchLinearTexture 
 *
 * This example demostrates how to use textures bound to pitch linear memory.  
 * It performs a shift of matrix elements using wrap addressing mode (aka
 * periodic boundary conditions) on two arrays, a pitch linear and a CUDA array,
 * in order to highlight the differences in using each. 
 *
 * Textures binding to pitch linear memory is a new feature in CUDA 2.2,
 * and allows use of texture features such as wrap addressing mode and 
 * filtering which are not possible with textures bound to regular linear memory 
 */

// includes, system
#include <stdio.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

// includes CUDA
#include <cutil.h>
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions


#define NUM_REPS 100  // number of repetitions performed  
#define TILE_DIM 16   // tile/block size

// Texture references

texture<float, 2, hipReadModeElementType> texRefPL;
texture<float, 2, hipReadModeElementType> texRefArray;

// -------
// kernels
// -------
//
// NB: (1) The second argument "pitch" is in elements, not bytes
//     (2) normalized coordinates are used (required for wrap address mode)

__global__ void shiftPitchLinear(float* odata, int pitch, int width, int height, 
				 int shiftX, int shiftY)
{
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;
  
  odata[yid*pitch+xid] = tex2D(texRefPL, 
			       (xid + shiftX)/(float)width, 
			       (yid + shiftY)/(float)height);
}

__global__ void shiftArray(float* odata, int pitch, int width, int height,
			   int shiftX, int shiftY)
{
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;
  
  odata[yid*pitch+xid] = tex2D(texRefArray, 
			       (xid + shiftX)/(float)width, 
			       (yid + shiftY)/(float)height);
}

// ----
// main
// ----

int
main( int argc, char** argv) 
{
  // set array size
  const int nx = 2048;  
  const int ny = 2048;

  // shifts applied to x and y data
  const int x_shift = 5;
  const int y_shift = 7;

  shrQAStart(argc, argv);

  if ((nx%TILE_DIM != 0)  || (ny%TILE_DIM != 0)) {
    printf("nx and ny must be multiples of TILE_DIM\n");
    shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
  }

  // execution configuration parameters
  dim3 grid(nx/TILE_DIM, ny/TILE_DIM), threads(TILE_DIM, TILE_DIM);

  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice((const int)argc, (const char **)argv);

  // CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ----------------------------------
  // Host allocation and initialization
  // ----------------------------------

  float *h_idata = (float*) malloc(sizeof(float)*nx*ny);
  float *h_odata = (float*) malloc(sizeof(float)*nx*ny);
  float *gold = (float*) malloc(sizeof(float)*nx*ny);

  for(int i = 0; i < nx*ny; ++i) h_idata[i] = (float) i;  

  // ------------------------
  // Device memory allocation
  // ------------------------

  // Pitch linear input data

  float *d_idataPL;
  size_t d_pitchBytes;

  checkCudaErrors(hipMallocPitch((void**) &d_idataPL, &d_pitchBytes, nx*sizeof(float), ny));

  // Array input data

  hipArray *d_idataArray;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  checkCudaErrors(hipMallocArray(&d_idataArray, &channelDesc, nx, ny));

  // Pitch linear output data

  float *d_odata;
  checkCudaErrors(hipMallocPitch( (void**) &d_odata, &d_pitchBytes, nx*sizeof(float), ny));

  // ------------------------
  // copy host data to device
  // ------------------------

  // Pitch linear

  size_t h_pitchBytes = nx*sizeof(float);

  checkCudaErrors(hipMemcpy2D(d_idataPL, d_pitchBytes, h_idata, h_pitchBytes, 
			     nx*sizeof(float), ny, hipMemcpyHostToDevice));

  // Array

  checkCudaErrors(hipMemcpyToArray(d_idataArray, 0, 0, h_idata, 
				  nx*ny*sizeof(float), hipMemcpyHostToDevice));

  // ----------------------
  // Bind texture to memory
  // ----------------------

  // Pitch linear

  texRefPL.normalized = 1;
  texRefPL.filterMode = hipFilterModePoint;
  texRefPL.addressMode[0] = hipAddressModeWrap;
  texRefPL.addressMode[1] = hipAddressModeWrap;
  
  checkCudaErrors(hipBindTexture2D(0, &texRefPL, d_idataPL, &channelDesc, nx, ny, d_pitchBytes));
  
  // Array

  texRefArray.normalized = 1;
  texRefArray.filterMode = hipFilterModePoint;
  texRefArray.addressMode[0] = hipAddressModeWrap;
  texRefArray.addressMode[1] = hipAddressModeWrap;

  checkCudaErrors(hipBindTextureToArray(texRefArray, d_idataArray, channelDesc));

  // ---------------------
  // reference calculation
  // ---------------------

  for (int j = 0; j < ny; j++) {
    int jshift = (j+y_shift)%ny;
    for (int i = 0; i < nx; i++) {
      int ishift = (i+x_shift)%nx;
      gold[j*nx + i] = h_idata[jshift*nx + ishift];
    }
  }

  // ----------------
  // shiftPitchLinear
  // ----------------

  checkCudaErrors(hipMemset2D(d_odata, d_pitchBytes, 0, nx*sizeof(float), ny));
  checkCudaErrors(hipEventRecord(start, 0));
  for (int i=0; i < NUM_REPS; i++) {
    shiftPitchLinear<<<grid, threads>>>(d_odata, (int)(d_pitchBytes/sizeof(float)), nx, ny, 
					x_shift, y_shift);
  }
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float timePL;
  checkCudaErrors(hipEventElapsedTime(&timePL, start, stop));    

  // check results

  checkCudaErrors(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes, 
			     nx*sizeof(float), ny, hipMemcpyDeviceToHost));

  bool res = compareData(gold, h_odata, nx*ny, 0.0f, 0.15f);

  bool success = true;
  if (res == false) {
    printf("*** shiftPitchLinear failed ***\n");
    success = false;
  }
    
  // ----------
  // shiftArray
  // ----------

  checkCudaErrors(hipMemset2D(d_odata, d_pitchBytes, 0, nx*sizeof(float), ny));
  checkCudaErrors(hipEventRecord(start, 0));
  for (int i=0; i < NUM_REPS; i++) {
    shiftArray<<<grid, threads>>>(d_odata, (int)(d_pitchBytes/sizeof(float)), nx, ny, x_shift, y_shift);
  }
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float timeArray;
  checkCudaErrors(hipEventElapsedTime(&timeArray, start, stop));    

  // check results

  checkCudaErrors(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes, 
		     nx*sizeof(float), ny, hipMemcpyDeviceToHost));
  res = compareData(gold, h_odata, nx*ny, 0.0f, 0.15f);

  if (res == CUTFalse) {
    printf("*** shiftArray failed ***\n");
    success = CUTFalse;
  }
    
  float bandwidthPL = 2.f*1000.f*nx*ny*sizeof(float)/(1.e+9f)/(timePL/NUM_REPS);
  float bandwidthArray = 2.f*1000.f*nx*ny*sizeof(float)/(1.e+9f)/(timeArray/NUM_REPS);
  printf("\nBandwidth (GB/s) for pitch linear: %.2e; for array: %.2e\n", 
	 bandwidthPL, bandwidthArray);

  float fetchRatePL = nx*ny/1.e+6f/(timePL/(1000.0f*NUM_REPS));
  float fetchRateArray = nx*ny/1.e+6f/(timeArray/(1000.0f*NUM_REPS));
  printf("\nTexture fetch rate (Mpix/s) for pitch linear: %.2e; for array: %.2e\n\n", 
	 fetchRatePL, fetchRateArray);


  // cleanup
  free(h_idata);
  free(h_odata);
  free(gold);

  checkCudaErrors(hipUnbindTexture(texRefPL));
  checkCudaErrors(hipUnbindTexture(texRefArray));
  checkCudaErrors(hipFree(d_idataPL));
  checkCudaErrors(hipFreeArray(d_idataArray));
  checkCudaErrors(hipFree(d_odata));

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  
  hipDeviceReset();
  shrQAFinishExit(argc, (const char **)argv, (success == CUTTrue) ? QA_PASSED : QA_FAILED);
}
