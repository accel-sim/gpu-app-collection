#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil_inline.h>
#include "histogram_common.h"

////////////////////////////////////////////////////////////////////////////////
// Shortcut shared memory atomic addition functions
////////////////////////////////////////////////////////////////////////////////
#define USE_SMEM_ATOMICS 0

#if(!USE_SMEM_ATOMICS)
    #define TAG_MASK ( (1U << (UINT_BITS - LOG2_WARP_SIZE)) - 1U )

    inline __device__ void addByte(volatile uint *s_WarpHist, uint data, uint threadTag){
        uint count;
        do{
            count = s_WarpHist[data] & TAG_MASK;
            count = threadTag | (count + 1);
            s_WarpHist[data] = count;
        }while(s_WarpHist[data] != count);
    }
#else
    #ifdef CUDA_NO_SM12_ATOMIC_INTRINSICS
        #error Compilation target does not support shared-memory atomics
    #endif

    #define TAG_MASK 0xFFFFFFFFU
    inline __device__ void addByte(uint *s_WarpHist, uint data, uint threadTag){
        atomicAdd(s_WarpHist + data, 1);
    }
#endif

inline __device__ void addWord(uint *s_WarpHist, uint data, uint tag){
    addByte(s_WarpHist, (data >>  0) & 0xFFU, tag);
    addByte(s_WarpHist, (data >>  8) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 16) & 0xFFU, tag);
    addByte(s_WarpHist, (data >> 24) & 0xFFU, tag);
}

__global__ void histogram256Kernel(uint *d_PartialHistograms, uint *d_Data, uint dataCount){
    //Per-warp subhistogram storage
    __shared__ uint s_Hist[HISTOGRAM256_THREADBLOCK_MEMORY];
    uint *s_WarpHist= s_Hist + (threadIdx.x >> LOG2_WARP_SIZE) * HISTOGRAM256_BIN_COUNT;

    //Clear shared memory storage for current threadblock before processing
    #pragma unroll
    for(uint i = 0; i < (HISTOGRAM256_THREADBLOCK_MEMORY / HISTOGRAM256_THREADBLOCK_SIZE); i++)
       s_Hist[threadIdx.x + i * HISTOGRAM256_THREADBLOCK_SIZE] = 0;

    //Cycle through the entire data set, update subhistograms for each warp
    const uint tag = threadIdx.x << (UINT_BITS - LOG2_WARP_SIZE);

    __syncthreads();
    for(uint pos = UMAD(blockIdx.x, blockDim.x, threadIdx.x); pos < dataCount; pos += UMUL(blockDim.x, gridDim.x)){
        uint data = d_Data[pos];
        addWord(s_WarpHist, data, tag);
    }

    //Merge per-warp histograms into per-block and write to global memory
    __syncthreads();
    for(uint bin = threadIdx.x; bin < HISTOGRAM256_BIN_COUNT; bin += HISTOGRAM256_THREADBLOCK_SIZE){
        uint sum = 0;

        for(uint i = 0; i < WARP_COUNT; i++)
            sum += s_Hist[bin + i * HISTOGRAM256_BIN_COUNT] & TAG_MASK;

        d_PartialHistograms[blockIdx.x * HISTOGRAM256_BIN_COUNT + bin] = sum;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Merge histogram256() output
// Run one threadblock per bin; each threadblock adds up the same bin counter
// from every partial histogram. Reads are uncoalesced, but mergeHistogram256
// takes only a fraction of total processing time
////////////////////////////////////////////////////////////////////////////////
#define MERGE_THREADBLOCK_SIZE 256

__global__ void mergeHistogram256Kernel(
    uint *d_Histogram,
    uint *d_PartialHistograms,
    uint histogramCount
){
    uint sum = 0;
    for(uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];

    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    for(uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1){
        __syncthreads();
        if(threadIdx.x < stride)
            data[threadIdx.x] += data[threadIdx.x + stride];
    }

    if(threadIdx.x == 0)
        d_Histogram[blockIdx.x] = data[0];
}

////////////////////////////////////////////////////////////////////////////////
// Host interface to GPU histogram
////////////////////////////////////////////////////////////////////////////////
//histogram256kernel() intermediate results buffer
static const uint PARTIAL_HISTOGRAM256_COUNT = 240;
static uint *d_PartialHistograms;

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions
    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
	        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }
// end of CUDA Helper Functions

//Internal memory allocation
extern "C" void initHistogram256(void){
    checkCudaErrors( hipMalloc((void **)&d_PartialHistograms, PARTIAL_HISTOGRAM256_COUNT * HISTOGRAM256_BIN_COUNT * sizeof(uint)) );
}

//Internal memory deallocation
extern "C" void closeHistogram256(void){
    checkCudaErrors( hipFree(d_PartialHistograms) );
}

extern "C" void histogram256(
    uint *d_Histogram,
    void *d_Data,
    uint byteCount
){
    assert( byteCount % sizeof(uint) == 0 );
    histogram256Kernel<<<PARTIAL_HISTOGRAM256_COUNT, HISTOGRAM256_THREADBLOCK_SIZE>>>(
        d_PartialHistograms,
        (uint *)d_Data,
        byteCount / sizeof(uint)
    );
    getLastCudaError("histogram256Kernel() execution failed\n");

    mergeHistogram256Kernel<<<HISTOGRAM256_BIN_COUNT, MERGE_THREADBLOCK_SIZE>>>(
        d_Histogram,
        d_PartialHistograms,
        PARTIAL_HISTOGRAM256_COUNT
    );
    getLastCudaError("mergeHistogram256Kernel() execution failed\n");
}
