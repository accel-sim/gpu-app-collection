#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// includes, cuda
#include <hip/hip_runtime_api.h>
#include <cutil_math.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include <bicubicTexture_kernel.cuh>

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
	    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }
// end of CUDA Helper Functions

hipArray *d_imageArray = 0;

extern "C"
void initTexture(int imageWidth, int imageHeight, uchar *h_data)
{
    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
    checkCudaErrors( hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight) ); 
    uint size = imageWidth * imageHeight * sizeof(uchar);
    checkCudaErrors( hipMemcpyToArray(d_imageArray, 0, 0, h_data, size, hipMemcpyHostToDevice) );
    free(h_data);

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = false;    // access with integer texture coordinates

    getLastCudaError("initTexture");

    // Bind the array to the texture
    checkCudaErrors( hipBindTextureToArray(tex, d_imageArray) );

    // bind same array to 2nd texture reference with point sampling
    tex2.addressMode[0] = hipAddressModeClamp;
    tex2.addressMode[1] = hipAddressModeClamp;
    tex2.filterMode = hipFilterModePoint;
    tex2.normalized = false;    // access with integer texture coordinates

    checkCudaErrors( hipBindTextureToArray(tex2, d_imageArray) );
}

extern "C"
void freeTexture()
{
    checkCudaErrors(hipFreeArray(d_imageArray));
}


// render image using CUDA
extern "C" 
void render(int width, int height, float tx, float ty, float scale, float cx, float cy, 
            dim3 blockSize, dim3 gridSize, int mode, uchar4 *output)
{
    // call CUDA kernel, writing results to PBO memory
    switch(mode) {
    case MODE_NEAREST:
        tex.filterMode = hipFilterModePoint;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BILINEAR:
        tex.filterMode = hipFilterModeLinear;
        d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_BICUBIC:
        tex.filterMode = hipFilterModePoint;
        d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_FAST_BICUBIC:
        tex.filterMode = hipFilterModeLinear;
        d_renderFastBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    case MODE_CATROM:
        tex.filterMode = hipFilterModePoint;
        d_renderCatRom<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale, cx, cy);
        break;
    }
    getLastCudaError("kernel failed");
}

#endif
