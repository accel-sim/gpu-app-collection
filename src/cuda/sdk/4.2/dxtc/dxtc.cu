#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes
#include <shrUtils.h>
#include <shrQATest.h>
#include <cutil_inline.h>
#include <cutil_math.h>
#include <float.h> // for FLT_MAX

#include "CudaMath.h"
#include "dds.h"
#include "permutations.h"

// Definitions
#define INPUT_IMAGE "lena_std.ppm"
#define REFERENCE_IMAGE "lena_ref.dds"

#define ERROR_THRESHOLD 0.02f

#define NUM_THREADS 64        // Number of threads per block.

#define __debugsync()

template <class T> 
__device__ inline void swap(T & a, T & b)
{
    T tmp = a;
    a = b;
    b = tmp;
}

//__constant__ float3 kColorMetric = { 0.2126f, 0.7152f, 0.0722f };
__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };

////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(const float * values, int * ranks)
{
    const int tid = threadIdx.x;

    int rank = 0;

    #pragma unroll
    for (int i = 0; i < 16; i++)
    {
        rank += (values[i] < values[tid]);
    }
    
    ranks[tid] = rank;

    // Resolve elements with the same index.
    #pragma unroll
    for (int i = 0; i < 15; i++)
    {
        if (tid > i && ranks[tid] == ranks[i]) ++ranks[tid];
    }
}

////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint * image, float3 colors[16], float3 sums[16], int xrefs[16], int blockOffset)
{
    const int bid = blockIdx.x + blockOffset;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    float3 tmp;

    if (idx < 16)
    {
        // Read color and copy to shared mem.
        uint c = image[(bid) * 16 + idx];
    
        colors[idx].x = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
        colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
        colors[idx].z = ((c >> 16) & 0xFF) * (1.0f / 255.0f);

        // Sort colors along the best fit line.
        colorSums(colors, sums);
        float3 axis = bestFitLine(colors, sums[0]);
        
        dps[idx] = dot(colors[idx], axis);
        
        sortColors(dps, xrefs);
        
        tmp = colors[idx];

        colors[xrefs[idx]] = tmp;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand(float3 v, ushort * w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);

    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f; // approximate integer bit expansion.
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}


__constant__ float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const int prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const int prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

#define USE_TABLES 1

////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
static __device__ float evalPermutation4(const float3 * colors, uint permutation, ushort * start, ushort * end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = (9.0f * color_sum) - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = (1 + beta) * (1.0f / 3.0f);
        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    // alpha2, beta2, alphabeta and factor could be precomputed for each permutation, but it's faster to recompute them.
    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.111111111111f) * dot(e, kColorMetric);
}

static __device__ float evalPermutation3(const float3 * colors, uint permutation, ushort * start, ushort * end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = (4.0f * color_sum) - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);
        if (bits & 2) beta = 0.5f;
        float alpha = 1.0f - beta;
    
        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;
    
    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.25f) * dot(e, kColorMetric);
}

__device__ void evalAllPermutations(const float3 * colors, const uint * permutations, ushort & bestStart, ushort & bestEnd, uint & bestPermutation, float * errors, float3 color_sum)
{
    const int idx = threadIdx.x;
    
    float bestError = FLT_MAX;
    
    __shared__ uint s_permutations[160];

    for(int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 992) break;
        
        ushort start, end;
        uint permutation = permutations[pidx];
        if (pidx < 160) s_permutations[pidx] = permutation;
                
        float error = evalPermutation4(colors, permutation, &start, &end, color_sum);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;    // Flip indices.
    }

    for(int i = 0; i < 3; i++)
    {
        int pidx = idx + NUM_THREADS * i;
        if (pidx >= 160) break;
        
        ushort start, end;
        uint permutation = s_permutations[pidx];
        float error = evalPermutation3(colors, permutation, &start, &end, color_sum);
        
        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
            
            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
    }

    errors[idx] = bestError;
}

////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float * errors)
{
    const int idx = threadIdx.x;

    __shared__ int indices[NUM_THREADS];
    indices[idx] = idx;

    for(int d = NUM_THREADS/2; d > 32; d >>= 1)
    {
        __syncthreads();
        
        if (idx < d)
        {
            float err0 = errors[idx];
            float err1 = errors[idx + d];
            
            if (err1 < err0) {
                errors[idx] = err1;
                indices[idx] = indices[idx + d];
            }
        }
    }

    __syncthreads();

    // unroll last 6 iterations
    if (idx < 32)
    {
        if (errors[idx + 32] < errors[idx]) {
            errors[idx] = errors[idx + 32];
            indices[idx] = indices[idx + 32];
        }
        if (errors[idx + 16] < errors[idx]) {
            errors[idx] = errors[idx + 16];
            indices[idx] = indices[idx + 16];
        }
        if (errors[idx + 8] < errors[idx]) {
            errors[idx] = errors[idx + 8];
            indices[idx] = indices[idx + 8];
        }
        if (errors[idx + 4] < errors[idx]) {
            errors[idx] = errors[idx + 4];
            indices[idx] = indices[idx + 4];
        }
        if (errors[idx + 2] < errors[idx]) {
            errors[idx] = errors[idx + 2];
            indices[idx] = indices[idx + 2];
        }
        if (errors[idx + 1] < errors[idx]) {
            errors[idx] = errors[idx + 1];
            indices[idx] = indices[idx + 1];
        }
    }

    __syncthreads();

    return indices[0];
}

////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 * result, int blockOffset)
{
    const int bid = blockIdx.x + blockOffset;

    if (start == end)
    {
        permutation = 0;
    }
    
    // Reorder permutation.
    uint indices = 0;
    for(int i = 0; i < 16; i++)
    {
        int ref = xrefs[i];
        indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
    }
    
    // Write endpoints.
    result[bid].x = (end << 16) | start;
    
    // Write palette indices.
    result[bid].y = indices;
}

////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compress(const uint * permutations, const uint * image, uint2 * result, int blockOffset)
{
    const int idx = threadIdx.x;
    
    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int xrefs[16];
    
    loadColorBlock(image, colors, sums, xrefs, blockOffset);

    __syncthreads();

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutations(colors, permutations, bestStart, bestEnd, bestPermutation, errors, sums[0]);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors);
    
    __syncthreads();
    
    // Only write the result of the winner thread.
    if (idx == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result, blockOffset);
    }
}

// Helper structs and functions to validate the output of the compressor.
// We cannot simply do a bitwise compare, because different compilers produce different
// results for different targets due to floating point arithmetic.

union Color32 {
    struct {
        unsigned char b, g, r, a;
    };
    unsigned int u;
};

union Color16 {
    struct {
        unsigned short b : 5;
        unsigned short g : 6;
        unsigned short r : 5;
    };
    unsigned short u;
};

struct BlockDXT1
{
    Color16 col0;
    Color16 col1;
    union {
        unsigned char row[4];
        unsigned int indices;
    };
    
    void decompress(Color32 colors[16]) const;
};

void BlockDXT1::decompress(Color32 * colors) const
{
    Color32 palette[4];
    
    // Does bit expansion before interpolation.
    palette[0].b = (col0.b << 3) | (col0.b >> 2);
    palette[0].g = (col0.g << 2) | (col0.g >> 4);
    palette[0].r = (col0.r << 3) | (col0.r >> 2);
    palette[0].a = 0xFF;
    
    palette[1].r = (col1.r << 3) | (col1.r >> 2);
    palette[1].g = (col1.g << 2) | (col1.g >> 4);
    palette[1].b = (col1.b << 3) | (col1.b >> 2);
    palette[1].a = 0xFF;
    
    if( col0.u > col1.u ) {
        // Four-color block: derive the other two colors.
        palette[2].r = (2 * palette[0].r + palette[1].r) / 3;
        palette[2].g = (2 * palette[0].g + palette[1].g) / 3;
        palette[2].b = (2 * palette[0].b + palette[1].b) / 3;
        palette[2].a = 0xFF;
        
        palette[3].r = (2 * palette[1].r + palette[0].r) / 3;
        palette[3].g = (2 * palette[1].g + palette[0].g) / 3;
        palette[3].b = (2 * palette[1].b + palette[0].b) / 3;
        palette[3].a = 0xFF;
    }
    else {
        // Three-color block: derive the other color.
        palette[2].r = (palette[0].r + palette[1].r) / 2;
        palette[2].g = (palette[0].g + palette[1].g) / 2;
        palette[2].b = (palette[0].b + palette[1].b) / 2;
        palette[2].a = 0xFF;

        palette[3].r = 0x00;
        palette[3].g = 0x00;
        palette[3].b = 0x00;
        palette[3].a = 0x00;
    }

    for (int i = 0; i < 16; i++)
    {
        colors[i] = palette[(indices >> (2*i)) & 0x3];
    }
}

static int compareColors(const Color32 * b0, const Color32 * b1)
{
    int sum = 0;
    for (int i = 0; i < 16; i++)
    {
        int r = (b0[i].r - b1[i].r);
        int g = (b0[i].g - b1[i].g);
        int b = (b0[i].b - b1[i].b);
        sum += r*r + g*g + b*b;
    }
    return sum;
}

static int compareBlock(const BlockDXT1 * b0, const BlockDXT1 * b1)
{
    Color32 colors0[16];
    Color32 colors1[16];

    if (memcmp(b0, b1, sizeof(BlockDXT1)) == 0)
    {
        return 0;
    }
    else
    {
        b0->decompress(colors0);
        b1->decompress(colors1);
    
        return compareColors(colors0, colors1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) 
{
    shrQAStart(argc, argv);
    shrSetLogFileName ("dxtc.txt");
    shrLog("[%s] Starting...\n\n", argv[0]); 
    
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    // Load input image.
    unsigned char* data = NULL;
    uint W, H;

    char* image_path = shrFindFilePath(INPUT_IMAGE, argv[0]);
    if (image_path == 0) 
    {
        shrLog("Error, unable to find source image  <%s>\n", image_path);
        cutilDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }

    if (!shrLoadPPM4ub(image_path, &data, &W, &H)) 
    {
        shrLog("Error, unable to open source image file <%s>\n", image_path);
        cutilDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }

    uint w = W/32, h = H/32;

    shrLog("Image Loaded '%s', %d x %d pixels\n\n", image_path, w, h);

    // Allocate input image.
    const uint memSize = w * h * 4;
    cutilCondition( 0 != memSize );
    uint* block_image = (uint*)malloc(memSize);

    // Convert linear image to block linear. 
    for(uint by = 0; by < h/4; by++) {
        for(uint bx = 0; bx < w/4; bx++) {
            for (int i = 0; i < 16; i++) {
                const int x = i & 3;
                const int y = i / 4;
                block_image[(by * w/4 + bx) * 16 + i] = 
                    ((uint *)data)[(by * 4 + y) * 4 * (W/4) + bx * 4 + x];
            }
        }
    }

    // copy into global mem
    uint * d_data = NULL;
    cutilSafeCall( hipMalloc((void**) &d_data, memSize) );

    // Result
    uint* d_result = NULL;
    const uint compressedSize = (w / 4) * (h / 4) * 8;
    cutilSafeCall(hipMalloc((void**)&d_result, compressedSize) );
    uint* h_result = (uint *)malloc(compressedSize);

    // Compute permutations.
    uint permutations[1024];
    computePermutations(permutations);

    // Copy permutations host to devie.
    uint * d_permutations = NULL;    
    cutilSafeCall(hipMalloc((void**) &d_permutations, 1024 * sizeof(uint)) );
    cutilSafeCall(hipMemcpy(d_permutations, permutations, 1024 * sizeof(uint), 
                   hipMemcpyHostToDevice) );

    // create a timer
    uint timer;
    cutilCheckError(cutCreateTimer(&timer));
    
    // Copy image from host to device
    cutilSafeCall(hipMemcpy(d_data, block_image, memSize, hipMemcpyHostToDevice) );

    // Determine launch configuration and run timed computation numIterations times
	uint blocks = ((w + 3) / 4) * ((h + 3) / 4); // rounds up by 1 block in each dim if %4 != 0

	int devID;
	hipDeviceProp_t deviceProp;

	// get number of SMs on this GPU
	cutilSafeCall(hipGetDevice(&devID));
	cutilSafeCall(hipGetDeviceProperties(&deviceProp, devID));

	// Restrict the numbers of blocks to launch on low end GPUs to avoid kernel timeout
	int blocksPerLaunch = min(blocks, 768 * deviceProp.multiProcessorCount);

    shrLog("Running DXT Compression on %u x %u image...\n", w, h);
    shrLog("\n%u Blocks, %u Threads per Block, %u Threads in Grid...\n\n", 
           blocks, NUM_THREADS, blocks * NUM_THREADS);
    int numIterations = 1;
    for (int i = -1; i < numIterations; ++i) {
        if( i == 0 ) { 
            cutilSafeCall(cutilDeviceSynchronize()); 
            cutilCheckError(cutStartTimer(timer));   
        }
		
        for( int j=0; j<(int)blocks; j+=blocksPerLaunch ) {
           compress<<<min(blocksPerLaunch, blocks-j), NUM_THREADS>>>(d_permutations, d_data, (uint2 *)d_result, j);
        }
    }
    cutilCheckMsg("compress");

    // sync to host, stop timer, record perf
    cutilSafeCall(cutilDeviceSynchronize());
    cutilCheckError(cutStopTimer(timer));
    double dAvgTime = 1.0e-3 * cutGetTimerValue(timer)/(double)numIterations;
    shrLogEx(LOGBOTH | MASTER, 0, "dxtc, Throughput = %.4f MPixels/s, Time = %.5f s, Size = %u Pixels, NumDevsUsed = %i, Workgroup = %d\n", 
           (1.0e-6 * (double)(W * H)/ dAvgTime), dAvgTime, (W * H), 1, NUM_THREADS); 

    // copy result data from device to host
    cutilSafeCall(hipMemcpy(h_result, d_result, compressedSize, hipMemcpyDeviceToHost));

    // Write out result data to DDS file
    char output_filename[1024];
    strcpy(output_filename, image_path);
    strcpy(output_filename + strlen(image_path) - 3, "dds");
    FILE* fp = fopen(output_filename, "wb");
    if (fp == 0) {
        shrLog("Error, unable to open output image <%s>\n", output_filename);
        cutilDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    DDSHeader header;
    header.fourcc = FOURCC_DDS;
    header.size = 124;
    header.flags  = (DDSD_WIDTH|DDSD_HEIGHT|DDSD_CAPS|DDSD_PIXELFORMAT|DDSD_LINEARSIZE);
    header.height = h;
    header.width = w;
    header.pitch = compressedSize;
    header.depth = 0;
    header.mipmapcount = 0;
    memset(header.reserved, 0, sizeof(header.reserved));
    header.pf.size = 32;
    header.pf.flags = DDPF_FOURCC;
    header.pf.fourcc = FOURCC_DXT1;
    header.pf.bitcount = 0;
    header.pf.rmask = 0;
    header.pf.gmask = 0;
    header.pf.bmask = 0;
    header.pf.amask = 0;
    header.caps.caps1 = DDSCAPS_TEXTURE;
    header.caps.caps2 = 0;
    header.caps.caps3 = 0;
    header.caps.caps4 = 0;
    header.notused = 0;
    fwrite(&header, sizeof(DDSHeader), 1, fp);
    fwrite(h_result, compressedSize, 1, fp);
    fclose(fp);

    // Make sure the generated image is correct.
    const char* reference_image_path = shrFindFilePath(REFERENCE_IMAGE, argv[0]);
    if (reference_image_path == 0) {
        shrLog("Error, unable to find reference image\n");
        cutilDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    fp = fopen(reference_image_path, "rb");
    if (fp == 0) {
        shrLog("Error, unable to open reference image\n");
        cutilDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    fseek(fp, sizeof(DDSHeader), SEEK_SET);
    uint referenceSize = (W / 4) * (H / 4) * 8;
    uint* reference = (uint *)malloc(referenceSize);
    fread(reference, referenceSize, 1, fp);
    fclose(fp);

    shrLog("\nChecking accuracy...\n");
    float rms = 0;
    for (uint y = 0; y < h; y += 4)
    {
        for (uint x = 0; x < w; x += 4)
        {
            uint referenceBlockIdx = ((y/4) * (W/4) + (x/4));
            uint resultBlockIdx = ((y/4) * (w/4) + (x/4));

            int cmp = compareBlock(((BlockDXT1 *)h_result) + resultBlockIdx, ((BlockDXT1 *)reference) + referenceBlockIdx);
            if (cmp != 0.0f) {
                shrLog("Deviation at (%4d,%4d):\t%f rms\n", x/4, y/4, float(cmp)/16/3);
            }
            rms += cmp;
        }
    }
    rms /= w * h * 3;

    // Free allocated resources and exit
    cutilSafeCall(hipFree(d_permutations));
    cutilSafeCall(hipFree(d_data));
    cutilSafeCall(hipFree(d_result));
    shrFree(image_path);
    free(data);
    free(block_image);
    free(h_result);
    free(reference);
    cutilCheckError(cutDeleteTimer(timer));
    cutilDeviceReset();

    shrLog("RMS(reference, result) = %f\n\n", rms);
    shrQAFinishExit(argc, (const char **)argv, (rms <= ERROR_THRESHOLD) ? QA_PASSED : QA_FAILED );
}
