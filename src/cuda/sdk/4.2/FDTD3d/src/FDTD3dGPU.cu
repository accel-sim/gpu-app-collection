#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "FDTD3dGPU.h"

#include <iostream>
#include <algorithm>
#include <cutil_inline.h>
#include <shrUtils.h>
#include "FDTD3dGPUKernel.cuh"

bool getTargetDeviceGlobalMemSize(memsize_t *result, const int argc, const char **argv)
{
    bool ok = true;
    int               deviceCount  = 0;
    int               targetDevice = 0;
    size_t            memsize      = 0;
    hipError_t         errnum       = hipSuccess;

    // Get the number of CUDA enabled GPU devices
    if (ok)
    {
        shrLog(" hipGetDeviceCount\n");
        errnum = hipGetDeviceCount(&deviceCount);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipGetDeviceCount '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Select target device (device 0 by default)
    if (ok)
    {
        char *device = 0;
        if (shrGetCmdLineArgumentstr(argc, argv, "device", &device))
        {
            targetDevice = (unsigned int)atoi(device);
            if (targetDevice >= deviceCount)
            {
                shrLogEx(LOGBOTH | ERRORMSG, 0, "invalid target device specified on command line (device %d does not exist).\n", targetDevice);
                ok = false;
            }
        }
        else
        {
            targetDevice = cutGetMaxGflopsDeviceId();
        }
        if (device)
            free(device);
    }

    // Query target device for maximum memory allocation
    if (ok)
    {
        shrLog(" hipGetDeviceProperties\n"); 
        struct hipDeviceProp_t deviceProp;
        errnum = hipGetDeviceProperties(&deviceProp, targetDevice);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipGetDeviceProperties '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
        memsize = deviceProp.totalGlobalMem;
    }

    // Save the result
    if (ok)
    {
        *result = (memsize_t)memsize;
    }

    return ok;
}

bool fdtdGPU(float *output, const float *input, const float *coeff, const int dimx, const int dimy, const int dimz, const int radius, const int timesteps, const int argc, const char **argv)
{
    bool ok = true;
    const int         outerDimx  = dimx + 2 * radius;
    const int         outerDimy  = dimy + 2 * radius;
    const int         outerDimz  = dimz + 2 * radius;
    const size_t      volumeSize = outerDimx * outerDimy * outerDimz;
    int               deviceCount  = 0;
    int               targetDevice = 0;
    float            *bufferOut    = 0;
    float            *bufferIn     = 0;
    dim3              dimBlock;
    dim3              dimGrid;
    hipError_t errnum = hipSuccess;

    // Ensure that the inner data starts on a 128B boundary
    const int padding = (128 / sizeof(float)) - radius;
    const size_t paddedVolumeSize = volumeSize + padding;

#ifdef GPU_PROFILING
    hipEvent_t profileStart = 0;
    hipEvent_t profileEnd   = 0;
    const int profileTimesteps = timesteps - 1;
    if (ok)
    {
        if (profileTimesteps < 1)
        {
            shrLog(" cannot profile with fewer than two timesteps (timesteps=%d), profiling is disabled.\n", timesteps);
        }
    }
#endif

    // Check the radius is valid
    if (ok)
    {
        if (radius != RADIUS)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "radius is invalid, must be %d - see kernel for details.\n", RADIUS);
            ok = false;
        }
    }

    // Get the number of CUDA enabled GPU devices
    if (ok)
    {
        shrLog(" hipGetDeviceCount\n");
        errnum = hipGetDeviceCount(&deviceCount);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipGetDeviceCount '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Select target device (device 0 by default)
    if (ok)
    {
        char *device = 0;
        if (shrGetCmdLineArgumentstr(argc, argv, "device", &device))
        {
            targetDevice = (unsigned int)atoi(device);
            if (targetDevice >= deviceCount)
            {
                shrLogEx(LOGBOTH | ERRORMSG, 0, "invalid target device specified on command line (device %d does not exist).\n", targetDevice);
                ok = false;
            }
        }
        else
        {
            targetDevice = cutGetMaxGflopsDeviceId();
        }
        shrLog(" hipSetDevice (device %d)\n", targetDevice);
        errnum = hipSetDevice(targetDevice);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipSetDevice '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
        if (device)
            free(device);
    }

    // Allocate memory buffers
    if (ok)
    {
        shrLog(" hipMalloc bufferOut\n");
        errnum = hipMalloc((void **)&bufferOut, paddedVolumeSize * sizeof(float));
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipMalloc '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }
    if (ok)
    {
        shrLog(" hipMalloc bufferIn\n");
        errnum = hipMalloc((void **)&bufferIn, paddedVolumeSize * sizeof(float));
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipMalloc '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Check for a command-line specified block size
    int userBlockSize;
    if (ok)
    {
        if (shrGetCmdLineArgumenti(argc, argv, "block-size", &userBlockSize))
        {
            // Constrain to a multiple of k_blockDimX
            userBlockSize = (userBlockSize / k_blockDimX * k_blockDimX);
            // Constrain within allowed bounds
            userBlockSize = CLAMP(userBlockSize, k_blockSizeMin, k_blockSizeMax);
        }
        else
        {
            userBlockSize = k_blockSizeMax;
        }
    }

    // Check the device limit on the number of threads
    if (ok)
    {
        shrLog(" hipFuncGetAttributes\n"); 
        struct hipFuncAttributes funcAttrib;
        errnum = hipFuncGetAttributes(&funcAttrib, reinterpret_cast<const void*>(FiniteDifferencesKernel));
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipFuncGetAttributes '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
        userBlockSize = MIN(userBlockSize, funcAttrib.maxThreadsPerBlock);
    }

    // Set the block size
    if (ok)
    {
        dimBlock.x = k_blockDimX;
        // Visual Studio 2005 does not like std::min
        //    dimBlock.y = std::min<size_t>(userBlockSize / k_blockDimX, (size_t)k_blockDimMaxY);
        dimBlock.y = ((userBlockSize / k_blockDimX) < (size_t)k_blockDimMaxY) ? (userBlockSize / k_blockDimX) : (size_t)k_blockDimMaxY;
        dimGrid.x  = (unsigned int)ceil((float)dimx / dimBlock.x);
        dimGrid.y  = (unsigned int)ceil((float)dimy / dimBlock.y);
        shrLog(" set block size to %dx%d\n", dimBlock.x, dimBlock.y);
        shrLog(" set grid size to %dx%d\n", dimGrid.x, dimGrid.y);
    }

    // Check the block size is valid
    if (ok)
    {
        if (dimBlock.x < RADIUS || dimBlock.y < RADIUS)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "invalid block size, x (%d) and y (%d) must be >= radius (%d).\n", dimBlock.x, dimBlock.y, RADIUS);
            ok = false;
        }
    }
    
    // Copy the input to the device input buffer
    if (ok)
    {
        shrLog(" hipMemcpy (HostToDevice) bufferIn\n");
        errnum = hipMemcpy(bufferIn + padding, input, volumeSize * sizeof(float), hipMemcpyHostToDevice);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipMemcpy '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Copy the input to the device output buffer (actually only need the halo)
    if (ok)
    {
        shrLog(" hipMemcpy (HostToDevice) bufferOut\n");
        errnum = hipMemcpy(bufferOut + padding, input, volumeSize * sizeof(float), hipMemcpyHostToDevice);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipMemcpy '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Copy the coefficients to the device coefficient buffer
    if (ok)
    {
        shrLog(" hipMemcpyToSymbol (HostToDevice) stencil\n");
        errnum = hipMemcpyToSymbol(HIP_SYMBOL(stencil), (void *)coeff, (radius + 1) * sizeof(float));
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipMemcpyToSymbol '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

#ifdef GPU_PROFILING
    // Create the events
    if (ok)
    {
        shrLog(" hipEventCreate\n");
        errnum = hipEventCreate(&profileStart);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipEventCreate '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }
    if (ok)
    {
        shrLog(" hipEventCreate\n");
        errnum = hipEventCreate(&profileEnd);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipEventCreate '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }
#endif
    
    // Execute the FDTD
    float *bufferSrc = bufferIn + padding;
    float *bufferDst = bufferOut + padding;
    shrLog(" GPU FDTD loop\n");
    for (int it = 0 ; ok && it < timesteps ; it++)
    {
        shrLog("\tt = %d ", it);

#ifdef GPU_PROFILING
        // Enqueue start event
        if (ok && it == 1)
        {
            errnum = hipEventRecord(profileStart, 0);
            if (errnum != hipSuccess)
            {
                shrLogEx(LOGBOTH | ERRORMSG, 0, "hipEventRecord '%s'.\n", hipGetErrorString(errnum));
                ok = false;
            }
        }
#endif

        // Launch the kernel
        if (ok)
        {
            shrLog("launch kernel\n");
            FiniteDifferencesKernel<<<dimGrid, dimBlock>>>(bufferDst, bufferSrc, dimx, dimy, dimz);
        }
        // Toggle the buffers
        // Visual Studio 2005 does not like std::swap
        //    std::swap<float *>(bufferSrc, bufferDst);
        float *tmp = bufferDst;
        bufferDst = bufferSrc;
        bufferSrc = tmp;
    }
    shrLog("\n");

#ifdef GPU_PROFILING
    // Enqueue end event
    if (ok)
    {
        errnum = hipEventRecord(profileEnd, 0);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipEventRecord '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }
#endif

    // Wait for the kernel to complete
    if (ok)
    {
        shrLog(" cutilDeviceSynchronize\n");
        errnum = cutilDeviceSynchronize();
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "cutilDeviceSynchronize '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Read the result back, result is in bufferSrc (after final toggle)
    if (ok)
    {
        shrLog(" hipMemcpy (DeviceToHost)\n");
        errnum = hipMemcpy(output, bufferSrc, volumeSize * sizeof(float), hipMemcpyDeviceToHost);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipMemcpy '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    // Report time
#ifdef GPU_PROFILING
    float elapsedTimeMS = 0;
    if (ok && profileTimesteps > 0)
    {
        shrLog(" hipEventElapsedTime\n\n");
        errnum = hipEventElapsedTime(&elapsedTimeMS, profileStart, profileEnd);
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "hipEventElapsedTime '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }
    if (ok && profileTimesteps > 0)
    {
        // Convert milliseconds to seconds
        double elapsedTime    = elapsedTimeMS * 1.0e-3;
        double avgElapsedTime = elapsedTime / (double)profileTimesteps;
        // Determine number of computations per timestep
        size_t pointsComputed = dimx * dimy * dimz;
        // Determine throughput
        double throughputM    = 1.0e-6 * (double)pointsComputed / avgElapsedTime;
        shrLogEx(LOGBOTH | MASTER, 0, "FDTD3d, Throughput = %.4f MPoints/s, Time = %.5f s, Size = %u Points, NumDevsUsed = %u, Blocksize = %u\n", 
            throughputM, avgElapsedTime, pointsComputed, 1, dimBlock.x * dimBlock.y); 
    }
#endif
    
    // Cleanup
    if (bufferIn)
        hipFree(bufferIn);
    if (bufferOut)
        hipFree(bufferOut);
#ifdef GPU_PROFILING
    if (profileStart)
        hipEventDestroy(profileStart);
    if (profileEnd)
        hipEventDestroy(profileEnd);
#endif

    if (ok)
    {
        shrLog("\n cutilDeviceReset\n");

        errnum = cutilDeviceReset();
        if (errnum != hipSuccess)
        {
            shrLogEx(LOGBOTH | ERRORMSG, 0, "cutilDeviceReset '%s'.\n", hipGetErrorString(errnum));
            ok = false;
        }
    }

    return ok;
}
