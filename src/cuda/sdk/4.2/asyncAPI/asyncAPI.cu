#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 *
 *
 * This sample illustrates the usage of CUDA events for both GPU timing and
 * overlapping CPU and GPU execution.  Events are insterted into a stream
 * of CUDA calls.  Since CUDA stream calls are asynchronous, the CPU can
 * perform computations while GPU is executing (including DMA memcopies
 * between the host and device).  CPU can query CUDA events to determine
 * whether GPU has completed tasks.
 *
*/

// includes, system
#include <stdio.h>

// includes, project
#include <sdkHelper.h>  // helper for shared functions common to CUDA SDK samples
#include <shrQATest.h>

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions


__global__ void increment_kernel(int *g_data, int inc_value)
{ 
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   g_data[idx] = g_data[idx] + inc_value;
}

int correct_output(int *data, const int n, const int x)
{
    for(int i = 0; i < n; i++)
        if(data[i] != x)
            return 0;
    return 1;
}

int main(int argc, char *argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;

    shrQAStart(argc, argv);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice((const int)argc, (const char **)argv);

    // get device name 
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    int n = 16 * 64 * 1024;
    int nbytes = n * sizeof(int);
    int value = 26;

    // allocate host memory
    int *a = 0;
    checkCudaErrors( hipHostMalloc((void**)&a, nbytes) );
    memset(a, 0, nbytes);

    // allocate device memory
    int *d_a=0;
    checkCudaErrors( hipMalloc((void**)&d_a, nbytes) );
    checkCudaErrors( hipMemset(d_a, 255, nbytes) );

    // set kernel launch configuration
    dim3 threads = dim3(512, 1);
    dim3 blocks  = dim3(n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors( hipEventCreate(&start) );
    checkCudaErrors( hipEventCreate(&stop)  );
    
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors( hipDeviceSynchronize() );
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    sdkStartTimer(&timer);
        hipEventRecord(start, 0);
        hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
        increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
        hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
        hipEventRecord(stop, 0);
    sdkStopTimer(&timer);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long int counter=0;
    while( hipEventQuery(stop) == hipErrorNotReady )
    {
        counter++;
    }
    checkCudaErrors( hipEventElapsedTime(&gpu_time, start, stop) );

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer) );
    printf("CPU executed %d iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    bool bFinalResults = (bool)correct_output(a, n, value);
	
    // release resources
    checkCudaErrors( hipEventDestroy(start) );
    checkCudaErrors( hipEventDestroy(stop) );
    checkCudaErrors( hipHostFree(a) );
    checkCudaErrors( hipFree(d_a) );

    hipDeviceReset();

	shrQAFinishExit(argc, (const char **)argv, (bFinalResults ? QA_PASSED : QA_FAILED));
}
