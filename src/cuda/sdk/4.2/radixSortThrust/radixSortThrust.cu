#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

#include <cutil_inline.h>

#include <shrUtils.h>
#include <shrQATest.h>
#include <algorithm>
#include <time.h>
#include <limits.h>

template <typename T, bool floatKeys>
bool testSort(int argc, char **argv)
{
    int cmdVal;
    int keybits = 32;

    unsigned int numElements = 1048576;
    bool keysOnly = (cutCheckCmdLineFlag(argc, (const char**)argv, "keysonly") == CUTTrue);
    bool quiet = (cutCheckCmdLineFlag(argc, (const char**)argv, "quiet") == CUTTrue);

    if( cutGetCmdLineArgumenti( argc, (const char**)argv, "n", &cmdVal) )
    { 
        numElements = cmdVal;
		if (cmdVal < 0) {
		    printf("Error: n must be > 0, n=%d is invalid\n", cmdVal);
			exit(0);
		}
    }

    if( cutGetCmdLineArgumenti( argc, (const char**)argv, "keybits", &cmdVal) )
    {
        keybits = cmdVal;
        
        if (keybits <= 0) {
			printf("Error: keybits must be > 0, keybits=%d is invalid\n", keybits);
			exit(0);
        }
    }
    unsigned int numIterations = 1; 
    if ( cutGetCmdLineArgumenti(argc, (const char**) argv, "iterations", &cmdVal) )
    {
        numIterations = cmdVal;
    }

    if( cutCheckCmdLineFlag(argc, (const char**)argv, "help") )
    {
        shrLog("Command line:\nradixSortThrust [-option]\n");
        shrLog("Valid options:\n");
        shrLog("-n=<N>        : number of elements to sort\n");
        shrLog("-keybits=bits : keybits must be > 0\n");
        shrLog("-keysonly     : only sort an array of keys (default sorts key-value pairs)\n");
        shrLog("-float        : use 32-bit float keys (default is 32-bit unsigned int)\n");
        shrLog("-quiet        : Output only the number of elements and the time to sort\n");
        shrLog("-help         : Output a help message\n");
        exit(0);
    }

    if (!quiet)
        shrLog("\nSorting %d %d-bit %s keys %s\n\n", numElements, keybits, floatKeys ? "float" : "unsigned int", keysOnly ? "(only)" : "and values");

    int deviceID = -1;
    if (hipSuccess == hipGetDevice(&deviceID))
    {
        hipDeviceProp_t devprop;
        hipGetDeviceProperties(&devprop, deviceID);
        unsigned int totalMem = (keysOnly ? 2 : 4) * numElements * sizeof(T);
        if (devprop.totalGlobalMem < totalMem)
        {
            shrLog("Error: not enough memory to sort %d elements.\n", numElements);
            shrLog("%d bytes needed, %d bytes available\n", (int) totalMem, (int) devprop.totalGlobalMem);
            exit(0);
        }
    }

    thrust::host_vector<T> h_keys(numElements);
    thrust::host_vector<T> h_keysSorted(numElements);
    thrust::host_vector<unsigned int> h_values;
    if (!keysOnly)
        h_values = thrust::host_vector<unsigned int>(numElements);

    // Fill up with some random data
    thrust::default_random_engine rng(clock());
    if (floatKeys)
    {
        thrust::uniform_real_distribution<float> u01(0, 1);
        for(int i = 0; i < (int)numElements; i++)
            h_keys[i] = u01(rng);
    }
    else
    {
        thrust::uniform_int_distribution<unsigned int> u(0, UINT_MAX);
        for(int i = 0; i < (int)numElements; i++)
            h_keys[i] = u(rng);
    }
    
    if (!keysOnly)
        thrust::sequence(h_values.begin(), h_values.end());

    // Copy data onto the GPU
    thrust::device_vector<T> d_keys;
    thrust::device_vector<unsigned int> d_values;

    // run multiple iterations to compute an average sort time
    hipEvent_t start_event, stop_event;
    cutilSafeCall( hipEventCreate(&start_event) );
    cutilSafeCall( hipEventCreate(&stop_event) );

    float totalTime = 0;
    for(unsigned int i = 0; i < numIterations; i++) 
    {
        // reset data before sort
        d_keys= h_keys;
	    if (!keysOnly)
	       d_values = h_values;

	    cutilSafeCall( hipEventRecord(start_event, 0) );

        if(keysOnly)
            thrust::sort(d_keys.begin(), d_keys.end());
        else 
            thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_values.begin());

	    cutilSafeCall( hipEventRecord(stop_event, 0) );
	    cutilSafeCall( hipEventSynchronize(stop_event) );

	    float time = 0;
	    cutilSafeCall( hipEventElapsedTime(&time, start_event, stop_event));
	    totalTime += time;
    }
    totalTime /= (1.0e3f * numIterations);
    shrLogEx(LOGBOTH | MASTER, 0, "radixSort, Throughput = %.4f MElements/s, Time = %.5f s, Size = %u elements\n", 
               1.0e-6f * numElements / totalTime, totalTime, numElements); 

    CUT_CHECK_ERROR("after radixsort");

    // Get results back to host for correctness checking
    thrust::copy(d_keys.begin(), d_keys.end(), h_keysSorted.begin());
    if (!keysOnly)
        thrust::copy(d_values.begin(), d_values.end(), h_values.begin());

    CUT_CHECK_ERROR("copying results to host memory");

    // Check results
    bool bTestResult = thrust::is_sorted(h_keysSorted.begin(), h_keysSorted.end());

    cutilSafeCall( hipEventDestroy(start_event) );
    cutilSafeCall( hipEventDestroy(stop_event) );

    if ( !bTestResult  && !quiet) {
        return false;
    }
    return bTestResult;
}

int main(int argc, char **argv)
{
    shrQAStart(argc, argv);

    // Start logs
    shrSetLogFileName ("radixSort.txt");
    shrLog("%s Starting...\n\n", argv[0]);
    
    cutilDeviceInit(argc, argv);
  
    bool bTestResult = false;

    if( cutCheckCmdLineFlag( argc, (const char**)argv, "float") )
        bTestResult = testSort<float, true>(argc, argv);
    else
        bTestResult = testSort<unsigned int, false>(argc, argv);

    shrQAFinishExit(argc, (const char **)argv, bTestResult ? QA_PASSED : QA_FAILED);
}

