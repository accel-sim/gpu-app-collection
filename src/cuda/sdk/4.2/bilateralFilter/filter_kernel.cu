#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _FILTER_KERNEL_H_
#define _FILTER_KERNEL_H_

#include <shrUtils.h>
#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <cutil_math.h>
#include <generalCuda.cuh>

texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
texture<float, 1, hipReadModeElementType> gaussianTex;

hipArray* d_array, *d_tempArray, *d_gaussianArray;

/*
    Perform a simple bilateral filter.

    Bilateral filter is a nonlinear filter that is a mixture of range 
    filter and domain filter, the previous one preserves crisp edges and 
    the latter one filters noise. The intensity value at each pixel in 
    an image is replaced by a weighted average of intensity values from 
    nearby pixels.

    The weight factor is calculated by the product of domain filter
    component(using the gaussian distribution as a spatial distance) as 
    well as range filter component(Euclidean distance between center pixel
    and the current neighbor pixel). Because this process is nonlinear, 
    the sample just uses a simple pixel by pixel step. 

    Texture fetches automatically clamp to edge of image. 1D gaussian array
    is mapped to a 1D texture instead of using shared memory, which may 
    cause severe bank conflict.

    Threads are y-pass(column-pass), because the output is coalesced.

    Parameters
    od - pointer to output data in global memory
    d_f - pointer to the 1D gaussian array
    e_d - euclidean delta
    w  - image width
    h  - image height
    r  - filter radius
*/

//column pass using coalesced global memory reads
__global__ void
d_bilateral_filter(uint *od,
                   float e_d, int w, int h, int r)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
    if (x < w && y < h) {
        float sum = 0.0f;
        float factor;
        float4 t = make_float4(0.0f);
        float4 center = tex2D(rgbaTex, x, y);

        for(int i = -r; i <= r; i++)
        {
            for(int j = -r; j <= r; j++)
            {

                float4 curPix = tex2D(rgbaTex, x + j, y + i);
                factor = (tex1D(gaussianTex, i + r) * tex1D(gaussianTex, j + r)) *     //domain factor
                    euclideanLen(curPix, center, e_d); //range factor

                t += factor * curPix;
                sum += factor;
            }
        }
        od[y * w + x] = rgbaFloatToInt( t / sum);
    }
}

__global__ void debugGaussianKernel(float *readValue)
{
    uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    readValue[x] = tex1D(gaussianTex, x); 
}

extern "C"
void debugGaussianTex(int radius)
{
    float* d_readValue; 
    int size = 3 * radius + 1; 
    cutilSafeCall(hipMalloc( (void**)&d_readValue, size * sizeof(float))); 

    debugGaussianKernel<<< 1, size >>>(d_readValue); 

    float* h_readValue = (float*)calloc(size, sizeof(float)); 
    cutilSafeCall(hipMemcpy( h_readValue, d_readValue, size * sizeof(float), hipMemcpyDeviceToHost ));

    for (int s = 0; s < size; s++) {
         printf("gaussianTex[%d] = %f\n", s, h_readValue[s]); 
    }

    free(h_readValue); 
    cutilSafeCall(hipFree(d_readValue)); 
}

extern "C" 
void initTexture(int width, int height, void *pImage)
{
    int size = width * height * sizeof(unsigned int);

    // copy image data to array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    cutilSafeCall( hipMallocArray  ( &d_array, &channelDesc, width, height )); 
    cutilSafeCall( hipMallocArray  ( &d_tempArray, &channelDesc, width, height ));
    cutilSafeCall( hipMemcpyToArray( d_array, 0, 0, pImage, size, hipMemcpyHostToDevice));
}

extern "C"
void freeTextures()
{
    cutilSafeCall(hipFreeArray(d_array));
    cutilSafeCall(hipFreeArray(d_tempArray));
    cutilSafeCall(hipFreeArray(d_gaussianArray));
}

extern "C"
void updateGaussian(float delta, int radius)
{
    int size = 2 * radius + 1;

    float* d_gaussian;
    cutilSafeCall(hipMalloc( (void**) &d_gaussian, 
        (2 * radius + 1)* sizeof(float)));

    //generate gaussian array
    d_generate_gaussian<<< 1, size>>>(d_gaussian, delta, radius);

    //create cuda array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    cutilSafeCall( hipMallocArray( &d_gaussianArray, &channelDesc, size, 1 )); 
    cutilSafeCall( hipMemcpyToArray( d_gaussianArray, 0, 0, d_gaussian, size * sizeof (float), hipMemcpyDeviceToDevice));

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray( gaussianTex, d_gaussianArray, channelDesc));
    cutilSafeCall( hipFree(d_gaussian) );

    // debugGaussianTex(radius); // print out the content of gaussianTex for debugging 
}

/*
    Perform 2D bilateral filter on image using CUDA

    Parameters:
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    e_d    - euclidean delta
    radius - filter radius
    iterations - number of iterations
*/

// RGBA version
extern "C" 
double bilateralFilterRGBA(uint *d_dest,
                           int width, int height,
                           float e_d, int radius, int iterations,
                           int nthreads)
{
    // var for kernel computation timing
    double dKernelTime;

    // Bind the array to the texture
    cutilSafeCall( hipBindTextureToArray(rgbaTex, d_array) );

    for(int i=0; i<iterations; i++) 
    {
        // sync host and start kernel computation timer
        dKernelTime = 0.0;
        cutilSafeCall(cutilDeviceSynchronize());
        shrDeltaT(0);

        dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
        dim3 blockSize(16, 16);
        d_bilateral_filter<<< gridSize, blockSize>>>( 
            d_dest, e_d, width, height, radius);

        // sync host and stop computation timer
        cutilSafeCall( cutilDeviceSynchronize() );
        dKernelTime += shrDeltaT(0);

        if (iterations > 1) {
            // copy result back from global memory to array
            cutilSafeCall( hipMemcpyToArray( d_tempArray, 0, 0, d_dest, width * height * sizeof(float),
                hipMemcpyDeviceToDevice));
            cutilSafeCall( hipBindTextureToArray(rgbaTex, d_tempArray) );
        }
    }

    return (dKernelTime/(double)iterations);
}

#endif
