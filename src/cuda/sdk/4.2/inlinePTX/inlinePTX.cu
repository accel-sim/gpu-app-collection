#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Demonstration of inline PTX (assembly language) usage in CUDA kernels
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

using namespace std;

bool g_bQATest = false;

#ifdef STRCASECMP
#undef STRCASECMP
#endif
#ifdef STRNCASECMP
#undef STRNCASECMP
#endif

#ifdef _WIN32
   #define STRCASECMP  _stricmp
   #define STRNCASECMP _strnicmp
#else
   #define STRCASECMP  strcasecmp
   #define STRNCASECMP strncasecmp
#endif

#define ASSERT(x, msg, retcode) \
    if (!(x)) \
    { \
        cout << msg << " " << __FILE__ << ":" << __LINE__ << endl; \
        return retcode; \
    }


__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;
    if (elemID < length)
    {
        unsigned int laneid;
        //This command gets the lane ID within the current warp
        asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
        d_ptr[elemID] = laneid;
    }
}


void sequence_cpu(int *h_ptr, int length)
{
    for (int elemID=0; elemID<length; elemID++)
    {
        h_ptr[elemID] = elemID % 32;
    }
}

void processArgs(int argc, char **argv)
{
    for (int i=1; i < argc; i++) {
        if((!STRNCASECMP((argv[i]+1), "noprompt", 8)) || (!STRNCASECMP((argv[i]+2), "noprompt", 8)) )
        {
            g_bQATest = true;
        }
    }
}

int main(int argc, char **argv)
{

    shrQAStart(argc, argv);

    cout << "CUDA inline PTX assembler sample" << endl;
    cout << "================================" << endl;
    cout << "Self-test started" << endl;

    const int N = 1000;

    processArgs(argc, argv);
    
    int *d_ptr;
    ASSERT(hipSuccess == hipMalloc(&d_ptr, N * sizeof(int)), "Device allocation of " << N << " ints failed", -1);

    int *h_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_ptr, N * sizeof(int)), "Host allocation of " << N << " ints failed", -1);

    cout << "Memory allocated successfully" << endl;

    dim3 cudaBlockSize(256,1,1);
    dim3 cudaGridSize((N + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
    sequence_gpu<<<cudaGridSize, cudaBlockSize>>>(d_ptr, N);
    ASSERT(hipSuccess == hipGetLastError(), "Kernel launch failed", -1);
    ASSERT(hipSuccess == hipDeviceSynchronize(), "Kernel synchronization failed", -1);

    sequence_cpu(h_ptr, N);

    cout << "CUDA and CPU algorithm implementations finished" << endl;

    int *h_d_ptr;
    ASSERT(hipSuccess == hipHostMalloc(&h_d_ptr, N * sizeof(int)), "Host allocation of " << N << " ints failed", -1);
    ASSERT(hipSuccess == hipMemcpy(h_d_ptr, d_ptr, N * sizeof(int), hipMemcpyDeviceToHost), "Copy of " << N << " ints from device to host failed", -1);
    bool bValid = true;
    for (int i=0; i<N && bValid; i++)
    {
        if (h_ptr[i] != h_d_ptr[i])
        {
            bValid = false;
        }
    }

    ASSERT(hipSuccess == hipFree(d_ptr), "Device deallocation failed", -1);
    ASSERT(hipSuccess == hipHostFree(h_ptr), "Host deallocation failed", -1);
    ASSERT(hipSuccess == hipHostFree(h_d_ptr), "Host deallocation failed", -1);

    cout << "Memory deallocated successfully" << endl;
    
    hipDeviceReset();

    cout << "TEST Results " << endl;
    shrQAFinishExit(argc, (const char **)argv, (bValid ? QA_PASSED : QA_FAILED));
}
