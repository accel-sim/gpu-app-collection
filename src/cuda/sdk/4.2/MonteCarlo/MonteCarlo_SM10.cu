/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



#include "MonteCarlo_kernel.cuh"
#include "quasirandomGenerator_kernel.cuh"



extern "C" void initMonteCarlo_SM10(TOptionPlan *plan){
    initMonteCarloGPU(plan);
}

extern "C" void closeMonteCarlo_SM10(TOptionPlan *plan){
    closeMonteCarloGPU(plan);
}

extern "C" void MonteCarlo_SM10(TOptionPlan *plan){
    MonteCarloGPU(plan);
}

extern "C" void inverseCND_SM10(float *d_Output, unsigned int *d_Input, unsigned int N){
    inverseCNDgpu(d_Output, d_Input, N);
}
