/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */



#include "binomialOptions_kernel.cuh"


extern "C" void binomialOptions_SM10
  ( float *callValue,
    TOptionData  *optionData,
    int optN )
{
    binomialOptionsGPU(callValue, optionData, optN);
}
