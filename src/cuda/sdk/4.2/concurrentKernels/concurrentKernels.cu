#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also illustrates how to 
// introduce dependencies between CUDA streams with the new hipStreamWaitEvent function introduced 
// in CUDA 3.2.
//
// Devices of compute capability 1.x will run the kernels one after another
// Devices of compute capability 2.0 or higher can overlap the kernels
//
#include <stdio.h>
#include <cutil_inline.h>
#include <shrUtils.h>
#include <shrQATest.h>

const char *sSDKsample = "concurrentKernels";

// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block(clock_t* d_o, clock_t clock_count)
{ 
	clock_t start_clock = clock();
	
	clock_t clock_offset = 0;

	while( clock_offset < clock_count ) {
		clock_offset = clock() - start_clock;
	}

	d_o[0] = clock_offset;
}


// Single warp reduction kernel
__global__ void sum(clock_t* d_clocks, int N)
{
	__shared__ clock_t s_clocks[32];

	clock_t my_sum = 0;

	for( int i = threadIdx.x; i < N; i+= blockDim.x ) {
		my_sum += d_clocks[i];
	}

	s_clocks[threadIdx.x] = my_sum;
	syncthreads();	

	for( int i=16; i>0; i/=2) {
		if( threadIdx.x < i ) {
			s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
		}
		syncthreads();	
	}	

	d_clocks[0] = s_clocks[0];
}

int main(int argc, char **argv)
{
    int nkernels = 8;               // number of concurrent kernels
    int nstreams = nkernels + 1;    // use one more stream than concurrent kernel
    int nbytes = nkernels * sizeof(clock_t);   // number of data bytes
    float kernel_time = 10; // time the kernel should run in ms
    float elapsed_time;   // timing variables
    int cuda_device = 0;

    shrQAStart(argc, argv); 

    // get number of kernels if overridden on the command line
    if (cutCheckCmdLineFlag(argc, (const char **)argv, "nkernels")) {
        cutGetCmdLineArgumenti(argc, (const char **)argv, "nkernels", &nkernels);
        nstreams = nkernels + 1;
    }

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    cuda_device = cutilChooseCudaDevice(argc, argv);

    hipDeviceProp_t deviceProp;
    cutilSafeCall( hipGetDevice(&cuda_device));	

    cutilSafeCall( hipGetDeviceProperties(&deviceProp, cuda_device) );
    if( (deviceProp.concurrentKernels == 0 )) {
        shrLog("> GPU does not support concurrent kernel execution\n");
        shrLog("  CUDA kernel runs will be serialized\n");
    }

    shrLog("> Detected Compute SM %d.%d hardware with %d multi-processors\n", 
           deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount); 

    // allocate host memory
    clock_t *a = 0;                     // pointer to the array data in host memory
    cutilSafeCall( hipHostMalloc((void**)&a, nbytes) ); 

    // allocate device memory
    clock_t *d_a = 0;             // pointers to data and init value in the device memory
    cutilSafeCall( hipMalloc((void**)&d_a, nbytes) );

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 0; i < nstreams; i++)
        cutilSafeCall( hipStreamCreate(&(streams[i])) );

    // create CUDA event handles
    hipEvent_t start_event, stop_event;
    cutilSafeCall( hipEventCreate(&start_event) );
    cutilSafeCall( hipEventCreate(&stop_event) );

   
    // the events are used for synchronization only and hence do not need to record timings
    // this also makes events not introduce global sync points when recorded which is critical to get overlap 
    hipEvent_t *kernelEvent;
    kernelEvent = (hipEvent_t*) malloc(nkernels * sizeof(hipEvent_t));
    for(int i = 0; i < nkernels; i++)
        cutilSafeCall( hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming) );

    //////////////////////////////////////////////////////////////////////
    // time execution with nkernels streams
    clock_t total_clocks = 0;
    clock_t time_clocks = kernel_time * deviceProp.clockRate;
	
    hipEventRecord(start_event, 0);
    // queue nkernels in separate streams and record when they are done
    for( int i=0; i<nkernels; ++i)
    {
        clock_block<<<1,1,0,streams[i]>>>(&d_a[i], time_clocks );
        total_clocks += time_clocks;
        cutilSafeCall( hipEventRecord(kernelEvent[i], streams[i]) );
	
        // make the last stream wait for the kernel event to be recorded
        cutilSafeCall( hipStreamWaitEvent(streams[nstreams-1], kernelEvent[i],0) );
    }

    // queue a sum kernel and a copy back to host in the last stream. 
    // the commands in this stream get dispatched as soon as all the kernel events have been recorded
    sum<<<1,32,0,streams[nstreams-1]>>>(d_a, nkernels);
    cutilSafeCall( hipMemcpyAsync(a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams-1]) );
 
    // at this point the CPU has dispatched all work for the GPU and can continue processing other tasks in parallel

    // in this sample we just wait until the GPU is done
    cutilSafeCall( hipEventRecord(stop_event, 0) );
    cutilSafeCall( hipEventSynchronize(stop_event) );
    cutilSafeCall( hipEventElapsedTime(&elapsed_time, start_event, stop_event) );
    
    shrLog("Expected time for serial execution of %d kernels = %.3fs\n", nkernels, nkernels * kernel_time/1000.0f);
    shrLog("Expected time for concurrent execution of %d kernels = %.3fs\n", nkernels, kernel_time/1000.0f);
    shrLog("Measured time for sample = %.3fs\n", elapsed_time/1000.0f);

    bool bTestResult  = (a[0] > total_clocks);

    // release resources
    for(int i = 0; i < nkernels; i++) {
        hipStreamDestroy(streams[i]); 
        hipEventDestroy(kernelEvent[i]);
    }
    free(streams);
    free(kernelEvent);

    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipHostFree(a);
    hipFree(d_a);

    cutilDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bTestResult) ? QA_PASSED : QA_FAILED);
}
