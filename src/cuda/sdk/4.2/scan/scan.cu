#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 

#include <assert.h>
#include <cutil_inline.h>
#include "scan_common.h"

//All three kernels run 512 threads per workgroup
//Must be a power of two
#define THREADBLOCK_SIZE 256

////////////////////////////////////////////////////////////////////////////////
// Basic ccan codelets
////////////////////////////////////////////////////////////////////////////////
#if(0)
    //Naive inclusive scan: O(N * log2(N)) operations
    //Allocate 2 * 'size' local memory, initialize the first half
    //with 'size' zeros avoiding if(pos >= offset) condition evaluation
    //and saving instructions
    inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size){
        uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
        s_Data[pos] = 0;
        pos += size;
        s_Data[pos] = idata;

        for(uint offset = 1; offset < size; offset <<= 1){
            __syncthreads();
            uint t = s_Data[pos] + s_Data[pos - offset];
            __syncthreads();
            s_Data[pos] = t;
        }

        return s_Data[pos];
    }

    inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size){
        return scan1Inclusive(idata, s_Data, size) - idata;
    }

#else
    #define LOG2_WARP_SIZE 5U
    #define WARP_SIZE (1U << LOG2_WARP_SIZE)

    //Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
    //assuming size <= WARP_SIZE
    inline __device__ uint warpScanInclusive(uint idata, volatile uint *s_Data, uint size){
        uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
        s_Data[pos] = 0;
        pos += size;
        s_Data[pos] = idata;

        for(uint offset = 1; offset < size; offset <<= 1)
            s_Data[pos] += s_Data[pos - offset];

        return s_Data[pos];
    }

    inline __device__ uint warpScanExclusive(uint idata, volatile uint *s_Data, uint size){
        return warpScanInclusive(idata, s_Data, size) - idata;
    }

    inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data, uint size){
        if(size > WARP_SIZE){
            //Bottom-level inclusive warp scan
            uint warpResult = warpScanInclusive(idata, s_Data, WARP_SIZE);

            //Save top elements of each warp for exclusive warp scan
            //sync to wait for warp scans to complete (because s_Data is being overwritten)
            __syncthreads();
            if( (threadIdx.x & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )
                s_Data[threadIdx.x >> LOG2_WARP_SIZE] = warpResult;

            //wait for warp scans to complete
            __syncthreads();
            if( threadIdx.x < (THREADBLOCK_SIZE / WARP_SIZE) ){
                //grab top warp elements
                uint val = s_Data[threadIdx.x];
                //calculate exclsive scan and write back to shared memory
                s_Data[threadIdx.x] = warpScanExclusive(val, s_Data, size >> LOG2_WARP_SIZE);
            }

            //return updated warp scans with exclusive scan results
            __syncthreads();
            return warpResult + s_Data[threadIdx.x >> LOG2_WARP_SIZE];
        }else{
            return warpScanInclusive(idata, s_Data, size);
        }
    }

    inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data, uint size){
        return scan1Inclusive(idata, s_Data, size) - idata;
    }

#endif

inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data, uint size){
    //Level-0 inclusive scan
    idata4.y += idata4.x;
    idata4.z += idata4.y;
    idata4.w += idata4.z;

    //Level-1 exclusive scan
    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);

    idata4.x += oval;
    idata4.y += oval;
    idata4.z += oval;
    idata4.w += oval;

    return idata4;
}

//Exclusive vector scan: the array to be scanned is stored
//in local thread memory scope as uint4
inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data, uint size){
    uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
    odata4.x -= idata4.x;
    odata4.y -= idata4.y;
    odata4.z -= idata4.z;
    odata4.w -= idata4.w;
    return odata4;
}

////////////////////////////////////////////////////////////////////////////////
// Scan kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void scanExclusiveShared(
    uint4 *d_Dst,
    uint4 *d_Src,
    uint size
){
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load data
    uint4 idata4 = d_Src[pos];

    //Calculate exclusive scan
    uint4 odata4 = scan4Exclusive(idata4, s_Data, size);

    //Write back
    d_Dst[pos] = odata4;
}

//Exclusive scan of top elements of bottom-level scans (4 * THREADBLOCK_SIZE)
__global__ void scanExclusiveShared2(
    uint *d_Buf,
    uint *d_Dst,
    uint *d_Src,
    uint N,
    uint arrayLength
){
    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

    //Skip loads and stores for inactive threads of last threadblock (pos >= N)
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Load top elements
    //Convert results of bottom-level scan back to inclusive
    uint idata = 0;
    if(pos < N)
        idata = 
        d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] + 
        d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];

    //Compute
    uint odata = scan1Exclusive(idata, s_Data, arrayLength);

    //Avoid out-of-bound access
    if(pos < N)
        d_Buf[pos] = odata;
}

//Final step of large-array scan: combine basic inclusive scan with exclusive scan of top elements of input arrays
__global__ void uniformUpdate(
    uint4 *d_Data,
    uint *d_Buffer
){
    __shared__ uint buf;
    uint pos = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIdx.x == 0)
        buf = d_Buffer[blockIdx.x];
    __syncthreads();

    uint4 data4 = d_Data[pos];
    data4.x += buf;
    data4.y += buf;
    data4.z += buf;
    data4.w += buf;
    d_Data[pos] = data4;
}

////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
//Derived as 32768 (max power-of-two gridDim.x) * 4 * THREADBLOCK_SIZE
//Due to scanExclusiveShared<<<>>>() 1D block addressing
extern "C" const uint MAX_BATCH_ELEMENTS = 64 * 1048576;
extern "C" const uint MIN_SHORT_ARRAY_SIZE = 4;
extern "C" const uint MAX_SHORT_ARRAY_SIZE = 4 * THREADBLOCK_SIZE;
extern "C" const uint MIN_LARGE_ARRAY_SIZE = 8 * THREADBLOCK_SIZE;
extern "C" const uint MAX_LARGE_ARRAY_SIZE = 4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE;

//Internal exclusive scan buffer
static uint *d_Buf;

extern "C" void initScan(void){
    cutilSafeCall( hipMalloc((void **)&d_Buf, (MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE)) * sizeof(uint)) );
}

extern "C" void closeScan(void){
    cutilSafeCall( hipFree(d_Buf) );
}

static uint factorRadix2(uint& log2L, uint L){
    if(!L){
        log2L = 0;
        return 0;
    }else{
        for(log2L = 0; (L & 1) == 0; L >>= 1, log2L++);
        return L;
    }
}

static uint iDivUp(uint dividend, uint divisor){
    return ( (dividend % divisor) == 0 ) ? (dividend / divisor) : (dividend / divisor + 1);
}

extern "C" size_t scanExclusiveShort(
    uint *d_Dst,
    uint *d_Src,
    uint batchSize,
    uint arrayLength
){
    //Check power-of-two factorization
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert( factorizationRemainder == 1 );

    //Check supported size range
    assert( (arrayLength >= MIN_SHORT_ARRAY_SIZE) && (arrayLength <= MAX_SHORT_ARRAY_SIZE) );

    //Check total batch size limit
    assert( (batchSize * arrayLength) <= MAX_BATCH_ELEMENTS );

    //Check all threadblocks to be fully packed with data
    assert( (batchSize * arrayLength) % (4 * THREADBLOCK_SIZE) == 0 );

    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        arrayLength
    );
    cutilCheckMsg("scanExclusiveShared() execution FAILED\n");

    return THREADBLOCK_SIZE;
}

extern "C" size_t scanExclusiveLarge(
    uint *d_Dst,
    uint *d_Src,
    uint batchSize,
    uint arrayLength
){
    //Check power-of-two factorization
    uint log2L;
    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
    assert( factorizationRemainder == 1 );

    //Check supported size range
    assert( (arrayLength >= MIN_LARGE_ARRAY_SIZE) && (arrayLength <= MAX_LARGE_ARRAY_SIZE) );

    //Check total batch size limit
    assert( (batchSize * arrayLength) <= MAX_BATCH_ELEMENTS );

    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint4 *)d_Src,
        4 * THREADBLOCK_SIZE
    );
    cutilCheckMsg("scanExclusiveShared() execution FAILED\n");

    //Not all threadblocks need to be packed with input data: 
    //inactive threads of highest threadblock just don't do global reads and writes
    const uint blockCount2 = iDivUp( (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE );
    scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
        (uint *)d_Buf,
        (uint *)d_Dst,
        (uint *)d_Src,
        (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
        arrayLength / (4 * THREADBLOCK_SIZE)
    );
    cutilCheckMsg("scanExclusiveShared2() execution FAILED\n");

    uniformUpdate<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
        (uint4 *)d_Dst,
        (uint  *)d_Buf
    );
    cutilCheckMsg("uniformUpdate() execution FAILED\n");

    return THREADBLOCK_SIZE;
}
