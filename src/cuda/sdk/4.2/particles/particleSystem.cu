#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <shrQATest.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel.cu"

extern "C"
{

void cudaInit(int argc, char **argv)
{   
    int devID;
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        devID = cutilDeviceInit(argc, argv);
        if (devID < 0) {
            printf("No CUDA Capable devices found, exiting...\n");
            shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
        }
    } else {
        devID = cutGetMaxGflopsDeviceId();
        hipSetDevice( devID );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void allocateArray(void **devPtr, size_t size)
{
    cutilSafeCall(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr)
{
    cutilSafeCall(hipFree(devPtr));
}

void threadSync()
{
    cutilSafeCall(cutilDeviceSynchronize());
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    cutilSafeCall(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    cutilSafeCall(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
					       cudaGraphicsMapFlagsNone));
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    cutilSafeCall(hipGraphicsUnregisterResource(cuda_vbo_resource));	
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    cutilSafeCall(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
    size_t num_bytes; 
    cutilSafeCall(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
						       *cuda_vbo_resource));
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
   cutilSafeCall(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
{   
    if (cuda_vbo_resource)
	device = mapGLBufferObject(cuda_vbo_resource);

    cutilSafeCall(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
    
    if (cuda_vbo_resource)
	unmapGLBufferObject(*cuda_vbo_resource);
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)) );
}

//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void integrateSystem(float *pos,
                     float *vel,
                     float deltaTime,
                     uint numParticles)
{
    thrust::device_ptr<float4> d_pos4((float4 *)pos);
    thrust::device_ptr<float4> d_vel4((float4 *)vel);

    thrust::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
        integrate_functor(deltaTime));
}

void calcHash(uint*  gridParticleHash,
              uint*  gridParticleIndex,
              float* pos, 
              int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");
}

void reorderDataAndFindCellStart(uint*  cellStart,
							     uint*  cellEnd,
							     float* sortedPos,
							     float* sortedVel,
                                 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
							     float* oldPos,
							     float* oldVel,
							     uint   numParticles,
							     uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // set all cells to empty
	cutilSafeCall(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
#endif

    uint smemSize = sizeof(uint)*(numThreads+1);
    reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
        cellStart,
        cellEnd,
        (float4 *) sortedPos,
        (float4 *) sortedVel,
		gridParticleHash,
		gridParticleIndex,
        (float4 *) oldPos,
        (float4 *) oldVel,
        numParticles);
    cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
#endif
}

void collide(float* newVel,
             float* sortedPos,
             float* sortedVel,
             uint*  gridParticleIndex,
             uint*  cellStart,
             uint*  cellEnd,
             uint   numParticles,
             uint   numCells)
{
#if USE_TEX
    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));    
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    collideD<<< numBlocks, numThreads >>>((float4*)newVel,
                                          (float4*)sortedPos,
                                          (float4*)sortedVel,
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

#if USE_TEX
    cutilSafeCall(hipUnbindTexture(oldPosTex));
    cutilSafeCall(hipUnbindTexture(oldVelTex));
    cutilSafeCall(hipUnbindTexture(cellStartTex));
    cutilSafeCall(hipUnbindTexture(cellEndTex));
#endif
}


void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                        thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                        thrust::device_ptr<uint>(dGridParticleIndex));
}

}   // extern "C"
