#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This is a simple test showing huge access speed gap
 * between aligned and misaligned structures
 * (those having/missing __align__ keyword).
 * It measures per-element copy throughput for 
 * aligned and misaligned structures on 
 * big chunks of data.
 */


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <sdkHelper.h>  // helper for shared functions common to CUDA SDK samples
#include <shrQATest.h>
#include <shrUtils.h>

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
// Misaligned types
////////////////////////////////////////////////////////////////////////////////
typedef unsigned char uint8;

typedef unsigned short int uint16;

typedef struct{
    unsigned char r, g, b, a;
} RGBA8_misaligned;

typedef struct{
    unsigned int l, a;
} LA32_misaligned;

typedef struct{
    unsigned int r, g, b;
} RGB32_misaligned;

typedef struct{
    unsigned int r, g, b, a;
} RGBA32_misaligned;



////////////////////////////////////////////////////////////////////////////////
// Aligned types
////////////////////////////////////////////////////////////////////////////////
typedef struct __align__(4){
    unsigned char r, g, b, a;
} RGBA8;

typedef unsigned int I32;

typedef struct __align__(8){
    unsigned int l, a;
} LA32;

typedef struct __align__(16){
    unsigned int r, g, b;
} RGB32;

typedef struct __align__(16){
    unsigned int r, g, b, a;
} RGBA32;


////////////////////////////////////////////////////////////////////////////////
// Because G80 class hardware natively supports global memory operations
// only with data elements of 4, 8 and 16 bytes, if structure size
// exceeds 16 bytes, it can't be efficiently read or written,
// since more than one global memory non-coalescable load/store instructions
// will be generated, even if __align__ option is supplied.
// "Structure of arrays" storage strategy offers best performance 
// in general case. See section 5.1.2 of the Programming Guide.
////////////////////////////////////////////////////////////////////////////////
typedef struct __align__(16){
    RGBA32 c1, c2;
} RGBA32_2;



////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}



////////////////////////////////////////////////////////////////////////////////
// Simple CUDA kernel.
// Copy is carried out on per-element basis,
// so it's not per-byte in case of padded structures.
////////////////////////////////////////////////////////////////////////////////
template<class TData> __global__ void testKernel(
    TData *d_odata,
    TData *d_idata,
    int numElements
){
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for(int pos = tid; pos < numElements; pos += numThreads)
        d_odata[pos] = d_idata[pos];
}



////////////////////////////////////////////////////////////////////////////////
// Validation routine for simple copy kernel.
// We must know "packed" size of TData (number_of_fields * sizeof(simple_type))
// and compare only these "packed" parts of the structure, 
// containig actual user data. The compiler behavior with padding bytes
// is undefined, since padding is merely a placeholder 
// and doesn't contain any user data.
////////////////////////////////////////////////////////////////////////////////
template<class TData> int testCPU(
    TData *h_odata,
    TData *h_idata,
    int numElements,
    int packedElementSize
){
    for(int pos = 0; pos < numElements; pos++){
        TData src = h_idata[pos];
        TData dst = h_odata[pos];
        for(int i = 0; i < packedElementSize; i++)
            if( ((char *)&src)[i] != ((char *)&dst)[i] ) return 0;
    }

    return 1;
}



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//Memory chunk size in bytes. Reused for test
// const int       MEM_SIZE = 50000000;
// const int NUM_ITERATIONS = 32;
const int       MEM_SIZE = 150000;
const int NUM_ITERATIONS = 2;

//GPU input and output data
unsigned char *d_idata, *d_odata;
//CPU input data and instance of GPU output data
unsigned char *h_idataCPU, *h_odataGPU;
StopWatchInterface *hTimer;



template<class TData> int runTest(int packedElementSize, int memory_size){
    const int totalMemSizeAligned = iAlignDown(memory_size, sizeof(TData));
    const int         numElements = iDivDown(memory_size, sizeof(TData));

    //Clean output buffer before current test
    checkCudaErrors( hipMemset(d_odata, 0, memory_size) );
    //Run test
    checkCudaErrors( hipDeviceSynchronize() );
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
    for(int i = 0; i < NUM_ITERATIONS; i++){
        testKernel<TData><<<64, 256>>>(
            (TData *)d_odata,
            (TData *)d_idata,
            numElements
        );
        getLastCudaError("testKernel() execution failed\n");
    }
    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer(&hTimer);
    double gpuTime = sdkGetTimerValue(&hTimer) / NUM_ITERATIONS;
    printf(
        "Avg. time: %f ms / Copy throughput: %f GB/s.\n", gpuTime,
        (double)totalMemSizeAligned / (gpuTime * 0.001 * 1073741824.0)
    );

    //Read back GPU results and run validation
    checkCudaErrors( hipMemcpy(h_odataGPU, d_odata, memory_size, hipMemcpyDeviceToHost) );
    int flag = testCPU(
        (TData *)h_odataGPU,
        (TData *)h_idataCPU,
        numElements,
        packedElementSize
    );
    
    printf(flag ? "\tTEST OK\n" : "\tTEST FAILURE\n" );

	return !flag;
}

int main(int argc, char **argv){
    int i, nTotalFailures = 0;

    int devID;
    hipDeviceProp_t deviceProp;
    shrQAStart(argc, argv);

    // find first CUDA device
    devID = findCudaDevice(argc, (const char **)argv);
		
    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
    printf("[%s] has %d MP(s) x %d (Cores/MP) = %d (Cores)\n", 
			deviceProp.name, deviceProp.multiProcessorCount,
			ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
			ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);

    // Anything that is less than 192 Cores will have a scaled down workload
    float scale_factor = max((192.0f / (ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * (float)deviceProp.multiProcessorCount) ), 1.0f);

    int   MemorySize = (int)(MEM_SIZE/scale_factor) & 0xffffff00; // force multiple of 256 bytes

    printf("> Compute scaling value = %4.2f\n", scale_factor);
    printf("> Memory Size = %d\n", MemorySize);

    sdkCreateTimer(&hTimer);

    printf("Allocating memory...\n");
        h_idataCPU = (unsigned char *)malloc(MemorySize);
        h_odataGPU = (unsigned char *)malloc(MemorySize);
        checkCudaErrors(hipMalloc((void **)&d_idata, MemorySize));
        checkCudaErrors(hipMalloc((void **)&d_odata, MemorySize));

    printf("Generating host input data array...\n");
        for(i = 0; i < MemorySize; i++)
            h_idataCPU[i] = (i & 0xFF) + 1;
    printf("Uploading input data to GPU memory...\n");
        checkCudaErrors(hipMemcpy(d_idata, h_idataCPU, MemorySize, hipMemcpyHostToDevice));

    printf("Testing misaligned types...\n");
        printf("uint8...\n");
        nTotalFailures += runTest<uint8>(1, MemorySize);

        printf("uint16...\n");
        nTotalFailures += runTest<uint16>(2, MemorySize);

        printf("RGBA8_misaligned...\n");
        nTotalFailures += runTest<RGBA8_misaligned>(4, MemorySize);

        printf("LA32_misaligned...\n");
        nTotalFailures += runTest<LA32_misaligned>(8, MemorySize);

        printf("RGB32_misaligned...\n");
        nTotalFailures += runTest<RGB32_misaligned>(12, MemorySize);

        printf("RGBA32_misaligned...\n");
        nTotalFailures += runTest<RGBA32_misaligned>(16, MemorySize);

    printf("Testing aligned types...\n");
        printf("RGBA8...\n");
        nTotalFailures += runTest<RGBA8>(4, MemorySize);

        printf("I32...\n");
        nTotalFailures += runTest<I32>(4, MemorySize);

        printf("LA32...\n");
        nTotalFailures += runTest<LA32>(8, MemorySize);

        printf("RGB32...\n");
        nTotalFailures += runTest<RGB32>(12, MemorySize);

        printf("RGBA32...\n");
        nTotalFailures += runTest<RGBA32>(16, MemorySize);

        printf("RGBA32_2...\n");
        nTotalFailures += runTest<RGBA32_2>(32, MemorySize);

	printf("\n[alignedTypes] -> Test Results: %d Failures\n", nTotalFailures);

    printf("Shutting down...\n");
        checkCudaErrors(hipFree(d_idata));
        checkCudaErrors(hipFree(d_odata));
        free(h_odataGPU);
        free(h_idataCPU);

    sdkDeleteTimer(&hTimer);
    hipDeviceReset();
	shrQAFinishExit(argc, (const char **)argv, (nTotalFailures==0 ? QA_PASSED : QA_FAILED));
}
