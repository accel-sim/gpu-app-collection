#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

// includes, kernels
#include <cppIntegration_kernel.cu>

//
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void computeGold(char* reference, char* idata, const unsigned int len);
extern "C" void computeGold2(int2* reference, int2* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" bool
runTest(const int argc, const char** argv, char* data, int2* data_int2, unsigned int len)
{
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, argv);

    const unsigned int num_threads = len / 4;
    assert(0 == (len % 4));
    const unsigned int mem_size = sizeof(char) * len;
    const unsigned int mem_size_int2 = sizeof(int2) * len;

    // allocate device memory
    char* d_data;
    checkCudaErrors(hipMalloc((void**) &d_data, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data, data, mem_size,
                            hipMemcpyHostToDevice) );
    // allocate device memory for int2 version
    int2* d_data_int2;
    checkCudaErrors(hipMalloc((void**) &d_data_int2, mem_size_int2));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                            hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 grid(1, 1, 1);
    dim3 threads(num_threads, 1, 1);
    dim3 threads2(len, 1, 1); // more threads needed fir separate int2 version
    // execute the kernel
    kernel<<< grid, threads >>>((int*) d_data);
    kernel2<<< grid, threads2 >>>(d_data_int2);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // compute reference solutions
    char* reference = (char*) malloc(mem_size);
    computeGold(reference, data, len);
    int2* reference2 = (int2*) malloc(mem_size_int2);
    computeGold2(reference2, data_int2, len);

    // copy results from device to host
    checkCudaErrors(hipMemcpy(data, d_data, mem_size,
                            hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                            hipMemcpyDeviceToHost));

    // check result
    bool success = true;
    for(unsigned int i = 0; i < len; i++ )
    {
        if( reference[i] != data[i] || 
	    reference2[i].x != data_int2[i].x || 
	    reference2[i].y != data_int2[i].y) 
        {
            success = false;
        }
    }

    // cleanup memory
    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFree(d_data_int2));
    free(reference);
    free(reference2);

    return success;
}
