/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
/*
* This sample demonstrates how use texture fetches in CUDA
*
* This sample takes an input PGM image (image_filename) and generates 
* an output PGM image (image_filename_out).  This CUDA kernel performs
* a simple 2D transform (rotation) on the texture coordinates (u,v).
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>
#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, kernels
#include <simpleSurfaceWrite_kernel.cu>

char *image_filename = "lena_bw.pgm";
char *ref_filename   = "ref_rotated.pgm";
float angle = 0.5f;    // angle to rotate image by (in radians)

static char *sSDKname = "simpleSurfaceWrite";

#define MIN_EPSILON_ERROR 5e-3f


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);                                                  
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv)
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    // If the command-line has a device number specified, use it
    if (checkCmdLineFlag(argc, argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, argv, "device=");
        if (devID < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(-1);
        }
        else
        {
            devID = gpuDeviceInit(devID);
            if (devID < 0)
            {
                printf("exiting...\n");
                shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                exit(-1);
            }
        }
    }
    else
    {
        // Otherwise pick the device with highest Gflops/s
        devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors( hipSetDevice( devID ) );
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
    return devID;
}
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    bool bTestResult = true;
    shrQAStart(argc, argv);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
	int devID = findCudaDevice(argc, (const char **)argv);

    // get number of SMs on this GPU
    hipDeviceProp_t deviceProps;

	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors, SM %d.%d\n", deviceProps.name, deviceProps.multiProcessorCount, deviceProps.major, deviceProps.minor );

	if (deviceProps.major < 2) {
		printf("%s requires SM >= 2.0 for SurfaceWrites, exiting... \n", sSDKname);
		hipDeviceReset();
        shrQAFinishExit(argc, (const char **)argv, QA_WAIVED);
	}

	// load image from disk
    float* h_data = NULL;
    unsigned int width, height;
    char* image_path = sdkFindFilePath(image_filename, argv[0]);
    if (image_path == NULL) {
        printf("Unable to source image input file: %s\n", image_filename);
        shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    sdkLoadPGM(image_path, &h_data, &width, &height);

    unsigned int size = width * height * sizeof(float);
    printf("Loaded '%s', %d x %d pixels\n", image_filename, width, height);

    // load reference image from image (output)
    float *h_data_ref = (float*) malloc(size);
    char* ref_path = sdkFindFilePath(ref_filename, argv[0]);
    if (ref_path == NULL) {
        printf("Unable to find reference image file: %s\n", ref_filename);
        shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
    }
    sdkLoadPGM(ref_path, &h_data_ref, &width, &height);

	// allocate device memory for result
    float* d_data = NULL;
    checkCudaErrors( hipMalloc( (void**) &d_data, size));

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_array;
    checkCudaErrors( hipMallocArray( &cu_array, &channelDesc, width, height, hipArraySurfaceLoadStore )); 

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

#if 1
    checkCudaErrors( hipMemcpy( d_data, h_data, size, hipMemcpyHostToDevice) );
    checkCudaErrors(cudaBindSurfaceToArray(output_surface, cu_array));

    surfaceWriteKernel<<< dimGrid, dimBlock >>>( d_data, width, height);
#else // this is what differs from the example simpleTexture
    checkCudaErrors( hipMemcpyToArray( cu_array, 0, 0, h_data, size, hipMemcpyHostToDevice));
#endif

    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors( hipBindTextureToArray( tex, cu_array, channelDesc));

    // warmup
    transformKernel<<< dimGrid, dimBlock, 0 >>>( d_data, width, height, angle);

    checkCudaErrors( hipDeviceSynchronize() );
    
	StopWatchInterface *timer = NULL;
    sdkCreateTimer( &timer );
    sdkStartTimer ( &timer );

    // execute the kernel
    transformKernel<<< dimGrid, dimBlock, 0 >>>( d_data, width, height, angle);

    // check if kernel execution generated an error
	getLastCudaError("Kernel execution failed");

    hipDeviceSynchronize();
    sdkStopTimer( &timer );
    printf("Processing time: %f (ms)\n", sdkGetTimerValue( &timer ));
    printf("%.2f Mpixels/sec\n", (width*height / (sdkGetTimerValue( &timer ) / 1000.0f)) / 1e6);
    sdkDeleteTimer( &timer );

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( size);
    // copy result from device to host
    checkCudaErrors( hipMemcpy( h_odata, d_data, size, hipMemcpyDeviceToHost) );

    // write result to file
    char output_filename[1024];
    strcpy(output_filename, "output.pgm");
    sdkSavePGM( "output.pgm", h_odata, width, height );
    printf("Wrote '%s'\n", output_filename);

    // write regression file if necessary
    if( checkCmdLineFlag( argc, (const char**) argv, "regression"))
    {
        // write file for regression test
        sdkWriteFile<float>( "./data/regression.dat", h_odata, width*height, 0.0f, false );
    } 
    else 
    {
        // We need to reload the data from disk, because it is inverted upon output
        sdkLoadPGM(output_filename, &h_odata, &width, &height);

        printf("Comparing files\n");
        printf("\toutput:    <%s>\n", output_filename);
        printf("\treference: <%s>\n", ref_path);
        bTestResult = compareData( h_odata, h_data_ref, width*height, MIN_EPSILON_ERROR, 0.0f );
    }

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFreeArray(cu_array));
    free(image_path);
    free(ref_path);

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bTestResult ? QA_PASSED : QA_FAILED) );
}
