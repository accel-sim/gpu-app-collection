#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#ifndef _SIMPLETEXTURE_KERNEL_H_
#define _SIMPLETEXTURE_KERNEL_H_

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;
surface<void, 2> output_surface;

////////////////////////////////////////////////////////////////////////////////
//! Write to a cuArray (texture data source) using surface writes
//! @param g_idata input data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
surfaceWriteKernel( float* g_idata, int width, int height) 
{
    // calculate surface coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // read from global memory and write to cuarray (via surface reference)
    surf2Dwrite(g_idata[y*width+x], output_surface, x*4, y, hipBoundaryModeTrap);
}

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
transformKernel( float* g_odata, int width, int height, float theta) 
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = x / (float) width;
    float v = y / (float) height;

    // transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u*cosf(theta) - v*sinf(theta) + 0.5f;
    float tv = v*cosf(theta) + u*sinf(theta) + 0.5f;

    // read from texture and write to global memory
    g_odata[y*width + x] = tex2D(tex, tu, tv);
}

#endif // #ifndef _SIMPLETEXTURE_KERNEL_H_
