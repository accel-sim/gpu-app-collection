#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication as described in Chapter 3
 * of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 * See also:
 * V. Volkov and J. Demmel, "Benchmarking GPUs to tune dense linear algebra,"
 * in Proc. 2008 ACM/IEEE Conf. on Superconducting (SC '08),
 * Piscataway, NJ: IEEE Press, 2008, pp. Art. 31:1-11. 
 *
 */

// Utilities and system includes
#include <hipblas.h>
#include <sdkHelper.h>  // helper for shared functions common to CUDA SDK samples
#include <shrQATest.h>
#include <shrUtils.h>

#include <hip/hip_runtime.h>

#include "matrixMul.h"

// includes, kernels
#include <matrixMul_kernel.cu>

static char *sSDKsample = "matrixMul";

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

void inline checkError(hipblasStatus_t status, const char* msg)
{
    if(status != HIPBLAS_STATUS_SUCCESS){
        printf(msg);
        exit(-1);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
    shrQAStart(argc, argv);
	printf("[ %s ]\n", sSDKsample);

    //shrSetLogFileName ("matrixMul.txt");
    shrLog("%s\n\tStarting (CUDA and CUBLAS tests)...\n\n", argv[0]);

    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv)
{
    if(checkCmdLineFlag(argc, (const char**)argv, "device"))
    {
        int devID = getCmdLineArgumentInt(argc, (const char **)argv, "device=");
        if (devID < 0) {
            printf("Invalid command line parameters\n");
            exit(-1);
        } else {
            devID = gpuDeviceInit(devID);
            if (devID < 0) {
               printf("exiting...\n");
               shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
               exit(-1);
            }
        }
    }
    else
    {
        checkCudaErrors( hipSetDevice(gpuGetMaxGflopsDeviceId()) );
    }

    int devID;
    hipDeviceProp_t props;

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    // use a larger block size for Fermi and above
    int block_size = (props.major < 2) ? 16 : 32;

    printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);

	// set seed for rand()
    srand(2006);

    // Optional Command-line multiplier for matrix sizes
    unsigned int uiWA, uiHA, uiWB, uiHB, uiWC, uiHC;
    int iSizeMultiple = 5;
    if (checkCmdLineFlag( argc, (const char **)argv, "sizemult" )) {
        iSizeMultiple = getCmdLineArgumentInt(argc, (const char**)argv, "sizemult"); 
    }
    iSizeMultiple = CLAMP(iSizeMultiple, 1, 10);

    bool useCublasOnly = false;
    if(checkCmdLineFlag(argc, (const char**)argv, "cublas"))
        useCublasOnly = true;

	// For GPUs with fewer # of SM's, we limit the maximum size of the matrix
	if (props.multiProcessorCount <= 4) {
		uiWA = 2 * block_size * iSizeMultiple;
		uiHA = 4 * block_size * iSizeMultiple;
		uiWB = 2 * block_size * iSizeMultiple;
		uiHB = 4 * block_size * iSizeMultiple;
		uiWC = 2 * block_size * iSizeMultiple;
		uiHC = 4 * block_size * iSizeMultiple;
	} else {
		uiWA = WA * iSizeMultiple;
		uiHA = HA * iSizeMultiple;
		uiWB = WB * iSizeMultiple;
		uiHB = HB * iSizeMultiple;
		uiWC = WC * iSizeMultiple;
		uiHC = HC * iSizeMultiple;
	}
    shrLog("\nUsing Matrix Sizes: A(%u x %u), B(%u x %u), C(%u x %u)\n\n", 
            uiWA, uiHA, uiWB, uiHB, uiWC, uiHC);

    // allocate host memory for matrices A and B
    unsigned int size_A = uiWA * uiHA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)malloc(mem_size_A);
    unsigned int size_B = uiWB * uiHB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A, *d_B, *d_C;
    unsigned int size_C = uiWC * uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;

    // allocate host memory for the result
    float* h_C      = (float*) malloc(mem_size_C);
	float* h_CUBLAS = (float*) malloc(mem_size_C);

    checkCudaErrors(hipMalloc((void**) &d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void**) &d_B, mem_size_B));

    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) );
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) );
    
    checkCudaErrors(hipMalloc((void**) &d_C, mem_size_C));
   
    // setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(uiWC / threads.x, uiHC / threads.y);

    // kernel warmup
    if(useCublasOnly) {
	} else {
    }
    
    // create and start timer
    shrLog("Runing Kernels...\n\n");

	StopWatchInterface * timer_cublas;
    StopWatchInterface * timer_matrixMul;

    // execute the kernel
    int nIter = 30;

	// CUBLAS version 2.0
	{
        hipblasHandle_t handle;
        checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");
        const float alpha = 1.0f;
        const float beta = 0.0f;
        //Perform warmup operation with cublas
        hipblasStatus_t ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);
        checkError(ret, "cublas Sgemm returned an error!\n");

		// Start Timing
		sdkCreateTimer(&timer_cublas);
		sdkStartTimer(&timer_cublas);
        for (int j = 0; j < nIter; j++) {
            //note cublas is column primary!
            //need to transpose the order
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, uiWB, uiHA, uiWA, &alpha, d_B, uiWB, d_A, uiWA, &beta, d_C, uiWA);
		}
		// check if kernel execution generated and error
		getLastCudaError("CUBLAS Kernel execution failed");
		hipDeviceSynchronize();
		// stop and destroy timer
		sdkStopTimer(&timer_cublas);

		double dSeconds = sdkGetTimerValue(&timer_cublas)/((double)nIter * 1000.0);
		double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
		double gflops = 1.0e-9 * dNumOps/dSeconds;

		//Log througput, etc
		shrLogEx(LOGBOTH | MASTER, 0, "> CUBLAS         %.4f GFlop/s, Time = %.5f s, Size = %.0f Ops\n\n", 
				gflops, dSeconds, dNumOps);

		sdkDeleteTimer(&timer_cublas);

		// copy result from device to host
		checkCudaErrors(hipMemcpy(h_CUBLAS, d_C, mem_size_C, hipMemcpyDeviceToHost) );

        checkError(hipblasDestroy(handle), "hipblasDestroy() error!\n");
	}

	// For the case where "-cublas" is not specified, we will run the matrixMul kernel
	if (!useCublasOnly) 
	{
        //Performs warmup operation using matrixMul CUDA kernel
		if (block_size == 16) {
            matrixMul<16><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
        } else {
            matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
        }
        hipDeviceSynchronize();

		// Start Timing	
		sdkCreateTimer(&timer_matrixMul);
		sdkStartTimer(&timer_matrixMul);
		for (int j = 0; j < nIter; j++) {
			if (block_size == 16) {
				matrixMul<16><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
			} else {
				matrixMul<32><<< grid, threads >>>(d_C, d_A, d_B, uiWA, uiWB);
			}
		}
		// check if kernel execution generated and error
		getLastCudaError("CUDA matrixMul Kernel execution failed");

        hipDeviceSynchronize();
		// stop and destroy timer
		sdkStopTimer(&timer_matrixMul);

		double dSeconds = sdkGetTimerValue(&timer_matrixMul)/((double)nIter * 1000.0);
		double dNumOps = 2.0 * (double)uiWA * (double)uiHA * (double)uiWB;
		double gflops = 1.0e-9 * dNumOps/dSeconds;

		//Log througput, etc
		shrLogEx(LOGBOTH | MASTER, 0, "> CUDA matrixMul %.4f GFlop/s, Time = %.5f s, Size = %.0f Ops, ", 
				gflops, dSeconds, dNumOps);
		shrLogEx(LOGBOTH | MASTER, 0, "NumDevsUsed = %d, Workgroup = %u\n", 1, threads.x * threads.y);

		sdkDeleteTimer(&timer_matrixMul);

		// copy result from device to host
		checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) );
	}

    // compute reference solution
    shrLog("\nComparing GPU results with Host computation...\n\n");    
    float* reference = (float*)malloc(mem_size_C);
    computeGold(reference, h_A, h_B, uiHA, uiWA, uiWB);

    // check result (CUBLAS)
	printf("Comparing CUBLAS & Host results\n");
    bool resCUBLAS = sdkCompareL2fe(reference, h_CUBLAS, size_C, 1.0e-6f);
    if (resCUBLAS != true) 
    {
        printDiff(reference, h_CUBLAS, uiWC, uiHC, 100, 1.0e-5f);
    }
    shrLog("CUBLAS compares %s\n\n", (true == resCUBLAS) ? "OK" : "FAIL");

    // check result (matrixMul)
	printf("Comparing CUDA matrixMul & Host results\n");
    bool resCUDA = sdkCompareL2fe(reference, h_C, size_C, 1.0e-6f);
    if (resCUDA != true) 
    {
        printDiff(reference, h_C, uiWC, uiHC, 100, 1.0e-5f);
    }
    shrLog("CUDA matrixMul compares %s\n\n", (true == resCUDA) ? "OK" : "FAIL");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (resCUDA == true && resCUBLAS == true) ? QA_PASSED : QA_FAILED);
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    shrLog("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int i,j,k;
    int error_count=0;
    for (j = 0; j < height; j++) 
    {
        if (error_count < iListLength)
        {
            shrLog("\n  Row %d:\n", j);
        }
        for (i = 0; i < width; i++) 
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]);
            if (fDiff > fListTol) 
            {                
                if (error_count < iListLength)
                {
                    shrLog("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    shrLog(" \n  Total Errors = %d\n\n", error_count);
}
