#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
#include <stdio.h>
#include "cutil_inline.h"
#include "Mandelbrot_kernel.h"
#include "Mandelbrot_kernel.cu"

// The Mandelbrot CUDA GPU thread function

/*
    Version using software scheduling of thread blocks.
    
    The idea here is to launch of fixed number of worker blocks to fill the
    machine, and have each block loop over the available work until it is all done.
    
    We use a counter in global memory to keep track of which blocks have been
    completed. The counter is incremented atomically by each worker block.
    
    This method can achieve higher performance when blocks take a wide range of
    different times to complete.
*/

__device__ unsigned int blockCounter;   // global counter, initialized to zero before kernel launch

template<class T>
__global__ void Mandelbrot0_sm11(uchar4 *dst, const int imageW, const int imageH, const int crunch, const T xOff, const T yOff, 
								 const T xJP, const T yJP, const T scale, const uchar4 colors, const int frame, 
								 const int animationFrame, const int gridWidth, const int numBlocks, const bool isJ)
{
    __shared__ unsigned int blockIndex;
    __shared__ unsigned int blockX, blockY;
    
    // loop until all blocks completed
    while(1) {
        if ((threadIdx.x==0) && (threadIdx.y==0)) {
            // get block to process
            blockIndex = atomicAdd(&blockCounter, 1);
            blockX = blockIndex % gridWidth;            // note: this is slow, but only called once per block here
            blockY = blockIndex / gridWidth;
        }
        __syncthreads();
  
        if (blockIndex >= numBlocks) break;  // finish

        // process this block
        const int ix = blockDim.x * blockX + threadIdx.x;
        const int iy = blockDim.y * blockY + threadIdx.y;

        if ((ix < imageW) && (iy < imageH)) {
		    // Calculate the location
		    const T xPos = (T)ix * scale + xOff;
		    const T yPos = (T)iy * scale + yOff;
    		
            // Calculate the Mandelbrot index for the current location
            int m = CalcMandelbrot<T>(xPos, yPos, xJP, yJP, crunch, isJ);
//            int m = blockIdx.x;         // uncomment to see scheduling order
            m = m > 0 ? crunch - m : 0;
    			
            // Convert the Mandelbrot index into a color
            uchar4 color;
            if (m) {
			    m += animationFrame;
			    color.x = m * colors.x;
			    color.y = m * colors.y;
			    color.z = m * colors.z;
		    } else {
			    color.x = 0;
			    color.y = 0;
			    color.z = 0;
		    }
    		
            // Output the pixel
 		    int pixel = imageW * iy + ix;
            if (frame == 0) {
			    color.w = 0;
			    dst[pixel] = color;
            } else {
			    int frame1 = frame + 1;
			    int frame2 = frame1 / 2;
			    dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
			    dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
			    dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
            }
        }

    }
    
} // Mandelbrot0


// The Mandelbrot CUDA GPU thread function (double single version)
__global__ void MandelbrotDS0_sm11(uchar4 *dst, const int imageW, const int imageH, const int crunch, const float xOff0, const float xOff1, 
								   const float yOff0, const float yOff1, const float xJP, const float yJP, const float scale, 
								   const uchar4 colors, const int frame, const int animationFrame, const int gridWidth, 
								   const int numBlocks, const bool isJ)
{
    __shared__ unsigned int blockIndex;
    __shared__ unsigned int blockX, blockY;
    
    // loop until all blocks completed
    while(1) {
        if ((threadIdx.x==0) && (threadIdx.y==0)) {
            // get block to process
            blockIndex = atomicAdd(&blockCounter, 1);
//			printf("add %i %i \n", blockIdx.x, blockIdx.y) ; 
            blockX = blockIndex % gridWidth;            // note: this is slow, but only called once per block here
            blockY = blockIndex / gridWidth;
        }
        __syncthreads();
        
        if (blockIndex >= numBlocks) break;  // finish

    // printf("run %i %i \n", blockX, blockY ) ; 
        // process this block
        const int ix = blockDim.x * blockX + threadIdx.x;
        const int iy = blockDim.y * blockY + threadIdx.y;

        if ((ix < imageW) && (iy < imageH)) {
		    // Calculate the location
		    float xPos0 = (float)ix * scale;
		    float xPos1 = 0.0f;
		    float yPos0 = (float)iy * scale;
		    float yPos1 = 0.0f;
		    dsadd(xPos0, xPos1, xPos0, xPos1, xOff0, xOff1);
		    dsadd(yPos0, yPos1, yPos0, yPos1, yOff0, yOff1);

            // Calculate the Mandelbrot index for the current location
            int m = CalcMandelbrotDS(xPos0, xPos1, yPos0, yPos1, xJP, yJP, crunch, isJ);
            m = m > 0 ? crunch - m : 0;
    			
            // Convert the Mandelbrot index into a color
            uchar4 color;
            if (m) {
			    m += animationFrame;
			    color.x = m * colors.x;
			    color.y = m * colors.y;
			    color.z = m * colors.z;
		    } else {
			    color.x = 0;
			    color.y = 0;
			    color.z = 0;
		    }
    		
            // Output the pixel
 		    int pixel = imageW * iy + ix;
            if (frame == 0) {
			    color.w = 0;
			    dst[pixel] = color;
            } else {
			    int frame1 = frame + 1;
			    int frame2 = frame1 / 2;
			    dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
			    dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
			    dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
            }
        }
        
    }
} // MandelbrotDS0

// The Mandelbrot secondary AA pass CUDA GPU thread function
template<class T>
__global__ void Mandelbrot1_sm11(uchar4 *dst, const int imageW, const int imageH, const int crunch, const T xOff, const T yOff, 
								 const T xJP, const T yJP, const T scale, const uchar4 colors, const int frame, 
								 const int animationFrame, const int gridWidth, const int numBlocks, const bool isJ)
{
    __shared__ unsigned int blockIndex;
    __shared__ unsigned int blockX, blockY;
    
    // loop until all blocks completed
    while(1) {
        if ((threadIdx.x==0) && (threadIdx.y==0)) {
            // get block to process
            blockIndex = atomicAdd(&blockCounter, 1);
            blockX = blockIndex % gridWidth;            // note: this is slow, but only called once per block here
            blockY = blockIndex / gridWidth;
        }
        __syncthreads();
        
        if (blockIndex >= numBlocks) break;  // finish

        // process this block
        const int ix = blockDim.x * blockX + threadIdx.x;
        const int iy = blockDim.y * blockY + threadIdx.y;

        if ((ix < imageW) && (iy < imageH)) {
		    // Get the current pixel color
 		    int pixel = imageW * iy + ix;
		    uchar4 pixelColor = dst[pixel];
		    int count = 0;
    		
		    // Search for pixels out of tolerance surrounding the current pixel
		    if (ix > 0)
			    count += CheckColors(pixelColor, dst[pixel - 1]);
		    if (ix + 1 < imageW)
			    count += CheckColors(pixelColor, dst[pixel + 1]);
		    if (iy > 0)
			    count += CheckColors(pixelColor, dst[pixel - imageW]);
		    if (iy + 1 < imageH)
			    count += CheckColors(pixelColor, dst[pixel + imageW]);
		    if (count) {
			    // Calculate the location
			    const T xPos = (T)ix * scale + xOff;
			    const T yPos = (T)iy * scale + yOff;
    			      
			    // Calculate the Mandelbrot index for the current location
			    int m = CalcMandelbrot(xPos, yPos, xJP, yJP, crunch, isJ);
			    m = m > 0 ? crunch - m : 0;
    	        
			    // Convert the Mandelbrot index into a color
			    uchar4 color;
			    if (m) {
				    m += animationFrame;
				    color.x = m * colors.x;
				    color.y = m * colors.y;
				    color.z = m * colors.z;
			    } else {
				    color.x = 0;
				    color.y = 0;
				    color.z = 0;
			    }
    			
			    // Output the pixel
			    int frame1 = frame + 1;
			    int frame2 = frame1 / 2;
			    dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
			    dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
			    dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
		    }
        }   
    }
    
} // Mandelbrot1

// The Mandelbrot secondary AA pass CUDA GPU thread function (double single version)
__global__ void MandelbrotDS1_sm11(uchar4 *dst, const int imageW, const int imageH, const int crunch, 
								   const float xOff0, const float xOff1, const float yOff0, const float yOff1, 
								   const float xJP, const float yJP, const float scale, const uchar4 colors, 
								   const int frame, const int animationFrame, const int gridWidth, const int numBlocks, const bool isJ)
{
    __shared__ unsigned int blockIndex;
    __shared__ unsigned int blockX, blockY;
    
    // loop until all blocks completed
    while(1) {
        if ((threadIdx.x==0) && (threadIdx.y==0)) {
            // get block to process
            blockIndex = atomicAdd(&blockCounter, 1);
            blockX = blockIndex % gridWidth;            // note: this is slow, but only called once per block here
            blockY = blockIndex / gridWidth;
        }
        __syncthreads();
        
        if (blockIndex >= numBlocks) break;  // finish

        // process this block
        const int ix = blockDim.x * blockX + threadIdx.x;
        const int iy = blockDim.y * blockY + threadIdx.y;

        if ((ix < imageW) && (iy < imageH)) {
		    // Get the current pixel color
 		    int pixel = imageW * iy + ix;
		    uchar4 pixelColor = dst[pixel];
		    int count = 0;
    		
		    // Search for pixels out of tolerance surrounding the current pixel
		    if (ix > 0)
			    count += CheckColors(pixelColor, dst[pixel - 1]);
		    if (ix + 1 < imageW)
			    count += CheckColors(pixelColor, dst[pixel + 1]);
		    if (iy > 0)
			    count += CheckColors(pixelColor, dst[pixel - imageW]);
		    if (iy + 1 < imageH)
			    count += CheckColors(pixelColor, dst[pixel + imageW]);
		    if (count) {
			    // Calculate the location
			    float xPos0 = (float)ix * scale;
			    float xPos1 = 0.0f;
			    float yPos0 = (float)iy * scale;
			    float yPos1 = 0.0f;
			    dsadd(xPos0, xPos1, xPos0, xPos1, xOff0, xOff1);
			    dsadd(yPos0, yPos1, yPos0, yPos1, yOff0, yOff1);
    			      
			    // Calculate the Mandelbrot index for the current location
			    int m = CalcMandelbrotDS(xPos0, xPos1, yPos0, yPos1, xJP, yJP, crunch, isJ);
			    m = m > 0 ? crunch - m : 0;
    	        
			    // Convert the Mandelbrot index into a color
			    uchar4 color;
			    if (m) {
				    m += animationFrame;
				    color.x = m * colors.x;
				    color.y = m * colors.y;
				    color.z = m * colors.z;
			    } else {
				    color.x = 0;
				    color.y = 0;
				    color.z = 0;
			    }
    			
			    // Output the pixel
			    int frame1 = frame + 1;
			    int frame2 = frame1 / 2;
			    dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
			    dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
			    dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
		    }
        }
    }
    
} // MandelbrotDS1

// The host CPU Mandebrot thread spawner
void RunMandelbrot0_sm11(uchar4 *dst, const int imageW, const int imageH, const int crunch, const double xOff, const double yOff, 
						 const double xjp, const double yjp, const double scale, const uchar4 colors, const int frame, 
						 const int animationFrame, const int mode, const int numSMs, const bool isJ)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    // zero block counter
    unsigned int hBlockCounter = 0;
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(blockCounter), &hBlockCounter, sizeof(unsigned int), 0, hipMemcpyHostToDevice ) );

    int numWorkerBlocks = numSMs;

	switch(mode) {
    default:
    case 0:
	    Mandelbrot0_sm11<float><<<numWorkerBlocks, threads>>>(dst, imageW, imageH, crunch, (float)xOff, (float)yOff, 
			(float)xjp, (float)yjp, (float)scale, colors, frame, animationFrame, grid.x, grid.x*grid.y, isJ);
        break;
    case 1:
		float x0, x1, y0, y1;
		dsdeq(x0, x1, xOff);
		dsdeq(y0, y1, yOff);
		MandelbrotDS0_sm11<<<numWorkerBlocks, threads>>>(dst, imageW, imageH, crunch, x0, x1, y0, y1, 
			xjp, yjp, (float)scale, colors, frame, animationFrame, grid.x, grid.x*grid.y, isJ);
        break;
    case 2:
		Mandelbrot0_sm11<double><<<numWorkerBlocks, threads>>>(dst, imageW, imageH, crunch, xOff, yOff, 
			xjp, yjp, scale, colors, frame, animationFrame, grid.x, grid.x*grid.y, isJ);
        break;
    }
    cutilCheckMsg("Mandelbrot0_sm11 kernel execution failed.\n");
} // RunMandelbrot0

// The host CPU Mandebrot thread spawner
void RunMandelbrot1_sm11(uchar4 *dst, const int imageW, const int imageH, const int crunch, const double xOff, const double yOff, 
						 const double xjp, const double yjp, const double scale, const uchar4 colors, const int frame, 
						 const int animationFrame, const int mode, const int numSMs, const bool isJ)
{
    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    // zero block counter
    unsigned int hBlockCounter = 0;
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(blockCounter), &hBlockCounter, sizeof(unsigned int), 0, hipMemcpyHostToDevice ) );

    int numWorkerBlocks = numSMs;

	switch(mode) {
    default:
    case 0:
		Mandelbrot1_sm11<float><<<numWorkerBlocks, threads>>>(dst, imageW, imageH, crunch, (float)xOff, (float)yOff, 
			(float)xjp, (float)yjp, (float)scale, colors, frame, animationFrame, grid.x, grid.x*grid.y, isJ);
        break;
    case 1:
		float x0, x1, y0, y1;
		dsdeq(x0, x1, xOff);
		dsdeq(y0, y1, yOff);
		MandelbrotDS1_sm11<<<numWorkerBlocks, threads>>>(dst, imageW, imageH, crunch, x0, x1, y0, y1, 
			xjp, yjp, (float)scale, colors, frame, animationFrame, grid.x, grid.x*grid.y, isJ);
        break;
    case 2:
		Mandelbrot1_sm11<double><<<numWorkerBlocks, threads>>>(dst, imageW, imageH, crunch, xOff, yOff, 
			xjp, yjp, scale, colors, frame, animationFrame, grid.x, grid.x*grid.y, isJ);
        break;
	}

    cutilCheckMsg("Mandelbrot1_sm11 kernel execution failed.\n");
} // RunMandelbrot1
