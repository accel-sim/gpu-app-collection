/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
 
/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)
#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, kernels
#include "simpleTemplates_kernel.cu"

int g_TotalFailures = 0;

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);        
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

// General GPU Device CUDA Initialization
int gpuDeviceInit(int devID)
{
    int deviceCount;
    checkCudaErrors(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
        exit(-1);
    }

    if (devID < 0)
       devID = 0;
        
    if (devID > deviceCount-1)
    {
        fprintf(stderr, "\n");
        fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
        fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
        fprintf(stderr, "\n");
        return -devID;
    }

    hipDeviceProp_t deviceProp;
    checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );

    if (deviceProp.major < 1)
    {
        fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
        exit(-1);                                                  
    }
    
    checkCudaErrors( hipSetDevice(devID) );
    printf("gpuDeviceInit() CUDA Device [%d]: \"%s\n", devID, deviceProp.name);

    return devID;
}

// This function returns the best GPU (with maximum GFLOPS)
int gpuGetMaxGflopsDeviceId()
{
    int current_device     = 0, sm_per_multiproc  = 0;
    int max_compute_perf   = 0, max_perf_device   = 0;
    int device_count       = 0, best_SM_arch      = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceCount( &device_count );
    
    // Find the best major SM Architecture GPU device
    while (current_device < device_count)
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major > 0 && deviceProp.major < 9999)
        {
            best_SM_arch = MAX(best_SM_arch, deviceProp.major);
        }
        current_device++;
    }

    // Find the best CUDA capable GPU device
    current_device = 0;
    while( current_device < device_count )
    {
        hipGetDeviceProperties( &deviceProp, current_device );
        if (deviceProp.major == 9999 && deviceProp.minor == 9999)
        {
            sm_per_multiproc = 1;
        }
        else
        {
            sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
        }
        
        int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
        
    if( compute_perf  > max_compute_perf )
    {
            // If we find GPU with SM major > 2, search only these
            if ( best_SM_arch > 2 )
            {
                // If our device==dest_SM_arch, choose this, or else pass
                if (deviceProp.major == best_SM_arch)
                {
                    max_compute_perf  = compute_perf;
                    max_perf_device   = current_device;
                 }
            }
            else
            {
                max_compute_perf  = compute_perf;
                max_perf_device   = current_device;
             }
        }
        ++current_device;
    }
    return max_perf_device;
}


// Initialization code to find the best CUDA Device
int findCudaDevice(int argc, const char **argv)
{
    hipDeviceProp_t deviceProp;
    int devID = 0;
    // If the command-line has a device number specified, use it
    if (checkCmdLineFlag(argc, argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, argv, "device=");
        if (devID < 0)
        {
            printf("Invalid command line parameter\n ");
            exit(-1);
        }
        else
        {
            devID = gpuDeviceInit(devID);
            if (devID < 0)
            {
                printf("exiting...\n");
                shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                exit(-1);
            }
        }
    }
    else
    {
        // Otherwise pick the device with highest Gflops/s
        devID = gpuGetMaxGflopsDeviceId();
        checkCudaErrors( hipSetDevice( devID ) );
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
    return devID;
}
// end of CUDA Helper Functions

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
template <class T>
void runTest( int argc, char** argv, int len);

template<class T>
void
computeGold( T* reference, T* idata, const unsigned int len) 
{
    const T T_len = static_cast<T>( len);
    for( unsigned int i = 0; i < len; ++i) 
    {
        reference[i] = idata[i] * T_len;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    shrQAStart(argc, argv);

    printf("> runTest<float,32>\n");
    runTest<float>( argc, argv, 32);
    printf("> runTest<int,64>\n");
    runTest<int>( argc, argv, 64);

    printf("\n[simpleTemplates] -> Test Results: %d Failures\n", g_TotalFailures);

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (g_TotalFailures == 0) ? QA_PASSED : QA_FAILED);
}

// To completely templatize runTest (below) with cutil, we need to use 
// template specialization to wrap up CUTIL's array comparison and file writing
// functions for different types.  

// Here's the generic wrapper for cutCompare*
template<class T>
class ArrayComparator
{
public:
    bool compare( const T* reference, T* data, unsigned int len)
    {
        fprintf(stderr, "Error: no comparison function implemented for this type\n");
        return false;
    }
};

// Here's the specialization for ints:
template<>
class ArrayComparator<int>
{
public:
    bool compare( const int* reference, int* data, unsigned int len)
    {
        return compareData(reference, data, len, 0.15f, 0.0f);
    }
};

// Here's the specialization for floats:
template<>
class ArrayComparator<float>
{
public:
    bool compare( const float* reference, float* data, unsigned int len)
    {
        return compareData(reference, data, len, 0.15f, 0.15f);
    }
};

// Here's the generic wrapper for cutWriteFile*
template<class T>
class ArrayFileWriter
{
public:
    bool write(const char* filename, T* data, unsigned int len, float epsilon)
    {
        fprintf(stderr, "Error: no file write function implemented for this type\n");
        return false;
    }
};

// Here's the specialization for ints:
template<>
class ArrayFileWriter<int>
{
public:
    bool write(const char* filename, int* data, unsigned int len, float epsilon)
    {
        return sdkWriteFile(filename, data, len, epsilon, false);
    }
};

// Here's the specialization for floats:
template<>
class ArrayFileWriter<float>
{
public:
    bool write(const char* filename, float* data, unsigned int len, float epsilon)
    {
        return sdkWriteFile(filename, data, len, epsilon, false);
    }
};


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
template<class T>
void
runTest( int argc, char** argv, int len) 
{
    int devID;
    hipDeviceProp_t deviceProps;

    devID = findCudaDevice(argc, (const char**)argv);
		
    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

    StopWatchInterface *timer = NULL;
    sdkCreateTimer( &timer );
    sdkStartTimer ( &timer );

    unsigned int num_threads = len;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    T* h_idata = (T*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (T) i;
    }

    // allocate device memory
    T* d_idata;
    checkCudaErrors( hipMalloc( (void**) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // allocate device memory for result
    T* d_odata;
    checkCudaErrors( hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
    testKernel<T><<< grid, threads, mem_size >>>( d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    T* h_odata = (T*) malloc( mem_size);
    // copy result from device to host
    checkCudaErrors( hipMemcpy( h_odata, d_odata, sizeof(T) * num_threads,
                                hipMemcpyDeviceToHost) );

    sdkStopTimer( &timer );
    printf( "Processing time: %f (ms)\n", sdkGetTimerValue( &timer ));
    sdkDeleteTimer( &timer );

    // compute reference solution
    T* reference = (T*) malloc( mem_size);
    computeGold<T>( reference, h_idata, num_threads);

    ArrayComparator<T> comparator;
    ArrayFileWriter<T> writer;

    // check result
    if( checkCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test
        writer.write( "./data/regression.dat", h_odata, num_threads, 0.0f );
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        bool res = comparator.compare( reference, h_odata, num_threads);
        printf( "Compare %s\n\n", (1 == res) ? "OK" : "MISMATCH");
        g_TotalFailures += (1 != res);
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    hipDeviceReset();
}
