#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* This sample is a templatized version of the template project.
 * It also shows how to correctly templatize dynamically allocated shared
 * memory arrays.
 * Device code.
 */

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "sharedmem.cuh"


////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template<class T>
__global__ void
testKernel( T* g_idata, T* g_odata) 
{
  // Shared mem size is determined by the host app at run time
  SharedMemory<T> smem;
  T* sdata = smem.getPointer();

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  sdata[tid] = g_idata[tid];
  __syncthreads();

  // perform some computations
  sdata[tid] = (T) num_threads * sdata[tid];
  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
