#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

// includes, kernels
#include "clock_kernel.cu"

// This example shows how to use the clock function to measure the performance of 
// a kernel accurately.
// 
// Blocks are executed in parallel and out of order. Since there's no synchronization
// mechanism between blocks, we measure the clock once for each block. The clock 
// samples are written to device memory.

#define NUM_BLOCKS    64
#define NUM_THREADS   256

// It's interesting to change the number of blocks and the number of threads to 
// understand how to keep the hardware busy.
//
// Here are some numbers I get on my G80:
//    blocks - clocks
//    1 - 3096
//    8 - 3232
//    16 - 3364
//    32 - 4615
//    64 - 9981
//
// With less than 16 blocks some of the multiprocessors of the device are idle. With
// more than 16 you are using all the multiprocessors, but there's only one block per
// multiprocessor and that doesn't allow you to hide the latency of the memory. With
// more than 32 the speed scales linearly.

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions

// Start the main SDK sample here
int main(int argc, char** argv)
{
    shrQAStart(argc, argv);

    // This will pick the best possible CUDA capable device
    int dev = findCudaDevice((const int)argc, (const char **)argv);

    float * dinput = NULL;
    float * doutput = NULL;
    clock_t * dtimer = NULL;

    clock_t timer[NUM_BLOCKS * 2];
    float input[NUM_THREADS * 2];

    for (int i = 0; i < NUM_THREADS * 2; i++)
    {
        input[i] = (float)i;
    }

    checkCudaErrors(hipMalloc((void**)&dinput, sizeof(float) * NUM_THREADS * 2));
    checkCudaErrors(hipMalloc((void**)&doutput, sizeof(float) * NUM_BLOCKS));
    checkCudaErrors(hipMalloc((void**)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

    checkCudaErrors(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2, hipMemcpyHostToDevice));

    timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS>>>(dinput, doutput, dtimer);

    //checkCudaErrors(hipMemcpy(output, doutput, sizeof(float) * NUM_BLOCKS, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(dinput));
    checkCudaErrors(hipFree(doutput));
    checkCudaErrors(hipFree(dtimer));


    // Compute the difference between the last block end and the first block start.
    clock_t minStart = timer[0];
    clock_t maxEnd = timer[NUM_BLOCKS];

    for (int i = 1; i < NUM_BLOCKS; i++)
    {
        minStart = timer[i] < minStart ? timer[i] : minStart;
        maxEnd = timer[NUM_BLOCKS+i] > maxEnd ? timer[NUM_BLOCKS+i] : maxEnd;
    }

    printf("time = %d\n", maxEnd - minStart);

    hipDeviceReset();

    // This test always passes.
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
}
