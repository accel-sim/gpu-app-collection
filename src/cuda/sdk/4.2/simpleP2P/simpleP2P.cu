#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates a combination of Peer-to-Peer (P2P) and 
 * Unified Virtual Address Space (UVA) features new to SDK 4.0
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

// CUDA includes
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions

__global__ void SimpleKernel(float *src, float *dst)
{
    // Just a dummy kernel, doing enough for us to verify that everything
    // worked
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * 2.0f;
}

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
#ifdef _WIN32
    return (bool)(pProp->tccDriver ? true : false);
#else
    return (bool)(pProp->major >= 2);
#endif
}

inline bool IsAppBuiltAs64()
{
#if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
    return 1;
#else
    return 0;
#endif
}

int main(int argc, char **argv)
{
    shrQAStart(argc, argv);

    if (!IsAppBuiltAs64()) {
        printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target.  Test is being waived.\n", argv[0]);
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
        exit(EXIT_SUCCESS);
    }

    // Number of GPUs
    printf("Checking for multiple GPUs...\n");
    int gpu_n;
    checkCudaErrors(hipGetDeviceCount(&gpu_n));
    printf("CUDA-capable device count: %i\n", gpu_n);
    if (gpu_n < 2)
    {
        printf("Two or more (SM 2.0) class GPUs are required for %s.\n", argv[0]);
        printf("Waiving test.\n");
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
        exit(EXIT_SUCCESS);
    }

    // Query device properties
    hipDeviceProp_t prop[64];
    int gpuid[64]; // we want to find the first two GPU's that can support P2P
    int gpu_count = 0;   // GPUs that meet the criteria

    for (int i=0; i < gpu_n; i++) {
        checkCudaErrors(hipGetDeviceProperties(&prop[i], i));
        // Only boards based on Fermi can support P2P
        if ((prop[i].major >= 2)
#ifdef _WIN32
            // on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled
             && prop[i].tccDriver
#endif
            ) 
        {
            // This is an array of P2P capable GPUs
            gpuid[gpu_count++] = i;
        }
        printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, prop[i].name, (IsGPUCapableP2P(&prop[i]) ? "IS " : "NOT"));
    }

    // Check for TCC for Windows
    if (gpu_count < 2)
    {
        printf("\nThis sample requires two SM 2.0 GPUs to use P2P/UVA functionality.\n");
#ifdef _WIN32
        printf("\nFor Windows Vista/Win7, a TCC driver must be installed and enabled to use P2P/UVA functionality.\n");
#endif
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
        exit(EXIT_SUCCESS);
    }

#if CUDART_VERSION >= 4000
    // Check possibility for peer access
    printf("\nChecking GPU(s) for support of peer to peer memory access...\n");
    int can_access_peer_0_1, can_access_peer_1_0;
    // In this case we just pick the first two that we can support
    checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer_0_1, gpuid[0], gpuid[1]));
    checkCudaErrors(hipDeviceCanAccessPeer(&can_access_peer_1_0, gpuid[1], gpuid[0]));

    // Output results from P2P capabilities
    printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[0]].name, gpuid[0], 
                                                                 prop[gpuid[1]].name, gpuid[1] ,
                                                                 can_access_peer_0_1 ? "Yes" : "No");
    printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[1]].name, gpuid[1],
                                                                 prop[gpuid[0]].name, gpuid[0],
                                                                 can_access_peer_1_0 ? "Yes" : "No");

    if (can_access_peer_0_1 == 0 || can_access_peer_1_0 == 0)
    {
        printf("Two or more SM 2.0 class GPUs are required for %s to run.\n", argv[0]);
        printf("Support for UVA requires a GPU with SM 2.0 capabilities.\n");
        printf("Peer to Peer access is not available between GPU%d <-> GPU%d, waiving test.\n", gpuid[0], gpuid[1]);
        printf("PASSED\n");
        exit(EXIT_SUCCESS);
    }

    // Enable peer access
    printf("Enabling peer access between GPU%d and GPU%d...\n", gpuid[0], gpuid[1]);
    checkCudaErrors(hipSetDevice(gpuid[0]));
    checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[1], 0));
    checkCudaErrors(hipSetDevice(gpuid[1]));
    checkCudaErrors(hipDeviceEnablePeerAccess(gpuid[0], 0));

    // Check that we got UVA on both devices
    printf("Checking GPU%d and GPU%d for UVA capabilities...\n", gpuid[0], gpuid[1]);
    const bool has_uva = (prop[gpuid[0]].unifiedAddressing && prop[gpuid[1]].unifiedAddressing);

    printf("> %s (GPU%d) supports UVA: %s\n", prop[gpuid[0]].name, gpuid[0], (prop[gpuid[0]].unifiedAddressing ? "Yes" : "No") );
    printf("> %s (GPU%d) supports UVA: %s\n", prop[gpuid[1]].name, gpuid[1], (prop[gpuid[1]].unifiedAddressing ? "Yes" : "No") );

    if (has_uva) {
        printf("Both GPUs can support UVA, enabling...\n");
    } else {
        printf("At least one of the two GPUs does NOT support UVA, waiving test.\n");
        printf("PASSED\n");
        exit(EXIT_SUCCESS);
    }

    // Allocate buffers
    const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
    printf("Allocating buffers (%iMB on GPU%d, GPU%d and CPU Host)...\n", int(buf_size / 1024 / 1024), gpuid[0], gpuid[1]);
    checkCudaErrors(hipSetDevice(gpuid[0]));
    float* g0;
    checkCudaErrors(hipMalloc(&g0, buf_size));
    checkCudaErrors(hipSetDevice(gpuid[1]));
    float* g1;
    checkCudaErrors(hipMalloc(&g1, buf_size));
    float* h0;
    checkCudaErrors(hipHostMalloc(&h0, buf_size)); // Automatically portable with UVA
        
    // Create CUDA event handles
    printf("Creating event handles...\n");
    hipEvent_t start_event, stop_event;
    float time_memcpy;
    int eventflags = hipEventBlockingSync;
    checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
    checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

    // P2P memcopy() benchmark
    checkCudaErrors(hipEventRecord(start_event, 0));
    for (int i=0; i<100; i++)
    {
        // With UVA we don't need to specify source and target devices, the
        // runtime figures this out by itself from the pointers
            
        // Ping-pong copy between GPUs
        if (i % 2 == 0)
            checkCudaErrors(hipMemcpy(g1, g0, buf_size, hipMemcpyDefault));
        else
            checkCudaErrors(hipMemcpy(g0, g1, buf_size, hipMemcpyDefault));
    }
    checkCudaErrors(hipEventRecord(stop_event, 0));
    checkCudaErrors(hipEventSynchronize(stop_event));
    checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
    printf("hipMemcpyPeer / hipMemcpy between GPU%d and GPU%d: %.2fGB/s\n", gpuid[0], gpuid[1],
        (1.0f / (time_memcpy / 1000.0f)) * ((100.0f * buf_size)) / 1024.0f / 1024.0f / 1024.0f);
 
    // Prepare host buffer and copy to GPU 0
    printf("Preparing host buffer and memcpy to GPU%d...\n", gpuid[0]);
    for (int i=0; i<buf_size / sizeof(float); i++)
    {
        h0[i] = float(i % 4096);
    }
    checkCudaErrors(hipSetDevice(gpuid[0]));
    checkCudaErrors(hipMemcpy(g0, h0, buf_size, hipMemcpyDefault));

    // Kernel launch configuration
    const dim3 threads(512, 1);
    const dim3 blocks((buf_size / sizeof(float)) / threads.x, 1);
 
    // Run kernel on GPU 1, reading input from the GPU 0 buffer, writing
    // output to the GPU 1 buffer
    printf("Run kernel on GPU%d, taking source data from GPU%d and writing to GPU%d...\n", 
            gpuid[1], gpuid[0], gpuid[1]);
    checkCudaErrors(hipSetDevice(gpuid[1]));
    SimpleKernel<<<blocks, threads>>> (g0, g1);

    checkCudaErrors( hipDeviceSynchronize() );

    // Run kernel on GPU 0, reading input from the GPU 1 buffer, writing
    // output to the GPU 0 buffer
    printf("Run kernel on GPU%d, taking source data from GPU%d and writing to GPU%d...\n", 
            gpuid[0], gpuid[1], gpuid[0]);
    checkCudaErrors(hipSetDevice(gpuid[0]));
    SimpleKernel<<<blocks, threads>>> (g1, g0);

    checkCudaErrors( hipDeviceSynchronize() );
 
    // Copy data back to host and verify
    printf("Copy data back to host from GPU%d and verify results...\n", gpuid[0]);
    checkCudaErrors(hipMemcpy(h0, g0, buf_size, hipMemcpyDefault));
 
    int error_count = 0;
    for (int i=0; i<buf_size / sizeof(float); i++)
    {
        // Re-generate input data and apply 2x '* 2.0f' computation of both
        // kernel runs
        if (h0[i] != float(i % 4096) * 2.0f * 2.0f)
        {
            printf("Verification error @ element %i: val = %f, ref = %f\n", i, h0[i], (float(i%4096)*2.0f*2.0f) );
            if (error_count++ > 10)
                break;
        }
    }
    // Disable peer access (also unregisters memory for non-UVA cases)
    printf("Enabling peer access...\n");
    checkCudaErrors(hipSetDevice(gpuid[0]));
    checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[1]));
    checkCudaErrors(hipSetDevice(gpuid[1]));
    checkCudaErrors(hipDeviceDisablePeerAccess(gpuid[0]));

    // Cleanup and shutdown
    printf("Shutting down...\n");
    checkCudaErrors(hipEventDestroy(start_event));
    checkCudaErrors(hipEventDestroy(stop_event));
    checkCudaErrors(hipSetDevice(gpuid[0]));
    checkCudaErrors(hipFree(g0));
    checkCudaErrors(hipSetDevice(gpuid[1]));
    checkCudaErrors(hipFree(g1));
    checkCudaErrors(hipHostFree(h0));

    for( int i=0; i<gpu_n; i++ ) {	
        checkCudaErrors( hipSetDevice(i) );
        hipDeviceReset();
    }

    shrQAFinishExit(argc, (const char **)argv, (error_count == 0) ? QA_PASSED : QA_FAILED);

#else // Using CUDA 3.2 or older
    printf("simpleP2P requires CUDA 4.0 to build and run, waiving testing\n");
    shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
#endif

}

