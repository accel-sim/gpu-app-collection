#include "hip/hip_runtime.h"
/*
* Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* This sample demonstrates how to use texture fetches from layered 2D textures in CUDA C
*
* This sample first generates a 3D input data array for the layered texture  
* and the expected output. Then it starts CUDA C kernels, one for each layer,
* which fetch their layer's texture data (using normalized texture coordinates) 
* transform it to the expected output, and write it to a 3D output data array.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

static char *sSDKname = "simpleLayeredTexture";

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
		    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
	    int current_device   = 0, sm_per_multiproc = 0;
	    int max_compute_perf = 0, max_perf_device  = 0;
	    int device_count     = 0, best_SM_arch     = 0;
	    hipDeviceProp_t deviceProp;

	    hipGetDeviceCount( &device_count );
	    // Find the best major SM Architecture GPU device
	    while ( current_device < device_count ) {
		    hipGetDeviceProperties( &deviceProp, current_device );
		    if (deviceProp.major > 0 && deviceProp.major < 9999) {
			    best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		    }
		    current_device++;
	    }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
		   } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
	    }
	    return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions


// includes, kernels
// declare texture reference for layered 2D float texture
// Note: The "dim" field in the texture reference template is now deprecated. 
// Instead, please use a texture type macro such as hipTextureType1D, etc.

texture<float, hipTextureType2DLayered> tex;

////////////////////////////////////////////////////////////////////////////////
//! Transform a layer of a layered 2D texture using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
transformKernel( float* g_odata, int width, int height, int layer) 
{
	// calculate this thread's data point
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// 0.5f offset and division are necessary to access the original data points
	// in the texture (such that bilinear interpolation will not be activated).
	// For details, see also CUDA Programming Guide, Appendix D 
	float u = (x+0.5f) / (float) width; 
	float v = (y+0.5f) / (float) height; 

	// read from texture, do expected transformation and write to global memory
	g_odata[layer*width*height + y*width + x] = -tex2DLayered(tex, u, v, layer) + layer;
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    shrQAStart(argc, argv);
	
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice((const int)argc, (const char **)argv);

	bool bResult = true;

    // get number of SMs on this GPU
    hipDeviceProp_t deviceProps;

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name, deviceProps.multiProcessorCount );
    printf("SM %d.%d\n", deviceProps.major, deviceProps.minor );

    if (deviceProps.major < 2) {
        printf("%s requires SM >= 2.0 to support Texture Arrays.  Test will exit... \n", sSDKname);
        hipDeviceReset();
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
    }

    // generate input data for layered texture
    unsigned int width=512, height=512, num_layers = 5;
    unsigned int size = width * height * num_layers * sizeof(float);
    float* h_data = (float*) malloc(size);
    for (unsigned int layer = 0; layer < num_layers; layer++)
	for (int i = 0; i < (int)(width * height); i++) 
            h_data[layer*width*height + i] = (float)i;    

    // this is the expected transformation of the input data (the expected output)
    float *h_data_ref = (float*) malloc(size);
    for (unsigned int layer = 0; layer < num_layers; layer++)
	for (int i = 0; i < (int)(width * height); i++) 
	    h_data_ref[layer*width*height + i] = -h_data[layer*width*height + i] + layer;

    // allocate device memory for result
    float* d_data = NULL;
    checkCudaErrors( hipMalloc( (void**) &d_data, size));

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_3darray;
    checkCudaErrors( hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered )); 
    hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0,0,0); 
    myparms.dstPos = make_hipPos(0,0,0); 
    myparms.srcPtr = make_hipPitchedPtr(h_data, width * sizeof(float), width, height); 
    myparms.dstArray = cu_3darray;
    myparms.extent = make_hipExtent(width, height, num_layers);
    myparms.kind = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D( &myparms));
    
    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;  // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors( hipBindTextureToArray( tex, cu_3darray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

    printf("Covering 2D data array of %d x %d: Grid size is %d x %d, each block has 8 x 8 threads\n", 
	   width, height, dimGrid.x, dimGrid.y);

    transformKernel<<< dimGrid, dimBlock >>>( d_data, width, height, 0); // warmup (for better timing)

    // check if kernel execution generated an error
    getLastCudaError("warmup Kernel execution failed");

    checkCudaErrors( hipDeviceSynchronize() );

    StopWatchInterface * timer;
    sdkCreateTimer( &timer );
    sdkStartTimer( &timer );

    // execute the kernel
    for (unsigned int layer = 0; layer < num_layers; layer++)
	transformKernel<<< dimGrid, dimBlock, 0 >>>(d_data, width, height, layer);

    // check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer( &timer );
    printf("Processing time: %.3f msec\n", sdkGetTimerValue( &timer));
    printf("%.2f Mtexlookups/sec\n", (width*height*num_layers / (sdkGetTimerValue( &timer) / 1000.0f) / 1e6));
    sdkDeleteTimer( &timer );

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( size);
    // copy result from device to host
    checkCudaErrors( hipMemcpy( h_odata, d_data, size, hipMemcpyDeviceToHost) );

    // write regression file if necessary
    if( checkCmdLineFlag( argc, (const char **)argv, "regression") ) {
        // write file for regression test
        sdkWriteFile<float>( "./data/regression.dat", h_odata, width*height, 0.0f, false);
    } 
    else 
    {
        printf("Comparing kernel output to expected data\n");

#define MIN_EPSILON_ERROR 5e-3f
        bResult = compareData( h_odata, h_data_ref, width*height*num_layers, MIN_EPSILON_ERROR, 0.0f);
    }

    // cleanup memory
    free(h_data);
    free(h_data_ref);
    free(h_odata);

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFreeArray(cu_3darray));

    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bResult ? QA_PASSED : QA_FAILED) );
}
