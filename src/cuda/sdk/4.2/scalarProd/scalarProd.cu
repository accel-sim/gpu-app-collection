#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample calculates scalar products of a 
 * given set of input vector pairs
 */



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <cutil_inline.h>
#include <shrQATest.h>



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProdCPU(
    float *h_C,
    float *h_A,
    float *h_B,
    int vectorN,
    int elementN
);



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cu"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high){
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////


// Parse program arguments
void ParseArguments(int argc, char** argv, int& N)
{
    for (int i = 0; i < argc; ++i) {
        if (strcmp(argv[i], "--size") == 0 ||
            strcmp(argv[i], "-size") == 0) 
        {
            N = atoi(argv[i+1]);
        }
    }
}


///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
    float *d_A, *d_B, *d_C;
    double delta, ref, sum_delta, sum_ref, L1norm;
    unsigned int hTimer;
    int i;

    //Total number of input vector pairs; arbitrary
    int VECTOR_N = 64;
    ParseArguments(argc,argv,VECTOR_N);

    //Number of elements per vector; arbitrary, 
    //but strongly preferred to be a multiple of warp size
    //to meet memory coalescing constraints
    const int ELEMENT_N = 1024;
    //Total number of data elements
    const int    DATA_N = VECTOR_N * ELEMENT_N;

    const int   DATA_SZ = DATA_N * sizeof(float);
    const int RESULT_SZ = VECTOR_N  * sizeof(float);

    shrQAStart(argc, argv);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice( cutGetMaxGflopsDeviceId() );

    cutilCheckError( cutCreateTimer(&hTimer) );

    printf("Initializing data...\n");
        printf("...allocating CPU memory.\n");
        h_A     = (float *)malloc(DATA_SZ);
        h_B     = (float *)malloc(DATA_SZ);
        h_C_CPU = (float *)malloc(RESULT_SZ);
        h_C_GPU = (float *)malloc(RESULT_SZ);

        printf("...allocating GPU memory.\n");
        cutilSafeCall( hipMalloc((void **)&d_A, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_B, DATA_SZ)   );
        cutilSafeCall( hipMalloc((void **)&d_C, RESULT_SZ) );

        printf("...generating input data in CPU mem.\n");
        srand(123);
        //Generating input data on CPU
        for(i = 0; i < DATA_N; i++){
            h_A[i] = RandFloat(0.0f, 1.0f);
            h_B[i] = RandFloat(0.0f, 1.0f);
        }

        printf("...copying input data to GPU mem.\n");
        //Copy options data to GPU memory for further processing 
        cutilSafeCall( hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice) );
        cutilSafeCall( hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice) );
    printf("Data init done.\n");


    printf("Executing GPU kernel...\n");
        cutilSafeCall( cutilDeviceSynchronize() );
        cutilCheckError( cutResetTimer(hTimer) );
        cutilCheckError( cutStartTimer(hTimer) );
        scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
        cutilCheckMsg("scalarProdGPU() execution failed\n");
        cutilSafeCall( cutilDeviceSynchronize() );
        cutilCheckError( cutStopTimer(hTimer) );
    printf("GPU time: %f msecs.\n", cutGetTimerValue(hTimer));

    printf("Reading back GPU result...\n");
        //Read back GPU results to compare them to CPU results
        cutilSafeCall( hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost) );


    printf("Checking GPU results...\n");
        printf("..running CPU scalar product calculation\n");
        scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

        printf("...comparing the results\n");
        //Calculate max absolute difference and L1 distance
        //between CPU and GPU results
        sum_delta = 0;
        sum_ref   = 0;
        for(i = 0; i < VECTOR_N; i++){
            delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
            ref   = h_C_CPU[i];
            sum_delta += delta;
            sum_ref   += ref;
        }
        L1norm = sum_delta / sum_ref;

    printf("Shutting down...\n");
        cutilSafeCall( hipFree(d_C) );
        cutilSafeCall( hipFree(d_B)   );
        cutilSafeCall( hipFree(d_A)   );
        free(h_C_GPU);
        free(h_C_CPU);
        free(h_B);
        free(h_A);
        cutilCheckError( cutDeleteTimer(hTimer) );

    cutilDeviceReset();
    printf("L1 error: %E\n", L1norm);
    shrQAFinishExit(argc, (const char **)argv, (L1norm < 1e-6) ? QA_PASSED : QA_FAILED);
}
