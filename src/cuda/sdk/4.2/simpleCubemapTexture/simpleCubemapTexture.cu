#include "hip/hip_runtime.h"
/*
* Copyright 1993-2011 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* This sample demonstrates how to use texture fetches from layered 2D textures in CUDA C
*
* This sample first generates a 3D input data array for the layered texture  
* and the expected output. Then it starts CUDA C kernels, one for each layer,
* which fetch their layer's texture data (using normalized texture coordinates) 
* transform it to the expected output, and write it to a 3D output data array.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <sdkHelper.h>  // helper for shared that are common to CUDA SDK samples
#include <shrQATest.h>  // This is for automated testing output (--qatest)

static char *sSDKname = "simpleCubemapTexture";

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

    // This will output the proper CUDA error strings in the event that a CUDA host call returns an error
    #define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

    inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
    {
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                    file, line, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // This will output the proper error string when calling hipGetLastError
    #define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

    inline void __getLastCudaError( const char *errorMessage, const char *file, const int line )
    {
        hipError_t err = hipGetLastError();
        if( hipSuccess != err) {
            fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
                    file, line, errorMessage, (int)err, hipGetErrorString( err ) );
            exit(-1);
        }
    }

    // General GPU Device CUDA Initialization
    int gpuDeviceInit(int devID)
    {
        int deviceCount;
        checkCudaErrors(hipGetDeviceCount(&deviceCount));
        if (deviceCount == 0) {
            fprintf(stderr, "gpuDeviceInit() CUDA error: no devices supporting CUDA.\n");
            exit(-1);
        }
        if (devID < 0) 
            devID = 0;
        if (devID > deviceCount-1) {
            fprintf(stderr, "\n");
            fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
            fprintf(stderr, ">> gpuDeviceInit (-device=%d) is not a valid GPU device. <<\n", devID);
            fprintf(stderr, "\n");
            return -devID;
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
        if (deviceProp.major < 1) {
            fprintf(stderr, "gpuDeviceInit(): GPU device does not support CUDA.\n");
            exit(-1);                                                  \
        }

        checkCudaErrors( hipSetDevice(devID) );
        printf("> gpuDeviceInit() CUDA device [%d]: %s\n", devID, deviceProp.name);
        return devID;
    }

    // This function returns the best GPU (with maximum GFLOPS)
    int gpuGetMaxGflopsDeviceId()
    {
        int current_device   = 0, sm_per_multiproc = 0;
        int max_compute_perf = 0, max_perf_device  = 0;
        int device_count     = 0, best_SM_arch     = 0;
        hipDeviceProp_t deviceProp;

        hipGetDeviceCount( &device_count );
        // Find the best major SM Architecture GPU device
        while ( current_device < device_count ) {
            hipGetDeviceProperties( &deviceProp, current_device );
            if (deviceProp.major > 0 && deviceProp.major < 9999) {
                best_SM_arch = MAX(best_SM_arch, deviceProp.major);
            }
            current_device++;
        }

        // Find the best CUDA capable GPU device
        current_device = 0;
        while( current_device < device_count ) {
           hipGetDeviceProperties( &deviceProp, current_device );
           if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
               sm_per_multiproc = 1;
           } else {
               sm_per_multiproc = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
           }

           int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;
           if( compute_perf  > max_compute_perf ) {
               // If we find GPU with SM major > 2, search only these
               if ( best_SM_arch > 2 ) {
                   // If our device==dest_SM_arch, choose this, or else pass
                   if (deviceProp.major == best_SM_arch) {	
                       max_compute_perf  = compute_perf;
                       max_perf_device   = current_device;
                   }
               } else {
                   max_compute_perf  = compute_perf;
                   max_perf_device   = current_device;
               }
           }
           ++current_device;
        }
        return max_perf_device;
    }

    // Initialization code to find the best CUDA Device
    int findCudaDevice(int argc, const char **argv)
    {
        hipDeviceProp_t deviceProp;
        int devID = 0;
        // If the command-line has a device number specified, use it
        if (checkCmdLineFlag(argc, argv, "device")) {
            devID = getCmdLineArgumentInt(argc, argv, "device=");
            if (devID < 0) {
                printf("Invalid command line parameters\n");
                exit(-1);
            } else {
                devID = gpuDeviceInit(devID);
                if (devID < 0) {
                   printf("exiting...\n");
                   shrQAFinishExit(argc, (const char **)argv, QA_FAILED);
                   exit(-1);
                }
            }
        } else {
            // Otherwise pick the device with highest Gflops/s
            devID = gpuGetMaxGflopsDeviceId();
            checkCudaErrors( hipSetDevice( devID ) );
            checkCudaErrors( hipGetDeviceProperties(&deviceProp, devID) );
            printf("> Using CUDA device [%d]: %s\n", devID, deviceProp.name);
        }
        return devID;
    }
// end of CUDA Helper Functions


// includes, kernels
// declare texture reference for layered 2D float texture
// Note: The "dim" field in the texture reference template is now deprecated. 
// Instead, please use a texture type macro such as hipTextureType1D, etc.

texture<float, hipTextureTypeCubemap> tex;

////////////////////////////////////////////////////////////////////////////////
//! Transform a cubemap face of a linear buffe using cubemap texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
transformKernel( float* g_odata, int width) 
{
    // calculate this thread's data point
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // 0.5f offset and division are necessary to access the original data points
    // in the texture (such that bilinear interpolation will not be activated).
    // For details, see also CUDA Programming Guide, Appendix D 

    float u = ( (x+0.5f) / (float) width ) * 2.f - 1.f; 
    float v = ( (y+0.5f) / (float) width ) * 2.f - 1.f;

    float cx, cy, cz;

    for ( unsigned int face = 0; face < 6; face ++ )
    {
        //Layer 0 is positive X face
        if ( face == 0 )
        {
            cx = 1;
            cy = -v;
            cz = -u;
        }
        //Layer 1 is negative X face
        else if ( face == 1 )
        {
            cx = -1;
            cy = -v;
            cz = u;
        }
        //Layer 2 is positive Y face
        else if ( face == 2 )
        {
            cx = u;
            cy = 1;
            cz = v;
        }
        //Layer 3 is negative Y face
        else if ( face == 3 )
        {
            cx = u;
            cy = -1;
            cz = -v;
        }
        //Layer 4 is positive Z face
        else if ( face == 4 )
        {
            cx = u;
            cy = -v;
            cz = 1;
        }
        //Layer 4 is negative Z face
        else if ( face == 5 )
        {
            cx = -u;
            cy = -v;
            cz = -1;
        }

        // read from texture, do expected transformation and write to global memory
        g_odata[face*width*width + y*width + x] = -texCubemap( tex, cx, cy, cz );
    }
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    shrQAStart(argc, argv);
    
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice((const int)argc, (const char **)argv);

    bool bResult = true;

    // get number of SMs on this GPU
    hipDeviceProp_t deviceProps;

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name, deviceProps.multiProcessorCount );
    printf("SM %d.%d\n", deviceProps.major, deviceProps.minor );

    if (deviceProps.major < 2) {
        printf("%s requires SM >= 2.0 to support Texture Arrays.  Test will exit... \n", sSDKname);
        hipDeviceReset();
        shrQAFinishExit(argc, (const char **)argv, QA_PASSED);
    }

    // generate input data for layered texture
    unsigned int width=64, num_faces = 6, num_layers = 1;
    unsigned int cubemap_size = width * width * num_faces;
    unsigned int size = cubemap_size * num_layers * sizeof(float);
    float* h_data = (float*) malloc(size);
    for (int i = 0; i < (int)(cubemap_size * num_layers); i++) 
            h_data[i] = (float)i;



    // this is the expected transformation of the input data (the expected output)
    float *h_data_ref = (float*) malloc(size);
    for (unsigned int layer = 0; layer < num_layers; layer++)
    {
        for (int i = 0; i < (int)(cubemap_size); i++) 
            h_data_ref[layer*cubemap_size + i] = -h_data[layer*cubemap_size + i] + layer;
    }

    // allocate device memory for result
    float* d_data = NULL;
    checkCudaErrors( hipMalloc( (void**) &d_data, size));

    // allocate array and copy image data
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* cu_3darray;
//    checkCudaErrors( hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, height, num_layers), hipArrayLayered )); 
    checkCudaErrors( hipMalloc3DArray( &cu_3darray, &channelDesc, make_hipExtent(width, width, num_faces), hipArrayCubemap )); 
    hipMemcpy3DParms myparms = {0};
    myparms.srcPos = make_hipPos(0,0,0); 
    myparms.dstPos = make_hipPos(0,0,0); 
    myparms.srcPtr = make_hipPitchedPtr(h_data, width * sizeof(float), width, width); 
    myparms.dstArray = cu_3darray;
    myparms.extent = make_hipExtent(width, width, num_faces);
    myparms.kind = hipMemcpyHostToDevice;
    checkCudaErrors( hipMemcpy3D( &myparms));
    
    // set texture parameters
    tex.addressMode[0] = hipAddressModeWrap;
    tex.addressMode[1] = hipAddressModeWrap;
    tex.filterMode = hipFilterModeLinear;
    tex.normalized = true;  // access with normalized texture coordinates

    // Bind the array to the texture
    checkCudaErrors( hipBindTextureToArray( tex, cu_3darray, channelDesc));

    dim3 dimBlock(8, 8, 1);
    dim3 dimGrid(width / dimBlock.x, width / dimBlock.y, 1);

    printf("Covering Cubemap data array of %d~3 x %d: Grid size is %d x %d, each block has 8 x 8 threads\n", 
        width, num_layers, dimGrid.x, dimGrid.y);

    transformKernel<<< dimGrid, dimBlock >>>( d_data, width); // warmup (for better timing)

    // check if kernel execution generated an error
    getLastCudaError("warmup Kernel execution failed");

    checkCudaErrors( hipDeviceSynchronize() );

    StopWatchInterface * timer;
    sdkCreateTimer( &timer );
    sdkStartTimer( &timer );

    // execute the kernel
    transformKernel<<< dimGrid, dimBlock, 0 >>>(d_data, width);

    // check if kernel execution generated an error
    getLastCudaError("Kernel execution failed");

    checkCudaErrors( hipDeviceSynchronize() );
    sdkStopTimer( &timer );
    printf("Processing time: %.3f msec\n", sdkGetTimerValue( &timer));
    printf("%.2f Mtexlookups/sec\n", (cubemap_size / (sdkGetTimerValue( &timer ) / 1000.0f) / 1e6));
    sdkDeleteTimer( &timer );

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( size);
    // copy result from device to host
    checkCudaErrors( hipMemcpy( h_odata, d_data, size, hipMemcpyDeviceToHost) );

    // write regression file if necessary
    if( checkCmdLineFlag( argc, (const char **)argv, "regression") ) {
        // write file for regression test
        sdkWriteFile<float>( "./data/regression.dat", h_odata, width*width, 0.0f, false);
    } 
    else 
    {
        printf("Comparing kernel output to expected data\n");

#define MIN_EPSILON_ERROR 5e-3f
        bResult = compareData( h_odata, h_data_ref, cubemap_size, MIN_EPSILON_ERROR, 0.0f);
    }

    // cleanup memory
    free(h_data);
    free(h_data_ref);
    free(h_odata);

    checkCudaErrors(hipFree(d_data));
    checkCudaErrors(hipFreeArray(cu_3darray));



    hipDeviceReset();
    shrQAFinishExit(argc, (const char **)argv, (bResult ? QA_PASSED : QA_FAILED) );
}
