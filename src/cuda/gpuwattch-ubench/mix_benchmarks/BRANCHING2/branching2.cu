#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
// Includes
#include <stdio.h>

// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

//NI DAQ
#include "../include/ContAcq-IntClk.h"

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 60
#define ITERATIONS REPLACE_ITERATIONS

// Variables

bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit_int(unsigned*, int);
void RandomInit_fp(float*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal1(unsigned *A, unsigned *B, int N)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int cta_id=blockDim.x * blockIdx.x;
    int offset=THREADS_PER_BLOCK/2;
    unsigned sum=0;
    if(id < N){
    	for(unsigned i=0; i<ITERATIONS; ++i){
    		A[id] = A[id] + B[id] + id;

    		//for(unsigned j=0; j<ITERATIONS/4; ++j){
    		sum += A[id];
    		sum += A[id+1];
    		sum += A[id+2];

		if(id>cta_id+offset){
			A[id+5]=sum;
			A[id+6]=sum;
			A[id+7]=sum;
    		        A[id+8]=sum;
    			A[id+9]=sum;
		}

                sum *= A[id+3];
                sum *= A[id+4];
    		sum *= A[id+10];

		if(id>cta_id+offset && id<cta_id+(offset+offset/2)){
			sum += A[id+11];
			A[id+12]=sum;				
			sum += A[id+13];
    			A[id+14]=sum;
    			sum += A[id+15];
		}
    		A[id] = sum+A[id]+B[id];
    	}
    }
}





__global__ void PowerKernalEmpty(unsigned* C, int N)
{
    unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    __syncthreads();
   // Excessive Mod/Div Operations
    for(unsigned long k=0; k<ITERATIONS*(blockDim.x + 299);k++) {
    	//Value1=(I1)+k;
        //Value2=(I2)+k;
        //Value3=(Value2)+k;
        //Value2=(Value1)+k;
    	/*
       	__asm volatile (
        			"B0: bra.uni B1;\n\t"
        			"B1: bra.uni B2;\n\t"
        			"B2: bra.uni B3;\n\t"
        			"B3: bra.uni B4;\n\t"
        			"B4: bra.uni B5;\n\t"
        			"B5: bra.uni B6;\n\t"
        			"B6: bra.uni B7;\n\t"
        			"B7: bra.uni B8;\n\t"
        			"B8: bra.uni B9;\n\t"
        			"B9: bra.uni B10;\n\t"
        			"B10: bra.uni B11;\n\t"
        			"B11: bra.uni B12;\n\t"
        			"B12: bra.uni B13;\n\t"
        			"B13: bra.uni B14;\n\t"
        			"B14: bra.uni B15;\n\t"
        			"B15: bra.uni B16;\n\t"
        			"B16: bra.uni B17;\n\t"
        			"B17: bra.uni B18;\n\t"
        			"B18: bra.uni B19;\n\t"
        			"B19: bra.uni B20;\n\t"
        			"B20: bra.uni B21;\n\t"
        			"B21: bra.uni B22;\n\t"
        			"B22: bra.uni B23;\n\t"
        			"B23: bra.uni B24;\n\t"
        			"B24: bra.uni B25;\n\t"
        			"B25: bra.uni B26;\n\t"
        			"B26: bra.uni B27;\n\t"
        			"B27: bra.uni B28;\n\t"
        			"B28: bra.uni B29;\n\t"
        			"B29: bra.uni B30;\n\t"
        			"B30: bra.uni B31;\n\t"
        			"B31: bra.uni LOOP;\n\t"
        			"LOOP:"
        			);
	*/
    }

    C[id]=id;
    __syncthreads();
}

// Host code
unsigned *h_A1, *h_A2, *h_A3;
unsigned *d_A1, *d_A2, *d_A3;

int main()
{
	 printf("Power Microbenchmarks\n");
	 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS*2;
	 
	 // Allocate input vectors h_A and h_B in host memory
 	 size_t size1 = N * sizeof(unsigned);
	 h_A1 = (unsigned*)malloc(size1);
	 if (h_A1 == 0) CleanupResources();

	 h_A2 = (unsigned*)malloc(size1);
	 if (h_A2 == 0) CleanupResources();



	 dim3 dimGrid2(1,1);
	 dim3 dimBlock2(1,1);

	 // Initialize input vectors
	 RandomInit_int(h_A1, N);
	 RandomInit_int(h_A2, N);



	 // Allocate vectors in device memory
	 checkCudaErrors( hipMalloc((void**)&d_A1, size1) );
	 checkCudaErrors( hipMalloc((void**)&d_A2, size1) );


	 // Copy vectors from host memory to device memory
	 checkCudaErrors( hipMemcpy(d_A1, h_A1, size1, hipMemcpyHostToDevice) );
	 checkCudaErrors( hipMemcpy(d_A2, h_A2, size1, hipMemcpyHostToDevice) );

	 dim3 dimGrid(NUM_OF_BLOCKS,1);
	 dim3 dimBlock(THREADS_PER_BLOCK,1);


	CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
	TaskHandle taskhandle = LaunchDAQ();
	CUT_SAFE_CALL(cutStartTimer(my_timer)); 

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	 //PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A3, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));

	 PowerKernal1<<<dimGrid,dimBlock>>>(d_A1, d_A2, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));

	 //PowerKernalEmpty<<<dimGrid2,dimBlock2>>>(d_A3, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	 getLastCudaError("kernel launch failure");

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	CUT_SAFE_CALL(cutStopTimer(my_timer));
	TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 

	#ifdef _DEBUG
	 checkCudaErrors( hipDeviceSynchronize() );
	#endif

	 // Copy result from device memory to host memory

	 CleanupResources();

	 return 0;
}

void CleanupResources(void)
{
	  // Free device memory
	  if (d_A1)
		hipFree(d_A1);
	  if (d_A2)
		hipFree(d_A2);
	  if (d_A3)
		hipFree(d_A3);
	  // Free host memory
	  if (h_A1)
		free(h_A1);
	  if (h_A2)
		free(h_A2);
	  if (h_A3)
		free(h_A3);
}

// Allocates an array with random float entries.
void RandomInit_int(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
	srand((unsigned)time(0));  
	data[i] = rand() / RAND_MAX;
  }
}

void RandomInit_fp(float* data, int n)
{
   for (int i = 0; i < n; ++i){
	data[i] = rand() / RAND_MAX;
   }
}






