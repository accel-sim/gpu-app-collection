#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <math.h>
// Includes
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>
#include "../include/ContAcq-IntClk.h"

// includes CUDA
#include <hip/hip_runtime.h>

#define MAX_THREADS_PER_BLOCK 256
#define THREADS_PER_BLOCK 256
#define LINE_SIZE 	128
#define SETS		4
#define ASSOC		24
#define SIMD_WIDTH	32

// Variables
int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(int*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line ){
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line ){
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
#define ITERATIONS REPLACE_ITERATIONS

texture<float,1,hipReadModeElementType> texmem1;
texture<float,1,hipReadModeElementType> texmem2;
texture<float,1,hipReadModeElementType> texmem3;
texture<float,1,hipReadModeElementType> texmem4;
texture<float,1,hipReadModeElementType> texmem5;
texture<float,1,hipReadModeElementType> texmem6;
texture<float,1,hipReadModeElementType> texmem7;
texture<float,1,hipReadModeElementType> texmem9;
texture<float,1,hipReadModeElementType> texmem8;
__constant__ float ConstArray1[THREADS_PER_BLOCK];
__constant__ float ConstArray2[THREADS_PER_BLOCK];
__constant__ float ConstArray3[THREADS_PER_BLOCK];
__constant__ float ConstArray4[THREADS_PER_BLOCK];
__constant__ float ConstArray5[THREADS_PER_BLOCK];
__constant__ float ConstArray6[THREADS_PER_BLOCK];
__constant__ float ConstArray7[THREADS_PER_BLOCK];
__constant__ float ConstArray8[THREADS_PER_BLOCK];


__global__ void tex_bm_kernel( float* out, unsigned size)
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	__device__ __shared__ float I1[THREADS_PER_BLOCK];
    __device__  __shared__ float I2[THREADS_PER_BLOCK];
    __device__  __shared__ float I3[THREADS_PER_BLOCK];
    __device__  __shared__ float I4[THREADS_PER_BLOCK];
    __device__  __shared__ float I5[THREADS_PER_BLOCK];
    __device__  __shared__ float I6[THREADS_PER_BLOCK];
    __device__  __shared__ float I7[THREADS_PER_BLOCK];
    __device__  __shared__ float I8[THREADS_PER_BLOCK];

	I1[tid%THREADS_PER_BLOCK] = tid;
	I2[tid%THREADS_PER_BLOCK] = tid/2;
	I3[tid%THREADS_PER_BLOCK] = 2*tid;
	I4[tid%THREADS_PER_BLOCK] = tid+2;
	I5[tid%THREADS_PER_BLOCK] = 5*tid;
	I6[tid%THREADS_PER_BLOCK] = tid/2;
	I7[tid%THREADS_PER_BLOCK] = tid*10;
	I8[tid%THREADS_PER_BLOCK] = tid/2;
	
	if(tid < size){
		for(unsigned i=0; i<ITERATIONS; ++i){
			out[tid] = tex1Dfetch(texmem1,tid);
			out[tid*2] = ConstArray1[(tid+i)%THREADS_PER_BLOCK]+out[tid];
			out[tid*3] =  I1[(tid+i)%THREADS_PER_BLOCK]+out[tid*2];
			out[tid*4] = tex1Dfetch(texmem4,tid)*out[tid*3];
			out[tid*5] = ConstArray2[(tid+i)%THREADS_PER_BLOCK]+out[tid*4];
			out[tid*6] = tex1Dfetch(texmem6,tid)/out[tid*5];
			out[tid*7] = I3[tid%THREADS_PER_BLOCK]+out[tid*6];
			out[tid*8] = tex1Dfetch(texmem8,tid)-out[tid*7];
			out[tid*9] = ConstArray3[(tid+i)%THREADS_PER_BLOCK]*out[tid*8];
		}
	}

}


////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	
	 float array1[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array1[i] = rand() / RAND_MAX;
	 }
	 float array2[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array2[i] = rand() / RAND_MAX;
	 }
	 float array3[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array3[i] = rand() / RAND_MAX;
	 }
	 float array4[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array4[i] = rand() / RAND_MAX;
	 }
	 float array5[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array5[i] = rand() / RAND_MAX;
	 }
	 float array6[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array6[i] = rand() / RAND_MAX;
	 }
	 float array7[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array7[i] = rand() / RAND_MAX;
	 }
	 float array8[THREADS_PER_BLOCK];
	 for(int i=0; i<THREADS_PER_BLOCK;i++){
		srand(time(0));
		array8[i] = rand() / RAND_MAX;
	 }
	 
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray1"), array1, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray2"), array2, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray3"), array3, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray4"), array4, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray5"), array5, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray6"), array6, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray7"), array7, sizeof(float) * THREADS_PER_BLOCK );
	 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray8"), array8, sizeof(float) * THREADS_PER_BLOCK );
	 
	
	int texmem_size = LINE_SIZE*SETS*ASSOC;	
	
	float *host_texture1 = (float*) malloc(texmem_size*sizeof(float));
	for (int i=0; i< texmem_size; i++) {
		host_texture1[i] = i;
	}
	float *device_texture1;
	float *device_texture2;
	float *device_texture3;
	float *device_texture4;
	float *device_texture5;
	float *device_texture6;
	float *device_texture7;
	float *device_texture8;
	float *device_texture9;

	float *host_out = (float*) malloc(texmem_size*sizeof(float)*10);
	float *device_out;

	hipMalloc((void**) &device_texture1, texmem_size);
	hipMalloc((void**) &device_texture2, texmem_size);
	hipMalloc((void**) &device_texture3, texmem_size);
	hipMalloc((void**) &device_texture4, texmem_size);
	hipMalloc((void**) &device_texture5, texmem_size);
	hipMalloc((void**) &device_texture6, texmem_size);
	hipMalloc((void**) &device_texture7, texmem_size);
	hipMalloc((void**) &device_texture8, texmem_size);
	hipMalloc((void**) &device_texture9, texmem_size);

	hipMalloc((void**) &device_out, texmem_size*10);

	hipMemcpy(device_texture1, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture2, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture3, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture4, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture5, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture6, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture7, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture8, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_texture9, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);

	hipBindTexture(0, texmem1, device_texture1, texmem_size);
	hipBindTexture(0, texmem2, device_texture2, texmem_size);
	hipBindTexture(0, texmem3, device_texture3, texmem_size);
	hipBindTexture(0, texmem4, device_texture4, texmem_size);
	hipBindTexture(0, texmem5, device_texture5, texmem_size);
	hipBindTexture(0, texmem6, device_texture6, texmem_size);
	hipBindTexture(0, texmem7, device_texture7, texmem_size);
	hipBindTexture(0, texmem8, device_texture8, texmem_size);
	hipBindTexture(0, texmem9, device_texture9, texmem_size);


	unsigned num_blocks = (texmem_size / MAX_THREADS_PER_BLOCK) + 1;
	dim3  grid( num_blocks, 1, 1);
	dim3  threads( MAX_THREADS_PER_BLOCK, 1, 1);

	CUT_SAFE_CALL(cutCreateTimer(&my_timer));
	TaskHandle taskhandle = LaunchDAQ();
	CUT_SAFE_CALL(cutStartTimer(my_timer));

	tex_bm_kernel<<< grid, threads, 0 >>>(device_out, texmem_size);
	hipDeviceSynchronize();

	CUT_SAFE_CALL(cutStopTimer(my_timer));
	TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	CUT_SAFE_CALL(cutDeleteTimer(my_timer));


	printf("Kernel DONE, probably correctly\n");
	hipMemcpy(host_out, device_out, texmem_size*sizeof(float), hipMemcpyDeviceToHost);

	/*
	printf("Output: ");
	float error = false;
	for (int i=0; i< texmem_size; i++){
		printf("%.1f ", host_out[i]);
		if (host_out[i] - i > 0.0001) error = true;
	}
	printf("\n");
	if (error) printf("\nFAILED\n");
	else printf("\nPASSED\n");
	*/
}

void CleanupResources(void){
  // Free device memory


}

// Allocates an array with random float entries.
void RandomInit(int* data, int n){
  for (int i = 0; i < n; ++i)
	data[i] = (int)(rand() / RAND_MAX);
}






