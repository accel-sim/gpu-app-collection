#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
// Includes
#include <stdio.h>
#include "../include/ContAcq-IntClk.h"

// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 60
#define ITERATIONS REPLACE_ITERATIONS

// Variables

__constant__ float ConstArray1[THREADS_PER_BLOCK];
__constant__ float ConstArray2[THREADS_PER_BLOCK];
__constant__ float ConstArray3[THREADS_PER_BLOCK];

float* h_Value;
float* d_Value;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(float* Value)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    
    //Do Some Computation
    float Value1;
    float Value2;
    float Value3;
    *Value=1;
    for(unsigned k=0; k<ITERATIONS;k++) {
    	Value1=ConstArray1[(i+k)%THREADS_PER_BLOCK];
    	Value2=ConstArray2[(i+k+1)%THREADS_PER_BLOCK];
    	Value3=ConstArray3[(i+k+5)%THREADS_PER_BLOCK];
    	Value2*=Value1;
    	Value3*=Value2;
    	*Value*=Value3;
    }		
     __syncthreads();
    
}


// Host code

int main()
{
 printf("Power Microbenchmarks\n");
 float array1[THREADS_PER_BLOCK];
 h_Value = (float *) malloc(sizeof(float));
 for(int i=0; i<THREADS_PER_BLOCK;i++){
	srand(time(0));
	array1[i] = rand() / RAND_MAX;
 }
 float array2[THREADS_PER_BLOCK];
 for(int i=0; i<THREADS_PER_BLOCK;i++){
	srand(time(0));
	array2[i] = rand() / RAND_MAX;
 }
 float array3[THREADS_PER_BLOCK];
 for(int i=0; i<THREADS_PER_BLOCK;i++){
	srand(time(0));
	array3[i] = rand() / RAND_MAX;
 }

 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray1"), array1, sizeof(float) * THREADS_PER_BLOCK );
 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray2"), array2, sizeof(float) * THREADS_PER_BLOCK );
 hipMemcpyToSymbol(HIP_SYMBOL("ConstArray3"), array3, sizeof(float) * THREADS_PER_BLOCK );
 
 checkCudaErrors( hipMalloc((void**)&d_Value, sizeof(float)) );
 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);

 CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
 TaskHandle taskhandle = LaunchDAQ();
 CUT_SAFE_CALL(cutStartTimer(my_timer)); 
 PowerKernal<<<dimGrid,dimBlock>>>(d_Value);
 CUDA_SAFE_CALL( hipDeviceSynchronize() );
 printf("execution time = %f\n", cutGetTimerValue(my_timer));
 TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
 CUT_SAFE_CALL(cutStopTimer(my_timer));
 CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 
 
 getLastCudaError("kernel launch failure");

 checkCudaErrors( hipMemcpy(h_Value, d_Value, sizeof(float), hipMemcpyDeviceToHost) );
#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif


 return 0;
}







