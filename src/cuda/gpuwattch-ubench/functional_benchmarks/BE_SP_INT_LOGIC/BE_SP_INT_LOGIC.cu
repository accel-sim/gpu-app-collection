#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
// Includes
#include <stdio.h>
#include "../include/ContAcq-IntClk.h"

// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 60
#define ITERATIONS REPLACE_ITERATIONS

// Variables
unsigned* h_A;
unsigned* h_B;
unsigned* h_C;
unsigned* d_A;
unsigned* d_B;
unsigned* d_C;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(unsigned*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal1(const unsigned* A, const unsigned* B, unsigned* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned Value1=0;
    unsigned Value2=0;
    unsigned Value3=0;
    unsigned Value=0;
    unsigned I1=A[i];
    unsigned I2=B[i];


    //Excessive Logical Unit access
    for(unsigned k=0; k<ITERATIONS;k++) {

	Value1=I1 & I2;
	Value2 |= (I1 | I2);
	Value3=I1^Value2;
	Value2|=Value1;
	    Value2=Value3 & Value2;
	    Value1=Value2 ^ Value3;

//	Value1=I1 & I2;
//	Value2=I1 | I2;
//	Value3=I1^Value2;
//	Value2|=Value1;
//	    Value2=Value3 & Value2;
//	    Value1=Value2 ^ Value3;
    }

    __syncthreads();
    Value=Value1;

    C[i]=Value;
    __syncthreads();

}


int main()
{
 printf("Power Microbenchmarks\n");
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(unsigned);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (unsigned*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (unsigned*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (unsigned*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
 TaskHandle taskhandle = LaunchDAQ();
 CUT_SAFE_CALL(cutStartTimer(my_timer)); 
 printf("execution time = %f\n", cutGetTimerValue(my_timer));
PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
TurnOffDAQ(taskhandle, my_timer);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
printf("execution time = %f\n", cutGetTimerValue(my_timer));


#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
	srand((unsigned)time(0));  
	data[i] = rand() / RAND_MAX;
  }
}






