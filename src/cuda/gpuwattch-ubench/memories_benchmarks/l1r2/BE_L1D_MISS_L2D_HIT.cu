#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>
#include <math.h>
// Includes
#include <stdio.h>
#include "../include/ContAcq-IntClk.h"
//#include "REPEATL.h"
#include "../include/REPEATR.h"
// includes, project
#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 32
#define NUM_OF_BLOCKS 1
#define max_tid THREADS_PER_BLOCK*NUM_OF_BLOCKS    
#define LINE_SIZE 4 
#define SETS		64
#define ASSOC		6
#define SIMD_WIDTH	32
#define ITERATIONS REPLACE_ITERATIONS
// Variables
int* h_A;
int* h_B;
int* h_C;
int* d_A;
int* d_B;
int* d_C;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(int*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line ){
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line ){
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(int* A, int* C, int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation

    int size = (400*max_tid*LINE_SIZE)/sizeof(int); 
    unsigned j=0, k=0;

    int sum=0;

	// Fill the L1 cache, Miss on every iteration
	for (int i=0; i<ITERATIONS ; i++){
    	REPEAT_L6(0);
	//REPLACE_ITERATIONS
	}


	/*
	// Fill the L1 cache, Miss on first LD, Hit on subsequent LDs
	for(k=0; k<ITERATIONS; ++k){
		for(j=0; j<(size/2); j+=THREADS_PER_BLOCK){
			C[tid+j] = A[tid+j];
		}
	}
	*/
	C[0]=sum;
    __syncthreads();


}


// Host code

int main(){

	 printf("Power Microbenchmarks\n");
	 int N = (400*max_tid*LINE_SIZE);
	 size_t size = N * sizeof(int) ;


	 // Allocate input vectors h_A and h_B in host memory
	 h_A = (int*)malloc(size);
	 if (h_A == 0) CleanupResources();
	 //h_B = (float*)malloc(size);
	 //if (h_B == 0) CleanupResources();
	 h_C = (int*)malloc(size);
	 if (h_C == 0) CleanupResources();

	 // Initialize input vectors
	 RandomInit(h_A, N);
	 //RandomInit(h_B, N);

	 // Allocate vectors in device memory
	 checkCudaErrors( hipMalloc((void**)&d_A, size) );
	 //checkCudaErrors( hipMalloc((void**)&d_B, size) );
	 checkCudaErrors( hipMalloc((void**)&d_C, size) );

	 // Copy vectors from host memory to device memory
	 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
	 //checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

	 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	 dim3 dimGrid(NUM_OF_BLOCKS,1);
	 dim3 dimBlock(THREADS_PER_BLOCK,1);
	CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
	TaskHandle taskhandle = LaunchDAQ();
	CUT_SAFE_CALL(cutStartTimer(my_timer)); 
	 PowerKernal<<<dimGrid,dimBlock>>>(d_A, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	printf("execution time = %f\n", cutGetTimerValue(my_timer));
	TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
	CUT_SAFE_CALL(cutStopTimer(my_timer));
	CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 

	 getLastCudaError("kernel launch failure");

	#ifdef _DEBUG
	 checkCudaErrors( hipDeviceSynchronize() );
	#endif

	 // Copy result from device memory to host memory
	 // h_C contains the result in host memory
	 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );

	 CleanupResources();

	 return 0;
}

void CleanupResources(void){
  // Free device memory
  if (d_A)
	hipFree(d_A);
  //if (d_B)
//	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
 // if (h_B)
//	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(int* data, int n){
  for (int i = 0; i < n; ++i)
	data[i] = (int)(rand() / RAND_MAX);
}






