
#include <hip/hip_runtime.h>
hipEvent_t cstart, cstop;

void cudatic(){
  
  hipEventCreate(&cstart);
  hipEventCreate(&cstop);

  hipEventRecord(cstart, 0); 
}

float cudatoc(){
  
  hipEventRecord(cstop, 0); 
  hipEventSynchronize(cstop); 
  float elapsedTime; 
  hipEventElapsedTime(&elapsedTime, cstart, cstop);
  
  /* return elapsed time in seconds */
  return elapsedTime/1000.0;

}


