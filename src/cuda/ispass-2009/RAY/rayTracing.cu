#include "hip/hip_runtime.h"
/*
 * Copyright 2008 BOROUJERDI Maxime. Tous droits reserves.
 */

#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include "makebmp.h"

/*#include <GL/glew.h>
#include <GL/glut.h>

#include <cuda_gl_interop.h>*/
#include <cutil.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#define numObj 4

#define PI 3.141592654f
#define Angle(a) ((a*PI)/180.0)

//#define DEVICE_EMU
//#define DEBUG_RT_CUDA
#define FIXED_CONST_PARSE
#ifdef DEBUG_RT_CUDA
#define DEBUG_NUM 8
float4 *d_debug_float4;
uint *d_debug_uint;
float4 *h_debug_float4;
uint *h_debug_uint;
#endif
int g_verbose;

#include <rayTracing_kernel.cu>

unsigned width = 64; //640; //512; //16; //32; //512;
unsigned height = 64; //480; //512; //16;//512;
dim3 blockSize(16,8);
dim3 gridSize(width/blockSize.x, height/blockSize.y);

float3 viewRotation;
float3 viewTranslation = make_float3(0.0, 0.0, -4.0f);
float invViewMatrix[12];

//static int fpsCount = 0;        // FPS count for averaging
//static int fpsLimit = 1;        // FPS limit for sampling
unsigned int timer;


//GLuint pbo = 0;     // Pixel buffer d'OpenGL


void initPixelBuffer();

class Observateur
{
  private:
    matrice3x4  M;   // U, V, W
    float       df;  // distance focale
    
  public:
    Observateur( );
    Observateur(const float3 &, const float3 &, const float3 &, double );
    
	inline const matrice3x4 & getMatrice( ) const { return M; }
	inline float getDistance( ) const { return df; }
};

Observateur::Observateur()
{
  M.m[0] = make_float4(0.0f,0.0f,1.0f,0.0f);
  M.m[1] = make_float4(0.0f,1.0f,0.0f,0.0f);
  M.m[2] = make_float4(1.0f,0.0f,0.0f,0.0f);
  df = 1.0 / tan(Angle(65)/2.0);
}

Observateur::Observateur(const float3 & p, const float3 & u, const float3 & v, double a )
{
  float3 VP, U, V, W;
  VP = normalize(v);
  U = normalize(u);
  V = normalize(VP - dot(U,VP)*U);
  W = normalize(cross(U,V));
  M.m[0] = make_float4(U.x,U.y,U.z,p.x);
  M.m[1] = make_float4(V.x,V.y,V.z,p.y);
  M.m[2] = make_float4(W.x,W.y,W.z,p.z);
  df = 1.0 / tan(Angle(a)/2.0);
}

float anim = 0.0f, pas = 0.015f;
Observateur obs = Observateur(make_float3(0.0f,0.5f,2.0f),normalize(make_float3(0.0f,0.0f,0.0f)-make_float3(0.0f,0.5f,2.0f)),make_float3(0.0f,1.0f,0.0f),65.0f);;

uint * values = NULL, * d_output, * d_temp, NUM;
uint * c_output;

Node node[numObj], * d_node;

Sphere s, s1, s2;
float phi;

uint  * nObj;
float * prof;
Rayon * ray;
float3 * A, *u;
int t = 1;


void initObjet()
{
	srand(47);
	node->s.r = 1.0f;
	node[0].s.C = make_float3(0.0f,-1.5f,-0.0f); node[0].s.r = 0.5f;
	node[1].s.C = make_float3(-1.0f,0.0f,-1.0f); node[1].s.r = 0.5f;
	node[2].s.C = make_float3(1.0f,-0.f,-1.0f); node[2].s.r = 0.5f;
	node[3].s.C = make_float3(0.0f,-0.f,-2.0f); node[3].s.r = 0.75f;
	for( int i(4); i < numObj; i++ ) {
		float r,v,b;
		float tmp1(5.0f*((r=(float(rand()%255)/255.0f)))-2.5f);
		float tmp2(5.0f*((v=(float(rand()%255)/255.0f)))-2.5f);
		float tmp3(-5.0f*((b=(float(rand()%255)/255.0f))));
		float tmp4((rand()%100)/100.0f);
		node[i].s.C = make_float3(tmp1,tmp2,tmp3); node[i].s.r = tmp4;
		node[i].s.R = r; node[i].s.V = v; node[i].s.B = b; node[i].s.A = 1.0f;
		node[i].fg = 0; node[i].fd = 0;
	}
	node[0].s.R = 0.0f; node[0].s.V = 1.0f; node[0].s.B = 1.0f; node[0].s.A = 1.0f;
	node[1].s.R = 1.0f; node[1].s.V = 0.0f; node[1].s.B = 0.0f; node[1].s.A = 1.0f;
	node[2].s.R = 0.0f; node[2].s.V = 0.0f; node[2].s.B = 1.0f; node[2].s.A = 1.0f;
	node[3].s.R = 0.0f; node[3].s.V = 1.0f; node[3].s.B = 0.0f; node[3].s.A = 1.0f;
	//createNode(&node[0], &node[1], &node[2], 1.0f);
	node[0].fg = 1;	node[0].fd = 2;
	node[1].fg = 0; node[1].fd = 0;
	node[2].fg = 0; node[2].fd = 0;
	node[3].fg = 0; node[3].fd = 0;

   #ifdef DEBUG_RT_CUDA
   h_debug_float4 = (float4*) calloc(DEBUG_NUM, sizeof(float4));
   h_debug_uint = (uint*) calloc(DEBUG_NUM, sizeof(uint));
   CUDA_SAFE_CALL( hipMalloc( (void**)&d_debug_float4, DEBUG_NUM*sizeof(float4)));
   CUDA_SAFE_CALL( hipMalloc( (void**)&d_debug_uint, DEBUG_NUM*sizeof(uint)));
   CUDA_SAFE_CALL( hipMemcpy( d_debug_float4, h_debug_float4, DEBUG_NUM*sizeof(float4), hipMemcpyHostToDevice) );
   CUDA_SAFE_CALL( hipMemcpy( d_debug_uint, h_debug_uint, DEBUG_NUM*sizeof(uint), hipMemcpyHostToDevice) );
   #endif
   c_output = (uint*) calloc(width*height, sizeof(uint));
   CUDA_SAFE_CALL( hipMalloc( (void**)&d_output, width*height*sizeof(uint)));

    CUDA_SAFE_CALL( hipMalloc( (void**)&d_node, numObj*sizeof(Node) ));
    CUDA_SAFE_CALL( hipMemcpy( d_node, node, numObj*sizeof(Node), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(cnode), node, numObj*sizeof(Node)) );
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MView), (void*)&obs, 3*sizeof(float4)) );	
	CUDA_SAFE_CALL( hipMalloc( (void**)&d_temp, width * height*sizeof(uint)));
	CUDA_SAFE_CALL( hipMemset(d_temp, 0, width * height*sizeof(uint)) );
	
	CUDA_SAFE_CALL( hipMalloc( (void**)&nObj, width * height*sizeof(uint)));
	CUDA_SAFE_CALL( hipMalloc( (void**)&prof, width * height*sizeof(float)));
	CUDA_SAFE_CALL( hipMalloc( (void**)&ray, width * height*sizeof(Rayon)));
	
	CUDA_SAFE_CALL( hipMalloc( (void**)&A, width * height*sizeof(float3)));
	CUDA_SAFE_CALL( hipMalloc( (void**)&u, width * height*sizeof(float3)));
}

#define PRINT_PIXELS

// Rendu de l'image avec CUDA
void render()
{
    // map PBO to get CUDA device pointer <GY: replace with memcpy?>
    //CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&d_output, pbo));
    //CUDA_SAFE_CALL( hipMemcpy( d_output, c_output, width*height*sizeof(uint), hipMemcpyHostToDevice) );
    // call CUDA kernel, writing results to PBO
    CUT_SAFE_CALL(cutStartTimer(timer)); 
    #ifdef DEBUG_RT_CUDA
    render<<<gridSize, blockSize>>>(d_debug_float4, d_debug_uint, d_output, d_node, width, height, anim, obs.getDistance());
    #else
    render<<<gridSize, blockSize>>>(d_output, d_node, width, height, anim, obs.getDistance());
    #endif
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL(cutStopTimer(timer));

    #ifdef DEBUG_RT_CUDA
    CUDA_SAFE_CALL( hipMemcpy( h_debug_float4, d_debug_float4, DEBUG_NUM*sizeof(float4), hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy( h_debug_uint, d_debug_uint, DEBUG_NUM*sizeof(uint), hipMemcpyDeviceToHost) );

    printf("debug_float4\n");
    for (int i=0; i< DEBUG_NUM; i++) {
       printf("%e %e %e %e\n", h_debug_float4[i].x, h_debug_float4[i].y, h_debug_float4[i].z, h_debug_float4[i].w);
    }
    printf("debug_uint\n");
    for (int i=0; i< DEBUG_NUM; i++) {
       printf("0x%x\n", h_debug_uint[i]);
    }
    #endif

    CUDA_SAFE_CALL( hipMemcpy( c_output, d_output, width*height*sizeof(uint), hipMemcpyDeviceToHost) );
    unsigned long long int checksum = 0;
    for (int y=(height-1); y >= 0; y--){
       if (g_verbose) printf("\n");
       for  (int x=0; x< width; x++) {
          if (g_verbose) printf("%010u ", (unsigned) c_output[x+y*width]);
          checksum += c_output[x+y*width];
       }
    }
    printf("\n");
    printf("checksum=%llx\n", checksum);
    CUT_CHECK_ERROR("Erreur kernel");

    //CUDA_SAFE_CALL(cudaGLUnmapBufferObject(pbo)); //<GY: replace with memcpy?>

}

// Affichage du resultat avec OpenGL
void display()
{

    // Affichage du resultat
    //glClear(GL_COLOR_BUFFER_BIT);

    //CUT_SAFE_CALL(cutStartTimer(timer)); 
	 render();
    //CUT_SAFE_CALL(cutStopTimer(timer));
	 printf("Kernel Time: %f \n", cutGetTimerValue(timer));
	/*fpsCount++;
	if (fpsCount == fpsLimit) {
		char fps[256];
		float ifps = 1.f / (cutGetAverageTimerValue(timer) / 1000.f);
		sprintf(fps, "Cuda Ray Tracing: %.1f fps", ifps);  
		glutSetWindowTitle(fps);
		fpsCount = 0; 
		fpsLimit = (int)max(ifps, 1.f);
		CUT_SAFE_CALL(cutResetTimer(timer));
	}*/

	if( anim >= 1.0f ) pas = -0.015f;
	else if( anim <= -1.0f ) pas = 0.015f;
	anim += pas;

    // Dessin de l'image de PBO
    /*glDisable(GL_DEPTH_TEST);
    glRasterPos2i(0, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    glutSwapBuffers();
    glutReportErrors();*/
    t--;
    if (!t) {
       return;
    }

}

/*void idle()
{
	glutPostRedisplay();
}

void keyboard(unsigned char , int , int )
{
	//glutPostRedisplay();
}*/

int ox, oy;
int buttonState = 0;

/*void mouse(int , int , int , int )
{
    if (state == GLUT_DOWN)
        buttonState |= 1<<button;
    else if (state == GLUT_UP)
        buttonState = 0;

    ox = x; oy = y;
    glutPostRedisplay();
}

void motion(int , int )
{
    float dx, dy;
    dx = x - ox;
    dy = y - oy;

    if (buttonState == 3) {
        // left+middle = zoom
        viewTranslation.z += dy / 100.0;
    } 
    else if (buttonState & 2) {
        // middle = translate
        viewTranslation.x += dx / 100.0;
        viewTranslation.y -= dy / 100.0;
    }
    else if (buttonState & 1) {
        // left = rotate
        viewRotation.x += dy / 5.0;
        viewRotation.y += dx / 5.0;
    }

    ox = x; oy = y;
    glutPostRedisplay();
}

void reshape(int x, int y)
{
    width = x; height = y;
    initPixelBuffer();

    glViewport(0, 0, x, y);
    //glViewport(-x/2, -y/2, x/2, y/2);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0); 
}

void cleanup()
{
	CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));    
	glDeleteBuffersARB(1, &pbo);
    CUT_SAFE_CALL(cutDeleteTimer(timer));  
}*/

int iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void initPixelBuffer()
{
    /*if (pbo) {
        // delete old buffer
        CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));
        glDeleteBuffersARB(1, &pbo);
    }*/

	NUM = width * height;
	phi = 2.0f/(float)min(width,height);

    // create pixel buffer object for display
   /* glGenBuffersARB(1, &pbo);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
	glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, width*height*sizeof(GLubyte)*4, 0, GL_STREAM_DRAW_ARB);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));*/

    // calculate new grid size
    gridSize = dim3(iDivUp(width, blockSize.x), iDivUp(height, blockSize.y));
}


////////////////////////////////////////////////////////////////////////////////
// Programme principal
////////////////////////////////////////////////////////////////////////////////



int main( int argc, char** argv) 
{
  // initialise card and timer
  int deviceCount;                                                         
  CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                
  if (deviceCount == 0) {                                                  
      fprintf(stderr, "There is no device.\n");                            
      exit(EXIT_FAILURE);                                                  
  }                                                                        
  int dev;                                                                 
  for (dev = 0; dev < deviceCount; ++dev) {                                
      hipDeviceProp_t deviceProp;                                           
      CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));   
      if (deviceProp.major >= 1)                                           
          break;                                                           
  }                                                                        
  if (dev == deviceCount) {                                                
      fprintf(stderr, "There is no device supporting CUDA.\n");            
      exit(EXIT_FAILURE);                                                  
  }                                                                        
  else                                                                     
      CUDA_SAFE_CALL(hipSetDevice(dev));  
	int i, commandline_error;
	commandline_error = 0;
	g_verbose = 0;
	if (argc >= 3) {
		width = atoi(argv[1]);
        height = atoi(argv[2]);
		for (i=3; i < argc;i++) {
			if (argv[i][0] == '-') {
				switch (argv[i][1]) {
				case 'v': g_verbose = 1;
					break;
				default: commandline_error=1;
				}
			}
			else commandline_error=1;
		}
	} else commandline_error=1;

	if (commandline_error || !width || !height) {
		printf("Usage: ./rayTracing <WIDTH> <HEIGHT> [-v]\n");
		printf("where WIDTH and HEIGHT are the screen dimensions and -v is used to display an abstract representation of the output.\n");
		return 1;
	}
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    CUT_SAFE_CALL(cutResetTimer(timer));  

    initialize_bmp(width,height,32);

    // initialise les functions callback de GLUT
    /*glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE);
    glutInitWindowSize(width, height);
    glutCreateWindow("CUDA Ray Tracing");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(mouse);
    glutMotionFunc(motion);
    glutReshapeFunc(reshape);
    glutIdleFunc(idle);

    glewInit();
    if (!glewIsSupported("GL_VERSION_2_0 GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Les extensions minimales d'OpenGL sont absentes.");
        exit(-1);
    }
    initPixelBuffer();
	 initObjet();

    atexit(cleanup);

    glutMainLoop();*/
    initObjet();
    initPixelBuffer();
    display();
    create_bmp(c_output);
    CUT_SAFE_CALL(cutDeleteTimer(timer)); 
    return 0;
}
