#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>

#include "cuenergy.h"

//#if UNROLLX != 8
//# error "UNROLLX must be 8"
//#endif

#if BLOCKSIZEX != 16
# error "BLOCKSIZEX must be 16"
#endif

// Max constant buffer size is 64KB, minus whatever
// the CUDA runtime and compiler are using that we don't know about.
// At 16 bytes for atom, for this program 4070 atoms is about the max
// we can store in the constant buffer.
__constant__ float4 atominfo[MAXATOMS];

// This kernel calculates coulombic potential at each grid point and
// stores the results in the output array.

__global__ void cenergy(int numatoms, float gridspacing, float * energygrid) {
  unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) * UNROLLX
                         + threadIdx.x;
  unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int outaddr = (__umul24(gridDim.x, blockDim.x) * UNROLLX) * yindex
                         + xindex;

  float coory = gridspacing * yindex;
  float coorx = gridspacing * xindex;

  float energyvalx1=0.0f;
  float energyvalx2=0.0f;

  float gridspacing_u = gridspacing * BLOCKSIZEX;

  int atomid;
  for (atomid=0; atomid<numatoms; atomid++) {
    float dy = coory - atominfo[atomid].y;
    float dyz2 = (dy * dy) + atominfo[atomid].z;

    float dx1 = coorx - atominfo[atomid].x;
    float dx2 = dx1 + gridspacing_u;

    energyvalx1 += atominfo[atomid].w * (1.0f / sqrtf(dx1*dx1 + dyz2));
    energyvalx2 += atominfo[atomid].w * (1.0f / sqrtf(dx2*dx2 + dyz2));
  }

  energygrid[outaddr]   += energyvalx1;
  energygrid[outaddr+1*BLOCKSIZEX] += energyvalx2;
}

// This function copies atoms from the CPU to the GPU and
// precalculates (z^2) for each atom.

int copyatomstoconstbuf(float *atoms, int count, float zplane) {
  if (count > MAXATOMS) {
    printf("Atom count exceeds constant buffer storage capacity\n");
    return -1;
  }

  float atompre[4*MAXATOMS];
  int i;
  for (i=0; i<count*4; i+=4) {
    atompre[i    ] = atoms[i    ];
    atompre[i + 1] = atoms[i + 1];
    float dz = zplane - atoms[i + 2];
    atompre[i + 2]  = dz*dz;
    atompre[i + 3] = atoms[i + 3];
  }

  hipMemcpyToSymbol(HIP_SYMBOL(atominfo), atompre, count * 4 * sizeof(float), 0);
  CUERR // check and clear any existing errors

  return 0;
}

