#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>

#include "cuenergy.h"

#if UNROLLX != 1
# error "UNROLLX must be 1"
#endif

__constant__ float4 atominfo[MAXATOMS];

/* This is a reference version of the kernel.  It is simpler and slower
 * than the optimzed version. */

__global__ void cenergy(int numatoms, float gridspacing, float * energygrid) {
  unsigned int xindex  = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  unsigned int yindex  = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
  unsigned int outaddr = __umul24(gridDim.x, blockDim.x) * yindex + xindex;

  float coorx = gridspacing * xindex;
  float coory = gridspacing * yindex;

  int atomid;
  float energyval=0.0f;

  /* For each atom, compute and accumulate its contribution to energyval
   * for this thread's grid point */
  for (atomid=0; atomid<numatoms; atomid++) {
    float dx = coorx - atominfo[atomid].x;
    float dy = coory - atominfo[atomid].y;
    float r_1 = 1.0f / sqrtf(dx*dx + dy*dy + atominfo[atomid].z);
    energyval += atominfo[atomid].w * r_1;
  }

  energygrid[outaddr] += energyval;
}

// This function copies atoms from the CPU to the GPU and
// precalculates (z^2) for each atom.

int copyatomstoconstbuf(float *atoms, int count, float zplane) {
  if (count > MAXATOMS) {
    printf("Atom count exceeds constant buffer storage capacity\n");
    return -1;
  }

  float atompre[4*MAXATOMS];
  int i;
  for (i=0; i<count*4; i+=4) {
    atompre[i    ] = atoms[i    ];
    atompre[i + 1] = atoms[i + 1];
    float dz = zplane - atoms[i + 2];
    atompre[i + 2]  = dz*dz;
    atompre[i + 3] = atoms[i + 3];
  }

  hipMemcpyToSymbol(HIP_SYMBOL(atominfo), atompre, count * 4 * sizeof(float), 0);
  CUERR // check and clear any existing errors

  return 0;
}

