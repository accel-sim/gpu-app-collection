#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
/*
 * CUDA accelerated coulombic potential grid test code
 *   John E. Stone <johns@ks.uiuc.edu>
 *   http://www.ks.uiuc.edu/~johns/
 *
 * Coulombic potential grid calculation microbenchmark based on the time
 * consuming portions of the 'cionize' ion placement tool.
 */

#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuenergy.h"

/* initatoms()
 * Store a pseudorandom arrangement of point charges in *atombuf.
 */
static int
initatoms(float **atombuf, int count, dim3 volsize, float gridspacing) {
  dim3 size;
  int i;
  float *atoms;

  srand(54321);			// Ensure that atom placement is repeatable

  atoms = (float *) malloc(count * 4 * sizeof(float));
  *atombuf = atoms;

  // compute grid dimensions in angstroms
  size.x = gridspacing * volsize.x;
  size.y = gridspacing * volsize.y;
  size.z = gridspacing * volsize.z;

  for (i=0; i<count; i++) {
    int addr = i * 4;
    atoms[addr    ] = (rand() / (float) RAND_MAX) * size.x; 
    atoms[addr + 1] = (rand() / (float) RAND_MAX) * size.y; 
    atoms[addr + 2] = (rand() / (float) RAND_MAX) * size.z; 
    atoms[addr + 3] = ((rand() / (float) RAND_MAX) * 2.0) - 1.0;  // charge
  }  

  return 0;
}

/* writeenergy()
 * Write part of the energy array to an output file for verification.
 */
static int
writeenergy(char *filename, float *energy, dim3 volsize)
{
  FILE *outfile;
  int x, y;

  outfile = fopen(filename, "w");
  if (outfile == NULL) {
    fputs("Cannot open output file\n", stderr);
    return -1;
    }

  /* Print the execution parameters */
  fprintf(outfile, "%d %d %d %d\n", volsize.x, volsize.y, volsize.z, ATOMCOUNT);

  /* Print a checksum */
  {
    double sum = 0.0;

    for (y = 0; y < volsize.y; y++) {
      for (x = 0; x < volsize.x; x++) {
        double t = energy[y*volsize.x+x];
        t = fmax(-20.0, fmin(20.0, t));
    	sum += t;
      }
    }
    fprintf(outfile, "%.4g\n", sum);
  }
  
  /* Print several rows of the computed data */
  for (y = 0; y < 17; y++) {
    for (x = 0; x < volsize.x; x++) {
      int addr = y * volsize.x + x;
      fprintf(outfile, "%.4g ", energy[addr]);
    }
    fprintf(outfile, "\n");
  }

  fclose(outfile);

  return 0;
}

int main(int argc, char** argv) {
  struct pb_TimerSet timers;
  struct pb_Parameters *parameters;

  float *energy = NULL;		// Output of device calculation
  float *atoms = NULL;
  dim3 volsize, Gsz, Bsz;

  // int final_iteration_count;

  // number of atoms to simulate
  int atomcount = ATOMCOUNT;

  // voxel spacing
  const float gridspacing = 0.1;

  // Size of buffer on GPU
  int volmemsz;

  printf("CUDA accelerated coulombic potential microbenchmark\n");
  printf("Original version by John E. Stone <johns@ks.uiuc.edu>\n");
  printf("This version maintained by Chris Rodrigues\n");

  parameters = pb_ReadParameters(&argc, argv);
  if (!parameters)
    return -1;

  if (parameters->inpFiles[0]) {
    fputs("No input files expected\n", stderr);
    return -1;
  }

  pb_InitializeTimerSet(&timers);
  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  // setup energy grid size
  volsize.x = VOLSIZEX;
  volsize.y = VOLSIZEY;
  volsize.z = 1;

  // setup CUDA grid and block sizes
  Bsz.x = BLOCKSIZEX;		// each thread does multiple Xs
  Bsz.y = BLOCKSIZEY;
  Bsz.z = 1;
  Gsz.x = volsize.x / (Bsz.x * UNROLLX); // each thread does multiple Xs
  Gsz.y = volsize.y / Bsz.y; 
  Gsz.z = volsize.z / Bsz.z; 

#if 0
  printf("Grid size: %d x %d x %d\n", volsize.x, volsize.y, volsize.z);
  printf("Running kernel(atoms:%d, gridspacing %g, z %d)\n", atomcount, gridspacing, 0);
#endif

  // allocate and initialize atom coordinates and charges
  if (initatoms(&atoms, atomcount, volsize, gridspacing))
    return -1;

  // allocate and initialize the GPU output array
  volmemsz = sizeof(float) * volsize.x * volsize.y * volsize.z;

  // Main computation
  {
    float *d_output = NULL;	// Output on device
    int iterations=0;
    int atomstart;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    hipMalloc((void**)&d_output, volmemsz);
    CUERR // check and clear any existing errors
    hipMemset(d_output, 0, volmemsz);
    CUERR // check and clear any existing errors
    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

    for (atomstart=0; atomstart<atomcount; atomstart+=MAXATOMS) {   
      int atomsremaining = atomcount - atomstart;
      int runatoms = (atomsremaining > MAXATOMS) ? MAXATOMS : atomsremaining;
      iterations++;

      // copy the atoms to the GPU
      pb_SwitchToTimer(&timers, pb_TimerID_COPY);
      if (copyatomstoconstbuf(atoms + 4*atomstart, runatoms, 0*gridspacing)) 
	return -1;

      if (parameters->synchronizeGpu) hipDeviceSynchronize();
      pb_SwitchToTimer(&timers, pb_TimerID_GPU);
 
      // RUN the kernel...
      pb_StartTimer(&timers.gpu);
      cenergy<<<Gsz, Bsz, 0>>>(runatoms, 0.1, d_output);
      CUERR // check and clear any existing errors

      if (parameters->synchronizeGpu) hipDeviceSynchronize();
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

      // final_iteration_count = iterations;
    }
#if 0
    printf("Done\n");
#endif

    // Copy the GPU output data back to the host and use/store it..
    energy = (float *) malloc(volmemsz);
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    hipMemcpy(energy, d_output, volmemsz,  hipMemcpyDeviceToHost);
    CUERR // check and clear any existing errors

    hipFree(d_output);

    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  }

  /* Print a subset of the results to a file */
  if (parameters->outFile) {
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
    if (writeenergy(parameters->outFile, energy, volsize) == -1)
      return -1;
    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  }

  free(atoms);
  free(energy);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  pb_PrintTimerSet(&timers);
  pb_FreeParameters(parameters);

  return 0;
}



