#include "hip/hip_runtime.h"
// N-queen for CUDA
//
// Copyright(c) 2008 Ping-Che Chen


//#define WIN32_LEAN_AND_MEAN
//#include <windows.h>
#include <stdio.h>
#include <cutil.h>

#define THREAD_NUM		96


int bunk = 0;		// this is a dummy variable used for making sure clock() are not optimized out

/*
 * ----------------------------------------------------------------
 * This is a recursive version of n-queen backtracking solver.
 * A non-recursive version is used instead.
 * ----------------------------------------------------------------

long long solve_nqueen_internal(int n, unsigned int mask, unsigned int l_mask, unsigned int r_mask, unsigned int t_mask)
{
	if(mask == t_mask) {
		return 1;
	}

	unsigned int m = (mask | l_mask | r_mask);
	if((m & t_mask) == t_mask) {
		return 0;
	}

	long long total = 0;
	unsigned int index = (m + 1) & ~m;
	while((index & t_mask) != 0) {
		total += solve_nqueen_internal(mask | index, (l_mask | index) << 1, (r_mask | index) >> 1, t_mask);
		m |= index;
		index = (m + 1) & ~m;
	}

	return total;
}


long long solve_nqueen(int n)
{
	return solve_nqueen_internal(0, 0, 0, (1 << n) - 1);
}
*/

/* -------------------------------------------------------------------
 * This is a non-recursive version of n-queen backtracking solver.
 * This provides the basis for the CUDA version.
 * -------------------------------------------------------------------
 */

long long solve_nqueen(int n)
{
	unsigned int mask[32];
	unsigned int l_mask[32];
	unsigned int r_mask[32];
	unsigned int m[32];

	if(n <= 0 || n > 32) {
		return 0;
	}

	const unsigned int t_mask = (1 << n) - 1;
	long long total = 0;
	long long upper_total = 0;
	int i = 0, j;
	unsigned int index;

	mask[0] = 0;
	l_mask[0] = 0;
	r_mask[0] = 0;
	m[0] = 0;

	for(j = 0; j < (n + 1) / 2; j++) {
		index = (1 << j);
		m[0] |= index;
		
		mask[1] = index;
		l_mask[1] = index << 1;
		r_mask[1] = index >> 1;
		m[1] = (mask[1] | l_mask[1] | r_mask[1]);
		i = 1;
		
		if(n % 2 == 1 && j == (n + 1) / 2 - 1) {
			upper_total = total;
			total = 0;
		}
		
		while(i > 0) {
			if((m[i] & t_mask) == t_mask) {
				i--;
			}
			else {
				index = ((m[i] + 1) ^ m[i]) & ~m[i];
				m[i] |= index;
				if((index & t_mask) != 0) {
					if(i + 1 == n) {
						total++;
						i--;
					}
					else {
						mask[i + 1] = mask[i] | index;
						l_mask[i + 1] = (l_mask[i] | index) << 1;
						r_mask[i + 1] = (r_mask[i] | index) >> 1;
						m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
						i++;
					}
				}
				else {
					i --;
				}
			}
		}
	}

	bunk = 2;

	if(n % 2 == 0) {
		return total * 2;
	}
	else {
		return upper_total * 2 + total;
	}
}


/* -------------------------------------------------------------------
 * This is a non-recursive version of n-queen backtracking solver
 * with multi-thread support.
 * -------------------------------------------------------------------
 */
/*
struct thread_context
{
	HANDLE thread;
	bool stop;

	long long total;
	int n;
	unsigned int mask;
	unsigned int l_mask;
	unsigned int r_mask;
	unsigned int t_mask;

	HANDLE ready;
	HANDLE complete;
};

DWORD WINAPI solve_nqueen_proc(LPVOID param)
{
	thread_context* ctx = (thread_context*) param;

	unsigned int mask[32];
	unsigned int l_mask[32];
	unsigned int r_mask[32];
	unsigned int m[32];
	unsigned int t_mask;
	long long total;
	unsigned int index;
	unsigned int mark;

	for(;;) {
		WaitForSingleObject(ctx->ready, INFINITE);
		if(ctx->stop) {
			break;
		}

		int i = 0;

		mask[0] = ctx->mask;
		l_mask[0] = ctx->l_mask;
		r_mask[0] = ctx->r_mask;
		m[0] = mask[0] | l_mask[0] | r_mask[0];
		total = 0;
		t_mask = ctx->t_mask;
		mark = ctx->n;

		while(i >= 0) {
			if((m[i] & t_mask) == t_mask) {
				i--;
			}
			else {
				index = (m[i] + 1) & ~m[i];
				m[i] |= index;
				if((index & t_mask) != 0) {
					if(i + 1 == mark) {
						total++;
						i--;
					}
					else {
						mask[i + 1] = mask[i] | index;
						l_mask[i + 1] = (l_mask[i] | index) << 1;
						r_mask[i + 1] = (r_mask[i] | index) >> 1;
						m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
						i++;
					}
				}
				else {
					i --;
				}
			}
		}

		ctx->total = total;

		SetEvent(ctx->complete);
	}

	return 0;
}

long long solve_nqueen_mcpu(int n)
{
	if(n <= 0 || n > 32) {
		return 0;
	}

	SYSTEM_INFO info;
	thread_context* threads;
	int num_threads;

	GetSystemInfo(&info);
	num_threads = info.dwNumberOfProcessors;
	if(num_threads == 1) {
		// only one cpu found, use single thread version
		return solve_nqueen(n);
	}

	threads = new thread_context[num_threads];
	int j;
	for(j = 0; j < num_threads; j++) {
		threads[j].stop = false;
		threads[j].ready = CreateEvent(0, FALSE, FALSE, 0);
		threads[j].complete = CreateEvent(0, FALSE, TRUE, 0);
		threads[j].thread = CreateThread(0, 0, solve_nqueen_proc, threads + j, 0, 0);
		threads[j].total = 0;
	}

	int thread_idx = 0;

	const unsigned int t_mask = (1 << n) - 1;
	long long total = 0;
	unsigned int index;
	
	unsigned int m_mask = 0;
	if(n % 2 == 1) {
		m_mask = 1 << ((n + 1) / 2 - 1);
	}

	for(j = 0; j < (n + 1) / 2; j++) {
		index = 1 << j;
		
		WaitForSingleObject(threads[thread_idx].complete, INFINITE);
		
		if(threads[thread_idx].mask != m_mask) {
			total += threads[thread_idx].total * 2;
		}
		else {
			total += threads[thread_idx].total;
		}
		
		threads[thread_idx].mask = index;
		threads[thread_idx].l_mask = index << 1;
		threads[thread_idx].r_mask = index >> 1;
		threads[thread_idx].t_mask = t_mask;
		threads[thread_idx].total = 0;
		threads[thread_idx].n = n - 1;
		
		SetEvent(threads[thread_idx].ready);
		
		thread_idx = (thread_idx + 1) % num_threads;
	}

	// collect all threads...
	HANDLE* events = new HANDLE[num_threads];
	for(j = 0; j < num_threads; j++) {
		events[j] = threads[j].complete;
	}
	WaitForMultipleObjects(num_threads, events, TRUE, INFINITE);
	for(j = 0; j < num_threads; j++) {
		if(threads[j].mask != m_mask) {
			total += threads[j].total * 2;
		}
		else {
			total += threads[j].total;
		}
		
		threads[j].stop = true;
		SetEvent(threads[j].ready);

		events[j] = threads[j].thread;
	}

	WaitForMultipleObjects(num_threads, events, TRUE, INFINITE);

	for(j = 0; j < num_threads; j++) {
		CloseHandle(threads[j].thread);
		CloseHandle(threads[j].ready);
		CloseHandle(threads[j].complete);
	}
	delete[] threads;
	delete[] events;

	bunk = 3;

	return total;
}
*/


/* --------------------------------------------------------------------------
 * This is a non-recursive version of n-queen backtracking solver for CUDA.
 * It receives multiple initial conditions from a CPU iterator, and count
 * each conditions.
 * --------------------------------------------------------------------------
 */

__global__ void solve_nqueen_cuda_kernel(int n, int mark, unsigned int* total_masks, unsigned int* total_l_masks, unsigned int* total_r_masks, unsigned int* results, int total_conditions)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	const int idx = bid * blockDim.x + tid;

	__shared__ unsigned int mask[THREAD_NUM][10];
	__shared__ unsigned int l_mask[THREAD_NUM][10];
	__shared__ unsigned int r_mask[THREAD_NUM][10];
	__shared__ unsigned int m[THREAD_NUM][10];

	__shared__ unsigned int sum[THREAD_NUM];

	const unsigned int t_mask = (1 << n) - 1;
	int total = 0;
	int i = 0;
	unsigned int index;

	if(idx < total_conditions) {
		mask[tid][i] = total_masks[idx];
		l_mask[tid][i] = total_l_masks[idx];
		r_mask[tid][i] = total_r_masks[idx];
		m[tid][i] = mask[tid][i] | l_mask[tid][i] | r_mask[tid][i];

		while(i >= 0) {
			if((m[tid][i] & t_mask) == t_mask) {
				i--;
			}
			else {
				index = (m[tid][i] + 1) & ~m[tid][i];
				m[tid][i] |= index;
				if((index & t_mask) != 0) {
					if(i + 1 == mark) {
						total++;
						i--;
					}
					else {
						mask[tid][i + 1] = mask[tid][i] | index;
						l_mask[tid][i + 1] = (l_mask[tid][i] | index) << 1;
						r_mask[tid][i + 1] = (r_mask[tid][i] | index) >> 1;
						m[tid][i + 1] = (mask[tid][i + 1] | l_mask[tid][i + 1] | r_mask[tid][i + 1]);
						i++;
					}
				}
				else {
					i --;
				}
			}
		}

		sum[tid] = total;
	}
	else {
		sum[tid] = 0;
	}

	__syncthreads();

	// reduction
	if(tid < 64 && tid + 64 < THREAD_NUM) { sum[tid] += sum[tid + 64]; } __syncthreads();
	if(tid < 32) { sum[tid] += sum[tid + 32]; } __syncthreads();
	if(tid < 16) { sum[tid] += sum[tid + 16]; } __syncthreads();
	if(tid < 8) { sum[tid] += sum[tid + 8]; } __syncthreads();
	if(tid < 4) { sum[tid] += sum[tid + 4]; } __syncthreads();
	if(tid < 2) { sum[tid] += sum[tid + 2]; } __syncthreads();
	if(tid < 1) { sum[tid] += sum[tid + 1]; } __syncthreads();

	if(tid == 0) {
		results[bid] = sum[0];
	}
}


long long solve_nqueen_cuda(int n, int steps)
{
	// generating start conditions
	unsigned int mask[32];
	unsigned int l_mask[32];
	unsigned int r_mask[32];
	unsigned int m[32];
	unsigned int index;

	if(n <= 0 || n > 32) {
		return 0;
	}

	unsigned int* total_masks = new unsigned int[steps];
	unsigned int* total_l_masks = new unsigned int[steps];
	unsigned int* total_r_masks = new unsigned int[steps];
	unsigned int* results = new unsigned int[steps];

	unsigned int* masks_cuda;
	unsigned int* l_masks_cuda;
	unsigned int* r_masks_cuda;
	unsigned int* results_cuda;

	hipMalloc((void**) &masks_cuda, sizeof(int) * steps);
	hipMalloc((void**) &l_masks_cuda, sizeof(int) * steps);
	hipMalloc((void**) &r_masks_cuda, sizeof(int) * steps);
	hipMalloc((void**) &results_cuda, sizeof(int) * steps / THREAD_NUM);

	const unsigned int t_mask = (1 << n) - 1;
	const unsigned int mark = n > 11 ? n - 10 : 2;
	long long total = 0;
	int total_conditions = 0;
	int i = 0, j;

	mask[0] = 0;
	l_mask[0] = 0;
	r_mask[0] = 0;
	m[0] = 0;

	bool computed = false;

	for(j = 0; j < n / 2; j++) {
		index = (1 << j);
		m[0] |= index;
		
		mask[1] = index;
		l_mask[1] = index << 1;
		r_mask[1] = index >> 1;
		m[1] = (mask[1] | l_mask[1] | r_mask[1]);
		i = 1;
			
		while(i > 0) {
			if((m[i] & t_mask) == t_mask) {
				i--;
			}
			else {
				index = (m[i] + 1) & ~m[i];
				m[i] |= index;
				if((index & t_mask) != 0) {
					mask[i + 1] = mask[i] | index;
					l_mask[i + 1] = (l_mask[i] | index) << 1;
					r_mask[i + 1] = (r_mask[i] | index) >> 1;
					m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
					i++;
					if(i == mark) {
						total_masks[total_conditions] = mask[i];
						total_l_masks[total_conditions] = l_mask[i];
						total_r_masks[total_conditions] = r_mask[i];
						total_conditions++;
						if(total_conditions == steps) {
							if(computed) {
								hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM, hipMemcpyDeviceToHost);

								for(int j = 0; j < steps / THREAD_NUM; j++) {
									total += results[j];
								}

								computed = false;
							}

							// start computation
							hipMemcpy(masks_cuda, total_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
							hipMemcpy(l_masks_cuda, total_l_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
							hipMemcpy(r_masks_cuda, total_r_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);

							solve_nqueen_cuda_kernel<<<steps/THREAD_NUM, THREAD_NUM>>>(n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda, results_cuda, total_conditions);

							computed = true;

							total_conditions = 0;
						}
						i--;
					}
				}
				else {
					i --;
				}
			}
		}
	}
	

	if(computed) {
		hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM, hipMemcpyDeviceToHost);

		for(int j = 0; j < steps / THREAD_NUM; j++) {
			total += results[j];
		}

		computed = false;
	}

	hipMemcpy(masks_cuda, total_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
	hipMemcpy(l_masks_cuda, total_l_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
	hipMemcpy(r_masks_cuda, total_r_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);

	solve_nqueen_cuda_kernel<<<steps/THREAD_NUM, THREAD_NUM>>>(n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda, results_cuda, total_conditions);

	hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM, hipMemcpyDeviceToHost);

	for(int j = 0; j < steps / THREAD_NUM; j++) {
		total += results[j];
	}	
	
	total *= 2;
	
	if(n % 2 == 1) {
		computed = false;
		total_conditions = 0;

		index = (1 << (n - 1) / 2);
		m[0] |= index;
		
		mask[1] = index;
		l_mask[1] = index << 1;
		r_mask[1] = index >> 1;
		m[1] = (mask[1] | l_mask[1] | r_mask[1]);
		i = 1;
			
		while(i > 0) {
			if((m[i] & t_mask) == t_mask) {
				i--;
			}
			else {
				index = (m[i] + 1) & ~m[i];
				m[i] |= index;
				if((index & t_mask) != 0) {
					mask[i + 1] = mask[i] | index;
					l_mask[i + 1] = (l_mask[i] | index) << 1;
					r_mask[i + 1] = (r_mask[i] | index) >> 1;
					m[i + 1] = (mask[i + 1] | l_mask[i + 1] | r_mask[i + 1]);
					i++;
					if(i == mark) {
						total_masks[total_conditions] = mask[i];
						total_l_masks[total_conditions] = l_mask[i];
						total_r_masks[total_conditions] = r_mask[i];
						total_conditions++;
						if(total_conditions == steps) {
							if(computed) {
								hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM, hipMemcpyDeviceToHost);

								for(int j = 0; j < steps / THREAD_NUM; j++) {
									total += results[j];
								}

								computed = false;
							}

							// start computation
							hipMemcpy(masks_cuda, total_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
							hipMemcpy(l_masks_cuda, total_l_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
							hipMemcpy(r_masks_cuda, total_r_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);

							solve_nqueen_cuda_kernel<<<steps/THREAD_NUM, THREAD_NUM>>>(n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda, results_cuda, total_conditions);

							computed = true;

							total_conditions = 0;
						}
						i--;
					}
				}
				else {
					i --;
				}
			}
		}

		if(computed) {
			hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM, hipMemcpyDeviceToHost);

			for(int j = 0; j < steps / THREAD_NUM; j++) {
				total += results[j];
			}

			computed = false;
		}

		hipMemcpy(masks_cuda, total_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
		hipMemcpy(l_masks_cuda, total_l_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);
		hipMemcpy(r_masks_cuda, total_r_masks, sizeof(int) * total_conditions, hipMemcpyHostToDevice);

		solve_nqueen_cuda_kernel<<<steps/THREAD_NUM, THREAD_NUM>>>(n, n - mark, masks_cuda, l_masks_cuda, r_masks_cuda, results_cuda, total_conditions);

		hipMemcpy(results, results_cuda, sizeof(int) * steps / THREAD_NUM, hipMemcpyDeviceToHost);

		for(int j = 0; j < steps / THREAD_NUM; j++) {
			total += results[j];
		}
	}

	hipFree(masks_cuda);
	hipFree(l_masks_cuda);
	hipFree(r_masks_cuda);
	hipFree(results_cuda);

	delete[] total_masks;
	delete[] total_l_masks;
	delete[] total_r_masks;
	delete[] results;

	bunk = 1;

	return total;
}


bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}


int main(int argc, char** argv)
{
  unsigned int hTimer;
  double  gpuTime;
  // initialise card and timer
  int deviceCount;                                                         
  CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                
  if (deviceCount == 0) {                                                  
      fprintf(stderr, "There is no device.\n");                            
      exit(EXIT_FAILURE);                                                  
  }                                                                        
  int dev;                                                                 
  for (dev = 0; dev < deviceCount; ++dev) {                                
      hipDeviceProp_t deviceProp;                                           
      CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));   
      if (deviceProp.major >= 1)                                           
          break;                                                           
  }                                                                        
  if (dev == deviceCount) {                                                
      fprintf(stderr, "There is no device supporting CUDA.\n");            
      exit(EXIT_FAILURE);                                                  
  }                                                                        
  else                                                                     
      CUDA_SAFE_CALL(hipSetDevice(dev));  
  CUT_SAFE_CALL( cutCreateTimer(&hTimer) );

	int n = 8;
	clock_t start, end;
	long long solution;
	bool cpu = true, gpu = true;
	int argstart = 1, steps = 24576;

	if(argc >= 2 && argv[1][0] == '-') {
		if(argv[1][1] == 'c' || argv[1][1] == 'C') {
			gpu = false;
		}
		else if(argv[1][1] == 'g' || argv[1][1] == 'G') {
			cpu = false;
		}

		argstart = 2;
	}

	if(argc < argstart + 1) {
		printf("Usage: %s [-c|-g] n steps\n", argv[0]);
		printf("  -c: CPU only\n");
		printf("  -g: GPU only\n");
		printf("  n: n-queen\n");
		printf("  steps: step for GPU\n");
		printf("Default to 8 queen\n");
	}
	else {
		n = atoi(argv[argstart]);
		if(n <= 1 || n > 32) {
			printf("Invalid n, n should be > 1 and <= 32\n");
			printf("Note: n > 18 will require a very very long time to compute!\n");
			return 0;
		}

		if(argc >= argstart + 2) {
			steps = atoi(argv[argstart + 1]);
			if(steps <= THREAD_NUM || steps % THREAD_NUM != 0) {
				printf("Invalid step, step should be multiple of %d\n", THREAD_NUM);
				return 0;
			}
		}
	}

	if(gpu) {
	    if(!InitCUDA()) {
		    return 0;
		}

		printf("CUDA initialized.\n");
	}

	if(cpu) {
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_SAFE_CALL( cutResetTimer(hTimer) );
  CUT_SAFE_CALL( cutStartTimer(hTimer) );

		//start = clock();
		solution = solve_nqueen(n); //solve_nqueen_mcpu(n);
		//solution = solve_nqueen(n);
		//end = clock();
  CUT_SAFE_CALL( cutStopTimer(hTimer) );
  gpuTime = cutGetTimerValue(hTimer);

		printf("CPU: %d queen = %lld  time = %f msec\n", n, solution, gpuTime);
	}

	if(gpu) {
		//start = clock();
  CUDA_SAFE_CALL( hipDeviceSynchronize() );
  CUT_SAFE_CALL( cutResetTimer(hTimer) );
  CUT_SAFE_CALL( cutStartTimer(hTimer) );
		solution = solve_nqueen_cuda(n, steps);
		//end = clock();
  CUT_SAFE_CALL( cutStopTimer(hTimer) );
  gpuTime = cutGetTimerValue(hTimer);
		printf("GPU: %d queen = %lld  time = %f msec\n", n, solution, gpuTime);
	}

	return 0;
}
