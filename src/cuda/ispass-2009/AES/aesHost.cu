
/***************************************************************************
 *   Copyright (C) 2006                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/


/**
	@author Svetlin Manavski <svetlin@manavski.com>
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

#include "sbox_E.h"
#include "sbox_D.h"
#include <aesEncrypt128_kernel.h>
#include <aesDecrypt128_kernel.h>
#include <aesEncrypt256_kernel.h>
#include <aesDecrypt256_kernel.h>

extern "C" void aesEncryptHandler128(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesEncrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesDecryptHandler128(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesDecrypt128<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesEncryptHandler256(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesEncrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}

extern "C" void aesDecryptHandler256(unsigned *d_Result, unsigned *d_Input, int inputSize) {

	dim3  threads(BSIZE, 1);
    dim3  grid((inputSize/BSIZE)/4, 1);

	aesDecrypt256<<< grid, threads >>>( d_Result, d_Input, inputSize);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}


extern "C" int aesHost(unsigned char* result, const unsigned char* inData, int inputSize, const unsigned char* key, int keySize, bool toEncrypt)
{
	if (inputSize < 256) 
		return -1;
	if (inputSize % 256 > 0) 
		return -11;
	if (keySize != 240 && keySize != 176) 
		return -2;
	if (!result || !inData || !key)
		return -3;

    int deviceCount;                                                         
    CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                
    if (deviceCount == 0) {                                                  
        fprintf(stderr, "There is no device.\n");                            
        exit(EXIT_FAILURE);                                                  
    }                                                                        
    int dev;                                                                 
    for (dev = 0; dev < deviceCount; ++dev) {                                
        hipDeviceProp_t deviceProp;                                           
        CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&deviceProp, dev));   
        if (deviceProp.major >= 1)                                           
            break;                                                           
    }                                                                        
    if (dev == deviceCount) {                                                
        fprintf(stderr, "There is no device supporting CUDA.\n");            
        exit(EXIT_FAILURE);                                                  
    }                                                                        
    else                                                                     
        CUDA_SAFE_CALL(hipSetDevice(dev));                                  


    // allocate device memory
    unsigned * d_Input;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Input, inputSize) );

	// the size of the memory for the key must be equal to keySize (every thread copies one key byte to shared memory)
    unsigned * d_Key;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Key, keySize) );

	unsigned int ext_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&ext_timer));
    CUT_SAFE_CALL(cutStartTimer(ext_timer));

    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy(d_Input, inData, inputSize, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_Key, key, keySize, hipMemcpyHostToDevice) );

	//texture
	hipChannelFormatDesc chDesc;
	chDesc.x = 32;
	chDesc.y = 0;
	chDesc.z = 0;
	chDesc.w = 0;
	chDesc.f = hipChannelFormatKindUnsigned;
	texEKey.normalized = false;
	texDKey.normalized = false;
	texEKey128.normalized = false;
	texDKey128.normalized = false;

	CUDA_SAFE_CALL( hipBindTexture( 0, &texEKey128, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texDKey128, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texEKey, d_Key, &chDesc, (size_t)keySize) );
	CUDA_SAFE_CALL( hipBindTexture( 0, &texDKey, d_Key, &chDesc, (size_t)keySize) );

    // allocate device memory for result
    unsigned int size_Result = inputSize;
    unsigned * d_Result;
    CUDA_SAFE_CALL( hipMalloc((void**) &d_Result, size_Result) );
	CUDA_SAFE_CALL( hipMemset(d_Result, 0, size_Result) );
	

	unsigned int int_timer = 0;
    CUT_SAFE_CALL(cutCreateTimer(&int_timer));
    CUT_SAFE_CALL(cutStartTimer(int_timer));

	if (!toEncrypt) {	
		printf("\nDECRYPTION.....\n\n");
		if (keySize != 240)
			aesDecryptHandler128( d_Result, d_Input, inputSize);
		else
			aesDecryptHandler256( d_Result, d_Input, inputSize);
	} else {
		printf("\nENCRYPTION.....\n\n");
		if (keySize != 240)
			aesEncryptHandler128( d_Result, d_Input, inputSize);
		else
			aesEncryptHandler256( d_Result, d_Input, inputSize);
	}
	
	CUT_SAFE_CALL(cutStopTimer(int_timer));
    printf("GPU processing time: %f (ms)\n", cutGetTimerValue(int_timer));
    CUT_SAFE_CALL(cutDeleteTimer(int_timer));

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // copy result from device to host
    CUDA_SAFE_CALL(hipMemcpy(result, d_Result, size_Result, hipMemcpyDeviceToHost) );

    CUT_SAFE_CALL(cutStopTimer(ext_timer));
    printf("Total processing time: %f (ms)\n\n", cutGetTimerValue(ext_timer));
    CUT_SAFE_CALL(cutDeleteTimer(ext_timer));

    // cleanup memory
    CUDA_SAFE_CALL(hipFree(d_Input));
    CUDA_SAFE_CALL(hipFree(d_Key));
    CUDA_SAFE_CALL(hipFree(d_Result));

    return 0;
}

