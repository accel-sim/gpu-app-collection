/************************************************************************************\
 *                                                                                  *
 * Copyright � 2014 Advanced Micro Devices, Inc.                                    *
 * Copyright (c) 2015 Mark D. Hill and David A. Wood                                *
 * All rights reserved.                                                             *
 *                                                                                  *
 * Redistribution and use in source and binary forms, with or without               *
 * modification, are permitted provided that the following are met:                 *
 *                                                                                  *
 * You must reproduce the above copyright notice.                                   *
 *                                                                                  *
 * Neither the name of the copyright holder nor the names of its contributors       *
 * may be used to endorse or promote products derived from this software            *
 * without specific, prior, written permission from at least the copyright holder.  *
 *                                                                                  *
 * You must include the following terms in your license and/or other materials      *
 * provided with the software.                                                      *
 *                                                                                  *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"      *
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE        *
 * IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT, AND FITNESS FOR A       *
 * PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER        *
 * OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,         *
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT  *
 * OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS      *
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN          *
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING  *
 * IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY   *
 * OF SUCH DAMAGE.                                                                  *
 *                                                                                  *
 * Without limiting the foregoing, the software may implement third party           *
 * technologies for which you must obtain licenses from parties other than AMD.     *
 * You agree that AMD has not obtained or conveyed to you, and that you shall       *
 * be responsible for obtaining the rights to use and/or distribute the applicable  *
 * underlying intellectual property rights related to the third party technologies. *
 * These third party technologies are not licensed hereunder.                       *
 *                                                                                  *
 * If you use the software (in whole or in part), you shall adhere to all           *
 * applicable U.S., European, and other export laws, including but not limited to   *
 * the U.S. Export Administration Regulations ("EAR") (15 C.F.R Sections 730-774),  *
 * and E.U. Council Regulation (EC) No 428/2009 of 5 May 2009.  Further, pursuant   *
 * to Section 740.6 of the EAR, you hereby certify that, except pursuant to a       *
 * license granted by the United States Department of Commerce Bureau of Industry   *
 * and Security or as otherwise permitted pursuant to a License Exception under     *
 * the U.S. Export Administration Regulations ("EAR"), you will not (1) export,     *
 * re-export or release to a national of a country in Country Groups D:1, E:1 or    *
 * E:2 any restricted technology, software, or source code you receive hereunder,   *
 * or (2) export to Country Groups D:1, E:1 or E:2 the direct product of such       *
 * technology or software, if such foreign produced direct product is subject to    *
 * national security controls as identified on the Commerce Control List (currently *
 * found in Supplement 1 to Part 774 of EAR).  For the most current Country Group   *
 * listings, or for additional information about the EAR or your obligations under  *
 * those regulations, please refer to the U.S. Bureau of Industry and Security's    *
 * website at http://www.bis.doc.gov/.                                              *
 *                                                                                  *
\************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <algorithm>
#include "../graph_parser/parse.h"
#include "../graph_parser/util.h"
#include "kernel.cu"

#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

#define RANGE 2048

void dump2file(int *adjmatrix, int num_nodes);
void print_vector(int *vector, int num);
void print_vectorf(float *vector, int num);

int main(int argc, char **argv)
{
    char *tmpchar;

    int num_nodes;
    int num_edges;
    int file_format = 1;
    bool directed = 0;

    hipError_t err = hipSuccess;

    // Input arguments
    if (argc == 3) {
        tmpchar = argv[1]; // Graph inputfile
        file_format = atoi(argv[2]); // Choose file format
    } else {
        fprintf(stderr, "You did something wrong!\n");
        exit(1);
    }

    srand(7);

    // Allocate the csr array
    csr_array *csr;

    // Parse the graph into the csr structure
    if (file_format == 1) {
        csr = parseMetis(tmpchar, &num_nodes, &num_edges, directed);
    } else if (file_format == 0) {
        csr = parseCOO(tmpchar, &num_nodes, &num_edges, directed);
    } else {
        fprintf(stderr, "reserve for future");
        exit(1);
    }

    // Allocate the node value array
    int *node_value = (int *)malloc(num_nodes * sizeof(int));
    if (!node_value) fprintf(stderr, "malloc failed node_value\n");

    // Allocate the set array
    int *s_array = (int *)malloc(num_nodes * sizeof(int));
    if (!s_array) fprintf(stderr, "malloc failed node_value\n");

    // Randomize the node values
    for (int i = 0; i < num_nodes; i++) {
        node_value[i] =  rand() % RANGE;
    }

    // Create device side buffers
    int *row_d;
    int *col_d;

    int *c_array_d;
    int *c_array_u_d;
    int *s_array_d;
    int *node_value_d;
    int *min_array_d;
    int *stop_d;

    // Allocate the device-side buffers for the graph
    err = hipMalloc(&row_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc row_d (size:%d) => %s\n",  num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&col_d, num_edges * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc col_d (size:%d) => %s\n",  num_edges , hipGetErrorString(err));
        return -1;
    }

    // Termination variable
    err = hipMalloc(&stop_d, sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc stop_d (size:%d) => %s\n", 1, hipGetErrorString(err));
        return -1;
    }

    // Allocate the device-side buffers for mis
    err = hipMalloc(&min_array_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc min_array_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&c_array_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc c_array_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&c_array_u_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc c_array_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&s_array_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc s_array_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&node_value_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc node_value_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }

    double time1 = gettime();

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

    // Copy data to device-side buffers
    err = hipMemcpy(row_d, csr->row_array, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy row_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(col_d, csr->col_array, num_edges * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy col_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(node_value_d, node_value, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy feature_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    // Work dimensions
    int block_size = 128;
    int num_blocks = (num_nodes + block_size - 1) / block_size;

    dim3 threads(block_size,  1, 1);
    dim3 grid(num_blocks, 1, 1);

    // Launch the initialization kernel
    init <<<grid, threads>>>(s_array_d, c_array_d, c_array_u_d,
                             num_nodes, num_edges);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: init kernel (%s)\n", hipGetErrorString(err));
        return -1;
    }

    // Termination variable
    int stop = 1;
    int iterations = 0;
    while (stop) {
        stop = 0;

        // Copy the termination variable to the device
        err = hipMemcpy(stop_d, &stop, sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "ERROR: write stop_d variable (%s)\n", hipGetErrorString(err));
            return -1;
        }

        // Launch mis1
        mis1 <<<grid, threads>>>(row_d, col_d, node_value_d, s_array_d,
                                 c_array_d, min_array_d, stop_d, num_nodes,
                                 num_edges);

        // Launch mis2
        mis2 <<<grid, threads>>>(row_d, col_d, node_value_d, s_array_d,
                                 c_array_d, c_array_u_d, min_array_d, num_nodes,
                                 num_edges);

        // Launch mis3
        mis3 <<<grid, threads>>>(c_array_u_d, c_array_d, num_nodes);

        // Copy the termination variable back
        err = hipMemcpy(&stop, stop_d, sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "ERROR: read stop_d variable (%s)\n", hipGetErrorString(err));
            return -1;
        }

        iterations++;

    }

    hipDeviceSynchronize();

    err = hipMemcpy(s_array, s_array_d, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy s_array_d failed (%s)\n", hipGetErrorString(err));
        return -1;
    }

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

    double time2 = gettime();

    // Print out the timing characterisitics
    printf("number of iterations: %d\n", iterations);
    printf("kernel + memcpy time %f ms\n", (time2 - time1) * 1000);

#if 0
    // Print the set array
    print_vector(s_array, num_nodes);
#endif

    // Clean up the host-side arrays
    free(node_value);
    free(s_array);
    csr->freeArrays();
    free(csr);

    // Clean up the device-side arrays
    hipFree(row_d);
    hipFree(col_d);
    hipFree(c_array_d);
    hipFree(s_array_d);
    hipFree(node_value_d);
    hipFree(min_array_d);
    hipFree(stop_d);

    return 0;

}

void print_vector(int *vector, int num)
{

    FILE * fp = fopen("result.out", "w");
    if (!fp) {
        printf("ERROR: unable to open result.txt\n");
    }

    for (int i = 0; i < num; i++) {
        fprintf(fp, "%d\n", vector[i]);
    }

    fclose(fp);

}

void print_vectorf(float *vector, int num)
{

    FILE * fp = fopen("result.out", "w");
    if (!fp) {
        printf("ERROR: unable to open result.txt\n");
    }

    for (int i = 0; i < num; i++) {
        fprintf(fp, "%f\n", vector[i]);
    }

    fclose(fp);

}
