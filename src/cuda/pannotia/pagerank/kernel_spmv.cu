
#include <hip/hip_runtime.h>
/************************************************************************************\
 *                                                                                  *
 * Copyright � 2014 Advanced Micro Devices, Inc.                                    *
 * Copyright (c) 2015 Mark D. Hill and David A. Wood                                *
 * All rights reserved.                                                             *
 *                                                                                  *
 * Redistribution and use in source and binary forms, with or without               *
 * modification, are permitted provided that the following are met:                 *
 *                                                                                  *
 * You must reproduce the above copyright notice.                                   *
 *                                                                                  *
 * Neither the name of the copyright holder nor the names of its contributors       *
 * may be used to endorse or promote products derived from this software            *
 * without specific, prior, written permission from at least the copyright holder.  *
 *                                                                                  *
 * You must include the following terms in your license and/or other materials      *
 * provided with the software.                                                      *
 *                                                                                  *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"      *
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE        *
 * IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT, AND FITNESS FOR A       *
 * PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER        *
 * OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,         *
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT  *
 * OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS      *
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN          *
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING  *
 * IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY   *
 * OF SUCH DAMAGE.                                                                  *
 *                                                                                  *
 * Without limiting the foregoing, the software may implement third party           *
 * technologies for which you must obtain licenses from parties other than AMD.     *
 * You agree that AMD has not obtained or conveyed to you, and that you shall       *
 * be responsible for obtaining the rights to use and/or distribute the applicable  *
 * underlying intellectual property rights related to the third party technologies. *
 * These third party technologies are not licensed hereunder.                       *
 *                                                                                  *
 * If you use the software (in whole or in part), you shall adhere to all           *
 * applicable U.S., European, and other export laws, including but not limited to   *
 * the U.S. Export Administration Regulations ("EAR") (15 C.F.R Sections 730-774),  *
 * and E.U. Council Regulation (EC) No 428/2009 of 5 May 2009.  Further, pursuant   *
 * to Section 740.6 of the EAR, you hereby certify that, except pursuant to a       *
 * license granted by the United States Department of Commerce Bureau of Industry   *
 * and Security or as otherwise permitted pursuant to a License Exception under     *
 * the U.S. Export Administration Regulations ("EAR"), you will not (1) export,     *
 * re-export or release to a national of a country in Country Groups D:1, E:1 or    *
 * E:2 any restricted technology, software, or source code you receive hereunder,   *
 * or (2) export to Country Groups D:1, E:1 or E:2 the direct product of such       *
 * technology or software, if such foreign produced direct product is subject to    *
 * national security controls as identified on the Commerce Control List (currently *
 * found in Supplement 1 to Part 774 of EAR).  For the most current Country Group   *
 * listings, or for additional information about the EAR or your obligations under  *
 * those regulations, please refer to the U.S. Bureau of Industry and Security's    *
 * website at http://www.bis.doc.gov/.                                              *
 *                                                                                  *
\************************************************************************************/

/**
 * @brief   inibuffer
 * @param   page_rank1   PageRank array 1
 * @param   page_rank2   PageRank array 2
 * @param   num_nodes    number of vertices
 */
__global__ void
inibuffer(float *page_rank1, float *page_rank2, const int num_nodes)
{
    // Get my workitem id
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize two pagerank arrays
    if (tid < num_nodes) {
        page_rank1[tid] = 1 / (float)num_nodes;
        page_rank2[tid] = 0.0f;
    }
}

/**
 * @brief   inicsr
 * @param   row        csr pointer array
 * @param   col        csr col array
 * @param   data       csr weigh array
 * @param   col_cnt    array for #. out-going edges
 * @param   num_nodes  number of vertices
 * @param   num_edges  number of edges
 */
__global__ void
inicsr(int *row, int *col, float *data, int *col_cnt, int num_nodes,
       int num_edges)
{
    // Get my workitem id
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < num_nodes) {
        // Get the starting and ending pointers
        int start = row[tid];
        int end;
        if (tid + 1 < num_nodes) {
            end = row[tid + 1] ;
        } else {
            end = num_edges;
        }

        int nid;
        // Navigate one row of data
        for (int edge = start; edge < end; edge++) {
            nid = col[edge];
            // Each neighbor will get equal amount of pagerank
            data[edge] = 1.0 / (float)col_cnt[nid];
        }
    }
}

/**
 * @brief   spmv_csr_scalar_kernel (simple spmv)
 * @param   num_nodes  number of vertices
 * @param   row        csr pointer array
 * @param   col        csr col array
 * @param   data       csr weigh array
 * @param   x          input vector
 * @param   y          output vector
 */
__global__ void
spmv_csr_scalar_kernel(const int num_nodes, int *row, int *col, float *data,
                       float *x, float *y)
{
    // Get my workitem id
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < num_nodes) {
        // Get the start and end pointers
        int row_start = row[tid];
        int row_end = row[tid + 1];
        float sum = 0;
        //navigate one row and sum all the elements
        for (int j = row_start; j < row_end; j++) {
            sum += data[j] * x[col[j]];
        }
        y[tid] += sum;
    }
}

/**
 * @brief   pagerank2
 * @param   page_rank1   PageRank array 1
 * @param   page_rank2   PageRank array 2
 * @param   num_nodes    number of vertices
 */
__global__ void
pagerank2(float *page_rank1, float *page_rank2, const int num_nodes)
{
    // Get my workitem id
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // Update pagerank value with damping factor
    if (tid < num_nodes) {
        page_rank1[tid]	= 0.15f / (float)num_nodes + 0.85f * page_rank2[tid];
        page_rank2[tid] = 0.0f;
    }
}


