/************************************************************************************\ 
 *                                                                                  *
 * Copyright © 2014 Advanced Micro Devices, Inc.                                    *
 * Copyright (c) 2015 Mark D. Hill and David A. Wood                                *
 * All rights reserved.                                                             *
 *                                                                                  *
 * Redistribution and use in source and binary forms, with or without               *
 * modification, are permitted provided that the following are met:                 *
 *                                                                                  *
 * You must reproduce the above copyright notice.                                   *
 *                                                                                  *
 * Neither the name of the copyright holder nor the names of its contributors       *
 * may be used to endorse or promote products derived from this software            *
 * without specific, prior, written permission from at least the copyright holder.  *
 *                                                                                  *
 * You must include the following terms in your license and/or other materials      *
 * provided with the software.                                                      *
 *                                                                                  *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"      *
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE        *
 * IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT, AND FITNESS FOR A       *
 * PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER        *
 * OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,         *
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT  *
 * OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS      *
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN          *
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING  *
 * IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY   *
 * OF SUCH DAMAGE.                                                                  *
 *                                                                                  *
 * Without limiting the foregoing, the software may implement third party           *
 * technologies for which you must obtain licenses from parties other than AMD.     *
 * You agree that AMD has not obtained or conveyed to you, and that you shall       *
 * be responsible for obtaining the rights to use and/or distribute the applicable  *
 * underlying intellectual property rights related to the third party technologies. *
 * These third party technologies are not licensed hereunder.                       *
 *                                                                                  *
 * If you use the software (in whole or in part), you shall adhere to all           *
 * applicable U.S., European, and other export laws, including but not limited to   *
 * the U.S. Export Administration Regulations ("EAR"�) (15 C.F.R Sections 730-774),  *
 * and E.U. Council Regulation (EC) No 428/2009 of 5 May 2009.  Further, pursuant   *
 * to Section 740.6 of the EAR, you hereby certify that, except pursuant to a       *
 * license granted by the United States Department of Commerce Bureau of Industry   *
 * and Security or as otherwise permitted pursuant to a License Exception under     *
 * the U.S. Export Administration Regulations ("EAR"), you will not (1) export,     *
 * re-export or release to a national of a country in Country Groups D:1, E:1 or    *
 * E:2 any restricted technology, software, or source code you receive hereunder,   *
 * or (2) export to Country Groups D:1, E:1 or E:2 the direct product of such       *
 * technology or software, if such foreign produced direct product is subject to    *
 * national security controls as identified on the Commerce Control List (currently *
 * found in Supplement 1 to Part 774 of EAR).  For the most current Country Group   *
 * listings, or for additional information about the EAR or your obligations under  *
 * those regulations, please refer to the U.S. Bureau of Industry and Security's    *
 * website at http://www.bis.doc.gov/.                                              *
 *                                                                                  *
\************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <omp.h>
#include "../graph_parser/util.h"
#include "kernel_block.cu"
#include "parse.h"

#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

#ifdef GEM5_FUSION
#define MAX_ITERS 36
#else
#include <stdint.h>
#define MAX_ITERS INT_MAX
#endif

#define BIGNUM 999999
#define TRUE 1
#define FALSE 0

int main(int argc, char **argv)
{
    char *tmpchar;
    bool verify_results = false;

    int num_nodes;
    int num_edges;

    hipError_t err = hipSuccess;

    // Get program input
    if (argc >= 2) {
        tmpchar = argv[1];  // Graph input file
    } else {
        fprintf(stderr, "You did something wrong!\n");
        exit(1);
    }

    if (argc >= 3) {
        if (atoi(argv[2]) == 1) {
            verify_results = true;
        }
    }

    // Parse the adjacency matrix
    int *adjmatrix = parse_graph_file(&num_nodes, &num_edges, tmpchar);
    int dim = num_nodes;

    // Initialize the distance matrix
    int *distmatrix = (int *)malloc(dim * dim * sizeof(int));
    if (!distmatrix) fprintf(stderr, "malloc failed - distmatrix\n");

    // Initialize the result matrix
    int *result = (int *)malloc(dim * dim * sizeof(int));
    if (!result) fprintf(stderr, "malloc failed - result\n");

    // TODO: Now only supports integer weights
    // Setup the input matrix
    for (int i = 0 ; i < dim; i++) {
        for (int j = 0 ; j < dim; j++) {
            if (i == j) {
                // Diagonal
                distmatrix[i * dim + j] = 0;
            } else if (adjmatrix[i * dim + j] == -1) {
                // Without edge
                distmatrix[i * dim + j] = BIGNUM;
            } else {
                // With edge
                distmatrix[i * dim + j] = adjmatrix[i * dim + j];
            }
        }
    }

    int *dist_d;

    // Create device-side FW buffers
    err = hipMalloc(&dist_d, dim * dim * sizeof(int));
    if (err != hipSuccess) {
        printf("ERROR: hipMalloc dist_d (size:%d) => %d\n",  dim * dim , err);
        return -1;
    }

    double timer1 = gettime();

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

    // Copy the dist matrix to the device
    err = hipMemcpy(dist_d, distmatrix, dim * dim * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy feature_d (size:%d) => %d\n", dim * dim, err);
        return -1;
    }

    // Work dimension
    int block_size = 16;
    int num_blk_per_dim = num_nodes / block_size;
    dim3 threads(block_size, block_size, 1);
    dim3 grid_dia(block_size, block_size, 1);
    dim3 grid_strip_x(num_blk_per_dim, block_size, 1);
    dim3 grid_strip_y(block_size, num_blk_per_dim, 1);
    dim3 grid_remain(num_blk_per_dim, num_blk_per_dim, 1);

    double timer3 = gettime();
    // Main computation loop
    for (int blk = 0; blk < num_blk_per_dim && blk < MAX_ITERS; blk++) {
        floydwarshall_dia_block<<<grid_dia, threads>>>(dist_d, blk, dim);
        floydwarshall_strip_blocks_x<<<grid_strip_x, threads>>>(dist_d, blk, dim);
        floydwarshall_strip_blocks_y<<<grid_strip_y, threads>>>(dist_d, blk, dim);
        floydwarshall_remaining_blocks<<<grid_remain, threads>>>(dist_d, blk, dim);
    }
    hipDeviceSynchronize();

    double timer4 = gettime();
    err = hipMemcpy(result, dist_d, dim * dim * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: read back dist_d %d failed\n", err);
        return -1;
    }

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

    double timer2 = gettime();

    printf("kernel time = %lf ms\n", (timer4 - timer3) * 1000);
    printf("kernel + memcpy time = %lf ms\n", (timer2 - timer1) * 1000);

    if (verify_results) {
        // Below is the verification part
        // Calculate on the CPU
        int *dist = distmatrix;
        for (int k = 0; k < dim; k++) {
            for (int i = 0; i < dim; i++) {
                for (int j = 0; j < dim; j++) {
                    if (dist[i * dim + k] + dist[k * dim + j] < dist[i * dim + j]) {
                        dist[i * dim + j] = dist[i * dim + k] + dist[k * dim + j];
                    }
                }
            }
        }

        // Compare results
        bool check_flag = 0;
        for (int i = 0; i < dim; i++) {
            for (int j = 0; j < dim; j++) {
                if (dist[i * dim + j] !=  result[i * dim + j]) {
                    printf("mismatch at (%d, %d)\n", i, j);
                    check_flag = 1;
                }
            }
        }
        // If there is mismatch, report
        if (check_flag) {
            fprintf(stderr, "WARNING: Produced incorrect results!\n");
        } else {
            printf("Results are correct!\n");
        }
    }

    printf("Finishing Floyd-Warshall\n");

    // Free host-side buffers
    free(adjmatrix);
    free(result);
    free(distmatrix);

    // Free CUDA buffers
    hipFree(dist_d);

    return 0;

}
