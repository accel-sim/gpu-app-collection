/************************************************************************************\ 
 *                                                                                  *
 * Copyright © 2014 Advanced Micro Devices, Inc.                                    *
 * Copyright (c) 2015 Mark D. Hill and David A. Wood                                *
 * All rights reserved.                                                             *
 *                                                                                  *
 * Redistribution and use in source and binary forms, with or without               *
 * modification, are permitted provided that the following are met:                 *
 *                                                                                  *
 * You must reproduce the above copyright notice.                                   *
 *                                                                                  *
 * Neither the name of the copyright holder nor the names of its contributors       *
 * may be used to endorse or promote products derived from this software            *
 * without specific, prior, written permission from at least the copyright holder.  *
 *                                                                                  *
 * You must include the following terms in your license and/or other materials      *
 * provided with the software.                                                      *
 *                                                                                  *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"      *
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE        *
 * IMPLIED WARRANTIES OF MERCHANTABILITY, NON-INFRINGEMENT, AND FITNESS FOR A       *
 * PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER        *
 * OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,         *
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT  *
 * OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS      *
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN          *
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING  *
 * IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY   *
 * OF SUCH DAMAGE.                                                                  *
 *                                                                                  *
 * Without limiting the foregoing, the software may implement third party           *
 * technologies for which you must obtain licenses from parties other than AMD.     *
 * You agree that AMD has not obtained or conveyed to you, and that you shall       *
 * be responsible for obtaining the rights to use and/or distribute the applicable  *
 * underlying intellectual property rights related to the third party technologies. *
 * These third party technologies are not licensed hereunder.                       *
 *                                                                                  *
 * If you use the software (in whole or in part), you shall adhere to all           *
 * applicable U.S., European, and other export laws, including but not limited to   *
 * the U.S. Export Administration Regulations ("EAR") (15 C.F.R Sections 730-774),  *
 * and E.U. Council Regulation (EC) No 428/2009 of 5 May 2009.  Further, pursuant   *
 * to Section 740.6 of the EAR, you hereby certify that, except pursuant to a       *
 * license granted by the United States Department of Commerce Bureau of Industry   *
 * and Security or as otherwise permitted pursuant to a License Exception under     *
 * the U.S. Export Administration Regulations ("EAR"), you will not (1) export,     *
 * re-export or release to a national of a country in Country Groups D:1, E:1 or    *
 * E:2 any restricted technology, software, or source code you receive hereunder,   *
 * or (2) export to Country Groups D:1, E:1 or E:2 the direct product of such       *
 * technology or software, if such foreign produced direct product is subject to    *
 * national security controls as identified on the Commerce Control List (currently *
 * found in Supplement 1 to Part 774 of EAR).  For the most current Country Group   *
 * listings, or for additional information about the EAR or your obligations under  *
 * those regulations, please refer to the U.S. Bureau of Industry and Security's    *
 * website at http://www.bis.doc.gov/.                                              *
 *                                                                                  *
\************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "../graph_parser/parse.h"
#include "../graph_parser/util.h"
#include "kernel_maxmin.cu"

#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

#define RANGE 2048

void print_vector(int *vector, int num);

int main(int argc, char **argv)
{
    char *tmpchar;

    int num_nodes;
    int num_edges;
    int file_format = 1;
    bool directed = 0;

    hipError_t err = hipSuccess;

    if (argc == 3) {
        tmpchar = argv[1];  //graph inputfile
        file_format = atoi(argv[2]); //graph format
    } else {
        fprintf(stderr, "You did something wrong!\n");
        exit(1);
    }

    srand(7);

    // Allocate the CSR structure
    csr_array *csr;

    // Parse graph file and store into a CSR format
    if (file_format == 1)
        csr = parseMetis(tmpchar, &num_nodes, &num_edges, directed);
    else if (file_format == 0)
        csr = parseCOO(tmpchar, &num_nodes, &num_edges, directed);
    else {
        printf("reserve for future");
        exit(1);
    }

    // Allocate the vertex value array
    int *node_value = (int *)malloc(num_nodes * sizeof(int));
    if (!node_value) fprintf(stderr, "node_value malloc failed\n");
    // Allocate the color array
    int *color = (int *)malloc(num_nodes * sizeof(int));
    if (!color) fprintf(stderr, "color malloc failed\n");

    // Initialize all the colors to -1
    // Randomize the value for each vertex
    for (int i = 0; i < num_nodes; i++) {
        color[i] = -1;
        node_value[i] = rand() % RANGE;
    }

    int *row_d;
    int *col_d;
    int *max_d;
    int *min_d;

    int *color_d;
    int *node_value_d;
    int *stop_d;

    // Create device-side buffers for the graph
    err = hipMalloc(&row_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc row_d (size:%d) => %s\n",  num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&col_d, num_edges * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc col_d (size:%d): %s\n",  num_edges , hipGetErrorString(err));
        return -1;
    }

    // Termination variable
    err = hipMalloc(&stop_d, sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc stop_d (size:%d) => %s\n",  1 , hipGetErrorString(err));
        return -1;
    }

    // Create device-side buffers for color
    err = hipMalloc(&color_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc color_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&node_value_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc node_value_d (size:%d) => %s\n", num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&max_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc max_d (size:%d) => %s\n",  num_nodes , hipGetErrorString(err));
        return -1;
    }
    err = hipMalloc(&min_d, num_nodes * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc min_d (size:%d) => %s\n",  num_nodes , hipGetErrorString(err));
        return -1;
    }

    // Copy data to device-side buffers
    double timer1 = gettime();

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

    err = hipMemcpy(color_d, color, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy color_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(row_d, csr->row_array, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy row_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(col_d, csr->col_array, num_edges * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy col_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(node_value_d, node_value, num_nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy node_value_d (size:%d) => %s\n", num_nodes, hipGetErrorString(err));
        return -1;
    }

    int block_size = 256;
    int num_blocks = (num_nodes + block_size - 1) / block_size;

    // Set up kernel dimensions
    dim3 threads(block_size,  1, 1);
    dim3 grid(num_blocks, 1,  1);

    int stop = 1;
    int graph_color = 1;

    // Initialize arrays
    ini<<< grid, threads >>>(max_d, min_d, num_nodes);

    // Main computation loop
    double timer3 = gettime();

    while (stop) {

        stop = 0;

        // Copy the termination variable to the device
        err = hipMemcpy(stop_d, &stop, sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "ERROR: write stop_d: %s\n", hipGetErrorString(err));
        }

        // Launch the color kernel 1
        color1 <<< grid, threads >>>(row_d, col_d, node_value_d, color_d,
                                     stop_d, max_d, min_d, graph_color,
                                     num_nodes, num_edges);

        // Launch the color kernel 2
        color2 <<< grid, threads >>>(node_value_d, color_d, max_d, min_d,
                                     graph_color, num_nodes, num_edges);

        err = hipMemcpy(&stop, stop_d, sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "ERROR: read stop_d: %s\n", hipGetErrorString(err));
        }

        // Update the color label for the next iter
        graph_color = graph_color + 2;

    }
    hipDeviceSynchronize();

    double timer4 = gettime();

    // Copy back the color array
    err = hipMemcpy(color, color_d, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("ERROR: hipMemcpy(): %s\n", hipGetErrorString(err));
        return -1;
    }

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

    double timer2 = gettime();

    // Print out color and timing statistics
    printf("total number of colors used: %d\n", graph_color);
    printf("kernel time = %lf ms\n", (timer4 - timer3) * 1000);
    printf("kernel + memcpy time = %lf ms\n", (timer2 - timer1) * 1000);

#if 1
    // Dump the color array into an output file
    print_vector(color, num_nodes);
#endif

    // Free host-side buffers
    free(node_value);
    free(color);
    csr->freeArrays();
    free(csr);

    // Free CUDA buffers
    hipFree(row_d);
    hipFree(col_d);
    hipFree(max_d);
    hipFree(color_d);
    hipFree(node_value_d);
    hipFree(stop_d);

    return 0;

}

void print_vector(int *vector, int num)
{
    FILE * fp = fopen("result.out", "w");
    if (!fp) {
        printf("ERROR: unable to open result.txt\n");
    }

    for (int i = 0; i < num; i++)
        fprintf(fp, "%d: %d\n", i + 1, vector[i]);

    fclose(fp);
}
