/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

/* Problem size. */
#define NX 1200
#define NY 1200


/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

#ifndef M_PI
#define M_PI 3.14159
#endif

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;

void init_array(DATA_TYPE *x, DATA_TYPE *A)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*(j)) / NX;
		}
	}
}

__global__ void atax_kernel1(DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NX)
	{
		int j;
		for(j=0; j < NY; j++)
		{
			tmp[i] += A[i * NY + j] * x[j];
		}
	}
}

__global__ void atax_kernel2(DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < NY)
	{
		int i;
		for(i=0; i < NX; i++)
		{
			y[j] += A[i * NY + j] * tmp[i];
		}
	}
}

void ataxGpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);

	atax_kernel1<<< grid1, block >>>(A,x,tmp);

	atax_kernel2<<< grid2, block >>>(A,y,tmp);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
}


int main(int argc, char** argv)
{
	DATA_TYPE* A;
	DATA_TYPE* x;
	DATA_TYPE* y;
	DATA_TYPE* tmp;

	hipMallocManaged( &A, NX*NY*sizeof(DATA_TYPE));
	hipMallocManaged( &x, NY*sizeof(DATA_TYPE));
	hipMallocManaged( &y, NY*sizeof(DATA_TYPE));
	hipMallocManaged( &tmp, NX*sizeof(DATA_TYPE));

	init_array(x, A);

	ataxGpu(A, x, y, tmp);
	
	FILE *fp;

	fp = fopen("result_ATAX.txt","a+");

	for(int i = 0; i < NY; i++) {
		fprintf(fp, "%lf\n", y[i]);
	}
	
	fclose(fp);

	hipFree(A);
	hipFree(x);
	hipFree(y);
	hipFree(tmp);

  	return 0;
}

