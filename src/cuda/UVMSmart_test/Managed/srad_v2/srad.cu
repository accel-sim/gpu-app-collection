#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "srad.h"

// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include "srad_kernel.cu"

void random_matrix(float *I, int rows, int cols);
void runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <rows> <cols> <y1> <y2> <x1> <x2> <lamda> <no. of iter>\n", argv[0]);
	fprintf(stderr, "\t<rows>   - number of rows\n");
	fprintf(stderr, "\t<cols>    - number of cols\n");
	fprintf(stderr, "\t<y1> 	 - y1 value of the speckle\n");
	fprintf(stderr, "\t<y2>      - y2 value of the speckle\n");
	fprintf(stderr, "\t<x1>       - x1 value of the speckle\n");
	fprintf(stderr, "\t<x2>       - x2 value of the speckle\n");
	fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
	fprintf(stderr, "\t<no. of iter>   - number of iterations\n");
	
	exit(1);
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
    runTest( argc, argv);

    return EXIT_SUCCESS;
}


void
runTest( int argc, char** argv) 
{
    	int rows, cols, size_I, size_R, niter = 10, iter;
    	float *I, lambda, q0sqr, sum, sum2, tmp, meanROI,varROI ;

	float *J_shared;
    	float *C_cuda;
	float *E_C, *W_C, *N_C, *S_C;
	
	unsigned int r1, r2, c1, c2;
	float *c;
    
	
 
	if (argc == 9)
	{
		rows = atoi(argv[1]);  //number of rows in the domain
		cols = atoi(argv[2]);  //number of cols in the domain
		if ((rows%16!=0) || (cols%16!=0)){
			fprintf(stderr, "rows and cols must be multiples of 16\n");
			exit(1);
		}
		r1   = atoi(argv[3]);  //y1 position of the speckle
		r2   = atoi(argv[4]);  //y2 position of the speckle
		c1   = atoi(argv[5]);  //x1 position of the speckle
		c2   = atoi(argv[6]);  //x2 position of the speckle
		lambda = atof(argv[7]); //Lambda value
		niter = atoi(argv[8]); //number of iterations
		
	}
    	else{
		usage(argc, argv);
    	}



	size_I = cols * rows;
    	size_R = (r2-r1+1)*(c2-c1+1);   

	I = (float *)malloc( size_I * sizeof(float) );
	c  = (float *)malloc(sizeof(float)* size_I) ;

	//Allocate device memory
    	hipMalloc((void**)& C_cuda, sizeof(float)* size_I);
	hipMalloc((void**)& E_C, sizeof(float)* size_I);
	hipMalloc((void**)& W_C, sizeof(float)* size_I);
	hipMalloc((void**)& S_C, sizeof(float)* size_I);
	hipMalloc((void**)& N_C, sizeof(float)* size_I);
	
	//Allocate managed memory
    	hipMallocManaged((void**)& J_shared, sizeof(float)* size_I);
	
	printf("Randomizing the input matrix\n");
	//Generate a random matrix
	random_matrix(I, rows, cols);

    	for (int k = 0;  k < size_I; k++ ) {
     		J_shared[k] = (float)exp(I[k]) ;
    	}
	printf("Start the SRAD main loop\n");

#ifdef PREF
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	hipStream_t stream2;
	hipStreamCreate(&stream2);
#endif
	for (iter=0; iter< niter; iter++) {     
		sum=0; sum2=0;
        	for (int i=r1; i<=r2; i++) {
            		for (int j=c1; j<=c2; j++) {
                		tmp   = J_shared[i * cols + j];
                		sum  += tmp ;
                		sum2 += tmp*tmp;
            		}
        	}
        	meanROI = sum / size_R;
        	varROI  = (sum2 / size_R) - meanROI*meanROI;
        	q0sqr   = varROI / (meanROI*meanROI);

#ifdef PREF
		int device = -1;
		hipGetDevice(&device);
		hipMemPrefetchAsync(J_shared, sizeof(float)* size_I, device, stream1);
#endif
		//Currently the input size must be divided by 16 - the block size
		int block_x = cols/BLOCK_SIZE ;
    		int block_y = rows/BLOCK_SIZE ;

    		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid(block_x , block_y);

		//Run kernels
#ifdef PREF
		srad_cuda_1<<<dimGrid, dimBlock, 0, stream2>>>(E_C, W_C, N_C, S_C, J_shared, C_cuda, cols, rows, q0sqr); 
		srad_cuda_2<<<dimGrid, dimBlock, 0, stream2>>>(E_C, W_C, N_C, S_C, J_shared, C_cuda, cols, rows, lambda, q0sqr); 
#else
		srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_shared, C_cuda, cols, rows, q0sqr); 
		srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_shared, C_cuda, cols, rows, lambda, q0sqr); 
#endif

		// Wait for GPU to finish before accessing on host
		hipDeviceSynchronize();

	}

    hipDeviceSynchronize();

#define OUTPUT

#ifdef OUTPUT
    //Printing output	
    printf("Printing Output:\n"); 
    for( int i = 0 ; i < rows ; i++){
	for ( int j = 0 ; j < cols ; j++){
        	printf("%.5f ", J_shared[i * cols + j]); 
	}	
   	printf("\n"); 
    }
#endif 

	printf("Computation Done\n");

	free(I);
	free(c);

    	hipFree(C_cuda);
	hipFree(E_C);
	hipFree(W_C);
	hipFree(N_C);
	hipFree(S_C);

	hipFree(J_shared);
  
}


void random_matrix(float *I, int rows, int cols){
    
	srand(7);
	
	for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
		 I[i * cols + j] = rand()/(float)RAND_MAX ;
		}
	}

}

