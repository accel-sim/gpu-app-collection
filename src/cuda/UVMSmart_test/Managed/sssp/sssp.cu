
#include <hip/hip_runtime.h>
#include <sstream>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <float.h>

//#include "Utilities.cuh"

#define NUM_ASYNCHRONOUS_ITERATIONS 5  // Number of async loop iterations before attempting to read results back
#define MAX_ITERATION 15
#define BLOCK_SIZE 16

__host__ __device__ int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }


/*********************************/
/* ATOMIC MIN FUNCTION ON FLOATS */
/*********************************/
__device__ float atomicMin(float* address, float val)
{
	int* address_as_i = (int*)address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
			__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }


/***********************/
/* GRAPHDATA STRUCTURE */
/***********************/
// --- The graph data structure is an adjacency list.
typedef struct {

    // --- Contains the integer offset to point to the edge list for each vertex
    int *vertexArray;

    // --- Overall number of vertices
    int numVertices;

    // --- Contains the "destination" vertices each edge is attached to
    int *edgeArray;

    // --- Overall number of edges
    int numEdges;

    // --- Contains the weight of each edge
    float *weightArray;

} GraphData;

/**********************************/
/* GENERATE RANDOM GRAPH FUNCTION */
/**********************************/
void generateRandomGraph(GraphData *graph, int numVertices, int neighborsPerVertex) {

    


    graph -> numVertices    = numVertices;
    //graph -> vertexArray    = (int *)malloc(graph -> numVertices * sizeof(int));
    graph -> numEdges       = numVertices * neighborsPerVertex;
    //graph -> edgeArray      = (int *)malloc(graph -> numEdges    * sizeof(int));
    //graph -> weightArray    = (float *)malloc(graph -> numEdges  * sizeof(float));

    gpuErrchk(hipMallocManaged(&graph -> vertexArray,    sizeof(int)   * graph -> numVertices));
    gpuErrchk(hipMallocManaged(&graph -> edgeArray,  sizeof(int)   * graph -> numEdges));
    gpuErrchk(hipMallocManaged(&graph -> weightArray,    sizeof(float) * graph -> numEdges));


    for (int i = 0; i < graph -> numVertices; i++) graph -> vertexArray[i] = i * neighborsPerVertex;

    int *tempArray = (int *)malloc(neighborsPerVertex * sizeof(int));
    for (int k = 0; k < numVertices; k++) {
        for (int l = 0; l < neighborsPerVertex; l++) tempArray[l] = INT_MAX;
        for (int l = 0; l < neighborsPerVertex; l++) {
            bool goOn = false;
            int temp;
            while (goOn == false) {
                goOn = true;
                temp = (rand() % graph->numVertices);
                for (int t = 0; t < neighborsPerVertex; t++)
                    if (temp == tempArray[t]) goOn = false;
                if (temp == k) goOn = false;
                if (goOn == true) tempArray[l] = temp;
            }
            graph -> edgeArray  [k * neighborsPerVertex + l] = temp;
            graph -> weightArray[k * neighborsPerVertex + l] = (float)(rand() % 1000) / 1000.0f;
	    //printf("%lf\n",graph -> weightArray[k * neighborsPerVertex + l]);
        }
    }
}

/************************/
/* minDistance FUNCTION */
/************************/
// --- Finds the vertex with minimum distance value, from the set of vertices not yet included in shortest path tree
int minDistance(float *shortestDistances, bool *finalizedVertices, const int sourceVertex, const int N) {

    // --- Initialize minimum value
    int minIndex = sourceVertex;
    float min = FLT_MAX;

    for (int v = 0; v < N; v++)
        if (finalizedVertices[v] == false && shortestDistances[v] <= min) min = shortestDistances[v], minIndex = v;

    return minIndex;
}

/************************/
/* dijkstraCPU FUNCTION */
/************************/
void dijkstraCPU(float *graph, float *h_shortestDistances, int sourceVertex, const int N) {

    // --- h_finalizedVertices[i] is true if vertex i is included in the shortest path tree
    //     or the shortest distance from the source node to i is finalized
    bool *h_finalizedVertices = (bool *)malloc(N * sizeof(bool));

    // --- Initialize h_shortestDistancesances as infinite and h_shortestDistances as false
    for (int i = 0; i < N; i++) h_shortestDistances[i] = FLT_MAX, h_finalizedVertices[i] = false;

    // --- h_shortestDistancesance of the source vertex from itself is always 0
    h_shortestDistances[sourceVertex] = 0.f;

    // --- Dijkstra iterations
    for (int iterCount = 0; iterCount < N - 1; iterCount++) {

        // --- Selecting the minimum distance vertex from the set of vertices not yet
        //     processed. currentVertex is always equal to sourceVertex in the first iteration.
        int currentVertex = minDistance(h_shortestDistances, h_finalizedVertices, sourceVertex, N);

        // --- Mark the current vertex as processed
        h_finalizedVertices[currentVertex] = true;

        // --- Relaxation loop
        for (int v = 0; v < N; v++) {

            // --- Update dist[v] only if it is not in h_finalizedVertices, there is an edge
            //     from u to v, and the cost of the path from the source vertex to v through
            //     currentVertex is smaller than the current value of h_shortestDistances[v]
            if (!h_finalizedVertices[v] &&
                graph[currentVertex * N + v] &&
                h_shortestDistances[currentVertex] != FLT_MAX &&
                h_shortestDistances[currentVertex] + graph[currentVertex * N + v] < h_shortestDistances[v])

                h_shortestDistances[v] = h_shortestDistances[currentVertex] + graph[currentVertex * N + v];
        }
    }
}

/***************************/
/* MASKARRAYEMPTY FUNCTION */
/***************************/
// --- Check whether all the vertices have been finalized. This tells the algorithm whether it needs to continue running or not.
bool allFinalizedVertices(bool *finalizedVertices, int numVertices) {

    for (int i = 0; i < numVertices; i++)  if (finalizedVertices[i] == true) { return false; }

    return true;
}

/*************************/
/* ARRAY INITIALIZATIONS */
/*************************/
__global__ void initializeArrays(bool * __restrict__ d_finalizedVertices, float* __restrict__ d_shortestDistances, float* __restrict__ d_updatingShortestDistances,
                                 const int sourceVertex, const int numVertices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (sourceVertex == tid) {

            d_finalizedVertices[tid]            = true;
            d_shortestDistances[tid]            = 0.f;
            d_updatingShortestDistances[tid]    = 0.f; }

        else {

            d_finalizedVertices[tid]            = false;
            d_shortestDistances[tid]            = FLT_MAX;
            d_updatingShortestDistances[tid]    = FLT_MAX;
        }
    }
}

/**************************/
/* DIJKSTRA GPU KERNEL #1 */
/**************************/
__global__  void Kernel1(const int * __restrict__ vertexArray, const int* __restrict__ edgeArray,
                         const float * __restrict__ weightArray, bool * __restrict__ finalizedVertices, float* __restrict__ shortestDistances,
                         float * __restrict__ updatingShortestDistances, const int numVertices, const int numEdges) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (finalizedVertices[tid] == true) {

            finalizedVertices[tid] = false;

            int edgeStart = vertexArray[tid], edgeEnd;

            if (tid + 1 < (numVertices)) edgeEnd = vertexArray[tid + 1];
            else                         edgeEnd = numEdges;

            for (int edge = edgeStart; edge < edgeEnd; edge++) {
                int nid = edgeArray[edge];
                atomicMin(&updatingShortestDistances[nid], shortestDistances[tid] + weightArray[edge]);
            }
        }
    }
}

/**************************/
/* DIJKSTRA GPU KERNEL #1 */
/**************************/
__global__  void Kernel2(const int * __restrict__ vertexArray, const int * __restrict__ edgeArray, const float* __restrict__ weightArray,
                         bool * __restrict__ finalizedVertices, float* __restrict__ shortestDistances, float* __restrict__ updatingShortestDistances,
                         const int numVertices) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < numVertices) {

        if (shortestDistances[tid] > updatingShortestDistances[tid]) {
            shortestDistances[tid] = updatingShortestDistances[tid];
            finalizedVertices[tid] = true; }

        updatingShortestDistances[tid] = shortestDistances[tid];
    }
}

/************************/
/* dijkstraGPU FUNCTION */
/************************/
void dijkstraGPU(GraphData *graph, const int sourceVertex, float * __restrict__ h_shortestDistances) {

    // --- Create device-side adjacency-list, namely, vertex array Va, edge array Ea and weight array Wa from G(V,E,W)
    //int     *d_vertexArray;         gpuErrchk(cudaMalloc(&d_vertexArray,    sizeof(int)   * graph -> numVertices));
    //int     *d_edgeArray;           gpuErrchk(cudaMalloc(&d_edgeArray,  sizeof(int)   * graph -> numEdges));
    //float   *d_weightArray;         gpuErrchk(cudaMalloc(&d_weightArray,    sizeof(float) * graph -> numEdges));

    // --- Copy adjacency-list to the device
    //gpuErrchk(cudaMemcpy(d_vertexArray, graph -> vertexArray, sizeof(int)   * graph -> numVertices, cudaMemcpyHostToDevice));
    //gpuErrchk(cudaMemcpy(d_edgeArray,   graph -> edgeArray,   sizeof(int)   * graph -> numEdges,    cudaMemcpyHostToDevice));
    //gpuErrchk(cudaMemcpy(d_weightArray, graph -> weightArray, sizeof(float) * graph -> numEdges,    cudaMemcpyHostToDevice));

    // --- Create mask array Ma, cost array Ca and updating cost array Ua of size V
    bool    *d_finalizedVertices;           gpuErrchk(hipMalloc(&d_finalizedVertices,       sizeof(bool)   * graph->numVertices));
    //float   *d_shortestDistances;           gpuErrchk(cudaMallocManaged(&d_shortestDistances,       sizeof(float) * graph->numVertices));
    float   *d_updatingShortestDistances;   gpuErrchk(hipMallocManaged(&d_updatingShortestDistances, sizeof(float) * graph->numVertices));

    bool *h_finalizedVertices = (bool *)malloc(sizeof(bool) * graph->numVertices);

    // --- Initialize mask Ma to false, cost array Ca and Updating cost array Ua to \u221e
    initializeArrays <<<iDivUp(graph->numVertices, BLOCK_SIZE), BLOCK_SIZE >>>(d_finalizedVertices, h_shortestDistances,
                                                            d_updatingShortestDistances, sourceVertex, graph -> numVertices);
    //gpuErrchk(cudaPeekAtLastError());
    //gpuErrchk(cudaDeviceSynchronize());

    // --- Read mask array from device -> host
    gpuErrchk(hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * graph->numVertices, hipMemcpyDeviceToHost));

    int iteration = 0;
    while (!allFinalizedVertices(h_finalizedVertices, graph->numVertices) && iteration < MAX_ITERATION) {

        // --- In order to improve performance, we run some number of iterations without reading the results.  This might result
        //     in running more iterations than necessary at times, but it will in most cases be faster because we are doing less
        //     stalling of the GPU waiting for results.
        for (int asyncIter = 0; asyncIter < NUM_ASYNCHRONOUS_ITERATIONS; asyncIter++) {

            Kernel1 <<<iDivUp(graph->numVertices, BLOCK_SIZE), BLOCK_SIZE >>>(graph -> vertexArray, graph -> edgeArray, graph -> weightArray, d_finalizedVertices, h_shortestDistances,
                                                            d_updatingShortestDistances, graph->numVertices, graph->numEdges);
            //gpuErrchk(cudaPeekAtLastError());
            //gpuErrchk(cudaDeviceSynchronize());
            Kernel2 <<<iDivUp(graph->numVertices, BLOCK_SIZE), BLOCK_SIZE >>>(graph -> vertexArray, graph -> edgeArray, graph -> weightArray, d_finalizedVertices, h_shortestDistances, d_updatingShortestDistances,
                                                            graph->numVertices);
            //gpuErrchk(cudaPeekAtLastError());
            //gpuErrchk(cudaDeviceSynchronize());
	    iteration++;
        }

        gpuErrchk(hipMemcpy(h_finalizedVertices, d_finalizedVertices, sizeof(bool) * graph->numVertices, hipMemcpyDeviceToHost));
    }

    // --- Copy the result to host
    //gpuErrchk(cudaMemcpy(h_shortestDistances, d_shortestDistances, sizeof(float) * graph->numVertices, cudaMemcpyDeviceToHost));

    hipDeviceSynchronize();

    free(h_finalizedVertices);

    gpuErrchk(hipFree(graph -> vertexArray));
    gpuErrchk(hipFree(graph -> edgeArray));
    gpuErrchk(hipFree(graph -> weightArray));
    gpuErrchk(hipFree(d_finalizedVertices));
    //gpuErrchk(cudaFree(d_shortestDistances));
    gpuErrchk(hipFree(d_updatingShortestDistances));
}

/****************/
/* MAIN PROGRAM */
/****************/
int main() {

    // --- Number of graph vertices
    //int numVertices = 8;

    int numVertices = 1<<20;

    // --- Number of edges per graph vertex
    //int neighborsPerVertex = 6;
    int neighborsPerVertex = 2;

    // --- Source vertex
    int sourceVertex = 0;

    // --- Allocate memory for arrays
    GraphData graph;
    generateRandomGraph(&graph, numVertices, neighborsPerVertex);
/*
    // --- From adjacency list to adjacency matrix.
    //     Initializing the adjacency matrix
    float *weightMatrix = (float *)malloc(numVertices * numVertices * sizeof(float));
    for (int k = 0; k < numVertices * numVertices; k++) weightMatrix[k] = FLT_MAX;

    // --- Displaying the adjacency list and constructing the adjacency matrix
    printf("Adjacency list\n");
    for (int k = 0; k < numVertices; k++) weightMatrix[k * numVertices + k] = 0.f;
    for (int k = 0; k < numVertices; k++)
        for (int l = 0; l < neighborsPerVertex; l++) {
            weightMatrix[k * numVertices + graph.edgeArray[graph.vertexArray[k] + l]] = graph.weightArray[graph.vertexArray[k] + l];
            printf("Vertex nr. %i; Edge nr. %i; Weight = %f\n", k, graph.edgeArray[graph.vertexArray[k] + l],
                                                                   graph.weightArray[graph.vertexArray[k] + l]);
        }

    for (int k = 0; k < numVertices * neighborsPerVertex; k++)
        printf("%i %i %f\n", k, graph.edgeArray[k], graph.weightArray[k]);

    // --- Displaying the adjacency matrix

    printf("\nAdjacency matrix\n");
    for (int k = 0; k < numVertices; k++) {
        for (int l = 0; l < numVertices; l++)
            if (weightMatrix[k * numVertices + l] < FLT_MAX)
                printf("%1.3f\t", weightMatrix[k * numVertices + l]);
            else
                printf("--\t");
            printf("\n");
        }
*/

/*
    // --- Running Dijkstra on the CPU
    float *h_shortestDistancesCPU = (float *)malloc(numVertices * sizeof(float));
    dijkstraCPU(weightMatrix, h_shortestDistancesCPU, sourceVertex, numVertices);

    printf("\nCPU results\n");
    for (int k = 0; k < numVertices; k++) printf("From vertex %i to vertex %i = %f\n", sourceVertex, k, h_shortestDistancesCPU[k]);
*/
    // --- Allocate space for the h_shortestDistancesGPU
    float *h_shortestDistancesGPU;// = (float*)malloc(sizeof(float) * graph.numVertices);
    gpuErrchk(hipMallocManaged(&h_shortestDistancesGPU,       sizeof(float) * graph.numVertices));

    dijkstraGPU(&graph, sourceVertex, h_shortestDistancesGPU);

    FILE *fp;
    fp = fopen ("output.txt", "w+");
    fprintf(fp,"\nGPU results\n");
    for (int k = 0; k < numVertices; k++) 
         fprintf(fp, "From vertex %i to vertex %i = %f\n", sourceVertex, k, h_shortestDistancesGPU[k]);
    fclose(fp);

    //free(h_shortestDistancesCPU);
    hipFree(h_shortestDistancesGPU);

    return 0;
}
