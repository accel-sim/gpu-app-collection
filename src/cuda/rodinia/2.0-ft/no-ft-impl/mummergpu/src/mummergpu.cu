#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <sys/types.h>
#include <unistd.h>
#include <errno.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

// includes, kernels
#include <common.cu>

#include <mummergpu.h>
#include <mummergpu_kernel.cu>

int USE_PRINT_KERNEL = 1;

#define BREATHING_ROOM (16 * 1024 * 1024)
#define BASES_PER_TREE_PAGE 8388608
//#define BASES_PER_TREE_PAGE 7000000
#define BLOCKSIZE 256
unsigned int cuda_calls = 0;
void trap_dbg()
{
	fprintf(stderr, "Trapped\n");
}

#define CUDA_SAFE_CALL( call) do {                                           \
	cuda_calls++;															 \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %d (%s).\n",        \
                __FILE__, __LINE__, err, hipGetErrorString( err) );              \
				trap_dbg();												     \
    exit(EXIT_FAILURE);                                                      \
    } } while (0)

#  define CU_SAFE_CALL_NO_SYNC( call ) do {                                  \
    hipError_t err = call;                                                     \
    if( hipSuccess != err) {                                               \
        fprintf(stderr, "Cuda driver error %x in file '%s' in line %i.\n",   \
                err, __FILE__, __LINE__ );                                   \
        exit(EXIT_FAILURE);                                                  \
    } } while (0)

#  define CUT_DEVICE_INIT_DRV(cuDevice) do {                                 \
    cuDevice = 0;                                                            \
    int deviceCount = 0;                                                     \
    hipError_t err = hipInit(0);                                                \
    if (hipSuccess == err)                                                 \
        CU_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));                \
    if (deviceCount == 0) {                                                  \
        fprintf(stderr, "There is no device.\n");                            \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    int dev;                                                                 \
    for (dev = 0; dev < deviceCount; ++dev) {                                \
        int major, minor;                                                    \
        CU_SAFE_CALL_NO_SYNC(hipDeviceComputeCapability(&major, &minor, dev));\
        if (major >= 1)                                                      \
            break;                                                           \
    }                                                                        \
    if (dev == deviceCount) {                                                \
        fprintf(stderr, "There is no device supporting CUDA.\n");            \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
    else                                                                     \
        CU_SAFE_CALL_NO_SYNC(hipDeviceGet(&cuDevice, dev));                   \
} while (0)

unsigned int num_bind_tex_calls = 0;
#define BIND_TEX(offset, tex, arr, desc, len) do {							 \
	CUDA_SAFE_CALL(hipBindTexture(offset, tex, arr, desc, len));			 \
	++num_bind_tex_calls;													 \
} while(0)

#define BIND_TEX_ARRAY(tex, arr, desc) do {							 		 \
	CUDA_SAFE_CALL(hipBindTextureToArray(tex, arr, desc));					 \
	++num_bind_tex_calls;													 \
} while(0)

#define CUDA_MALLOC(ptr, size) do {							 			 	 \
	hipMalloc(ptr, size);													 \
	++num_bind_tex_calls;													 \
} while(0)

#define CUDA_MALLOC_PITCH(ptr, out_pitch, rowsize, numrows) do { 			 \
	hipMallocPitch(ptr, out_pitch, rowsize, numrows);						 \
	++num_bind_tex_calls;													 \
} while(0)

#define CUDA_MALLOC_ARRAY(ptr, desc, pitch, rows) do {						 \
	hipMallocArray(ptr, desc, pitch, rows);					 			 \
	++num_bind_tex_calls;													 \
} while(0)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold(MatchResults* results, 
				 char* refstr, 
				 char* queries, 
				 int* queryAddrs,
				 int* queryLengths,
				 PixelOfNode* nodeTexture,
				 PixelOfChildren* childrenTexture,
				 int numQueries,
				 int mismatch_length,
				 int rc); 
 
extern "C"
void getReferenceString(const char * filename, char** refstr, size_t* reflen);

extern "C"
void createTreeTexture(const char * filename,
                       PixelOfNode** nodeTexture, 
					   PixelOfChildren** childrenTexture,
                       unsigned int* width, 
                       unsigned int* node_height,
                       unsigned int* children_height,
					   AuxiliaryNodeData** aux_data,
					   int* num_match_coords,
                       int min_match_len,
					   Statistics* statistics,
					   const char * dotfilename,
                       const char * texfilename);

extern "C"
void getQueriesTexture(int qfile,
                       char** queryTexture, 
                       size_t* queryLength, 
                       int** queryAddrs, 
					   char*** queryNames,
                       int** queryLengths,
					   unsigned int* numQueries,
					   unsigned int* num_match_coords,
					   unsigned int device_memory_avail,
					   int min_match_length,
					   bool rc);

extern "C"
int lookupNumLeaves(ReferencePage * page, TextureAddress addr);

void printAlignments(ReferencePage* page, 
					 Alignment* alignments,
                     char* query, 
                     int qrylen,
                     TextureAddress nodeid, 
                     int qrypos, 
                     int edge_depth, 
                     int min_match, 
                     bool rc,
                     bool forwardcoordinates);

int  countLeafNodes(int nodeid);        

extern "C"
void mapQueriesEndToEnd(MatchContext* ctx,
                        ReferencePage* page,
                        MatchInfo* h_matches,
                        unsigned int numMatches,
                        Alignment* h_alignments,
						unsigned int numAligments);

char *  createTimer()
{
  unsigned int * ptr = (unsigned int *) malloc(sizeof(struct Timer_t));
  memset(ptr, 0, sizeof(struct Timer_t));
  return (char *) ptr;
}

void startTimer(char * ptr)
{
  gettimeofday(&(((struct Timer_t *)ptr)->start_m), NULL);
}

void stopTimer(char * ptr)
{
  gettimeofday(&(((struct Timer_t *)ptr)->end_m), NULL);
}

float getTimerValue(char * ptr)
{
  Timer_t * timer = (Timer_t*) ptr;

  if (timer == NULL)
  {
    fprintf(stderr, "Uninitialized timer!!!\n");
    return 0.0;
  }

  if (timer->end_m.tv_sec == 0) { stopTimer(ptr); }

  return  (float) (1000.0 * (timer->end_m.tv_sec - timer->start_m.tv_sec) 
                + (0.001 *  (timer->end_m.tv_usec - timer->start_m.tv_usec)));
}

void deleteTimer(char * ptr)
{
  free((Timer_t *)ptr);
}

extern "C"
int createReference(const char* fromFile, Reference* ref)
{
   if (!fromFile || !ref)
	  return -1;

   char * loadreftimer = createTimer();
   startTimer(loadreftimer);

   getReferenceString(fromFile, &(ref->str), &(ref->len));
  
   stopTimer(loadreftimer);
   ref->t_load_from_disk += getTimerValue(loadreftimer);
   deleteTimer(loadreftimer);

   return 0;
}

extern "C"
int destroyReference(Reference* ref)
{
   free(ref->h_node_tex_array);
   free(ref->h_children_tex_array);
   free(ref->str);
#if REORDER_REF
   free(ref->h_ref_array);
#endif

   free(ref->aux_data);
#if TREE_ACCESS_HISTOGRAM
   free(ref->h_node_hist);
   free(ref->h_child_hist);
#endif
   ref->str = NULL;
   ref->len = 0;

   return 0;
}

extern "C"
int createQuerySet(const char* fromFile, QuerySet* queries)
{

   fprintf(stderr, "Opening %s...\n", fromFile);
   int qfile = open(fromFile, O_RDONLY);
   
   if (qfile == -1)
   {
	  fprintf(stderr, "Can't open %s: %d\n", fromFile, errno);
	  exit (1);
   }

   queries->qfile = qfile;

   return 0;
}

extern "C"
int destroyQuerySet(QuerySet* queries)
{
 
   if (queries->qfile)
	  close(queries->qfile);

   return 0;
}

extern "C"
void printStringForError(int err)
{
   
}

extern "C"
int createMatchContext(Reference* ref,
                       QuerySet* queries,
                       MatchResults* matches,
                       bool on_cpu,
                       int min_match_length,
                       char* stats_file,
                       bool reverse,
                       bool forwardreverse,
                       bool forwardcoordinates,
                       bool showQueryLength,
                       char* dotfilename,
                       char* texfilename,
                       MatchContext* ctx) {
                       
    ctx->queries = queries;
    ctx->ref = ref;
    ctx->full_ref = ref->str;
    ctx->full_ref_len = ref->len;
    
    ctx->on_cpu = on_cpu;
    ctx->min_match_length = min_match_length;
    ctx->stats_file = stats_file;
    ctx->reverse = reverse;
    ctx->forwardreverse = forwardreverse;
    ctx->forwardcoordinates = forwardcoordinates;
    ctx->show_query_length = showQueryLength;
    ctx->dotfilename = dotfilename;
    ctx->texfilename = texfilename;
    return 0;
}


extern "C"
int destroyMatchContext(MatchContext* ctx)
{
   free(ctx->full_ref);
   //destroyReference(ctx->ref);
   destroyQuerySet(ctx->queries);
   return 0;
}

void buildReferenceTexture(Reference* ref, 
                           char* full_ref, 
                           size_t begin, 
                           size_t end, 
                           int min_match_len,
                           char* dotfilename, 
                           char* texfilename,
                           Statistics* statistics)
{
    fprintf(stderr, "Building reference texture...\n");
    
    PixelOfNode* nodeTexture = NULL;
    PixelOfChildren * childrenTexture = NULL;
    
    unsigned int width = 0;
    unsigned int node_height = 0;
    unsigned int children_height = 0;
    
    AuxiliaryNodeData* aux_data = NULL;
    int num_nodes;
    
	char * loadreftimer = createTimer();
   	startTimer(loadreftimer);

    ref->len = end - begin + 3;
    ref->str = (char*)malloc(ref->len);
    ref->str[0] = 's';
    strncpy(ref->str + 1, full_ref + begin, ref->len - 3);
    strcpy(ref->str + ref->len - 2, "$");

   	stopTimer(loadreftimer);
   	statistics->t_ref_from_disk += getTimerValue(loadreftimer) + ref->t_load_from_disk;
   	deleteTimer(loadreftimer);

    createTreeTexture(ref->str,
                      &nodeTexture,
                      &childrenTexture,
                      &width, 
                      &node_height,
                      &children_height,
                      &aux_data,
                      &num_nodes,
                      min_match_len,
					  statistics,
                      dotfilename,
                      texfilename);
	         
    ref->h_node_tex_array = nodeTexture;
    ref->h_children_tex_array = childrenTexture;
    ref->tex_width = width;
    ref->tex_node_height = node_height;
    ref->tex_children_height = children_height;

#if TREE_ACCESS_HISTOGRAM
	ref->h_node_hist = (int*)calloc(width * node_height, sizeof(int));
	ref->h_child_hist = (int*)calloc(width * children_height, sizeof(int));
#endif 
    
    ref->aux_data = aux_data;
    ref->num_nodes = num_nodes;
	
 	ref->bytes_on_board = (width * node_height * sizeof(PixelOfNode)) + 
                          (width * children_height * sizeof(PixelOfChildren));
	fprintf(stderr, "This tree will need %d bytes on the board\n", ref->bytes_on_board);

#if REORDER_REF
    char * reordertimer = createTimer();
    startTimer(reordertimer);

  	unsigned int refpitch = ref->pitch = 65536;
    int numrows = ceil(ref->len / ((float)refpitch));
    int blocksize = 4;
    numrows += blocksize;
    
    int refstrsize = numrows * refpitch;
    ref->h_ref_array = (char *) malloc(refstrsize);
    ref->bytes_on_board += refstrsize;

    fprintf(stderr, "The refstr (reordered) requires %d bytes\n", refstrsize);
    
    int z_max = numrows * refpitch;
    for (int z = 0; z < z_max; z++) {
        ref->h_ref_array[z] = 'Z';
    }
    
    int x, y;
    int maxx = 0, maxy = 0;
    
    size_t reflen = ref->len;
    char* refstr = ref->str;
    
    
    int block_dim = refpitch * blocksize;
    for (int i = 0; i < reflen; i++) {
        int bigx = i % (block_dim); // ref string reorder
        int bigy = i / (block_dim);
        
        y = bigy * blocksize + bigx % blocksize;
        x = bigx / blocksize;
        
        //   printf("%d: (%d,%d)=%c\n", i, x, y, refstr[i]);
        
        assert(x < refpitch);
        assert(y < numrows);
        
        ref->h_ref_array[y*refpitch+x] = refstr[i];
        
        if (x > maxx) {
            maxx = x;
        }
        if (y > maxy) {
            maxy = y;
        }
    }
    
    if ((maxx >= refpitch) || (maxy >= numrows)) {
        fprintf(stderr, "ERROR: maxx: %d refpitch: %d, maxy: %d numrows: %d\n",
                maxx,    refpitch,     maxy,    numrows);
                
        exit(1);
    }
    stopTimer(reordertimer);
	if (statistics)
    	statistics->t_reorder_ref_str += getTimerValue(reordertimer);
    deleteTimer(reordertimer);
#else
    fprintf(stderr, "The refstr requires %d bytes\n", ref->len);
	ref->bytes_on_board += ref->len;
#endif


}

void boardMemory(unsigned int * free_mem, unsigned int * total_mem)
{
  // The emulator doesn't allow calls to hipMemGetInfo

#ifdef __DEVICE_EMULATION__
  *free_mem =  512*1024*1024;
  *total_mem = 768*1024*1024;
#else
  CU_SAFE_CALL_NO_SYNC(hipMemGetInfo(free_mem, total_mem));
#endif
}


void loadReferenceTexture(MatchContext* ctx)
{
    Reference* ref = ctx->ref;
    int numrows = ceil(ref->len / ((float)ref->pitch));
    int blocksize = 4;
    numrows += blocksize;
    
    hipChannelFormatDesc refTextureDesc =
        hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindSigned);
        
    if (!ctx->on_cpu) {
        char * toboardtimer = createTimer();
        startTimer(toboardtimer);

#if REFTEX
#if REORDER_REF

        CUDA_MALLOC_ARRAY((hipArray**)(&ref->d_ref_array),
                                        &refTextureDesc,
                                        ref->pitch,
                                        numrows);

        
        CUDA_SAFE_CALL(hipMemcpyToArray( (hipArray*)(ref->d_ref_array),
                                          0,
                                          0,
                                          ref->h_ref_array,
                                          numrows*ref->pitch,
                                          hipMemcpyHostToDevice));
                                          
        reftex.addressMode[0] = hipAddressModeClamp;
        reftex.addressMode[1] = hipAddressModeClamp;
        reftex.filterMode = hipFilterModePoint;
        reftex.normalized = false;
        
		BIND_TEX_ARRAY(reftex, (hipArray*)ref->d_ref_array, refTextureDesc);

        ctx->ref->bytes_on_board += numrows * ref->pitch;
#else

		CUDA_MALLOC( (void**)(&ref->d_ref_array), ref->len);
		CUDA_SAFE_CALL( hipMemcpy( (void*)(ref->d_ref_array), 
									ref->str, 
									ref->len, 
									hipMemcpyHostToDevice) );
									
		reftex.addressMode[0] = hipAddressModeClamp;
		reftex.filterMode = hipFilterModePoint;
		reftex.normalized = false;    // access with normalized texture coordinates
		hipChannelFormatDesc refDesc =
           hipCreateChannelDesc(8,0,0,0, hipChannelFormatKindUnsigned);
		BIND_TEX(0, reftex, (void*)(ref->d_ref_array), refDesc, ref->len);

        ctx->ref->bytes_on_board += ref->len;
#endif


#else
#if REORDER_REF
		size_t refpitch;
		
		CUDA_MALLOC_PITCH( (void**)(&ref->d_ref_array),
		                                 &refpitch,
		                                 ref->pitch * sizeof(char),
		                                 numrows);
		CUDA_SAFE_CALL( hipMemcpy2D((ref->d_ref_array),
		                             refpitch,
		                             ref->h_ref_array,
		                             ref->pitch ,
		                             ref->pitch  * sizeof(char),
		                             numrows,
		                             hipMemcpyHostToDevice));

        ctx->ref->bytes_on_board += numrows * ref->pitch;
#else
		CUDA_MALLOC( (void**)(&ref->d_ref_array), ref->len);
		CUDA_SAFE_CALL( hipMemcpy( (void*)(ref->d_ref_array), 
									ref->str, 
									ref->len, 
									hipMemcpyHostToDevice) );

        ctx->ref->bytes_on_board += ref->len;
#endif  
#endif    
        stopTimer(toboardtimer);
        ctx->statistics.t_ref_str_to_board += getTimerValue(toboardtimer);
        deleteTimer(toboardtimer);
    }
    else {
        ref->d_ref_array = NULL;
    }
}


void unloadReferenceString(Reference* ref)
{
#if REFTEX
   CUDA_SAFE_CALL(hipUnbindTexture( reftex ) );
#endif

#if REORDER_REF && REFTEX
   CUDA_SAFE_CALL(hipFreeArray((hipArray*)(ref->d_ref_array)));
#else 
	CUDA_SAFE_CALL(hipFree((ref->d_ref_array)));
#endif

   ref->d_ref_array = NULL;
}

void unloadReferenceTree(MatchContext* ctx)
{
   Reference* ref = ctx->ref;

#if REORDER_TREE
   // Unload nodetex
	#if NODETEX
	   CUDA_SAFE_CALL(hipUnbindTexture( nodetex ) );
	   CUDA_SAFE_CALL(hipFreeArray((hipArray*)(ref->d_node_tex_array)));
	#else
		CUDA_SAFE_CALL(hipFree(ref->d_node_tex_array));
	#endif
	   ref->d_node_tex_array = NULL;

	   // Unload childrentex
    if (ref->d_children_tex_array)
    {
	#if CHILDTEX
	   CUDA_SAFE_CALL(hipUnbindTexture( childrentex ) );
	   CUDA_SAFE_CALL(hipFreeArray((hipArray*)(ref->d_children_tex_array)));
	#else
		CUDA_SAFE_CALL(hipFree(ref->d_children_tex_array));
	#endif
    }

	   ref->d_children_tex_array = NULL;
#else

	#if NODETEX
	   CUDA_SAFE_CALL(hipUnbindTexture( nodetex ) );
	#endif
		CUDA_SAFE_CALL(hipFree(ref->d_node_tex_array));
	
	   ref->d_node_tex_array = NULL;

	   // Unload childrentex
       if (ref->d_children_tex_array)
       {
	#if CHILDTEX
	   CUDA_SAFE_CALL(hipUnbindTexture( childrentex ) );
	#endif

	CUDA_SAFE_CALL(hipFree(ref->d_children_tex_array));
	ref->d_children_tex_array = NULL;
       }

#endif

#if TREE_ACCESS_HISTOGRAM
	CUDA_SAFE_CALL(hipFree(ref->d_node_hist));
	ref->d_node_hist = NULL;
	
	CUDA_SAFE_CALL(hipFree(ref->d_child_hist));
	ref->d_child_hist = NULL;
#endif

}

//loads a tree and text for [begin, end) in the reference
void loadReference(MatchContext* ctx) {

    Reference* ref = ctx->ref;
    
    ref->bytes_on_board = 0;
    
    loadReferenceTexture(ctx);

    if (!ctx->on_cpu) {
        char * toboardtimer = createTimer();
        startTimer(toboardtimer);

        // node texels
		ref->bytes_on_board += ref->tex_width * ref->tex_node_height * (sizeof(PixelOfNode));

        // children texels
		ref->bytes_on_board += ref->tex_width * ref->tex_children_height * sizeof(PixelOfChildren);
		
#if REORDER_TREE

#if NODETEX        
        hipChannelFormatDesc nodeTextureDesc =
            hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);

        CUDA_MALLOC_ARRAY( (hipArray**)(&ref->d_node_tex_array),
                                         &nodeTextureDesc,
                                         ref->tex_width,
                                         ref->tex_node_height );
                                         
        
        
        CUDA_SAFE_CALL( hipMemcpyToArray( (hipArray*)(ref->d_node_tex_array),
                                           0,
                                           0,
                                           ref->h_node_tex_array,
                                           ref->tex_width * ref->tex_node_height * sizeof(PixelOfNode),
                                           hipMemcpyHostToDevice));

        nodetex.addressMode[0] = hipAddressModeClamp;
        nodetex.addressMode[1] = hipAddressModeClamp;
        nodetex.filterMode = hipFilterModePoint;
        nodetex.normalized = false;    // access with normalized texture coordinates
        
        BIND_TEX_ARRAY(nodetex, (hipArray*)ref->d_node_tex_array, 
					nodeTextureDesc);
#else
		size_t nodepitch;
		
		CUDA_MALLOC_PITCH( (void**)(&ref->d_node_tex_array),
		                                 &nodepitch,
		                                 ref->tex_width * sizeof(PixelOfNode),
		                                 ref->tex_node_height );
		CUDA_SAFE_CALL( hipMemcpy2D((ref->d_node_tex_array),
		                             nodepitch,
		                             ref->h_node_tex_array,
		                             nodepitch,
		                             ref->tex_width * sizeof(PixelOfNode),
		                             ref->tex_node_height,
		                             hipMemcpyHostToDevice));

#endif

        if (ref->tex_children_height)
        {
#if CHILDTEX
		hipChannelFormatDesc childrenTextureDesc =
            hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);		
        CUDA_MALLOC_ARRAY( (hipArray**)(&ref->d_children_tex_array),
                                         &childrenTextureDesc,
                                         ref->tex_width,
                                         ref->tex_children_height );
                                         
        CUDA_SAFE_CALL( hipMemcpyToArray((hipArray*)(ref->d_children_tex_array),
                                          0,
                                          0,
                                          ref->h_children_tex_array,
                                          ref->tex_width * ref->tex_children_height * sizeof(PixelOfChildren),
                                          hipMemcpyHostToDevice));
                                          
        childrentex.addressMode[0] = hipAddressModeClamp;
        childrentex.addressMode[1] = hipAddressModeClamp;
        childrentex.filterMode = hipFilterModePoint;
        childrentex.normalized = false;    // access with normalized texture coordinates
        
        BIND_TEX_ARRAY(childrentex, (hipArray*)(ref->d_children_tex_array),
					childrenTextureDesc);
#else
		size_t childpitch;

		CUDA_MALLOC_PITCH( (void**)(&ref->d_children_tex_array),
		                                 &childpitch,
		                                 ref->tex_width * sizeof(PixelOfChildren),
		                                 ref->tex_children_height );
		CUDA_SAFE_CALL( hipMemcpy2D((ref->d_children_tex_array),
		                             childpitch,
		                             ref->h_children_tex_array,
		                             childpitch,
		                             ref->tex_width * sizeof(PixelOfNode),
		                             ref->tex_children_height,
		                             hipMemcpyHostToDevice));
#endif
        }

#if TREE_ACCESS_HISTOGRAM
        // node hist
		ref->bytes_on_board += ref->tex_width * ref->tex_node_height * sizeof(int);

		CUDA_MALLOC( (void**)(&ref->d_node_hist), 
									ref->tex_width * ref->tex_node_height *sizeof(int));
		CUDA_SAFE_CALL( hipMemset((ref->d_node_hist),0,
		                            ref->tex_width * ref->tex_node_height * sizeof(int)));
		
        if (ref->tex_children_height)
        {
        // children hist
		ref->bytes_on_board += ref->tex_width * ref->tex_children_height * sizeof(int);
        fprintf(stderr, "after child_hist  ref->bytes_on_board:%ld\n", ref->bytes_on_board);
		CUDA_MALLOC( (void**)(&ref->d_child_hist), 
									ref->tex_width * ref->tex_children_height *sizeof(int));
		CUDA_SAFE_CALL( hipMemset((ref->d_child_hist),0,
		                            ref->tex_width * ref->tex_children_height * sizeof(int)));
        }
#endif

#else // NO TREE REORDERING

		// Node tex, 1-dimensional
        CUDA_MALLOC( (void**)(&ref->d_node_tex_array), 
                     ref->tex_node_height * sizeof(PixelOfNode));
        
        CUDA_SAFE_CALL( hipMemcpy( (ref->d_node_tex_array),
                                           ref->h_node_tex_array,
                                           ref->tex_node_height * sizeof(PixelOfNode),
                                           hipMemcpyHostToDevice));
#if NODETEX  
        hipChannelFormatDesc nodeTextureDesc =
            hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
        nodetex.addressMode[0] = hipAddressModeClamp;
        nodetex.filterMode = hipFilterModePoint;
        nodetex.normalized = false;    // access with normalized texture coordinates
        
        BIND_TEX(0, nodetex, (void*)(ref->d_node_tex_array), nodeTextureDesc, 
					ref->tex_node_height* sizeof(PixelOfNode));
#endif
		if (ref->tex_children_height)
		{
			// Child tex, 1-dimensional
	        CUDA_MALLOC( (void**)(&ref->d_children_tex_array), 
				          ref->tex_children_height * sizeof(PixelOfChildren));

	        CUDA_SAFE_CALL( hipMemcpy( (ref->d_children_tex_array),
	                                           ref->h_children_tex_array,
	                                           ref->tex_children_height * sizeof(PixelOfChildren),
	                                           hipMemcpyHostToDevice));
#if CHILDTEX  
	        hipChannelFormatDesc childTextureDesc =
	            hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindUnsigned);
	        childrentex.addressMode[0] = hipAddressModeClamp;
	        childrentex.filterMode = hipFilterModePoint;
	        childrentex.normalized = false;    // access with normalized texture coordinates

	        BIND_TEX(0, childrentex, (void*)(ref->d_children_tex_array), 
						childTextureDesc, ref->tex_children_height* sizeof(PixelOfChildren));
#endif
        }

#if TREE_ACCESS_HISTOGRAM
		ref->bytes_on_board += ref->tex_node_height * sizeof(int);
		CUDA_MALLOC( (void**)(&ref->d_node_hist), 
									ref->tex_node_height *sizeof(int));
		CUDA_SAFE_CALL( hipMemset((ref->d_node_hist),0,
		                            ref->tex_node_height * sizeof(int)));
		
        if (ref->tex_children_height)
        {
		ref->bytes_on_board += ref->tex_children_height * sizeof(int);
		CUDA_MALLOC( (void**)(&ref->d_child_hist), 
									ref->tex_children_height *sizeof(int));
		CUDA_SAFE_CALL( hipMemset((ref->d_child_hist),0,
		                            ref->tex_children_height * sizeof(int)));
        }
#endif
 
#endif


#if TWO_LEVEL_NODE_TREE
		PixelOfNode node_buf[NODE_THRESH];
		memset(node_buf, 0, sizeof(node_buf));
		for (unsigned int i = 0; (i < NODE_THRESH) && (i < ref->num_nodes); ++i)
		{
			TextureAddress myaddress(id2addr(i));

#if MERGETEX && REORDER_TREE 
            myaddress.x &= 0x7FF;
            myaddress.x *= 2;

			int loc = myaddress.x + myaddress.y*MAX_TEXTURE_DIMENSION;
			node_buf[i]= ((PixelOfNode*)(ref->h_node_tex_array))[loc];

#elif REORDER_TREE
			int loc = myaddress.x + myaddress.y*MAX_TEXTURE_DIMENSION;
			node_buf[i]= ((PixelOfNode*)(ref->h_node_tex_array))[loc];

#elif MERGETEX
			node_buf[i]= ((PixelOfNode*)(ref->h_node_tex_array))[myaddress.x*2];
#else
			node_buf[i]= ((PixelOfNode*)(ref->h_node_tex_array))[myaddress.x];
#endif

		}

		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(node_tree_top), node_buf, sizeof(node_buf)));					
#endif

#if TWO_LEVEL_CHILD_TREE
		PixelOfChildren child_buf[CHILD_THRESH];
		memset(child_buf, 0, sizeof(child_buf));
		for (unsigned int i = 0; (i < CHILD_THRESH) && (i < ref->num_nodes); ++i)
		{
			TextureAddress myaddress(id2addr(i));

#if MERGETEX && REORDER_TREE 
            myaddress.x &= 0x7FF;
            myaddress.x *= 2;

			int loc = myaddress.x + myaddress.y*MAX_TEXTURE_DIMENSION;
			child_buf[i]= ((PixelOfChildren*)(ref->h_node_tex_array))[loc+1];

#elif REORDER_TREE
			int loc = myaddress.x + myaddress.y*MAX_TEXTURE_DIMENSION;
			child_buf[i]= ((PixelOfChildren*)(ref->h_children))[loc];

#elif MERGETEX
			child_buf[i]= ((PixelOfChildren*)(ref->h_node_tex_array))[myaddress.x*2+1];
#else
			child_buf[i]= ((PixelOfChildren*)(ref->h_children_tex_array))[myaddress.x];
#endif
		}
		
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(child_tree_top), child_buf, sizeof(child_buf)));					
#endif
        stopTimer(toboardtimer);
        ctx->statistics.t_tree_to_board += getTimerValue(toboardtimer);
        deleteTimer(toboardtimer);

        fprintf(stderr, "done\n");
    }
    else {
        ref->d_node_tex_array = NULL;
        ref->d_children_tex_array = NULL;
    }
}



void dumpQueryBlockInfo(QuerySet* queries)
{
	fprintf(stderr, "\tProcessing queries %s to %s\n",
		queries->h_names[0], 
		queries->h_names[queries->count-1]); 
}

void loadQueries(MatchContext* ctx)
{    
    QuerySet* queries = ctx->queries;
    queries->bytes_on_board = 0;
    
    unsigned int numQueries = queries->count;
    
    if (!ctx->on_cpu) {
        fprintf(stderr, "Allocating device memory for queries... ");
	
	    char* toboardtimer = createTimer();
	    startTimer(toboardtimer);
	
		dumpQueryBlockInfo(queries);
        CUDA_MALLOC((void**) &queries->d_tex_array, queries->texlen);                                                    \

        
        queries->bytes_on_board += queries->texlen;
        
        CUDA_SAFE_CALL( hipMemcpy((void*) queries->d_tex_array,
                                   queries->h_tex_array + queries->h_addrs_tex_array[0],
                                   queries->texlen,
                                   hipMemcpyHostToDevice));

#if QRYTEX
		qrytex.addressMode[0] = hipAddressModeClamp;
		qrytex.filterMode = hipFilterModePoint;
		qrytex.normalized = false;    // access with normalized texture coordinates
		hipChannelFormatDesc qryDesc =
           hipCreateChannelDesc(8,0,0,0, hipChannelFormatKindUnsigned);
		BIND_TEX(0, qrytex, (void*)(queries->d_tex_array), qryDesc, 
					queries->texlen);
#endif
        CUDA_MALLOC((void**) &queries->d_addrs_tex_array,
                                   numQueries * sizeof(int));
                                   
        queries->bytes_on_board += numQueries * sizeof(int);
        
        CUDA_SAFE_CALL( hipMemcpy((void*) queries->d_addrs_tex_array,
                                   queries->h_addrs_tex_array,
                                   numQueries * sizeof(int),
                                   hipMemcpyHostToDevice));
                           
        CUDA_MALLOC((void**) &queries->d_lengths_array,
                                   numQueries * sizeof(int));
                                   
        queries->bytes_on_board += numQueries * sizeof(int);        

        CUDA_SAFE_CALL( hipMemcpy((void*) queries->d_lengths_array,
                                   queries->h_lengths_array,
                                   numQueries * sizeof(int),
                                   hipMemcpyHostToDevice));
	    stopTimer(toboardtimer);
	    ctx->statistics.t_queries_to_board += getTimerValue(toboardtimer);
	    deleteTimer(toboardtimer);
	
		fprintf(stderr, "\tallocated %ld bytes\n", queries->bytes_on_board);
		
    }
    else {
        queries->d_addrs_tex_array = NULL;
        queries->d_tex_array = NULL;
        queries->d_lengths_array = NULL;
        fprintf(stderr, " allocated %ld bytes\n", 2 * numQueries*sizeof(int) + queries->texlen);
    }
    

}


void unloadQueries(MatchContext* ctx)
{
   QuerySet* queries = ctx->queries;

   CUDA_SAFE_CALL(hipFree(queries->d_tex_array));
   queries->d_tex_array = NULL;

   CUDA_SAFE_CALL(hipFree(queries->d_addrs_tex_array));
   queries->d_addrs_tex_array = NULL;

   CUDA_SAFE_CALL(hipFree(queries->d_lengths_array));
   queries->d_lengths_array = NULL;

   queries->bytes_on_board = 0;
}

// Computes the location of the first MatchCoord for a given query.  NOTE:
// Do NOT use this function if COALESCED_QUERIES == 1
inline int match_coord_addrs(int qryid, int qry_addrs, int match_length)
{
    return qry_addrs - qryid * (match_length + 1);
}

// Construct the offset table for a set of queries.  This table will be used  
// by the printing functions, and if COALESCED_QUERIES == 1, by the matching 
// kernel.
void buildCoordOffsetArray(MatchContext* ctx,
                           int** h_coord_offset_array,
                           unsigned int* num_coords)
{
    int numCoords = 0;
    int match_length = ctx->min_match_length;
	int numQueries = ctx->queries->count;
	int* lengths = ctx->queries->h_lengths_array;

    int* coord_offsets = (int*)calloc(numQueries, sizeof(int));

#if COALESCED_QUERIES
	
    for (unsigned int i = 0; i < numQueries; i += WARP_SIZE)
    {
        // Every query in this warp will need at least this many coords
        int max_num_coords = 0;
        for (unsigned int j = 0; j < WARP_SIZE && (i + j) < numQueries; ++j)
        {
            int num_coords = lengths[i + j] - match_length + 1;
            if ( max_num_coords <  num_coords)
                max_num_coords = num_coords;
        }
        
        unsigned int block_size = max_num_coords * WARP_SIZE;
        
        for (unsigned int j = 0; j < WARP_SIZE && (i + j) < numQueries; ++j)
        {
            ctx->results.h_coord_tex_array[i + j] = numCoords + j;
        }
        numCoords += block_size;
    }
#else
    for (unsigned int i = 0; i < numQueries; ++i)
    {
		int qryoffset = ctx->queries->h_addrs_tex_array[i];
		coord_offsets[i] = match_coord_addrs(i, qryoffset, match_length);
    }
	if (numQueries > 0)
	{
		unsigned int last_qry = numQueries - 1;
		unsigned int last_qry_len = lengths[last_qry] - match_length + 1;
		numCoords = coord_offsets[last_qry] + last_qry_len;
		fprintf(stderr, "Need %d match coords for this result array\n", 
			numCoords);
	}
#endif
    *num_coords = numCoords;
	*h_coord_offset_array = coord_offsets;
}


void loadResultBuffer(MatchContext* ctx)
{
    unsigned int numQueries = ctx->queries->count;
    
    assert (numQueries);

    char* offsettimer = createTimer();
    startTimer(offsettimer);

	buildCoordOffsetArray(ctx,
	                      &(ctx->results.h_coord_tex_array),
	                      &(ctx->results.numCoords));

    stopTimer(offsettimer);
    ctx->statistics.t_build_coord_offsets += getTimerValue(offsettimer);
    deleteTimer(offsettimer);

	unsigned int numCoords = ctx->results.numCoords;
	fprintf(stderr, "Allocating result array for %d queries (%d bytes) ...", 
			numQueries, numCoords*sizeof(MatchCoord) );
	
    unsigned int boardFreeMemory = 0;
    unsigned int total_mem = 0;

	boardMemory(&boardFreeMemory, &total_mem);

   fprintf(stderr,"board free memory: %u total memory: %u\n", 
          boardFreeMemory, total_mem);
	
    ctx->results.h_match_coords = (MatchCoord*) calloc( numCoords, sizeof(MatchCoord));
    if (ctx->results.h_match_coords == NULL)
	{
		trap_dbg();
		exit(EXIT_FAILURE);
	}
   
    if (!ctx->on_cpu) {
        char* toboardtimer = createTimer();
        startTimer(toboardtimer);
        
        ctx->results.bytes_on_board = 0;
        
        CUDA_MALLOC( (void**) &ctx->results.d_match_coords,
                                    numCoords * sizeof(MatchCoord));
        ctx->results.bytes_on_board += numCoords * sizeof(MatchCoord);
        
        CUDA_SAFE_CALL( hipMemset( (void*)ctx->results.d_match_coords, 0,
                                    numCoords * sizeof(MatchCoord)));
                             
#if COALESCED_QUERIES
        CUDA_MALLOC((void**) &ctx->results.d_coord_tex_array,
                                   numQueries * sizeof(int));

        ctx->results.bytes_on_board += numQueries * sizeof(int);

        CUDA_SAFE_CALL( hipMemcpy((void*) ctx->results.d_coord_tex_array,
                                   ctx->results.h_coord_tex_array,
                                   numQueries * sizeof(int),
                                   hipMemcpyHostToDevice));      
#endif 
        stopTimer(toboardtimer);
        ctx->statistics.t_match_coords_to_board += getTimerValue(toboardtimer);
        deleteTimer(toboardtimer);
    }
    else {
        ctx->results.d_match_coords = NULL;
    }
    
    fprintf(stderr, "done\n");
}


void unloadResultBuffer(MatchContext* ctx) {
    CUDA_SAFE_CALL(hipFree(ctx->results.d_match_coords));
    ctx->results.d_match_coords = NULL;
    ctx->results.bytes_on_board = 0;

#if COALESCED_QUERIES
	CUDA_SAFE_CALL(hipFree(ctx->results.d_match_coords));
#endif
}

void transferResultsFromDevice(MatchContext* ctx)
{
   if (!ctx->on_cpu)
   {
      char* fromboardtimer = createTimer();
      startTimer(fromboardtimer);

	  CUDA_SAFE_CALL(hipMemcpy(ctx->results.h_match_coords, 
								ctx->results.d_match_coords, 
								ctx->results.numCoords * sizeof(MatchCoord), 
								hipMemcpyDeviceToHost) );
	  

#if TREE_ACCESS_HISTOGRAM
		CUDA_SAFE_CALL(hipMemcpy(ctx->ref->h_node_hist, 
                                  ctx->ref->d_node_hist, 
                                  ctx->ref->tex_node_height * ctx->ref->tex_width * sizeof(int), 
                                  hipMemcpyDeviceToHost) );

		CUDA_SAFE_CALL(hipMemcpy(ctx->ref->h_child_hist, 
                                  ctx->ref->d_child_hist, 
                                  ctx->ref->tex_children_height * ctx->ref->tex_width * sizeof(int), 
                                  hipMemcpyDeviceToHost) );

		if (ctx->statistics.node_hist_size < ctx->ref->tex_width * ctx->ref->tex_node_height)
		{
			int* temp = (int*)calloc(ctx->ref->tex_width * ctx->ref->tex_node_height, sizeof(int));
			if (ctx->statistics.node_hist_size)
				memcpy(temp, ctx->statistics.node_hist, ctx->statistics.node_hist_size * sizeof(int));
			ctx->statistics.node_hist = temp;
            ctx->statistics.node_hist_size = ctx->ref->tex_width * ctx->ref->tex_node_height;
        }
			
		if (ctx->statistics.child_hist_size < ctx->ref->tex_width * ctx->ref->tex_children_height)
		{
			temp = (int*)calloc(ctx->ref->tex_width * ctx->ref->tex_children_height, sizeof(int));
			if (ctx->statistics.hist_size)
				memcpy(temp, ctx->statistics.child_hist, ctx->statistics.hist_size * sizeof(int));

			ctx->statistics.child_hist = temp;
			ctx->statistics.child_hist_size = ctx->ref->tex_width * ctx->ref->tex_children_height;
		}

		for (unsigned int i = 0; i < ctx->statistics.node_hist_size; ++i)
		{
		  ctx->statistics.node_hist[i] += ctx->ref->h_node_hist[i];
		}	

		for (unsigned int i = 0; i < ctx->statistics.child_hist_size; ++i)
		{
		  ctx->statistics.child_hist[i] += ctx->ref->h_child_hist[i];
        }

#endif

      stopTimer(fromboardtimer);
      ctx->statistics.t_match_coords_from_board += getTimerValue(fromboardtimer);
      deleteTimer(fromboardtimer);
   }
  
}


int flushOutput();
int addToBuffer(char* string);

char numbuffer[32];

MatchCoord* coordForQueryChar(MatchContext* ctx,
                                      unsigned int qryid,
                                      unsigned int qrychar)
{
	MatchResults* results = &(ctx->results);
    MatchCoord* coords = results->h_match_coords;
#if COALESCED_QUERIES
    return coords + results->h_coord_tex_array[qryid] + qrychar * WARP_SIZE;
#else
    return coords + results->h_coord_tex_array[qryid] + qrychar;
#endif
}

void coordsToPrintBuffers(MatchContext* ctx,
                       ReferencePage* page,
                       MatchInfo** matches,
					   Alignment** alignments,
                       unsigned int mem_avail,
                       unsigned int* coord_idx,
                       unsigned int* match_idx,
					   unsigned int* align_idx,
					   unsigned int* nextqry,
					   unsigned int* nextqrychar)
{
    unsigned int numQueries = ctx->queries->count;
    int match_length = ctx->min_match_length;
    unsigned int cidx = *coord_idx;
    unsigned int midx = 0;
	
	unsigned int numCoords = ctx->results.numCoords;
	
	unsigned int numMatches = 0;
	unsigned int numAlignments = 0;

    int DEBUG = 0;
    if (DEBUG && cidx == 0)
    {
      for (int j = 0; j < numCoords; ++j)
      {
        MatchCoord * coord = ctx->results.h_match_coords+j;
        if (coord->node.data > 0 && !(coord->edge_match_length & FRMASK))
        {
          //fprintf(stdout, "node: %d\n", 
          //        coord->node);
          fprintf(stdout, "node: %d leaves:%d\n", 
                  coord->node.data, lookupNumLeaves(page, coord->node));
        }
      }
      exit(0);
    }

	
	// How much can we fit into mem_avail?
	for (int j = cidx; j < numCoords; ++j)
	{
	    MatchCoord* coord = ctx->results.h_match_coords + j;

		int queryAlignments = 0;
		int queryMatches = 0;

	    if (coord->node.data > 0 && !(coord->edge_match_length & FRMASK))
	    {
	        int numLeaves = lookupNumLeaves(page, coord->node);
	        queryAlignments += numLeaves;
	        queryMatches++;
	    }
		int allMatches    = numMatches    + queryMatches;
		int allAlignments = numAlignments + queryAlignments;

		int neededSize = allMatches * sizeof(MatchInfo) + allAlignments * sizeof(Alignment);

		if (neededSize > mem_avail || (allMatches/BLOCKSIZE) >= MAX_GRID_DIMENSION)
		{
		    // adding this match won't fit on the board
		    break;
		}
		
		++cidx;
		numMatches    = allMatches;
		numAlignments = allAlignments;
	}
			
    MatchInfo* M = (MatchInfo*)calloc(numMatches, sizeof(MatchInfo));
    unsigned int alignmentOffset = 0;

	int qry = *nextqry;
	int qrychar = *nextqrychar;
	bool set_full = false;
    while (qry < numQueries)
    {
        // h_lengths_array doesn't count the 'q' at the beginning of each query
        int qlen = ctx->queries->h_lengths_array[qry] + 1 - match_length;

        while (qrychar < qlen)
        {
			if (midx >= numMatches)
			{
				set_full = true;
				break;
			}
			
            MatchCoord* coord = coordForQueryChar(ctx, qry, qrychar);
            
            if (coord->node.data > 0 && !(coord->edge_match_length & FRMASK))
            {
                MatchInfo m;
                m.resultsoffset = alignmentOffset;
                m.qrystartpos = qrychar;
                m.matchnode = coord->node;
                m.edgematch = coord->edge_match_length;
                m.numLeaves = lookupNumLeaves(page, m.matchnode);
                m.queryid = qry;
                
                alignmentOffset += m.numLeaves;
                M[midx++] = m;
            }
            
			++qrychar;
        }
		
		if (set_full)
			break;
			
		++qry;
		qrychar = 0;
    }
	
    *coord_idx = cidx;
    *match_idx = midx;
	*align_idx = alignmentOffset;
    *matches = M;
	*nextqry = qry;
	*nextqrychar = qrychar;
	fprintf(stderr, "Allocing %d bytes of host memory for %d alignments\n",  alignmentOffset * sizeof(Alignment), numAlignments);
    *alignments = (struct Alignment *) calloc(alignmentOffset, sizeof(Alignment));
	//hipHostMalloc((void**)alignments, numAlignments * sizeof(Alignment));
}


void runPrintKernel(MatchContext* ctx,
					ReferencePage* page,
                    MatchInfo* h_matches,
                    unsigned int numMatches,
                    Alignment* alignments,
                    unsigned int numAlignments)
{

    MatchInfo* d_matches;
    size_t matchesSize = numMatches * sizeof(MatchInfo);
    CUDA_MALLOC((void**) &d_matches, matchesSize);
    
    struct Alignment * d_alignments;
    size_t alignmentSize = numAlignments * sizeof(Alignment);
    CUDA_MALLOC((void**) &d_alignments, alignmentSize);
    CUDA_SAFE_CALL(hipMemset((void*)   d_alignments, 0, alignmentSize));
    
 	char*  atimer = createTimer();
    startTimer(atimer);
    // Copy matches to card
    fprintf(stderr, "prepared %d matches %d alignments\n", numMatches, numAlignments);
	fprintf(stderr, "Copying %d bytes to host memory for %d alignments\n",  numAlignments * sizeof(Alignment), numAlignments);

    int DEBUG = 0;
    if (DEBUG)
    {
      for (int i = 0; i < numMatches; i++)
      {
        printf("m[%d]:\t%d\t%d\t%d\t%d\t%d\t%d\n",
              i,
              h_matches[i].resultsoffset,
              h_matches[i].queryid,
              h_matches[i].matchnode.data,
              h_matches[i].numLeaves,
              h_matches[i].edgematch,
              h_matches[i].qrystartpos);
      }

      exit(0);
    }
	
    CUDA_SAFE_CALL(hipMemcpy(d_matches, h_matches, matchesSize, hipMemcpyHostToDevice));
    stopTimer(atimer);
	float mtime =  getTimerValue(atimer);
    // Launch the kernel
    
    int blocksize = (numMatches > BLOCKSIZE) ? BLOCKSIZE : numMatches;
    
    dim3 dimBlock(blocksize, 1, 1);
    dim3 dimGrid(ceil(numMatches / (float)BLOCKSIZE), 1, 1);
    
    fprintf(stderr, "  Calling print kernel... ");
    
    printKernel <<< dimGrid, dimBlock, 0 >>> (d_matches,
            numMatches,
            d_alignments,
            
#if COALESCED_QUERIES
            ctx->results.d_coord_tex_array,
#endif
            
#if !QRYTEX
#if COALESCED_QUERIES
            (int*)
#endif
            ctx->queries->d_tex_array,
#endif
            
#if !NODETEX
            (_PixelOfNode*)ctx->ref->d_node_tex_array,
#endif
#if !CHILDTEX
            (_PixelOfChildren*)ctx->ref->d_children_tex_array,
#endif
            ctx->queries->d_addrs_tex_array,
            ctx->queries->d_lengths_array,
            page->begin,
            page->end,
            page->shadow_left,
            page->shadow_right,
            ctx->min_match_length
            
#if TREE_ACCESS_HISTOGRAM
            , ctx->ref->d_node_hist,
            ctx->ref->d_child_hist
#endif
                                             );
                                             
    hipDeviceSynchronize();
    
    
    
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err)
    {
        fprintf(stderr, "Kernel execution failed: %s.\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    startTimer(atimer);
    // Copy the results back to the host
    CUDA_SAFE_CALL(hipMemcpy((void*)alignments,
                              (void*)d_alignments,
                              alignmentSize,
                              hipMemcpyDeviceToHost));   
    hipDeviceSynchronize();
	stopTimer(atimer);

	float atime = getTimerValue(atimer);
    fprintf(stderr, "memcpy time= %f\n", atime + mtime);
	deleteTimer(atimer);
    // Cleanup
    CUDA_SAFE_CALL(hipFree(d_alignments));
    CUDA_SAFE_CALL(hipFree(d_matches));                   
}

// TODO: need reverse-complement printing support
void runPrintOnCPU(MatchContext* ctx, ReferencePage* page,
 	MatchInfo* h_matches,
    unsigned int numMatches,
    Alignment* alignments,
    unsigned int numAlignments)
{
	unsigned int min_match_length = ctx->min_match_length;
	
	int* addrs = ctx->queries->h_addrs_tex_array;
	int* lengths = ctx->queries->h_lengths_array; 
	char* qrychars = ctx->queries->h_tex_array;

	if (!numMatches)
		return;
	
	int qry = -1;
	unsigned int qrylen;
	
	for (int i = 0; i < numMatches; ++i)
	{
        MatchInfo& match = h_matches[i];
     	if (match.queryid != qry)
		{
			qry = match.queryid;
			qrylen = lengths[qry];
		}
   		if (!(match.edgematch & FRMASK))
        {   
            printAlignments(page,
							alignments + match.resultsoffset,
#if COALESCED_QUERIES
                            qrychars + sizeof(int) * addrs[qry],
#else
                            qrychars + addrs[qry],
#endif
                            qrylen,
                            match.matchnode,
                            match.qrystartpos,
                            match.edgematch,
                            min_match_length,
                            0,
                            ctx->forwardcoordinates);
        }
	}	
}

int addMatchToBuffer(int left_in_ref, int qrypos, int matchlen);

void getExactAlignments(MatchContext * ctx, ReferencePage * page, bool on_cpu)
{
    assert(!ctx->reverse && !ctx->forwardreverse);
    
    unsigned int boardFreeMemory;
    unsigned int total_mem;

    if (!on_cpu)
	{
		boardMemory(&boardFreeMemory, &total_mem);
		fprintf(stderr, "board free memory: %u total memory: %u\n", 
		boardFreeMemory, total_mem);
	}
	else
	{
		boardFreeMemory = 256 * 1024 * 1024;
		total_mem = boardFreeMemory;
	}
    
#ifdef __DEVICE_EMULATION__
    boardFreeMemory = 512 * 1024 * 1024;
#endif
    
	boardFreeMemory -= BREATHING_ROOM;
    fprintf(stderr, "board free memory: %u\n", boardFreeMemory);
    
    int rTotalMatches = 0;
    int rTotalAlignments = 0;
    int totalRounds = 0;
	unsigned int last_coord = ctx->results.numCoords;
	unsigned int next_coord = 0;
	unsigned int nextqry = 0;
	unsigned int nextqrychar = 0;
	int lastqry = -1;
    while (next_coord < last_coord)
    {
        // see how many queries will fit on the board
        totalRounds++;
        
        unsigned int numMatches = 0;
        unsigned int numAlignments = 0;
		MatchInfo* h_matches = NULL;
		Alignment* h_alignments = NULL;
		int coord_left = next_coord;
		char* btimer = createTimer();
	    startTimer(btimer);
		coordsToPrintBuffers(ctx, page, &h_matches, &h_alignments, boardFreeMemory, 
			&next_coord, &numMatches, &numAlignments, &nextqry, &nextqrychar);
		stopTimer(btimer);

		float btime = getTimerValue(btimer);
		ctx->statistics.t_coords_to_buffers += btime;
	    fprintf(stderr, "buffer prep time= %f\n", btime);
		deleteTimer(btimer);
		
        fprintf(stderr, "Round %d: Printing results for match coords [%d-%d) of %d using %d matches and %d alignments\n",
                totalRounds, coord_left, next_coord, last_coord, numMatches, numAlignments);

		if (numMatches == 0)
			continue;
			
        char buf[256];      
       //assert(qryend > qrystart);
        
        rTotalAlignments += numAlignments;
        rTotalMatches += numMatches;

		if (num_bind_tex_calls > 100)
		{
        	hipDeviceReset();
			num_bind_tex_calls = 0;
			loadReference(ctx);
			loadQueries(ctx);
		}

        char* ktimer = createTimer();
	    startTimer(ktimer);
		if (on_cpu)
		{
			runPrintOnCPU(ctx, page, h_matches, numMatches, 
				h_alignments, numAlignments);
		}
		else
		{
			runPrintKernel(ctx, page, h_matches, numMatches, 
				h_alignments, numAlignments);
		}
        stopTimer(ktimer);

	    float ktime = getTimerValue(ktimer);
	    ctx->statistics.t_print_kernel += ktime;
	    fprintf(stderr, "print kernel time= %f\n", ktime);
		deleteTimer(ktimer);
		
		// char* stimer = createTimer();
		// startTimer(stimer);
		// mapQueriesEndToEnd(ctx,
		//                    page,
		//                    h_matches,
		//                    numMatches,
		//                    h_alignments,
		// 				   numAlignments);
		// 
		// stopTimer(stimer);
		// 
		// float stime = getTimerValue(stimer);
		// fprintf(stderr, "postprocess time= %f\n", stime);
		// deleteTimer(stimer);

		//flushOutput();
		
		//Process the alignments
		char* otimer = createTimer();
	    startTimer(otimer);
	
        for (int m = 0; m < numMatches; m++)
        {   
            int base = h_matches[m].resultsoffset;
            for (int i = 0; i < h_matches[m].numLeaves; i++)
            {
                // See if there are any more left maximal alignments for this match
                if (h_alignments[base+i].left_in_ref == 0)
                {
                    break;
                }

				if (h_matches[m].queryid != lastqry)
	            {
	                lastqry = h_matches[m].queryid;
	                addToBuffer("> ");
	                addToBuffer(*(ctx->queries->h_names + lastqry));
	                addToBuffer("\n");
	            }
                
                sprintf(buf, "%d\t%d\t%d\n",
                        h_alignments[base+i].left_in_ref,
                        h_matches[m].qrystartpos + 1,
                        h_alignments[base+i].matchlen);
				addToBuffer(buf);
				
				// addMatchToBuffer(h_alignments[base+i].left_in_ref,
				// 								 h_matches[m].qrystartpos + 1,
				// 								h_alignments[base+i].matchlen);
                
            }
        }
		
        
        flushOutput();
        
	    stopTimer(otimer);
	    ctx->statistics.t_results_to_disk += getTimerValue(otimer);
	    deleteTimer(otimer);
		
        free(h_matches);
        free(h_alignments);
		//hipHostFree((void*)h_alignments);
        
    }
    free(ctx->results.h_coord_tex_array);
	free(ctx->results.h_match_coords);
    ctx->results.h_coord_tex_array = NULL;
	ctx->results.h_match_coords = NULL;

    fprintf(stderr, "Finished processing %d matches and %d potential alignments in %d rounds\n",
            rTotalMatches, rTotalAlignments, totalRounds);
}

int getQueryBlock(MatchContext* ctx, size_t device_mem_avail)
{
    QuerySet* queries = ctx->queries;
    char * queryTex = NULL;
    int* queryAddrs = NULL;
    int* queryLengths = NULL;
    unsigned int numQueries;
	unsigned int num_match_coords;
    size_t queryLen;
    char** names;

    fprintf(stderr, "Loading query block... ");
    
    char* queryreadtimer = createTimer();
    startTimer(queryreadtimer);
    
    getQueriesTexture(queries->qfile,
                      &queryTex,
                      &queryLen,
                      &queryAddrs,
                      &names,
                      &queryLengths,
                      &numQueries,
					  &num_match_coords,
                      device_mem_avail,
                      ctx->min_match_length,
                      ctx->reverse || ctx->forwardreverse);
                      
    stopTimer(queryreadtimer);
    ctx->statistics.t_queries_from_disk += getTimerValue(queryreadtimer);
    deleteTimer(queryreadtimer);
    
    queries->h_tex_array = queryTex;
    queries->count = numQueries;
    queries->h_addrs_tex_array = queryAddrs;
    queries->texlen = queryLen;
    queries->h_names = names;
    queries->h_lengths_array = queryLengths;

	ctx->results.numCoords = num_match_coords;

    fprintf(stderr, "done.\n");

    return numQueries;
}

void destroyQueryBlock(QuerySet* queries)
{
   free(queries->h_tex_array);
   queries->h_tex_array = NULL;

   for (int i = 0; i < queries->count; ++i)
	  free(queries->h_names[i]);

   free(queries->h_names);

   queries->count = 0;
   queries->texlen = 0;

   free(queries->h_addrs_tex_array);
   queries->h_addrs_tex_array = NULL;   

   free(queries->h_lengths_array);
   queries->h_lengths_array = NULL;
}

void resetStats(Statistics* stats)
{
    stats->t_end_to_end = 0.0;
    stats->t_match_kernel = 0.0;
    stats->t_print_kernel = 0.0;
    stats->t_queries_to_board = 0.0;
    stats->t_match_coords_to_board = 0.0;
    stats->t_match_coords_from_board = 0.0;
    stats->t_tree_to_board = 0.0;
    stats->t_ref_str_to_board = 0.0;
    stats->t_queries_from_disk = 0.0;
	stats->t_ref_from_disk = 0.0;
    stats->t_results_to_disk = 0.0;
    stats->t_tree_construction = 0.0;
    stats->t_tree_reorder = 0.0;
    stats->t_tree_flatten = 0.0;
	stats->t_reorder_ref_str = 0.0;
	stats->t_build_coord_offsets = 0.0;
	stats->t_coords_to_buffers = 0.0;
    stats->bp_avg_query_length = 0.0;

#if TREE_ACCESS_HISTOGRAM
	if (stats->node_hist_size)
	{
		free(stats->node_hist);
		stats->node_hist = NULL;
		stats->node_hist_size = 0;
    }

    if (stats->child_hist_size)
    {
        free(stats->child_hist);
        stats->child_hist = NULL;
        stats->child_hist_size = 0;
	}
#endif
}

void writeStatisticsFile(Statistics* stats, 
						 char* stats_filename, 
						 char* node_hist_filename = NULL, 
						 char* child_hist_filename = NULL)
{
	if (stats_filename)
	{
   		FILE* f = fopen(stats_filename, "w");

		if (!f)
		{
			fprintf(stderr, "WARNING: could not open %s for writing\n", stats_filename);
		}
		else
		{
			fprintf(f, "Q");
			fprintf(f, ",R");
			fprintf(f, ",T");
			fprintf(f, ",m");
			fprintf(f, ",r");
			fprintf(f, ",t");
			fprintf(f, ",n");
			fprintf(f, ",Total");
			fprintf(f, ",Match kernel");
			fprintf(f, ",Print Kernel");
			fprintf(f, ",Queries to board");
			fprintf(f, ",Match coords to board");
			fprintf(f, ",Match coords from board");
			fprintf(f, ",Tree to board");
			fprintf(f, ",Ref str to board");
			fprintf(f, ",Queries from disk");
			fprintf(f, ",Ref from disk");
			fprintf(f, ",Output to disk");
			fprintf(f, ",Tree construction");
			fprintf(f, ",Tree reorder");
			fprintf(f, ",Tree flatten");
			fprintf(f, ",Ref reorder");
			fprintf(f, ",Build coord table");
			fprintf(f, ",Coords to buffers");
			fprintf(f, ",Avg qry length");
			fprintf(f, "\n");
			
			fprintf(f, "%d", QRYTEX);
			fprintf(f, ",%d", REFTEX);
			fprintf(f, ",%d", TREETEX);
			fprintf(f, ",%d", MERGETEX);
			fprintf(f, ",%d", REORDER_REF);
			fprintf(f, ",%d", REORDER_TREE);
			fprintf(f, ",%d", RENUMBER_TREE);
			fprintf(f, ",%f", stats->t_end_to_end);
			fprintf(f, ",%f", stats->t_match_kernel);
			fprintf(f, ",%f", stats->t_print_kernel);
			fprintf(f, ",%f", stats->t_queries_to_board);
			fprintf(f, ",%f", stats->t_match_coords_to_board);
			fprintf(f, ",%f", stats->t_match_coords_from_board);
			fprintf(f, ",%f", stats->t_tree_to_board);
			fprintf(f, ",%f", stats->t_ref_str_to_board);
			fprintf(f, ",%f", stats->t_queries_from_disk);
			fprintf(f, ",%f", stats->t_ref_from_disk);
			fprintf(f, ",%f", stats->t_results_to_disk);
			fprintf(f, ",%f", stats->t_tree_construction);
			fprintf(f, ",%f", stats->t_tree_reorder);
			fprintf(f, ",%f", stats->t_tree_flatten);
			fprintf(f, ",%f", stats->t_reorder_ref_str);
			fprintf(f, ",%f", stats->t_build_coord_offsets);
			fprintf(f, ",%f", stats->t_coords_to_buffers);
			fprintf(f, ",%f", stats->bp_avg_query_length);
			fprintf(f,"\n");
			
			fclose(f);
		}
	}
#if TREE_ACCESS_HISTOGRAM	
	if (node_hist_filename)
	{
		FILE* f = fopen(node_hist_filename, "w");
		if (!f)
		{
			fprintf(stderr, "WARNING: could not open %s for writing\n", node_hist_filename);
		}
		else
		{
			for (unsigned int i = 0; i < ctx->statistics.node_hist_size; ++i)
				fprintf(f, "%d\t%d\n", i, ctx->statistics.node_hist[i]);
		}

	}
	
	if (child_hist_filename)
	{
		FILE* f = fopen(child_hist_filename, "w");
		if (!f)
		{
			fprintf(stderr, "WARNING: could not open %s for writing\n", child_hist_filename);
		}
		else
		{
			for (unsigned int i = 0; i < ctx->statistics.child_hist_size; ++i)
				fprintf(f, "%d\t%d\n", i, ctx->statistics.child_hist[i]);
		}

	}
	
	float total_node_hits = 0;
	float tree_top_node_hits = 0;
	
	float total_child_hits = 0;
	float tree_top_child_hits = 0;
	
	for (unsigned int i = 0; i < ctx->statistics.node_hist_size; ++i)
	{
		total_node_hits +=ctx->statistics.node_hist[i];
		if (i < 256) { tree_top_node_hits += ctx->statistics.node_hist[i]; }
	}

	for (unsigned int i = 0; i < ctx->statistics.child_hist_size; ++i)
	{
		total_child_hits +=ctx->statistics.child_hist[i];
		if (i < 256) { tree_top_child_hits += ctx->statistics.child_hist[i]; }
	}

	fprintf(stderr, "Tree top node  hits (%d/%d) = %f percent\n",(int)tree_top_node_hits, (int)total_node_hits, tree_top_node_hits /total_node_hits);	
	fprintf(stderr, "Tree top child hits (%d/%d) = %f percent\n",(int)tree_top_child_hits, (int)total_child_hits, tree_top_child_hits /total_child_hits);
#endif
}

void matchOnCPU(MatchContext* ctx, bool doRC)
{
	//TODO: CPU is matching is disabled.
	if (doRC) {
		// Match the reverse complement of the queries to the ref
        computeGold(&ctx->results,
                    ctx->ref->str,
                    ctx->queries->h_tex_array,
                    ctx->queries->h_addrs_tex_array,
                    ctx->queries->h_lengths_array,
                    (PixelOfNode*)(ctx->ref->h_node_tex_array),
                    (PixelOfChildren*)(ctx->ref->h_children_tex_array),
                    ctx->queries->count,
                    ctx->min_match_length,
                    REVERSE);
    }
    else {
        computeGold(&ctx->results,
                    ctx->ref->str,
                    ctx->queries->h_tex_array,
                    ctx->queries->h_addrs_tex_array,
                    ctx->queries->h_lengths_array,
                    (PixelOfNode*)(ctx->ref->h_node_tex_array),
                    (PixelOfChildren*)(ctx->ref->h_children_tex_array),
                    ctx->queries->count,
                    ctx->min_match_length,
                    FORWARD);
    }
}

void matchOnGPU(MatchContext* ctx, bool doRC)
{
	int numQueries = ctx->queries->count;
	int blocksize = (numQueries > BLOCKSIZE) ? BLOCKSIZE : numQueries;

	dim3 dimBlock(blocksize, 1, 1);

	dim3 dimGrid(ceil(numQueries / (float)BLOCKSIZE), 1, 1);
	
	// Match the reverse complement of the queries to the ref
    if (doRC) {
		//TODO: GPU RC is disabled
        mummergpuRCKernel <<< dimGrid, dimBlock, 0 >>> (ctx->results.d_match_coords,
                ctx->queries->d_tex_array,
                ctx->queries->d_addrs_tex_array,
                ctx->queries->d_lengths_array,
                numQueries,
                ctx->min_match_length);
    }
    else {
        mummergpuKernel <<< dimGrid, dimBlock, 0 >>> (ctx->results.d_match_coords,
#if COALESCED_QUERIES
                ctx->results.d_coord_tex_array,
#endif

#if !QRYTEX
#if COALESCED_QUERIES
                (int*)
#endif
                ctx->queries->d_tex_array,
#endif

#if !NODETEX
                (_PixelOfNode*)(ctx->ref->d_node_tex_array),
#endif

#if !CHILDTEX
                (_PixelOfChildren*)(ctx->ref->d_children_tex_array),
#endif

#if !REFTEX
                (char*)ctx->ref->d_ref_array,
#endif
                ctx->queries->d_addrs_tex_array,
                ctx->queries->d_lengths_array,
                numQueries,
                ctx->min_match_length
#if TREE_ACCESS_HISTOGRAM
                , ctx->ref->d_node_hist,
                ctx->ref->d_child_hist
#endif
				);
    }

	// check if kernel execution generated an error
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err) {
	    fprintf(stderr, "Kernel execution failed: %s.\n",
	            hipGetErrorString(err));
	    exit(EXIT_FAILURE);
	}
}

void getMatchResults(MatchContext* ctx, 
					 unsigned int page_num)
{
	transferResultsFromDevice(ctx);
}

void matchQueryBlockToReferencePage(MatchContext* ctx, 
								   ReferencePage* page,
								   bool reverse_complement)
{
	char*  ktimer = createTimer();
	
    fprintf(stderr, "Memory footprint is:\n\tqueries: %d\n\tref: %d\n\tresults: %d\n",
            ctx->queries->bytes_on_board,
            ctx->ref->bytes_on_board,
            ctx->results.bytes_on_board);

	startTimer(ktimer);	
	if (ctx->on_cpu) 
	{
		matchOnCPU(ctx, reverse_complement);
	}
	else 
	{

		matchOnGPU(ctx, reverse_complement);
		hipDeviceSynchronize();

	}
	stopTimer(ktimer);

	float ktime = getTimerValue(ktimer);
	ctx->statistics.t_match_kernel += ktime;
	fprintf(stderr, "match kernel time= %f\n", ktime);
	deleteTimer(ktimer);

	getMatchResults(ctx, page->id);
	unloadResultBuffer(ctx);    
}


int matchSubset(MatchContext* ctx,
                ReferencePage* page) 
{
                
    loadQueries(ctx);

    fprintf(stderr, 
			"Matching queries %s - %s against ref coords %d - %d\n",
			ctx->queries->h_names[0], 
			ctx->queries->h_names[ctx->queries->count - 1],
			page->begin,
			page->end);

    loadResultBuffer(ctx);
    
	// TODO: renable RC support by calling this twice /w reverse/fwdreverse
	// idiom.
	matchQueryBlockToReferencePage(ctx, page, false);
	
	if (USE_PRINT_KERNEL && !ctx->on_cpu) 
	{
	    getExactAlignments(ctx, page, false);
	}
    
	else
	{
	    getExactAlignments(ctx, page, true);
	}
	
	flushOutput();
    unloadQueries(ctx);
    return 0;
}

int getFreeDeviceMemory(bool on_cpu)
{
	unsigned int free_mem = 0;
	unsigned int total_mem = 0;
	
	// We have to 'prime' CUDA by making an allocation here.  hipMemGetInfo 
	// will return zeroes until we do a malloc.
	int * p = NULL;
	CUDA_SAFE_CALL(hipMalloc((void**)&p, sizeof(int)));
	CUDA_SAFE_CALL(hipFree(p));
	if (!on_cpu) {

        boardMemory(&free_mem, &total_mem);
		fprintf(stderr, "board free memory: %u total memory: %u\n", 
		free_mem, total_mem);
    }
    else {
        total_mem = free_mem = 804585472; // pretend we are on a 8800 GTX
    }

	return free_mem;
}

int matchQueriesToReferencePage(MatchContext* ctx, ReferencePage* page)
{
	fprintf(stderr, "Beginning reference page %p\n", page);
   	
	int free_mem = getFreeDeviceMemory(ctx->on_cpu);
	
	int available_mem = free_mem - page->ref.bytes_on_board - BREATHING_ROOM;
	ctx->ref = &(page->ref);
    loadReference(ctx);

    while (getQueryBlock(ctx, available_mem)) {
        matchSubset(ctx, page);
        ctx->statistics.bp_avg_query_length = 
			ctx->queries->texlen / (float)(ctx->queries->count) - 2;
        destroyQueryBlock(ctx->queries);
		if (num_bind_tex_calls > 100)
		{
        	hipDeviceReset();
			num_bind_tex_calls = 0;
			loadReference(ctx);
		}
    }

    unloadReferenceString(ctx->ref);
    unloadReferenceTree(ctx);
	lseek(ctx->queries->qfile, 0, SEEK_SET);
    return 0;
}



void initReferencePages( MatchContext* ctx , int* num_pages, ReferencePage** pages_out) {
    unsigned int bases_in_ref = ctx->full_ref_len - 3;
    unsigned int page_size = BASES_PER_TREE_PAGE < bases_in_ref ?
                             BASES_PER_TREE_PAGE : bases_in_ref;
    unsigned int num_reference_pages = ceil((bases_in_ref + 0.0) / page_size);
    fprintf(stderr, "Stream will use %d pages for %d bases, page size = %d\n",
            num_reference_pages, bases_in_ref, page_size);
            
    unsigned int page_overlap = MAX_QUERY_LEN + 1;
    ReferencePage* pages = (ReferencePage*) calloc(num_reference_pages,
                           sizeof(ReferencePage));
                           
    pages[0].begin = 1;
    pages[0].end = pages[0].begin +
                   page_size  +
                   ceil(page_overlap / 2.0) + 1; //the 1 is for the 's' at the beginning
    pages[0].shadow_left = -1;
    pages[0].id = 0;
    
    for (int i = 1; i < num_reference_pages - 1; ++i) {
        pages[i].begin = pages[i - 1].end - page_overlap;
        pages[i].end = pages[i].begin + page_size +  page_overlap;
        pages[i - 1].shadow_right = pages[i].begin;
        pages[i].shadow_left = pages[i-1].end;
        pages[i].id = i;
    }
    
    if (num_reference_pages > 1) {
        int last_page = num_reference_pages - 1;
        pages[last_page].begin = pages[last_page - 1].end - page_overlap;
        pages[last_page].end = ctx->full_ref_len - 1;
        pages[last_page - 1].shadow_right = pages[last_page].begin;
        pages[last_page].shadow_right = -1;
        pages[last_page].shadow_left = pages[last_page - 1].end;
        pages[last_page].id = last_page;
    }
    
   	*pages_out = pages;
	*num_pages = num_reference_pages;
}

int streamReferenceAgainstQueries(MatchContext* ctx) {
    int num_reference_pages = 0;
    ReferencePage* pages = NULL;
    initReferencePages(ctx, &num_reference_pages, &pages);
    
    
    buildReferenceTexture(&(pages[0].ref),
                          ctx->full_ref,
                          pages[0].begin,
                          pages[0].end,
                          ctx->min_match_length,
                          ctx->dotfilename,
                          ctx->texfilename,
                          &(ctx->statistics));
                          
                          
    matchQueriesToReferencePage(ctx, &pages[0]);
    destroyReference(&(pages[0].ref));
    
    for (int i = 1; i < num_reference_pages - 1; ++i) {
    
        buildReferenceTexture(&(pages[i].ref),
                              ctx->full_ref,
                              pages[i].begin,
                              pages[i].end,
                              ctx->min_match_length,
                              NULL,
                              NULL,
                              &(ctx->statistics));
                              
        matchQueriesToReferencePage(ctx, &pages[i]);
        destroyReference(&(pages[i].ref));
    }
    
    if (num_reference_pages > 1) {
        int last_page = num_reference_pages - 1;
        buildReferenceTexture(&(pages[last_page].ref),
                              ctx->full_ref,
                              pages[last_page].begin,
                              pages[last_page].end,
                              ctx->min_match_length,
                              NULL,
                              NULL,
                              &(ctx->statistics));
                              
        matchQueriesToReferencePage(ctx, &pages[last_page]);
        destroyReference(&(pages[last_page].ref));
    }
    free(pages);
    return 0;
}


extern "C"
int matchQueries(MatchContext* ctx) {
    assert(sizeof(struct PixelOfNode) == sizeof(uint4));
    assert(sizeof(struct PixelOfChildren) == sizeof(uint4));

#if TREE_ACCESS_HISTOGRAM    
	ctx->statistics.node_hist_size = 0;
    ctx->statistics.child_hist_size = 0;
#endif

	resetStats(&(ctx->statistics));
    
    char* ttimer = createTimer();
    startTimer(ttimer);
    
    int ret;

    fprintf(stderr, "Streaming reference pages against all queries\n"); 
    ret = streamReferenceAgainstQueries(ctx);

    stopTimer(ttimer);
    ctx->statistics.t_end_to_end += getTimerValue(ttimer);
    deleteTimer(ttimer);
    
    writeStatisticsFile(&(ctx->statistics), ctx->stats_file, "node_hist.out", "child_hist.out");
    
    return ret;
}



