
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////////
// Set Device
////////////////////////////////////////////////////////////////////////////////

void setdevice(void){

	// variables
	int num_devices;
	int device;

	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
		
		// variables
		int max_multiprocessors; 
		int max_device;
		hipDeviceProp_t properties;

		// initialize variables
		max_multiprocessors = 0;
		max_device = 0;
		
		for (device = 0; device < num_devices; device++) {
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
	}

}
