#include "hip/hip_runtime.h"
/*
 * PAVLE - Parallel Variable-Length Encoder for CUDA
 *
 * Copyright (C) 2009 Tjark Bringewat <golvellius@gmx.net>, Ana Balevic <ana.balevic@gmail.com>
 * All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of the
 * MIT License. Read the full licence: http://www.opensource.org/licenses/mit-license.php
 *
 * If you find this program useful, please contact me and reference PAVLE home page in your work.
 * 
 */


#ifndef _PACK_KERNELS_H_
#define _PACK_KERNELS_H_
#include "parameters.h"

__global__ static void pack2(unsigned int *srcData, unsigned int *cindex, unsigned int *cindex2, unsigned int *dstData, unsigned int original_num_block_elements) {
	unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;

	// source index
	unsigned int offset = tid * original_num_block_elements;//DPB,
	unsigned int bitsize = cindex[tid];

	// destination index
	unsigned int pos = cindex2[tid],
				 dword = pos / 32,
				 bit = pos % 32;

	unsigned int i, dw, tmp;
	dw = srcData[offset];			// load the first dword from srcData[]
	tmp = dw >> bit;				// cut off those bits that do not fit into the initial location in destData[]
	atomicOr(&dstData[dword], tmp);	// fill up this initial location
	tmp = dw << 32-bit;				// save the remaining bits that were cut off earlier in tmp
	for (i=1; i<bitsize/32; i++) {	// from now on, we have exclusive access to destData[]
		dw = srcData[offset+i];		// load next dword from srcData[]
		tmp |= dw >> bit;			// fill up tmp
		dstData[dword+i] = tmp;		// write complete dword to destData[]
		tmp = dw << 32-bit;			// save the remaining bits in tmp (like before)
	}
	// exclusive access to dstData[] ends here
	// the remaining block can, or rather should be further optimized
	// write the remaining bits in tmp, UNLESS bit is 0 and bitsize is divisible by 32, in this case do nothing
	if (bit != 0 || bitsize % 32 != 0)
		atomicOr(&dstData[dword+i], tmp);
	if (bitsize % 32 != 0) {
		dw = srcData[offset+i];
		atomicOr(&dstData[dword+i], dw >> bit);
		atomicOr(&dstData[dword+i+1], dw << 32-bit);
	}
}

#endif
