/*
 * PAVLE - Parallel Variable-Length Encoder for CUDA. Main file.
 *
 * Copyright (C) 2009 Ana Balevic <ana.balevic@gmail.com>
 * All rights reserved.
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of the
 * MIT License. Read the full licence: http://www.opensource.org/licenses/mit-license.php
 *
 * If you find this program useful, please contact me and reference PAVLE home page in your work.
 * 
 */

#include "stdafx.h"
#include <hip/hip_runtime.h>
#include "cuda_helpers.h"
#include "print_helpers.h"
#include "comparison_helpers.h"
#include "stats_logger.h"
#include "load_data.h"
#include <sys/time.h>
//#include "vlc_kernel_gm32.cu"
//#include "vlc_kernel_sm32.cu"
#include "vlc_kernel_sm64huff.cu"
//#include "vlc_kernel_dpt.cu"
//#include "vlc_kernel_dptt.cu"
//#include "scan_kernel.cu"
#include "scan.cu"
#include "pack_kernels.cu"
#include "cpuencode.h"

long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}
void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks=1);

extern "C" void cpu_vlc_encode(unsigned int* indata, unsigned int num_elements, unsigned int* outdata, unsigned int *outsize, unsigned int *codewords, unsigned int* codewordlens);

int main(int argc, char* argv[]){
    if(!InitCUDA()) { return 0;	}
    unsigned int num_block_threads = 256;
    if (argc > 1)
        for (int i=1; i<argc; i++)
            runVLCTest(argv[i], num_block_threads);
    else {	runVLCTest(NULL, num_block_threads, 1024);	}
    CUDA_SAFE_CALL(hipDeviceReset());
    return 0;
}

void runVLCTest(char *file_name, uint num_block_threads, uint num_blocks) {
    printf("CUDA! Starting VLC Tests!\n");
    unsigned int num_elements; //uint num_elements = num_blocks * num_block_threads; 
    unsigned int mem_size; //uint mem_size = num_elements * sizeof(int); 
    unsigned int symbol_type_size = sizeof(int);
    //////// LOAD DATA ///////////////
    double H; // entropy
    initParams(file_name, num_block_threads, num_blocks, num_elements, mem_size, symbol_type_size);
    printf("Parameters: num_elements: %d, num_blocks: %d, num_block_threads: %d\n----------------------------\n", num_elements, num_blocks, num_block_threads);
    ////////LOAD DATA ///////////////
    uint	*sourceData =	(uint*) malloc(mem_size);
    uint	*destData   =	(uint*) malloc(mem_size);
    uint	*crefData   =	(uint*) malloc(mem_size);

    uint	*codewords	   = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);
    uint	*codewordlens  = (uint*) malloc(NUM_SYMBOLS*symbol_type_size);

    uint	*cw32 =		(uint*) malloc(mem_size);
    uint	*cw32len =	(uint*) malloc(mem_size);
    uint	*cw32idx =	(uint*) malloc(mem_size);

    uint	*cindex2=	(uint*) malloc(num_blocks*sizeof(int));

    memset(sourceData,   0, mem_size);
    memset(destData,     0, mem_size);
    memset(crefData,     0, mem_size);
    memset(cw32,         0, mem_size);
    memset(cw32len,      0, mem_size);
    memset(cw32idx,      0, mem_size);
    memset(codewords,    0, NUM_SYMBOLS*symbol_type_size);
    memset(codewordlens, 0, NUM_SYMBOLS*symbol_type_size);
    memset(cindex2, 0, num_blocks*sizeof(int));
    //////// LOAD DATA ///////////////
    loadData(file_name, sourceData, codewords, codewordlens, num_elements, mem_size, H);

    //////// LOAD DATA ///////////////

    unsigned int	*d_sourceData, *d_destData, *d_destDataPacked;
    unsigned int	*d_codewords, *d_codewordlens;
    unsigned int	*d_cw32, *d_cw32len, *d_cw32idx, *d_cindex, *d_cindex2;

    CUDA_SAFE_CALL(hipMalloc((void**) &d_sourceData,		  mem_size));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_destData,			  mem_size));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_destDataPacked,	  mem_size));

    CUDA_SAFE_CALL(hipMalloc((void**) &d_codewords,		  NUM_SYMBOLS*symbol_type_size));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_codewordlens,		  NUM_SYMBOLS*symbol_type_size));

    CUDA_SAFE_CALL(hipMalloc((void**) &d_cw32,				  mem_size));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_cw32len,			  mem_size));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_cw32idx,			  mem_size));

    CUDA_SAFE_CALL(hipMalloc((void**)&d_cindex,         num_blocks*sizeof(unsigned int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_cindex2,        num_blocks*sizeof(unsigned int)));

    CUDA_SAFE_CALL(hipMemcpy(d_sourceData,		sourceData,		mem_size,		hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_codewords,		codewords,		NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_codewordlens,	codewordlens,	NUM_SYMBOLS*symbol_type_size,	hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_destData,		destData,		mem_size,		hipMemcpyHostToDevice));

    dim3 grid_size(num_blocks,1,1);
    dim3 block_size(num_block_threads, 1, 1);
    unsigned int sm_size; 


    unsigned int NT = 10; //number of runs for each execution time

    //////////////////* CPU ENCODER *///////////////////////////////////
    unsigned int refbytesize;
    long long timer = get_time();
    cpu_vlc_encode((unsigned int*)sourceData, num_elements, (unsigned int*)crefData,  &refbytesize, codewords, codewordlens);
    float msec = (float)((get_time() - timer)/1000.0);
    printf("CPU Encoding time (CPU): %f (ms)\n", msec);
    printf("CPU Encoded to %d [B]\n", refbytesize);
    unsigned int num_ints = refbytesize/4 + ((refbytesize%4 ==0)?0:1);
    //////////////////* END CPU *///////////////////////////////////

    //////////////////* SM64HUFF KERNEL *///////////////////////////////////
    grid_size.x		= num_blocks;
    block_size.x	= num_block_threads;
    sm_size			= block_size.x*sizeof(unsigned int);
#ifdef CACHECWLUT
    sm_size			= 2*NUM_SYMBOLS*sizeof(int) + block_size.x*sizeof(unsigned int);
#endif
    hipEvent_t     start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord( start, 0 );
        for (int i=0; i<NT; i++) {
            vlc_encode_kernel_sm64huff<<<grid_size, block_size, sm_size>>>(d_sourceData, d_codewords, d_codewordlens,  
#ifdef TESTING
                    d_cw32, d_cw32len, d_cw32idx, 
#endif
                    d_destData, d_cindex); //testedOK2
        }
    hipDeviceSynchronize();
    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;
    float   elapsedTime;
    hipEventElapsedTime( &elapsedTime,
            start, stop ) ;

    CUT_CHECK_ERROR("Kernel execution failed\n");
    printf("GPU Encoding time (SM64HUFF): %f (ms)\n", elapsedTime/NT);
    //////////////////* END KERNEL *///////////////////////////////////

#ifdef TESTING
    unsigned int num_scan_elements = grid_size.x;
    preallocBlockSums(num_scan_elements);
    hipMemset(d_destDataPacked, 0, mem_size);
    printf("Num_blocks to be passed to scan is %d.\n", num_scan_elements);
    prescanArray(d_cindex2, d_cindex, num_scan_elements);

    pack2<<< num_scan_elements/16, 16>>>((unsigned int*)d_destData, d_cindex, d_cindex2, (unsigned int*)d_destDataPacked, num_elements/num_scan_elements);
    CUT_CHECK_ERROR("Pack2 Kernel execution failed\n");
    deallocBlockSums();

    CUDA_SAFE_CALL(hipMemcpy(destData, d_destDataPacked, mem_size, hipMemcpyDeviceToHost));
    compare_vectors((unsigned int*)crefData, (unsigned int*)destData, num_ints);
#endif 

    free(sourceData); free(destData);  	free(codewords);  	free(codewordlens); free(cw32);  free(cw32len); free(crefData); 
    CUDA_SAFE_CALL(hipFree(d_sourceData)); 	CUDA_SAFE_CALL(hipFree(d_destData)); CUDA_SAFE_CALL(hipFree(d_destDataPacked));
    CUDA_SAFE_CALL(hipFree(d_codewords)); 		CUDA_SAFE_CALL(hipFree(d_codewordlens));
    CUDA_SAFE_CALL(hipFree(d_cw32)); 		CUDA_SAFE_CALL(hipFree(d_cw32len)); 	CUDA_SAFE_CALL(hipFree(d_cw32idx)); 
    CUDA_SAFE_CALL(hipFree(d_cindex)); CUDA_SAFE_CALL(hipFree(d_cindex2));
    free(cindex2);
}

