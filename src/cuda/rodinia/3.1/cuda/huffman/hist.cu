#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include <iostream>
#include <stdio.h>

#define CHECK(ans) {gpuAssert((ans),__FILE__,__LINE__);}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if(code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n",hipGetErrorString(code),file, line);
        if(abort) exit(code);
    }
}

using namespace std;

#define SIZE    (100*1024*1024)


__global__ void histo_kernel( unsigned char *buffer,
        long size,
        unsigned int *histo ) {

    __shared__  unsigned int temp[256];

    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd( &temp[buffer[i]], 1 );
        i += offset;
    }

    __syncthreads();
    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

int runHisto(char* file, unsigned int* freq, unsigned int memSize, unsigned int *source) {

    FILE *f = fopen(file,"rb");
    if (!f) {perror(file); exit(1);}
    fseek(f,0,SEEK_SET);
    size_t result = fread(source,1,memSize,f);
    if(result != memSize) fputs("Cannot read input file", stderr);

    fclose(f);

    unsigned char *buffer = (unsigned char*)source;

    hipDeviceProp_t  prop;
    ( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = prop.multiProcessorCount;
    if(!prop.deviceOverlap)
    {
        cout << "No overlaps, so no speedup from streams" << endl;
        return 0;
    }

    // allocate memory on the GPU for the file's data
    int partSize = memSize/32;
    int totalNum = memSize/sizeof(unsigned int);
    int partialNum = partSize/sizeof(unsigned int);

    unsigned char *dev_buffer0; 
    unsigned char *dev_buffer1;
    unsigned int *dev_histo;
    hipMalloc( (void**)&dev_buffer0, partSize ) ;
    hipMalloc( (void**)&dev_buffer1, partSize ) ;
    hipMalloc( (void**)&dev_histo,
            256 * sizeof( int ) ) ;
    hipMemset( dev_histo, 0,
            256 * sizeof( int ) ) ;
    hipStream_t stream0, stream1;
    CHECK(hipStreamCreate(&stream0));
    CHECK(hipStreamCreate(&stream1));
    hipEvent_t     start, stop;
    ( hipEventCreate( &start ) );
    ( hipEventCreate( &stop ) );
    ( hipEventRecord( start, 0 ) );


    for(int i = 0; i < totalNum; i+=partialNum*2)
    {

        CHECK(hipMemcpyAsync(dev_buffer0, buffer+i, partSize, hipMemcpyHostToDevice,stream0));
        CHECK(hipMemcpyAsync(dev_buffer1, buffer+i+partialNum, partSize, hipMemcpyHostToDevice,stream1));


        // kernel launch - 2x the number of mps gave best timing
        histo_kernel<<<blocks*2,256,0,stream0>>>( dev_buffer0, partSize, dev_histo );
        histo_kernel<<<blocks*2,256,0,stream1>>>( dev_buffer1, partSize, dev_histo );
    }
    CHECK(hipStreamSynchronize(stream0));
    CHECK(hipStreamSynchronize(stream1));
    hipMemcpy( freq, dev_histo, 256 * sizeof( int ), hipMemcpyDeviceToHost );
    ( hipEventRecord( stop, 0 ) );
    ( hipEventSynchronize( stop ) );
    float   elapsedTime;
    ( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );



    hipFree( dev_histo );
    hipFree( dev_buffer0 );
    hipFree( dev_buffer1 );
    return 0;
}
