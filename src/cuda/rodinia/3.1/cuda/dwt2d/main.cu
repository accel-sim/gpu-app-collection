/* 
 * Copyright (c) 2009, Jiri Matela
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <unistd.h>
#include <error.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <errno.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#include <getopt.h>

#include "common.h"
#include "components.h"
#include "dwt.h"

struct dwt {
    char * srcFilename;
    char * outFilename;
    unsigned char *srcImg;
    int pixWidth;
    int pixHeight;
    int components;
    int dwtLvls;
};

int getImg(char * srcFilename, unsigned char *srcImg, int inputSize)
{
     printf("Loading input: %s\n", srcFilename);

    //srcFilename = strcat("../../data/dwt2d/",srcFilename);
    //read image
    int i = open(srcFilename, O_RDONLY, 0644);
    if (i == -1) { 
        error(0,errno,"cannot access %s", srcFilename);
        return -1;
    }
    int ret = read(i, srcImg, inputSize);
    printf("precteno %d, inputsize %d\n", ret, inputSize);
    close(i);

    return 0;
}


void usage() {
    printf("dwt [otpions] src_img.rgb <out_img.dwt>\n\
  -d, --dimension\t\tdimensions of src img, e.g. 1920x1080\n\
  -c, --components\t\tnumber of color components, default 3\n\
  -b, --depth\t\t\tbit depth, default 8\n\
  -l, --level\t\t\tDWT level, default 3\n\
  -D, --device\t\t\tcuda device\n\
  -f, --forward\t\t\tforward transform\n\
  -r, --reverse\t\t\treverse transform\n\
  -9, --97\t\t\t9/7 transform\n\
  -5, --53\t\t\t5/3 transform\n\
  -w  --write-visual\t\twrite output in visual (tiled) fashion instead of the linear\n");
}

template <typename T>
void processDWT(struct dwt *d, int forward, int writeVisual)
{
    int componentSize = d->pixWidth*d->pixHeight*sizeof(T);
    
    T *c_r_out, *backup ;
    hipMalloc((void**)&c_r_out, componentSize); //< aligned component size
    cudaCheckError("Alloc device memory");
    hipMemset(c_r_out, 0, componentSize);
    cudaCheckError("Memset device memory");
    
    hipMalloc((void**)&backup, componentSize); //< aligned component size
    cudaCheckError("Alloc device memory");
    hipMemset(backup, 0, componentSize);
    cudaCheckError("Memset device memory");
	
    if (d->components == 3) {
        /* Alloc two more buffers for G and B */
        T *c_g_out, *c_b_out;
        hipMalloc((void**)&c_g_out, componentSize); //< aligned component size
        cudaCheckError("Alloc device memory");
        hipMemset(c_g_out, 0, componentSize);
        cudaCheckError("Memset device memory");
        
        hipMalloc((void**)&c_b_out, componentSize); //< aligned component size
        cudaCheckError("Alloc device memory");
        hipMemset(c_b_out, 0, componentSize);
        cudaCheckError("Memset device memory");
        
        /* Load components */
        T *c_r, *c_g, *c_b;
        hipMalloc((void**)&c_r, componentSize); //< R, aligned component size
        cudaCheckError("Alloc device memory");
        hipMemset(c_r, 0, componentSize);
        cudaCheckError("Memset device memory");

        hipMalloc((void**)&c_g, componentSize); //< G, aligned component size
        cudaCheckError("Alloc device memory");
        hipMemset(c_g, 0, componentSize);
        cudaCheckError("Memset device memory");

        hipMalloc((void**)&c_b, componentSize); //< B, aligned component size
        cudaCheckError("Alloc device memory");
        hipMemset(c_b, 0, componentSize);
        cudaCheckError("Memset device memory");

        rgbToComponents(c_r, c_g, c_b, d->srcImg, d->pixWidth, d->pixHeight);
		

        /* Compute DWT and always store into file */

        nStage2dDWT(c_r, c_r_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward);
        nStage2dDWT(c_g, c_g_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward);
        nStage2dDWT(c_b, c_b_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward);
     
        // -------test----------
        // T *h_r_out=(T*)malloc(componentSize);
		// hipMemcpy(h_r_out, c_g_out, componentSize, hipMemcpyDeviceToHost);
        // int ii;
		// for(ii=0;ii<componentSize/sizeof(T);ii++) {
			// fprintf(stderr, "%d ", h_r_out[ii]);
			// if((ii+1) % (d->pixWidth) == 0) fprintf(stderr, "\n");
        // }
        // -------test----------
        
		
        /* Store DWT to file */
#ifdef OUTPUT        
        if (writeVisual) {
            writeNStage2DDWT(c_r_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".r");
            writeNStage2DDWT(c_g_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".g");
            writeNStage2DDWT(c_b_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".b");
        } else {
            writeLinear(c_r_out, d->pixWidth, d->pixHeight, d->outFilename, ".r");
            writeLinear(c_g_out, d->pixWidth, d->pixHeight, d->outFilename, ".g");
            writeLinear(c_b_out, d->pixWidth, d->pixHeight, d->outFilename, ".b");
        }
#endif


        hipFree(c_r);
        cudaCheckError("Cuda free");
        hipFree(c_g);
        cudaCheckError("Cuda free");
        hipFree(c_b);
        cudaCheckError("Cuda free");
        hipFree(c_g_out);
        cudaCheckError("Cuda free");
        hipFree(c_b_out);
        cudaCheckError("Cuda free");

    } 
    else if (d->components == 1) {
        //Load component
        T *c_r;
        hipMalloc((void**)&(c_r), componentSize); //< R, aligned component size
        cudaCheckError("Alloc device memory");
        hipMemset(c_r, 0, componentSize);
        cudaCheckError("Memset device memory");

        bwToComponent(c_r, d->srcImg, d->pixWidth, d->pixHeight);

        // Compute DWT 
        nStage2dDWT(c_r, c_r_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward);

        // Store DWT to file 
// #ifdef OUTPUT        
        if (writeVisual) {
            writeNStage2DDWT(c_r_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".out");
        } else {
            writeLinear(c_r_out, d->pixWidth, d->pixHeight, d->outFilename, ".lin.out");
        }
// #endif
        hipFree(c_r);
        cudaCheckError("Cuda free");
    }

    hipFree(c_r_out);
    cudaCheckError("Cuda free device");
    hipFree(backup);
    cudaCheckError("Cuda free device");
}

int main(int argc, char **argv) 
{
    int optindex = 0;
    char ch;
    struct option longopts[] = {
        {"dimension",   required_argument, 0, 'd'}, //dimensions of src img
        {"components",  required_argument, 0, 'c'}, //numger of components of src img
        {"depth",       required_argument, 0, 'b'}, //bit depth of src img
        {"level",       required_argument, 0, 'l'}, //level of dwt
        {"device",      required_argument, 0, 'D'}, //cuda device
        {"forward",     no_argument,       0, 'f'}, //forward transform
        {"reverse",     no_argument,       0, 'r'}, //reverse transform
        {"97",          no_argument,       0, '9'}, //9/7 transform
        {"53",          no_argument,       0, '5' }, //5/3transform
        {"write-visual",no_argument,       0, 'w' }, //write output (subbands) in visual (tiled) order instead of linear
        {"help",        no_argument,       0, 'h'}  
    };
    
    int pixWidth    = 0; //<real pixWidth
    int pixHeight   = 0; //<real pixHeight
    int compCount   = 3; //number of components; 3 for RGB or YUV, 4 for RGBA
    int bitDepth    = 8; 
    int dwtLvls     = 3; //default numuber of DWT levels
    int device      = 0;
    int forward     = 1; //forward transform
    int dwt97       = 1; //1=dwt9/7, 0=dwt5/3 transform
    int writeVisual = 0; //write output (subbands) in visual (tiled) order instead of linear
    char * pos;

    while ((ch = getopt_long(argc, argv, "d:c:b:l:D:fr95wh", longopts, &optindex)) != -1) {
        switch (ch) {
        case 'd':
            pixWidth = atoi(optarg);
            pos = strstr(optarg, "x");
            if (pos == NULL || pixWidth == 0 || (strlen(pos) >= strlen(optarg))) {
                usage();
                return -1;
            }
            pixHeight = atoi(pos+1);
            break;
        case 'c':
            compCount = atoi(optarg);
            break;
        case 'b':
            bitDepth = atoi(optarg);
            break;
        case 'l':
            dwtLvls = atoi(optarg);
            break;
        case 'D':
            device = atoi(optarg);
            break;
        case 'f':
            forward = 1;
            break;
        case 'r':
            forward = 0;
            break;
        case '9':
            dwt97 = 1;
            break;
        case '5':
            dwt97 = 0;
            break;
        case 'w':
            writeVisual = 1;
            break;
        case 'h':
            usage();
            return 0;
        case '?':
            return -1;
        default :
            usage();
            return -1;
        }
    }
	argc -= optind;
	argv += optind;

    if (argc == 0) { // at least one filename is expected
        printf("Please supply src file name\n");
        usage();
        return -1;
    }

    if (pixWidth <= 0 || pixHeight <=0) {
        printf("Wrong or missing dimensions\n");
        usage();
        return -1;
    }

    if (forward == 0) {
        writeVisual = 0; //do not write visual when RDWT
    }

    // device init
    int devCount;
    hipGetDeviceCount(&devCount);
    cudaCheckError("Get device count");
    if (devCount == 0) {
        printf("No CUDA enabled device\n");
        return -1;
    } 
    if (device < 0 || device > devCount -1) {
        printf("Selected device %d is out of bound. Devices on your system are in range %d - %d\n", 
               device, 0, devCount -1);
        return -1;
    }
    hipDeviceProp_t devProp;                                          
    hipGetDeviceProperties(&devProp, device);  
    cudaCheckError("Get device properties");
    if (devProp.major < 1) {                                         
        printf("Device %d does not support CUDA\n", device);
        return -1;
    }                                                                   
    printf("Using device %d: %s\n", device, devProp.name);
    hipSetDevice(device);
    cudaCheckError("Set selected device");

    struct dwt *d;
    d = (struct dwt *)malloc(sizeof(struct dwt));
    d->srcImg = NULL;
    d->pixWidth = pixWidth;
    d->pixHeight = pixHeight;
    d->components = compCount;
    d->dwtLvls  = dwtLvls;

    // file names
    d->srcFilename = (char *)malloc(strlen(argv[0]));
    strcpy(d->srcFilename, argv[0]);
    if (argc == 1) { // only one filename supplyed
        d->outFilename = (char *)malloc(strlen(d->srcFilename)+4);
        strcpy(d->outFilename, d->srcFilename);
        strcpy(d->outFilename+strlen(d->srcFilename), ".dwt");
    } else {
        d->outFilename = strdup(argv[1]);
    }

    //Input review
    printf("Source file:\t\t%s\n", d->srcFilename);
    printf(" Dimensions:\t\t%dx%d\n", pixWidth, pixHeight);
    printf(" Components count:\t%d\n", compCount);
    printf(" Bit depth:\t\t%d\n", bitDepth);
    printf(" DWT levels:\t\t%d\n", dwtLvls);
    printf(" Forward transform:\t%d\n", forward);
    printf(" 9/7 transform:\t\t%d\n", dwt97);
    
    //data sizes
    int inputSize = pixWidth*pixHeight*compCount; //<amount of data (in bytes) to proccess

    //load img source image
    hipHostMalloc((void **)&d->srcImg, inputSize);
    cudaCheckError("Alloc host memory");
    if (getImg(d->srcFilename, d->srcImg, inputSize) == -1) 
        return -1;

    /* DWT */
    if (forward == 1) {
        if(dwt97 == 1 )
            processDWT<float>(d, forward, writeVisual);
        else // 5/3
            processDWT<int>(d, forward, writeVisual);
    }
    else { // reverse
        if(dwt97 == 1 )
            processDWT<float>(d, forward, writeVisual);
        else // 5/3
            processDWT<int>(d, forward, writeVisual);
    }

    //writeComponent(r_cuda, pixWidth, pixHeight, srcFilename, ".g");
    //writeComponent(g_wave_cuda, 512000, ".g");
    //writeComponent(g_cuda, componentSize, ".g");
    //writeComponent(b_wave_cuda, componentSize, ".b");
    hipHostFree(d->srcImg);
    cudaCheckError("Cuda free host");

    return 0;
}
