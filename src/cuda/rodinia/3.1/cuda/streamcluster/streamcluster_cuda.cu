#include "hip/hip_runtime.h"
/***********************************************
	streamcluster_cuda.cu
	: parallelized code of streamcluster
	
	- original code from PARSEC Benchmark Suite
	- parallelization with CUDA API has been applied by
	
	Shawn Sang-Ha Lee - sl4ge@virginia.edu
	University of Virginia
	Department of Electrical and Computer Engineering
	Department of Computer Science
	
***********************************************/
#include "streamcluster_header.cu"

using namespace std;

// AUTO-ERROR CHECK FOR ALL CUDA FUNCTIONS
#define CUDA_SAFE_CALL( call) do {										\
   hipError_t err = call;												\
   if( hipSuccess != err) {											\
       fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",	\
               __FILE__, __LINE__, hipGetErrorString( err) );			\
   exit(EXIT_FAILURE);													\
   } } while (0)

#define THREADS_PER_BLOCK 512
#define MAXBLOCKS 65536
#define CUDATIME

// host memory
float *work_mem_h;
float *coord_h;

// device memory
float *work_mem_d;
float *coord_d;
int   *center_table_d;
bool  *switch_membership_d;
Point *p;

static int iter = 0;		// counter for total# of iteration


//=======================================
// Euclidean Distance
//=======================================
__device__ float
d_dist(int p1, int p2, int num, int dim, float *coord_d)
{
	float retval = 0.0;
	for(int i = 0; i < dim; i++){
		float tmp = coord_d[(i*num)+p1] - coord_d[(i*num)+p2];
		retval += tmp * tmp;
	}
	return retval;
}

//=======================================
// Kernel - Compute Cost
//=======================================
__global__ void
kernel_compute_cost(int num, int dim, long x, Point *p, int K, int stride,
					float *coord_d, float *work_mem_d, int *center_table_d, bool *switch_membership_d)
{
	// block ID and global thread ID
	const int bid  = blockIdx.x + gridDim.x * blockIdx.y;
	const int tid = blockDim.x * bid + threadIdx.x;

	if(tid < num)
	{
		float *lower = &work_mem_d[tid*stride];
		
		// cost between this point and point[x]: euclidean distance multiplied by weight
		float x_cost = d_dist(tid, x, num, dim, coord_d) * p[tid].weight;
		
		// if computed cost is less then original (it saves), mark it as to reassign
		if ( x_cost < p[tid].cost )
		{
			switch_membership_d[tid] = 1;
			lower[K] += x_cost - p[tid].cost;
		}
		// if computed cost is larger, save the difference
		else
		{
			lower[center_table_d[p[tid].assign]] += p[tid].cost - x_cost;
		}
	}
}

//=======================================
// Allocate Device Memory
//=======================================
void allocDevMem(int num, int dim)
{
	CUDA_SAFE_CALL( hipMalloc((void**) &center_table_d,	  num * sizeof(int))   );
	CUDA_SAFE_CALL( hipMalloc((void**) &switch_membership_d, num * sizeof(bool))  );
	CUDA_SAFE_CALL( hipMalloc((void**) &p,					  num * sizeof(Point)) );
	CUDA_SAFE_CALL( hipMalloc((void**) &coord_d,		num * dim * sizeof(float)) );
}

//=======================================
// Allocate Host Memory
//=======================================
void allocHostMem(int num, int dim)
{
	coord_h	= (float*) malloc( num * dim * sizeof(float) );
}

//=======================================
// Free Device Memory
//=======================================
void freeDevMem()
{
	CUDA_SAFE_CALL( hipFree(center_table_d)	  );
	CUDA_SAFE_CALL( hipFree(switch_membership_d) );
	CUDA_SAFE_CALL( hipFree(p)					  );
	CUDA_SAFE_CALL( hipFree(coord_d)			  );
}

//=======================================
// Free Host Memory
//=======================================
void freeHostMem()
{
	free(coord_h);
}

//=======================================
// pgain Entry - CUDA SETUP + CUDA CALL
//=======================================
float pgain( long x, Points *points, float z, long int *numcenters, int kmax, bool *is_center, int *center_table, bool *switch_membership, bool isCoordChanged,
							double *serial_t, double *cpu_to_gpu_t, double *gpu_to_cpu_t, double *alloc_t, double *kernel_t, double *free_t)
{	
#ifdef CUDATIME
	float tmp_t;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
#endif

	hipError_t error;
	
	int stride	= *numcenters + 1;			// size of each work_mem segment
	int K		= *numcenters ;				// number of centers
	int num		=  points->num;				// number of points
	int dim		=  points->dim;				// number of dimension
	int nThread =  num;						// number of threads == number of data points
	
	//=========================================
	// ALLOCATE HOST MEMORY + DATA PREPARATION
	//=========================================
	work_mem_h = (float*) malloc(stride * (nThread + 1) * sizeof(float) );
	// Only on the first iteration
	if(iter == 0)
	{
		allocHostMem(num, dim);
	}
	
	// build center-index table
	int count = 0;
	for( int i=0; i<num; i++)
	{
		if( is_center[i] )
		{
			center_table[i] = count++;
		}
	}

	// Extract 'coord'
	// Only if first iteration OR coord has changed
	if(isCoordChanged || iter == 0)
	{
		for(int i=0; i<dim; i++)
		{
			for(int j=0; j<num; j++)
			{
				coord_h[ (num*i)+j ] = points->p[j].coord[i];
			}
		}
	}
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*serial_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif

	//=======================================
	// ALLOCATE GPU MEMORY
	//=======================================
	CUDA_SAFE_CALL( hipMalloc((void**) &work_mem_d,  stride * (nThread + 1) * sizeof(float)) );
	// Only on the first iteration
	if( iter == 0 )
	{
		allocDevMem(num, dim);
	}
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*alloc_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif

	//=======================================
	// CPU-TO-GPU MEMORY COPY
	//=======================================
	// Only if first iteration OR coord has changed
	if(isCoordChanged || iter == 0)
	{
		CUDA_SAFE_CALL( hipMemcpy(coord_d,  coord_h,	 num * dim * sizeof(float), hipMemcpyHostToDevice) );
	}
	CUDA_SAFE_CALL( hipMemcpy(center_table_d,  center_table,  num * sizeof(int),   hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(p,  points->p,				   num * sizeof(Point), hipMemcpyHostToDevice) );
	
	CUDA_SAFE_CALL( hipMemset((void*) switch_membership_d, 0,			num * sizeof(bool))  );
	CUDA_SAFE_CALL( hipMemset((void*) work_mem_d,  		0, stride * (nThread + 1) * sizeof(float)) );
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*cpu_to_gpu_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif
	
	//=======================================
	// KERNEL: CALCULATE COST
	//=======================================
	// Determine the number of thread blocks in the x- and y-dimension
	int num_blocks 	 = (int) ((float) (num + THREADS_PER_BLOCK - 1) / (float) THREADS_PER_BLOCK);
	int num_blocks_y = (int) ((float) (num_blocks + MAXBLOCKS - 1)  / (float) MAXBLOCKS);
	int num_blocks_x = (int) ((float) (num_blocks+num_blocks_y - 1) / (float) num_blocks_y);	
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);

	kernel_compute_cost<<<grid_size, THREADS_PER_BLOCK>>>(	
															num,					// in:	# of data
															dim,					// in:	dimension of point coordinates
															x,						// in:	point to open a center at
															p,						// in:	data point array
															K,						// in:	number of centers
															stride,					// in:  size of each work_mem segment
															coord_d,				// in:	array of point coordinates
															work_mem_d,				// out:	cost and lower field array
															center_table_d,			// in:	center index table
															switch_membership_d		// out:  changes in membership
															);
	hipDeviceSynchronize();
	
	// error check
	error = hipGetLastError();
	if (error != hipSuccess)
	{
		printf("kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*kernel_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif
	
	//=======================================
	// GPU-TO-CPU MEMORY COPY
	//=======================================
	CUDA_SAFE_CALL( hipMemcpy(work_mem_h, 		  work_mem_d, 	stride * (nThread + 1) * sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipMemcpy(switch_membership, switch_membership_d,	 num * sizeof(bool),  hipMemcpyDeviceToHost) );
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*gpu_to_cpu_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif
	
	//=======================================
	// CPU (SERIAL) WORK
	//=======================================
	int number_of_centers_to_close = 0;
	float gl_cost_of_opening_x = z;
	float *gl_lower = &work_mem_h[stride * nThread];
	// compute the number of centers to close if we are to open i
	for(int i=0; i < num; i++)
	{
		if( is_center[i] )
		{
			float low = z;
		    for( int j = 0; j < num; j++ )
			{
				low += work_mem_h[ j*stride + center_table[i] ];
			}
			
		    gl_lower[center_table[i]] = low;
				
		    if ( low > 0 )
			{
				++number_of_centers_to_close;
				work_mem_h[i*stride+K] -= low;
		    }
		}
		gl_cost_of_opening_x += work_mem_h[i*stride+K];
	}

	//if opening a center at x saves cost (i.e. cost is negative) do so; otherwise, do nothing
	if ( gl_cost_of_opening_x < 0 )
	{
		for(int i = 0; i < num; i++)
		{
			bool close_center = gl_lower[center_table[points->p[i].assign]] > 0 ;
			if ( switch_membership[i] || close_center )
			{
				points->p[i].cost = dist(points->p[i], points->p[x], dim) * points->p[i].weight;
				points->p[i].assign = x;
			}
		}
		
		for(int i = 0; i < num; i++)
		{
			if( is_center[i] && gl_lower[center_table[i]] > 0 )
			{
				is_center[i] = false;
			}
		}
		
		if( x >= 0 && x < num)
		{
			is_center[x] = true;
		}
		*numcenters = *numcenters + 1 - number_of_centers_to_close;
	}
	else
	{
		gl_cost_of_opening_x = 0;
	}
	
	//=======================================
	// DEALLOCATE HOST MEMORY
	//=======================================
	free(work_mem_h);
	
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*serial_t += (double) tmp_t;
	
	hipEventRecord(start,0);
#endif

	//=======================================
	// DEALLOCATE GPU MEMORY
	//=======================================
	CUDA_SAFE_CALL( hipFree(work_mem_d) );
	
	
#ifdef CUDATIME
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	*free_t += (double) tmp_t;
#endif
	iter++;
	return -gl_cost_of_opening_x;
}
