//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the maximum read bandwidth of L1 cache for 64 bit read

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024
#define THREADS_PER_SM 1024
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 1024
#define ARRAY_SIZE 16384 //ARRAY_SIZE has to be less than L1_SIZE
#define L1_SIZE 32768  //L1 size in 64-bit. Volta L1 size is 128KB, i.e. 16K of 64-bit

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
	
	// thread index
	uint32_t tid = threadIdx.x;
	uint32_t uid = blockIdx.x * blockDim.x + tid;
	
	// a register to avoid compiler optimization
	float sink0 = 0;
	float sink1 = 0;
	float sink2 = 0;
	float sink3 = 0;
	
	// populate l1 cache to warm up
	for (uint32_t i = tid; i<ARRAY_SIZE; i+=THREADS_PER_BLOCK) {
		float* ptr = posArray + i;
		// use ca modifier to cache the load in L1
		asm volatile ("{\t\n"
			".reg .f32 data;\n\t"
			"ld.global.ca.f32 data, [%1];\n\t"
			"add.f32 %0, data, %0;\n\t"
			"}" : "+f"(sink0) : "l"(ptr) : "memory"
		);
	}
	
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	
	// load data from l1 cache and accumulate
	for(uint32_t j=0; j<REPEAT_TIMES; j++){
        	        float* ptr = posArray + ((tid + (j*WARP_SIZE))%ARRAY_SIZE);
                	asm volatile ("{\t\n"
                        	".reg .f32 data;\n\t"
				"ld.global.ca.f32 data, [%1+0];\n\t"
				"add.f32 %0, data, %0;\n\t"
				"}" : "+f"(sink0) : "l"(ptr) : "memory"
                	);
	}

	// synchronize all threads
	asm volatile("bar.sync 0;");
	
	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[uid] = start;
	stopClk[uid] = stop;
	dsink[uid] = sink0+sink1+sink2+sink3;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	float *posArray = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *dsink = (float*) malloc(TOTAL_THREADS*sizeof(float));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        float *posArray_g;
        float *dsink_g;
	
	for (uint32_t i=0; i<ARRAY_SIZE; i++)
		posArray[i] = (float)i;
		
	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&dsink_g, TOTAL_THREADS*sizeof(float)) );
	
	gpuErrchk( hipMemcpy(posArray_g, posArray, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );

	l1_bw<<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, dsink_g, posArray_g);
        gpuErrchk( hipPeekAtLastError() );
	
	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, TOTAL_THREADS*sizeof(float), hipMemcpyDeviceToHost) );

        float bw;
	bw = (float)(REPEAT_TIMES*THREADS_PER_SM*4)/((float)(stopClk[0]-startClk[0]));
	printf("L1 bandwidth = %f (byte/clk/SM)\n", bw);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 
