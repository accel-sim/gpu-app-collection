//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark measures the latency of L1 cache

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 1   //Launch only one thread to calcaulte the latency using a pointer-chasing array technique
#define WARP_SIZE 32
#define ITERS 32768       //iterate over the array ITERS times
#define ARRAY_SIZE 4096    //size of the array

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

//TO DO: @Jason, please change the code to be similar to the L2/DRAM latency format
//Measure latency of ITERS reads. 
__global__ void l1_lat(uint32_t *startClk, uint32_t *stopClk, uint64_t *posArray, uint64_t *dsink){

	// thread index
	uint32_t tid = threadIdx.x;

	// one thread to initialize the pointer-chasing array
	if (tid == 0){
	for (uint32_t i=0; i<(ARRAY_SIZE-1); i++)
		posArray[i] = (uint64_t)(posArray + i + 1);

	posArray[ARRAY_SIZE-1] = (uint64_t)posArray;
	}

	if(tid < THREADS_NUM){
		// a register to avoid compiler optimization
		uint64_t *ptr = posArray + tid;
		uint64_t ptr1, ptr0;
	
		// initialize the thread pointer with the start address of the array
		// use ca modifier to cache the in L1
		asm volatile ("{\t\n"
			"ld.global.ca.u64 %0, [%1];\n\t"
			"}" : "=l"(ptr1) : "l"(ptr) : "memory"
		);
	
		// synchronize all threads
		asm volatile ("bar.sync 0;");

		// start timing
		uint32_t start = 0;
		asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");


		// pointer-chasing ITERS times
		// use ca modifier to cache the load in L1
		for(uint32_t i=0; i<ITERS; ++i) {	
			asm volatile ("{\t\n"
				"ld.global.ca.u64 %0, [%1];\n\t"
				"}" : "=l"(ptr0) : "l"((uint64_t*)ptr1) : "memory"
			);
			ptr1 = ptr0;    //swap the register for the next load

		}

		// stop timing
		uint32_t stop = 0;
		asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

		// write time and data back to memory
		startClk[tid] = start;
		stopClk[tid] = stop;
		dsink[tid] = ptr1;
	}
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(THREADS_NUM*sizeof(uint32_t));
	uint64_t *dsink = (uint64_t*) malloc(THREADS_NUM*sizeof(uint64_t));
	
	uint32_t *startClk_g;
        uint32_t *stopClk_g;
        uint64_t *posArray_g;
        uint64_t *dsink_g;
	
	gpuErrchk( hipMalloc(&startClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, THREADS_NUM*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&posArray_g, ARRAY_SIZE*sizeof(uint64_t)) );
	gpuErrchk( hipMalloc(&dsink_g, THREADS_NUM*sizeof(uint64_t)) );
	
	l1_lat<<<1,THREADS_NUM>>>(startClk_g, stopClk_g, posArray_g, dsink_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, THREADS_NUM*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(dsink, dsink_g, THREADS_NUM*sizeof(uint64_t), hipMemcpyDeviceToHost) );
	printf("L1 Latency  = %12.4f cycles\n", (float)(stopClk[0]-startClk[0])/ITERS);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 
