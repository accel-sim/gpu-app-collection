#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1
#define THREADS_PER_SM 1
#define BLOCKS_NUM 1
#define TOTAL_THREADS (THREADS_PER_BLOCK*BLOCKS_NUM)
#define WARP_SIZE 32
#define REPEAT_TIMES 4096

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


template <class T>
__global__ void max_flops(uint32_t *startClk, uint32_t *stopClk, T *data1, T *res) {
	int gid = blockIdx.x*blockDim.x + threadIdx.x;
	//register T s1 = data1[gid];
	//register T s2 = data2[gid];
	//register T result = 0;
	uint32_t index = 0;
	int32_t offset = 10;
	// synchronize all threads
	asm volatile ("bar.sync 0;");
	
	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");
	//printf("%ld \n", &data1[0]);
	for (int j=0 ; j<REPEAT_TIMES ; ++j) {
		index = atomicAdd(&data1[index], offset);
		//printf("index = %d", index);
	}
	// synchronize all threads
	asm volatile("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[gid] = start;
	stopClk[gid] = stop;
	res[gid] = data1[0];
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	int32_t *data1 = (int32_t*) malloc(REPEAT_TIMES*sizeof(int32_t));
	//int32_t *data2 = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));
	int32_t *res = (int32_t*) malloc(TOTAL_THREADS*sizeof(int32_t));

	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	int32_t *data1_g;
	//int32_t *data2_g;
	int32_t *res_g;

	int32_t stride = 1;

	for (int32_t i=0; i<(REPEAT_TIMES); i++)
		data1[i] = (i+stride)%REPEAT_TIMES;


	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&data1_g, REPEAT_TIMES*sizeof(int32_t)) );
	//gpuErrchk( cudaMalloc(&data2_g, TOTAL_THREADS*sizeof(int32_t)) );
	gpuErrchk( hipMalloc(&res_g, TOTAL_THREADS*sizeof(int32_t)) );
	//printf("address = %ld\n", (long)data1_g);
	gpuErrchk( hipMemcpy(data1_g, data1, REPEAT_TIMES*sizeof(int32_t), hipMemcpyHostToDevice) );
	//gpuErrchk( cudaMemcpy(data2_g, data2, TOTAL_THREADS*sizeof(int32_t), cudaMemcpyHostToDevice) );

	max_flops<int32_t><<<BLOCKS_NUM,THREADS_PER_BLOCK>>>(startClk_g, stopClk_g, data1_g, res_g);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(res, res_g, TOTAL_THREADS*sizeof(int32_t), hipMemcpyDeviceToHost) );

	float latency;
	latency = ((float)(stopClk[0]-startClk[0]))/((float)(REPEAT_TIMES));
	printf("int32 latency = %f (clk)\n", latency);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);

	return 0;
} 

