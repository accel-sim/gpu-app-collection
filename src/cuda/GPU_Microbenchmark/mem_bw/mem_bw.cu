
//This benchmark measures the maximum read bandwidth of GPU memory
//Compile this file using the following command to disable L1 cache:
//    nvcc -Xptxas -dlcm=cg -Xptxas -dscm=wt l2_bw.cu

//This code have been tested on Volta V100 architecture
//You can check the mem BW from the NVPROF (dram_read_throughput+dram_write_throughput)

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCKS_NUM 160
#define THREADS_NUM 1024 //thread number/block
#define TOTAL_THREADS (BLOCKS_NUM*THREADS_NUM)
#define ARRAY_SIZE 8388608   //Array size has to exceed L2 size to avoid L2 cache residence
#define WARP_SIZE 32 
#define L2_SIZE 1572864 //number of floats L2 can store
#define clock_freq_MHZ 1132

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess) {
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

/*
Four Vector Addition using flost4 types
Send as many as float4 read requests on the flight to increase Row buffer locality of DRAM and hit the max BW
 */

__global__ void mem_bw (float* A,  float* B, float* C, float* D, float* E, float* F, uint32_t *startClk, uint32_t *stopClk){
	// block and thread index
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// start timing
	uint32_t start = 0;
	asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

	for(int i = idx; i < ARRAY_SIZE/4; i += blockDim.x * gridDim.x) {
		float4 a1 = reinterpret_cast<float4*>(A)[i];
		float4 b1 = reinterpret_cast<float4*>(B)[i];
		float4 d1 = reinterpret_cast<float4*>(D)[i];
		float4 e1 = reinterpret_cast<float4*>(E)[i];
		float4 f1 = reinterpret_cast<float4*>(F)[i];
		float4 c1;

		c1.x = a1.x + b1.x + d1.x + e1.x + f1.x;
		c1.y = a1.y + b1.y + d1.y + e1.y + f1.y;
		c1.z = a1.z + b1.z + d1.z + e1.z + f1.z;
		c1.w = a1.w + b1.w + d1.w + e1.w + f1.w;

		reinterpret_cast<float4*>(C)[i] = c1;
	}

	// synchronize all threads

	// synchronize all threads
	asm volatile ("bar.sync 0;");

	// stop timing
	uint32_t stop = 0;
	asm volatile("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

	// write time and data back to memory
	startClk[idx] = start;
	stopClk[idx] = stop;
}

int main(){
	uint32_t *startClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	uint32_t *stopClk = (uint32_t*) malloc(TOTAL_THREADS*sizeof(uint32_t));
	float *A = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *B = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *C = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *D = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *E = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *F = (float*) malloc(ARRAY_SIZE*sizeof(float));


	uint32_t *startClk_g;
	uint32_t *stopClk_g;
	float *A_g;
	float *B_g;
	float *C_g;
	float *D_g;
	float *E_g;
	float *F_g;


	for (uint32_t i=0; i<ARRAY_SIZE; i++){
		A[i] = (float)i;
		B[i] = (float)i;
		D[i] = (float)i;
		E[i] = (float)i;
		F[i] = (float)i;

	}

	gpuErrchk( hipMalloc(&startClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&stopClk_g, TOTAL_THREADS*sizeof(uint32_t)) );
	gpuErrchk( hipMalloc(&A_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&B_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&C_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&D_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&E_g, ARRAY_SIZE*sizeof(float)) );
	gpuErrchk( hipMalloc(&F_g, ARRAY_SIZE*sizeof(float)) );


	gpuErrchk( hipMemcpy(A_g, A, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(B_g, B, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(D_g, D, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(E_g, E, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(F_g, F, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice) );

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	mem_bw<<<BLOCKS_NUM,THREADS_NUM>>>(A_g, B_g, C_g, D_g, E_g, F_g, startClk_g, stopClk_g);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(startClk, startClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(stopClk, stopClk_g, TOTAL_THREADS*sizeof(uint32_t), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(C, C_g, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost) );

	float mem_bw;
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	unsigned N = ARRAY_SIZE * 6 * 4; //6 arrays of floats types

	mem_bw = (float)(N)/((float)(stopClk[0]-startClk[0]));  
	printf("Mem BW= %f (Byte/Clk)\n", mem_bw);
	printf("Mem BW= %f (GB/sec)\n", (float)N/milliseconds/1e6);
	printf("Total Clk number = %u \n", stopClk[0]-startClk[0]);
}

