/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "args.h"

#include "model.h"
#include "tpacf_kernel.cu"  

#define CUDA_ERRCK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
  printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
  return -1; }}

extern unsigned int NUM_SETS;
extern unsigned int NUM_ELEMENTS;

int 
main( int argc, char** argv) 
{
  struct pb_TimerSet timers;
  struct pb_Parameters *params;

  pb_InitializeTimerSet( &timers );
  params = pb_ReadParameters( &argc, argv );

  options args;
  parse_args(argc, argv, &args);
  
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  NUM_ELEMENTS = args.npoints;
  NUM_SETS = args.random_count;
  int num_elements = NUM_ELEMENTS; 
  
  printf("Min distance: %f arcmin\n", min_arcmin);
  printf("Max distance: %f arcmin\n", max_arcmin);
  printf("Bins per dec: %i\n", bins_per_dec);
  printf("Total bins  : %i\n", NUM_BINS);

  //read in files 
  unsigned mem_size = (1+NUM_SETS)*num_elements*sizeof(struct cartesian);
  unsigned f_mem_size = (1+NUM_SETS)*num_elements*sizeof(REAL);

  // container for all the points read from files
  struct cartesian *h_all_data;
  h_all_data = (struct cartesian*) malloc(mem_size); 
  // Until I can get libs fixed
    
  // iterator for data files
  struct cartesian *working = h_all_data;
    
  // go through and read all data and random points into h_all_data
  pb_SwitchToTimer( &timers, pb_TimerID_IO );
  readdatafile(params->inpFiles[0], working, num_elements);
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  working += num_elements;
  for(int i = 0; i < (NUM_SETS); i++)
    {
      pb_SwitchToTimer( &timers, pb_TimerID_IO );
      readdatafile(params->inpFiles[i+1], working, num_elements);
      pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

      working += num_elements;
    }

  // split into x, y, and z arrays
  REAL * h_x_data = (REAL*) malloc (3*f_mem_size);
  REAL * h_y_data = h_x_data + NUM_ELEMENTS*(NUM_SETS+1);
  REAL * h_z_data = h_y_data + NUM_ELEMENTS*(NUM_SETS+1);
  for(int i = 0; i < (NUM_SETS+1); ++i)
    {
      for(int j = 0; j < NUM_ELEMENTS; ++j)
	{
	  h_x_data[i*NUM_ELEMENTS+j] = h_all_data[i*NUM_ELEMENTS+j].x;
	  h_y_data[i*NUM_ELEMENTS+j] = h_all_data[i*NUM_ELEMENTS+j].y;
	  h_z_data[i*NUM_ELEMENTS+j] = h_all_data[i*NUM_ELEMENTS+j].z;
	}
    }

  // from on use x, y, and z arrays, free h_all_data
  free(h_all_data);
  pb_SwitchToTimer( &timers, pb_TimerID_COPY );

  // allocate cuda memory to hold all points
  REAL * d_x_data;
  hipMalloc((void**) & d_x_data, 3*f_mem_size);
  CUDA_ERRCK
  REAL * d_y_data = d_x_data + NUM_ELEMENTS*(NUM_SETS+1);
  REAL * d_z_data = d_y_data + NUM_ELEMENTS*(NUM_SETS+1);

  // allocate cuda memory to hold final histograms
  // (1 for dd, and NUM_SETS for dr and rr apiece)
  hist_t * d_hists;
  hipMalloc((void**) & d_hists, NUM_BINS*(NUM_SETS*2+1)*sizeof(hist_t) );
  CUDA_ERRCK
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  // allocate system memory for final histograms
  hist_t *new_hists = (hist_t *) malloc(NUM_BINS*(NUM_SETS*2+1)*
					sizeof(hist_t));

  // Initialize the boundary constants for bin search
  initBinB( &timers );
  CUDA_ERRCK

  // **===------------------ Kick off TPACF on CUDA------------------===**
  pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  hipMemcpy(d_x_data, h_x_data, 3*f_mem_size, hipMemcpyHostToDevice);
  CUDA_ERRCK
  pb_SwitchToTimer( &timers, pb_TimerID_KERNEL );

  TPACF(d_hists, d_x_data, d_y_data, d_z_data);

  pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  hipMemcpy(new_hists, d_hists, NUM_BINS*(NUM_SETS*2+1)*
	     sizeof(hist_t), hipMemcpyDeviceToHost);
  CUDA_ERRCK
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  // **===-----------------------------------------------------------===**

  // references into output histograms
  hist_t *dd_hist = new_hists;
  hist_t *rr_hist = dd_hist + NUM_BINS;
  hist_t *dr_hist = rr_hist + NUM_BINS*NUM_SETS;

  // add up values within dr and rr
  int rr[NUM_BINS];
  for(int i=0; i<NUM_BINS; i++)
    {
      rr[i] = 0;
    }
  for(int i=0; i<NUM_SETS; i++)
    {
      for(int j=0; j<NUM_BINS; j++)
	{
	  rr[j] += rr_hist[i*NUM_BINS + j];
	}
    }
  int dr[NUM_BINS];
  for(int i=0; i<NUM_BINS; i++)
    {
      dr[i] = 0;
    }
  for(int i=0; i<NUM_SETS; i++)
    {
      for(int j=0; j<NUM_BINS; j++)
	{
	  dr[j] += dr_hist[i*NUM_BINS + j];
	}
    }

  //int dd_t = 0;
  //int dr_t = 0;
  //int rr_t = 0;
  FILE *outfile;
  if ((outfile = fopen(params->outFile, "w")) == NULL)
    {
      fprintf(stderr, "Unable to open output file %s for writing, "
	      "assuming stdout\n", params->outFile);
      outfile = stdout;
    }
  
  pb_SwitchToTimer( &timers, pb_TimerID_IO );
  // print out final histograms + omega (while calculating omega)
  for(int i=0; i<NUM_BINS; i++)
    {
      //REAL w = (100.0 * dd_hist[i] - dr[i]) / rr[i] + 1.0;
      //fprintf(outfile, "%f\n", w);
      fprintf(outfile, "%d\n%d\n%d\n", dd_hist[i], dr[i], rr[i]);
//      dd_t += dd_hist[i];
//      dr_t += dr[i];
//      rr_t += rr[i];
    }
  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );

  if(outfile != stdout)
    fclose(outfile);

  // cleanup memory
  free(new_hists);
  free( h_x_data);

  pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  hipFree( d_hists );
  hipFree( d_x_data );

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);
}

