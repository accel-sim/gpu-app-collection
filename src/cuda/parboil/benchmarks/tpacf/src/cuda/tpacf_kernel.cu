#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

#include <assert.h>
#include "model.h"
#include <math.h>

#define WARP_SIZE 32
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#define BLOCK_SIZE 256
#define NUM_WARPS (BLOCK_SIZE/WARP_SIZE)
#define HISTS_PER_WARP 16
#define NUM_HISTOGRAMS  (NUM_WARPS*HISTS_PER_WARP)
#define THREADS_PER_HIST (WARP_SIZE/HISTS_PER_WARP)

REAL** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

__constant__ REAL dev_binb[NUM_BINS+1];

unsigned int NUM_SETS;
unsigned int NUM_ELEMENTS;

// create the bin boundaries
void initBinB( struct pb_TimerSet *timers )
{
  REAL *binb = (REAL*)malloc((NUM_BINS+1)*sizeof(REAL));
  for (int k = 0; k < NUM_BINS+1; k++)
    {
      binb[k] = cos(pow(10.0, (log10(min_arcmin) + k*1.0/bins_per_dec)) 
		    / 60.0*D2R);
    }
  pb_SwitchToTimer( timers, pb_TimerID_COPY );
  hipMemcpyToSymbol(HIP_SYMBOL(dev_binb), binb, (NUM_BINS+1)*sizeof(REAL));
  pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );
  free(binb);
}

__global__ 
void gen_hists( hist_t* histograms, REAL* all_x_data, REAL* all_y_data, 
		REAL* all_z_data, int NUM_SETS, int NUM_ELEMENTS )
{
  unsigned int bx = blockIdx.x;
  unsigned int tid = threadIdx.x;
  bool do_self = (bx < (NUM_SETS + 1));

  REAL* data_x;
  REAL* data_y;
  REAL* data_z;
  REAL* random_x;
  REAL* random_y;
  REAL* random_z;

  __shared__ struct cartesian data_s[BLOCK_SIZE];
  
  __shared__ unsigned int 
    warp_hists[NUM_BINS][NUM_HISTOGRAMS]; // 640B <1k  
    
  for(unsigned int w = 0; w < NUM_BINS*NUM_HISTOGRAMS; w += BLOCK_SIZE )
    {
      if(w+tid < NUM_BINS*NUM_HISTOGRAMS)
	{
	  warp_hists[(w+tid)/NUM_HISTOGRAMS][(w+tid)%NUM_HISTOGRAMS] = 0;
	}
    }
    
  // Get stuff into shared memory to kick off the loop.
  if( !do_self)
    {
      data_x = all_x_data;
      data_y = all_y_data;
      data_z = all_z_data;
      random_x = all_x_data + NUM_ELEMENTS * (bx - NUM_SETS);
      random_y = all_y_data + NUM_ELEMENTS * (bx - NUM_SETS);
      random_z = all_z_data + NUM_ELEMENTS * (bx - NUM_SETS);
    }
  else
    {
      random_x = all_x_data + NUM_ELEMENTS * (bx);
      random_y = all_y_data + NUM_ELEMENTS * (bx);
      random_z = all_z_data + NUM_ELEMENTS * (bx);
      
      data_x = random_x;
      data_y = random_y;
      data_z = random_z;
    }
    
  // Iterate over all data points
  for(unsigned int i = 0; i < NUM_ELEMENTS; i += BLOCK_SIZE )
    {
      // load current set of data into shared memory
      // (total of BLOCK_SIZE points loaded)
      if( tid + i < NUM_ELEMENTS )
	{ // reading outside of bounds is a-okay
	  data_s[tid] = (struct cartesian)
            {data_x[tid + i], data_y[tid + i], data_z[tid + i]};
	}
      
      __syncthreads();

      // Iterate over all random points
      for(unsigned int j = (do_self ? i+1 : 0); j < NUM_ELEMENTS; 
	  j += BLOCK_SIZE)
	{
	  // load current random point values
	  REAL random_x_s;
	  REAL random_y_s;
	  REAL random_z_s;
	  
	  if(tid + j < NUM_ELEMENTS)
	    {
	      random_x_s = random_x[tid + j];
	      random_y_s = random_y[tid + j];
	      random_z_s = random_z[tid + j];
	    }

	  // Iterate for all elements of current set of data points 
	  // (BLOCK_SIZE iterations per thread)
	  // Each thread calcs against 1 random point within cur set of random
	  // (so BLOCK_SIZE threads covers all random points within cur set)
	  for(unsigned int k = 0; 
	      (k < BLOCK_SIZE) && (k+i < NUM_ELEMENTS);
	      k += 1)
	    {
	      // do actual calculations on the values:
	      REAL distance = 
	        data_s[k].x * random_x_s +
		    data_s[k].y * random_y_s +
		    data_s[k].z * random_z_s;

	      unsigned int bin_index;

	      // run binary search to find bin_index
	      unsigned int min = 0;
	      unsigned int max = NUM_BINS;
	      {
		unsigned int k2;
	      
		while (max > min+1)
		  {
		    k2 = (min + max) / 2;
		    if (distance >= dev_binb[k2]) 
		      max = k2;
		    else 
		      min = k2;
		  }
		bin_index = max - 1;
	      }

	      unsigned int warpnum = tid / (WARP_SIZE/HISTS_PER_WARP);
	      if((distance < dev_binb[min]) && (distance >= dev_binb[max]) && 
		 (!do_self || (tid + j > i + k)) && (tid + j < NUM_ELEMENTS))
		{
		  atomicAdd(&warp_hists[bin_index][warpnum], 1U);
		}
	    }
	}
    }
    
  // coalesce the histograms in a block
  unsigned int warp_index = tid & ( (NUM_HISTOGRAMS>>1) - 1);
  unsigned int bin_index = tid / (NUM_HISTOGRAMS>>1);
  for(unsigned int offset = NUM_HISTOGRAMS >> 1; offset > 0; 
      offset >>= 1)
    {
      for(unsigned int bin_base = 0; bin_base < NUM_BINS; 
	  bin_base += BLOCK_SIZE/ (NUM_HISTOGRAMS>>1))
	{
	  __syncthreads();
	  if(warp_index < offset && bin_base+bin_index < NUM_BINS )
	    {
	      unsigned long sum =
		warp_hists[bin_base + bin_index][warp_index] + 
		warp_hists[bin_base + bin_index][warp_index+offset];
	      warp_hists[bin_base + bin_index][warp_index] = sum;
	    }
	}
    }
    
  __syncthreads();
    
  // Put the results back in the real histogram
  // warp_hists[x][0] holds sum of all locations of bin x
  hist_t* hist_base = histograms + NUM_BINS * bx;
  if(tid < NUM_BINS)
    {
      hist_base[tid] = warp_hists[tid][0];
    }
}

void TPACF(hist_t * histograms, REAL* d_x_data, REAL* d_y_data, 
	   REAL* d_z_data)
{
  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(NUM_SETS*2 + 1);

  gen_hists <<< dimGrid, dimBlock >>> ( histograms, d_x_data, 
					d_y_data, d_z_data, NUM_SETS, 
					NUM_ELEMENTS);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
