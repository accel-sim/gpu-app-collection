#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#ifndef _PRESCAN_CU_
#define _PRESCAN_CU_

#include <assert.h>
#include "model.h"
#include <math.h>

#define WARP_SIZE 32
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#define BLOCK_SIZE 256
#define NUM_WARPS (BLOCK_SIZE/WARP_SIZE)
#define HISTS_PER_WARP 16
#define NUM_HISTOGRAMS  (NUM_WARPS*HISTS_PER_WARP)
#define THREADS_PER_HIST (WARP_SIZE/HISTS_PER_WARP)

float** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

__constant__ float dev_binb[NUM_BINS+1];

unsigned int NUM_SETS;
unsigned int NUM_ELEMENTS;

// create the bin boundaries
void initBinB( struct pb_TimerSet *timers )
{
  float *binb = (float*)malloc((NUM_BINS+1)*sizeof(float));
  for (int k = 0; k < NUM_BINS+1; k++)
    {
      binb[k] = cos(pow(10.0, (log10(min_arcmin) + k*1.0/bins_per_dec)) 
		    / 60.0*D2R);
    }
  pb_SwitchToTimer( timers, pb_TimerID_COPY );
  hipMemcpyToSymbol(HIP_SYMBOL(dev_binb), binb, (NUM_BINS+1)*sizeof(float));
  pb_SwitchToTimer( timers, pb_TimerID_COMPUTE );
  free(binb);
}

__global__ 
void gen_hists( hist_t* histograms, float* all_x_data, float* all_y_data, 
		float* all_z_data, int NUM_SETS, int NUM_ELEMENTS )
{
  unsigned int bx = blockIdx.x;
  unsigned int tid = threadIdx.x;
  bool do_self = (bx < (NUM_SETS + 1));

  float* data_x;
  float* data_y;
  float* data_z;
  float* random_x;
  float* random_y;
  float* random_z;

  __shared__ unsigned int
    warp_hists[NUM_BINS][NUM_HISTOGRAMS]; // 640B <1k  
    
  for(unsigned int w = 0; w < NUM_BINS*NUM_HISTOGRAMS; w += BLOCK_SIZE )
    {
      if(w+tid < NUM_BINS*NUM_HISTOGRAMS)
	{
	  warp_hists[(w+tid)/NUM_HISTOGRAMS][(w+tid)%NUM_HISTOGRAMS] = 0;
	}
    }
    
  // Get stuff into shared memory to kick off the loop.
  if( !do_self)
    {
      data_x = all_x_data;
      data_y = all_y_data;
      data_z = all_z_data;
      random_x = all_x_data + NUM_ELEMENTS * (bx - NUM_SETS);
      random_y = all_y_data + NUM_ELEMENTS * (bx - NUM_SETS);
      random_z = all_z_data + NUM_ELEMENTS * (bx - NUM_SETS);
    }
  else
    {
      random_x = all_x_data + NUM_ELEMENTS * (bx);
      random_y = all_y_data + NUM_ELEMENTS * (bx);
      random_z = all_z_data + NUM_ELEMENTS * (bx);
      
      data_x = random_x;
      data_y = random_y;
      data_z = random_z;
    }
    
  // Iterate over all random points
  for(unsigned int j = 0; j < NUM_ELEMENTS; j += BLOCK_SIZE)
    {
      // load current random point values
      float random_x_s;
      float random_y_s;
      float random_z_s;
	  
      if(tid + j < NUM_ELEMENTS)
        {
	  random_x_s = random_x[tid + j];
	  random_y_s = random_y[tid + j];
	  random_z_s = random_z[tid + j];
	}

      // Iterate over all data points
      // If do_self, then use a tighter bound on the number of data points.
      for(unsigned int k = 0;
	  k < NUM_ELEMENTS && (do_self ? k < j + BLOCK_SIZE : 1); k++)
	{
	  // do actual calculations on the values:
	  float distance = data_x[k] * random_x_s + 
	    data_y[k] * random_y_s + 
	    data_z[k] * random_z_s ;

	  unsigned int bin_index;

	  // run binary search to find bin_index
	  unsigned int min = 0;
	  unsigned int max = NUM_BINS;
	  {
	    unsigned int k2;
	      
	    while (max > min+1)
	      {
		k2 = (min + max) / 2;
		if (distance >= dev_binb[k2]) 
		  max = k2;
		else 
		  min = k2;
	      }
	    bin_index = max - 1;
	  }

	  unsigned int warpnum = tid / (WARP_SIZE/HISTS_PER_WARP);
	  if((distance < dev_binb[min]) && (distance >= dev_binb[max]) && 
	     (!do_self || (tid + j > k)) && (tid + j < NUM_ELEMENTS))
	    {
	      atomicAdd(&warp_hists[bin_index][warpnum], 1U);
	    }
	}
    }
    
  // coalesce the histograms in a block
  unsigned int warp_index = tid & ( (NUM_HISTOGRAMS>>1) - 1);
  unsigned int bin_index = tid / (NUM_HISTOGRAMS>>1);
  for(unsigned int offset = NUM_HISTOGRAMS >> 1; offset > 0; 
      offset >>= 1)
    {
      for(unsigned int bin_base = 0; bin_base < NUM_BINS; 
	  bin_base += BLOCK_SIZE/ (NUM_HISTOGRAMS>>1))
	{
	  __syncthreads();
	  if(warp_index < offset && bin_base+bin_index < NUM_BINS )
	    {
	      unsigned long sum =
		warp_hists[bin_base + bin_index][warp_index] + 
		warp_hists[bin_base + bin_index][warp_index+offset];
	      warp_hists[bin_base + bin_index][warp_index] = sum;
	    }
	}
    }
    
  __syncthreads();
    
  // Put the results back in the real histogram
  // warp_hists[x][0] holds sum of all locations of bin x
  hist_t* hist_base = histograms + NUM_BINS * bx;
  if(tid < NUM_BINS)
    {
      hist_base[tid] = warp_hists[tid][0];
    }
}

void TPACF(hist_t * histograms, float* d_x_data, float* d_y_data, 
	   float* d_z_data)
{
  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(NUM_SETS*2 + 1);

  gen_hists <<< dimGrid, dimBlock >>> ( histograms, d_x_data, 
					d_y_data, d_z_data, NUM_SETS, 
					NUM_ELEMENTS);
}
// **===-----------------------------------------------------------===**


#endif // _PRESCAN_CU_
