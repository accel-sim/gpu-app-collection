/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * C code for creating the Q data structure for fast convolution-based 
 * Hessian multiplication for arbitrary k-space trajectories.
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis 
 *      function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 *
 * recommended g++ options:
 *  -O3 -lm -ffast-math -funroll-all-loops
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <malloc.h>

#include <parboil.h>

#include "file.h"
#include "computeQ.cu"

static void
setupMemoryGPU(int num, int size, float*& dev_ptr, float*& host_ptr)
{
  hipMalloc ((void **) &dev_ptr, num * size);
  CUDA_ERRCK;
  hipMemcpy (dev_ptr, host_ptr, num * size, hipMemcpyHostToDevice);
  CUDA_ERRCK;
}

static void
cleanupMemoryGPU(int num, int size, float *& dev_ptr, float * host_ptr)
{
  hipMemcpy (host_ptr, dev_ptr, num * size, hipMemcpyDeviceToHost);
  CUDA_ERRCK;
  hipFree(dev_ptr);
  CUDA_ERRCK;
}

int
main (int argc, char *argv[]) {
  int numX, numK;		/* Number of X and K values */
  int original_numK;		/* Number of K values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */
  float *phiR, *phiI;		/* Phi values (complex) */
  float *phiMag;		/* Magnitude of Phi */
  float *Qr, *Qi;		/* Q signal (complex) */

  struct kValues* kVals;

  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);

  /* Read command line */
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
    {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
    }
  
  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData(params->inpFiles[0],
	    &original_numK, &numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI);

  /* Reduce the number of k-space samples if a number is given
   * on the command line */
  if (argc < 2)
    numK = original_numK;
  else
    {
      int inputK;
      char *end;
      inputK = strtol(argv[1], &end, 10);
      if (end == argv[1])
	{
	  fprintf(stderr, "Expecting an integer parameter\n");
	  exit(-1);
	}

      numK = MIN(inputK, original_numK);
    }

  printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  /* Create CPU data structures */
  createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);

  /* GPU section 1 (precompute PhiMag) */
  {
    /* Mirror several data structures on the device */
    float *phiR_d, *phiI_d;
    float *phiMag_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    setupMemoryGPU(numK, sizeof(float), phiR_d, phiR);
    setupMemoryGPU(numK, sizeof(float), phiI_d, phiI);
    hipMalloc((void **)&phiMag_d, numK * sizeof(float));
    CUDA_ERRCK;

    hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    computePhiMag_GPU(numK, phiR_d, phiI_d, phiMag_d);

    hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    cleanupMemoryGPU(numK, sizeof(float), phiMag_d, phiMag);
    hipFree(phiR_d);
    hipFree(phiI_d);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  kVals = (struct kValues*)calloc(numK, sizeof (struct kValues));
  for (int k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].PhiMag = phiMag[k];
  }

  free(phiMag);

  /* GPU section 2 */
  {
    float *x_d, *y_d, *z_d;
    float *Qr_d, *Qi_d;

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    setupMemoryGPU(numX, sizeof(float), x_d, x);
    setupMemoryGPU(numX, sizeof(float), y_d, y);
    setupMemoryGPU(numX, sizeof(float), z_d, z);
    hipMalloc((void **)&Qr_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset((void *)Qr_d, 0, numX * sizeof(float));
    hipMalloc((void **)&Qi_d, numX * sizeof(float));
    CUDA_ERRCK;
    hipMemset((void *)Qi_d, 0, numX * sizeof(float));

    hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    computeQ_GPU(numK, numX, x_d, y_d, z_d, kVals, Qr_d, Qi_d);

    hipDeviceSynchronize();
    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    cleanupMemoryGPU(numX, sizeof(float), Qr_d, Qr);
    cleanupMemoryGPU(numX, sizeof(float), Qi_d, Qi);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  if (params->outFile)
    {
      /* Write Q to file */
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      outputData(params->outFile, Qr, Qi, numX);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }

  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (kVals);
  free (Qr);
  free (Qi);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);

  pb_FreeParameters(params);

  return 0;
}
