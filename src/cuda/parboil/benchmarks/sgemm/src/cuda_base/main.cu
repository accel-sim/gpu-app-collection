/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * Main entry of dense matrix-matrix multiplication kernel
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <parboil.h>
#include <iostream>
#include "sgemm_kernel.cu"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float>&v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float>&);

extern "C"
void computeGold(float *, const float*, const float*, unsigned int, unsigned int, unsigned int);

int
main (int argc, char *argv[]) {

  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  float *dA, *dB, *dC;
  size_t A_sz, B_sz, C_sz;
  int matArow, matAcol;
  int matBrow, matBcol;
  std::vector<float> matA, matBT;

  pb_InitializeTimerSet(&timers);

  /* Read command line. Expect 3 inputs: A, B and B^T 
     in column-major layout*/
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) 
      || (params->inpFiles[1] == NULL)
      || (params->inpFiles[2] == NULL)
      || (params->inpFiles[3] != NULL))
    {
      fprintf(stderr, "Expecting three input filenames\n");
      exit(-1);
    }
 
  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);

  // load A
  readColMajorMatrixFile(params->inpFiles[0],
      matArow, matAcol, matA);
  // copy A to device memory
  A_sz = matArow*matAcol*sizeof(float);

  // load B^T
  readColMajorMatrixFile(params->inpFiles[2],
      matBcol, matBrow, matBT);

  pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  B_sz = matBrow*matBcol*sizeof(float);

  // allocate space for C
  C_sz = matArow*matBcol*sizeof(float);

  // CUDA memory allocation
  std::vector<float> matC(matArow*matBcol);
  hipMalloc((void**)&dA, A_sz);
  hipMalloc((void**)&dB, B_sz);
  hipMalloc((void**)&dC, C_sz);

  // Copy A and B^T into device memory
  pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  hipMemcpy(dA, &matA.front(), A_sz, hipMemcpyHostToDevice); 
  hipMemcpy(dB, &matBT.front(), B_sz, hipMemcpyHostToDevice); 

  hipDeviceSynchronize();

  pb_SwitchToTimer( &timers, pb_TimerID_KERNEL );

  // Use standard sgemm interface
  basicSgemm('N', 'T', matArow, matBcol, matAcol, 1.0f, \
      dA, matArow, dB, matBcol, 0.0f, dC, matArow);

  hipDeviceSynchronize();

  if (params->outFile) {
    pb_SwitchToTimer( &timers, pb_TimerID_COPY );
    hipMemcpy(&matC.front(), dC, C_sz, hipMemcpyDeviceToHost);
    /* Write C to file */
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
    writeColMajorMatrixFile(params->outFile,
	matArow, matBcol, matC); 
  }

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  double GPUtime = pb_GetElapsedTime(&(timers.timers[pb_TimerID_KERNEL]));
  std::cout<< "GFLOPs = " << 2.* matArow * matBcol * matAcol/GPUtime/1e9 << std::endl;
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  return 0;
}
