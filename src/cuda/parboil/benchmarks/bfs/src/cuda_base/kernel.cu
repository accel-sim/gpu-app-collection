#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in DAC'10
  paper "An Effective GPU Implementation of Breadth-First Search"

  Copyright (c) 2010 University of Illinois at Urbana-Champaign. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Author: Lijiuan Luo (lluo3@uiuc.edu)
 ************************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_
/**********
Define colors for BFS
1) the definition of White, gray and black comes from the text book "Introduction to Algorithms"
2) For path search problems, people may choose to use different colors to record the found paths.
Therefore we reserve numbers (0-16677216) for this purpose. Only nodes with colors bigger than
UP_LIMIT are free to visit 
3) We define two gray shades to differentiate between the new frontier nodes and the old frontier nodes that
 have not been marked BLACK 
*************/

#define UP_LIMIT 16677216//2^24
#define WHITE 16677217
#define GRAY 16677218
#define GRAY0 16677219
#define GRAY1 16677220
#define BLACK 16677221

#include "config.h"
texture<Node> g_graph_node_ref;
texture<Edge> g_graph_edge_ref;

volatile __device__ int count = 0;
volatile __device__ int no_of_nodes_vol = 0;
volatile __device__ int stay_vol = 0;

/*****************************************************************************
This is the  most general version of BFS kernel, i.e. no assumption about #block in the grid  
\param q1: the array to hold the current frontier
\param q2: the array to hold the new frontier
\param g_graph_nodes: the nodes in the input graph
\param g_graph_edges: the edges i nthe input graph
\param g_color: the colors of nodes
\param g_cost: the costs of nodes
\param no_of_nodes: the number of nodes in the current frontier
\param tail: pointer to the location of the tail of the new frontier. *tail is the size of the new frontier 
\param gray_shade: the shade of the gray in current BFS propagation. See GRAY0, GRAY1 macro definitions for more details
\param k: the level of current propagation in the BFS tree. k= 0 for the first propagation.
***********************************************************************/
__global__ void
BFS_kernel(int * q1, 
           int * q2, 
           Node* g_graph_nodes, 
           Edge* g_graph_edges, 
           int* g_color, 
           int * g_cost, 
           int no_of_nodes, 
           int * tail, 
           int gray_shade, 
           int k) 
{
  __shared__ int local_q_tail;//the tails of each local warp-level queue
  __shared__ int local_q[NUM_BIN*W_QUEUE_SIZE];//the local warp-level queues 
  //current w-queue, a.k.a prefix sum
  __shared__ int shift;

  if(threadIdx.x == 0){
    local_q_tail = 0;//initialize the tail of w-queue
  }
  __syncthreads();

  //first, propagate and add the new frontier elements into w-queues
  int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
  if( tid<no_of_nodes)
  {
    int pid = q1[tid]; //the current frontier node, or the parent node of the new frontier nodes 
    g_color[pid] = BLACK;
    int cur_cost = g_cost[pid];
    //into
    Node cur_node = tex1Dfetch(g_graph_node_ref,pid);
    for(int i=cur_node.x; i<cur_node.y + cur_node.x; i++)//visit each neighbor of the
      //current frontier node.
    {
      Edge cur_edge = tex1Dfetch(g_graph_edge_ref,i);
      int id = cur_edge.x;
      int cost = cur_edge.y;
      cost += cur_cost;
      int orig_cost = atomicMin(&g_cost[id],cost);
      if(orig_cost > cost){//the node should be visited
        if(g_color[id] > UP_LIMIT){
          int old_color = atomicExch(&g_color[id],gray_shade);
          //this guarantees that only one thread will push this node
          //into a queue
          if(old_color != gray_shade) {

            //atomic operation guarantees the correctness
            //even if multiple warps are executing simultaneously
            int index = atomicAdd(&local_q_tail,1);
            local_q[index] = id;
          }
        }
      }
    }
  }
  __syncthreads();

  if(threadIdx.x == 0){
    int tot_sum = local_q_tail; 

    //the offset or "shift" of the block-level queue within the grid-level queue
    //is determined by atomic operation
    shift = atomicAdd(tail,tot_sum);
  }
  __syncthreads();


  int local_shift = threadIdx.x;//shift within a w-queue

  //loop unrolling was originally used for better performance, but removed for better readability
  while(local_shift < local_q_tail){
    q2[shift + local_shift] = local_q[local_shift];
    local_shift += blockDim.x;//multiple threads are copying elements at the same time,
    //so we shift by multiple elements for next iteration  
  }
}
#endif 
