#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2008-2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "atom.h"
#include "cutoff.h"
#include "parboil.h"

#ifdef __DEVICE_EMULATION__
#define DEBUG
/* define which grid block and which thread to examine */
#define BX  0
#define BY  0
#define TX  0
#define TY  0
#define TZ  0
#define EMU(code) do { \
  if (blockIdx.x==BX && blockIdx.y==BY && \
      threadIdx.x==TX && threadIdx.y==TY && threadIdx.z==TZ) { \
    code; \
  } \
} while (0)
#define INT(n)    printf("%s = %d\n", #n, n)
#define FLOAT(f)  printf("%s = %g\n", #f, (double)(f))
#define INT3(n)   printf("%s = %d %d %d\n", #n, (n).x, (n).y, (n).z)
#define FLOAT4(f) printf("%s = %g %g %g %g\n", #f, (double)(f).x, \
    (double)(f).y, (double)(f).z, (double)(f).w)
#else
#define EMU(code)
#define INT(n)
#define FLOAT(f)
#define INT3(n)
#define FLOAT4(f)
#endif

/* report error from CUDA */
#define CUERR \
  do { \
    hipError_t err; \
    if ((err = hipGetLastError()) != hipSuccess) { \
      printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
      return -1; \
    } \
  } while (0)

/*
 * neighbor list:
 * stored in constant memory as table of offsets
 * flat index addressing is computed by kernel
 *
 * reserve enough memory for 11^3 stencil of grid cells
 * this fits within 16K of memory
 */
#define NBRLIST_DIM  11
#define NBRLIST_MAXLEN (NBRLIST_DIM * NBRLIST_DIM * NBRLIST_DIM)
__constant__ int NbrListLen;
__constant__ int3 NbrList[NBRLIST_MAXLEN];


/* Normally, we're summing electrostatic potential.  However, for
 * profiling we may want to appropriate this storage to count the
 * number of nearby atoms, instead.
 */
#undef NEIGHBOR_COUNT
 
#ifndef NEIGHBOR_COUNT
typedef float ener_t;
#else
typedef int ener_t;
#endif

/*
 * atom bins cached into shared memory for processing
 *
 * this reserves 4K of shared memory for 32 atom bins each containing 8 atoms,
 * should permit scheduling of up to 3 thread blocks per SM
 */
#define BIN_DEPTH         8  /* max number of atoms per bin */
#define BIN_SIZE         32  /* size of bin in floats */
#define BIN_SHIFT         5  /* # of bits to shift for mul/div by BIN_SIZE */
#define BIN_CACHE_MAXLEN 32  /* max number of atom bins to cache */

#define BIN_LENGTH      4.f  /* spatial length in Angstroms */
#define BIN_INVLEN  (1.f / BIN_LENGTH)
/* assuming density of 1 atom / 10 A^3, expectation is 6.4 atoms per bin
 * so that bin fill should be 80% (for non-empty regions of space) */

#define REGION_SIZE     512  /* number of floats in lattice region */

/*
 * potential lattice is decomposed into size 8^3 lattice point "regions"
 *
 * THIS IMPLEMENTATION:  one thread per lattice point
 * thread block size 128 gives 4 thread blocks per region
 * kernel is invoked for each x-y plane of regions,
 * where gridDim.x is 4*(x region dimension) so that blockIdx.x 
 * can absorb the z sub-region index in its 2 lowest order bits
 *
 * Regions are stored contiguously in memory in row-major order
 *
 * The bins have to not only cover the region, but they need to surround
 * the outer edges so that region sides and corners can still use
 * neighbor list stencil.  The binZeroAddr is actually a shifted pointer into
 * the bin array (binZeroAddr = binBaseAddr + (c*binDim_y + c)*binDim_x + c)
 * where c = ceil(cutoff / binsize).  This allows for negative offsets to
 * be added to myBinIndex.
 *
 * The (0,0,0) spatial origin corresponds to lower left corner of both
 * regionZeroAddr and binZeroAddr.  The atom coordinates are translated
 * during binning to enforce this assumption.
 */
__global__ static void cuda_cutoff_potential_lattice6overlap(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    ener_t *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex
    )
{
  __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
  __shared__ ener_t *myRegionAddr;
  __shared__ int3 myBinIndex;

  const int xRegionIndex = blockIdx.x;
  const int yRegionIndex = blockIdx.y;

  /* thread id */
  const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x
    + threadIdx.x;
  /* blockDim.x == 8, blockDim.y == 2, blockDim.z == 8 */

  /* neighbor index */
  int nbrid;

  /* this is the start of the sub-region indexed by tid */
  myRegionAddr = regionZeroAddr + ((zRegionIndex*gridDim.y
        + yRegionIndex)*gridDim.x + xRegionIndex)*REGION_SIZE;
    
  /* spatial coordinate of this lattice point */
  float x = (8 * xRegionIndex + threadIdx.x) * h;
  float y = (8 * yRegionIndex + threadIdx.y) * h;
  float z = (8 * zRegionIndex + threadIdx.z) * h;

  int totalbins = 0;
  int numbins;

  /* bin number determined by center of region */
  myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
  myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
  myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

  /* first neighbor in list for me to cache */
  nbrid = (tid >> 4);

  numbins = BIN_CACHE_MAXLEN;

#ifndef NEIGHBOR_COUNT
  ener_t energy0 = 0.f;
  ener_t energy1 = 0.f;
  ener_t energy2 = 0.f;
  ener_t energy3 = 0.f;
#else
  ener_t energy0 = 0, energy1 = 0, energy2 = 0, energy3 = 0;
#endif

  for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
    int bincnt;

    /* start of where to write in shared memory */
    int startoff = BIN_SIZE * (tid >> 4);

    /* each half-warp to cache up to 4 atom bins */
    for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
      int i = myBinIndex.x + NbrList[nbrid].x;
      int j = myBinIndex.y + NbrList[nbrid].y;
      int k = myBinIndex.z + NbrList[nbrid].z;

      /* determine global memory location of atom bin */
      float *p_global = ((float *) binZeroAddr)
        + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

      /* coalesced read from global memory -
       * retain same ordering in shared memory for now */
      int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
      int tidmask = tid & 15;

      AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
      AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
    }
    __syncthreads();

    /* no warp divergence */
    if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
      numbins = NbrListLen - totalbins;
    }

    int stopbin = (numbins << BIN_SHIFT);
    for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
      int i;

      for (i = 0;  i < BIN_DEPTH;  i++) {
        int off = bincnt + (i<<2);

        float aq = AtomBinCache[off + 3];
        if (0.f == aq) 
          break;  /* no more atoms in bin */

        float dx = AtomBinCache[off    ] - x;
        float dz = AtomBinCache[off + 2] - z;
        float dxdz2 = dx*dx + dz*dz;
        float dy = AtomBinCache[off + 1] - y;
        float r2 = dy*dy + dxdz2;

#ifndef NEIGHBOR_COUNT
        if (r2 < cutoff2)
	  {
          float s = (1.f - r2 * inv_cutoff2);
          energy0 += aq * rsqrtf(r2) * s * s;
        }
#else
	energy0 += (r2 < cutoff2);
#endif
        dy -= 2.0f*h;
        r2 = dy*dy + dxdz2;
#ifndef NEIGHBOR_COUNT
	if (r2 < cutoff2)
	  {
          float s = (1.f - r2 * inv_cutoff2);
          energy1 += aq * rsqrtf(r2) * s * s;
        }
#else
	energy1 += (r2 < cutoff2);
#endif
        dy -= 2.0f*h;
        r2 = dy*dy + dxdz2;
#ifndef NEIGHBOR_COUNT
        if (r2 < cutoff2)
	  {
          float s = (1.f - r2 * inv_cutoff2);
          energy2 += aq * rsqrtf(r2) * s * s;
        }
#else
	energy2 += (r2 < cutoff2);
#endif
        dy -= 2.0f*h;
        r2 = dy*dy + dxdz2;
#ifndef NEIGHBOR_COUNT
        if (r2 < cutoff2)
	  {
          float s = (1.f - r2 * inv_cutoff2);
          energy3 += aq * rsqrtf(r2) * s * s;
        }
#else
	energy3 += (r2 < cutoff2);
#endif
      } /* end loop over atoms in bin */
    } /* end loop over cached atom bins */
    __syncthreads();

  } /* end loop over neighbor list */

  /* store into global memory */
  myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
  myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
  myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
  myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
}




extern "C" int gpu_compute_cutoff_potential_lattice6overlap(
    struct pb_TimerSet *timers,        /* for measuring execution time */
    Lattice *lattice,
    float cutoff,                      /* cutoff distance */
    Atoms *atoms,                      /* array of atoms */
    int verbose                        /* print info/debug messages */
    )
{
  int nx = lattice->dim.nx;
  int ny = lattice->dim.ny;
  int nz = lattice->dim.nz;
  float xlo = lattice->dim.lo.x;
  float ylo = lattice->dim.lo.y;
  float zlo = lattice->dim.lo.z;
  float h = lattice->dim.h;
  int natoms = atoms->size;
  Atom *atom = atoms->atoms;

  int3 nbrlist[NBRLIST_MAXLEN];
  int nbrlistlen = 0;

  int binHistoFull[BIN_DEPTH+1] = { 0 };   /* clear every array element */
  int binHistoCover[BIN_DEPTH+1] = { 0 };  /* clear every array element */
  int num_excluded = 0;

  int xRegionDim, yRegionDim, zRegionDim;
  int xRegionIndex, yRegionIndex, zRegionIndex;
  int xOffset, yOffset, zOffset;
  int lnx, lny, lnz, lnall;
  ener_t *regionZeroAddr, *thisRegion;
  ener_t *regionZeroCuda;
  int index, indexRegion;

  int c;
  int3 binDim;
  int nbins;
  float4 *binBaseAddr, *binZeroAddr;
  float4 *binBaseCuda, *binZeroCuda;
  int *bincntBaseAddr, *bincntZeroAddr;
  Atoms *extra = NULL;

  int i, j, k, n;
  int sum, total;

  float avgFillFull, avgFillCover;
  const float cutoff2 = cutoff * cutoff;
  const float inv_cutoff2 = 1.f / cutoff2;

  dim3 gridDim, blockDim;

#ifdef NEIGHBOR_COUNT
  double neighbor_count = 0;	/* used to profile the number of atoms near a
				 * lattice point */
#endif

  // Caller has made the 'compute' timer active

  /* pad lattice to be factor of 8 in each dimension */
  xRegionDim = (int) ceilf(nx/8.f);
  yRegionDim = (int) ceilf(ny/8.f);
  zRegionDim = (int) ceilf(nz/8.f);

  lnx = 8 * xRegionDim;
  lny = 8 * yRegionDim;
  lnz = 8 * zRegionDim;
  lnall = lnx * lny * lnz;

  /* will receive energies from CUDA */
  regionZeroAddr = (ener_t *) malloc(lnall * sizeof(float));

  /* create bins */
  c = (int) ceil(cutoff * BIN_INVLEN);  /* count extra bins around lattice */
  binDim.x = (int) ceil(lnx * h * BIN_INVLEN) + 2*c;
  binDim.y = (int) ceil(lny * h * BIN_INVLEN) + 2*c;
  binDim.z = (int) ceil(lnz * h * BIN_INVLEN) + 2*c;
  nbins = binDim.x * binDim.y * binDim.z;
  binBaseAddr = (float4 *) calloc(nbins * BIN_DEPTH, sizeof(float4));
  binZeroAddr = binBaseAddr + ((c * binDim.y + c) * binDim.x + c) * BIN_DEPTH;

  bincntBaseAddr = (int *) calloc(nbins, sizeof(int));
  bincntZeroAddr = bincntBaseAddr + (c * binDim.y + c) * binDim.x + c;

  /* create neighbor list */
  if (ceilf(BIN_LENGTH / (8*h)) == floorf(BIN_LENGTH / (8*h))) {
    float s = sqrtf(3);
    float r2 = (cutoff + s*BIN_LENGTH) * (cutoff + s*BIN_LENGTH);
    int cnt = 0;
    /* develop neighbor list around 1 cell */
    if (2*c + 1 > NBRLIST_DIM) {
      fprintf(stderr, "must have cutoff <= %f\n",
          (NBRLIST_DIM-1)/2 * BIN_LENGTH);
      return -1;
    }
    for (k = -c;  k <= c;  k++) {
      for (j = -c;  j <= c;  j++) {
        for (i = -c;  i <= c;  i++) {
          if ((i*i + j*j + k*k)*BIN_LENGTH*BIN_LENGTH >= r2) continue;
          nbrlist[cnt].x = i;
          nbrlist[cnt].y = j;
          nbrlist[cnt].z = k;
          cnt++;
        }
      }
    }
    nbrlistlen = cnt;
  }
  else if (8*h <= 2*BIN_LENGTH) {
    float s = 2.f*sqrtf(3);
    float r2 = (cutoff + s*BIN_LENGTH) * (cutoff + s*BIN_LENGTH);
    int cnt = 0;
    /* develop neighbor list around 3-cube of cells */
    if (2*c + 3 > NBRLIST_DIM) {
      fprintf(stderr, "must have cutoff <= %f\n",
          (NBRLIST_DIM-3)/2 * BIN_LENGTH);
      return -1;
    }
    for (k = -c;  k <= c;  k++) {
      for (j = -c;  j <= c;  j++) {
        for (i = -c;  i <= c;  i++) {
          if ((i*i + j*j + k*k)*BIN_LENGTH*BIN_LENGTH >= r2) continue;
          nbrlist[cnt].x = i;
          nbrlist[cnt].y = j;
          nbrlist[cnt].z = k;
          cnt++;
        }
      }
    }
    nbrlistlen = cnt;
  }
  else {
    fprintf(stderr, "must have h <= %f\n", 0.25 * BIN_LENGTH);
    return -1;
  }

  /* perform geometric hashing of atoms into bins */
  {
    /* array of extra atoms, permit average of one extra per bin */
    Atom *extra_atoms = (Atom *) calloc(nbins, sizeof(Atom));
    int extra_len = 0;
    
    for (n = 0;  n < natoms;  n++) {
      float4 p;
      p.x = atom[n].x - xlo;
      p.y = atom[n].y - ylo;
      p.z = atom[n].z - zlo;
      p.w = atom[n].q;
      i = (int) floorf(p.x * BIN_INVLEN);
      j = (int) floorf(p.y * BIN_INVLEN);
      k = (int) floorf(p.z * BIN_INVLEN);
      if (i >= -c && i < binDim.x - c &&
	  j >= -c && j < binDim.y - c &&
	  k >= -c && k < binDim.z - c &&
	  atom[n].q != 0) {
	int index = (k * binDim.y + j) * binDim.x + i;
	float4 *bin = binZeroAddr + index * BIN_DEPTH;
	int bindex = bincntZeroAddr[index];
	if (bindex < BIN_DEPTH) {
	  /* copy atom into bin and increase counter for this bin */
	  bin[bindex] = p;
	  bincntZeroAddr[index]++;
	}
	else {
	  /* add index to array of extra atoms to be computed with CPU */
	  if (extra_len >= nbins) {
	    fprintf(stderr, "exceeded space for storing extra atoms\n");
	    return -1;
	  }
	  extra_atoms[extra_len] = atom[n];
	  extra_len++;
	}
      }
      else {
	/* excluded atoms are either outside bins or neutrally charged */
	num_excluded++;
      }
    }

    /* Save result */
    extra = (Atoms *)malloc(sizeof(Atoms));
    extra->atoms = extra_atoms;
    extra->size = extra_len;
  }

  /* bin stats */
  sum = total = 0;
  for (n = 0;  n < nbins;  n++) {
    binHistoFull[ bincntBaseAddr[n] ]++;
    sum += bincntBaseAddr[n];
    total += BIN_DEPTH;
  }
  avgFillFull = sum / (float) total;
  sum = total = 0;
  for (k = 0;  k < binDim.z - 2*c;  k++) {
    for (j = 0;  j < binDim.y - 2*c;  j++) {
      for (i = 0;  i < binDim.x - 2*c;  i++) {
        int index = (k * binDim.y + j) * binDim.x + i;
        binHistoCover[ bincntZeroAddr[index] ]++;
        sum += bincntZeroAddr[index];
        total += BIN_DEPTH;
      }
    }
  }
  avgFillCover = sum / (float) total;

  if (verbose) {
    /* report */
    printf("number of atoms = %d\n", natoms);
    printf("lattice spacing = %g\n", h);
    printf("cutoff distance = %g\n", cutoff);
    printf("\n");
    printf("requested lattice dimensions = %d %d %d\n", nx, ny, nz);
    printf("requested space dimensions = %g %g %g\n", nx*h, ny*h, nz*h);
    printf("expanded lattice dimensions = %d %d %d\n", lnx, lny, lnz);
    printf("expanded space dimensions = %g %g %g\n", lnx*h, lny*h, lnz*h);
    printf("number of bytes for lattice data = %u\n", lnall*sizeof(float));
    printf("\n");
    printf("bin padding thickness = %d\n", c);
    printf("bin cover dimensions = %d %d %d\n",
        binDim.x - 2*c, binDim.y - 2*c, binDim.z - 2*c);
    printf("bin full dimensions = %d %d %d\n", binDim.x, binDim.y, binDim.z);
    printf("number of bins = %d\n", nbins);
    printf("total number of atom slots = %d\n", nbins * BIN_DEPTH);
    printf("%% overhead space = %g\n",
        (natoms / (double) (nbins * BIN_DEPTH)) * 100);
    printf("number of bytes for bin data = %u\n",
        nbins * BIN_DEPTH * sizeof(float4));
    printf("\n");
    printf("bin histogram with padding:\n");
    sum = 0;
    for (n = 0;  n <= BIN_DEPTH;  n++) {
      printf("     number of bins with %d atoms:  %d\n", n, binHistoFull[n]);
      sum += binHistoFull[n];
    }
    printf("     total number of bins:  %d\n", sum);
    printf("     %% average fill:  %g\n", avgFillFull * 100);
    printf("\n");
    printf("bin histogram excluding padding:\n");
    sum = 0;
    for (n = 0;  n <= BIN_DEPTH;  n++) {
      printf("     number of bins with %d atoms:  %d\n", n, binHistoCover[n]);
      sum += binHistoCover[n];
    }
    printf("     total number of bins:  %d\n", sum);
    printf("     %% average fill:  %g\n", avgFillCover * 100);
    printf("\n");
    printf("number of extra atoms = %d\n", extra->size);
    printf("%% atoms that are extra = %g\n", (extra->size / (double) natoms) * 100);
    printf("\n");

    /* sanity check on bins */
    sum = 0;
    for (n = 0;  n <= BIN_DEPTH;  n++) {
      sum += n * binHistoFull[n];
    }
    sum += extra->size + num_excluded;
    printf("sanity check on bin histogram with edges:  "
        "sum + others = %d\n", sum);
    sum = 0;
    for (n = 0;  n <= BIN_DEPTH;  n++) {
      sum += n * binHistoCover[n];
    }
    sum += extra->size + num_excluded;
    printf("sanity check on bin histogram excluding edges:  "
        "sum + others = %d\n", sum);
    printf("\n");

    /* neighbor list */
    printf("neighbor list length = %d\n", nbrlistlen);
    printf("\n");
  }

  /* setup CUDA kernel parameters */
  gridDim.x = xRegionDim;
  gridDim.y = yRegionDim;
  gridDim.z = 1;
  blockDim.x = 8;
  blockDim.y = 2;
  blockDim.z = 8;

  /* allocate and initialize memory on CUDA device */
  pb_SwitchToTimer(timers, pb_TimerID_COPY);
  if (verbose) {
    printf("Allocating %.2fMB on CUDA device for potentials\n",
           lnall * sizeof(float) / (double) (1024*1024));
  }
  hipMalloc((void **) &regionZeroCuda, lnall * sizeof(ener_t));
  CUERR;
  hipMemset(regionZeroCuda, 0, lnall * sizeof(ener_t));
  CUERR;
  if (verbose) {
    printf("Allocating %.2fMB on CUDA device for atom bins\n",
           nbins * BIN_DEPTH * sizeof(float4) / (double) (1024*1024));
  }
  hipMalloc((void **) &binBaseCuda, nbins * BIN_DEPTH * sizeof(float4));
  CUERR;
  hipMemcpy(binBaseCuda, binBaseAddr, nbins * BIN_DEPTH * sizeof(float4),
      hipMemcpyHostToDevice);
  CUERR;
  binZeroCuda = binBaseCuda + ((c * binDim.y + c) * binDim.x + c) * BIN_DEPTH;
  hipMemcpyToSymbol(HIP_SYMBOL(NbrListLen), &nbrlistlen, sizeof(int), 0);
  CUERR;
  hipMemcpyToSymbol(HIP_SYMBOL(NbrList), nbrlist, nbrlistlen * sizeof(int3), 0);
  CUERR;

  if (verbose) 
    printf("\n");


  hipStream_t cutoffstream;
  hipStreamCreate(&cutoffstream);

  /* loop over z-dimension, invoke CUDA kernel for each x-y plane */
  pb_SwitchToTimer(timers, pb_TimerID_KERNEL);
  printf("Invoking CUDA kernel on %d region planes...\n", zRegionDim);
  for (zRegionIndex = 0;  zRegionIndex < zRegionDim;  zRegionIndex++) {
    printf("  computing plane %d\r", zRegionIndex);
    fflush(stdout);
    cuda_cutoff_potential_lattice6overlap<<<gridDim, blockDim, 0>>>(binDim.x, binDim.y,
        binZeroCuda, h, cutoff2, inv_cutoff2, regionZeroCuda, zRegionIndex);
  }


  /* 
   * handle extra atoms on the CPU, concurrently with the GPU calculations
   */
  pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
  if (extra->size > 0) {
    printf("computing extra atoms on CPU\n");
    if (cpu_compute_cutoff_potential_lattice(lattice, cutoff, extra)) {
      fprintf(stderr, "cpu_compute_cutoff_potential_lattice() failed "
          "for extra atoms\n");
      return -1;
    }
    printf("\n");
  }

  hipStreamSynchronize(cutoffstream);
  CUERR;
  hipDeviceSynchronize();
  hipStreamDestroy(cutoffstream);
  printf("Finished CUDA kernel calls                        \n");

  /* copy result regions from CUDA device */
  pb_SwitchToTimer(timers, pb_TimerID_COPY);
  hipMemcpy(regionZeroAddr, regionZeroCuda, lnall * sizeof(ener_t),
      hipMemcpyDeviceToHost);
  CUERR;

  /* free CUDA memory allocations */
  hipFree(regionZeroCuda);
  hipFree(binBaseCuda);

  /*
   * transpose on CPU, updating, producing the final lattice
   */
  /* transpose regions back into lattice */
  pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);
  for (k = 0;  k < nz;  k++) {
    zRegionIndex = (k >> 3);
    zOffset = (k & 7);

    for (j = 0;  j < ny;  j++) {
      yRegionIndex = (j >> 3);
      yOffset = (j & 7);

      for (i = 0;  i < nx;  i++) {
        xRegionIndex = (i >> 3);
        xOffset = (i & 7);

        thisRegion = regionZeroAddr
          + ((zRegionIndex * yRegionDim + yRegionIndex) * xRegionDim
              + xRegionIndex) * REGION_SIZE;

        indexRegion = (zOffset * 8 + yOffset) * 8 + xOffset;
        index = (k * ny + j) * nx + i;

#ifndef NEIGHBOR_COUNT
        lattice->lattice[index] += thisRegion[indexRegion];
#else
	neighbor_count += thisRegion[indexRegion];
#endif
      }
    }
  }

#ifdef NEIGHBOR_COUNT
  printf("Neighbor count: %f\n", (float)neighbor_count);
#endif


  /* cleanup memory allocations */
  free(regionZeroAddr);
  free(binBaseAddr);
  free(bincntBaseAddr);
  free_atom(extra);

  return 0;
}
