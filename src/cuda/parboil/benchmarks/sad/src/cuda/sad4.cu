#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "sad.h"
#include "sad4.h"

/* Macros to access temporary frame storage in shared memory */
#define FRAME_GET(n, x, y) \
  (frame_loc[((n) << 4) + ((y) << 2) + (x)])
#define FRAME_PUT_1(n, x, value) \
  (frame_loc[((n) << 4) + (x)] = value)

/* Macros to access temporary SAD storage in shared memory */
#define SAD_LOC_GET(blocknum, pos) \
  (sad_loc[(blocknum) * MAX_POS_PADDED + (pos)])
#define SAD_LOC_PUT(blocknum, pos, value) \
  (sad_loc[(blocknum) * MAX_POS_PADDED + (pos)] = (value))

/* When reading from this array, we use an "index" rather than a
   search position.  Also, the number of array elements is divided by
   four relative to SAD_LOC_GET() since this is an array of 8byte
   data, while SAD_LOC_GET() sees an array of 2byte data. */
#define SAD_LOC_8B_GET(blocknum, ix) \
  (sad_loc_8b[(blocknum) * (MAX_POS_PADDED/4) + (ix)])

/* The size of one row of sad_loc_8b.  This is the group of elements
 * holding SADs for all search positions for one 4x4 block. */
#define SAD_LOC_8B_ROW_SIZE (MAX_POS_PADDED/4)

/* The presence of this preprocessor variable controls which
 * of two means of computing the current search position is used. */
#define SEARCHPOS_RECURRENCE

/* A local copy of the current 4x4 block */
__shared__ unsigned short frame_loc[THREADS_W * THREADS_H * 16];

/* The part of the reference image that is in the search range */
texture<unsigned short, 2, hipReadModeElementType> ref;

/* The local SAD array on the device.  This is an array of short ints.  It is
 * interpreted as an array of 8-byte data for global data transfers. */
extern __shared__ unsigned short sad_loc[];
extern __shared__ vec8b sad_loc_8b[];

/* The compute kernel. */
/* The macros THREADS_W and THREADS_H specify the width and height of the
 * area to be processed by one thread, measured in 4-by-4 pixel blocks.
 * Larger numbers mean more computation per thread block.
 *
 * The macro POS_PER_THREAD specifies the number of search positions for which
 * an SAD is computed.  A larger value indicates more computation per thread,
 * and fewer threads per thread block.  It must be a multiple of 3 and also
 * must be at most 33 because the loop to copy from shared memory uses
 * 32 threads per 4-by-4 pixel block.
 *
 */
__global__ void mb_sad_calc(unsigned short *blk_sad,
			    unsigned short *frame,
			    int mb_width,
			    int mb_height)
{
  int txy_tmp = threadIdx.x / CEIL(MAX_POS, POS_PER_THREAD);
  int ty = txy_tmp / THREADS_W;
  int tx = txy_tmp - __umul24(ty, THREADS_W);
  int bx = blockIdx.x;
  int by = blockIdx.y;

  /* Macroblock and sub-block coordinates */
  int mb_x = (tx + __umul24(bx, THREADS_W)) >> 2;
  int mb_y = (ty + __umul24(by, THREADS_H)) >> 2;
  int block_x = (tx + __umul24(bx, THREADS_W)) & 0x03;
  int block_y = (ty + __umul24(by, THREADS_H)) & 0x03;

  /* Block-copy data into shared memory.
   * Threads are grouped into sets of 16, leaving some threads idle. */
  if ((threadIdx.x >> 4) < (THREADS_W * THREADS_H))
  {
    int ty = (threadIdx.x >> 4) / THREADS_W;
    int tx = (threadIdx.x >> 4) - __umul24(ty, THREADS_W);
    int tgroup = threadIdx.x & 15;

    /* Width of the image in pixels */
    int img_width = mb_width*16;

    /* Pixel offset of the origin of the current 4x4 block */
    int frame_x = (tx + __umul24(bx, THREADS_W)) << 2;
    int frame_y = (ty + __umul24(by, THREADS_H)) << 2;

    /* Origin in the current frame for this 4x4 block */
    int cur_o = frame_y * img_width + frame_x;

    /* If this is an invalid 4x4 block, do nothing */
    if (((frame_x >> 4) < mb_width) && ((frame_y >> 4) < mb_height))
      {
	/* Copy one pixel into 'frame' */
	FRAME_PUT_1(__umul24(ty, THREADS_W) + tx, tgroup,
		    frame[cur_o + (tgroup >> 2) * img_width + (tgroup & 3)]);
      }
  }

  __syncthreads();

  /* If this thread is assigned to an invalid 4x4 block, do nothing */
  if ((mb_x < mb_width) && (mb_y < mb_height))
    {
      /* Pixel offset of the origin of the current 4x4 block */
      int frame_x = ((mb_x << 2) + block_x) << 2;
      int frame_y = ((mb_y << 2) + block_y) << 2;

      /* Origin of the search area for this 4x4 block */
      int ref_x = frame_x - SEARCH_RANGE;
      int ref_y = frame_y - SEARCH_RANGE;

      /* Origin in the current frame for this 4x4 block */
      int cur_o = ty * THREADS_W + tx;

      int search_pos;
      int search_pos_base =
	(threadIdx.x % CEIL(MAX_POS, POS_PER_THREAD)) * POS_PER_THREAD;
      int search_pos_end = search_pos_base + POS_PER_THREAD;

      int sotmp = search_pos_base / SEARCH_DIMENSION;
      int local_search_off_x = search_pos_base - TIMES_DIM_POS(sotmp);
      int search_off_y = ref_y + sotmp;

      /* Don't go past bounds */
      if (search_pos_end > MAX_POS)
	search_pos_end = MAX_POS;

      /* For each search position, within the range allocated to this thread */
      for (search_pos = search_pos_base;
	   search_pos < search_pos_end;
	   search_pos += 3) {
	/* It is also beneficial to fuse (jam) the enclosed loops if this loop
	 * is unrolled. */
	unsigned short sad1 = 0, sad2 = 0, sad3 = 0;
	int search_off_x = ref_x + local_search_off_x;

	/* 4x4 SAD computation */
	for(int y=0; y<4; y++) {
	  int t;
	  t = tex2D(ref, search_off_x, search_off_y + y);
	  sad1 += abs(t - FRAME_GET(cur_o, 0, y));

	  t = tex2D(ref, search_off_x + 1, search_off_y + y);
	  sad1 += abs(t - FRAME_GET(cur_o, 1, y));
	  sad2 += abs(t - FRAME_GET(cur_o, 0, y));

	  t = tex2D(ref, search_off_x + 2, search_off_y + y);
	  sad1 += abs(t - FRAME_GET(cur_o, 2, y));
	  sad2 += abs(t - FRAME_GET(cur_o, 1, y));
	  sad3 += abs(t - FRAME_GET(cur_o, 0, y));

	  t = tex2D(ref, search_off_x + 3, search_off_y + y);
	  sad1 += abs(t - FRAME_GET(cur_o, 3, y));
	  sad2 += abs(t - FRAME_GET(cur_o, 2, y));
	  sad3 += abs(t - FRAME_GET(cur_o, 1, y));

	  t = tex2D(ref, search_off_x + 4, search_off_y + y);
	  sad2 += abs(t - FRAME_GET(cur_o, 3, y));
	  sad3 += abs(t - FRAME_GET(cur_o, 2, y));

	  t = tex2D(ref, search_off_x + 5, search_off_y + y);
	  sad3 += abs(t - FRAME_GET(cur_o, 3, y));
	}

	/* Save this value into the local SAD array */
	SAD_LOC_PUT(__umul24(ty, THREADS_W) + tx, search_pos, sad1);
	SAD_LOC_PUT(__umul24(ty, THREADS_W) + tx, search_pos+1, sad2);
	SAD_LOC_PUT(__umul24(ty, THREADS_W) + tx, search_pos+2, sad3);

	local_search_off_x += 3;
	if (local_search_off_x >= SEARCH_DIMENSION)
	  {
	    local_search_off_x -= SEARCH_DIMENSION;
	    search_off_y++;
	  }
      }
    }

  __syncthreads();

  /* Block-copy data into global memory.
   * Threads are grouped into sets of 32, leaving some threads idle. */
  if ((threadIdx.x >> 5) < (THREADS_W * THREADS_H))
  {
    int tgroup = threadIdx.x & 31;
    int ty = (threadIdx.x >> 5) / THREADS_W;
    int tx = (threadIdx.x >> 5) - __umul24(ty, THREADS_W);
    int index;

    /* Macroblock and sub-block coordinates */
    int mb_x = (tx + __umul24(bx, THREADS_W)) >> 2;
    int mb_y = (ty + __umul24(by, THREADS_H)) >> 2;
    int block_x = (tx + __umul24(bx, THREADS_W)) & 0x03;
    int block_y = (ty + __umul24(by, THREADS_H)) & 0x03;

    if ((mb_x < mb_width) && (mb_y < mb_height))
      {
	/* All SADs from this thread are stored in a contiguous chunk
	 * of memory starting at this offset */
	blk_sad += (__umul24(__umul24(mb_width, mb_height), 25) +
		    (__umul24(mb_y, mb_width) + mb_x) * 16 +
		    (4 * block_y + block_x)) *
	  MAX_POS_PADDED;

	/* Block copy, 32 threads at a time */
	for (index = tgroup; index < SAD_LOC_8B_ROW_SIZE; index += 32)
	  ((vec8b *)blk_sad)[index] 
	    = SAD_LOC_8B_GET(__umul24(ty, THREADS_W) + tx, index);
      }
  }
}

texture<unsigned short, 2, hipReadModeElementType> &get_ref(void)
{
  return ref;
}
