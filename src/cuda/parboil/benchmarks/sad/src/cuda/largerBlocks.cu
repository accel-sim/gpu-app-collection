#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "sad.h"
#include "largerBlocks.h"

typedef struct {
  unsigned short x;
  unsigned short y;
} __align__(4) uhvec;

typedef unsigned int uint;

__global__ void larger_sad_calc_8(unsigned short *blk_sad,
				  int mb_width,
				  int mb_height)
{
  int tx = threadIdx.y & 1;
  int ty = threadIdx.y >> 1;

  /* Macroblock and sub-block coordinates */
  int mb_x = blockIdx.x;
  int mb_y = blockIdx.y;

  /* Number of macroblocks in a frame */
  int macroblocks = __mul24(mb_width, mb_height);
  int macroblock_index = (__mul24(mb_y, mb_width) + mb_x) * MAX_POS_PADDED;

  int search_pos;

  unsigned short *bi;
  unsigned short *bo_6, *bo_5, *bo_4;

  bi = blk_sad
    + (__mul24(macroblocks, 25) + (ty * 8 + tx * 2)) * MAX_POS_PADDED
    + macroblock_index * 16;

  // Block type 6: 4x8
  bo_6 = blk_sad
    + ((macroblocks << 4) + macroblocks + (ty * 4 + tx * 2)) * MAX_POS_PADDED
    + macroblock_index * 8;

  if (ty < 100) // always true, but improves register allocation
    {
      // Block type 5: 8x4
      bo_5 = blk_sad
	+ ((macroblocks << 3) + macroblocks + (ty * 4 + tx)) * MAX_POS_PADDED
	+ macroblock_index * 8;

      // Block type 4: 8x8
      bo_4 = blk_sad
	+ ((macroblocks << 2) + macroblocks + (ty * 2 + tx)) * MAX_POS_PADDED
	+ macroblock_index * 4;
    }

  for (search_pos = threadIdx.x; search_pos < (MAX_POS+1)/2; search_pos += 32)
    {
      /* Each uint is actually two 2-byte integers packed together.
       * Only addition is used and there is no chance of integer overflow
       * so this can be done to reduce computation time. */
      uint i00 = ((uint *)bi)[search_pos];
      uint i01 = ((uint *)bi)[search_pos + MAX_POS_PADDED/2];
      uint i10 = ((uint *)bi)[search_pos + 4*MAX_POS_PADDED/2];
      uint i11 = ((uint *)bi)[search_pos + 5*MAX_POS_PADDED/2];

      ((uint *)bo_6)[search_pos]                  = i00 + i10;
      ((uint *)bo_6)[search_pos+MAX_POS_PADDED/2] = i01 + i11;
      ((uint *)bo_5)[search_pos]                  = i00 + i01;
      ((uint *)bo_5)[search_pos+2*MAX_POS_PADDED/2] = i10 + i11;
      ((uint *)bo_4)[search_pos]                  = (i00 + i01) + (i10 + i11);
    }
}

__global__ void larger_sad_calc_16(unsigned short *blk_sad,
				   int mb_width,
				   int mb_height)
{
  /* Macroblock coordinates */
  int mb_x = blockIdx.x;
  int mb_y = blockIdx.y;

  /* Number of macroblocks in a frame */
  int macroblocks = __mul24(mb_width, mb_height) * MAX_POS_PADDED;
  int macroblock_index = (__mul24(mb_y, mb_width) + mb_x) * MAX_POS_PADDED;

  int search_pos;

  unsigned short *bi;
  unsigned short *bo_3, *bo_2, *bo_1;

  //bi = blk_sad + macroblocks * 5 + macroblock_index * 4;
  bi = blk_sad + ((macroblocks + macroblock_index) << 2) + macroblocks;

  // Block type 3: 8x16
  //bo_3 = blk_sad + macroblocks * 3 + macroblock_index * 2;
  bo_3 = blk_sad + ((macroblocks + macroblock_index) << 1) + macroblocks;

  // Block type 5: 8x4
  bo_2 = blk_sad + macroblocks + macroblock_index * 2;

  // Block type 4: 8x8
  bo_1 = blk_sad + macroblock_index;

  for (search_pos = threadIdx.x; search_pos < (MAX_POS+1)/2; search_pos += 32)
    {
      /* Each uint is actually two 2-byte integers packed together.
       * Only addition is used and there is no chance of integer overflow
       * so this can be done to reduce computation time. */
      uint i00 = ((uint *)bi)[search_pos];
      uint i01 = ((uint *)bi)[search_pos + MAX_POS_PADDED/2];
      uint i10 = ((uint *)bi)[search_pos + 2*MAX_POS_PADDED/2];
      uint i11 = ((uint *)bi)[search_pos + 3*MAX_POS_PADDED/2];

      ((uint *)bo_3)[search_pos]                  = i00 + i10;
      ((uint *)bo_3)[search_pos+MAX_POS_PADDED/2] = i01 + i11;
      ((uint *)bo_2)[search_pos]                  = i00 + i01;
      ((uint *)bo_2)[search_pos+MAX_POS_PADDED/2] = i10 + i11;
      ((uint *)bo_1)[search_pos]                  = (i00 + i01) + (i10 + i11);
    }
}
