#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "sad.h"
#include "largerBlocks.h"
#include <stdio.h>


typedef struct {
  unsigned short x;
  unsigned short y;
} __align__(4) uhvec;

typedef unsigned int uint;

__global__ void larger_sad_calc_8(unsigned short *blk_sad,
				  int mb_width,
				  int mb_height)
{
  int tx = threadIdx.y & 1;
  int ty = threadIdx.y >> 1;

  /* Macroblock and sub-block coordinates */
  int mb_x = blockIdx.x;
  int mb_y = blockIdx.y;

  /* Number of macroblocks in a frame */
  int macroblocks = __mul24(mb_width, mb_height);
  int macroblock_index = (__mul24(mb_y, mb_width) + mb_x) * MAX_POS_PADDED;

  int search_pos;

  unsigned short *bi;
  unsigned short *bo_6, *bo_5, *bo_4;

  bi = blk_sad
    + (__mul24(macroblocks, 25) + (ty * 8 + tx * 2)) * MAX_POS_PADDED
    + macroblock_index * 16;

  // Block type 6: 4x8
  bo_6 = blk_sad
    + ((macroblocks << 4) + macroblocks + (ty * 4 + tx * 2)) * MAX_POS_PADDED
    + macroblock_index * 8;

  if (ty < 100) // always true, but improves register allocation
    {
      // Block type 5: 8x4
      bo_5 = blk_sad
	+ ((macroblocks << 3) + macroblocks + (ty * 4 + tx)) * MAX_POS_PADDED
	+ macroblock_index * 8;

      // Block type 4: 8x8
      bo_4 = blk_sad
	+ ((macroblocks << 2) + macroblocks + (ty * 2 + tx)) * MAX_POS_PADDED
	+ macroblock_index * 4;
    }

  for (search_pos = threadIdx.x; search_pos < (MAX_POS+1)/2; search_pos += 32)
    {
      /* Each uint is actually two 2-byte integers packed together.
       * Only addition is used and there is no chance of integer overflow
       * so this can be done to reduce computation time. */
      uint i00 = ((uint *)bi)[search_pos];
      uint i01 = ((uint *)bi)[search_pos + MAX_POS_PADDED/2];
      uint i10 = ((uint *)bi)[search_pos + 4*MAX_POS_PADDED/2];
      uint i11 = ((uint *)bi)[search_pos + 5*MAX_POS_PADDED/2];

      ((uint *)bo_6)[search_pos]                  = i00 + i10;
      ((uint *)bo_6)[search_pos+MAX_POS_PADDED/2] = i01 + i11;
      ((uint *)bo_5)[search_pos]                  = i00 + i01;
      ((uint *)bo_5)[search_pos+2*MAX_POS_PADDED/2] = i10 + i11;
      ((uint *)bo_4)[search_pos]                  = (i00 + i01) + (i10 + i11);
    }
}

__global__ void larger_sad_calc_16(unsigned short *blk_sad,
				   int mb_width,
				   int mb_height)
{
  /* Macroblock coordinates */
  int mb_x = blockIdx.x;
  int mb_y = blockIdx.y;

  /* Number of macroblocks in a frame */
  int macroblocks = __mul24(mb_width, mb_height) * MAX_POS_PADDED;
  int macroblock_index = (__mul24(mb_y, mb_width) + mb_x) * MAX_POS_PADDED;

  int search_pos;

  unsigned short *bi;
  unsigned short *bo_3, *bo_2, *bo_1;

  //bi = blk_sad + macroblocks * 5 + macroblock_index * 4;
  bi = blk_sad + ((macroblocks + macroblock_index) << 2) + macroblocks;

  // Block type 3: 8x16
  //bo_3 = blk_sad + macroblocks * 3 + macroblock_index * 2;
  bo_3 = blk_sad + ((macroblocks + macroblock_index) << 1) + macroblocks;

  // Block type 5: 8x4
  bo_2 = blk_sad + macroblocks + macroblock_index * 2;

  // Block type 4: 8x8
  bo_1 = blk_sad + macroblock_index;

  for (search_pos = threadIdx.x; search_pos < (MAX_POS+1)/2; search_pos += 32)
    {
      /* Each uint is actually two 2-byte integers packed together.
       * Only addition is used and there is no chance of integer overflow
       * so this can be done to reduce computation time. */
      uint i00 = ((uint *)bi)[search_pos];
      uint i01 = ((uint *)bi)[search_pos + MAX_POS_PADDED/2];
      uint i10 = ((uint *)bi)[search_pos + 2*MAX_POS_PADDED/2];
      uint i11 = ((uint *)bi)[search_pos + 3*MAX_POS_PADDED/2];
      
      ((uint *)bo_3)[search_pos]                  = i00 + i10;
      ((uint *)bo_3)[search_pos+MAX_POS_PADDED/2] = i01 + i11;
      ((uint *)bo_2)[search_pos]                  = i00 + i01;
      ((uint *)bo_2)[search_pos+MAX_POS_PADDED/2] = i10 + i11;
      ((uint *)bo_1)[search_pos]                  = (i00 + i01) + (i10 + i11);
 /*
      ushort2 s00 = { bi[search_pos*2], bi[search_pos*2+1] };
      ushort2 s01 = { bi[(search_pos + MAX_POS_PADDED/2)*2], bi[(search_pos + MAX_POS_PADDED/2)*2+1] };
      ushort2 s10 = { bi[(search_pos + 2*MAX_POS_PADDED/2)*2], bi[(search_pos + 2*MAX_POS_PADDED/2)*2+1] };
      ushort2 s11 = { bi[(search_pos + 3*MAX_POS_PADDED/2)*2], bi[(search_pos + 3*MAX_POS_PADDED/2)*2+1] };

      ((ushort2 *)bo_3)[search_pos]                  = make_ushort2(s00.x + s10.x, s00.y + s10.y);
      ((ushort2 *)bo_3)[search_pos+MAX_POS_PADDED/2] = make_ushort2(s01.x + s11.x, s01.y + s11.y);
      ((ushort2 *)bo_2)[search_pos]                  = make_ushort2(s00.x + s01.x, s00.y + s01.y);
      ((ushort2 *)bo_2)[search_pos+MAX_POS_PADDED/2] = make_ushort2(s10.x + s11.x, s10.y + s11.y);
      ((ushort2 *)bo_1)[search_pos]                  = make_ushort2((s00.x + s01.x)+(s10.x + s11.x), (s00.y + s01.y)+(s10.y + s11.y));
      */
    }
}
