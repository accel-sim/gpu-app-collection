/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <inttypes.h>
#include <parboil.h>
#include <hip/hip_runtime.h>

#include "sad.h"
#include "sad4.h"
#include "largerBlocks.h"
#include "file.h"
#include "image.h"

#define CUDA_ERRCK \
  {hipError_t err = hipGetLastError(); \
    if (err) fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
  }

static unsigned short *
load_sads(char *filename);
static void
write_sads(char *filename,
	   int image_width_macroblocks,
	   int image_height_macroblocks,
	   unsigned short *sads);
static void
write_sads_directly(char *filename,
		    int width,
		    int height,
		    unsigned short *sads);

/* FILE I/O */

unsigned short *
load_sads(char *filename)
{
  FILE *infile;
  unsigned short *sads;
  int w;
  int h;
  int sads_per_block;

  infile = fopen(filename, "r");

  if (!infile)
    {
      fprintf(stderr, "Cannot find file '%s'\n", filename);
      exit(-1);
    }

  /* Read image dimensions (measured in macroblocks) */
  w = read16u(infile);
  h = read16u(infile);

  /* Read SAD values.  Only interested in the 4x4 SAD values, which are
   * at the end of the file. */
  sads_per_block = MAX_POS_PADDED * (w * h);
  fseek(infile, 25 * sads_per_block * sizeof(unsigned short), SEEK_CUR);

  sads = (unsigned short *)malloc(sads_per_block * 16 * sizeof(unsigned short));
  fread(sads, sizeof(unsigned short), sads_per_block * 16, infile);
  fclose(infile);

  return sads;
}

/* Compare the reference SADs to the expected SADs.
 */
void
check_sads(unsigned short *sads_reference,
	   unsigned short *sads_computed,
	   int image_size_macroblocks)
{
  int block;

  /* Check the 4x4 SAD values.  These are in sads_reference.
   * Ignore the data at the beginning of sads_computed. */
  sads_computed += 25 * MAX_POS_PADDED * image_size_macroblocks;

  for (block = 0; block < image_size_macroblocks; block++)
    {
      int subblock;

      for (subblock = 0; subblock < 16; subblock++)
	{
	  int sad_index;

	  for (sad_index = 0; sad_index < MAX_POS; sad_index++)
	    {
	      int index =
		(block * 16 + subblock) * MAX_POS_PADDED + sad_index;

	      if (sads_reference[index] != sads_computed[index])
		{
#if 0
		  /* Print exactly where the mismatch was seen */
		  printf("M %3d %2d %4d (%d = %d)\n", block, subblock, sad_index, sads_reference[index], sads_computed[index]);
#else
		  goto mismatch;
#endif
		}
	    }
	}
    }

  printf("Success.\n");
  return;

 mismatch:
  printf("Computed SADs do not match expected values.\n");
}

/* Extract the SAD data for a particular block type for a particular
 * macroblock from the array of SADs of that block type. */
static inline void
write_subblocks(FILE *outfile, unsigned short *subblock_array, int macroblock,
		int count)
{
  int block;
  int pos;

  for (block = 0; block < count; block++)
    {
      unsigned short *vec = subblock_array +
	(block + macroblock * count) * MAX_POS_PADDED;

      /* Write all SADs for this sub-block */
      for (pos = 0; pos < MAX_POS; pos++)
	write16u(outfile, *vec++);
    }
}

/* Write some SAD data to a file for output checking.
 *
 * All SAD values for six rows of macroblocks are written.
 * The six rows consist of the top two, middle two, and bottom two image rows.
 */
void
write_sads(char *filename,
	   int mb_width,
	   int mb_height,
	   unsigned short *sads)
{
  FILE *outfile = fopen(filename, "w");
  int mbs = mb_width * mb_height;
  int row_indir;
  int row_indices[6] = {0, 1,
			mb_height / 2 - 1, mb_height / 2,
			mb_height - 2, mb_height - 1};

  if (outfile == NULL)
    {
      fprintf(stderr, "Cannot open output file\n");
      exit(-1);
    }

  /* Write the number of output macroblocks */
  write32u(outfile, mb_width * 6);

  /* Write zeros */
  write32u(outfile, 0);

  /* Each row */
  for (row_indir = 0; row_indir < 6; row_indir++)
    {
      int row = row_indices[row_indir];

      /* Each block in row */
      int block;
      for (block = mb_width * row; block < mb_width * (row + 1); block++)
	{
	  int blocktype;

	  /* Write SADs for all sub-block types */
	  for (blocktype = 1; blocktype <= 7; blocktype++)
	    write_subblocks(outfile,
			    sads + SAD_TYPE_IX(blocktype, mbs),
			    block,
			    SAD_TYPE_CT(blocktype));
	}
    }

  fclose(outfile);
}

/* FILE I/O for debugging */

static void
write_sads_directly(char *filename,
		    int width,
		    int height,
		    unsigned short *sads)
{
  FILE *f = fopen(filename, "w");
  int n;

  write16u(f, width);
  write16u(f, height);
  for (n = 0; n < 41 * MAX_POS_PADDED * (width * height); n++) {
    write16u(f, sads[n]);
  }
  fclose(f);
}

static void
print_test_sad_vector(unsigned short *base, int macroblock, int count)
{
  int n;
  int searchpos = 17*33+17;
  for (n = 0; n < count; n++)
    printf(" %d", base[(count * macroblock + n) * MAX_POS_PADDED + searchpos]);
}

static void
print_test_sads(unsigned short *sads_computed,
		int mbs)
{
  int macroblock = 5;
  int blocktype;

  for (blocktype = 1; blocktype <= 7; blocktype++)
    {
      printf("%d:", blocktype);
      print_test_sad_vector(sads_computed + SAD_TYPE_IX(blocktype, mbs),
			    macroblock, SAD_TYPE_CT(blocktype));
      puts("\n");
    }
}

/* MAIN */

int
main(int argc, char **argv)
{
  struct image_i16 *ref_image;
  struct image_i16 *cur_image;
  unsigned short *sads_computed; /* SADs generated by the program */

  int image_size_bytes;
  int image_width_macroblocks, image_height_macroblocks;
  int image_size_macroblocks;

  struct pb_TimerSet timers;
  struct pb_Parameters *params;

  pb_InitializeTimerSet(&timers);
  params = pb_ReadParameters(&argc, argv);

  if (pb_Parameters_CountInputs(params) != 2)
    {
      fprintf(stderr, "Expecting two input filenames\n");
      exit(-1);
    }

  /* Read input files */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  ref_image = load_image(params->inpFiles[0]);
  cur_image = load_image(params->inpFiles[1]);
  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  if ((ref_image->width != cur_image->width) ||
      (ref_image->height != cur_image->height))
    {
      fprintf(stderr, "Input images must be the same size\n");
      exit(-1);
    }
  if ((ref_image->width % 16) || (ref_image->height % 16))
    {
      fprintf(stderr, "Input image size must be an integral multiple of 16\n");
      exit(-1);
    }

  /* Compute parameters, allocate memory */
  image_size_bytes = ref_image->width * ref_image->height * sizeof(short);
  image_width_macroblocks = ref_image->width >> 4;
  image_height_macroblocks = ref_image->height >> 4;
  image_size_macroblocks = image_width_macroblocks * image_height_macroblocks;
  
  sads_computed = (unsigned short *)
    malloc(41 * MAX_POS_PADDED * image_size_macroblocks * sizeof(short));

  /* Run the kernel code */
  {
    struct hipArray *ref_ary;  /* Reference image on the device */
    short *d_cur_image;         /* Current image on the device */
    unsigned short *d_sads;     /* SADs on the device */
    dim3 macroblock_grid(image_width_macroblocks, image_height_macroblocks);

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);
    hipMalloc((void **)&d_cur_image, image_size_bytes);
    CUDA_ERRCK
    hipMallocArray(&ref_ary, &get_ref().channelDesc,
                    ref_image->width, ref_image->height);
    CUDA_ERRCK

    /* Transfer current image to device */
    hipMemcpy(d_cur_image, cur_image->data, image_size_bytes,
               hipMemcpyHostToDevice);
    CUDA_ERRCK

    /* Transfer reference image to device */
    hipMemcpy2DToArray(ref_ary,
                        0, 0,
                        ref_image->data,
                        ref_image->width * sizeof(unsigned short),
                        ref_image->width * sizeof(unsigned short),
                        ref_image->height,
                        hipMemcpyHostToDevice);
    CUDA_ERRCK
    hipBindTextureToArray(get_ref(), ref_ary);
    CUDA_ERRCK

    /* Allocate SAD data on the device */
    hipMalloc((void **)&d_sads, 41 * MAX_POS_PADDED * image_size_macroblocks *
	       sizeof(unsigned short));
    CUDA_ERRCK
    hipMemset(d_sads, 0, 41 * MAX_POS_PADDED * image_size_macroblocks *
	       sizeof(unsigned short));
    CUDA_ERRCK

    pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

    // Run the 4x4 kernel
    mb_sad_calc<<<dim3(CEIL(ref_image->width / 4, THREADS_W),
		       CEIL(ref_image->height / 4, THREADS_H)),
      dim3(CEIL(MAX_POS, POS_PER_THREAD) * THREADS_W * THREADS_H),
      SAD_LOC_SIZE_BYTES>>>
      (d_sads,
       (unsigned short *)d_cur_image,
       image_width_macroblocks,
       image_height_macroblocks);
    CUDA_ERRCK

    // Run the larger-blocks kernels
    larger_sad_calc_8<<<macroblock_grid, dim3(32, 4)>>>
      (d_sads,
       image_width_macroblocks,
       image_height_macroblocks);
    CUDA_ERRCK
    
    larger_sad_calc_16<<<macroblock_grid, dim3(32, 1)>>>
      (d_sads,
       image_width_macroblocks,
       image_height_macroblocks);
    CUDA_ERRCK

    pb_SwitchToTimer(&timers, pb_TimerID_COPY);

    /* Transfer SAD data to the host */
    hipMemcpy(sads_computed,// + 25 * MAX_POS_PADDED * image_size_macroblocks,
	       d_sads,// + 25 * MAX_POS_PADDED * image_size_macroblocks,
	       41 * MAX_POS_PADDED * image_size_macroblocks * sizeof(unsigned short)
,
           hipMemcpyDeviceToHost);
    CUDA_ERRCK

    /* Free GPU memory */
    hipFree(d_sads);
    CUDA_ERRCK
    hipUnbindTexture(get_ref());
    CUDA_ERRCK
    hipFreeArray(ref_ary);
    CUDA_ERRCK
    hipFree(d_cur_image);
    CUDA_ERRCK

    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  }

  /* Print output */
  if (params->outFile)
    {
      pb_SwitchToTimer(&timers, pb_TimerID_IO);
      write_sads(params->outFile,
		 image_width_macroblocks,
		 image_height_macroblocks,
		 sads_computed);
      pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
    }

#if 0  /* Debugging */
  print_test_sads(sads_computed, image_size_macroblocks);
  write_sads_directly("sad-debug.bin",
		      ref_image->width / 16, ref_image->height / 16,
		      sads_computed);
#endif

  /* Free memory */
  free(sads_computed);
  free_image(ref_image);
  free_image(cur_image);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);

  return 0;
}
