#include "hip/hip_runtime.h"
/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

#include "sad.h"
#include "sad4.h"

/* The part of the reference image that is in the search range */
texture<unsigned short, 2, hipReadModeElementType> ref;

/* The compute kernel. */
/* The macros THREADS_W and THREADS_H specify the width and height of the
 * area to be processed by one thread, measured in 4-by-4 pixel blocks.
 * Larger numbers mean more computation per thread block.
 *
 * The macro POS_PER_THREAD specifies the number of search positions for which
 * an SAD is computed.  A larger value indicates more computation per thread,
 * and fewer threads per thread block.  It must be a multiple of 3 and also
 * must be at most 33 because the loop to copy from shared memory uses
 * 32 threads per 4-by-4 pixel block.
 *
 */
__global__ void mb_sad_calc(unsigned short *blk_sad,
                            unsigned short *frame,
                            int mb_width,
                            int mb_height)
{
  int tx = (threadIdx.x / CEIL(MAX_POS, POS_PER_THREAD)) % THREADS_W;
  int ty = (threadIdx.x / CEIL(MAX_POS, POS_PER_THREAD)) / THREADS_W;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int img_width = mb_width*16;

  /* Macroblock and sub-block coordinates */
  int mb_x = (tx + bx * THREADS_W) >> 2;
  int mb_y = (ty + by * THREADS_H) >> 2;
  int block_x = (tx + bx * THREADS_W) & 0x03;
  int block_y = (ty + by * THREADS_H) & 0x03;

  /* If this thread is assigned to an invalid 4x4 block, do nothing */
  if ((mb_x < mb_width) && (mb_y < mb_height))
    {
      /* Pixel offset of the origin of the current 4x4 block */
      int frame_x = ((mb_x << 2) + block_x) << 2;
      int frame_y = ((mb_y << 2) + block_y) << 2;

      /* Origin of the search area for this 4x4 block */
      int ref_x = frame_x - SEARCH_RANGE;
      int ref_y = frame_y - SEARCH_RANGE;

      /* Origin in the current frame for this 4x4 block */
      int cur_o = frame_y * img_width + frame_x;

      int search_pos;
      int search_pos_base =
        (threadIdx.x % CEIL(MAX_POS, POS_PER_THREAD)) * POS_PER_THREAD;
      int search_pos_end = search_pos_base + POS_PER_THREAD;

      /* All SADs from this thread are stored in a contiguous chunk
       * of memory starting at this offset */
      blk_sad += mb_width * mb_height * MAX_POS_PADDED * (9 + 16) +
        (mb_y * mb_width + mb_x) * MAX_POS_PADDED * 16 +
        (4 * block_y + block_x) * MAX_POS_PADDED;

      /* Don't go past bounds */
      if (search_pos_end > MAX_POS)
        search_pos_end = MAX_POS;

      /* For each search position, within the range allocated to this thread */
      for (search_pos = search_pos_base;
           search_pos < search_pos_end;
           search_pos++) {
        unsigned short sad4x4 = 0;
        int search_off_x = ref_x + (search_pos % SEARCH_DIMENSION);
        int search_off_y = ref_y + (search_pos / SEARCH_DIMENSION);

        /* 4x4 SAD computation */
        for(int y=0; y<4; y++) {
          for (int x=0; x<4; x++) {
            sad4x4 +=
              abs(tex2D(ref, search_off_x + x, search_off_y + y) -
                  frame[cur_o + y * img_width + x]);
          }
        }

        /* Save this value into the local SAD array */
        blk_sad[search_pos] = sad4x4;
      }
    }
}

texture<unsigned short, 2, hipReadModeElementType> &get_ref(void)
{
  return ref;
}
