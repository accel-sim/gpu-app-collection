#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/


#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "util.h"

__global__ void histo_prescan_kernel (
        unsigned int* input,
        int size,
        unsigned int* minmax);

__global__ void histo_main_kernel (
        uchar4 *sm_mappings,
        unsigned int num_elements,
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow);

__global__ void histo_intermediates_kernel (
        uint2 *input,
        unsigned int height,
        unsigned int width,
        unsigned int input_pitch,
        uchar4 *sm_mappings);

__global__ void histo_final_kernel (
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow,
        unsigned int *final_histo);

/******************************************************************************
* Implementation: GPU
* Details:
* in the GPU implementation of histogram, we begin by computing the span of the
* input values into the histogram. Then the histogramming computation is carried
* out by a (BLOCK_X, BLOCK_Y) sized grid, where every group of Y (same X)
* computes its own partial histogram for a part of the input, and every Y in the
* group exclusively writes to a portion of the span computed in the beginning.
* Finally, a reduction is performed to combine all the partial histograms into
* the final result.
******************************************************************************/

int main(int argc, char* argv[]) {
  struct pb_TimerSet *timersPtr;
  struct pb_Parameters *parameters;

  parameters = pb_ReadParameters(&argc, argv);
  if (!parameters)
    return -1;

  if(!parameters->inpFiles[0]){
    fputs("Input file expected\n", stderr);
    return -1;
  }

  timersPtr = (struct pb_TimerSet *) malloc (sizeof(struct pb_TimerSet));
  
  
  //appendDefaultTimerSet(NULL);
  
  
  if (timersPtr == NULL) {
    fprintf(stderr, "Could not append default timer set!\n");
    exit(1);
  }
  
  struct pb_TimerSet timers = *timersPtr;
  
//  pb_CreateTimer(&timers, "myTimer!", 0);
  
  
  pb_InitializeTimerSet(&timers);
  
  pb_AddSubTimer(&timers, "Input", pb_TimerID_IO);
  pb_AddSubTimer(&timers, "Output", pb_TimerID_IO);
  
  char *prescans = "PreScanKernel";
  char *postpremems = "PostPreMems";
  char *intermediates = "IntermediatesKernel";
  char *mains = "MainKernel";
  char *finals = "FinalKernel";
  
  pb_AddSubTimer(&timers, prescans, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, postpremems, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, intermediates, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, mains, pb_TimerID_KERNEL);
  pb_AddSubTimer(&timers, finals, pb_TimerID_KERNEL);
  
//  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  pb_SwitchToSubTimer(&timers, "Input", pb_TimerID_IO);

  int numIterations;
  if (argc >= 2){
    numIterations = atoi(argv[1]);
  } else {
    fputs("Expected at least one command line argument\n", stderr);
    return -1;
  }

  unsigned int img_width, img_height;
  unsigned int histo_width, histo_height;

  FILE* f = fopen(parameters->inpFiles[0],"rb");
  int result = 0;

  result += fread(&img_width,    sizeof(unsigned int), 1, f);
  result += fread(&img_height,   sizeof(unsigned int), 1, f);
  result += fread(&histo_width,  sizeof(unsigned int), 1, f);
  result += fread(&histo_height, sizeof(unsigned int), 1, f);

  if (result != 4){
    fputs("Error reading input and output dimensions from file\n", stderr);
    return -1;
  }

  unsigned int* img = (unsigned int*) malloc (img_width*img_height*sizeof(unsigned int));
  unsigned char* histo = (unsigned char*) calloc (histo_width*histo_height, sizeof(unsigned char));

  result = fread(img, sizeof(unsigned int), img_width*img_height, f);

  fclose(f);

  if (result != img_width*img_height){
    fputs("Error reading input array from file\n", stderr);
    return -1;
  }

  int even_width = ((img_width+1)/2)*2;
  unsigned int* input;
  unsigned int* ranges;
  uchar4* sm_mappings;
  unsigned int* global_subhisto;
  unsigned short* global_histo;
  unsigned int* global_overflow;
  unsigned char* final_histo;

  hipMalloc((void**)&input           , even_width*(((img_height+UNROLL-1)/UNROLL)*UNROLL)*sizeof(unsigned int));
  hipMalloc((void**)&ranges          , 2*sizeof(unsigned int));
  hipMalloc((void**)&sm_mappings     , img_width*img_height*sizeof(uchar4));
  hipMalloc((void**)&global_subhisto , img_width*histo_height*sizeof(unsigned int));
  hipMalloc((void**)&global_histo    , img_width*histo_height*sizeof(unsigned short));
  hipMalloc((void**)&global_overflow , img_width*histo_height*sizeof(unsigned int));
  hipMalloc((void**)&final_histo     , img_width*histo_height*sizeof(unsigned char));

  hipMemset(final_histo           ,0 , img_width*histo_height*sizeof(unsigned char));

  for (int y=0; y < img_height; y++){
    hipMemcpy(&(((unsigned int*)input)[y*even_width]),&img[y*img_width],img_width*sizeof(unsigned int), hipMemcpyHostToDevice);
  }
  
  //pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
  pb_SwitchToSubTimer(&timers, NULL, pb_TimerID_KERNEL);
  
  
  unsigned int *zeroData = (unsigned int *) calloc(img_width*histo_height, sizeof(unsigned int));
  

  for (int iter = 0; iter < numIterations; iter++) {
    unsigned int ranges_h[2] = {UINT32_MAX, 0};

    hipMemcpy(ranges,ranges_h, 2*sizeof(unsigned int), hipMemcpyHostToDevice);


    pb_SwitchToSubTimer(&timers, prescans , pb_TimerID_KERNEL);

    histo_prescan_kernel<<<dim3(PRESCAN_BLOCKS_X),dim3(PRESCAN_THREADS)>>>((unsigned int*)input, img_height*img_width, ranges);

    pb_SwitchToSubTimer(&timers, postpremems , pb_TimerID_KERNEL);

    hipMemcpy(ranges_h,ranges, 2*sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipMemcpy(global_subhisto,zeroData, img_width*histo_height*sizeof(unsigned int), hipMemcpyHostToDevice);
    //    hipMemset(global_subhisto,0,img_width*histo_height*sizeof(unsigned int));

    pb_SwitchToSubTimer(&timers, intermediates, pb_TimerID_KERNEL);

    histo_intermediates_kernel<<<dim3((img_height + UNROLL-1)/UNROLL), dim3((img_width+1)/2)>>>(
                (uint2*)(input),
                (unsigned int)img_height,
                (unsigned int)img_width,
                (img_width+1)/2,
                (uchar4*)(sm_mappings)
    );
    
    pb_SwitchToSubTimer(&timers, mains, pb_TimerID_KERNEL);    
    
    histo_main_kernel<<<dim3(BLOCK_X, ranges_h[1]-ranges_h[0]+1), dim3(THREADS)>>>(
                (uchar4*)(sm_mappings),
                img_height*img_width,
                ranges_h[0], ranges_h[1],
                histo_height, histo_width,
                (unsigned int*)(global_subhisto),
                (unsigned int*)(global_histo),
                (unsigned int*)(global_overflow)
    );
    
    pb_SwitchToSubTimer(&timers, finals, pb_TimerID_KERNEL);    

    histo_final_kernel<<<dim3(BLOCK_X*3), dim3(512)>>>(
                ranges_h[0], ranges_h[1],
                histo_height, histo_width,
                (unsigned int*)(global_subhisto),
                (unsigned int*)(global_histo),
                (unsigned int*)(global_overflow),
                (unsigned int*)(final_histo)
    );
  }

  pb_SwitchToSubTimer(&timers, "Output", pb_TimerID_IO);
  //  pb_SwitchToTimer(&timers, pb_TimerID_IO);


  hipMemcpy(histo,final_histo, histo_height*histo_width*sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(input);
  hipFree(ranges);
  hipFree(sm_mappings);
  hipFree(global_subhisto);
  hipFree(global_histo);
  hipFree(global_overflow);
  hipFree(final_histo);

  if (parameters->outFile) {
    dump_histo_img(histo, histo_height, histo_width, parameters->outFile);
  }

  //pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  pb_SwitchToSubTimer(&timers, NULL, pb_TimerID_COMPUTE);

  free(img);
  free(histo);

  pb_SwitchToSubTimer(&timers, NULL, pb_TimerID_NONE);
  
  printf("\n");
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(parameters);

  return 0;
}
