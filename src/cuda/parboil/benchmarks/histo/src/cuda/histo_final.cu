#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include "util.h"

/* Combine all the sub-histogram results into one final histogram */
__global__ void histo_final_kernel (
    unsigned int sm_range_min, 
    unsigned int sm_range_max,
    unsigned int histo_height, 
    unsigned int histo_width,
    unsigned int *global_subhisto,
    unsigned int *global_histo,
    unsigned int *global_overflow,
    unsigned int *final_histo) //final output
{
    unsigned int start_offset = threadIdx.x + blockIdx.x * blockDim.x;
    const ushort4 zero_short  = {0, 0, 0, 0};
    const uint4 zero_int      = {0, 0, 0, 0};

    unsigned int size_low_histo = sm_range_min * BINS_PER_BLOCK;
    unsigned int size_mid_histo = (sm_range_max - sm_range_min +1) * BINS_PER_BLOCK;

    /* Clear lower region of global histogram */
    for (unsigned int i = start_offset; i < size_low_histo/4; i += gridDim.x * blockDim.x)
    {
        ushort4 global_histo_data = ((ushort4*)global_histo)[i];
        ((ushort4*)global_histo)[i] = zero_short;

        global_histo_data.x = min (global_histo_data.x, 255);
        global_histo_data.y = min (global_histo_data.y, 255);
        global_histo_data.z = min (global_histo_data.z, 255);
        global_histo_data.w = min (global_histo_data.w, 255);

        uchar4 final_histo_data = {
            global_histo_data.x,
            global_histo_data.y,
            global_histo_data.z,
            global_histo_data.w
        };

        ((uchar4*)final_histo)[i] = final_histo_data;
    }

    /* Clear the middle region of the overflow buffer */
    for (unsigned int i = (size_low_histo/4) + start_offset; i < (size_low_histo+size_mid_histo)/4; i += gridDim.x * blockDim.x)
    {
        uint4 global_histo_data = ((uint4*)global_overflow)[i];
        ((uint4*)global_overflow)[i] = zero_int;

        uint4 internal_histo_data = {
            global_histo_data.x,
            global_histo_data.y,
            global_histo_data.z,
            global_histo_data.w
        };

        #pragma unroll
        for (int j = 0; j < BLOCK_X; j++)
        {
            unsigned int bin4in = ((unsigned int*)global_subhisto)[i + j * histo_height * histo_width / 4];
            internal_histo_data.x += (bin4in >>  0) & 0xFF;
            internal_histo_data.y += (bin4in >>  8) & 0xFF;
            internal_histo_data.z += (bin4in >> 16) & 0xFF;
            internal_histo_data.w += (bin4in >> 24) & 0xFF;
        }

        internal_histo_data.x = min (internal_histo_data.x, 255);
        internal_histo_data.y = min (internal_histo_data.y, 255);
        internal_histo_data.z = min (internal_histo_data.z, 255);
        internal_histo_data.w = min (internal_histo_data.w, 255);

        uchar4 final_histo_data = {
            internal_histo_data.x,
            internal_histo_data.y,
            internal_histo_data.z,
            internal_histo_data.w
        };

        ((uchar4*)final_histo)[i] = final_histo_data;
    }

    /* Clear the upper region of global histogram */
    for (unsigned int i = ((size_low_histo+size_mid_histo)/4) + start_offset; i < (histo_height*histo_width)/4; i += gridDim.x * blockDim.x)
    {
        ushort4 global_histo_data = ((ushort4*)global_histo)[i];
        ((ushort4*)global_histo)[i] = zero_short;

        global_histo_data.x = min (global_histo_data.x, 255);
        global_histo_data.y = min (global_histo_data.y, 255);
        global_histo_data.z = min (global_histo_data.z, 255);
        global_histo_data.w = min (global_histo_data.w, 255);

        uchar4 final_histo_data = {
            global_histo_data.x,
            global_histo_data.y,
            global_histo_data.z,
            global_histo_data.w
        };

        ((uchar4*)final_histo)[i] = final_histo_data;
    }
}
