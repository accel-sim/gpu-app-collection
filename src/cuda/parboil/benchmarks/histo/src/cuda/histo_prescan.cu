#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#include "util.h"

__global__ void histo_prescan_kernel (unsigned int* input, int size, unsigned int* minmax)
{
    __shared__ float Avg[PRESCAN_THREADS];
    __shared__ float StdDev[PRESCAN_THREADS];

    int stride = size/gridDim.x;
    int addr = blockIdx.x*stride+threadIdx.x;
    int end = blockIdx.x*stride + stride/8; // Only sample 1/8th of the input data

    // Compute the average per thread
    float avg = 0.0;
    unsigned int count = 0;
    while (addr < end){
        avg += input[addr];
        count++;
	addr += blockDim.x;
    }
    avg /= count;
    Avg[threadIdx.x] = avg;

    // Compute the standard deviation per thread
    int addr2 = blockIdx.x*stride+threadIdx.x;
    float stddev = 0;
    while (addr2 < end){
        stddev += (input[addr2]-avg)*(input[addr2]-avg);
        addr2 += blockDim.x;
    }
    stddev /= count;
    StdDev[threadIdx.x] = sqrtf(stddev);

#define SUM(stride__)\
if(threadIdx.x < stride__){\
    Avg[threadIdx.x] += Avg[threadIdx.x+stride__];\
    StdDev[threadIdx.x] += StdDev[threadIdx.x+stride__];\
}

    // Add all the averages and standard deviations from all the threads
    // and take their arithmetic average (as a simplified approximation of the
    // real average and standard deviation.
#if (PRESCAN_THREADS >= 32)    
    for (int stride = PRESCAN_THREADS/2; stride >= 32; stride = stride >> 1){
	__syncthreads();
	SUM(stride);
    }
#endif
#if (PRESCAN_THREADS >= 16)
    SUM(16);
#endif
#if (PRESCAN_THREADS >= 8)
    SUM(8);
#endif
#if (PRESCAN_THREADS >= 4)
    SUM(4);
#endif
#if (PRESCAN_THREADS >= 2)
    SUM(2);
#endif

    if (threadIdx.x == 0){
        float avg = Avg[0]+Avg[1];
	avg /= PRESCAN_THREADS;
	float stddev = StdDev[0]+StdDev[1];
	stddev /= PRESCAN_THREADS;

        // Take the maximum and minimum range from all the blocks. This will
        // be the final answer. The standard deviation is taken out to 10 sigma
        // away from the average. The value 10 was obtained empirically.
	    atomicMin(minmax,((unsigned int)(avg-10*stddev))/(KB*1024));
        atomicMax(minmax+1,((unsigned int)(avg+10*stddev))/(KB*1024));
    }
}
