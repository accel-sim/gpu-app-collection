#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "util.h"

__device__ void testIncrementGlobal (
        unsigned int *global_histo,
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        const uchar4 sm)
{
        const unsigned int range = sm.x;
        const unsigned int indexhi = sm.y;
        const unsigned int indexlo = sm.z;
        const unsigned int offset  = sm.w;

        /* Scan for inputs that are outside the central region of histogram */
        if (range < sm_range_min || range > sm_range_max)
        {
                const unsigned int bin = range * BINS_PER_BLOCK + offset / 8 + (indexlo << 2) + (indexhi << 10);
                const unsigned int bin_div2 = bin / 2;
                const unsigned int bin_offset = (bin % 2 == 1) ? 16 : 0;

                unsigned int old_val = global_histo[bin_div2];
                unsigned short old_bin = (old_val >> bin_offset) & 0xFFFF;

                if (old_bin < 255)
                {
                        atomicAdd (&global_histo[bin_div2], 1 << bin_offset);
                }
        }
}

__device__ void testIncrementLocal (
        unsigned int *global_overflow,
        unsigned int smem[KB][256],
        const unsigned int myRange,
        const uchar4 sm)
{
        const unsigned int range = sm.x;
        const unsigned int indexhi = sm.y;
        const unsigned int indexlo = sm.z;
        const unsigned int offset  = sm.w;

        /* Scan for inputs that are inside the central region of histogram */
        if (range == myRange)
        {
                /* Atomically increment shared memory */
                unsigned int add = (unsigned int)(1 << offset);
                unsigned int prev = atomicAdd (&smem[indexhi][indexlo], add);

                /* Check if current bin overflowed */
                unsigned int prev_bin_val = (prev >> offset) & 0x000000FF;

                /* If there was an overflow, record it and record if it cascaded into other bins */
                if (prev_bin_val == 0x000000FF)
                {
                        const unsigned int bin =
                                range * BINS_PER_BLOCK +
                                offset / 8 + (indexlo << 2) + (indexhi << 10);

                        bool can_overflow_to_bin_plus_1 = (offset < 24) ? true : false;
                        bool can_overflow_to_bin_plus_2 = (offset < 16) ? true : false;
                        bool can_overflow_to_bin_plus_3 = (offset <  8) ? true : false;

                        bool overflow_into_bin_plus_1 = false;
                        bool overflow_into_bin_plus_2 = false;
                        bool overflow_into_bin_plus_3 = false;

                        unsigned int prev_bin_plus_1_val = (prev >> (offset +  8)) & 0x000000FF;
                        unsigned int prev_bin_plus_2_val = (prev >> (offset + 16)) & 0x000000FF;
                        unsigned int prev_bin_plus_3_val = (prev >> (offset + 24)) & 0x000000FF;

                        if (can_overflow_to_bin_plus_1 &&        prev_bin_val == 0x000000FF) overflow_into_bin_plus_1 = true;
                        if (can_overflow_to_bin_plus_2 && prev_bin_plus_1_val == 0x000000FF) overflow_into_bin_plus_2 = true;
                        if (can_overflow_to_bin_plus_3 && prev_bin_plus_2_val == 0x000000FF) overflow_into_bin_plus_3 = true;

                        unsigned int bin_plus_1_add;
                        unsigned int bin_plus_2_add;
                        unsigned int bin_plus_3_add;

                        if (overflow_into_bin_plus_1) bin_plus_1_add = (prev_bin_plus_1_val < 0x000000FF) ? 0xFFFFFFFF : 0x000000FF;
                        if (overflow_into_bin_plus_2) bin_plus_2_add = (prev_bin_plus_2_val < 0x000000FF) ? 0xFFFFFFFF : 0x000000FF;
                        if (overflow_into_bin_plus_3) bin_plus_3_add = (prev_bin_plus_3_val < 0x000000FF) ? 0xFFFFFFFF : 0x000000FF;

                                                      atomicAdd (&global_overflow[bin],   256);
                        if (overflow_into_bin_plus_1) atomicAdd (&global_overflow[bin+1], bin_plus_1_add);
                        if (overflow_into_bin_plus_2) atomicAdd (&global_overflow[bin+2], bin_plus_2_add);
                        if (overflow_into_bin_plus_3) atomicAdd (&global_overflow[bin+3], bin_plus_3_add);
                }
        }
}

__device__ void clearMemory (unsigned int smem[KB][256])
{
        for (int i = threadIdx.x; i < BINS_PER_BLOCK / 4; i += blockDim.x)
        {
                ((unsigned int*)smem)[i] = 0;
        }
}

__device__ void copyMemory (unsigned int *dst, unsigned int src[KB][256])
{
        for (int i = threadIdx.x; i < BINS_PER_BLOCK / 4; i += blockDim.x)
        {
                dst[i] = ((unsigned int*)src)[i];
        }
}

__global__ void histo_main_kernel (
        uchar4 *sm_mappings,
        unsigned int num_elements,
        unsigned int sm_range_min,
        unsigned int sm_range_max,
        unsigned int histo_height,
        unsigned int histo_width,
        unsigned int *global_subhisto,
        unsigned int *global_histo,
        unsigned int *global_overflow)
{
        /* Most optimal solution uses 24 * 1024 bins per threadblock */
        __shared__ unsigned int sub_histo[KB][256];

        /* Each threadblock contributes to a specific 24KB range of histogram,
         * and also scans every N-th line for interesting data.  N = gridDim.x
         */
        unsigned int local_scan_range = sm_range_min + blockIdx.y;
        unsigned int local_scan_load = blockIdx.x * blockDim.x + threadIdx.x;

        clearMemory (sub_histo);
        __syncthreads();

        if (blockIdx.y == 0)
        {
                /* Loop through and scan the input */
                while (local_scan_load < num_elements)
                {
                        /* Read buffer */
                        uchar4 sm = sm_mappings[local_scan_load];
                        local_scan_load += blockDim.x * gridDim.x;

                        /* Check input */
                        testIncrementLocal (
                                global_overflow,
                                sub_histo,
                                local_scan_range,
                                sm
                        );
                        testIncrementGlobal (
                                global_histo,
                                sm_range_min,
                                sm_range_max,
                                sm
                        );
                }
        }
        else
        {
                /* Loop through and scan the input */
                while (local_scan_load < num_elements)
                {
                        /* Read buffer */
                        uchar4 sm = sm_mappings[local_scan_load];
                        local_scan_load += blockDim.x * gridDim.x;

                        /* Check input */
                        testIncrementLocal (
                                global_overflow,
                                sub_histo,
                                local_scan_range,
                                sm
                        );
                }
        }

        /* Store sub histogram to global memory */
        unsigned int store_index = blockIdx.x * (histo_height * histo_width / 4) + (local_scan_range * BINS_PER_BLOCK / 4);

        __syncthreads();
        copyMemory (&(global_subhisto[store_index]), sub_histo);
}
