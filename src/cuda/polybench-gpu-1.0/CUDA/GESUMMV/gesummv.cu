#include "hip/hip_runtime.h"
/**
 * gesummv.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define N 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 43532.0f
#define BETA 12313.0f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void gesummv(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int i, j;
	
	for (i = 0; i < N; i++)
	{
		tmp[i] = 0;
		y[i] = 0;
		for (j = 0; j < N; j++)
		{
			tmp[i] = A[i*N + j] * x[j] + tmp[i];
			y[i] = B[i*N + j] * x[j] + y[i];
		}
		
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}


void init(DATA_TYPE* A, DATA_TYPE* x)
{
  	int i, j;

 	for (i = 0; i < N; i++)
    {
    	x[i] = ((DATA_TYPE) i) / N;
      	
		for (j = 0; j < N; j++) 
		{
			A[i*N + j] = ((DATA_TYPE) i*j) / N;
		}
    }
}


void compareResults(DATA_TYPE* y, DATA_TYPE* y_outputFromGpu)
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<(N); i++) 
	{
		if (percentDiff(y[i], y_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gesummv_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *x, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N)
	{
		int j;
		for(j = 0; j < N; j++)
		{	
			tmp[i] += a[i * N + j] * x[j];
			y[i] += b[i * N + j] * x[j];
		}
		y[i] = ALPHA * tmp[i] + BETA * y[i];
	}
}

void gesummvCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp, DATA_TYPE* y_outputFromGpu)
{
	double t_start, t_end;		

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * N * N);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * N);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * N);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), 1);


	t_start = rtclock();
	gesummv_kernel<<< grid, block>>>(A_gpu,B_gpu,x_gpu, y_gpu, tmp_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * N, hipMemcpyDeviceToHost);

	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
}


int main(int argc, char *argv[])
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* x;  
	DATA_TYPE* y;
	DATA_TYPE* y_outputFromGpu;
	DATA_TYPE* tmp;
	
	A = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(N*N*sizeof(DATA_TYPE));
	x = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE)); 
	y = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	y_outputFromGpu = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(N*sizeof(DATA_TYPE));

	init(A, x);
	
	GPU_argv_init();
	gesummvCuda(A, B, x, y, tmp, y_outputFromGpu);
	
	t_start = rtclock();
	gesummv(A, B, x, y, tmp);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(y, y_outputFromGpu);

	free(A);
	free(B);  
	free(x);  
	free(y);
	free(y_outputFromGpu);
	free(tmp);

	return 0;
}

