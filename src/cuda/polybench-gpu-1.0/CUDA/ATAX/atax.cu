#include "hip/hip_runtime.h"
/**
 * atax.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.5

#define GPU_DEVICE 0

/* Problem size. */
#define NX 4096
#define NY 4096

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

#ifndef M_PI
#define M_PI 3.14159
#endif

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_array(DATA_TYPE *x, DATA_TYPE *A)
{
	int i, j;

	for (i = 0; i < NX; i++)
	{
		x[i] = i * M_PI;
		for (j = 0; j < NY; j++)
		{
			A[i*NY + j] = ((DATA_TYPE) i*(j)) / NX;
		}
	}
}


void compareResults(DATA_TYPE *z, DATA_TYPE *z_outputFromGpu)
{
	int i, fail;
	fail = 0;

	for (i=0; i<NY; i++)
	{
		if (percentDiff(z[i], z_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD)
		{
			fail++;
		}		
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void atax_kernel1(DATA_TYPE *A, DATA_TYPE *x, DATA_TYPE *tmp)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < NX)
	{
		int j;
		for(j=0; j < NY; j++)
		{
			tmp[i] += A[i * NY + j] * x[j];
		}
	}
}

__global__ void atax_kernel2(DATA_TYPE *A, DATA_TYPE *y, DATA_TYPE *tmp)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (j < NY)
	{
		int i;
		for(i=0; i < NX; i++)
		{
			y[j] += A[i * NY + j] * tmp[i];
		}
	}
}


void atax_cpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp)
{
	int i,j;
	
	for (i= 0; i < NY; i++)
	{
    	y[i] = 0;
	}
  
	for (i = 0; i < NX; i++)
 	{
      	tmp[i] = 0;

      	for (j = 0; j < NY; j++)
		{
			tmp[i] = tmp[i] + A[i*NY + j] * x[j];
		}
		
      	for (j = 0; j < NY; j++)
		{
			y[j] = y[j] + A[i*NY + j] * tmp[i];
		}
    }
}


void ataxGpu(DATA_TYPE* A, DATA_TYPE* x, DATA_TYPE* y, DATA_TYPE* tmp, DATA_TYPE* y_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NX * NY);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NY);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NX);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NY, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NX, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NX) / ((float)block.x) )), 1);
	dim3 grid2((size_t)(ceil( ((float)NY) / ((float)block.x) )), 1);

	t_start = rtclock();
	atax_kernel1<<< grid1, block >>>(A_gpu,x_gpu,tmp_gpu);
	hipDeviceSynchronize();
	atax_kernel2<<< grid2, block >>>(A_gpu,y_gpu,tmp_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(tmp_gpu);
}


int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* x;
	DATA_TYPE* y;
	DATA_TYPE* y_outputFromGpu;
	DATA_TYPE* tmp;

	A = (DATA_TYPE*)malloc(NX*NY*sizeof(DATA_TYPE));
	x = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	y_outputFromGpu = (DATA_TYPE*)malloc(NY*sizeof(DATA_TYPE));
	tmp = (DATA_TYPE*)malloc(NX*sizeof(DATA_TYPE));

	init_array(x, A);

	GPU_argv_init();
	ataxGpu(A, x, y, tmp, y_outputFromGpu);
	
	t_start = rtclock();
	atax_cpu(A, x, y, tmp);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(y, y_outputFromGpu);

	free(A);
	free(x);
	free(y);
	free(y_outputFromGpu);
	free(tmp);

  	return 0;
}

