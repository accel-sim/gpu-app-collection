#include "hip/hip_runtime.h"
/**
 * gramschmidt.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Problem size */
#define M 2048
#define N 2048

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 256
#define DIM_THREAD_BLOCK_Y 1

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void gramschmidt(DATA_TYPE* A, DATA_TYPE* R, DATA_TYPE* Q)
{
	int i,j,k;
	DATA_TYPE nrm;
	for (k = 0; k < N; k++)
	{
		nrm = 0;
		for (i = 0; i < M; i++)
		{
			nrm += A[i*N + k] * A[i*N + k];
		}
		
		R[k*N + k] = sqrt(nrm);
		for (i = 0; i < M; i++)
		{
			Q[i*N + k] = A[i*N + k] / R[k*N + k];
		}
		
		for (j = k + 1; j < N; j++)
		{
			R[k*N + j] = 0;
			for (i = 0; i < M; i++)
			{
				R[k*N + j] += Q[i*N + k] * A[i*N + j];
			}
			for (i = 0; i < M; i++)
			{
				A[i*N + j] = A[i*N + j] - Q[i*N + k] * R[k*N + j];
			}
		}
	}
}


void init_array(DATA_TYPE* A)
{
	int i, j;

	for (i = 0; i < M; i++)
	{
		for (j = 0; j < N; j++)
		{
			A[i*N + j] = ((DATA_TYPE) (i+1)*(j+1)) / (M+1);
		}
	}
}


void compareResults(DATA_TYPE* A, DATA_TYPE* A_outputFromGpu)
{
	int i, j, fail;
	fail = 0;

	for (i=0; i < M; i++) 
	{
		for (j=0; j < N; j++) 
		{
			if (percentDiff(A[i*N + j], A_outputFromGpu[i*N + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{				
				fail++;
				printf("i: %d j: %d \n1: %f\n 2: %f\n", i, j, A[i*N + j], A_outputFromGpu[i*N + j]);
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );	
	return;
}


__global__ void gramschmidt_kernel1(DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid==0)
	{
		DATA_TYPE nrm = 0.0;
		int i;
		for (i = 0; i < M; i++)
		{
			nrm += a[i * N + k] * a[i * N + k];
		}
      		r[k * N + k] = sqrt(nrm);
	}
}


__global__ void gramschmidt_kernel2(DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < M)
	{	
		q[i * N + k] = a[i * N + k] / r[k * N + k];
	}
}


__global__ void gramschmidt_kernel3(DATA_TYPE *a, DATA_TYPE *r, DATA_TYPE *q, int k)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((j > k) && (j < N))
	{
		r[k*N + j] = 0.0;

		int i;
		for (i = 0; i < M; i++)
		{
			r[k*N + j] += q[i*N + k] * a[i*N + j];
		}
		
		for (i = 0; i < M; i++)
		{
			a[i*N + j] -= q[i*N + k] * r[k*N + j];
		}
	}
}


void gramschmidtCuda(DATA_TYPE* A, DATA_TYPE* R, DATA_TYPE* Q, DATA_TYPE* A_outputFromGpu)
{
	double t_start, t_end;

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 gridKernel1(1, 1);
	dim3 gridKernel2((size_t)ceil(((float)N) / ((float)DIM_THREAD_BLOCK_X)), 1);
	dim3 gridKernel3((size_t)ceil(((float)N) / ((float)DIM_THREAD_BLOCK_X)), 1);
	
	DATA_TYPE *A_gpu;
	DATA_TYPE *R_gpu;
	DATA_TYPE *Q_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * M * N);
	hipMalloc((void **)&R_gpu, sizeof(DATA_TYPE) * M * N);
	hipMalloc((void **)&Q_gpu, sizeof(DATA_TYPE) * M * N);
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * M * N, hipMemcpyHostToDevice);
	
	t_start = rtclock();
	int k;
	for (k = 0; k < N; k++)
	{
		gramschmidt_kernel1<<<gridKernel1,block>>>(A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
		gramschmidt_kernel2<<<gridKernel2,block>>>(A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
		gramschmidt_kernel3<<<gridKernel3,block>>>(A_gpu, R_gpu, Q_gpu, k);
		hipDeviceSynchronize();
	}
	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	hipMemcpy(A_outputFromGpu, A_gpu, sizeof(DATA_TYPE) * M * N, hipMemcpyDeviceToHost);    

	hipFree(A_gpu);
	hipFree(R_gpu);
	hipFree(Q_gpu);
}


int main(int argc, char *argv[])
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* A_outputFromGpu;
	DATA_TYPE* R;
	DATA_TYPE* Q;
	
	A = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));
	A_outputFromGpu = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));
	R = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));  
	Q = (DATA_TYPE*)malloc(M*N*sizeof(DATA_TYPE));  
	
	init_array(A);
	
	GPU_argv_init();
	gramschmidtCuda(A, R, Q, A_outputFromGpu);
	
	t_start = rtclock();
	gramschmidt(A, R, Q);
	t_end = rtclock();

	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(A, A_outputFromGpu);
	
	free(A);
	free(A_outputFromGpu);
	free(R);
	free(Q);  

    	return 0;
}

