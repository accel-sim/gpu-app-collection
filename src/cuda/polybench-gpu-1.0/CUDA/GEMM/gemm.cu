#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size */
#define NI 512
#define NJ 512
#define NK 512

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void gemm(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i,j,k;
	
	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
    	{
			C[i*NJ + j] *= BETA;
	
			for (k = 0; k < NK; ++k)
			{
	  			C[i*NJ + j] += ALPHA * A[i*NK + k] * B[k*NJ + j];
			}
      	}
	}
}


void init(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C)
{
	int i, j;

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NK; j++)
		{
      		A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < NK; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		B[i*NJ + j] = ((DATA_TYPE) i*j + 1) / NJ;
		}
	}

  	for (i = 0; i < NI; i++)
	{
    	for (j = 0; j < NJ; j++)
		{
      		C[i*NJ + j] = ((DATA_TYPE) i*j + 2) / NJ;
		}
	}
}


void compareResults(DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	int i, j, fail;
	fail = 0;
	
	// Compare C1 and C2
	for (i=0; i < NI; i++) 
	{
		for (j=0; j < NJ; j++) 
		{
			if (percentDiff(C[i*NJ + j], C_outputFromGpu[i*NJ + j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemm_kernel(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{	
		c[i * NJ + j] *= BETA;
		int k;
		for(k=0; k < NK; k++)
		{
			c[i * NJ + j] += ALPHA * a[i * NK + k] * b[k * NJ +j];
		}
	}
}


void gemmCuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* C_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil( ((float)NI)/ ((float)block.x) )),(size_t)(ceil( ((float)NJ)/ ((float)block.y) )));

	t_start = rtclock();

	gemm_kernel<<< grid, block >>>(A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();

	t_end = rtclock();
	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);

	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);    
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}
	

int main(int argc, char *argv[])
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;  
	DATA_TYPE* C;  
	DATA_TYPE* C_outputFromGpu; 

	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE)); 
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));   
	C = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 
	C_outputFromGpu = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE)); 

	init(A, B, C);
	
	GPU_argv_init();
	
	gemmCuda(A, B, C, C_outputFromGpu);

	t_start = rtclock();	
	gemm(A, B, C);
	t_end = rtclock();
	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);
	
	compareResults(C, C_outputFromGpu);

	free(A);
	free(B);  
	free(C);  
	free(C_outputFromGpu); 

    	return 0;
}

