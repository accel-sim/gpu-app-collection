#include "hip/hip_runtime.h"
/**
 * 3mm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Problem size. */
# define NI 512
# define NJ 512
# define NK 512
# define NL 512
# define NM 512

/* Thread block dimensions */
#define DIM_THREAD_BLOCK_X 32
#define DIM_THREAD_BLOCK_Y 8

/* Can switch DATA_TYPE between float and double */
typedef float DATA_TYPE;



void init_array(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D)
{
	int i, j;

	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NK; j++)
		{
			A[i*NK + j] = ((DATA_TYPE) i*j) / NI;
		}
	}
  
	for (i = 0; i < NK; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			B[i*NJ + j] = ((DATA_TYPE) i*(j+1)) / NJ;
		}
	}
  
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NM; j++)
		{
			C[i*NM + j] = ((DATA_TYPE) i*(j+3)) / NL;
		}
	}
  
	for (i = 0; i < NM; i++)
	{
		for (j = 0; j < NL; j++)
		{
			D[i*NL + j] = ((DATA_TYPE) i*(j+2)) / NK;
		}
	}
}


void compareResults(DATA_TYPE *G, DATA_TYPE *G_outputFromGpu)
{
	int i,j,fail;
	fail = 0;

	for (i=0; i < NI; i++)
	{
		for (j=0; j < NL; j++)
		{
			if (percentDiff(G[i*NL + j], G_outputFromGpu[i*NL + j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;				
			}
		}
	}
	
	// print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}

	
__global__ void mm3_kernel1(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *E)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NJ))
	{
		int k;
		for(k=0; k < NK; k++)
		{
			E[i * NJ + j] += A[i * NK + k] * B[k * NJ + j];
		}
	}
}

	
__global__ void mm3_kernel2(DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *F)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NJ) && (j < NL))
	{
		int k;
		for(k=0; k < NM; k++)
		{
			F[i * NL + j] += C[i * NM + k] * D[k * NL +j];
		}
	}
}

	
__global__ void mm3_kernel3(DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < NI) && (j < NL))
	{
		int k;
		for(k=0; k < NJ; k++)
		{
			G[i * NL + j] += E[i * NJ + k] * F[k * NL + j];
		}
	}
}


void mm3_cpu(DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *C, DATA_TYPE *D, DATA_TYPE *E, DATA_TYPE *F, DATA_TYPE *G)
{
	int i,j,k;
	
	/* E := A*B */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NJ; j++)
		{
			E[i*NJ + j] = 0;
			for (k = 0; k < NK; ++k)
			{
				E[i*NJ + j] += A[i*NK + k] * B[k*NJ + j];
			}
		}
	}
		
	/* F := C*D */
	for (i = 0; i < NJ; i++)
	{
		for (j = 0; j < NL; j++)
		{
			F[i*NL + j] = 0;
			for (k = 0; k < NM; ++k)
			{
				F[i*NL + j] += C[i*NM + k] * D[k*NL + j];
			}
		}
	}

  	/* G := E*F */
	for (i = 0; i < NI; i++)
	{
		for (j = 0; j < NL; j++)
		{
			G[i*NL + j] = 0;
			for (k = 0; k < NJ; ++k)
			{
				G[i*NL + j] += E[i*NJ + k] * F[k*NL + j];
			}
		}
	}
}


void mm3Cuda(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE* D, DATA_TYPE* E, DATA_TYPE* F, 
		DATA_TYPE* G, DATA_TYPE* G_outputFromGpu)
{
	double t_start, t_end;

	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;
	DATA_TYPE *D_gpu;
	DATA_TYPE *E_gpu;
	DATA_TYPE *F_gpu;
	DATA_TYPE *G_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NJ * NM);
	hipMalloc((void **)&D_gpu, sizeof(DATA_TYPE) * NM * NL);
	hipMalloc((void **)&E_gpu, sizeof(DATA_TYPE) * NI * NJ);
	hipMalloc((void **)&F_gpu, sizeof(DATA_TYPE) * NJ * NL);
	hipMalloc((void **)&G_gpu, sizeof(DATA_TYPE) * NI * NL);

	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NJ * NM, hipMemcpyHostToDevice);
	hipMemcpy(D_gpu, D, sizeof(DATA_TYPE) * NM * NL, hipMemcpyHostToDevice);
	hipMemcpy(E_gpu, E, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);
	hipMemcpy(F_gpu, F, sizeof(DATA_TYPE) * NJ * NL, hipMemcpyHostToDevice);
	hipMemcpy(G_gpu, G, sizeof(DATA_TYPE) * NI * NL, hipMemcpyHostToDevice);	
	
	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid1((size_t)(ceil( ((float)NJ) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid2((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NJ/ ((float)DIM_THREAD_BLOCK_Y) )));
	dim3 grid3((size_t)(ceil( ((float)NL) / ((float)DIM_THREAD_BLOCK_X) )),(size_t)(ceil((float)NI/ ((float)DIM_THREAD_BLOCK_Y) )));

	t_start = rtclock();
	mm3_kernel1<<<grid1,block>>>(A_gpu, B_gpu, E_gpu);
	hipDeviceSynchronize();
	mm3_kernel2<<<grid2,block>>>(C_gpu, D_gpu, F_gpu);
	hipDeviceSynchronize();
	mm3_kernel3<<<grid3,block>>>(E_gpu, F_gpu, G_gpu);
	hipDeviceSynchronize();
	t_end = rtclock();
	hipMemcpy(G_outputFromGpu, G_gpu, sizeof(DATA_TYPE) * NI * NL, hipMemcpyDeviceToHost);

	fprintf(stdout, "GPU Runtime: %0.6lfs\n", t_end - t_start);
	
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
	hipFree(D_gpu);
	hipFree(E_gpu);
	hipFree(F_gpu);
	hipFree(G_gpu);
}


int main(int argc, char** argv)
{
	double t_start, t_end;

	DATA_TYPE* A;
	DATA_TYPE* B;
	DATA_TYPE* C;
	DATA_TYPE* D;
	DATA_TYPE* E;
	DATA_TYPE* F;
	DATA_TYPE* G;
	DATA_TYPE* G_outputFromGpu;

	A = (DATA_TYPE*)malloc(NI*NK*sizeof(DATA_TYPE));
	B = (DATA_TYPE*)malloc(NK*NJ*sizeof(DATA_TYPE));
	C = (DATA_TYPE*)malloc(NJ*NM*sizeof(DATA_TYPE));
	D = (DATA_TYPE*)malloc(NM*NL*sizeof(DATA_TYPE));
	E = (DATA_TYPE*)malloc(NI*NJ*sizeof(DATA_TYPE));
	F = (DATA_TYPE*)malloc(NJ*NL*sizeof(DATA_TYPE));
	G = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));
	G_outputFromGpu = (DATA_TYPE*)malloc(NI*NL*sizeof(DATA_TYPE));

	init_array(A, B, C, D);

	GPU_argv_init();

	mm3Cuda(A, B, C, D, E, F, G, G_outputFromGpu);

	t_start = rtclock();

	mm3_cpu(A, B, C, D, E, F, G);
	
	t_end = rtclock();

	fprintf(stdout, "CPU Runtime: %0.6lfs\n", t_end - t_start);

	compareResults(G, G_outputFromGpu);

	free(A);
	free(B);
	free(C);
	free(D);
	free(E);
	free(F);
	free(G);
	free(G_outputFromGpu);

	return 0;
}

